#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "gpu_nonbonded.h"

#include "gpu_helpers.h"
#include "gpu_list.h"
#if !defined(GPU_ATOMIC_EV)
  #include "gpu_reduction.h"
#endif
#include "gpu_utils.h"

#include "../index_utils.h"
#include "../vector.h"

#include <cub/warp/warp_reduce.cuh>


GPU_GLOBAL void k_compute_polarization_energy( reax_atom const * const my_atoms, 
        single_body_parameters const * const sbp, int n, real * const e_pol_g )
{
    int i, type_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n ) {
        return;
    }

    type_i = my_atoms[i].type;

#if defined(GPU_ATOMIC_EV)
    atomicAdd( (double *) e_pol_g, (double) (KCALpMOL_to_EV * (sbp[type_i].chi
                    * my_atoms[i].q + (sbp[type_i].eta / 2.0) * SQR(my_atoms[i].q))) );
#else
    e_pol_g[i] = KCALpMOL_to_EV * (sbp[type_i].chi * my_atoms[i].q
            + (sbp[type_i].eta / 2.0) * SQR(my_atoms[i].q));
#endif
}


/* Compute energies and forces due to van der Waals and Coulomb interactions
 * where the far neighbors list is in full format
 *
 * This implementation assigns one thread per atom */
GPU_GLOBAL void k_vdW_coulomb_energy_full( reax_atom const * const my_atoms,
        two_body_parameters const * const tbp, real const * const gp_l, int vdw_type,
        real cutoff, double * const tap_coef, double const * const dtap_coef, rvec * const f,
        reax_list far_nbr_list, int n, int num_atom_types, 
        real * const e_vdW_g, real * const e_ele_g )
{
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j, tbp_ij;
    real self_coef;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real tap, dtap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_ele_, e_vdW_, e_core, de_core, e_clb, de_clb;
    rvec temp, f_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n ) {
        return;
    }

    const real p_vdW1 = gp_l[28];
    const real p_vdW1i = 1.0 / p_vdW1;
    e_vdW_ = 0.0;
    e_ele_ = 0.0;
    rvec_MakeZero( f_i );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = start_i; pj < end_i; ++pj ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j ) {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types);

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            tap = tap_coef[7] * r_ij
                + tap_coef[6];
            tap = tap * r_ij + tap_coef[5];
            tap = tap * r_ij + tap_coef[4];
            tap = tap * r_ij + tap_coef[3];
            tap = tap * r_ij + tap_coef[2];
            tap = tap * r_ij + tap_coef[1];
            tap = tap * r_ij + tap_coef[0];

            dtap = dtap_coef[6] * r_ij
                + dtap_coef[5];
            dtap = dtap * r_ij + dtap_coef[4];
            dtap = dtap * r_ij + dtap_coef[3];
            dtap = dtap * r_ij + dtap_coef[2];
            dtap = dtap * r_ij + dtap_coef[1];
            dtap = dtap * r_ij + dtap_coef[0];

            /* vdWaals Calculations */
            if ( vdw_type == 1 || vdw_type == 3 ) {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / tbp[tbp_ij].gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
                exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
                e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

                e_vdW_ += self_coef * (e_base * tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else {
                exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
                exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
                e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

                e_vdW_ += self_coef * (e_base * tap);

                de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( vdw_type == 2 || vdw_type == 3 ) {
                e_core = tbp[tbp_ij].ecore * EXP( tbp[tbp_ij].acore * (1.0 - (r_ij / tbp[tbp_ij].rcore)) );
                e_vdW_ += self_coef * (e_core * tap);

                de_core = -(tbp[tbp_ij].acore / tbp[tbp_ij].rcore) * e_core;
            } else {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * tap
                    + (e_base + e_core) * dtap );

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij + tbp[tbp_ij].gamma;
            dr3gamij_3 = RCBRT( dr3gamij_1 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) * dr3gamij_3;
            e_ele_ += self_coef * (e_clb * tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * tap + e_clb * dtap);

            rvec_Scale( temp, -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_Add( f_i, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( f[j], temp );
        }
    }

    atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
    atomicAdd( (double *) e_vdW_g, (double) e_vdW_ );
    atomicAdd( (double *) e_ele_g, (double) e_ele_ );
#else
    e_vdW_g[i] = e_vdW_;
    e_ele_g[i] = e_ele_;
#endif
}


/* Compute virial terms, energies, and forces due to van der Waals and Coulomb interactions
 * where the far neighbors list is in full format
 *
 * This implementation assigns one thread per atom */
GPU_GLOBAL void k_vdW_coulomb_energy_virial_full( reax_atom const * const my_atoms,
        two_body_parameters const * const tbp, real const * const gp_l, int vdw_type,
        real cutoff, double * const tap_coef, double const * const dtap_coef, rvec * const f,
        reax_list far_nbr_list, int n, int num_atom_types, 
        real * const e_vdW_g, real * const e_ele_g, rvec * const ext_press_g )
{
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j, tbp_ij;
    real self_coef;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real tap, dtap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_ele_, e_vdW_, e_core, de_core, e_clb, de_clb;
    rvec temp, f_i, ext_press_;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n ) {
        return;
    }

    const real p_vdW1 = gp_l[28];
    const real p_vdW1i = 1.0 / p_vdW1;
    e_vdW_ = 0.0;
    e_ele_ = 0.0;
    rvec_MakeZero( f_i );
    rvec_MakeZero( ext_press_ );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = start_i; pj < end_i; ++pj ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j ) {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types);

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            tap = tap_coef[7] * r_ij
                + tap_coef[6];
            tap = tap * r_ij + tap_coef[5];
            tap = tap * r_ij + tap_coef[4];
            tap = tap * r_ij + tap_coef[3];
            tap = tap * r_ij + tap_coef[2];
            tap = tap * r_ij + tap_coef[1];
            tap = tap * r_ij + tap_coef[0];

            dtap = dtap_coef[6] * r_ij
                + dtap_coef[5];
            dtap = dtap * r_ij + dtap_coef[4];
            dtap = dtap * r_ij + dtap_coef[3];
            dtap = dtap * r_ij + dtap_coef[2];
            dtap = dtap * r_ij + dtap_coef[1];
            dtap = dtap * r_ij + dtap_coef[0];

            /* vdWaals Calculations */
            if ( vdw_type == 1 || vdw_type == 3 ) {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / tbp[tbp_ij].gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
                exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
                e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

                e_vdW_ += self_coef * (e_base * tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else {
                exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
                exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
                e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

                e_vdW_ += self_coef * (e_base * tap);

                de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( vdw_type == 2 || vdw_type == 3 ) {
                e_core = tbp[tbp_ij].ecore * EXP( tbp[tbp_ij].acore * (1.0 - (r_ij / tbp[tbp_ij].rcore)) );
                e_vdW_ += self_coef * (e_core * tap);

                de_core = -(tbp[tbp_ij].acore / tbp[tbp_ij].rcore) * e_core;
            } else {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * tap
                    + (e_base + e_core) * dtap );

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij + tbp[tbp_ij].gamma;
            dr3gamij_3 = RCBRT( dr3gamij_1 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) * dr3gamij_3;
            e_ele_ += self_coef * (e_clb * tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * tap + e_clb * dtap);

            /* for pressure coupling, terms not related to bond order 
               derivatives are added directly into pressure vector/tensor */
            rvec_Scale( temp,
                    -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_Add( f_i, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( f[j], temp );

            rvec_iMultiply( temp,
                    far_nbr_list.far_nbr_list.rel_box[pj], temp );
            rvec_Add( ext_press_, temp );
        }
    }

    atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
    atomicAdd( (double *) e_vdW_g, (double) e_vdW_ );
    atomicAdd( (double *) e_ele_g, (double) e_ele_ );
    atomic_rvecAdd( *ext_press_g, ext_press_ );
#else
    e_vdW_g[i] = e_vdW_;
    e_ele_g[i] = e_ele_;
    rvec_Copy( ext_press_g[j], ext_press_ );
#endif
}


/* Compute energies and forces due to van der Waals and Coulomb interactions
 * where the far neighbors list is in full format
 *
 * This implementation assigns one warp of threads per atom */
GPU_GLOBAL void k_vdW_coulomb_energy_full_opt( reax_atom const * const my_atoms, 
        two_body_parameters const * const tbp, real const * const gp_l, int vdw_type,
        real cutoff, double * const tap_coef, double const * const dtap_coef, rvec * const f,
        reax_list far_nbr_list, int n, int num_atom_types, 
        real * const e_vdW_g, real * const e_ele_g )
{
    extern __shared__ hipcub::WarpReduce<double>::TempStorage temp_storage[];
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j, tbp_ij;
    real self_coef;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real tap, dtap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_vdW_, e_ele_, e_core, de_core, e_clb, de_clb;
    rvec temp, f_i;
    int thread_id, warp_id, lane_id;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id / warpSize;

    if ( warp_id >= n ) {
        return;
    }

    lane_id = thread_id % warpSize; 
    i = warp_id;
    const real p_vdW1 = gp_l[28];
    const real p_vdW1i = 1.0 / p_vdW1;
    e_vdW_ = 0.0;
    e_ele_ = 0.0;
    rvec_MakeZero( f_i );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    pj = start_i + lane_id;
    while ( pj < end_i ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j ) {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types);

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            tap = tap_coef[7] * r_ij
                + tap_coef[6];
            tap = tap * r_ij + tap_coef[5];
            tap = tap * r_ij + tap_coef[4];
            tap = tap * r_ij + tap_coef[3];
            tap = tap * r_ij + tap_coef[2];
            tap = tap * r_ij + tap_coef[1];
            tap = tap * r_ij + tap_coef[0];

            dtap = dtap_coef[6] * r_ij
                + dtap_coef[5];
            dtap = dtap * r_ij + dtap_coef[4];
            dtap = dtap * r_ij + dtap_coef[3];
            dtap = dtap * r_ij + dtap_coef[2];
            dtap = dtap * r_ij + dtap_coef[1];
            dtap = dtap * r_ij + dtap_coef[0];

            /* vdWaals Calculations */
            if ( vdw_type == 1 || vdw_type == 3 ) {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / tbp[tbp_ij].gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
                exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
                e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

                e_vdW_ += self_coef * (e_base * tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else {
                exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
                exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
                e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

                e_vdW_ += self_coef * (e_base * tap);

                de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( vdw_type == 2 || vdw_type == 3 ) {
                e_core = tbp[tbp_ij].ecore * EXP( tbp[tbp_ij].acore * (1.0 - (r_ij / tbp[tbp_ij].rcore)) );
                e_vdW_ += self_coef * (e_core * tap);

                de_core = -(tbp[tbp_ij].acore / tbp[tbp_ij].rcore) * e_core;
            } else {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * tap
                    + (e_base + e_core) * dtap );

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij + tbp[tbp_ij].gamma;
            dr3gamij_3 = RCBRT( dr3gamij_1 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) * dr3gamij_3;
            e_ele_ += self_coef * (e_clb * tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * tap + e_clb * dtap);

            rvec_Scale( temp, -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_Add( f_i, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( f[j], temp );
        }

        pj += warpSize;
    }

    e_vdW_ = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(e_vdW_);
    e_ele_ = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(e_ele_);
    f_i[0] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[0]);
    f_i[1] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[1]);
    f_i[2] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[2]);

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 ) {
        atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
        atomicAdd( (double *) e_vdW_g, (double) e_vdW_ );
        atomicAdd( (double *) e_ele_g, (double) e_ele_ );
#else
        e_vdW_g[i] = e_vdW_;
        e_ele_g[i] = e_ele_;
#endif
    }
}


/* Compute energies and forces due to type 1 van der Waals
 * interactions (shielding, no inner core repulsion) 
 * where the far neighbors list is in full format
 *
 * This implementation assigns one warp of threads per atom */
GPU_GLOBAL void k_vdW_energy_type1_full_opt( reax_atom const * const my_atoms, 
        two_body_parameters const * const tbp, real const * const gp_l,
        real cutoff, double * const tap_coef, double const * const dtap_coef, rvec * const f,
        reax_list far_nbr_list, int n, int num_atom_types, 
        real * const e_vdW_g )
{
    extern __shared__ hipcub::WarpReduce<double>::TempStorage temp_storage[];
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j, tbp_ij;
    real self_coef;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real tap, dtap, tap_coef_[TAPER_COEF_SIZE], dtap_coef_[DTAPER_COEF_SIZE], dfn13, CEvd;
    real e_vdW_;
    rvec temp, f_i;
    int warp_id, lane_id;

    warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( warp_id >= n ) {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize; 
    i = warp_id;
    const real p_vdW1 = gp_l[28];
    const real p_vdW1i = 1.0 / p_vdW1;
    e_vdW_ = 0.0;
    rvec_MakeZero( f_i );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = 0; pj < TAPER_COEF_SIZE; ++pj ) {
        tap_coef_[pj] = tap_coef[pj];
    }

    for ( pj = 0; pj < DTAPER_COEF_SIZE; ++pj ) {
        dtap_coef_[pj] = dtap_coef[pj];
    }

    pj = start_i + lane_id;
    while ( pj < end_i ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j ) {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types);

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            tap = tap_coef_[7] * r_ij + tap_coef_[6];
            tap = tap * r_ij + tap_coef_[5];
            tap = tap * r_ij + tap_coef_[4];
            tap = tap * r_ij + tap_coef_[3];
            tap = tap * r_ij + tap_coef_[2];
            tap = tap * r_ij + tap_coef_[1];
            tap = tap * r_ij + tap_coef_[0];

            dtap = dtap_coef_[6] * r_ij + dtap_coef_[5];
            dtap = dtap * r_ij + dtap_coef_[4];
            dtap = dtap * r_ij + dtap_coef_[3];
            dtap = dtap * r_ij + dtap_coef_[2];
            dtap = dtap * r_ij + dtap_coef_[1];
            dtap = dtap * r_ij + dtap_coef_[0];

            /* vdWaals Calculations */
            /* shielding */
            powr_vdW1 = POW( r_ij, p_vdW1 );
            powgi_vdW1 = POW( 1.0 / tbp[tbp_ij].gamma_w, p_vdW1 );

            fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
            exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
            exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
            e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

            e_vdW_ += self_coef * (e_base * tap);

            dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
            de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1) * dfn13;

            CEvd = self_coef * (de_base * tap + e_base * dtap);

            rvec_Scale( temp, -CEvd / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_Add( f_i, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( f[j], temp );
        }

        pj += warpSize;
    }

    e_vdW_ = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(e_vdW_);
    f_i[0] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[0]);
    f_i[1] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[1]);
    f_i[2] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[2]);

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 ) {
        atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
        atomicAdd( (double *) e_vdW_g, (double) e_vdW_ );
#else
        e_vdW_g[i] = e_vdW_;
#endif
    }
}


/* Compute energies and forces due to type 2 van der Waals
 * interactions (no shielding, inner core repulsion) 
 * where the far neighbors list is in full format
 *
 * This implementation assigns one warp of threads per atom */
GPU_GLOBAL void k_vdW_energy_type2_full_opt( reax_atom const * const my_atoms, 
        two_body_parameters const * const tbp, real cutoff,
        double * const tap_coef, double const * const dtap_coef, rvec * const f,
        reax_list far_nbr_list, int n, int num_atom_types, 
        real * const e_vdW_g )
{
    extern __shared__ hipcub::WarpReduce<double>::TempStorage temp_storage[];
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j, tbp_ij;
    real self_coef;
    real r_ij, exp1, exp2, e_base, de_base;
    real tap, dtap, tap_coef_[TAPER_COEF_SIZE], dtap_coef_[DTAPER_COEF_SIZE], CEvd;
    real e_vdW_, e_core, de_core;
    rvec temp, f_i;
    int warp_id, lane_id;

    warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( warp_id >= n ) {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize; 
    i = warp_id;
    e_vdW_ = 0.0;
    rvec_MakeZero( f_i );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = 0; pj < TAPER_COEF_SIZE; ++pj ) {
        tap_coef_[pj] = tap_coef[pj];
    }

    for ( pj = 0; pj < DTAPER_COEF_SIZE; ++pj ) {
        dtap_coef_[pj] = dtap_coef[pj];
    }

    pj = start_i + lane_id;
    while ( pj < end_i ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j )
        {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types);

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            tap = tap_coef_[7] * r_ij + tap_coef_[6];
            tap = tap * r_ij + tap_coef_[5];
            tap = tap * r_ij + tap_coef_[4];
            tap = tap * r_ij + tap_coef_[3];
            tap = tap * r_ij + tap_coef_[2];
            tap = tap * r_ij + tap_coef_[1];
            tap = tap * r_ij + tap_coef_[0];

            dtap = dtap_coef_[6] * r_ij + dtap_coef_[5];
            dtap = dtap * r_ij + dtap_coef_[4];
            dtap = dtap * r_ij + dtap_coef_[3];
            dtap = dtap * r_ij + dtap_coef_[2];
            dtap = dtap * r_ij + dtap_coef_[1];
            dtap = dtap * r_ij + dtap_coef_[0];

            /* vdWaals Calculations */
            /* no shielding */
            exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
            exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
            e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

            e_vdW_ += self_coef * (e_base * tap);

            de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1);

            /* calculate inner core repulsion */
            e_core = tbp[tbp_ij].ecore * EXP( tbp[tbp_ij].acore * (1.0 - (r_ij / tbp[tbp_ij].rcore)) );
            e_vdW_ += self_coef * (e_core * tap);

            de_core = -(tbp[tbp_ij].acore / tbp[tbp_ij].rcore) * e_core;

            CEvd = self_coef * ((de_base + de_core) * tap
                    + (e_base + e_core) * dtap);

            rvec_Scale( temp, -CEvd / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_Add( f_i, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( f[j], temp );
        }

        pj += warpSize;
    }

    e_vdW_ = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(e_vdW_);
    f_i[0] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[0]);
    f_i[1] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[1]);
    f_i[2] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[2]);

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 ) {
        atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
        atomicAdd( (double *) e_vdW_g, (double) e_vdW_ );
#else
        e_vdW_g[i] = e_vdW_;
#endif
    }
}


/* Compute energies and forces due to type 3 van der Waals
 * interactions (shielding, inner core repulsion) 
 * where the far neighbors list is in full format
 *
 * This implementation assigns one warp of threads per atom */
GPU_GLOBAL void k_vdW_energy_type3_full_opt( reax_atom const * const my_atoms, 
        two_body_parameters const * const tbp, real const * const gp_l, real cutoff,
        double * const tap_coef, double const * const dtap_coef, rvec * const f,
        reax_list far_nbr_list, int n, int num_atom_types, 
        real * const e_vdW_g )
{
    extern __shared__ hipcub::WarpReduce<double>::TempStorage temp_storage[];
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j, tbp_ij;
    real self_coef;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real tap, dtap, tap_coef_[TAPER_COEF_SIZE], dtap_coef_[DTAPER_COEF_SIZE], dfn13, CEvd;
    real e_vdW_, e_core, de_core;
    rvec temp, f_i;
    int warp_id, lane_id;

    warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( warp_id >= n ) {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize; 
    i = warp_id;
    const real p_vdW1 = gp_l[28];
    const real p_vdW1i = 1.0 / p_vdW1;
    e_vdW_ = 0.0;
    rvec_MakeZero( f_i );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = 0; pj < TAPER_COEF_SIZE; ++pj ) {
        tap_coef_[pj] = tap_coef[pj];
    }

    for ( pj = 0; pj < DTAPER_COEF_SIZE; ++pj ) {
        dtap_coef_[pj] = dtap_coef[pj];
    }

    pj = start_i + lane_id;
    while ( pj < end_i ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j ) {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types);

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            tap = tap_coef_[7] * r_ij + tap_coef_[6];
            tap = tap * r_ij + tap_coef_[5];
            tap = tap * r_ij + tap_coef_[4];
            tap = tap * r_ij + tap_coef_[3];
            tap = tap * r_ij + tap_coef_[2];
            tap = tap * r_ij + tap_coef_[1];
            tap = tap * r_ij + tap_coef_[0];

            dtap = dtap_coef_[6] * r_ij + dtap_coef_[5];
            dtap = dtap * r_ij + dtap_coef_[4];
            dtap = dtap * r_ij + dtap_coef_[3];
            dtap = dtap * r_ij + dtap_coef_[2];
            dtap = dtap * r_ij + dtap_coef_[1];
            dtap = dtap * r_ij + dtap_coef_[0];

            /* vdWaals Calculations */
            /* shielding */
            powr_vdW1 = POW( r_ij, p_vdW1 );
            powgi_vdW1 = POW( 1.0 / tbp[tbp_ij].gamma_w, p_vdW1 );

            fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
            exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
            exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
            e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

            e_vdW_ += self_coef * (e_base * tap);

            dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
            de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1) * dfn13;

            /* calculate inner core repulsion */
            e_core = tbp[tbp_ij].ecore * EXP( tbp[tbp_ij].acore * (1.0 - (r_ij / tbp[tbp_ij].rcore)) );
            e_vdW_ += self_coef * (e_core * tap);

            de_core = -(tbp[tbp_ij].acore / tbp[tbp_ij].rcore) * e_core;

            CEvd = self_coef * ((de_base + de_core) * tap
                    + (e_base + e_core) * dtap);

            rvec_Scale( temp, -CEvd / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_Add( f_i, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( f[j], temp );
        }

        pj += warpSize;
    }

    e_vdW_ = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(e_vdW_);
    f_i[0] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[0]);
    f_i[1] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[1]);
    f_i[2] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[2]);

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 ) {
        atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
        atomicAdd( (double *) e_vdW_g, (double) e_vdW_ );
#else
        e_vdW_g[i] = e_vdW_;
#endif
    }
}


/* Compute energies and forces due to Coulomb interactions
 * where the far neighbors list is in full format
 *
 * This implementation assigns one warp of threads per atom */
GPU_GLOBAL void k_coulomb_energy_full_opt( reax_atom const * const my_atoms, 
        two_body_parameters const * const tbp,
        real cutoff, double * const tap_coef, double const * const dtap_coef,
        rvec * const f, reax_list far_nbr_list, int n, int num_atom_types, 
        real * const e_ele_g )
{
    extern __shared__ hipcub::WarpReduce<double>::TempStorage temp_storage[];
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j, tbp_ij;
    real self_coef;
    real r_ij;
    real tap, dtap, tap_coef_[8], dtap_coef_[7], CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_ele_, e_clb, de_clb;
    rvec temp, f_i;
    int thread_id, warp_id, lane_id;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id / warpSize;

    if ( warp_id >= n ) {
        return;
    }

    lane_id = thread_id % warpSize; 
    i = warp_id;
    e_ele_ = 0.0;
    rvec_MakeZero( f_i );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    tap_coef_[0] = tap_coef[0];
    tap_coef_[1] = tap_coef[1];
    tap_coef_[2] = tap_coef[2];
    tap_coef_[3] = tap_coef[3];
    tap_coef_[4] = tap_coef[4];
    tap_coef_[5] = tap_coef[5];
    tap_coef_[6] = tap_coef[6];
    tap_coef_[7] = tap_coef[7];

    dtap_coef_[0] = dtap_coef[0];
    dtap_coef_[1] = dtap_coef[1];
    dtap_coef_[2] = dtap_coef[2];
    dtap_coef_[3] = dtap_coef[3];
    dtap_coef_[4] = dtap_coef[4];
    dtap_coef_[5] = dtap_coef[5];
    dtap_coef_[6] = dtap_coef[6];

    pj = start_i + lane_id;
    while ( pj < end_i ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j ) {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types);

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            tap = tap_coef_[7] * r_ij + tap_coef_[6];
            tap = tap * r_ij + tap_coef_[5];
            tap = tap * r_ij + tap_coef_[4];
            tap = tap * r_ij + tap_coef_[3];
            tap = tap * r_ij + tap_coef_[2];
            tap = tap * r_ij + tap_coef_[1];
            tap = tap * r_ij + tap_coef_[0];

            dtap = dtap_coef_[6] * r_ij + dtap_coef_[5];
            dtap = dtap * r_ij + dtap_coef_[4];
            dtap = dtap * r_ij + dtap_coef_[3];
            dtap = dtap * r_ij + dtap_coef_[2];
            dtap = dtap * r_ij + dtap_coef_[1];
            dtap = dtap * r_ij + dtap_coef_[0];

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij + tbp[tbp_ij].gamma;
            dr3gamij_3 = RCBRT( dr3gamij_1 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) * dr3gamij_3;
            e_ele_ += self_coef * (e_clb * tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * tap + e_clb * dtap);

            rvec_Scale( temp, -CEclmb / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_Add( f_i, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( f[j], temp );
        }

        pj += warpSize;
    }

    e_ele_ = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(e_ele_);
    f_i[0] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[0]);
    f_i[1] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[1]);
    f_i[2] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[2]);

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 ) {
        atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
        atomicAdd( (double *) e_ele_g, (double) e_ele_ );
#else
        e_ele_g[i] = e_ele_;
#endif
    }
}


/* Compute virial terms, energies, and forces due to van der Waals and Coulomb interactions
 * where the far neighbors list is in full format
 *
 * This implementation assigns one warp of threads per atom */
GPU_GLOBAL void k_vdW_coulomb_energy_virial_full_opt( reax_atom const * const my_atoms,
        two_body_parameters const * const tbp, real const * const gp_l, int vdw_type,
        real cutoff, double * const tap_coef, double const * const dtap_coef,
        rvec * const f, reax_list far_nbr_list, int n, int num_atom_types, 
        real * const e_vdW_g, real * const e_ele_g, rvec * const ext_press_g )
{
    extern __shared__ hipcub::WarpReduce<double>::TempStorage temp_storage[];
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j, tbp_ij;
    real self_coef;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real tap, dtap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_vdW_, e_ele_, e_core, de_core, e_clb, de_clb;
    rvec temp, f_i, ext_press_;
    int thread_id, warp_id, lane_id;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id / warpSize;

    if ( warp_id >= n ) {
        return;
    }

    lane_id = thread_id % warpSize; 
    i = warp_id;
    const real p_vdW1 = gp_l[28];
    const real p_vdW1i = 1.0 / p_vdW1;
    e_vdW_ = 0.0;
    e_ele_ = 0.0;
    rvec_MakeZero( f_i );
    rvec_MakeZero( ext_press_ );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    pj = start_i + lane_id;
    while ( pj < end_i ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j ) {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types);

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            tap = tap_coef[7] * r_ij
                + tap_coef[6];
            tap = tap * r_ij + tap_coef[5];
            tap = tap * r_ij + tap_coef[4];
            tap = tap * r_ij + tap_coef[3];
            tap = tap * r_ij + tap_coef[2];
            tap = tap * r_ij + tap_coef[1];
            tap = tap * r_ij + tap_coef[0];

            dtap = dtap_coef[6] * r_ij
                + dtap_coef[5];
            dtap = dtap * r_ij + dtap_coef[4];
            dtap = dtap * r_ij + dtap_coef[3];
            dtap = dtap * r_ij + dtap_coef[2];
            dtap = dtap * r_ij + dtap_coef[1];
            dtap = dtap * r_ij + dtap_coef[0];

            /* vdWaals Calculations */
            if ( vdw_type == 1 || vdw_type == 3 ) {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / tbp[tbp_ij].gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
                exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - fn13 / tbp[tbp_ij].r_vdW) );
                e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

                e_vdW_ += self_coef * (e_base * tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else {
                exp1 = EXP( tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
                exp2 = EXP( 0.5 * tbp[tbp_ij].alpha * (1.0 - r_ij / tbp[tbp_ij].r_vdW) );
                e_base = tbp[tbp_ij].D * (exp1 - 2.0 * exp2);

                e_vdW_ += self_coef * (e_base * tap);

                de_base = (tbp[tbp_ij].D * tbp[tbp_ij].alpha / tbp[tbp_ij].r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( vdw_type == 2 || vdw_type == 3 ) {
                e_core = tbp[tbp_ij].ecore * EXP( tbp[tbp_ij].acore * (1.0 - (r_ij / tbp[tbp_ij].rcore)) );
                e_vdW_ += self_coef * (e_core * tap);

                de_core = -(tbp[tbp_ij].acore / tbp[tbp_ij].rcore) * e_core;
            } else {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * tap
                    + (e_base + e_core) * dtap );

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij + tbp[tbp_ij].gamma;
            dr3gamij_3 = RCBRT( dr3gamij_1 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) * dr3gamij_3;
            e_ele_ += self_coef * (e_clb * tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * tap + e_clb * dtap);

            /* for pressure coupling, terms not related to bond order 
               derivatives are added directly into pressure vector/tensor */
            rvec_Scale( temp,
                    -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_Add( f_i, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( f[j], temp );

            rvec_iMultiply( temp,
                    far_nbr_list.far_nbr_list.rel_box[pj], temp );
            rvec_Add( ext_press_, temp );
        }

        pj += warpSize;
    }

    e_vdW_ = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(e_vdW_);
    e_ele_ = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(e_ele_);
    f_i[0] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[0]);
    f_i[1] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[1]);
    f_i[2] = hipcub::WarpReduce<double>(temp_storage[warp_id]).Sum(f_i[2]);

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 ) {
        atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
        atomicAdd( (double *) e_vdW_g, (double) e_vdW_ );
        atomicAdd( (double *) e_ele_g, (double) e_ele_ );
        atomic_rvecAdd( *ext_press_g, ext_press_ );
#else
        e_vdW_g[i] = e_vdW_;
        e_ele_g[i] = e_ele_;
        rvec_Copy( ext_press_g[j], ext_press_ );
#endif
    }
}


/* one thread per atom implementation */
GPU_GLOBAL void k_vdW_coulomb_energy_tab_full( reax_atom const * const my_atoms, 
        real cutoff, int virial, rvec * const f, reax_list far_nbr_list, 
        LR_lookup_table * const t_LR, int n, int num_atom_types, 
        real * const e_vdW_g, real * const e_ele_g, rvec * const ext_press_g )
{
    int i, j, pj, r;
    int type_i, type_j;
    int start_i, end_i, orig_i, orig_j;
    real r_ij, self_coef, base, dif;
    real e_vdW_, e_ele_;
    real CEvd, CEclmb;
    rvec temp, f_i, ext_press_;
    LR_lookup_table *t;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n ) {
        return;
    }

    e_ele_ = 0.0;
    e_vdW_ = 0.0;
    rvec_MakeZero( f_i );
    rvec_MakeZero( ext_press_ );

    type_i = my_atoms[i].type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = start_i; pj < end_i; ++pj ) {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff && orig_i < orig_j ) {
            type_j = my_atoms[j].type;
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            self_coef = (i == j) ? 0.5 : 1.0;
            t = &t_LR[ index_lr(min(type_i, type_j), max(type_i, type_j), num_atom_types) ];

            /* Cubic Spline Interpolation */
            r = (int) (r_ij * t->inv_dx);
            if ( r == 0 ) {
                ++r;
            }
            base = (real) (r + 1) * t->dx;
            dif = r_ij - base;

            e_vdW_ += self_coef * (((t->vdW[r].d * dif + t->vdW[r].c) * dif + t->vdW[r].b)
                * dif + t->vdW[r].a);

            e_ele_ += (((t->ele[r].d * dif + t->ele[r].c) * dif + t->ele[r].b)
                * dif + t->ele[r].a) * self_coef * my_atoms[i].q * my_atoms[j].q;

            CEvd = (((t->CEvd[r].d * dif + t->CEvd[r].c) * dif + t->CEvd[r].b)
                * dif + t->CEvd[r].a) * self_coef;

            CEclmb = (((t->CEclmb[r].d * dif + t->CEclmb[r].c) * dif + t->CEclmb[r].b)
                * dif + t->CEclmb[r].a) * self_coef * my_atoms[i].q * my_atoms[j].q;

            if ( virial == 0 ) {
                rvec_Scale( temp,
                        -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                rvec_Add( f_i, temp );
                rvec_Scale( temp, -1.0, temp );
                atomic_rvecAdd( f[j], temp );
            }
            /* NPT, iNPT or sNPT */
            else {
                /* for pressure coupling, terms not related to bond order derivatives
                   are added directly into pressure vector/tensor */
                rvec_Scale( temp,
                        -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                rvec_Add( f_i, temp );
                rvec_Scale( temp, -1.0, temp );
                atomic_rvecAdd( f[j], temp );

                rvec_iMultiply( temp, far_nbr_list.far_nbr_list.rel_box[pj], temp );
                rvec_Add( ext_press_, temp );
            }
        }
    }

    atomic_rvecAdd( f[i], f_i );
#if defined(GPU_ATOMIC_EV)
    atomicAdd( (double *) e_vdW_g, (double) e_vdW_ );
    atomicAdd( (double *) e_ele_g, (double) e_ele_ );
    if ( virial == 1 ) {
        atomic_rvecAdd( *ext_press_g, ext_press_ );
    }
#else
    e_vdW_g[i] = e_vdW_;
    e_ele_g[i] = e_ele_;
    if ( virial == 1 ) {
        rvec_Copy( ext_press_g[j], ext_press_ );
    }
#endif
}


static void GPU_Compute_Polarization_Energy( reax_system const * const system,
        control_params const * const control, storage * const workspace,
        simulation_data * const data )
{
#if defined(GPU_ATOMIC_EV)
    sCudaMemsetAsync( &data->d_my_en[E_POL], 0, sizeof(real),
            control->gpu_streams[5], __FILE__, __LINE__ );
#else
    real *spad;

    sCudaCheckMalloc( &workspace->d_workspace->scratch[5],
            &workspace->d_workspace->scratch_size[5],
            sizeof(real) * system->n, __FILE__, __LINE__ );
    spad = (real *) workspace->d_workspace->scratch[5];
#endif

    k_compute_polarization_energy <<< control->blocks_n, control->gpu_block_size,
                                  0, control->gpu_streams[5] >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, system->n,
#if defined(GPU_ATOMIC_EV)
          &data->d_my_en[E_POL]
#else
          spad
#endif
        );
    cudaCheckError( );

#if !defined(GPU_ATOMIC_EV)
    GPU_Reduction_Sum( spad, &data->d_my_en[E_POL], system->n, 5,
            control->gpu_streams[5] );
#endif
}


void GPU_Compute_NonBonded_Forces_Part1( reax_system const * const system,
        control_params const * const control, simulation_data * const data,
        storage * const workspace, reax_list **lists,
        output_controls const * const out_control )
{
#if !defined(FUSED_VDW_COULOMB)
#if !defined(GPU_ATOMIC_EV)
    int update_energy;
    size_t s;
    real *spad;
    rvec *spad_rvec;

    if ( control->tabulate <= 0 ) {
        update_energy = (out_control->energy_update_freq > 0
                && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;
#endif

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->gpu_time_events[TE_VDW_START], control->gpu_streams[4] );
#endif

#if defined(GPU_ATOMIC_EV)
        sCudaMemsetAsync( &data->d_my_en[E_VDW], 0, sizeof(real),
                control->gpu_streams[4], __FILE__, __LINE__ );
        if ( control->virial == 1 ) {
            sCudaMemsetAsync( &data->d_my_ext_press, 0, sizeof(rvec),
                    control->gpu_streams[4], __FILE__, __LINE__ );
        }
#else
        if ( control->virial == 1 ) {
            s = (sizeof(real) + sizeof(rvec)) * system->n + sizeof(rvec) * control->blocks_n;
        } else {
            s = sizeof(real) * system->n;
        }
        sCudaCheckMalloc( &workspace->d_workspace->scratch[4],
                &workspace->d_workspace->scratch_size[4],
                s, __FILE__, __LINE__ );
        spad = (real *) workspace->d_workspace->scratch[4];
#endif

        hipStreamWaitEvent( control->gpu_streams[4],
                control->gpu_stream_events[SE_INIT_DIST_DONE], 0 );

        if ( control->virial == 0 ) {
            if ( system->reax_param.gp.vdw_type == 1 ) {
                k_vdW_energy_type1_full_opt <<< control->blocks_warp_n, control->gpu_block_size,
                                         sizeof(hipcub::WarpReduce<double>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                         control->gpu_streams[4] >>>
                    ( system->d_my_atoms, system->reax_param.d_tbp,
                      system->reax_param.gp.d_l, control->nonb_cut,
                      workspace->d_workspace->tap_coef, workspace->d_workspace->dtap_coef,
#if defined(GPU_STREAM_SINGLE_ACCUM)
                      workspace->d_workspace->f,
#else
                      workspace->d_workspace->f_vdw,
#endif
                      *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
#if defined(GPU_ATOMIC_EV)
                      &data->d_my_en[E_VDW]
#else
                      spad
#endif
                    );
            } else if ( system->reax_param.gp.vdw_type == 2 ) {
                k_vdW_energy_type2_full_opt <<< control->blocks_warp_n, control->gpu_block_size,
                                         sizeof(hipcub::WarpReduce<double>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                         control->gpu_streams[4] >>>
                    ( system->d_my_atoms, system->reax_param.d_tbp, control->nonb_cut,
                      workspace->d_workspace->tap_coef, workspace->d_workspace->dtap_coef,
#if defined(GPU_STREAM_SINGLE_ACCUM)
                      workspace->d_workspace->f,
#else
                      workspace->d_workspace->f_vdw,
#endif
                      *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
#if defined(GPU_ATOMIC_EV)
                      &data->d_my_en[E_VDW]
#else
                      spad
#endif
                    );
            } else if ( system->reax_param.gp.vdw_type == 3 ) {
                k_vdW_energy_type3_full_opt <<< control->blocks_warp_n, control->gpu_block_size,
                                         sizeof(hipcub::WarpReduce<double>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                         control->gpu_streams[4] >>>
                    ( system->d_my_atoms, system->reax_param.d_tbp, 
                      system->reax_param.gp.d_l, control->nonb_cut,
                      workspace->d_workspace->tap_coef, workspace->d_workspace->dtap_coef,
#if defined(GPU_STREAM_SINGLE_ACCUM)
                      workspace->d_workspace->f,
#else
                      workspace->d_workspace->f_vdw,
#endif
                      *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
#if defined(GPU_ATOMIC_EV)
                      &data->d_my_en[E_VDW]
#else
                      spad
#endif
                    );
            }
        } else if ( control->virial == 1 ) {
            k_vdW_coulomb_energy_virial_full_opt <<< control->blocks_warp_n, control->gpu_block_size,
                                                 sizeof(hipcub::WarpReduce<double>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                                 control->gpu_streams[4] >>>
                ( system->d_my_atoms, system->reax_param.d_tbp, system->reax_param.gp.d_l,
                  system->reax_param.gp.vdw_type, control->nonb_cut,
                  workspace->d_workspace->tap_coef, workspace->d_workspace->dtap_coef,
#if defined(GPU_STREAM_SINGLE_ACCUM)
                  workspace->d_workspace->f,
#else
                  workspace->d_workspace->f_vdw_clmb,
#endif
                  *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
#if defined(GPU_ATOMIC_EV)
                  &data->d_my_en[E_VDW], &data->d_my_en[E_ELE],
                  &data->d_my_ext_press
#else
                  spad, &spad[system->n], (rvec *) (&spad[2 * system->n])
#endif
                );
        }
        cudaCheckError( );

#if !defined(GPU_ATOMIC_EV)
        if ( update_energy == TRUE ) {
            /* reduction for vdw */
            GPU_Reduction_Sum( spad, &data->d_my_en[E_VDW], system->n,
                    4, control->gpu_streams[4] );
        }

        if ( control->virial == 1 ) {
            spad_rvec = (rvec *) (&spad[system->n]);

            GPU_Reduction_Sum( spad_rvec, &data->d_my_ext_press,
                    system->n, 4, control->gpu_streams[4] );
        }
#endif

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->gpu_time_events[TE_VDW_STOP], control->gpu_streams[4] );
#endif
    }
#endif
}


void GPU_Compute_NonBonded_Forces_Part2( reax_system const * const system,
        control_params const * const control, simulation_data * const data,
        storage * const workspace, reax_list **lists,
        output_controls const * const out_control )
{
    int update_energy;
#if !defined(GPU_ATOMIC_EV)
    size_t s;
    real *spad;
    rvec *spad_rvec;
#endif

#if defined(LOG_PERFORMANCE)
    hipEventRecord( control->gpu_time_events[TE_COULOMB_START], control->gpu_streams[5] );
#endif

    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;

#if defined(GPU_ATOMIC_EV)
#if defined(FUSED_VDW_COULOMB)
    sCudaMemsetAsync( &data->d_my_en[E_VDW], 0, sizeof(real) * 2,
            control->gpu_streams[5], __FILE__, __LINE__ );
#else
    sCudaMemsetAsync( &data->d_my_en[E_VDW], 0, sizeof(real),
            control->gpu_streams[5], __FILE__, __LINE__ );
#endif
    if ( control->virial == 1 ) {
        sCudaMemsetAsync( &data->d_my_ext_press, 0, sizeof(rvec),
                control->gpu_streams[5], __FILE__, __LINE__ );
    }
#else
    if ( control->virial == 1 ) {
#if defined(FUSED_VDW_COULOMB)
        s = (sizeof(real) * 2 + sizeof(rvec)) * system->n + sizeof(rvec) * control->blocks_n;
#else
        s = (sizeof(real) + sizeof(rvec)) * system->n + sizeof(rvec) * control->blocks_n;
#endif
    } else {
#if defined(FUSED_VDW_COULOMB)
        s = sizeof(real) * 2 * system->n;
#else
        s = sizeof(real) * system->n;
#endif
    }
    sCudaCheckMalloc( &workspace->d_workspace->scratch[5],
            &workspace->d_workspace->scratch_size[5],
            s, __FILE__, __LINE__ );
    spad = (real *) workspace->d_workspace->scratch[5];
#endif

    if ( control->tabulate <= 0 && control->virial == 0 ) {
#if defined(FUSED_VDW_COULOMB)
//           k_vdW_coulomb_energy_full <<< control->blocks_n, control->gpu_block_size,
//                                     0, control->gpu_streams[5] >>>
//               ( system->d_my_atoms, system->reax_param.d_tbp, system->reax_param.gp.d_l,
//                 system->reax_param.gp.vdw_type, control->nonb_cut, workspace->d_workspace->tap_coef,
//                 workspace->d_workspace->dtap_coef,
//    #if defined(GPU_STREAM_SINGLE_ACCUM)
//                 workspace->d_workspace->f,
//    #else
//                 workspace->d_workspace->f_vdw_clmb,
//    #endif
//                 *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
//    #if defined(GPU_ATOMIC_EV)
//                  &data->d_my_en[E_VDW], &data->d_my_en[E_ELE]
//    #else
//                  spad, &spad[system->n]
//    #endif
//                    );

        k_vdW_coulomb_energy_full_opt <<< control->blocks_warp_n, control->gpu_block_size,
                                      sizeof(hipcub::WarpReduce<double>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                      control->gpu_streams[5] >>>
            ( system->d_my_atoms, system->reax_param.d_tbp, system->reax_param.gp.d_l,
              system->reax_param.gp.vdw_type, control->nonb_cut,
              workspace->d_workspace->tap_coef, workspace->d_workspace->dtap_coef,
#if defined(GPU_STREAM_SINGLE_ACCUM)
              workspace->d_workspace->f,
#else
              workspace->d_workspace->f_vdw_clmb,
#endif
              *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
#if defined(GPU_ATOMIC_EV)
              &data->d_my_en[E_VDW], &data->d_my_en[E_ELE]
#else
              spad, &spad[system->n]
#endif
            );

#else
        k_coulomb_energy_full_opt <<< control->blocks_warp_n, control->gpu_block_size,
                                  sizeof(hipcub::WarpReduce<double>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                  control->gpu_streams[5] >>>
            ( system->d_my_atoms, system->reax_param.d_tbp, control->nonb_cut,
              workspace->d_workspace->tap_coef, workspace->d_workspace->dtap_coef,
#if defined(GPU_STREAM_SINGLE_ACCUM)
              workspace->d_workspace->f,
#else
              workspace->d_workspace->f_clmb,
#endif
              *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
#if defined(GPU_ATOMIC_EV)
              &data->d_my_en[E_ELE]
#else
              spad
#endif
            );
#endif
    } else if ( control->tabulate <= 0 && control->virial == 1 ) {
//            k_vdW_coulomb_energy_virial_full <<< control->blocks_n, control->gpu_block_size,
//                                             0, control->gpu_streams[5] >>>
//                ( system->d_my_atoms, system->reax_param.d_tbp, system->reax_param.gp.d_l,
//                  system->reax_param.gp.vdw_type, control->nonb_cut,
//                  workspace->d_workspace->tap_coef, workspace->d_workspace->dtap_coef,
//    #if defined(GPU_STREAM_SINGLE_ACCUM)
//                  workspace->d_workspace->f,
//    #else
//                  workspace->d_workspace->f_vdw_clmb,
//    #endif
//                  *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
//    #if defined(GPU_ATOMIC_EV)
//                  &data->d_my_en[E_VDW], &data->d_my_en[E_ELE],
//                  &data->d_my_ext_press
//    #else
//                  spad, &spad[system->n], (rvec *) (&spad[2 * system->n])
//    #endif
//            );

        k_vdW_coulomb_energy_virial_full_opt <<< control->blocks_warp_n, control->gpu_block_size,
                                             sizeof(hipcub::WarpReduce<double>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                             control->gpu_streams[5] >>>
            ( system->d_my_atoms, system->reax_param.d_tbp, system->reax_param.gp.d_l,
              system->reax_param.gp.vdw_type, control->nonb_cut,
              workspace->d_workspace->tap_coef, workspace->d_workspace->dtap_coef,
#if defined(GPU_STREAM_SINGLE_ACCUM)
              workspace->d_workspace->f,
#else
              workspace->d_workspace->f_vdw_clmb,
#endif
              *(lists[FAR_NBRS]), system->n, system->reax_param.num_atom_types, 
#if defined(GPU_ATOMIC_EV)
              &data->d_my_en[E_VDW], &data->d_my_en[E_ELE],
              &data->d_my_ext_press
#else
              spad, &spad[system->n], (rvec *) (&spad[2 * system->n])
#endif
            );
    } else if ( control->tabulate > 0 ) {
        k_vdW_coulomb_energy_tab_full <<< control->blocks_n, control->gpu_block_size,
                                      0, control->gpu_streams[5] >>>
            ( system->d_my_atoms, control->nonb_cut, control->virial,
#if defined(GPU_STREAM_SINGLE_ACCUM)
              workspace->d_workspace->f,
#else
              workspace->d_workspace->f_vdw_clmb,
#endif
              *(lists[FAR_NBRS]), workspace->d_workspace->LR, system->n,
              system->reax_param.num_atom_types, 
#if defined(GPU_ATOMIC_EV)
              &data->d_my_en[E_VDW], &data->d_my_en[E_ELE],
              &data->d_my_ext_press
#else
              spad, &spad[system->n], (rvec *) (&spad[2 * system->n])
#endif
            );
    }
    cudaCheckError( );

#if !defined(GPU_ATOMIC_EV)
    if ( update_energy == TRUE ) {
#if defined(FUSED_VDW_COULOMB)
        /* reduction for vdw */
        GPU_Reduction_Sum( spad, &data->d_my_en[E_VDW], system->n, 5,
                control->gpu_streams[5] );
#endif

        /* reduction for ele */
        GPU_Reduction_Sum(
#if defined(FUSED_VDW_COULOMB)
                &spad[system->n],
#else
                spad,
#endif
                &data->d_my_en[E_ELE], system->n, 5, control->gpu_streams[5] );
    }

    if ( control->virial == 1 ) {
#if defined(FUSED_VDW_COULOMB)
        spad_rvec = (rvec *) (&spad[2 * system->n]);
#else
        spad_rvec = (rvec *) (&spad[system->n]);
#endif

        GPU_Reduction_Sum( spad_rvec, &data->d_my_ext_press,
                system->n, 5, control->gpu_streams[5] );
    }
#endif

    if ( update_energy == TRUE && control->polarization_energy_enabled == TRUE ) {
        GPU_Compute_Polarization_Energy( system, control, workspace, data );
    }

#if defined(LOG_PERFORMANCE)
    hipEventRecord( control->gpu_time_events[TE_COULOMB_STOP], control->gpu_streams[5] );
#endif
}

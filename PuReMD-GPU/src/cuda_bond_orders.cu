#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD-GPU - Reax Force Field Simulator

  Copyright (2014) Purdue University
  Sudhir Kylasa, skylasa@purdue.edu
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_bond_orders.h"

#include "bond_orders.h"
#include "list.h"
#include "lookup.h"
#include "print_utils.h"
#include "vector.h"
#include "index_utils.h"

#include "cuda_utils.h"
#include "cuda_helpers.h"


HOST_DEVICE void Cuda_Add_dBond_to_Forces_NPT( int i, int pj, reax_atom *atoms, 
        simulation_data *data, static_storage *workspace, 
        list *bonds )
{
    bond_data *nbr_j, *nbr_k;
    bond_order_data *bo_ij, *bo_ji; 
    dbond_coefficients coef;
    rvec temp, ext_press;
    ivec rel_box;
    int pk, k, j;

    /* Initializations */
    nbr_j = &(bonds->select.bond_list[pj]);
    j = nbr_j->nbr;
    bo_ij = &(nbr_j->bo_data);
    bo_ji = &(bonds->select.bond_list[ nbr_j->sym_index ].bo_data);

    coef.C1dbo = bo_ij->C1dbo * (bo_ij->Cdbo + bo_ji->Cdbo);
    coef.C2dbo = bo_ij->C2dbo * (bo_ij->Cdbo + bo_ji->Cdbo);
    coef.C3dbo = bo_ij->C3dbo * (bo_ij->Cdbo + bo_ji->Cdbo);

    coef.C1dbopi = bo_ij->C1dbopi * (bo_ij->Cdbopi + bo_ji->Cdbopi);
    coef.C2dbopi = bo_ij->C2dbopi * (bo_ij->Cdbopi + bo_ji->Cdbopi);
    coef.C3dbopi = bo_ij->C3dbopi * (bo_ij->Cdbopi + bo_ji->Cdbopi);
    coef.C4dbopi = bo_ij->C4dbopi * (bo_ij->Cdbopi + bo_ji->Cdbopi);

    coef.C1dbopi2 = bo_ij->C1dbopi2 * (bo_ij->Cdbopi2 + bo_ji->Cdbopi2);
    coef.C2dbopi2 = bo_ij->C2dbopi2 * (bo_ij->Cdbopi2 + bo_ji->Cdbopi2);
    coef.C3dbopi2 = bo_ij->C3dbopi2 * (bo_ij->Cdbopi2 + bo_ji->Cdbopi2);
    coef.C4dbopi2 = bo_ij->C4dbopi2 * (bo_ij->Cdbopi2 + bo_ji->Cdbopi2);

    coef.C1dDelta = bo_ij->C1dbo * (workspace->CdDelta[i]+workspace->CdDelta[j]);
    coef.C2dDelta = bo_ij->C2dbo * (workspace->CdDelta[i]+workspace->CdDelta[j]);
    coef.C3dDelta = bo_ij->C3dbo * (workspace->CdDelta[i]+workspace->CdDelta[j]);


    /************************************
     * forces related to atom i          *
     * first neighbors of atom i         *
     ************************************/
    for( pk = Start_Index(i, bonds); pk < End_Index(i, bonds); ++pk ) {
        nbr_k = &(bonds->select.bond_list[pk]);
        k = nbr_k->nbr;

        rvec_Scale( temp, -coef.C2dbo, nbr_k->bo_data.dBOp );       /*2nd,dBO*/
        rvec_ScaledAdd( temp, -coef.C2dDelta, nbr_k->bo_data.dBOp );/*dDelta*/
        rvec_ScaledAdd( temp, -coef.C3dbopi, nbr_k->bo_data.dBOp ); /*3rd,dBOpi*/
        rvec_ScaledAdd( temp, -coef.C3dbopi2, nbr_k->bo_data.dBOp );/*3rd,dBOpi2*/

        /* force */
        rvec_Add( atoms[k].f, temp );
        /* pressure */
        rvec_iMultiply( ext_press, nbr_k->rel_box, temp );
        rvec_Add( data->ext_press, ext_press );
    }

    /* then atom i itself  */
    rvec_Scale( temp, coef.C1dbo, bo_ij->dBOp );                      /*1st, dBO*/
    rvec_ScaledAdd( temp, coef.C2dbo, workspace->dDeltap_self[i] );   /*2nd, dBO*/

    rvec_ScaledAdd( temp, coef.C1dDelta, bo_ij->dBOp );               /*1st, dBO*/
    rvec_ScaledAdd( temp, coef.C2dDelta, workspace->dDeltap_self[i] );/*2nd, dBO*/

    rvec_ScaledAdd( temp, coef.C1dbopi, bo_ij->dln_BOp_pi );         /*1st,dBOpi*/
    rvec_ScaledAdd( temp, coef.C2dbopi, bo_ij->dBOp );               /*2nd,dBOpi*/
    rvec_ScaledAdd( temp, coef.C3dbopi, workspace->dDeltap_self[i] );/*3rd,dBOpi*/

    rvec_ScaledAdd(temp, coef.C1dbopi2, bo_ij->dln_BOp_pi2) ;      /*1st,dBO_pi2*/
    rvec_ScaledAdd(temp, coef.C2dbopi2, bo_ij->dBOp);              /*2nd,dBO_pi2*/
    rvec_ScaledAdd(temp, coef.C3dbopi2, workspace->dDeltap_self[i]);/*3rd,dBO_pi2*/

    /* force */
    rvec_Add( atoms[i].f, temp );
    /* ext pressure due to i dropped, counting force on j only will be enough */


    /****************************************************************************
     * forces and pressure related to atom j                                    *
     * first neighbors of atom j                                                *
     ***************************************************************************/
    for( pk = Start_Index(j, bonds); pk < End_Index(j, bonds); ++pk ) {
        nbr_k = &(bonds->select.bond_list[pk]);
        k = nbr_k->nbr;

        rvec_Scale( temp, -coef.C3dbo, nbr_k->bo_data.dBOp );       /*3rd,dBO*/
        rvec_ScaledAdd( temp, -coef.C3dDelta, nbr_k->bo_data.dBOp );/*dDelta*/ 
        rvec_ScaledAdd( temp, -coef.C4dbopi, nbr_k->bo_data.dBOp ); /*4th,dBOpi*/
        rvec_ScaledAdd( temp, -coef.C4dbopi2, nbr_k->bo_data.dBOp );/*4th,dBOpi2*/

        /* force */
        rvec_Add( atoms[k].f, temp );
        /* pressure */
        if( k != i ) {
            ivec_Sum(rel_box, nbr_k->rel_box, nbr_j->rel_box);//k's rel_box  wrt i
            rvec_iMultiply( ext_press, rel_box, temp );
            rvec_Add( data->ext_press, ext_press );
        }
    }

    /* then atom j itself */
    rvec_Scale( temp, -coef.C1dbo, bo_ij->dBOp );                     /*1st, dBO*/
    rvec_ScaledAdd( temp, coef.C3dbo, workspace->dDeltap_self[j] );   /*2nd, dBO*/

    rvec_ScaledAdd( temp, -coef.C1dDelta, bo_ij->dBOp );              /*1st, dBO*/
    rvec_ScaledAdd( temp, coef.C3dDelta, workspace->dDeltap_self[j] );/*2nd, dBO*/

    rvec_ScaledAdd( temp, -coef.C1dbopi, bo_ij->dln_BOp_pi );        /*1st,dBOpi*/
    rvec_ScaledAdd( temp, -coef.C2dbopi, bo_ij->dBOp );              /*2nd,dBOpi*/
    rvec_ScaledAdd( temp, coef.C4dbopi, workspace->dDeltap_self[j] );/*3rd,dBOpi*/

    rvec_ScaledAdd(temp, -coef.C1dbopi2, bo_ij->dln_BOp_pi2);       /*1st,dBOpi2*/
    rvec_ScaledAdd(temp, -coef.C2dbopi2, bo_ij->dBOp);              /*2nd,dBOpi2*/
    rvec_ScaledAdd(temp, coef.C4dbopi2, workspace->dDeltap_self[j]);/*3rd,dBOpi2*/

    /* force */
    rvec_Add( atoms[j].f, temp );
    /* pressure */
    rvec_iMultiply( ext_press, nbr_j->rel_box, temp );
    rvec_Add( data->ext_press, ext_press );
}


HOST_DEVICE void Cuda_Add_dBond_to_Forces ( int i, int pj, reax_atom *atoms, 
        static_storage *workspace, list *bonds )
{
    bond_data *nbr_j, *nbr_k;
    bond_order_data *bo_ij, *bo_ji; 
    dbond_coefficients coef;
    int pk, k, j;
    rvec t_f;

    /* Initializations */ 
    nbr_j = &(bonds->select.bond_list[pj]);
    j = nbr_j->nbr;

    if (i < j)
    {
        bo_ij = &(nbr_j->bo_data);
        bo_ji = &(bonds->select.bond_list[ nbr_j->sym_index ].bo_data);
    } else {
        bo_ji = &(nbr_j->bo_data);
        bo_ij = &(bonds->select.bond_list[ nbr_j->sym_index ].bo_data);
    }

    coef.C1dbo = bo_ij->C1dbo * (bo_ij->Cdbo + bo_ji->Cdbo);
    coef.C2dbo = bo_ij->C2dbo * (bo_ij->Cdbo + bo_ji->Cdbo);
    coef.C3dbo = bo_ij->C3dbo * (bo_ij->Cdbo + bo_ji->Cdbo);

    coef.C1dbopi = bo_ij->C1dbopi * (bo_ij->Cdbopi + bo_ji->Cdbopi);
    coef.C2dbopi = bo_ij->C2dbopi * (bo_ij->Cdbopi + bo_ji->Cdbopi);
    coef.C3dbopi = bo_ij->C3dbopi * (bo_ij->Cdbopi + bo_ji->Cdbopi);
    coef.C4dbopi = bo_ij->C4dbopi * (bo_ij->Cdbopi + bo_ji->Cdbopi);

    coef.C1dbopi2 = bo_ij->C1dbopi2 * (bo_ij->Cdbopi2 + bo_ji->Cdbopi2);
    coef.C2dbopi2 = bo_ij->C2dbopi2 * (bo_ij->Cdbopi2 + bo_ji->Cdbopi2);
    coef.C3dbopi2 = bo_ij->C3dbopi2 * (bo_ij->Cdbopi2 + bo_ji->Cdbopi2);
    coef.C4dbopi2 = bo_ij->C4dbopi2 * (bo_ij->Cdbopi2 + bo_ji->Cdbopi2);

    coef.C1dDelta = bo_ij->C1dbo * (workspace->CdDelta[i]+workspace->CdDelta[j]);
    coef.C2dDelta = bo_ij->C2dbo * (workspace->CdDelta[i]+workspace->CdDelta[j]);
    coef.C3dDelta = bo_ij->C3dbo * (workspace->CdDelta[i]+workspace->CdDelta[j]);

    if ( i < j) {
        for( pk = Start_Index(i, bonds); pk < End_Index(i, bonds); ++pk ) {
            nbr_k = &(bonds->select.bond_list[pk]);
            k = nbr_k->nbr;
            rvec_MakeZero (t_f);

            rvec_ScaledAdd( t_f, -coef.C2dbo, nbr_k->bo_data.dBOp ); 
            /*2nd, dBO*/
            rvec_ScaledAdd( t_f, -coef.C2dDelta, nbr_k->bo_data.dBOp );
            /*dDelta*/
            rvec_ScaledAdd( t_f, -coef.C3dbopi, nbr_k->bo_data.dBOp );
            /*3rd, dBOpi*/
            rvec_ScaledAdd( t_f, -coef.C3dbopi2, nbr_k->bo_data.dBOp );
            /*3rd, dBOpi2*/

            //Store in the temp place
            rvec_Add (nbr_k->t_f, t_f);
        }

        rvec_ScaledAdd( atoms[i].f, coef.C1dbo, bo_ij->dBOp );
        /*1st, dBO*/
        rvec_ScaledAdd( atoms[i].f, coef.C2dbo, workspace->dDeltap_self[i] );
        /*2nd, dBO*/

        rvec_ScaledAdd(atoms[i].f, coef.C1dDelta, bo_ij->dBOp);
        /*1st, dBO*/
        rvec_ScaledAdd(atoms[i].f, coef.C2dDelta, workspace->dDeltap_self[i]);
        /*2nd, dBO*/

        rvec_ScaledAdd( atoms[i].f, coef.C1dbopi, bo_ij->dln_BOp_pi );
        /*1st, dBOpi*/
        rvec_ScaledAdd( atoms[i].f, coef.C2dbopi, bo_ij->dBOp );
        /*2nd, dBOpi*/
        rvec_ScaledAdd( atoms[i].f, coef.C3dbopi, workspace->dDeltap_self[i]);
        /*3rd, dBOpi*/

        rvec_ScaledAdd( atoms[i].f, coef.C1dbopi2, bo_ij->dln_BOp_pi2 );
        /*1st, dBO_pi2*/
        rvec_ScaledAdd( atoms[i].f, coef.C2dbopi2, bo_ij->dBOp );
        /*2nd, dBO_pi2*/
        rvec_ScaledAdd( atoms[i].f, coef.C3dbopi2, workspace->dDeltap_self[i]);
        /*3rd, dBO_pi2*/
    }
    else 
    {
        for( pk = Start_Index(i, bonds); pk < End_Index(i, bonds); ++pk ) {
            nbr_k = &(bonds->select.bond_list[pk]);
            k = nbr_k->nbr;
            rvec_MakeZero (t_f);

            rvec_ScaledAdd( t_f, -coef.C3dbo, nbr_k->bo_data.dBOp );
            /*3rd, dBO*/
            rvec_ScaledAdd( t_f, -coef.C3dDelta, nbr_k->bo_data.dBOp );
            /*dDelta*/ 
            rvec_ScaledAdd( t_f, -coef.C4dbopi, nbr_k->bo_data.dBOp );
            /*4th, dBOpi*/
            rvec_ScaledAdd( t_f, -coef.C4dbopi2, nbr_k->bo_data.dBOp );
            /*4th, dBOpi2*/

            //Store in the temp place
            rvec_Add (nbr_k->t_f, t_f);
        }

        rvec_ScaledAdd( atoms[i].f, -coef.C1dbo, bo_ij->dBOp );
        /*1st, dBO*/
        rvec_ScaledAdd( atoms[i].f, coef.C3dbo, workspace->dDeltap_self[i] );
        /*2nd, dBO*/

        rvec_ScaledAdd( atoms[i].f, -coef.C1dDelta, bo_ij->dBOp );
        /*1st, dBO*/
        rvec_ScaledAdd(atoms[i].f, coef.C3dDelta, workspace->dDeltap_self[i]);
        /*2nd, dBO*/

        rvec_ScaledAdd( atoms[i].f, -coef.C1dbopi, bo_ij->dln_BOp_pi );
        /*1st, dBOpi*/
        rvec_ScaledAdd( atoms[i].f, -coef.C2dbopi, bo_ij->dBOp );
        /*2nd, dBOpi*/
        rvec_ScaledAdd(atoms[i].f, coef.C4dbopi, workspace->dDeltap_self[i]);
        /*3rd, dBOpi*/

        rvec_ScaledAdd( atoms[i].f, -coef.C1dbopi2, bo_ij->dln_BOp_pi2 );
        /*1st, dBOpi2*/
        rvec_ScaledAdd( atoms[i].f, -coef.C2dbopi2, bo_ij->dBOp );
        /*2nd, dBOpi2*/
        rvec_ScaledAdd(atoms[i].f, coef.C4dbopi2, workspace->dDeltap_self[i]);
        /*3rd, dBOpi2*/
    }
}


HOST_DEVICE void Cuda_dbond_to_Forces_postprocess (int i, reax_atom *atoms, list *bonds)
{
    int pk;
    bond_data *nbr_k, *nbr_k_sym;

    /*
       for( pk = Start_Index(i, bonds); pk < End_Index(i, bonds); ++pk ) {
       nbr_k = &(bonds->select.bond_list[pk]);
       rvec_Add (atoms[i].f, nbr_k->t_f);
       }
     */

    for( pk = Start_Index(i, bonds); pk < End_Index(i, bonds); ++pk ) {
        nbr_k = &(bonds->select.bond_list[pk]);
        nbr_k_sym = &( bonds->select.bond_list [nbr_k->sym_index] );

        rvec_Add (atoms[i].f, nbr_k_sym->t_f);
    }
}


GLOBAL void Cuda_Calculate_Bond_Orders_Init (  reax_atom *atoms, global_parameters g_params, single_body_parameters *sbp,
        static_storage workspace, int num_atom_types, int N )
{
    int i, type_i;
    real p_boc1, p_boc2;
    single_body_parameters *sbp_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    p_boc1 = g_params.l[0];
    p_boc2 = g_params.l[1];

    /* Calculate Deltaprime, Deltaprime_boc values */
    type_i = atoms[i].type;
    sbp_i = &(sbp[type_i]);
    workspace.Deltap[i] = workspace.total_bond_order[i] - sbp_i->valency;
    workspace.Deltap_boc[i] = 
        workspace.total_bond_order[i] - sbp_i->valency_val;
    workspace.total_bond_order[i] = 0;
}


/* A very important and crucial assumption here is that each segment
   belonging to a different atom in nbrhoods->nbr_list is sorted in its own.
   This can either be done in the general coordinator function or here */
GLOBAL void Cuda_Calculate_Bond_Orders (  reax_atom *atoms, global_parameters g_params, single_body_parameters *sbp,
        two_body_parameters *tbp, static_storage workspace, list bonds,
        list dDeltas, list dBOs, int num_atom_types, int N )
{
    int i, j, pj, type_i, type_j;
    int start_i, end_i;
    int num_bonds, sym_index;
    real p_boc1, p_boc2;
    real val_i, Deltap_i, Deltap_boc_i;
    real val_j, Deltap_j, Deltap_boc_j;
    real temp, f1, f2, f3, f4, f5, f4f5, exp_f4, exp_f5;
    real exp_p1i,    exp_p2i, exp_p1j, exp_p2j;
    real u1_ij, u1_ji, Cf1A_ij, Cf1B_ij, Cf1_ij, Cf1_ji;
    real Cf45_ij, Cf45_ji, p_lp1;
    real A0_ij, A1_ij, A2_ij, A2_ji, A3_ij, A3_ji;
    real explp1;
    two_body_parameters *twbp;
    bond_order_data *bo_ij, *bo_ji;
    single_body_parameters *sbp_i, *sbp_j;

#if defined(TEST_FORCES)
    int  k, pk, start_j, end_j;
    int  top_dbo=0, top_dDelta=0;
    dbond_data *pdbo;
    dDelta_data *ptop_dDelta;

#endif

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    num_bonds = 0;
    p_boc1 = g_params.l[0];
    p_boc2 = g_params.l[1];

    /* Calculate Deltaprime, Deltaprime_boc values */
    //for( i = 0; i < system->N; ++i ) {
    /*
       if (i < N) {
       type_i = atoms[i].type;
       sbp_i = &(sbp[type_i]);
       workspace.Deltap[i] = workspace.total_bond_order[i] - sbp_i->valency;
       workspace.Deltap_boc[i] = 
       workspace.total_bond_order[i] - sbp_i->valency_val;
       workspace.total_bond_order[i] = 0;

       }

       __syncthreads ();
     */


    // fprintf( stderr, "done with uncorrected bond orders\n" );

    /* Corrected Bond Order calculations */
    //for( i = 0; i < system->N; ++i ) {
    type_i = atoms[i].type;
    sbp_i = &(sbp[type_i]);
    val_i = sbp_i->valency;
    Deltap_i = workspace.Deltap[i];
    Deltap_boc_i = workspace.Deltap_boc[i];
    start_i = Start_Index(i, &bonds);
    end_i = End_Index(i, &bonds);
    //fprintf( stderr, "i:%d Dp:%g Dbocp:%g s:%d e:%d\n",
    //       i+1, Deltap_i, Deltap_boc_i, start_i, end_i );

    for( pj = start_i; pj < end_i; ++pj ) {
        j = bonds.select.bond_list[pj].nbr;
        type_j = atoms[j].type;
        bo_ij = &( bonds.select.bond_list[pj].bo_data );
        //fprintf( stderr, "\tj:%d - ubo: %8.3f\n", j+1, bo_ij->BO );

        if( i < j ) {
            twbp = &( tbp[ index_tbp (type_i,type_j,num_atom_types) ] );          
#ifdef TEST_FORCES
            Set_Start_Index( pj, top_dbo, &dBOs );
            /* fprintf( stderr, "%6d%6d%23.15e%23.15e%23.15e\n", 
               workspace->reverse_map[i], workspace->reverse_map[j], 
               twbp->ovc, twbp->v13cor, bo_ij->BO ); */
#endif
            if( twbp->ovc < 0.001 && twbp->v13cor < 0.001 ) {
                /* There is no correction to bond orders nor to derivatives of 
                   bond order prime! So we leave bond orders unchanged and 
                   set derivative of bond order coefficients s.t. 
                   dBO = dBOp & dBOxx = dBOxxp in Add_dBO_to_Forces */
                bo_ij->C1dbo = 1.0;
                bo_ij->C2dbo = 0.0;
                bo_ij->C3dbo = 0.0; 

                bo_ij->C1dbopi = 1.0;
                bo_ij->C2dbopi = 0.0;
                bo_ij->C3dbopi = 0.0;
                bo_ij->C4dbopi = 0.0;

                bo_ij->C1dbopi2 = 1.0; 
                bo_ij->C2dbopi2 = 0.0;
                bo_ij->C3dbopi2 = 0.0;
                bo_ij->C4dbopi2 = 0.0;

#ifdef TEST_FORCES
                pdbo = &(dBOs.select.dbo_list[ top_dbo ]);

                // compute dBO_ij/dr_i
                pdbo->wrt = i;
                rvec_Copy( pdbo->dBO, bo_ij->dBOp );
                rvec_Scale( pdbo->dBOpi, bo_ij->BO_pi, bo_ij->dln_BOp_pi );
                rvec_Scale( pdbo->dBOpi2, bo_ij->BO_pi2, bo_ij->dln_BOp_pi2 );

                // compute dBO_ij/dr_j
                pdbo++;
                pdbo->wrt = j;
                rvec_Scale( pdbo->dBO,-1.0,bo_ij->dBOp );
                rvec_Scale( pdbo->dBOpi,-bo_ij->BO_pi,bo_ij->dln_BOp_pi );
                rvec_Scale( pdbo->dBOpi2,-bo_ij->BO_pi2,bo_ij->dln_BOp_pi2 );

                top_dbo += 2;
#endif
            }
            else {
                val_j = sbp[type_j].valency;
                Deltap_j = workspace.Deltap[j];
                Deltap_boc_j = workspace.Deltap_boc[j];

                /* on page 1 */
                if( twbp->ovc >= 0.001 ) {
                    /* Correction for overcoordination */        
                    exp_p1i = EXP( -p_boc1 * Deltap_i );
                    exp_p2i = EXP( -p_boc2 * Deltap_i );
                    exp_p1j = EXP( -p_boc1 * Deltap_j );
                    exp_p2j = EXP( -p_boc2 * Deltap_j );

                    f2 = exp_p1i + exp_p1j;            
                    f3 = -1.0 / p_boc2 * log( 0.5 * ( exp_p2i  + exp_p2j ) );
                    f1 = 0.5 * ( ( val_i + f2 )/( val_i + f2 + f3 ) + 
                            ( val_j + f2 )/( val_j + f2 + f3 ) );

                    /*fprintf( stderr,"%6d%6d\t%g %g   j:%g %g  p_boc:%g %g\n",
                      i+1, j+1, val_i, Deltap_i, val_j, Deltap_j, p_boc1, p_boc2 );
                      fprintf( stderr,"\tf:%g  %g  %g, exp:%g %g %g %g\n", 
                      f1, f2, f3, exp_p1i, exp_p2i, exp_p1j, exp_p2j );*/

                    /* Now come the derivates */        
                    /* Bond Order pages 5-7, derivative of f1 */
                    temp = f2 + f3;
                    u1_ij = val_i + temp;
                    u1_ji = val_j + temp;
                    Cf1A_ij = 0.5 * f3 * (1.0 / SQR( u1_ij ) + 1.0 / SQR( u1_ji ));
                    Cf1B_ij = -0.5 * (( u1_ij - f3 ) / SQR( u1_ij ) + 
                            ( u1_ji - f3 ) / SQR( u1_ji ));

                    //Cf1_ij = -Cf1A_ij * p_boc1 * exp_p1i + 
                    //          Cf1B_ij * exp_p2i / ( exp_p2i + exp_p2j );
                    Cf1_ij = 0.50 * ( -p_boc1 * exp_p1i / u1_ij - 
                            ((val_i+f2) / SQR(u1_ij)) * 
                            ( -p_boc1 * exp_p1i + 
                              exp_p2i / ( exp_p2i + exp_p2j ) ) + 
                            -p_boc1 * exp_p1i / u1_ji - 
                            ((val_j+f2)/SQR(u1_ji)) * ( -p_boc1*exp_p1i +  
                            exp_p2i / ( exp_p2i + exp_p2j ) ));

                    Cf1_ji = -Cf1A_ij * p_boc1 * exp_p1j + 
                        Cf1B_ij * exp_p2j / ( exp_p2i + exp_p2j ); 
                    //fprintf( stderr, "\tCf1:%g  %g\n", Cf1_ij, Cf1_ji );
                }
                else {
                    /* No overcoordination correction! */
                    f1 = 1.0;
                    Cf1_ij = Cf1_ji = 0.0;          
                }

                if( twbp->v13cor >= 0.001 ) {
                    /* Correction for 1-3 bond orders */
                    exp_f4 =EXP(-(twbp->p_boc4 * SQR( bo_ij->BO ) - 
                                Deltap_boc_i) * twbp->p_boc3 + twbp->p_boc5);
                    exp_f5 =EXP(-(twbp->p_boc4 * SQR( bo_ij->BO ) - 
                                Deltap_boc_j) * twbp->p_boc3 + twbp->p_boc5);

                    f4 = 1. / (1. + exp_f4);
                    f5 = 1. / (1. + exp_f5);
                    f4f5 = f4 * f5;

                    /* Bond Order pages 8-9, derivative of f4 and f5 */
                    /*temp = twbp->p_boc5 - 
                      twbp->p_boc3 * twbp->p_boc4 * SQR( bo_ij->BO );
                      u_ij = temp + twbp->p_boc3 * Deltap_boc_i;
                      u_ji = temp + twbp->p_boc3 * Deltap_boc_j;
                      Cf45_ij = Cf45( u_ij, u_ji ) / f4f5;
                      Cf45_ji = Cf45( u_ji, u_ij ) / f4f5;*/
                    Cf45_ij = -f4 * exp_f4;
                    Cf45_ji = -f5 * exp_f5;
                }
                else {
                    f4 = f5 = f4f5 = 1.0;
                    Cf45_ij = Cf45_ji = 0.0;
                }

                /* Bond Order page 10, derivative of total bond order */
                A0_ij = f1 * f4f5;
                A1_ij = -2 * twbp->p_boc3 * twbp->p_boc4 * bo_ij->BO * 
                    (Cf45_ij + Cf45_ji);
                A2_ij = Cf1_ij / f1 + twbp->p_boc3 * Cf45_ij;
                A2_ji = Cf1_ji / f1 + twbp->p_boc3 * Cf45_ji;
                A3_ij = A2_ij + Cf1_ij / f1;
                A3_ji = A2_ji + Cf1_ji / f1;

                /*fprintf( stderr, "\tBO: %f, A0: %f, A1: %f, A2_ij: %f 
A2_ji: %f, A3_ij: %f, A3_ji: %f\n",
bo_ij->BO, A0_ij, A1_ij, A2_ij, A2_ji, A3_ij, A3_ji );*/

                /* find corrected bond order values and their deriv coefs */
                bo_ij->BO    = bo_ij->BO    * A0_ij;
                bo_ij->BO_pi = bo_ij->BO_pi * A0_ij *f1;
                bo_ij->BO_pi2= bo_ij->BO_pi2* A0_ij *f1;
                bo_ij->BO_s  = bo_ij->BO - ( bo_ij->BO_pi + bo_ij->BO_pi2 );

                bo_ij->C1dbo = A0_ij + bo_ij->BO * A1_ij;
                bo_ij->C2dbo = bo_ij->BO * A2_ij;
                bo_ij->C3dbo = bo_ij->BO * A2_ji; 

                bo_ij->C1dbopi = f1*f1*f4*f5;
                bo_ij->C2dbopi = bo_ij->BO_pi * A1_ij;
                bo_ij->C3dbopi = bo_ij->BO_pi * A3_ij;
                bo_ij->C4dbopi = bo_ij->BO_pi * A3_ji;

                bo_ij->C1dbopi2 = f1*f1*f4*f5;
                bo_ij->C2dbopi2 = bo_ij->BO_pi2 * A1_ij;
                bo_ij->C3dbopi2 = bo_ij->BO_pi2 * A3_ij;

#ifdef TEST_FORCES
                /*fprintf( stderr, "%6d%6d%13.6f%13.6f%13.6f%13.6f\n", 
                  i+1, j+1, bo_ij->BO, bo_ij->C1dbo, Cf45_ij, Cf45_ji );*/

                /* fprintf( stderr, "%6d%6d%13.6f%13.6f%13.6f%13.6f\n",
                //"%6d%6d%10.6f%10.6f%10.6f%10.6f\n%10.6f%10.6f%10.6f\n%10.6f%10.6f%10.6f%10.6f\n%10.6f%10.6f%10.6f%10.6f\n\n",
                workspace->orig_id[i], workspace->orig_id[j]
                A0_ij, A1_ij, A2_ij, A2_ji, A3_ij, A3_ji
                bo_ij->BO, bo_ij->BO_pi, bo_ij->BO_pi2, bo_ij->BO_s,
                bo_ij->C1dbo, bo_ij->C2dbo, bo_ij->C3dbo, 
                bo_ij->C1dbopi,bo_ij->C2dbopi,bo_ij->C3dbopi,bo_ij->C4dbopi,
                bo_ij->C1dbopi2,bo_ij->C2dbopi2,bo_ij->C3dbopi2,bo_ij->C4dbopi2
                ); */

                Calculate_dBO( i, pj, workspace, lists, &top_dbo );
#endif
            }

            /* neglect bonds that are < 1e-10 */
            if( bo_ij->BO < 1e-10 )
                bo_ij->BO = 0.0;
            if( bo_ij->BO_s < 1e-10 )
                bo_ij->BO_s = 0.0;
            if( bo_ij->BO_pi < 1e-10 )
                bo_ij->BO_pi = 0.0;
            if( bo_ij->BO_pi2 < 1e-10 )
                bo_ij->BO_pi2 = 0.0;

            workspace.total_bond_order[i] += bo_ij->BO; // now keeps total_BO


            /* fprintf( stderr, "%d %d\t%g %g %g %g\n
Cdbo:\t%g %g %g\n
Cdbopi:\t%g %g %g %g\n
Cdbopi2:%g %g %g %g\n\n", 
i+1, j+1, bonds->select.bond_list[ pj ].d, 
bo_ij->BO,bo_ij->BO_pi, bo_ij->BO_pi2, 
bo_ij->C1dbo, bo_ij->C2dbo, bo_ij->C3dbo,
bo_ij->C1dbopi, bo_ij->C2dbopi, bo_ij->C3dbopi, bo_ij->C4dbopi,
bo_ij->C1dbopi2, bo_ij->C2dbopi2, 
bo_ij->C3dbopi2, bo_ij->C4dbopi2 ); */

            /* fprintf( stderr, "%d %d, BO:%f BO_s:%f BO_pi:%f BO_pi2:%f\n",
               i+1,j+1,bo_ij->BO,bo_ij->BO_s,bo_ij->BO_pi,bo_ij->BO_pi2 ); */

#ifdef TEST_FORCES
            Set_End_Index( pj, top_dbo, &dBOs );
            //Add_dBO( system, lists, i, pj, 1.0, workspace->dDelta );
#endif
        }
        /*
           else {
        // We only need to update bond orders from bo_ji
        //   everything else is set in uncorrected_bo calculations 
        sym_index = bonds.select.bond_list[pj].sym_index;
        bo_ji = &(bonds.select.bond_list[ sym_index ].bo_data);
        bo_ij->BO = bo_ji->BO;
        bo_ij->BO_s = bo_ji->BO_s;
        bo_ij->BO_pi = bo_ji->BO_pi;
        bo_ij->BO_pi2 = bo_ji->BO_pi2;

        workspace.total_bond_order[i] += bo_ij->BO; // now keeps total_BO

#ifdef TEST_FORCES
        //Add_dBO( system, lists, j, sym_index, 1.0, workspace.dDelta );
#endif
}      
         */
    }

#ifdef TEST_FORCES 
    // fprintf( stderr, "dDelta computations\nj:" );
    Set_Start_Index( i, top_dDelta, &dDeltas );
    ptop_dDelta = &( dDeltas.select.dDelta_list[top_dDelta] );
    
    for( pj = start_i; pj < end_i; ++pj ) {
        j = bonds.select.bond_list[pj].nbr;
        // fprintf( stderr, "%d  ", j );
    
        if( !rvec_isZero( workspace.dDelta[j] ) ) {
            ptop_dDelta->wrt = j;
            rvec_Copy( ptop_dDelta->dVal, workspace.dDelta[j] );
            rvec_MakeZero( workspace.dDelta[j] );
            ++top_dDelta, ++ptop_dDelta;
        }
    
        start_j = Start_Index(j, &bonds);
        end_j = End_Index(j, &bonds);     
        for( pk = start_j; pk < end_j; ++pk ) {
            k = bonds.select.bond_list[pk].nbr;    
            if( !rvec_isZero( workspace.dDelta[k] ) ) {
                ptop_dDelta->wrt = k;
                rvec_Copy( ptop_dDelta->dVal, workspace.dDelta[k] );
                rvec_MakeZero( workspace.dDelta[k] );
                ++top_dDelta, ++ptop_dDelta;
            }
        }
    }
    
    Set_End_Index( i, top_dDelta, &dDeltas );
    
    /*for( pj=Start_Index(i,dDeltas); pj<End_Index(i,dDeltas); ++pj )
      fprintf( stdout, "dDel: %d %d [%g %g %g]\n",
      i+1, dDeltas->select.dDelta_list[pj].wrt+1,
      dDeltas->select.dDelta_list[pj].dVal[0], 
      dDeltas->select.dDelta_list[pj].dVal[1], 
      dDeltas->select.dDelta_list[pj].dVal[2] );*/
#endif
    //}
    
    /*fprintf(stderr,"\tCalculated actual bond orders ...\n" );
      fprintf(stderr,"%6s%8s%8s%8s%8s%8s%8s%8s\n", 
      "atom", "Delta", "Delta_e", "Delta_boc", "nlp", 
      "Delta_lp", "Clp", "dDelta_lp" );*/
    
    /*
       p_lp1 = g_params.l[15];
    
    //get the kernel ID for the following computation
    j = i;
    
    // Calculate some helper variables that are  used at many places 
    //  throughout force calculations 
    //for( j = 0; j < system->N; ++j ) {
    type_j = atoms[j].type;
    sbp_j = &(sbp[ type_j ]);
    
    workspace.Delta[j] = workspace.total_bond_order[j] - sbp_j->valency;
    workspace.Delta_e[j] = workspace.total_bond_order[j] - sbp_j->valency_e;
    workspace.Delta_boc[j] = workspace.total_bond_order[j] - 
    sbp_j->valency_boc;
    
    workspace.vlpex[j] =  workspace.Delta_e[j] - 
    2.0 * (int)(workspace.Delta_e[j]/2.0);
    explp1 = EXP(-p_lp1 * SQR(2.0 + workspace.vlpex[j]));
    workspace.nlp[j] = explp1 - (int)(workspace.Delta_e[j] / 2.0);
    workspace.Delta_lp[j] = sbp_j->nlp_opt - workspace.nlp[j];
    workspace.Clp[j] = 2.0 * p_lp1 * explp1 * (2.0 + workspace.vlpex[j]);
    // Adri uses different dDelta_lp values than the ones in notes... //
    workspace.dDelta_lp[j] = workspace.Clp[j];
    //workspace->dDelta_lp[j] = workspace->Clp[j] + (0.5-workspace->Clp[j]) *
    //((fabs(workspace->Delta_e[j]/2.0 - 
    //       (int)(workspace->Delta_e[j]/2.0)) < 0.1) ? 1 : 0 );
    
    if( sbp_j->mass > 21.0 ) {
    workspace.nlp_temp[j] = 0.5 * (sbp_j->valency_e - sbp_j->valency);
    workspace.Delta_lp_temp[j] = sbp_j->nlp_opt - workspace.nlp_temp[j];
    workspace.dDelta_lp_temp[j] = 0.;
    }
    else {
    workspace.nlp_temp[j] = workspace.nlp[j];
    workspace.Delta_lp_temp[j] = sbp_j->nlp_opt - workspace.nlp_temp[j];
    workspace.dDelta_lp_temp[j] = workspace.Clp[j];
    }
    
    //fprintf( stderr, "%d\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\n",
    //j, workspace->Delta[j], workspace->Delta_e[j], workspace->Delta_boc[j], 
    //workspace->nlp[j], system->reaxprm.sbp[type_j].nlp_opt,
    //workspace->Delta_lp[j], workspace->Clp[j], workspace->dDelta_lp[j] );
    //}
     */
    
    //Print_Bonds( system, bonds, "sbonds.out" );
    
#if defined(DEBUG)
    //fprintf( stderr, "Number of bonds: %d\n", num_bonds );
    //Print_Bond_Orders( system, control, data, workspace, lists, out_control );
#endif
}


GLOBAL void Cuda_Update_Uncorrected_BO (  static_storage workspace, list bonds, int N )
{
    int i, j, pj;
    int start_i, end_i;
    int sym_index;

    bond_order_data *bo_ij, *bo_ji;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    start_i = Start_Index(i, &bonds);
    end_i = End_Index(i, &bonds);

    for( pj = start_i; pj < end_i; ++pj ) {

        j = bonds.select.bond_list[pj].nbr;
        bo_ij = &( bonds.select.bond_list[pj].bo_data );

        if( i >= j ) {
            // We only need to update bond orders from bo_ji
            //   everything else is set in uncorrected_bo calculations 
            sym_index = bonds.select.bond_list[pj].sym_index;
            bo_ji = &(bonds.select.bond_list[ sym_index ].bo_data);
            bo_ij->BO = bo_ji->BO;
            bo_ij->BO_s = bo_ji->BO_s;
            bo_ij->BO_pi = bo_ji->BO_pi;
            bo_ij->BO_pi2 = bo_ji->BO_pi2;

            workspace.total_bond_order[i] += bo_ij->BO; // now keeps total_BO
        }
    }
}


GLOBAL void Cuda_Update_Workspace_After_Bond_Orders(  reax_atom *atoms, global_parameters g_params, single_body_parameters *sbp,
        static_storage workspace, int N )
{
    int j, type_j;
    real explp1;
    real p_lp1;
    single_body_parameters *sbp_i, *sbp_j;

    j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= N) return;

    p_lp1 = g_params.l[15];

    /* Calculate some helper variables that are  used at many places 
       throughout force calculations */
    //for( j = 0; j < system->N; ++j ) {
    type_j = atoms[j].type;
    sbp_j = &(sbp[ type_j ]);

    workspace.Delta[j] = workspace.total_bond_order[j] - sbp_j->valency;
    workspace.Delta_e[j] = workspace.total_bond_order[j] - sbp_j->valency_e;
    workspace.Delta_boc[j] = workspace.total_bond_order[j] - 
        sbp_j->valency_boc;

    workspace.vlpex[j] =  workspace.Delta_e[j] - 
        2.0 * (int)(workspace.Delta_e[j]/2.0);
    explp1 = EXP(-p_lp1 * SQR(2.0 + workspace.vlpex[j]));
    workspace.nlp[j] = explp1 - (int)(workspace.Delta_e[j] / 2.0);
    workspace.Delta_lp[j] = sbp_j->nlp_opt - workspace.nlp[j];
    workspace.Clp[j] = 2.0 * p_lp1 * explp1 * (2.0 + workspace.vlpex[j]);
    /* Adri uses different dDelta_lp values than the ones in notes... */
    workspace.dDelta_lp[j] = workspace.Clp[j];
    //workspace->dDelta_lp[j] = workspace->Clp[j] + (0.5-workspace->Clp[j]) *
    //((fabs(workspace->Delta_e[j]/2.0 - 
    //       (int)(workspace->Delta_e[j]/2.0)) < 0.1) ? 1 : 0 );

    if( sbp_j->mass > 21.0 ) {
        workspace.nlp_temp[j] = 0.5 * (sbp_j->valency_e - sbp_j->valency);
        workspace.Delta_lp_temp[j] = sbp_j->nlp_opt - workspace.nlp_temp[j];
        workspace.dDelta_lp_temp[j] = 0.;
    }
    else {
        workspace.nlp_temp[j] = workspace.nlp[j];
        workspace.Delta_lp_temp[j] = sbp_j->nlp_opt - workspace.nlp_temp[j];
        workspace.dDelta_lp_temp[j] = workspace.Clp[j];
    }

    //fprintf( stderr, "%d\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\t%lf\n",
    //j, workspace->Delta[j], workspace->Delta_e[j], workspace->Delta_boc[j], 
    //workspace->nlp[j], system->reaxprm.sbp[type_j].nlp_opt,
    //workspace->Delta_lp[j], workspace->Clp[j], workspace->dDelta_lp[j] );
    //}

}


//Import from the forces file. 
GLOBAL void Cuda_Compute_Total_Force (reax_atom *atoms, simulation_data *data, 
        static_storage workspace, list p_bonds, int ensemble, int N)
{
    int i, pj;
    list *bonds = &p_bonds;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) 
    {
        for (pj = Start_Index (i, bonds); pj < End_Index (i, bonds); ++pj)
        {
            //int j = bonds->select.bond_list[pj].nbr;
            if (ensemble == NVE || ensemble == NVT || ensemble == bNVT) 
                Cuda_Add_dBond_to_Forces (i, pj, atoms, &workspace, bonds );
            else 
                Cuda_Add_dBond_to_Forces_NPT (i, pj, atoms, data, &workspace, bonds );
        }
    }
}


GLOBAL void Cuda_Compute_Total_Force_PostProcess (reax_atom *atoms, simulation_data *data, 
        static_storage workspace, list p_bonds, int ensemble, int N)
{
    int i, pj;
    list *bonds = &p_bonds;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) 
    {
        if (ensemble == NVE || ensemble == NVT || ensemble == bNVT) 
            Cuda_dbond_to_Forces_postprocess (i, atoms, bonds );
    }
}

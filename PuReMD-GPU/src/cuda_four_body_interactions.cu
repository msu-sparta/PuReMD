#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD-GPU - Reax Force Field Simulator

  Copyright (2014) Purdue University
  Sudhir Kylasa, skylasa@purdue.edu
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_four_body_interactions.h"

#include "box.h"
#include "index_utils.h"
#include "list.h"
#include "vector.h"

#include "cuda_helpers.h"

#define MIN_SINE 1e-10


DEVICE real Calculate_Omega( rvec dvec_ij, real r_ij, rvec dvec_jk, real r_jk,
        rvec dvec_kl, real r_kl, rvec dvec_li, real r_li,
        three_body_interaction_data *p_ijk, 
        three_body_interaction_data *p_jkl, 
        rvec dcos_omega_di, rvec dcos_omega_dj, 
        rvec dcos_omega_dk, rvec dcos_omega_dl, 
        output_controls *out_control )
{
    real unnorm_cos_omega, unnorm_sin_omega, omega;
    real sin_ijk, cos_ijk, sin_jkl, cos_jkl;
    real htra, htrb, htrc, hthd, hthe, hnra, hnrc, hnhd, hnhe;
    real arg, poem, tel;
    rvec cross_jk_kl;

    sin_ijk = SIN( p_ijk->theta );
    cos_ijk = COS( p_ijk->theta );
    sin_jkl = SIN( p_jkl->theta );
    cos_jkl = COS( p_jkl->theta );

    /* omega */
    unnorm_cos_omega = -rvec_Dot( dvec_ij,dvec_jk )*rvec_Dot( dvec_jk,dvec_kl ) +
        SQR( r_jk ) *  rvec_Dot( dvec_ij,dvec_kl );
    rvec_Cross( cross_jk_kl, dvec_jk, dvec_kl );
    unnorm_sin_omega = -r_jk * rvec_Dot( dvec_ij, cross_jk_kl );
    omega = atan2( unnorm_sin_omega, unnorm_cos_omega ); 

    /* derivatives */
    /* coef for adjusments to cos_theta's */
    /* rla = r_ij, rlb = r_jk, rlc = r_kl, r4 = r_li;
       coshd = cos_ijk, coshe = cos_jkl;
       sinhd = sin_ijk, sinhe = sin_jkl; */
    htra = r_ij + cos_ijk * ( r_kl * cos_jkl - r_jk );
    htrb = r_jk - r_ij * cos_ijk - r_kl * cos_jkl;
    htrc = r_kl + cos_jkl * ( r_ij * cos_ijk - r_jk );
    hthd = r_ij * sin_ijk * ( r_jk - r_kl * cos_jkl );
    hthe = r_kl * sin_jkl * ( r_jk - r_ij * cos_ijk );
    hnra = r_kl * sin_ijk * sin_jkl;
    hnrc = r_ij * sin_ijk * sin_jkl;
    hnhd = r_ij * r_kl * cos_ijk * sin_jkl;
    hnhe = r_ij * r_kl * sin_ijk * cos_jkl;

    poem = 2.0 * r_ij * r_kl * sin_ijk * sin_jkl;
    if( poem < 1e-20 ) poem = 1e-20;

    tel  = (SQR(r_ij) + SQR(r_jk) + SQR(r_kl) - SQR(r_li)) - 
        2.0 * ( r_ij * r_jk * cos_ijk - r_ij * r_kl * cos_ijk * cos_jkl + 
                r_jk * r_kl * cos_jkl );

    arg  = tel / poem;
    if( arg >  1.0 )
    {
        arg =  1.0;
    }
    if( arg < -1.0 )
    {
        arg = -1.0;
    }

    /*fprintf( out_control->etor, 
      "%23.15e%23.15e%23.15e%23.15e%23.15e%23.15e%23.15e%23.15e%23.15e\n",
      htra, htrb, htrc, hthd, hthe, hnra, hnrc, hnhd, hnhe );
      fprintf( out_control->etor, "%23.15e%23.15e%23.15e\n",
      dvec_ij[0]/r_ij, dvec_ij[1]/r_ij, dvec_ij[2]/r_ij );
      fprintf( out_control->etor, "%23.15e%23.15e%23.15e\n",
      -dvec_jk[0]/r_jk, -dvec_jk[1]/r_jk, -dvec_jk[2]/r_jk );
      fprintf( out_control->etor, "%23.15e%23.15e%23.15e\n",
      -dvec_kl[0]/r_kl, -dvec_kl[1]/r_kl, -dvec_kl[2]/r_kl );
      fprintf( out_control->etor, "%23.15e%23.15e%23.15e%23.15e\n",
      r_li, dvec_li[0], dvec_li[1], dvec_li[2] );
      fprintf( out_control->etor, "%23.15e%23.15e%23.15e%23.15e\n",
      r_ij, r_jk, r_kl, r_li ); 
      fprintf( out_control->etor, "%23.15e%23.15e%23.15e%23.15e\n", 
      cos_ijk, cos_jkl, sin_ijk, sin_jkl ); 
      fprintf( out_control->etor, "%23.15e%23.15e%23.15e\n",
      poem, tel, arg );*/
    /* fprintf( out_control->etor, "%23.15e%23.15e%23.15e\n",
       -p_ijk->dcos_dk[0]/sin_ijk, 
       -p_ijk->dcos_dk[1]/sin_ijk, 
       -p_ijk->dcos_dk[2]/sin_ijk );
       fprintf( out_control->etor, "%23.15e%23.15e%23.15e\n",
       -p_jkl->dcos_dk[0]/sin_jkl, 
       -p_jkl->dcos_dk[1]/sin_jkl, 
       -p_jkl->dcos_dk[2]/sin_jkl );*/

    if( sin_ijk >= 0 && sin_ijk <= MIN_SINE )
    {
        sin_ijk = MIN_SINE;
    }
    else if( sin_ijk <= 0 && sin_ijk >= -MIN_SINE )
    {
        sin_ijk = -MIN_SINE;
    }
    if( sin_jkl >= 0 && sin_jkl <= MIN_SINE )
    {
        sin_jkl = MIN_SINE;
    }
    else if( sin_jkl <= 0 && sin_jkl >= -MIN_SINE )
    {
        sin_jkl = -MIN_SINE;
    }

    // dcos_omega_di
    rvec_ScaledSum( dcos_omega_di, (htra-arg*hnra)/r_ij, dvec_ij, -1., dvec_li );
    rvec_ScaledAdd( dcos_omega_di,-(hthd - arg*hnhd)/sin_ijk, p_ijk->dcos_dk );
    rvec_Scale( dcos_omega_di, 2.0 / poem, dcos_omega_di );

    // dcos_omega_dj
    rvec_ScaledSum( dcos_omega_dj,-(htra-arg*hnra)/r_ij, dvec_ij, 
            -htrb / r_jk, dvec_jk );
    rvec_ScaledAdd( dcos_omega_dj,-(hthd-arg*hnhd) / sin_ijk, p_ijk->dcos_dj );
    rvec_ScaledAdd( dcos_omega_dj,-(hthe-arg*hnhe) / sin_jkl, p_jkl->dcos_di );
    rvec_Scale( dcos_omega_dj, 2.0 / poem, dcos_omega_dj );

    // dcos_omega_dk
    rvec_ScaledSum( dcos_omega_dk,-(htrc-arg*hnrc) / r_kl, dvec_kl,  
            htrb / r_jk, dvec_jk );
    rvec_ScaledAdd( dcos_omega_dk,-(hthd-arg*hnhd) / sin_ijk, p_ijk->dcos_di );
    rvec_ScaledAdd( dcos_omega_dk,-(hthe-arg*hnhe) / sin_jkl, p_jkl->dcos_dj );
    rvec_Scale( dcos_omega_dk, 2.0 / poem, dcos_omega_dk );

    // dcos_omega_dl
    rvec_ScaledSum( dcos_omega_dl, (htrc-arg*hnrc) / r_kl, dvec_kl, 1., dvec_li );
    rvec_ScaledAdd( dcos_omega_dl,-(hthe-arg*hnhe) / sin_jkl, p_jkl->dcos_dk );
    rvec_Scale( dcos_omega_dl, 2.0 / poem, dcos_omega_dl );

    return omega;  
    //return arg;
}


GLOBAL void k_Four_Body_Interactions ( reax_atom *atoms, 
        global_parameters g_params,
        four_body_header *d_fbp,
        control_params *control,
        list p_bonds, list p_thb_intrs,
        simulation_box *box,
        simulation_data *data,
        static_storage p_workspace,
        int N, int num_atom_types, 
        real *E_Tor, real *E_Con, rvec *aux_ext_press)
{
    /*
       extern __shared__ real _tor[];
       extern __shared__ real _con [];
       extern __shared__ rvec _press[];
       real *sh_tor, *sh_con; rvec *sh_press;
     */

    int i, j, k, l, pi, pj, pk, pl, pij, plk;
    int type_i, type_j, type_k, type_l;
    int start_j, end_j, start_k, end_k;
    int start_pj, end_pj, start_pk, end_pk;
    int num_frb_intrs = 0;

    real Delta_j, Delta_k;
    real r_ij, r_jk, r_kl, r_li;
    real BOA_ij, BOA_jk, BOA_kl;

    real exp_tor2_ij, exp_tor2_jk, exp_tor2_kl;
    real exp_tor1, exp_tor3_DjDk, exp_tor4_DjDk, exp_tor34_inv;
    real exp_cot2_jk, exp_cot2_ij, exp_cot2_kl;
    real fn10, f11_DjDk, dfn11, fn12;

    real theta_ijk, theta_jkl;
    real sin_ijk, sin_jkl;
    real cos_ijk, cos_jkl;
    real tan_ijk_i, tan_jkl_i;

    real omega, cos_omega, cos2omega, cos3omega;
    rvec dcos_omega_di, dcos_omega_dj, dcos_omega_dk, dcos_omega_dl;

    real CV, cmn, CEtors1, CEtors2, CEtors3, CEtors4;
    real CEtors5, CEtors6, CEtors7, CEtors8, CEtors9;
    real Cconj, CEconj1, CEconj2, CEconj3;
    real CEconj4, CEconj5, CEconj6;

    real e_tor, e_con;
    rvec dvec_li;
    rvec force, ext_press;
    ivec rel_box_jl;
    // rtensor total_rtensor, temp_rtensor;

    four_body_header *fbh;
    four_body_parameters *fbp;
    bond_data *pbond_ij, *pbond_jk, *pbond_kl;
    bond_order_data *bo_ij, *bo_jk, *bo_kl;
    three_body_interaction_data *p_ijk, *p_jkl;

    j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= N) return;
    //    j = blockIdx.x;

    real p_tor2 = g_params.l[23];
    real p_tor3 = g_params.l[24];
    real p_tor4 = g_params.l[25];
    real p_cot2 = g_params.l[27];

    list *bonds = &p_bonds;
    list *thb_intrs = &p_thb_intrs;
    static_storage *workspace = &p_workspace;

    //for( j = 0; j < system->N; ++j ) {
    type_j = atoms[j].type;
    Delta_j = workspace->Delta_boc[j];
    start_j = Start_Index(j, bonds);
    end_j = End_Index(j, bonds);

    /*
       sh_tor = _tor;
       sh_con = sh_tor + blockDim.x;
       sh_press = (rvec *) (sh_tor + 2*blockDim.x);

       sh_tor[threadIdx.x] = 0;
       sh_con [threadIdx.x] = 0;
       rvec_MakeZero (sh_press [threadIdx.x] );
       pk = threadIdx.x + start_j;
     */

    E_Tor [j] = 0;
    E_Con [j] = 0;
    rvec_MakeZero (aux_ext_press [j]);


    for( pk = start_j; pk < end_j; ++pk ) 
        //while (pk < end_j)
    {
        pbond_jk = &( bonds->select.bond_list[pk] );
        k = pbond_jk->nbr;
        bo_jk = &( pbond_jk->bo_data );
        BOA_jk = bo_jk->BO - control->thb_cut;

        /* see if there are any 3-body interactions involving j&k
           where j is the central atom. Otherwise there is no point in
           trying to form a 4-body interaction out of this neighborhood */    
        if( j < k && bo_jk->BO > control->thb_cut/*0*/ && 
                Num_Entries(pk, thb_intrs) ) {
            start_k = Start_Index(k, bonds);
            end_k = End_Index(k, bonds);                   
            pj = pbond_jk->sym_index; // pj points to j on k's list

            /* do the same check as above: are there any 3-body interactions 
               involving k&j where k is the central atom */
            if( Num_Entries(pj, thb_intrs) ) {
                type_k = atoms[k].type;
                Delta_k = workspace->Delta_boc[k];
                r_jk = pbond_jk->d;

                start_pk = Start_Index(pk, thb_intrs );
                end_pk = End_Index(pk, thb_intrs );
                start_pj = Start_Index(pj, thb_intrs );
                end_pj = End_Index(pj, thb_intrs );        

                exp_tor2_jk = EXP( -p_tor2 * BOA_jk );
                exp_cot2_jk = EXP( -p_cot2 * SQR(BOA_jk - 1.5) );
                exp_tor3_DjDk = EXP( -p_tor3 * (Delta_j + Delta_k) );
                exp_tor4_DjDk = EXP( p_tor4  * (Delta_j + Delta_k) );
                exp_tor34_inv = 1.0 / (1.0 + exp_tor3_DjDk + exp_tor4_DjDk);
                f11_DjDk = (2.0 + exp_tor3_DjDk) * exp_tor34_inv;


                /* pick i up from j-k interaction where j is the centre atom */
                for( pi = start_pk; pi < end_pk; ++pi ) {
                    p_ijk = &( thb_intrs->select.three_body_list[pi] );
                    pij = p_ijk->pthb; // pij is pointer to i on j's bond_list
                    pbond_ij = &( bonds->select.bond_list[pij] );
                    bo_ij = &( pbond_ij->bo_data );


                    if( bo_ij->BO > control->thb_cut/*0*/ ) {
                        i = p_ijk->thb;
                        type_i = atoms[i].type;
                        r_ij = pbond_ij->d;
                        BOA_ij = bo_ij->BO - control->thb_cut;

                        theta_ijk = p_ijk->theta;
                        sin_ijk = SIN( theta_ijk );
                        cos_ijk = COS( theta_ijk );
                        //tan_ijk_i = 1. / TAN( theta_ijk );
                        if( sin_ijk >= 0 && sin_ijk <= MIN_SINE ) 
                            tan_ijk_i = cos_ijk / MIN_SINE;
                        else if( sin_ijk <= 0 && sin_ijk >= -MIN_SINE ) 
                            tan_ijk_i = cos_ijk / -MIN_SINE;
                        else tan_ijk_i = cos_ijk / sin_ijk;

                        exp_tor2_ij = EXP( -p_tor2 * BOA_ij );
                        exp_cot2_ij = EXP( -p_cot2 * SQR(BOA_ij -1.5) );

                        /* pick l up from j-k intr. where k is the centre */
                        for( pl = start_pj; pl < end_pj; ++pl ) {
                            p_jkl = &( thb_intrs->select.three_body_list[pl] );
                            l = p_jkl->thb;
                            plk = p_jkl->pthb; //pointer to l on k's bond_list!
                            pbond_kl = &( bonds->select.bond_list[plk] );
                            bo_kl = &( pbond_kl->bo_data );
                            type_l = atoms[l].type;
                            fbh = &(d_fbp[ index_fbp(type_i,type_j,type_k,type_l,num_atom_types) ]);
                            fbp = &(d_fbp[ index_fbp(type_i,type_j,type_k,type_l,num_atom_types)].prm[0]);

                            if( i != l && fbh->cnt && bo_kl->BO > control->thb_cut/*0*/ &&
                                    bo_ij->BO * bo_jk->BO * bo_kl->BO > control->thb_cut/*0*/ ){
                                ++num_frb_intrs;
                                r_kl = pbond_kl->d;
                                BOA_kl = bo_kl->BO - control->thb_cut;

                                theta_jkl = p_jkl->theta;
                                sin_jkl = SIN( theta_jkl );
                                cos_jkl = COS( theta_jkl );
                                //tan_jkl_i = 1. / TAN( theta_jkl );
                                if( sin_jkl >= 0 && sin_jkl <= MIN_SINE ) 
                                    tan_jkl_i = cos_jkl / MIN_SINE;
                                else if( sin_jkl <= 0 && sin_jkl >= -MIN_SINE ) 
                                    tan_jkl_i = cos_jkl / -MIN_SINE;
                                else tan_jkl_i = cos_jkl /sin_jkl;

                                Sq_Distance_on_T3( atoms[l].x, atoms[i].x, 
                                        box, dvec_li );
                                r_li = rvec_Norm( dvec_li );


                                /* omega and its derivative */
                                //cos_omega=Calculate_Omega(pbond_ij->dvec,r_ij,pbond_jk->dvec, 
                                omega = Calculate_Omega(pbond_ij->dvec, r_ij, pbond_jk->dvec, 
                                        r_jk, pbond_kl->dvec, r_kl,
                                        dvec_li, r_li, p_ijk, p_jkl,
                                        dcos_omega_di, dcos_omega_dj,
                                        dcos_omega_dk, dcos_omega_dl,
                                        NULL); //TODO *check*
                                cos_omega = COS( omega );
                                cos2omega = COS( 2. * omega );
                                cos3omega = COS( 3. * omega );
                                /* end omega calculations */

                                /* torsion energy */
                                exp_tor1 = EXP(fbp->p_tor1 * SQR(2.-bo_jk->BO_pi-f11_DjDk));
                                exp_tor2_kl = EXP( -p_tor2 * BOA_kl );
                                exp_cot2_kl = EXP( -p_cot2 * SQR(BOA_kl-1.5) );
                                fn10 = (1.0 - exp_tor2_ij) * (1.0 - exp_tor2_jk) * 
                                    (1.0 - exp_tor2_kl);

                                CV = 0.5 * ( fbp->V1 * (1.0 + cos_omega) + 
                                        fbp->V2 * exp_tor1 * (1.0 - cos2omega) +
                                        fbp->V3 * (1.0 + cos3omega) );
                                //CV = 0.5 * fbp->V1 * (1.0 + cos_omega) + 
                                //  fbp->V2 * exp_tor1 * (1.0 - SQR(cos_omega)) +
                                //  fbp->V3 * (0.5 + 2.0*CUBE(cos_omega) - 1.5 * cos_omega);

                                //PERFORMANCE IMPACT
                                e_tor = fn10 * sin_ijk * sin_jkl * CV;
                                //MYATOMICADD(&data->E_Tor ,e_tor );
                                E_Tor [j] += e_tor;
                                //sh_tor [threadIdx.x] += e_tor;

                                dfn11 = (-p_tor3 * exp_tor3_DjDk +
                                        (p_tor3 * exp_tor3_DjDk - p_tor4 * exp_tor4_DjDk) *
                                        (2.+exp_tor3_DjDk) * exp_tor34_inv) * exp_tor34_inv;

                                CEtors1 = sin_ijk * sin_jkl * CV;

                                CEtors2 = -fn10 * 2.0 * fbp->p_tor1 * fbp->V2 * exp_tor1 * 
                                    (2.0 - bo_jk->BO_pi - f11_DjDk) * (1.0 - SQR(cos_omega)) * 
                                    sin_ijk * sin_jkl; 

                                CEtors3 = CEtors2 * dfn11;

                                CEtors4 = CEtors1 * p_tor2 * exp_tor2_ij * 
                                    (1.0 - exp_tor2_jk) * (1.0 - exp_tor2_kl);

                                CEtors5 = CEtors1 * p_tor2 * exp_tor2_jk * 
                                    (1.0 - exp_tor2_ij) * (1.0 - exp_tor2_kl);

                                CEtors6 = CEtors1 * p_tor2 * exp_tor2_kl *
                                    (1.0 - exp_tor2_ij) * (1.0 - exp_tor2_jk);

                                cmn = -fn10 * CV;
                                CEtors7 = cmn * sin_jkl * tan_ijk_i;
                                CEtors8 = cmn * sin_ijk * tan_jkl_i;
                                CEtors9 = fn10 * sin_ijk * sin_jkl * 
                                    (0.5 * fbp->V1 - 2.0 * fbp->V2 * exp_tor1 * cos_omega +
                                     1.5 * fbp->V3 * (cos2omega + 2. * SQR(cos_omega)));
                                //cmn = -fn10 * CV;
                                //CEtors7 = cmn * sin_jkl * cos_ijk;
                                //CEtors8 = cmn * sin_ijk * cos_jkl;
                                //CEtors9 = fn10 * sin_ijk * sin_jkl * 
                                //  (0.5 * fbp->V1 - 2.0 * fbp->V2 * exp_tor1 * cos_omega +
                                //   fbp->V3 * (6*SQR(cos_omega) - 1.50));
                                /* end  of torsion energy */


                                /* 4-body conjugation energy */
                                fn12 = exp_cot2_ij * exp_cot2_jk * exp_cot2_kl;
                                //PERFORMANCE IMPACT
                                e_con = fbp->p_cot1 * fn12 * (1. + (SQR(cos_omega)-1.) * sin_ijk*sin_jkl);
                                //MYATOMICADD(&data->E_Con ,e_con );
                                E_Con [j] += e_con ;
                                //sh_con [threadIdx.x] += e_con;

                                Cconj = -2.0 * fn12 * fbp->p_cot1 * p_cot2 * 
                                    (1. + (SQR(cos_omega)-1.) * sin_ijk*sin_jkl);

                                CEconj1 = Cconj * (BOA_ij - 1.5e0);
                                CEconj2 = Cconj * (BOA_jk - 1.5e0);
                                CEconj3 = Cconj * (BOA_kl - 1.5e0);

                                CEconj4 = -fbp->p_cot1 * fn12 * 
                                    (SQR(cos_omega) - 1.0) * sin_jkl * tan_ijk_i;
                                CEconj5 = -fbp->p_cot1 * fn12 * 
                                    (SQR(cos_omega) - 1.0) * sin_ijk * tan_jkl_i;
                                //CEconj4 = -fbp->p_cot1 * fn12 * 
                                //  (SQR(cos_omega) - 1.0) * sin_jkl * cos_ijk;
                                //CEconj5 = -fbp->p_cot1 * fn12 * 
                                //  (SQR(cos_omega) - 1.0) * sin_ijk * cos_jkl;
                                CEconj6 = 2.0 * fbp->p_cot1 * fn12 * 
                                    cos_omega * sin_ijk * sin_jkl;
                                /* end 4-body conjugation energy */

                                //fprintf(stdout, "%6d %6d %6d %6d %7.3f %7.3f %7.3f %7.3f ",
                                //   workspace->orig_id[i], workspace->orig_id[j],
                                //       workspace->orig_id[k], workspace->orig_id[l], 
                                //    omega, cos_omega, cos2omega, cos3omega );
                                //fprintf(stdout, 
                                //    "%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f\n",
                                //    CEtors2, CEtors3, CEtors4, CEtors5, 
                                //    CEtors6, CEtors7, CEtors8, CEtors9 );
                                //fprintf(stdout, "%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f\n",
                                //    theta_ijk, theta_jkl, sin_ijk, 
                                //    sin_jkl, cos_jkl, tan_jkl_i );

                                /* forces */
                                //PERFORMANCE IMPACT
                                /*
                                   MYATOMICADD( &bo_jk->Cdbopi, CEtors2 );
                                   MYATOMICADD( &workspace->CdDelta[j], CEtors3 );
                                   MYATOMICADD( &workspace->CdDelta[k], CEtors3 );
                                   MYATOMICADD( &bo_ij->Cdbo, (CEtors4 + CEconj1) );
                                   MYATOMICADD( &bo_jk->Cdbo, (CEtors5 + CEconj2) );
                                   MYATOMICADD( &bo_kl->Cdbo, (CEtors6 + CEconj3) );
                                 */

                                //PERFORMANCE IMPACT
                                bo_jk->Cdbopi += CEtors2;
                                workspace->CdDelta[j] += CEtors3;
                                pbond_jk->CdDelta_jk += CEtors3;
                                bo_ij->Cdbo += CEtors4 + CEconj1;
                                bo_jk->Cdbo += CEtors5 + CEconj2;

                                //TODO REMOVE THIS ATOMIC OPERATION IF POSSIBLE
                                MYATOMICADD(&pbond_kl->Cdbo_kl, CEtors6 + CEconj3 );
                                //TODO REMOVE THIS ATOMIC OPERATION IF POSSIBLE

                                if( control->ensemble == NVE || control->ensemble == NVT ||control->ensemble == bNVT) {
                                    /* dcos_theta_ijk */
                                    //PERFORMANCE IMPACT
                                    atomic_rvecScaledAdd( pbond_ij->i_f, CEtors7 + CEconj4, p_ijk->dcos_dk );
                                    rvec_ScaledAdd( atoms[j].f, CEtors7 + CEconj4, p_ijk->dcos_dj );
                                    atomic_rvecScaledAdd( pbond_jk->k_f, CEtors7 + CEconj4, p_ijk->dcos_di );


                                    /* dcos_theta_jkl */
                                    //PERFORMANCE IMPACT
                                    rvec_ScaledAdd( atoms[j].f, CEtors8 + CEconj5, p_jkl->dcos_di );
                                    atomic_rvecScaledAdd( pbond_jk->i_f, CEtors8 + CEconj5, p_jkl->dcos_dj );
                                    atomic_rvecScaledAdd( pbond_kl->k_f, CEtors8 + CEconj5, p_jkl->dcos_dk );

                                    /* dcos_omega */
                                    //PERFORMANCE IMPACT
                                    atomic_rvecScaledAdd( pbond_ij->i_f, CEtors9 + CEconj6, dcos_omega_di );
                                    rvec_ScaledAdd( atoms[j].f, CEtors9 + CEconj6, dcos_omega_dj );
                                    atomic_rvecScaledAdd( pbond_jk->i_f, CEtors9 + CEconj6, dcos_omega_dk );
                                    atomic_rvecScaledAdd( pbond_kl->k_f, CEtors9 + CEconj6, dcos_omega_dl );
                                }
                                else {
                                    ivec_Sum(rel_box_jl, pbond_jk->rel_box, pbond_kl->rel_box);

                                    /* dcos_theta_ijk */
                                    rvec_Scale( force, CEtors7 + CEconj4, p_ijk->dcos_dk );
                                    //PERFORMANCE IMPACT
                                    atomic_rvecAdd( pbond_ij->i_f, force );
                                    rvec_iMultiply( ext_press, pbond_ij->rel_box, force );
                                    rvec_Add( aux_ext_press [j], ext_press );
                                    //rvec_Add (sh_press [threadIdx.x], ext_press);

                                    //PERFORMANCE IMPACT
                                    rvec_ScaledAdd( atoms[j].f, CEtors7 + CEconj4, p_ijk->dcos_dj );

                                    rvec_Scale( force, CEtors7 + CEconj4, p_ijk->dcos_di );
                                    //PERFORMANCE IMPACT
                                    atomic_rvecAdd( pbond_jk->i_f, force );
                                    rvec_iMultiply( ext_press, pbond_jk->rel_box, force );
                                    //PERFORMANCE IMPACT
                                    rvec_Add( aux_ext_press [j], ext_press );
                                    //rvec_Add (sh_press [threadIdx.x], ext_press);


                                    /* dcos_theta_jkl */
                                    //PERFORMANCE IMPACT
                                    rvec_ScaledAdd( atoms[j].f, CEtors8 + CEconj5, p_jkl->dcos_di );

                                    rvec_Scale( force, CEtors8 + CEconj5, p_jkl->dcos_dj );
                                    //PERFORMANCE IMPACT
                                    atomic_rvecAdd( pbond_jk->i_f, force );
                                    rvec_iMultiply( ext_press, pbond_jk->rel_box, force );
                                    rvec_Add( aux_ext_press [j], ext_press );
                                    //rvec_Add (sh_press [threadIdx.x], ext_press);

                                    rvec_Scale( force, CEtors8 + CEconj5, p_jkl->dcos_dk );
                                    //PERFORMANCE IMPACT
                                    atomic_rvecAdd( pbond_kl->k_f, force );
                                    rvec_iMultiply( ext_press, rel_box_jl, force );
                                    rvec_Add( aux_ext_press [j], ext_press );
                                    //rvec_Add (sh_press [threadIdx.x], ext_press);


                                    /* dcos_omega */                      
                                    rvec_Scale( force, CEtors9 + CEconj6, dcos_omega_di );
                                    //PERFORMANCE IMPACT
                                    atomic_rvecAdd( pbond_ij->i_f, force );
                                    rvec_iMultiply( ext_press, pbond_ij->rel_box, force );
                                    rvec_Add( aux_ext_press [j], ext_press );
                                    //rvec_Add (sh_press [threadIdx.x], ext_press);

                                    //PERFORMANCE IMPACT
                                    rvec_ScaledAdd( atoms[j].f, 
                                            CEtors9 + CEconj6, dcos_omega_dj );

                                    rvec_Scale( force, CEtors9 + CEconj6, dcos_omega_dk );
                                    //PERFORMANCE IMPACT
                                    atomic_rvecAdd( pbond_jk->i_f, force );
                                    rvec_iMultiply( ext_press, pbond_jk->rel_box, force );
                                    rvec_Add( aux_ext_press [j], ext_press );
                                    //rvec_Add (sh_press [threadIdx.x], ext_press);

                                    rvec_Scale( force, CEtors9 + CEconj6, dcos_omega_dl );
                                    //PERFORMANCE IMPACT
                                    atomic_rvecAdd( pbond_kl->k_f, force );
                                    rvec_iMultiply( ext_press, rel_box_jl, force );
                                    rvec_Add( aux_ext_press [j], ext_press );
                                    //rvec_Add (sh_press [threadIdx.x], ext_press);


                                    /* This part is intended for a fully-flexible box */
                                    /* rvec_ScaledSum( temp_rvec, 
                                       CEtors7 + CEconj4, p_ijk->dcos_dk,      // i     
                                       CEtors9 + CEconj6, dcos_omega_di );
                                       rvec_OuterProduct( temp_rtensor, 
                                       temp_rvec, system->atoms[i].x );
                                       rtensor_Copy( total_rtensor, temp_rtensor );

                                       rvec_ScaledSum( temp_rvec, 
                                       CEtors7 + CEconj4, p_ijk->dcos_dj,      // j
                                       CEtors8 + CEconj5, p_jkl->dcos_di );
                                       rvec_ScaledAdd( temp_rvec, 
                                       CEtors9 + CEconj6, dcos_omega_dj );
                                       rvec_OuterProduct( temp_rtensor, 
                                       temp_rvec, system->atoms[j].x );
                                       rtensor_Add( total_rtensor, temp_rtensor );

                                       rvec_ScaledSum( temp_rvec, 
                                       CEtors7 + CEconj4, p_ijk->dcos_di,      // k
                                       CEtors8 + CEconj5, p_jkl->dcos_dj );
                                       rvec_ScaledAdd( temp_rvec, 
                                       CEtors9 + CEconj6, dcos_omega_dk );
                                       rvec_OuterProduct( temp_rtensor, 
                                       temp_rvec, system->atoms[k].x );
                                       rtensor_Add( total_rtensor, temp_rtensor );

                                       rvec_ScaledSum( temp_rvec, 
                                       CEtors8 + CEconj5, p_jkl->dcos_dk,      // l
                                       CEtors9 + CEconj6, dcos_omega_dl );
                                       rvec_OuterProduct( temp_rtensor, 
                                       temp_rvec, system->atoms[l].x );
                                       rtensor_Copy( total_rtensor, temp_rtensor );

                                       if( pbond_ij->imaginary || pbond_jk->imaginary || 
                                       pbond_kl->imaginary )
                                       rtensor_ScaledAdd( data->flex_bar.P, -1., total_rtensor );
                                       else
                                       rtensor_Add( data->flex_bar.P, total_rtensor ); */
                                }

#ifdef TEST_ENERGY
                                /*fprintf( out_control->etor, 
                                //"%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f\n",
                                //r_ij, r_jk, r_kl, 
                                "%12.8f%12.8f%12.8f%12.8f\n",
                                cos_ijk, cos_jkl, sin_ijk, sin_jkl );*/
                                // fprintf( out_control->etor, "%12.8f\n", dfn11 );
                                /*
                                   fprintf( out_control->etor, "%12.8f%12.8f%12.8f\n", 
                                   fn10, cos_omega, CV );

                                   fprintf( out_control->etor, 
                                   "%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f\n",
                                   CEtors2, CEtors3, CEtors4, CEtors5, 
                                   CEtors6, CEtors7, CEtors8, CEtors9 );
                                 */
                                //end

                                /* fprintf( out_control->etor, 
                                   "%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f\n",
                                   htra, htrb, htrc, hthd, hthe, hnra, hnrc, hnhd, hnhe ); */

                                /*
                                   fprintf( out_control->etor, 
                                   "%12.8f%12.8f%12.8f%12.8f%12.8f%12.8f\n",
                                   CEconj1, CEconj2, CEconj3, CEconj4, CEconj5, CEconj6 );
                                 */
                                //end
                                /* fprintf(out_control->etor,"%23.15e%23.15e%23.15e%23.15e\n",
                                   fbp->V1, fbp->V2, fbp->V3, fbp->p_tor1 );*/

                                /*

                                   fprintf( out_control->etor, 
                                //"%6d%6d%6d%6d%23.15e%23.15e%23.15e%23.15e\n", 
                                "%6d%6d%6d%6d%12.8f%12.8f\n", 
                                workspace->orig_id[i], workspace->orig_id[j], 
                                workspace->orig_id[k], workspace->orig_id[l], 
                                e_tor, e_con );
                                //RAD2DEG(omega), BOA_jk, e_tor, data->E_Tor );

                                fprintf( out_control->econ, 
                                "%6d%6d%6d%6d%23.15e%23.15e%23.15e%23.15e%23.15e%23.15e\n", 
                                workspace->orig_id[i], workspace->orig_id[j], 
                                workspace->orig_id[k], workspace->orig_id[l], 
                                RAD2DEG(omega), BOA_ij, BOA_jk, BOA_kl, 
                                e_con,data->E_Con );
                                 */
                                //end

                                /* fprintf( out_control->etor, 
                                   "%12.8f%12.8f%12.8f\n%12.8f%12.8f%12.8f\n%12.8f%12.8f%12.8f\n",       
                                   (CEtors7 + CEconj4)*p_ijk->dcos_dk[0], 
                                   (CEtors7 + CEconj4)*p_ijk->dcos_dk[1], 
                                   (CEtors7 + CEconj4)*p_ijk->dcos_dk[2],
                                   (CEtors7 + CEconj4)*p_ijk->dcos_dj[0], 
                                   (CEtors7 + CEconj4)*p_ijk->dcos_dj[1], 
                                   (CEtors7 + CEconj4)*p_ijk->dcos_dj[2],
                                   (CEtors7 + CEconj4)*p_ijk->dcos_di[0], 
                                   (CEtors7 + CEconj4)*p_ijk->dcos_di[1], 
                                   (CEtors7 + CEconj4)*p_ijk->dcos_di[2] ); */


                                /* fprintf( out_control->etor, 
                                   "%12.8f%12.8f%12.8f\n%12.8f%12.8f%12.8f\n%12.8f%12.8f%12.8f\n",
                                   (CEtors8 + CEconj5)*p_jkl->dcos_di[0], 
                                   (CEtors8 + CEconj5)*p_jkl->dcos_di[1], 
                                   (CEtors8 + CEconj5)*p_jkl->dcos_di[2], 
                                   (CEtors8 + CEconj5)*p_jkl->dcos_dj[0], 
                                   (CEtors8 + CEconj5)*p_jkl->dcos_dj[1], 
                                   (CEtors8 + CEconj5)*p_jkl->dcos_dj[2], 
                                   (CEtors8 + CEconj5)*p_jkl->dcos_dk[0], 
                                   (CEtors8 + CEconj5)*p_jkl->dcos_dk[1], 
                                   (CEtors8 + CEconj5)*p_jkl->dcos_dk[2] ); */

                                /*
                                   fprintf( out_control->etor, 
                                   "%12.8f%12.8f%12.8f\n%12.8f%12.8f%12.8f\n%12.8f%12.8f%12.8f\n%12.8f%12.8f%12.8f\n",
                                   dcos_omega_di[0], dcos_omega_di[1], dcos_omega_di[2], 
                                   dcos_omega_dj[0], dcos_omega_dj[1], dcos_omega_dj[2], 
                                   dcos_omega_dk[0], dcos_omega_dk[1], dcos_omega_dk[2],
                                   dcos_omega_dl[0], dcos_omega_dl[1], dcos_omega_dl[2] );
                                 */
                                //end
#endif

#ifdef TEST_FORCES
                                /*
                                // Torsion Forces 
                                Add_dBOpinpi2(system, lists, j, pk, CEtors2, 0., 
                                workspace->f_tor, workspace->f_tor);
                                Add_dDelta( system, lists, j, CEtors3, workspace->f_tor );
                                Add_dDelta( system, lists, k, CEtors3, workspace->f_tor );
                                Add_dBO( system, lists, j, pij, CEtors4, workspace->f_tor );
                                Add_dBO( system, lists, j, pk, CEtors5, workspace->f_tor );
                                Add_dBO( system, lists, k, plk, CEtors6, workspace->f_tor );

                                rvec_ScaledAdd(workspace->f_tor[i], CEtors7, p_ijk->dcos_dk);
                                rvec_ScaledAdd(workspace->f_tor[j], CEtors7, p_ijk->dcos_dj);
                                rvec_ScaledAdd(workspace->f_tor[k], CEtors7, p_ijk->dcos_di);

                                rvec_ScaledAdd(workspace->f_tor[j], CEtors8, p_jkl->dcos_di);
                                rvec_ScaledAdd(workspace->f_tor[k], CEtors8, p_jkl->dcos_dj);
                                rvec_ScaledAdd(workspace->f_tor[l], CEtors8, p_jkl->dcos_dk);

                                rvec_ScaledAdd( workspace->f_tor[i], CEtors9, dcos_omega_di );
                                rvec_ScaledAdd( workspace->f_tor[j], CEtors9, dcos_omega_dj );
                                rvec_ScaledAdd( workspace->f_tor[k], CEtors9, dcos_omega_dk );
                                rvec_ScaledAdd( workspace->f_tor[l], CEtors9, dcos_omega_dl );

                                // Conjugation Forces 
                                Add_dBO( system, lists, j, pij, CEconj1, workspace->f_con );
                                Add_dBO( system, lists, j, pk, CEconj2, workspace->f_con );
                                Add_dBO( system, lists, k, plk, CEconj3, workspace->f_con );

                                rvec_ScaledAdd(workspace->f_con[i], CEconj4, p_ijk->dcos_dk);
                                rvec_ScaledAdd(workspace->f_con[j], CEconj4, p_ijk->dcos_dj);
                                rvec_ScaledAdd(workspace->f_con[k], CEconj4, p_ijk->dcos_di);

                                rvec_ScaledAdd(workspace->f_con[j], CEconj5, p_jkl->dcos_di);
                                rvec_ScaledAdd(workspace->f_con[k], CEconj5, p_jkl->dcos_dj);
                                rvec_ScaledAdd(workspace->f_con[l], CEconj5, p_jkl->dcos_dk);

                                rvec_ScaledAdd( workspace->f_con[i], CEconj6, dcos_omega_di );
                                rvec_ScaledAdd( workspace->f_con[j], CEconj6, dcos_omega_dj );
                                rvec_ScaledAdd( workspace->f_con[k], CEconj6, dcos_omega_dk );
                                rvec_ScaledAdd( workspace->f_con[l], CEconj6, dcos_omega_dl );
                                 */
                                //end
#endif
                            } // pl check ends
                        } // pl loop ends
                    } // pi check ends
                } // pi loop ends
            } // k-j neighbor check ends
        } // j<k && j-k neighbor check ends


        //pk += blockDim.x;



    } // pk loop ends
    //} // j loop -- REMOVED FOR CUDA

    /* fprintf( stderr, "4body: ext_press (%23.15e %23.15e %23.15e)\n", 
       data->ext_press[0], data->ext_press[1], data->ext_press[2] );*/

#ifdef TEST_FORCES
    /*
       fprintf( stderr, "Number of torsion angles: %d\n", num_frb_intrs );
       fprintf( stderr, "Torsion Energy: %g\t Conjugation Energy: %g\n", 
       data->E_Tor, data->E_Con );
     */
#endif

    /*
    //do the reduction for the shared memory variables
    // now do a reduce inside the warp for E_vdW, E_Ele and force.
    if (threadIdx.x < 16) {
    sh_tor [threadIdx.x] += sh_tor [threadIdx.x + 16];
    sh_con [threadIdx.x] += sh_con [threadIdx.x + 16];
    rvec_Add (sh_press [threadIdx.x], sh_press[threadIdx.x + 16]);
    }
    if (threadIdx.x < 8) {
    sh_tor [threadIdx.x] += sh_tor [threadIdx.x + 8];
    sh_con [threadIdx.x] += sh_con [threadIdx.x + 8];
    rvec_Add (sh_press [threadIdx.x], sh_press[threadIdx.x + 8]);
    }
    if (threadIdx.x < 4) {
    sh_tor [threadIdx.x] += sh_tor [threadIdx.x + 4];
    sh_con [threadIdx.x] += sh_con [threadIdx.x + 4];
    rvec_Add (sh_press [threadIdx.x], sh_press[threadIdx.x + 4]);
    }
    if (threadIdx.x < 2) {
    sh_tor [threadIdx.x] += sh_tor [threadIdx.x + 2];
    sh_con [threadIdx.x] += sh_con [threadIdx.x + 2];
    rvec_Add (sh_press [threadIdx.x], sh_press[threadIdx.x + 2]);
    }
    if (threadIdx.x < 1) {
    sh_tor [threadIdx.x] += sh_tor [threadIdx.x + 1];
    sh_con [threadIdx.x] += sh_con [threadIdx.x + 1];
    rvec_Add (sh_press [threadIdx.x], sh_press[threadIdx.x + 1]);
    }

    if (threadIdx.x == 0) {
    E_Tor[j] = sh_tor [threadIdx.x];
    E_Con[j] = sh_con [threadIdx.x];
    rvec_Copy (aux_ext_press[j], sh_press[threadIdx.x]);
    }
     */

}


GLOBAL void k_Four_Body_Postprocess( reax_atom *atoms, 
        static_storage p_workspace, 
        list p_bonds, int N )
{
    int i, pj;

    bond_data *pbond;
    bond_data *sym_index_bond;
    bond_order_data *bo_data;

    list *bonds = &p_bonds;
    static_storage *workspace = &p_workspace;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N) return;

    for( pj = Start_Index(i, bonds); pj < End_Index(i, bonds); ++pj ){

        pbond = &(bonds->select.bond_list[pj]);
        bo_data = &pbond->bo_data;
        sym_index_bond = &( bonds->select.bond_list[ pbond->sym_index ] );

        workspace->CdDelta [i] += sym_index_bond->CdDelta_jk;

        //bo_data->Cdbo += sym_index_bond->Cdbo_kl;
        bo_data->Cdbo += pbond->Cdbo_kl;

        //update f vector
        rvec_Add (atoms [i].f, sym_index_bond->i_f ); 
        rvec_Add (atoms [i].f, sym_index_bond->k_f );
    }
}

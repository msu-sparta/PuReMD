/*----------------------------------------------------------------------
  PuReMD-GPU - Reax Force Field Simulator

  Copyright (2014) Purdue University
  Sudhir Kylasa, skylasa@purdue.edu
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_environment.h"

#include "cuda_utils.h"


void Setup_Cuda_Environment( int rank, int nprocs, int gpus_per_node )
{

    int deviceCount = 0;
    hipError_t flag;
    
    flag = hipGetDeviceCount( &deviceCount );

    if ( flag != hipSuccess || deviceCount < 1 )
    {
        fprintf( stderr, "ERROR: no CUDA capable device(s) found. Terminating...\n" );
        exit( 1 );
    }

    //Calculate the # of GPUs per processor
    //and assign the GPU for each process
    //TODO: handle condition where # CPU procs > # GPUs
    hipSetDevice( rank % deviceCount );
    cudaCheckError( );

    /* reset device and clear previous allocations */
    Cleanup_Cuda_Environment( );    
    cudaCheckError( );

#if defined(__CUDA_DEBUG__)
    fprintf( stderr, "p:%d is using GPU: %d \n", rank, rank % deviceCount );
#endif

    //CHANGE ORIGINAL
    //hipDeviceSetLimit( hipLimitStackSize, 8192 );
    //hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
    //cudaCheckError( );

    cublasCheckError( hipblasCreate(&cublasHandle) );

    cusparseCheckError( hipsparseCreate(&cusparseHandle) );
    cusparseCheckError( hipsparseCreateMatDescr(&matdescriptor) );
    hipsparseSetMatType( matdescriptor, HIPSPARSE_MATRIX_TYPE_GENERAL );
    hipsparseSetMatIndexBase( matdescriptor, HIPSPARSE_INDEX_BASE_ZERO );

}


void Cleanup_Cuda_Environment( )
{
    hipDeviceReset( );
    hipDeviceSynchronize( );
}

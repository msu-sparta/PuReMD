/*----------------------------------------------------------------------
  PuReMD-GPU - Reax Force Field Simulator

  Copyright (2014) Purdue University
  Sudhir Kylasa, skylasa@purdue.edu
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_lookup.h"

#include "index_utils.h"

#include "cuda_utils.h"
#include "cuda_two_body_interactions.h"


/* Fills solution into x. Warning: will modify c and d! */
DEVICE void Tridiagonal_Solve( const real *a, const real *b,
        real *c, real *d, real *x, unsigned int n){
    int i;
    real id;

    /* Modify the coefficients. */
    c[0] /= b[0];    /* Division by zero risk. */
    d[0] /= b[0];    /* Division by zero would imply a singular matrix. */
    for(i = 1; i < n; i++){
        id = (b[i] - c[i-1] * a[i]);  /* Division by zero risk. */
        c[i] /= id;            /* Last value calculated is redundant. */
        d[i] = (d[i] - d[i-1] * a[i])/id;
    }

    /* Now back substitute. */
    x[n - 1] = d[n - 1];
    for(i = n - 2; i >= 0; i--)
        x[i] = d[i] - c[i] * x[i + 1];
}


GLOBAL void Cuda_Tridiagonal_Solve (const real *a, const real *b, 
        real *c, real *d, real *x, unsigned int n)
{
    Tridiagonal_Solve ( a, b, c, d, x, n );
}


GLOBAL void cubic_spline_init_a ( real *a, const real *h, int n )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;

    if (i == 0 || i == 1 || i == (n-1)) {
        a[i] = 0;
    } else {
        a[i] = h[i-1];
    }
}


GLOBAL void cubic_spline_init_b (real *b, const real *h, int n )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;

    if (i == 0 || i == (n-1)) {
        b[i] = 0;
    } else {
        b[i] = 2 * (h[i-1] + h[i]);
    }
}


GLOBAL void cubic_spline_init_c (real *c, const real *h, int n )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;

    if (i == 0 || i == (n-1) || i == (n-2)) {
        c[i] = 0;
    } else {
        c[i] = h[i];
    }
}


GLOBAL void cubic_spline_init_d (real *d, const real *f, const real *h, int n )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;

    if ( i == 0 || i == (n-1) ) {
        d[i] = 0;
    } else {
        d[i] = 6 * ((f[i+1]-f[i])/h[i] - (f[i]-f[i-1])/h[i-1]);
    }
}


GLOBAL void calculate_cubic_spline_coef ( const real *f, real *v, const real *h, LR_lookup_table *data, int offset, int n )
{
    cubic_spline_coef *coef;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n || i == 0) return;

    if (offset == SPLINE_H_OFFSET)
        coef = &data->H[1];
    else if(offset == SPLINE_CEVD_OFFSET)
        coef = &data->CEvd[1];
    else if (offset == SPLINE_CECLMB_OFFSET)
        coef = &data->CEclmb[1];
    else if (offset == SPLINE_VDW_OFFSET)
        coef = &data->vdW[1];
    else if (offset == SPLINE_ELE_OFFSET)
        coef = &data->ele[1];
    else
        coef = 0;

    coef[i-1].d = (v[i] - v[i-1]) / (6*h[i-1]);
    coef[i-1].c = v[i]/2;
    coef[i-1].b = (f[i]-f[i-1])/h[i-1] + h[i-1]*(2*v[i] + v[i-1])/6;
    coef[i-1].a = f[i];
}


void Cuda_Natural_Cubic_Spline( const real *h, const real *f, 
        LR_lookup_table *data, int offset, unsigned int n )
{
    int i;
    real *a, *b, *c, *d, *v;
    int blocks, block_size;

    ////fprintf (stderr, "Entering Cuda_Natural_Cubic_Spline ... \n");

    /* allocate space for the linear system */
    cuda_malloc ((void **) &a, REAL_SIZE * n, 0, __LINE__ );
    cuda_malloc ((void **) &b, REAL_SIZE * n, 0, __LINE__ );
    cuda_malloc ((void **) &c, REAL_SIZE * n, 0, __LINE__ );
    cuda_malloc ((void **) &d, REAL_SIZE * n, 0, __LINE__ );
    cuda_malloc ((void **) &v, REAL_SIZE * n, 1, __LINE__ );

    ////fprintf (stderr, "Mem allocation done... \n");

    /* build linear system */
    compute_blocks ( &blocks, &block_size, n);
    cubic_spline_init_a <<< blocks, block_size >>>
        ( a, h, n );
    hipDeviceSynchronize ();
    ////fprintf (stderr, "cubic_spline_init_a done.... -> %d \n", hipGetLastError ());

    cubic_spline_init_b <<< blocks, block_size >>>
        ( b, h, n );
    hipDeviceSynchronize ();
    ////fprintf (stderr, "cubic_spline_init_b done.... -> %d \n", hipGetLastError ());

    cubic_spline_init_c <<< blocks, block_size >>>
        ( c, h, n );
    hipDeviceSynchronize ();
    //fprintf (stderr, "cubic_spline_init_c done.... -> %d \n", hipGetLastError ());

    cubic_spline_init_d <<< blocks, block_size >>>
        ( d, f, h, n );
    hipDeviceSynchronize ();
    //fprintf (stderr, "cubic_spline_init_d done.... -> %d \n", hipGetLastError ());

    /*//fprintf( stderr, "i  a        b        c        d\n" );
      for( i = 0; i < n; ++i )
    //fprintf( stderr, "%d  %f  %f  %f  %f\n", i, a[i], b[i], c[i], d[i] );*/

    Cuda_Tridiagonal_Solve <<<1, 1>>>
        ( &(a[1]), &(b[1]), &(c[1]), &(d[1]), &(v[1]), n-2 );
    hipDeviceSynchronize ();
    //fprintf (stderr, "Tridiagonal_Solve done.... -> %d \n", hipGetLastError ());

    calculate_cubic_spline_coef <<< blocks, block_size >>>
        ( f, v, h, data,offset, n );
    hipDeviceSynchronize ();
    //fprintf (stderr, "calculate_cubic_spline_coef done.... -> %d \n", hipGetLastError ());

    /*//fprintf( stderr, "i  v  coef\n" );
      for( i = 0; i < n; ++i )
    //fprintf( stderr, "%d  %f  %f  %f  %f  %f\n", 
    i, v[i], coef[i].a, coef[i].b, coef[i].c, coef[i].d ); */
}


GLOBAL void complete_cubic_spline_init_a (real *a, const real *h, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;

    if (i == 0) a[0] = 0;
    else {
        a[i] = h[i];
    }
}


GLOBAL void complete_cubic_spline_init_b (real *b, const real *h, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;

    if (i == 0) b[0] = 2 * h[0];
    else {
        b[i] = 2 * (h[i-1] + h[i]); 
    }
}


GLOBAL void complete_cubic_spline_init_c (real *c, const real *h, int n )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;

    if (i == (n-1)) c[n-1] = 0;
    else {
        c[i] = h[i];
    }
}


GLOBAL void complete_cubic_spline_init_d (real *d, const real *f, const real *h, int v0_r, int vlast_r, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    real v0, vlast;
    if ( i >= n ) return;

    v0 = 0;
    vlast = 0;

    if (i == 0) {
        d[0] = 6 * (f[1]-f[0])/h[0] - 6 * v0;   
    }
    else if (i == (n-1)) {
        d[n-1] = 6 * vlast - 6 * (f[n-1]-f[n-2]/h[n-2]);
    }
    else
        d[i] = 6 * ((f[i+1]-f[i])/h[i] - (f[i]-f[i-1])/h[i-1]);
}


GLOBAL void calculate_complete_cubic_spline_coef (LR_lookup_table *data, int offset, real *v, const real *h, const real *f, int n)
{

    cubic_spline_coef *coef;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n ) return;

    if (offset == SPLINE_H_OFFSET)
        coef = &data->H[1];
    else if(offset == SPLINE_CEVD_OFFSET)
        coef = &data->CEvd[1];
    else if (offset == SPLINE_CECLMB_OFFSET)
        coef = &data->CEclmb[1];
    else if (offset == SPLINE_VDW_OFFSET)
        coef = &data->vdW[1];
    else if (offset == SPLINE_ELE_OFFSET)
        coef = &data->ele[1];
    else
        coef = 0;

    coef[i-1].d = (v[i] - v[i-1]) / (6*h[i-1]);
    coef[i-1].c = v[i]/2;
    coef[i-1].b = (f[i]-f[i-1])/h[i-1] + h[i-1]*(2*v[i] + v[i-1])/6;
    coef[i-1].a = f[i];
}


void Cuda_Complete_Cubic_Spline( const real *h, const real *f, int v0_r, int vlast_r,
        LR_lookup_table *data, int offset, unsigned int n )
{
    int i;
    real *a, *b, *c, *d, *v;

    int blocks, block_size;

    /* allocate space for the linear system */
    cuda_malloc ((void **) &a, REAL_SIZE * n, 0, __LINE__ );
    cuda_malloc ((void **) &b, REAL_SIZE * n, 0, __LINE__ );
    cuda_malloc ((void **) &c, REAL_SIZE * n, 0, __LINE__ );
    cuda_malloc ((void **) &d, REAL_SIZE * n, 0, __LINE__ );
    cuda_malloc ((void **) &v, REAL_SIZE * n, 1, __LINE__ );

    /* build the linear system */
    compute_blocks ( &blocks, &block_size, n );

    complete_cubic_spline_init_a <<< blocks, block_size >>>
        (a, h, n);
    hipDeviceSynchronize ();
    //fprintf (stderr, "complete_cubic_spline_init_a done.... -> %d \n", hipGetLastError ());

    complete_cubic_spline_init_b <<< blocks, block_size >>>
        (b, h, n);
    hipDeviceSynchronize ();
    //fprintf (stderr, "complete_cubic_spline_init_b done.... -> %d \n", hipGetLastError ());

    complete_cubic_spline_init_c <<< blocks, block_size >>>
        ( c, h, n );
    hipDeviceSynchronize ();
    //fprintf (stderr, "complete_cubic_spline_init_c done.... -> %d \n", hipGetLastError ());

    complete_cubic_spline_init_d <<< blocks, block_size >>>
        (d, f, h, v0_r, vlast_r, n);
    hipDeviceSynchronize ();
    //fprintf (stderr, "complete_cubic_spline_init_d done.... -> %d \n", hipGetLastError ());

    /*//fprintf( stderr, "i  a        b        c        d\n" );
      for( i = 0; i < n; ++i )
    //fprintf( stderr, "%d  %f  %f  %f  %f\n", i, a[i], b[i], c[i], d[i] );*/


    Cuda_Tridiagonal_Solve <<< 1, 1 >>>
        ( &(a[0]), &(b[0]), &(c[0]), &(d[0]), &(v[0]), n );
    hipDeviceSynchronize ();
    //fprintf (stderr, "Tridiagonal_Solve done.... -> %d \n", hipGetLastError ());
    // Tridiagonal_Solve( &(a[1]), &(b[1]), &(c[1]), &(d[1]), &(v[1]), n-2 );


    calculate_complete_cubic_spline_coef <<< blocks, block_size >>>
        (data, offset, v, h, f, n);
    hipDeviceSynchronize ();
    //fprintf (stderr, " calculate_complete_cubic_spline_coef done.... -> %d \n", hipGetLastError ());

    /*//fprintf( stderr, "i  v  coef\n" );
      for( i = 0; i < n; ++i )
    //fprintf( stderr, "%d  %f  %f  %f  %f  %f\n", 
    i, v[i], coef[i].a, coef[i].b, coef[i].c, coef[i].d ); */
}


void copy_LR_table_to_device (reax_system *system, control_params *control)
{
    int i, j, r;
    int num_atom_types;
    int existing_types[MAX_ATOM_TYPES];
    LR_data *d_y;
    cubic_spline_coef *temp;

    num_atom_types = system->reaxprm.num_atom_types;

    //fprintf (stderr, "Copying the LR Lookyp Table to the device ... \n");

    cuda_malloc ((void **) &d_LR, LR_LOOKUP_TABLE_SIZE * ( num_atom_types * num_atom_types ), 0, RES_LR_LOOKUP_TABLE );

    for( i = 0; i < MAX_ATOM_TYPES; ++i )
        existing_types[i] = 0;

    for( i = 0; i < system->N; ++i )
        existing_types[ system->atoms[i].type ] = 1;

    copy_host_device ( LR, d_LR, LR_LOOKUP_TABLE_SIZE * (num_atom_types * num_atom_types), hipMemcpyHostToDevice, RES_LR_LOOKUP_TABLE );

    for( i = 0; i < num_atom_types; ++i )
        if( existing_types[i] )
            for( j = i; j < num_atom_types; ++j )

                if( existing_types[j] ) {

                    cuda_malloc ((void **) &d_y, LR_DATA_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_Y );
                    copy_host_device ( LR [ index_lr (i, j, num_atom_types) ].y, d_y, LR_DATA_SIZE * (control->tabulate + 1), hipMemcpyHostToDevice, RES_LR_LOOKUP_Y );
                    copy_host_device ( &d_y, &d_LR [ index_lr (i, j, num_atom_types) ].y, LR_DATA_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_Y );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_H );
                    copy_host_device ( LR [ index_lr (i, j, num_atom_types) ].H, temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), hipMemcpyHostToDevice, RES_LR_LOOKUP_H );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].H, CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_H );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_VDW );
                    copy_host_device ( LR [ index_lr (i, j, num_atom_types) ].vdW, temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), hipMemcpyHostToDevice, RES_LR_LOOKUP_VDW );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].vdW,CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_VDW );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_CEVD );
                    copy_host_device ( LR [ index_lr (i, j, num_atom_types) ].CEvd, temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), hipMemcpyHostToDevice, RES_LR_LOOKUP_CEVD );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].CEvd, CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_CEVD );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_ELE );
                    copy_host_device ( LR [ index_lr (i, j, num_atom_types) ].ele, temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), hipMemcpyHostToDevice, RES_LR_LOOKUP_ELE );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].ele, CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_ELE );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_CECLMB );
                    copy_host_device ( LR [ index_lr (i, j, num_atom_types) ].CEclmb, temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), hipMemcpyHostToDevice, RES_LR_LOOKUP_CECLMB );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].CEclmb, CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_CECLMB );
                }

    //fprintf (stderr, "Copy of the LR Lookup Table to the device complete ... \n");
}


GLOBAL void calculate_LR_Values ( LR_lookup_table *d_LR, real *h, real *fh, real *fvdw, real *fCEvd, real *fele, real *fCEclmb, 
        global_parameters g_params, two_body_parameters *tbp, 
        control_params *control, int i, 
        int j, int num_atom_types, LR_data *data, real dr, int count )
{
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    if ( r == 0 || r > count ) return;

    d_LR_vdW_Coulomb( g_params, tbp, control, i, j, r * dr, &data[r], num_atom_types );

    h[r] = d_LR[ index_lr(i, j, num_atom_types) ].dx;
    fh[r] = d_LR[ index_lr(i, j, num_atom_types) ].y[r].H;
    fvdw[r] = d_LR[ index_lr(i, j, num_atom_types) ].y[r].e_vdW;
    fCEvd[r] = d_LR[ index_lr(i, j, num_atom_types) ].y[r].CEvd;
    fele[r] = d_LR[ index_lr(i, j, num_atom_types) ].y[r].e_ele;
    fCEclmb[r] = d_LR[ index_lr(i, j, num_atom_types) ].y[r].CEclmb;
}


GLOBAL void init_LR_values( LR_lookup_table *d_LR, control_params *control, real dr, int i, int j, int num_atom_types )
{
    d_LR[ index_lr (i, j, num_atom_types) ].xmin = 0;
    d_LR[ index_lr (i, j, num_atom_types) ].xmax = control->r_cut;
    d_LR[ index_lr (i, j, num_atom_types) ].n = control->tabulate + 1;
    d_LR[ index_lr (i, j, num_atom_types) ].dx = dr;
    d_LR[ index_lr (i, j, num_atom_types) ].inv_dx = control->tabulate / control->r_cut;
}


void Cuda_Make_LR_Lookup_Table( reax_system *system, control_params *control )
{
    int i, j, r;
    int num_atom_types;
    int existing_types[MAX_ATOM_TYPES];
    real dr;
    real *h, *fh, *fvdw, *fele, *fCEvd, *fCEclmb;

    int v0_vdw_r, v0_ele_r, vlast_vdw_r, vlast_ele_r;

    void *temp;
    LR_data *d_y;
    int blocks, block_size;

    /* initializations */
    vlast_ele_r = 0;
    vlast_vdw_r = 0;
    v0_ele_r = 0;
    v0_vdw_r = 0;

    num_atom_types = system->reaxprm.num_atom_types;
    dr = control->r_cut / control->tabulate;

    cuda_malloc ((void **) &h,             REAL_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_Y);
    cuda_malloc ((void **) &fh,         REAL_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_H);
    cuda_malloc ((void **) &fvdw,         REAL_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_VDW);
    cuda_malloc ((void **) &fCEvd,     REAL_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_CEVD);
    cuda_malloc ((void **) &fele,         REAL_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_ELE);
    cuda_malloc ((void **) &fCEclmb,     REAL_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_CECLMB);

    /* allocate Long-Range LookUp Table space based on 
       number of atom types in the ffield file */
    cuda_malloc ((void **) &d_LR, LR_LOOKUP_TABLE_SIZE * ( num_atom_types * num_atom_types ), 0, RES_LR_LOOKUP_TABLE );

    /* most atom types in ffield file will not exist in the current
       simulation. to avoid unnecessary lookup table space, determine
       the atom types that exist in the current simulation */
    for( i = 0; i < MAX_ATOM_TYPES; ++i )
        existing_types[i] = 0;

    for( i = 0; i < system->N; ++i )
        existing_types[ system->atoms[i].type ] = 1;

    /* fill in the lookup table entries for existing atom types.
       only lower half should be enough. */
    for( i = 0; i < num_atom_types; ++i )
        if( existing_types[i] )
            for( j = i; j < num_atom_types; ++j )
                if( existing_types[j] ) {

                    init_LR_values <<< 1, 1 >>>
                        ( d_LR, (control_params *)control->d_control, dr, i, j, num_atom_types );
                    hipDeviceSynchronize ();
                    //fprintf (stderr, "Done with init LR Values --> %d \n", hipGetLastError ());

                    cuda_malloc ((void **) &d_y, LR_DATA_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_Y );
                    copy_host_device ( &d_y, &d_LR [ index_lr (i, j, num_atom_types) ].y, LR_DATA_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_Y );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_H );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].H, CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_H );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_VDW );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].vdW,CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_VDW );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_CEVD );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].CEvd, CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_CEVD );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_ELE );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].ele, CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_ELE );

                    cuda_malloc ((void **) &temp, CUBIC_SPLINE_COEF_SIZE * (control->tabulate + 1), 0, RES_LR_LOOKUP_CECLMB );
                    copy_host_device ( &temp, &d_LR [ index_lr (i, j, num_atom_types) ].CEclmb, CUBIC_SPLINE_COEF_PTR_SIZE, hipMemcpyHostToDevice, RES_LR_LOOKUP_CECLMB );

                    //TODO check the bounds
                    compute_blocks ( &blocks, &block_size, control->tabulate );
                    calculate_LR_Values <<<blocks, block_size>>>
                        ( d_LR, h, fh, fvdw, fCEvd, fele, fCEclmb, 
                          system->reaxprm.d_gp, system->reaxprm.d_tbp, 
                          (control_params *)control->d_control, i, j, system->reaxprm.num_atom_types, 
                          d_y, dr, control->tabulate );
                    hipDeviceSynchronize ();

                    //fprintf (stderr, "Done with LR Values Calculation --> %d \n", hipGetLastError ());

                    /*//fprintf( stderr, "%-6s  %-6s  %-6s\n", "r", "h", "fh" );
                      for( r = 1; r <= control->tabulate; ++r )
                    //fprintf( stderr, "%f  %f  %f\n", r * dr, h[r], fh[r] ); */
                    Cuda_Natural_Cubic_Spline( h+1, fh+1, 
                            d_LR + index_lr (i,j,num_atom_types), SPLINE_H_OFFSET, control->tabulate+1 );

                    /*//fprintf( stderr, "%-6s  %-6s  %-6s\n", "r", "h", "fvdw" );
                      for( r = 1; r <= control->tabulate; ++r )
                    //fprintf( stderr, "%f  %f  %f\n", r * dr, h[r], fvdw[r] );
                    //fprintf( stderr, "v0_vdw: %f, vlast_vdw: %f\n", v0_vdw, vlast_vdw );
                     */

                    //TODO -- Pass the right v0 and vlast for the cubic spline
                    //Cuda_Complete_Cubic_Spline( &h[1], &fvdw[1], v0_vdw_r, vlast_vdw_r, 
                    //         &(LR[ index_lr (i,j,num_atom_types) ].vdW[1]), control->tabulate+1 );
                    //Cuda_Natural_Cubic_Spline( &h[1], &fCEvd[1], 
                    //        &(LR[ index_lr (i,j,num_atom_types) ].CEvd[1]), control->tabulate+1 );
                    Cuda_Complete_Cubic_Spline( &h[1], &fvdw[1], v0_vdw_r, vlast_vdw_r, 
                            d_LR + index_lr (i,j,num_atom_types) , SPLINE_VDW_OFFSET, control->tabulate+1 );
                    Cuda_Natural_Cubic_Spline( &h[1], &fCEvd[1], 
                            d_LR + index_lr (i,j,num_atom_types) , SPLINE_CEVD_OFFSET, control->tabulate+1 );

                    /*//fprintf( stderr, "%-6s  %-6s  %-6s\n", "r", "h", "fele" );
                      for( r = 1; r <= control->tabulate; ++r )
                    //fprintf( stderr, "%f  %f  %f\n", r * dr, h[r], fele[r] );
                    //fprintf( stderr, "v0_ele: %f, vlast_ele: %f\n", v0_ele, vlast_ele );
                     */
                    //Cuda_Complete_Cubic_Spline( &h[1], &fele[1], v0_ele_r, vlast_ele_r, 
                    //         &(LR[index_lr (i,j,num_atom_types) ].ele[1]), control->tabulate+1 );
                    //Cuda_Natural_Cubic_Spline( &h[1], &fCEclmb[1], 
                    //        &(LR[ index_lr (i,j,num_atom_types) ].CEclmb[1]), control->tabulate+1 );
                    Cuda_Complete_Cubic_Spline( &h[1], &fele[1], v0_ele_r, vlast_ele_r, 
                            d_LR + index_lr (i,j,num_atom_types) , SPLINE_ELE_OFFSET, control->tabulate+1 );
                    Cuda_Natural_Cubic_Spline( &h[1], &fCEclmb[1], 
                            d_LR + index_lr (i,j,num_atom_types) , SPLINE_CECLMB_OFFSET, control->tabulate+1 );
                }

    cuda_free(h, RES_LR_LOOKUP_Y);
    cuda_free(fh, RES_LR_LOOKUP_H);
    cuda_free(fvdw, RES_LR_LOOKUP_VDW);
    cuda_free(fCEvd, RES_LR_LOOKUP_CEVD);
    cuda_free(fele, RES_LR_LOOKUP_ELE);
    cuda_free(fCEclmb, RES_LR_LOOKUP_CECLMB);
}

#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_neighbors.h"
#include "cuda_list.h"
#include "vector.h"

#include "index_utils.h"
#include "reax_types.h"
#include "cuda_utils.h"
#include "tool_box.h"

//extern "C" real Get_Time( );
//extern "C" real Get_Timing_Info( real );


CUDA_DEVICE real Dev_DistSqr_to_Special_Point( rvec cp, rvec x ) 
{
    int  i;  
    real d_sqr = 0;

    for( i = 0; i < 3; ++i )
    {
        if( cp[i] > NEG_INF )
        {
            d_sqr += SQR( cp[i] - x[i] );
        }
    }

    return d_sqr;
}


CUDA_GLOBAL void k_generate_neighbor_lists( reax_atom *my_atoms, 
        simulation_box my_ext_box, grid g, reax_list far_nbrs, int n, int N )
{
    int  i, j, k, l, m, itr, num_far;
    real d, cutoff;
    ivec c, nbrs_x;
    rvec dvec;
    far_neighbor_data *nbr_data;//, *my_start;
    reax_atom *atom1, *atom2;

    l = blockIdx.x * blockDim.x  + threadIdx.x;
    if (l >= N) return;

    atom1 = &(my_atoms[l]);
    num_far = Dev_Start_Index (l, &far_nbrs);

    //get the coordinates of the atom and 
    //compute the grid cell
    /*
       i = (int) (my_atoms[ l ].x[0] * g.inv_len[0]);
       j = (int) (my_atoms[ l ].x[1] * g.inv_len[1]);
       k = (int) (my_atoms[ l ].x[2] * g.inv_len[2]);
     */
    if (l < n)
    {
        for (i = 0; i < 3; i++)
        {
            c[i] = (int)((my_atoms[l].x[i]- my_ext_box.min[i])*g.inv_len[i]);   
            if( c[i] >= g.native_end[i] )
            {
                c[i] = g.native_end[i] - 1;
            }
            else if( c[i] < g.native_str[i] )
            {
                c[i] = g.native_str[i];
            }
        }
    }
    else
    {
        for (i = 0; i < 3; i++)
        {
            c[i] = (int)((my_atoms[l].x[i] - my_ext_box.min[i]) * g.inv_len[i]);
            if( c[i] < 0 )
            {
                c[i] = 0;
            }
            else if( c[i] >= g.ncells[i] )
            {
                c[i] = g.ncells[i] - 1;
            }
        }
    }

    i = c[0];
    j = c[1];
    k = c[2];

    cutoff = SQR( g.cutoff[index_grid_3d(i, j, k, &g)] );

    itr = 0;
    while( (g.nbrs_x[index_grid_nbrs(i, j, k, itr, &g)][0]) >= 0 )
    { 
        ivec_Copy( nbrs_x, g.nbrs_x[index_grid_nbrs(i, j, k, itr, &g)] );

        if( g.str[index_grid_3d(i, j, k, &g)] <= g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)] &&  
                (Dev_DistSqr_to_Special_Point(g.nbrs_cp[index_grid_nbrs (i, j, k, itr, &g)],atom1->x)<=cutoff) )
            /* pick up another atom from the neighbor cell */
            for( m = g.str[index_grid_3d (nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; 
                    m < g.end[index_grid_3d (nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; ++m )
            {
                // prevent recounting same pairs within a gcell 
                if( l < m )
                {
                    atom2 = &(my_atoms[m]);
                    dvec[0] = atom2->x[0] - atom1->x[0];
                    dvec[1] = atom2->x[1] - atom1->x[1];
                    dvec[2] = atom2->x[2] - atom1->x[2];
                    d = rvec_Norm_Sqr( dvec );
                    if( d <= cutoff )
                    { 
                        nbr_data = &(far_nbrs.select.far_nbr_list[num_far]);
                        nbr_data->nbr = m;
                        nbr_data->d = SQRT(d);
                        rvec_Copy( nbr_data->dvec, dvec );
                        ivec_ScaledSum( nbr_data->rel_box, 1, g.rel_box[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)], 
                                -1, g.rel_box[index_grid_3d(i, j, k, &g)] );
                        ++num_far;
                    }
                }
            }
        ++itr;
    }   

    itr = 0;
    while( (g.nbrs_x[index_grid_nbrs (i, j, k, itr, &g)][0]) >= 0 )
    { 
        ivec_Copy (nbrs_x, g.nbrs_x[index_grid_nbrs (i, j, k, itr, &g)] );
        cutoff = SQR(g.cutoff[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]);

        if( g.str[index_grid_3d(i, j, k, &g)] >= g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)] &&  
                (Dev_DistSqr_to_Special_Point(g.nbrs_cp[index_grid_nbrs (i, j, k, itr, &g)],atom1->x) <= cutoff) )
            for( m = g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; 
                    m < g.end[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; ++m )
            {
                if( l > m )
                {
                    atom2 = &(my_atoms[m]);
                    dvec[0] = atom1->x[0] - atom2->x[0];
                    dvec[1] = atom1->x[1] - atom2->x[1];
                    dvec[2] = atom1->x[2] - atom2->x[2];
                    d = rvec_Norm_Sqr( dvec );
                    if( d <= cutoff )
                    {
                        nbr_data = &(far_nbrs.select.far_nbr_list[num_far]);
                        nbr_data->nbr = m;
                        nbr_data->d = SQRT(d);
                        rvec_Copy( nbr_data->dvec, dvec );
                        ivec_ScaledSum( nbr_data->rel_box, 1, g.rel_box[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)], 
                                -1, g.rel_box[index_grid_3d(i, j, k, &g)] );
                        ++num_far;
                    }
                }   
            }
        ++itr;
    }   

    Dev_Set_End_Index( l, num_far, &far_nbrs );
}


CUDA_GLOBAL void k_mt_generate_neighbor_lists( reax_atom *my_atoms, 
//CUDA_GLOBAL void __launch_bounds__ (1024) k_mt_generate_neighbor_lists( reax_atom *my_atoms, 
        simulation_box my_ext_box, grid g, reax_list far_nbrs, int n, int N )
{
    extern __shared__ int __nbr[];
    extern __shared__ int __sofar[];
    bool nbrgen;

    int __THREADS_PER_ATOM__ = NB_KER_THREADS_PER_ATOM;

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = thread_id / __THREADS_PER_ATOM__;
    int lane_id = thread_id & (__THREADS_PER_ATOM__ - 1); 
    int my_bucket = threadIdx.x / __THREADS_PER_ATOM__;

    if ( warp_id >= N )
    {
        return;
    }

    int *tnbr = __nbr;
    int *nbrssofar = __nbr + blockDim.x;
    int max, leader;

    int  i, j, k, l, m, itr, num_far, ll;
    real d, cutoff, cutoff_ji;
    ivec c, nbrs_x;
    rvec dvec;
    far_neighbor_data *nbr_data, *my_start;
    reax_atom *atom1, *atom2;

    //l = blockIdx.x * blockDim.x  + threadIdx.x;
    //if (l >= N) return;

    l = warp_id;

    atom1 = &(my_atoms[l]);
    num_far = Dev_Start_Index( l, &far_nbrs );

    my_start = &( far_nbrs.select.far_nbr_list[num_far] );

    //get the coordinates of the atom and 
    //compute the grid cell
    if (l < n)
    {
        for (i = 0; i < 3; i++)
        {
            c[i] = (int)((my_atoms[l].x[i]- my_ext_box.min[i])*g.inv_len[i]);   
            if( c[i] >= g.native_end[i] )
            {
                c[i] = g.native_end[i] - 1;
            }
            else if( c[i] < g.native_str[i] )
            {
                c[i] = g.native_str[i];
            }
        }
    }
    else
    {
        for (i = 0; i < 3; i++)
        {
            c[i] = (int)((my_atoms[l].x[i] - my_ext_box.min[i]) * g.inv_len[i]);
            if( c[i] < 0 )
            {
                c[i] = 0;
            }
            else if( c[i] >= g.ncells[i] )
            {
                c[i] = g.ncells[i] - 1;
            }
        }
    }

    i = c[0];
    j = c[1];
    k = c[2];

    //gci = &( g.cells[ index_grid_3d (i, j, k, &g) ] );

    tnbr[threadIdx.x] = 0;
    if (lane_id == 0)
    {
        nbrssofar[my_bucket] = 0;
    }
    __syncthreads( );

    itr = 0;
    while( (g.nbrs_x[index_grid_nbrs(i, j, k, itr, &g)][0]) >= 0 )
    { 
        tnbr[threadIdx.x] = 0;
        nbrgen = false;

        ivec_Copy( nbrs_x, g.nbrs_x[index_grid_nbrs(i, j, k, itr, &g)] );

        cutoff = SQR( g.cutoff[index_grid_3d(i, j, k, &g)] );
        cutoff_ji = SQR( g.cutoff[index_grid_3d( nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)] );
        if( ((g.str[index_grid_3d(i, j, k, &g)] <= g.str[index_grid_3d( nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]) 
                && (Dev_DistSqr_to_Special_Point(g.nbrs_cp[index_grid_nbrs(i, j, k, itr, &g)],atom1->x)<=cutoff)) 
                || ((g.str[index_grid_3d (i, j, k, &g)] >= g.str[index_grid_3d( nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]) 
                && (Dev_DistSqr_to_Special_Point(g.nbrs_cp[index_grid_nbrs(i, j, k, itr, &g)],atom1->x)<=cutoff_ji)))
        {
            max = g.end[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]
                    - g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)];
            tnbr[threadIdx.x] = 0;
            nbrgen = false;
            m = lane_id  + g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; //0-31
            int loopcount = max / __THREADS_PER_ATOM__ + ((max % __THREADS_PER_ATOM__) == 0 ? 0 : 1);
            int iterations = 0;

            // pick up another atom from the neighbor cell
            while (iterations < loopcount)
            {
                tnbr [threadIdx.x] = 0;
                nbrgen = false;

                // prevent recounting same pairs within a gcell 
                if( l < m  && m < g.end[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)] )
                {
                    atom2 = &(my_atoms[m]);
                    dvec[0] = atom2->x[0] - atom1->x[0];
                    dvec[1] = atom2->x[1] - atom1->x[1];
                    dvec[2] = atom2->x[2] - atom1->x[2];
                    d = rvec_Norm_Sqr( dvec );
                    if( d <= cutoff )
                    { 
                        tnbr [threadIdx.x] = 1;
                        nbrgen = true;
                    }
                }

                if( l > m  && m < g.end[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)] )
                {
                    atom2 = &(my_atoms[m]);
                    dvec[0] = atom1->x[0] - atom2->x[0];
                    dvec[1] = atom1->x[1] - atom2->x[1];
                    dvec[2] = atom1->x[2] - atom2->x[2];
                    d = rvec_Norm_Sqr( dvec );
                    if( d <= cutoff_ji )
                    {
                        tnbr [threadIdx.x] = 1;
                        nbrgen = true;
                    }
                } 

                //is neighbor generated
                if (nbrgen)
                {
                    //do leader selection here
                    leader = -1;
                    for (ll = my_bucket *__THREADS_PER_ATOM__; ll < (my_bucket)*__THREADS_PER_ATOM__ + __THREADS_PER_ATOM__; ll++)
                    {
                        if (tnbr[ll])
                        {
                            leader = ll;
                            break;
                        }
                    }

                    //do the reduction;
                    if (threadIdx.x == leader)
                    {
                        for (ll = 1; ll < __THREADS_PER_ATOM__; ll++)
                        {
                            tnbr[my_bucket * __THREADS_PER_ATOM__ + ll]
                                    += tnbr[my_bucket * __THREADS_PER_ATOM__ + (ll-1)];
                        }
                    }
                }

                if (nbrgen)
                {
                    //got the indices
                    nbr_data = my_start + nbrssofar[my_bucket] + tnbr[threadIdx.x] - 1;
                    nbr_data->nbr = m;
                    if (l < m)
                    {
                        dvec[0] = atom2->x[0] - atom1->x[0];
                        dvec[1] = atom2->x[1] - atom1->x[1];
                        dvec[2] = atom2->x[2] - atom1->x[2];
                        d = rvec_Norm_Sqr( dvec );
                        nbr_data->d = SQRT(d);
                        rvec_Copy( nbr_data->dvec, dvec );
                        ivec_ScaledSum( nbr_data->rel_box, 1, g.rel_box[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)], 
                                -1, g.rel_box[index_grid_3d(i, j, k, &g)] );
                    } 
                    else
                    {
                        dvec[0] = atom1->x[0] - atom2->x[0];
                        dvec[1] = atom1->x[1] - atom2->x[1];
                        dvec[2] = atom1->x[2] - atom2->x[2];
                        d = rvec_Norm_Sqr( dvec );
                        nbr_data->d = SQRT(d);
                        rvec_Copy( nbr_data->dvec, dvec );
                        /*
                           CHANGE ORIGINAL
                           This is a bug in the original code 
                           ivec_ScaledSum( nbr_data->rel_box, 1, g.rel_box[index_grid_3d( nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)], 
                           -1, g.rel_box[index_grid_3d( i, j, k, &g)] );
                         */
                        ivec_ScaledSum( nbr_data->rel_box, -1, g.rel_box[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)], 
                                1, g.rel_box[index_grid_3d(i, j, k, &g)] );
                    }

                    if (threadIdx.x == leader)
                    {
                        nbrssofar[my_bucket] += tnbr[my_bucket *__THREADS_PER_ATOM__ + (__THREADS_PER_ATOM__ - 1)];
                    }
                }

                m += __THREADS_PER_ATOM__;
                iterations++;

                //cleanup
                nbrgen = false;
                tnbr[threadIdx.x] = 0;
            }
        }
        ++itr;
    }

    if (lane_id == 0)
    {
        Dev_Set_End_Index( l, num_far + nbrssofar[my_bucket], &far_nbrs );
        //Dev_Set_End_Index( l, num_far, &far_nbrs );
    }
}


CUDA_GLOBAL void k_count_total_nbrs( reax_list far_nbrs, int N, int *result )
{
    //strided access
    extern __shared__ int count[];
    unsigned int i = threadIdx.x;
    int my_count = 0;
    
    count[i] = 0;

    for (i = threadIdx.x; i < N; i += threadIdx.x + blockDim.x)
    {
        count[threadIdx.x] += Dev_Num_Entries( i, &far_nbrs );
    }

    __syncthreads( );

    for (int offset = blockDim.x/2; offset > 0; offset >>=1 )
    {
        if(threadIdx.x < offset)
        {
            count[threadIdx.x] += count[threadIdx.x + offset];
        }
    }

    __syncthreads( );

    if (threadIdx.x == 0)
    {
        *result = count [threadIdx.x];
    }
}


void Cuda_Generate_Neighbor_Lists( reax_system *system, simulation_data *data, 
        storage *workspace, reax_list **lists )
{
    int i, blocks;
//    int num_far;
//    int *d_num_far = (int *) scratch;
//    int *index, *end_index;
#if defined(LOG_PERFORMANCE)
    real t_start=0, t_elapsed=0;

    if( system->my_rank == MASTER_NODE )
    {
        t_start = Get_Time( );
    }
#endif

//    cuda_memset( d_num_far, 0, sizeof(int), "num_far" );

    /* one thread per atom implementation */
//    blocks = (system->N / NBRS_BLOCK_SIZE) +
//        ((system->N % NBRS_BLOCK_SIZE) == 0 ? 0 : 1);
//    k_generate_neighbor_lists <<<blocks, NBRS_BLOCK_SIZE>>>
//        (system->d_my_atoms, system->my_ext_box, system->d_my_grid,
//         *(*dev_lists + FAR_NBRS), system->n, system->N);
//     hipDeviceSynchronize( );
//     cudaCheckError( );

    /* multiple threads per atom implementation */
    blocks = ((system->N * NB_KER_THREADS_PER_ATOM) / NBRS_BLOCK_SIZE) + 
        (((system->N * NB_KER_THREADS_PER_ATOM) % NBRS_BLOCK_SIZE) == 0 ? 0 : 1);
    k_mt_generate_neighbor_lists <<<blocks, NBRS_BLOCK_SIZE, 
        //sizeof(int) * (NBRS_BLOCK_SIZE + (NBRS_BLOCK_SIZE / NB_KER_THREADS_PER_ATOM)) >>>
        sizeof(int) * 2 * (NBRS_BLOCK_SIZE) >>>
            (system->d_my_atoms, system->my_ext_box, system->d_my_grid,
            *(*dev_lists + FAR_NBRS), system->n, system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );

//    k_count_total_nbrs <<<1, NBRS_BLOCK_SIZE, sizeof (int) * NBRS_BLOCK_SIZE>>>
//            (*(*dev_lists + FAR_NBRS), system->N, d_num_far);
//    hipDeviceSynchronize( );
//    cudaCheckError( );
//    copy_host_device( &num_far, d_num_far, sizeof (int), hipMemcpyDeviceToHost, "num_far" );

#if defined(LOG_PERFORMANCE)
    if( system->my_rank == MASTER_NODE )
    {
        t_elapsed = Get_Timing_Info( t_start );
        data->timing.nbrs += t_elapsed;
    }
#endif

#if defined(DEBUG_FOCUS)  
    fprintf( stderr, "p%d @ step%d: nbrs done\n", 
            system->my_rank, data->step );
    MPI_Barrier( MPI_COMM_WORLD );
#endif
}


CUDA_GLOBAL void k_estimate_neighbors( reax_atom *my_atoms, 
        simulation_box my_ext_box, grid g, int n, int N, int *indices )
{
    int i, j, k, l, m, itr, num_far;
    real d, cutoff;
    rvec dvec, c;
    ivec nbrs_x;
    grid_cell *gci, *gcj;
    far_neighbor_data *nbr_data;//, *my_start;
    reax_atom *atom1, *atom2;

    l = blockIdx.x * blockDim.x  + threadIdx.x;
    if (l >= N)
    {
        return;
    }

    num_far = 0;
    atom1 = &(my_atoms[l]);
    indices[l] = 0;

    /* get the coordinates of the atom and compute the grid cell
     * if atom is locally owned by processor AND not ghost atom */
    if (l < n)
    {
        for (i = 0; i < 3; i++)
        {
            c[i] = (int)((my_atoms[l].x[i]- my_ext_box.min[i])*g.inv_len[i]);   
            if( c[i] >= g.native_end[i] )
            {
                c[i] = g.native_end[i] - 1;
            }
            else if( c[i] < g.native_str[i] )
            {
                c[i] = g.native_str[i];
            }
        }
    }
    /* same as above, but for ghost atoms */
    else
    {
        for (i = 0; i < 3; i++)
        {
            c[i] = (int)((my_atoms[l].x[i] - my_ext_box.min[i]) * g.inv_len[i]);
            if( c[i] < 0 )
            {
                c[i] = 0;
            }
            else if( c[i] >= g.ncells[i] )
            {
                c[i] = g.ncells[i] - 1;
            }
        }
    }

    i = c[0];
    j = c[1];
    k = c[2];

    cutoff = SQR( g.cutoff[ index_grid_3d(i, j, k, &g) ] );

    itr = 0;
    while( (g.nbrs_x[index_grid_nbrs(i, j, k, itr, &g)][0]) >= 0)
    { 
        ivec_Copy( nbrs_x, g.nbrs_x[index_grid_nbrs(i, j, k, itr, &g)] );
        //gcj =  &( g.cells [ index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g) ]);

        if( //(g.str[index_grid_3d(i, j, k, &g)] <= g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]) &&  
                Dev_DistSqr_to_Special_Point(g.nbrs_cp[index_grid_nbrs(i, j, k, itr, &g)],atom1->x) <= cutoff ) 
        {
            // pick up another atom from the neighbor cell 
            for( m = g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; 
                    m < g.end[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; ++m )
            {
                // prevent recounting same pairs within a gcell 
                if( l < m )
                {
                    atom2 = &(my_atoms[m]);
                    dvec[0] = atom2->x[0] - atom1->x[0];
                    dvec[1] = atom2->x[1] - atom1->x[1];
                    dvec[2] = atom2->x[2] - atom1->x[2];
                    d = rvec_Norm_Sqr( dvec );
                    if( d <= cutoff )
                    { 
                        num_far++;
                    }
                }   
            }
        }
        ++itr;

    }   

    itr = 0;
    while( (g.nbrs_x[index_grid_nbrs(i, j, k, itr, &g)][0]) >= 0 )
    {
        ivec_Copy( nbrs_x, g.nbrs_x[index_grid_nbrs(i, j, k, itr, &g)] );
        //gcj =  &( g.cells [ index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g) ]);
        cutoff = SQR( g.cutoff[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)] );

        if( g.str[index_grid_3d(i, j, k, &g)] >= g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)] &&  
                Dev_DistSqr_to_Special_Point(g.nbrs_cp[index_grid_nbrs(i, j, k, itr, &g)],atom1->x) <= cutoff ) 
        {
            // pick up another atom from the neighbor cell 
            for( m = g.str[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; 
                    m < g.end[index_grid_3d(nbrs_x[0], nbrs_x[1], nbrs_x[2], &g)]; ++m )
            {
                // prevent recounting same pairs within a gcell 
                if( l > m )
                {
                    atom2 = &(my_atoms[m]);
                    dvec[0] = atom2->x[0] - atom1->x[0];
                    dvec[1] = atom2->x[1] - atom1->x[1];
                    dvec[2] = atom2->x[2] - atom1->x[2];
                    d = rvec_Norm_Sqr( dvec );
                    if( d <= cutoff )
                    { 
                        num_far++;
                    }
                }   
            }
        }
        ++itr;
    }   

    indices[l] = num_far;// * SAFE_ZONE;
}


int Cuda_Estimate_Neighbors( reax_system *system, int *nbr_indices )
{
    int i, blocks, ret;
    int *indices = (int *) scratch;
    reax_list *far_nbrs;

    ret = SUCCESS;

    cuda_memset( indices, 0, sizeof(int) * system->total_cap, 
            "dev_neighbors:indices");

    blocks = system->N / DEF_BLOCK_SIZE + 
        ((system->N % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_estimate_neighbors <<< blocks, DEF_BLOCK_SIZE >>>
        (system->d_my_atoms, (system->my_ext_box), system->d_my_grid, 
         system->n, system->N, indices);
    hipDeviceSynchronize( );
    cudaCheckError( );

    copy_host_device( nbr_indices, indices, sizeof(int) * system->total_cap, 
            hipMemcpyDeviceToHost, "dev_nbrs:indices" );

    /* build neighbor indices (num. far nbrs per atom) */
    for ( i = 0; i < system->total_cap; i++ )
    {
        /* check if per atom bond limits are exceeded,
         * and, if so, trigger reallocation */
        if ( nbr_indices[i] > system->max_far_nbrs[i] )
        {
            ret = FAILURE;
            system->max_far_nbrs[i] = MAX( (int)CEIL(nbr_indices[i] * SAFE_ZONE), MIN_NBRS );
        }
    }

    return ret;
}


void Cuda_Init_Neighbor_Indices( int *indices, int entries )
{
    int i;
    reax_list *far_nbrs = *dev_lists + FAR_NBRS;

    for (i = 1; i < entries; i++)
    {
        indices[i] += indices[i - 1];
    }

    copy_host_device( indices, (far_nbrs->index + 1), (entries - 1) * sizeof(int), 
            hipMemcpyHostToDevice, "dev_nbrs:index" );
    copy_host_device( indices, far_nbrs->end_index, entries * sizeof(int), 
            hipMemcpyHostToDevice, "dev_nbrs:end_index" );
}


void Cuda_Init_HBond_Indices( int *indices, int entries )
{
    int i;
    reax_list *hbonds = *dev_lists + HBONDS;

    for ( i = 1 ; i < entries; i++ )
    {
        indices[i] += indices[i-1];
    }

    copy_host_device( indices, hbonds->index + 1, (entries-1) * sizeof(int), 
            hipMemcpyHostToDevice, "dev_hbonds:index" );
    copy_host_device( indices, hbonds->end_index + 1, (entries-1) * sizeof(int), 
            hipMemcpyHostToDevice, "dev_hbonds:end_index" );
}


void Cuda_Init_Bond_Indices( int *indices, int entries )
{
    int i;
    reax_list *bonds = *dev_lists + BONDS;

    for (i = 1 ; i < entries; i++)
    {
        indices[i] += indices[i - 1];
    }

    copy_host_device( indices, (bonds->index + 1), (entries - 1) * sizeof(int),
            hipMemcpyHostToDevice, "dev_bonds:index" );
    copy_host_device( indices, bonds->end_index, entries * sizeof(int),
            hipMemcpyHostToDevice, "dev_bonds:end_index" );
}


void Cuda_Init_Three_Body_Indices( int *indices, int entries )
{
    int i;
    reax_list *thbody = *dev_lists + THREE_BODIES;

    copy_host_device( indices, thbody->index + 1,
            sizeof(int) * (entries - 1),
            hipMemcpyHostToDevice, "dev_thb:index" );
    copy_host_device( indices, thbody->end_index,
            sizeof(int) * entries,
            hipMemcpyHostToDevice, "dev_thb:end_index" );

}

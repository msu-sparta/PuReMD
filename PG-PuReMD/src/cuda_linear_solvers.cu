/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_linear_solvers.h"

#include "reax_types.h"
#include "cuda_utils.h"
#include "reduction.h"
#include "dual_matvec.h"
#include "matvec.h"



void get_from_device (real *host, real *device, unsigned int bytes, char *msg)
{
    copy_host_device (host, device, bytes, hipMemcpyDeviceToHost, msg);
}

void put_on_device (real *host, real *device, unsigned int bytes, char *msg)
{
    copy_host_device (host, device, bytes, hipMemcpyHostToDevice, msg);
}

void Cuda_Vector_Sum (real *res, real a, real *x, real b, real *y, int count)
{
    //res = ax + by
    //use the cublas here
    int blocks;
    blocks = (count / DEF_BLOCK_SIZE) + 
        ((count % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_vector_sum <<< blocks, DEF_BLOCK_SIZE >>>
        ( res, a, x, b, y, count );
    hipDeviceSynchronize ();
    cudaCheckError ();
}

void Cuda_CG_Preconditioner (real *res, real *a, real *b, int count)
{
    //res = a*b - vector multiplication
    //use the cublas here.
    int blocks;
    blocks = (count / DEF_BLOCK_SIZE) + 
        ((count % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_vector_mul <<< blocks, DEF_BLOCK_SIZE >>>
        ( res, a, b, count );
    hipDeviceSynchronize ();
}

CUDA_GLOBAL void k_diagnol_preconditioner (storage p_workspace, rvec2 *b, int n)
{
    storage *workspace = &( p_workspace );
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= n) return;

    //for( j = 0; j < system->n; ++j ) {
    // residual 
    workspace->r2[j][0] = b[j][0] - workspace->q2[j][0];
    workspace->r2[j][1] = b[j][1] - workspace->q2[j][1];
    // apply diagonal pre-conditioner
    workspace->d2[j][0] = workspace->r2[j][0] * workspace->Hdia_inv[j]; 
    workspace->d2[j][1] = workspace->r2[j][1] * workspace->Hdia_inv[j]; 
    //}
}

void Cuda_CG_Diagnol_Preconditioner (storage *workspace, rvec2 *b, int n)
{
    int blocks;

    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_diagnol_preconditioner <<< blocks, DEF_BLOCK_SIZE >>>
        (*workspace, b, n);
    hipDeviceSynchronize ();
    cudaCheckError ();
}

CUDA_GLOBAL void k_dual_cg_preconditioner (storage p_workspace, rvec2 *x, 
        real alpha_0, real alpha_1, int n, rvec2 *my_dot)
{
    storage *workspace = &( p_workspace );
    rvec2 alpha;
    alpha[0] = alpha_0;
    alpha[1] = alpha_1;

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= n) return;
    my_dot[j][0] = my_dot[j][1] = 0.0;

    //for( j = 0; j < system->n; ++j ) {
    // update x 
    x[j][0] += alpha[0] * workspace->d2[j][0];
    x[j][1] += alpha[1] * workspace->d2[j][1];      
    // update residual 
    workspace->r2[j][0] -= alpha[0] * workspace->q2[j][0]; 
    workspace->r2[j][1] -= alpha[1] * workspace->q2[j][1]; 
    // apply diagonal pre-conditioner 
    workspace->p2[j][0] = workspace->r2[j][0] * workspace->Hdia_inv[j];
    workspace->p2[j][1] = workspace->r2[j][1] * workspace->Hdia_inv[j];
    // dot product: r.p 
    my_dot[j][0] = workspace->r2[j][0] * workspace->p2[j][0];
    my_dot[j][1] = workspace->r2[j][1] * workspace->p2[j][1];
    //}
}

void Cuda_DualCG_Preconditioer (storage *workspace, rvec2 *x, rvec2 alpha, int n, rvec2 result)
{
    int blocks;
    rvec2 *tmp = (rvec2 *) scratch;
    cuda_memset (tmp, 0, sizeof (rvec2) * ( 2 * n + 1), "cuda_dualcg_preconditioner");

    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_dual_cg_preconditioner <<< blocks, DEF_BLOCK_SIZE >>>
        (*workspace, x, alpha[0], alpha[1], n, tmp);
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction to calculate my_dot
    k_reduction_rvec2 <<< blocks, DEF_BLOCK_SIZE, sizeof (rvec2) * DEF_BLOCK_SIZE >>>
        ( tmp, tmp + n, n);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction_rvec2 <<< 1, BLOCKS_POW_2, sizeof (rvec2) * BLOCKS_POW_2 >>>
        ( tmp + n, tmp + 2*n, blocks);
    hipDeviceSynchronize ();
    cudaCheckError ();

    copy_host_device (result, (tmp + 2*n), sizeof (rvec2), hipMemcpyDeviceToHost, "my_dot");
}

void Cuda_Norm (rvec2 *arr, int n, rvec2 result)
{
    int blocks;
    rvec2 *tmp = (rvec2 *) scratch;

    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_norm_rvec2 <<< blocks, DEF_BLOCK_SIZE, sizeof (rvec2) * DEF_BLOCK_SIZE >>>
        (arr, tmp, n, INITIAL);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_norm_rvec2 <<< 1, BLOCKS_POW_2, sizeof (rvec2) * BLOCKS_POW_2 >>>
        (tmp, tmp + BLOCKS_POW_2, blocks, FINAL );
    hipDeviceSynchronize ();
    cudaCheckError ();

    copy_host_device (result, tmp + BLOCKS_POW_2, sizeof (rvec2), 
            hipMemcpyDeviceToHost, "cuda_norm_rvec2");
}

void Cuda_Dot (rvec2 *a, rvec2 *b, rvec2 result, int n)
{
    int blocks;
    rvec2 *tmp = (rvec2 *) scratch;

    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_dot_rvec2 <<< blocks, DEF_BLOCK_SIZE, sizeof (rvec2) * DEF_BLOCK_SIZE >>>
        ( a, b, tmp, n );
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_norm_rvec2 <<< 1, BLOCKS_POW_2, sizeof (rvec2) * BLOCKS_POW_2 >>> 
        //k_norm_rvec2 <<< blocks, DEF_BLOCK_SIZE, sizeof (rvec2) * BLOCKS_POW_2 >>> 
        ( tmp, tmp + BLOCKS_POW_2, blocks, FINAL );
    hipDeviceSynchronize ();
    cudaCheckError ();

    copy_host_device (result, tmp + BLOCKS_POW_2, sizeof (rvec2), 
            hipMemcpyDeviceToHost, "cuda_dot");
}

void Cuda_Vector_Sum_Rvec2 (rvec2 *x, rvec2 *a, rvec2 b, rvec2 *c, int n)
{
    int blocks;

    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_rvec2_pbetad <<< blocks, DEF_BLOCK_SIZE >>> 
        ( x, a, b[0], b[1], c, n);
    hipDeviceSynchronize ();
    cudaCheckError ();
}

CUDA_GLOBAL void k_rvec2_to_real_copy ( real *dst, rvec2 *src, int index, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    dst[i] = src[i][index];
}

void Cuda_RvecCopy_From (real *dst, rvec2 *src, int index, int n)
{
    int blocks;
    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_rvec2_to_real_copy <<< blocks, DEF_BLOCK_SIZE >>>
        ( dst, src, index, n);
    hipDeviceSynchronize ();
    cudaCheckError ();
}

CUDA_GLOBAL void k_real_to_rvec2_copy ( rvec2 *dst, real *src, int index, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    dst[i][index] = src[i];
}

void Cuda_RvecCopy_To (rvec2 *dst, real *src, int index, int n)
{
    int blocks;
    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_real_to_rvec2_copy <<< blocks, DEF_BLOCK_SIZE >>>
        ( dst, src, index, n);
    hipDeviceSynchronize ();
    cudaCheckError ();
}

void Cuda_Dual_Matvec (sparse_matrix *H, rvec2 *a, rvec2 *b, int n, int size)
{
    int blocks;
    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE) == 0 ? 0 : 1);

    cuda_memset (b, 0, sizeof (rvec2) * size, "dual_matvec:result");

    //One thread per row implementation
    //k_dual_matvec <<< blocks, DEF_BLOCK_SIZE >>>
    //        (*H, a, b, n);
    //hipDeviceSynchronize ();
    //cudaCheckError ();

    //One warp per row implementation
#if defined(__SM_35__)
    k_dual_matvec_csr <<< MATVEC_BLOCKS, MATVEC_BLOCK_SIZE >>>
#else
        k_dual_matvec_csr <<< MATVEC_BLOCKS, MATVEC_BLOCK_SIZE, 
                  sizeof (rvec2) * MATVEC_BLOCK_SIZE >>>
#endif
                      (*H, a, b, n);
    hipDeviceSynchronize ();
    cudaCheckError ();
}

void Cuda_Matvec (sparse_matrix *H, real *a, real *b, int n, int size)
{
    int blocks;
    blocks = (n / DEF_BLOCK_SIZE) + 
        (( n % DEF_BLOCK_SIZE) == 0 ? 0 : 1);

    cuda_memset (b, 0, sizeof (real) * size, "dual_matvec:result");

    //one thread per row implementation
    //k_matvec <<< blocks, DEF_BLOCK_SIZE >>>
    //        (*H, a, b, n);
    //hipDeviceSynchronize ();
    //cudaCheckError ();

#if defined(__SM_35__)
    k_matvec_csr <<< MATVEC_BLOCKS, MATVEC_BLOCK_SIZE >>>
#else
        k_matvec_csr <<< MATVEC_BLOCKS, MATVEC_BLOCK_SIZE, 
                 sizeof (real) * MATVEC_BLOCK_SIZE>>>
#endif
                     (*H, a, b, n);
    hipDeviceSynchronize ();
    cudaCheckError ();
}


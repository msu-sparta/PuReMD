#include "cuda_utils.h"


extern "C" void cuda_malloc(void **ptr, int size, int mem_set, const char *msg)
{

    hipError_t retVal = hipSuccess;

    retVal = hipMalloc( ptr, size );

    if( retVal != hipSuccess )
    {
        fprintf( stderr,
                "ERROR: failed to allocate memory on device for resouce %s\nCUDA API error code: %d, requested memory size (in bytes): %d\n", 
                msg, retVal, size );
        exit( INSUFFICIENT_MEMORY );
    }  

    if( mem_set )
    {
        retVal = hipMemset( *ptr, 0, size );

        if( retVal != hipSuccess )
        {
            fprintf( stderr,
                    "ERROR: failed to memset memory on device for resource %s\nCUDA API error code: %d, requested memory size (in bytes): %d\n", 
                    msg, retVal, size );
            exit( INSUFFICIENT_MEMORY );
        }
    }  
}


extern "C" void cuda_free(void *ptr, const char *msg)
{

    hipError_t retVal = hipSuccess;

    if ( !ptr )
    {
        return;
    }  

    retVal = hipFree( ptr );

    if( retVal != hipSuccess )
    {
        fprintf( stderr,
                "WARNING: failed to release memory on device for resource %s\nCUDA API error code: %d, memory address: %ld\n", 
                msg, retVal, (long int) ptr );
        return;
    }  
}


extern "C" void cuda_memset(void *ptr, int data, size_t count, const char *msg){
    hipError_t retVal = hipSuccess;

    retVal = hipMemset( ptr, data, count );

    if( retVal != hipSuccess )
    {
        fprintf( stderr,
                "ERROR: failed to memset memory on device for resource %s\nCUDA API error code: %d\n", 
                msg, retVal );
        exit( INSUFFICIENT_MEMORY );
    }
}


extern "C" void copy_host_device(void *host, void *dev, int size, enum hipMemcpyKind dir, const char *msg)
{
    hipError_t retVal = hipErrorNotReady;

    if( dir == hipMemcpyHostToDevice )
    {
        retVal = hipMemcpy( dev, host, size, hipMemcpyHostToDevice );
    }
    else
    {
        retVal = hipMemcpy( host, dev, size, hipMemcpyDeviceToHost );
    }

    if( retVal != hipSuccess )
    {
        fprintf( stderr,
                "ERROR: could not copy resource %s from host to device\nCUDA API error code: %d n",
                msg, retVal );
        exit( INSUFFICIENT_MEMORY );
    }
}


extern "C" void copy_device(void *dest, void *src, int size, const char *msg)
{
    hipError_t retVal = hipErrorNotReady;

    retVal = hipMemcpy( dest, src, size, hipMemcpyDeviceToDevice );
    if( retVal != hipSuccess )
    {
        fprintf( stderr,
                "ERROR: could not copy resource %s from device to device\nCUDA API error code: %d\n",
                msg, retVal );
        exit( INSUFFICIENT_MEMORY );
    }
}


extern "C" void compute_blocks( int *blocks, int *block_size, int count )
{
    *block_size = CUDA_BLOCK_SIZE;
    *blocks = (int) CEIL((double) count / CUDA_BLOCK_SIZE);
}


extern "C" void compute_matvec_blocks( int *blocks, int count )
{

    *blocks = (int) CEIL((double) count * MATVEC_KER_THREADS_PER_ROW / MATVEC_BLOCK_SIZE);
}


extern "C" void compute_nearest_pow_2(int blocks, int *result)
{

  *result = (int) EXP2( CEIL( LOG2((double) blocks) ) );
}


extern "C" void print_device_mem_usage()
{
    size_t total, free;

    hipMemGetInfo( &free, &total );

    if ( hipGetLastError() != hipSuccess )
    {
        fprintf( stderr, "WARNING: error on the CUDA get memory info call\n" );
        return;
    }

    fprintf( stderr,
            "Total %ld Mb %ld gig %ld , free %ld, Mb %ld , gig %ld \n", 
            total, total/(1024*1024), total/ (1024*1024*1024), 
            free, free/(1024*1024), free/ (1024*1024*1024) );
}

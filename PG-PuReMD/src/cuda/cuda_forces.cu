#include "hip/hip_runtime.h"

#include "cuda_forces.h"

#include "cuda_bonds.h"
#include "cuda_bond_orders.h"
#include "cuda_charges.h"
#include "cuda_helpers.h"
#include "cuda_hydrogen_bonds.h"
#include "cuda_list.h"
#include "cuda_multi_body.h"
#include "cuda_neighbors.h"
#include "cuda_nonbonded.h"
#include "cuda_reduction.h"
#include "cuda_spar_lin_alg.h"
#include "cuda_torsion_angles.h"
#include "cuda_utils.h"
#include "cuda_valence_angles.h"

#include "../basic_comm.h"
#include "../forces.h"
#include "../index_utils.h"
#include "../tool_box.h"
#include "../vector.h"

#include <cub/util_ptx.cuh>
#include <cub/warp/warp_reduce.cuh>
#include <cub/warp/warp_scan.cuh>


#define FULL_WARP_MASK (0xFFFFFFFF)


typedef enum
{
    DIAGONAL = 0,
    OFF_DIAGONAL = 1,
} MATRIX_ENTRY_POSITION;


CUDA_DEVICE real Init_Charge_Matrix_Entry( single_body_parameters const * const sbp_i,
        real const * const workspace_Tap, control_params const * const control,
        int i, int j, real r_ij, real gamma, MATRIX_ENTRY_POSITION pos )
{
    real Tap, dr3gamij_1, dr3gamij_3, ret;

    ret = 0.0;

    switch ( control->charge_method )
    {
    case QEQ_CM:
    case EE_CM:
    case ACKS2_CM:
        switch ( pos )
        {
            case OFF_DIAGONAL:
                Tap = workspace_Tap[7] * r_ij + workspace_Tap[6];
                Tap = Tap * r_ij + workspace_Tap[5];
                Tap = Tap * r_ij + workspace_Tap[4];
                Tap = Tap * r_ij + workspace_Tap[3];
                Tap = Tap * r_ij + workspace_Tap[2];
                Tap = Tap * r_ij + workspace_Tap[1];
                Tap = Tap * r_ij + workspace_Tap[0];    

                /* shielding */
                dr3gamij_1 = r_ij * r_ij * r_ij
                    + POW( gamma, -3.0 );
                dr3gamij_3 = POW( dr3gamij_1 , 1.0 / 3.0 );

                /* i == j: periodic self-interaction term
                 * i != j: general interaction term */
                ret = ((i == j) ? 0.5 : 1.0) * Tap * EV_to_KCALpMOL / dr3gamij_3;
                break;

            case DIAGONAL:
                ret = sbp_i->eta;
                break;

            default:
//                fprintf( stderr, "[ERROR] Invalid matrix position. Terminating...\n" );
//                exit( INVALID_INPUT );
                break;
        }
        break;


    default:
//        fprintf( stderr, "[ERROR] Invalid charge method. Terminating...\n" );
//        exit( INVALID_INPUT );
        break;
    }

    return ret;
}


CUDA_DEVICE real Init_Charge_Matrix_Entry_Tab( LR_lookup_table const * const t_LR, real r_ij,
        int ti, int tj, int num_atom_types )
{
    int r, tmin, tmax;
    real val, dif, base;

    tmin = MIN( ti, tj );
    tmax = MAX( ti, tj );
    LR_lookup_table const * const t = &t_LR[ index_lr(tmin,tmax, num_atom_types) ];

    /* cubic spline interpolation */
    r = (int)(r_ij * t->inv_dx);
    if ( r == 0 )
    {
        ++r;
    }
    base = (real)(r + 1) * t->dx;
    dif = r_ij - base;
    val = ((t->ele[r].d * dif + t->ele[r].c) * dif + t->ele[r].b) * dif
        + t->ele[r].a;
    val *= EV_to_KCALpMOL / C_ELE;

    return val;
}


CUDA_GLOBAL void k_init_end_index( int const * const intr_cnt,
        int const * const indices, int * const end_indices, int N )
{
    int i;

    i = blockIdx.x * blockDim.x  + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    end_indices[i] = indices[i] + intr_cnt[i];
}


CUDA_GLOBAL void k_init_hbond_indices( reax_atom * const atoms,
        single_body_parameters const * const sbp,
        int const * const hbonds, int const * const max_hbonds,
        int * const indices, int * const end_indices, int N )
{
    int i, hindex, flag;

    i = blockIdx.x * blockDim.x  + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    hindex = atoms[i].Hindex;

    flag = (sbp[ atoms[i].type ].p_hbond == H_ATOM
            || sbp[ atoms[i].type ].p_hbond == H_BONDING_ATOM ? TRUE : FALSE);

    indices[hindex] = (flag == TRUE ? max_hbonds[i] : 0);
    end_indices[hindex] = (flag == TRUE ? indices[hindex] + hbonds[i] : 0);
    atoms[i].num_hbonds = (flag == TRUE ? hbonds[i] : 0);
}


CUDA_GLOBAL void k_print_hbond_info( reax_atom *my_atoms, single_body_parameters *sbp, 
        control_params *control, reax_list hbond_list, int N )
{
    int i;
    int type_i;
    single_body_parameters *sbp_i;
    reax_atom *atom_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    atom_i = &my_atoms[i];
    type_i = atom_i->type;
    sbp_i = &sbp[type_i];

    printf( "atom %6d: ihb = %2d, ihb_top = %2d\n", i, sbp_i->p_hbond,
            Start_Index( atom_i->Hindex, &hbond_list ) );
}


/* 1 thread computes the distances and displacement vectors of an atom for its neighbors
 * in the far neighbors list if it's a NOT re-neighboring step
 */
CUDA_GLOBAL void k_init_dist( reax_atom const * const my_atoms,
        reax_list far_nbr_list, int N )
{
    int i, j, pj, start_i, end_i;
    rvec x_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    rvec_Copy( x_i, my_atoms[i].x );

    /* update distance and displacement vector between atoms i and j (i-j) */
    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];

        far_nbr_list.far_nbr_list.dvec[pj][0] = my_atoms[j].x[0] - x_i[0];
        far_nbr_list.far_nbr_list.dvec[pj][1] = my_atoms[j].x[1] - x_i[1];
        far_nbr_list.far_nbr_list.dvec[pj][2] = my_atoms[j].x[2] - x_i[2];
        far_nbr_list.far_nbr_list.d[pj] = rvec_Norm( far_nbr_list.far_nbr_list.dvec[pj] );
    }
}


/* 1 warp of threads computes the distances and displacement vectors of an atom for its neighbors
 * in the far neighbors list if it's a NOT re-neighboring step
 */
CUDA_GLOBAL void k_init_dist_opt( reax_atom const * const my_atoms,
        reax_list far_nbr_list, int N )
{
    int j, pj, start_i, end_i, thread_id, i, lane_id;
    rvec x_i;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    i = thread_id / warpSize;

    if ( i >= N )
    {
        return;
    }

    lane_id = thread_id % warpSize; 
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    rvec_Copy( x_i, my_atoms[i].x );

    /* update distance and displacement vector between atoms i and j (i-j) */
    for ( pj = start_i + lane_id; pj < end_i; pj += warpSize )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];

        far_nbr_list.far_nbr_list.dvec[pj][0] = my_atoms[j].x[0] - x_i[0];
        far_nbr_list.far_nbr_list.dvec[pj][1] = my_atoms[j].x[1] - x_i[1];
        far_nbr_list.far_nbr_list.dvec[pj][2] = my_atoms[j].x[2] - x_i[2];
        far_nbr_list.far_nbr_list.d[pj] = rvec_Norm( far_nbr_list.far_nbr_list.dvec[pj] );
    }
}


/* Compute the charge matrix entries and store the matrix in half format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
CUDA_GLOBAL void k_init_cm_half_fs( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, orig_id_i;
    int cm_top;
    int num_cm_entries;
    real r_ij;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        type_i = my_atoms[i].type;
        orig_id_i = my_atoms[i].orig_id;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry in the matrix */
        H->j[cm_top] = i;
        H->val[cm_top] = Init_Charge_Matrix_Entry( &sbp[type_i], workspace.Tap, control,
                i, i, 0.0, 0.0, DIAGONAL );
        ++cm_top;

        for ( pj = start_i; pj < end_i; ++pj )
        {
            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];

                /* if j is a local OR ghost atom in the upper triangular region of the matrix */
                if ( orig_id_i < my_atoms[j].orig_id )
                {
                    r_ij = far_nbr_list.far_nbr_list.d[pj];

                    H->j[cm_top] = j;
                    H->val[cm_top] = Init_Charge_Matrix_Entry( &sbp[type_i], workspace.Tap,
                            control, i, H->j[cm_top], r_ij,
                            tbp[ index_tbp(type_i, my_atoms[j].type, num_atom_types) ].gamma, OFF_DIAGONAL );
                    ++cm_top;
                }
            }
        }
    }

    __syncthreads( );

    H->end[i] = cm_top;
    num_cm_entries = cm_top - H->start[i];

    /* reallocation check */
    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


/* Compute the tabulated charge matrix entries and store the matrix in half format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
CUDA_GLOBAL void k_init_cm_half_fs_tab( reax_atom * const my_atoms,
        single_body_parameters const * const sbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, LR_lookup_table const * const t_LR, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    int cm_top;
    int num_cm_entries;
    real r_ij;
    reax_atom *atom_i, *atom_j;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        atom_i = &my_atoms[i];
        type_i = atom_i->type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry in the matrix */
        H->j[cm_top] = i;
        H->val[cm_top] = Init_Charge_Matrix_Entry( &sbp[type_i], workspace.Tap, control,
                i, i, 0.0, 0.0, DIAGONAL );
        ++cm_top;

        for ( pj = start_i; pj < end_i; ++pj )
        {
            j = far_nbr_list.far_nbr_list.nbr[pj];

            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                atom_j = &my_atoms[j];
                type_j = atom_j->type;

                /* if j is a local OR ghost atom in the upper triangular region of the matrix */
                if ( atom_i->orig_id < atom_j->orig_id )
                {
                    r_ij = far_nbr_list.far_nbr_list.d[pj];

                    H->j[cm_top] = j;
                    H->val[cm_top] = Init_Charge_Matrix_Entry_Tab( t_LR, r_ij,
                            type_i, type_j, num_atom_types );
                    ++cm_top;
                }
            }
        }
    }

    __syncthreads( );

    H->end[i] = cm_top;
    num_cm_entries = cm_top - H->start[i];

    /* reallocation check */
    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


/* Compute the charge matrix entries and store the matrix in full format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
CUDA_GLOBAL void k_init_cm_full_fs( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i;
    int cm_top;
    int num_cm_entries;
    reax_atom *atom_i;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        atom_i = &my_atoms[i];
        type_i = atom_i->type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry in the matrix */
        H->j[cm_top] = i;
        H->val[cm_top] = Init_Charge_Matrix_Entry( &sbp[type_i], workspace.Tap, control,
                i, i, 0.0, 0.0, DIAGONAL );
        ++cm_top;

        for ( pj = start_i; pj < end_i; ++pj )
        {
            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];

                H->j[cm_top] = j;
                H->val[cm_top] = Init_Charge_Matrix_Entry( &sbp[type_i], workspace.Tap,
                        control, i, j, far_nbr_list.far_nbr_list.d[pj],
                        tbp[ index_tbp(type_i, my_atoms[j].type, num_atom_types) ].gamma,
                        OFF_DIAGONAL );
                ++cm_top;
            }
        }
    }

    __syncthreads( );

    H->end[i] = cm_top;
    num_cm_entries = cm_top - H->start[i];

    /* reallocation check */
    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


/* Compute the charge matrix entries and store the matrix in full format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
CUDA_GLOBAL void k_init_cm_full_fs_opt( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    extern __shared__ hipcub::WarpScan<int>::TempStorage temp1[];
    int i, j, pj, thread_id, warp_id, lane_id, itr;
    int start_i, end_i, type_i;
    int cm_top, num_cm_entries, offset, flag;
    reax_atom *atom_i;
    sparse_matrix *H;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    /* all threads within a warp are assigned the same unique row 
     * in the charge matrix */
    i = thread_id / warpSize;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    warp_id = threadIdx.x / warpSize;
    lane_id = thread_id % warpSize;
    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        atom_i = &my_atoms[i];
        type_i = atom_i->type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry in the matrix */
        if ( lane_id == 0 )
        {
            H->j[cm_top] = i;
            H->val[cm_top] = Init_Charge_Matrix_Entry( &sbp[type_i], workspace.Tap, control,
                    i, i, 0.0, 0.0, DIAGONAL );
        }
        ++cm_top;

        for ( itr = 0, pj = start_i + lane_id; itr < (end_i - start_i + warpSize - 1) / warpSize; ++itr )
        {
            offset = (pj < end_i && far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut) ? 1 : 0;
            flag = (offset == 1) ? TRUE : FALSE;
            hipcub::WarpScan<int>(temp1[warp_id]).ExclusiveSum(offset, offset);

            if ( flag == TRUE )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];

                H->j[cm_top + offset] = j;
                H->val[cm_top + offset] = Init_Charge_Matrix_Entry( &sbp[type_i], workspace.Tap,
                        control, i, j, far_nbr_list.far_nbr_list.d[pj],
                        tbp[ index_tbp(type_i, my_atoms[j].type, num_atom_types) ].gamma, OFF_DIAGONAL );
            }

            /* get cm_top from thread in last lane */
            cm_top = cm_top + offset + (flag == TRUE ? 1 : 0);
            cm_top = hipcub::ShuffleIndex<32>( cm_top, warpSize - 1, FULL_WARP_MASK );

            pj += warpSize;
        }
    }

    if ( lane_id == 0 )
    {
        H->end[i] = cm_top;
        num_cm_entries = cm_top - H->start[i];

        /* reallocation check */
        if ( num_cm_entries > max_cm_entries[i] )
        {
            *realloc_cm_entries = TRUE;
        }
    }
}


/* Compute the tabulated charge matrix entries and store the matrix in full format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
CUDA_GLOBAL void k_init_cm_full_fs_tab( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, 
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, LR_lookup_table *t_LR, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i;
    int cm_top;
    int num_cm_entries;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        type_i = my_atoms[i].type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry in the matrix */
        H->j[cm_top] = i;
        H->val[cm_top] = Init_Charge_Matrix_Entry( &sbp[type_i], workspace.Tap, control,
                i, i, 0.0, 0.0, DIAGONAL );
        ++cm_top;

        for ( pj = start_i; pj < end_i; ++pj )
        {
            j = far_nbr_list.far_nbr_list.nbr[pj];

            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                H->j[cm_top] = j;
                H->val[cm_top] = Init_Charge_Matrix_Entry_Tab( t_LR,
                        far_nbr_list.far_nbr_list.d[pj],
                        type_i, my_atoms[j].type, num_atom_types );
                ++cm_top;
            }
        }
    }

    __syncthreads( );

    H->end[i] = cm_top;
    num_cm_entries = cm_top - H->start[i];

    /* reallocation check */
    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


CUDA_GLOBAL void k_init_bonds( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, reax_list bond_list, int n, int N,
        int num_atom_types, int * const max_bonds, int * const realloc_bonds )
{
    int i, j, pj, start_i, end_i;
    int type_i, type_j, tbp_ij;
    int btop_i, num_bonds;
    real total_bond_order_i;
    rvec dDeltap_self_i;
    real cutoff, r_ij;
    real C12, C34, C56;
    real BO_s, BO_pi, BO_pi2, BO;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    type_i = my_atoms[i].type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    btop_i = Start_Index( i, &bond_list );
    total_bond_order_i = 0.0;
    rvec_MakeZero( dDeltap_self_i );

    if ( i < n )
    {
        cutoff = MIN( control->nonb_cut, control->bond_cut );
//        workspace.bond_mark[i] = 0;
    }
    else
    {
        cutoff = control->bond_cut;
        /* put ghost atoms to an infinite distance (i.e., 1000) */
//        workspace.bond_mark[i] = 1000;
    }

    /* check if j is within cutoff */
    for ( pj = start_i; pj < end_i; ++pj )
    {
        /* uncorrected bond orders */
        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff )
        {
            j = far_nbr_list.far_nbr_list.nbr[pj];
            type_j = my_atoms[j].type;
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(type_i, type_j, num_atom_types);

            /* uncorrected bond orders */
            if ( sbp[type_i].r_s > 0.0 && sbp[type_j].r_s > 0.0 )
            {
                C12 = tbp[tbp_ij].p_bo1 * POW( r_ij / tbp[tbp_ij].r_s, tbp[tbp_ij].p_bo2 );
                BO_s = (1.0 + control->bo_cut) * EXP( C12 );
            }
            else
            {
                C12 = 0.0;
                BO_s = 0.0;
            }

            if ( sbp[type_i].r_pi > 0.0 && sbp[type_j].r_pi > 0.0 )
            {
                C34 = tbp[tbp_ij].p_bo3 * POW( r_ij / tbp[tbp_ij].r_p, tbp[tbp_ij].p_bo4 );
                BO_pi = EXP( C34 );
            }
            else
            {
                C34 = 0.0;
                BO_pi = 0.0;
            }

            if ( sbp[type_i].r_pi_pi > 0.0 && sbp[type_j].r_pi_pi > 0.0 )
            {
                C56 = tbp[tbp_ij].p_bo5 * POW( r_ij / tbp[tbp_ij].r_pp, tbp[tbp_ij].p_bo6 );
                BO_pi2 = EXP( C56 );
            }
            else
            {
                C56 = 0.0;
                BO_pi2 = 0.0;
            }

            /* initially BO values are the uncorrected ones, page 1 */
            BO = BO_s + BO_pi + BO_pi2;

            if ( BO >= control->bo_cut )
            {
                /* compute and append bond info to list */
                Cuda_Compute_BOp( bond_list, control->bo_cut, i, btop_i,
                        far_nbr_list.far_nbr_list.nbr[pj],
                        C12, C34, C56, BO_s, BO_pi, BO_pi2, BO,
                        &far_nbr_list.far_nbr_list.rel_box[pj],
                        far_nbr_list.far_nbr_list.d[pj],
                        &far_nbr_list.far_nbr_list.dvec[pj], far_nbr_list.format,
                        &tbp[tbp_ij], dDeltap_self_i, &total_bond_order_i );

                ++btop_i;

                /* TODO: future optimization if bond_mark implemented */
//                if ( workspace->bond_mark[j] > workspace->bond_mark[i] + 1 )
//                {
//                    workspace->bond_mark[j] = workspace->bond_mark[i] + 1;
//                }
//                else if ( workspace->bond_mark[i] > workspace->bond_mark[j] + 1 )
//                {
//                    workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
//                }
            }
        }
    }

    Set_End_Index( i, btop_i, &bond_list );

    num_bonds = btop_i - Start_Index( i, &bond_list );

    /* copy bond info to atom structure
     * (needed for atom ownership transfer via MPI) */
    my_atoms[i].num_bonds = num_bonds;

    workspace.total_bond_order[i] = total_bond_order_i;
    rvec_Copy( workspace.dDeltap_self[i], dDeltap_self_i );

    /* reallocation check */
    if ( num_bonds > max_bonds[i] )
    {
        *realloc_bonds = TRUE;
    }
}


CUDA_GLOBAL void k_init_bonds_opt( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, reax_list bond_list, int n, int N,
        int num_atom_types, int * const max_bonds, int * const realloc_bonds )
{
    extern __shared__ hipcub::WarpScan<int>::TempStorage temp21[];
    hipcub::WarpReduce<double>::TempStorage *temp22;
    int i, j, pj, thread_id, warp_id, lane_id, itr;
    int start_i, end_i, tbp_ij;
    int type_i, type_j;
    int btop_i, offset, flag, num_bonds;
    real cutoff, r_ij;
    real C12, C34, C56;
    real BO_s, BO_pi, BO_pi2, BO;
    real total_bond_order_i;
    rvec dDeltap_self_i;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    /* all threads within a warp are assigned the bonds
     * for a unique atom */
    i = thread_id / warpSize;

    if ( i >= N )
    {
        return;
    }

    warp_id = threadIdx.x / warpSize;
    lane_id = thread_id % warpSize;
    temp22 = (hipcub::WarpReduce<double>::TempStorage *) &temp21[warp_id];
    type_i = my_atoms[i].type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    btop_i = Start_Index( i, &bond_list );
    total_bond_order_i = 0.0;
    rvec_MakeZero( dDeltap_self_i );

    if ( i < n )
    {
        cutoff = MIN( control->nonb_cut, control->bond_cut );
//        workspace.bond_mark[i] = 0;
    }
    else
    {
        cutoff = control->bond_cut;
        /* put ghost atoms to an infinite distance (i.e., 1000) */
//        workspace.bond_mark[i] = 1000;
    }

    for ( itr = 0, pj = start_i + lane_id; itr < (end_i - start_i + warpSize - 1) / warpSize; ++itr )
    {
        /* uncorrected bond orders */
        if ( pj < end_i && far_nbr_list.far_nbr_list.d[pj] <= cutoff )
        {
            j = far_nbr_list.far_nbr_list.nbr[pj];
            type_j = my_atoms[j].type;
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(type_i, type_j, num_atom_types);

            /* uncorrected bond orders */
            if ( sbp[type_i].r_s > 0.0 && sbp[type_j].r_s > 0.0 )
            {
                C12 = tbp[tbp_ij].p_bo1 * POW( r_ij / tbp[tbp_ij].r_s, tbp[tbp_ij].p_bo2 );
                BO_s = (1.0 + control->bo_cut) * EXP( C12 );
            }
            else
            {
                C12 = 0.0;
                BO_s = 0.0;
            }

            if ( sbp[type_i].r_pi > 0.0 && sbp[type_j].r_pi > 0.0 )
            {
                C34 = tbp[tbp_ij].p_bo3 * POW( r_ij / tbp[tbp_ij].r_p, tbp[tbp_ij].p_bo4 );
                BO_pi = EXP( C34 );
            }
            else
            {
                C34 = 0.0;
                BO_pi = 0.0;
            }

            if ( sbp[type_i].r_pi_pi > 0.0 && sbp[type_j].r_pi_pi > 0.0 )
            {
                C56 = tbp[tbp_ij].p_bo5 * POW( r_ij / tbp[tbp_ij].r_pp, tbp[tbp_ij].p_bo6 );
                BO_pi2 = EXP( C56 );
            }
            else
            {
                C56 = 0.0;
                BO_pi2 = 0.0;
            }
        }
        else
        {
            BO_s = 0.0;
            BO_pi = 0.0;
            BO_pi2 = 0.0;
        }

        /* initially BO values are the uncorrected ones, page 1 */
        BO = BO_s + BO_pi + BO_pi2;

        offset = (pj < end_i && far_nbr_list.far_nbr_list.d[pj] <= cutoff && BO >= control->bo_cut) ? 1 : 0;
        flag = (offset == 1) ? TRUE : FALSE;
        hipcub::WarpScan<int>(temp21[warp_id]).ExclusiveSum(offset, offset);

        if ( flag == TRUE )
        {
            /* compute and append bond info to list */
            Cuda_Compute_BOp( bond_list, control->bo_cut, i, btop_i + offset,
                    far_nbr_list.far_nbr_list.nbr[pj],
                    C12, C34, C56, BO_s, BO_pi, BO_pi2, BO,
                    &far_nbr_list.far_nbr_list.rel_box[pj],
                    far_nbr_list.far_nbr_list.d[pj],
                    &far_nbr_list.far_nbr_list.dvec[pj], far_nbr_list.format,
                    &tbp[tbp_ij], dDeltap_self_i, &total_bond_order_i );

            /* TODO: future optimization if bond_mark implemented */
//            if ( workspace->bond_mark[j] > workspace->bond_mark[i] + 1 )
//            {
//                workspace->bond_mark[j] = workspace->bond_mark[i] + 1;
//            }
//            else if ( workspace->bond_mark[i] > workspace->bond_mark[j] + 1 )
//            {
//                workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
//            }
        }

        /* get btop_i from thread in last lane */
        btop_i = btop_i + offset + (flag == TRUE ? 1 : 0);
        btop_i = hipcub::ShuffleIndex<32>( btop_i, warpSize - 1, FULL_WARP_MASK );

        pj += warpSize;
    }

    total_bond_order_i = hipcub::WarpReduce<double>(temp22[warp_id]).Sum(total_bond_order_i);
    dDeltap_self_i[0] = hipcub::WarpReduce<double>(temp22[warp_id]).Sum(dDeltap_self_i[0]);
    dDeltap_self_i[1] = hipcub::WarpReduce<double>(temp22[warp_id]).Sum(dDeltap_self_i[1]);
    dDeltap_self_i[2] = hipcub::WarpReduce<double>(temp22[warp_id]).Sum(dDeltap_self_i[2]);

    if ( lane_id == 0 )
    {
        Set_End_Index( i, btop_i, &bond_list );

        num_bonds = btop_i - Start_Index( i, &bond_list );

        /* copy bond info to atom structure
         * (needed for atom ownership transfer via MPI) */
        my_atoms[i].num_bonds = num_bonds;

        workspace.total_bond_order[i] = total_bond_order_i;
        rvec_Copy( workspace.dDeltap_self[i], dDeltap_self_i );

        /* reallocation check */
        if ( num_bonds > max_bonds[i] )
        {
            *realloc_bonds = TRUE;
        }
    }
}


/* Construct the interaction list for hydrogen bonds */
CUDA_GLOBAL void k_init_hbonds( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, control_params const * const control,
        reax_list far_nbr_list, reax_list hbond_list,
        int n, int N, int num_atom_types, int * const max_hbonds, int * const realloc_hbonds )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb, ihb_top;
    int num_hbonds;
    real cutoff;
    reax_atom *atom_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    atom_i = &my_atoms[i];
    type_i = atom_i->type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    ihb = sbp[type_i].p_hbond;

    cutoff = MIN( control->nonb_cut, control->hbond_cut );

    ihb_top = Start_Index( atom_i->Hindex, &hbond_list );

    if ( (i < n && ihb == H_ATOM) || ihb == H_BONDING_ATOM )
    {
        /* check if j is within cutoff */
        for ( pj = start_i; pj < end_i; ++pj )
        {
            if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                jhb = sbp[type_j].p_hbond;

                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( i >= n && j < n
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM )
                {
                    hbond_list.hbond_list[ihb_top].nbr = j;
                    hbond_list.hbond_list[ihb_top].scl = -1;
                    hbond_list.hbond_list[ihb_top].ptr = pj;

#if !defined(CUDA_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );
#endif

                    ++ihb_top;
                }
                /* atom i: H atom, native
                 * atom j: H bonding atom */
                else if ( i < n
                        && ihb == H_ATOM && jhb == H_BONDING_ATOM )
                {
                    hbond_list.hbond_list[ihb_top].nbr = j;
                    hbond_list.hbond_list[ihb_top].scl = 1;
                    hbond_list.hbond_list[ihb_top].ptr = pj;

#if !defined(CUDA_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );
#endif

                    ++ihb_top;
                }
                /* atom i: H bonding atom, native
                 * atom j: H atom, native */
                else if ( i < n
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                {
                    hbond_list.hbond_list[ihb_top].nbr = j;
                    hbond_list.hbond_list[ihb_top].scl = -1;
                    hbond_list.hbond_list[ihb_top].ptr = pj;

#if !defined(CUDA_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );
#endif

                    ++ihb_top;
                }
            }
        }
    }

    Set_End_Index( atom_i->Hindex, ihb_top, &hbond_list );

    num_hbonds = ihb_top - Start_Index( atom_i->Hindex, &hbond_list );

    /* copy hbond info to atom structure
     * (needed for atom ownership transfer via MPI) */
    my_atoms[i].num_hbonds = num_hbonds;

    /* reallocation check */
    if ( num_hbonds > max_hbonds[i] )
    {
        *realloc_hbonds = TRUE;
    }
}


/* Construct the interaction list for hydrogen bonds */
CUDA_GLOBAL void k_init_hbonds_opt( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, control_params const * const control,
        reax_list far_nbr_list, reax_list hbond_list,
        int n, int N, int num_atom_types, int * const max_hbonds, int * const realloc_hbonds )
{
    extern __shared__ hipcub::WarpScan<int>::TempStorage temp[];
    int i, j, pj, thread_id, warp_id, lane_id, itr;
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb, ihb_top, offset, flag;
    int num_hbonds;
    real cutoff;
    reax_atom *atom_i;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    /* all threads within a warp are assigned the bonds
     * for a unique atom */
    i = thread_id / warpSize;

    if ( i >= N )
    {
        return;
    }

    warp_id = threadIdx.x / warpSize;
    lane_id = thread_id % warpSize;
    atom_i = &my_atoms[i];
    type_i = atom_i->type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    ihb = sbp[type_i].p_hbond;

    cutoff = MIN( control->nonb_cut, control->hbond_cut );

    ihb_top = Start_Index( atom_i->Hindex, &hbond_list );

    if ( (i < n && ihb == H_ATOM) || ihb == H_BONDING_ATOM )
    {
        for ( itr = 0, pj = start_i + lane_id; itr < (end_i - start_i + warpSize - 1) / warpSize; ++itr )
        {
            if ( pj < end_i )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                jhb = sbp[type_j].p_hbond;

                offset = (pj < end_i && far_nbr_list.far_nbr_list.d[pj] <= cutoff
                        && ((i >= n && j < n && ihb == H_BONDING_ATOM && jhb == H_ATOM)
                            || (i < n && ihb == H_ATOM && jhb == H_BONDING_ATOM)
                            || (i < n && ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n))) ? 1 : 0;
            }
            else
            {
                offset = 0;
            }

            flag = (offset == 1) ? TRUE : FALSE;
            hipcub::WarpScan<int>(temp[warp_id]).ExclusiveSum(offset, offset);

            if ( flag == TRUE )
            {
                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( i >= n && j < n
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM )
                {
                    hbond_list.hbond_list[ihb_top + offset].nbr = j;
                    hbond_list.hbond_list[ihb_top + offset].scl = -1;
                    hbond_list.hbond_list[ihb_top + offset].ptr = pj;

#if !defined(CUDA_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top + offset].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top + offset].hb_f );
#endif
                }
                /* atom i: H atom, native
                 * atom j: H bonding atom */
                else if ( i < n
                        && ihb == H_ATOM && jhb == H_BONDING_ATOM )
                {
                    hbond_list.hbond_list[ihb_top + offset].nbr = j;
                    hbond_list.hbond_list[ihb_top + offset].scl = 1;
                    hbond_list.hbond_list[ihb_top + offset].ptr = pj;

#if !defined(CUDA_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top + offset].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top + offset].hb_f );
#endif
                }
                /* atom i: H bonding atom, native
                 * atom j: H atom, native */
                else if ( i < n
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                {
                    hbond_list.hbond_list[ihb_top + offset].nbr = j;
                    hbond_list.hbond_list[ihb_top + offset].scl = -1;
                    hbond_list.hbond_list[ihb_top + offset].ptr = pj;

#if !defined(CUDA_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top + offset].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top + offset].hb_f );
#endif
                }
            }

            /* get ihb_top from thread in last lane */
            ihb_top = ihb_top + offset + (flag == TRUE ? 1 : 0);
            ihb_top = hipcub::ShuffleIndex<32>( ihb_top, warpSize - 1, FULL_WARP_MASK );

            pj += warpSize;
        }
    }

    if ( lane_id == 0 )
    {
        Set_End_Index( atom_i->Hindex, ihb_top, &hbond_list );

        num_hbonds = ihb_top - Start_Index( atom_i->Hindex, &hbond_list );

        /* copy hbond info to atom structure
         * (needed for atom ownership transfer via MPI) */
        my_atoms[i].num_hbonds = num_hbonds;

        /* reallocation check */
        if ( num_hbonds > max_hbonds[i] )
        {
            *realloc_hbonds = TRUE;
        }
    }
}


/* Construct the interaction list for bonds */
CUDA_GLOBAL void k_estimate_storages_cm_half( reax_atom const * const my_atoms,
        control_params const * const control, reax_list far_nbr_list,
        int cm_n, int cm_n_max, int * const cm_entries, int * const max_cm_entries )
{
    int i, j, pj; 
    int start_i, end_i;
    int num_cm_entries;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= cm_n_max )
    {
        return;
    }

    num_cm_entries = 0;

    if ( i < cm_n )
    {
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry */
        ++num_cm_entries;

        for ( pj = start_i; pj < end_i; ++pj )
        { 
            j = far_nbr_list.far_nbr_list.nbr[pj];

            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut
                    && (j < cm_n || my_atoms[i].orig_id < my_atoms[j].orig_id) )
            {
                ++num_cm_entries;
            }
        }
    }

    __syncthreads( );

    cm_entries[i] = num_cm_entries;
    /* round up to the nearest multiple of 32 to ensure that reads along
     * rows can be coalesced for 1 warp per row SpMV implementation */
    max_cm_entries[i] = MAX( ((int) CEIL( num_cm_entries * SAFE_ZONE )
                + warpSize - 1) / warpSize * warpSize, MIN_CM_ENTRIES );
}


CUDA_GLOBAL void k_estimate_storages_cm_full( control_params const * const control,
        reax_list far_nbr_list, int cm_n, int cm_n_max,
        int * const cm_entries, int * const max_cm_entries )
{
    int i, pj; 
    int start_i, end_i;
    int num_cm_entries;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= cm_n_max )
    {
        return;
    }

    num_cm_entries = 0;

    if ( i < cm_n )
    {
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry */
        ++num_cm_entries;

        for ( pj = start_i; pj < end_i; ++pj )
        { 
            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                ++num_cm_entries;
            }
        }
    }

    __syncthreads( );

    cm_entries[i] = num_cm_entries;
    /* round up to the nearest multiple of 32 to ensure that reads along
     * rows can be coalesced for 1 warp per row SpMV implementation */
    max_cm_entries[i] = MAX( ((int) CEIL( num_cm_entries * SAFE_ZONE )
                + warpSize - 1) / warpSize * warpSize, MIN_CM_ENTRIES );
}


CUDA_GLOBAL void k_estimate_storage_bonds( reax_atom const * const my_atoms, 
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        control_params *control, reax_list far_nbr_list, 
        int num_atom_types, int n, int N, int total_cap,
        int * const bonds, int * const max_bonds )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j, tbp_ij;
    int num_bonds;
    real cutoff, r_ij; 
    real C12, C34, C56;
    real BO_s, BO_pi, BO_pi2;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= total_cap )
    {
        return;
    }

    num_bonds = 0;

    if ( i < N )
    {
        type_i = my_atoms[i].type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        if ( i < n )
        {
            cutoff = MIN( control->nonb_cut, control->bond_cut );
        }
        else
        {
            cutoff = control->bond_cut;
        }

        for ( pj = start_i; pj < end_i; ++pj )
        { 
            if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                r_ij = far_nbr_list.far_nbr_list.d[pj];
                tbp_ij = index_tbp(type_i ,type_j, num_atom_types);

                /* uncorrected bond orders */
                if ( sbp[type_i].r_s > 0.0 && sbp[type_j].r_s > 0.0 )
                {
                    C12 = tbp[tbp_ij].p_bo1 * POW( r_ij / tbp[tbp_ij].r_s, tbp[tbp_ij].p_bo2 );
                    BO_s = (1.0 + control->bo_cut) * EXP( C12 );
                }
                else
                {
                    C12 = 0.0;
                    BO_s = 0.0;
                }

                if ( sbp[type_i].r_pi > 0.0 && sbp[type_j].r_pi > 0.0 )
                {
                    C34 = tbp[tbp_ij].p_bo3 * POW( r_ij / tbp[tbp_ij].r_p, tbp[tbp_ij].p_bo4 );
                    BO_pi = EXP( C34 );
                }
                else
                {
                    C34 = 0.0;
                    BO_pi = 0.0;
                }

                if ( sbp[type_i].r_pi_pi > 0.0 && sbp[type_j].r_pi_pi > 0.0 )
                {
                    C56 = tbp[tbp_ij].p_bo5 * POW( r_ij / tbp[tbp_ij].r_pp, tbp[tbp_ij].p_bo6 );
                    BO_pi2= EXP( C56 );
                }
                else
                {
                    C56 = 0.0;
                    BO_pi2 = 0.0;
                }

                /* initially BO values are the uncorrected ones, page 1 */
                if ( BO_s + BO_pi + BO_pi2 >= control->bo_cut )
                {
                    ++num_bonds;
                }
            }
        }
    }

    __syncthreads( );

    bonds[i] = num_bonds;
    max_bonds[i] = MAX( (int) CEIL(2 * num_bonds * SAFE_ZONE), MIN_BONDS );
}


CUDA_GLOBAL void k_estimate_storage_hbonds( reax_atom const * const my_atoms, 
        single_body_parameters const * const sbp, control_params const * const control,
        reax_list far_nbr_list, int num_atom_types, int n, int N,
        int total_cap, int * const hbonds, int * const max_hbonds )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb;
    int num_hbonds;
    real cutoff;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= total_cap )
    {
        return;
    }

    num_hbonds = 0;

    if ( i < N )
    {
        type_i = my_atoms[i].type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );
        ihb = sbp[type_i].p_hbond;

        if ( i < n )
        { 
            cutoff = control->nonb_cut;
        }   
        else
        {
            cutoff = control->bond_cut;
        } 

        if ( i < n && ihb == H_ATOM || ihb == H_BONDING_ATOM )
        {
            for ( pj = start_i; pj < end_i; ++pj )
            { 
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                jhb = sbp[type_j].p_hbond;

                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( i >= n && j < n && ihb == H_BONDING_ATOM && jhb == H_ATOM
                        && far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut
                        && far_nbr_list.far_nbr_list.d[pj] <= control->hbond_cut )
                {
                    ++num_hbonds;
                }
                else if ( i < n && far_nbr_list.far_nbr_list.d[pj] <= cutoff
                        && far_nbr_list.far_nbr_list.d[pj] <= control->hbond_cut )
                {
                    /* atom i: H atom, native
                     * atom j: H bonding atom */
                    if ( ihb == H_ATOM && jhb == H_BONDING_ATOM )
                    {
                        ++num_hbonds;
                    }
                    /* atom i: H bonding atom, native
                     * atom j: H atom, native */
                    else if ( ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                    {
                        ++num_hbonds;
                    }
                }
            }
        }
    }

    __syncthreads( );

    hbonds[i] = num_hbonds;
    max_hbonds[i] = MAX( (int) CEIL(num_hbonds * SAFE_ZONE), MIN_HBONDS );
}


CUDA_GLOBAL void k_update_sym_dbond_indices( reax_list bond_list, int N )
{
    int i, pj, pk, nbr_ij, nbr_jk;
    bond_data *ibond, *jbond;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    /* i-j bonds */
    for ( pj = Start_Index(i, &bond_list); pj < End_Index(i, &bond_list); ++pj )
    {
        ibond = &bond_list.bond_list[pj];
        nbr_ij = ibond->nbr;

        /* j-k bonds */
        for ( pk = Start_Index(nbr_ij, &bond_list); pk < End_Index(nbr_ij, &bond_list); ++pk )
        {
            jbond = &bond_list.bond_list[pk];
            nbr_jk = jbond->nbr;

            if ( i == nbr_jk && i > nbr_ij )
            {
                ibond->dbond_index = pj;
                jbond->dbond_index = pj;

                ibond->sym_index = pk;
                jbond->sym_index = pj;
                break;
            }
        }
    }
}


#if !defined(CUDA_ACCUM_ATOMIC)
CUDA_GLOBAL void k_update_sym_hbond_indices_opt( reax_atom *my_atoms,
        reax_list hbond_list, int N )
{
    int i, pj, pk;
    int nbr, nbrstart, nbrend;
    int start, end;
    hbond_data *ihbond, *jhbond;
    int thread_id, lane_id;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    i = thread_id / warpSize;

    if ( i > N )
    {
        return;
    }

    lane_id = thread_id % warpSize; 
    start = Start_Index( my_atoms[i].Hindex, &hbond_list );
    end = End_Index( my_atoms[i].Hindex, &hbond_list );
    pj = start + lane_id;

    while ( pj < end )
    {
        ihbond = &hbond_list.hbond_list[pj];
        nbr = ihbond->nbr;

        nbrstart = Start_Index( my_atoms[nbr].Hindex, &hbond_list );
        nbrend = End_Index( my_atoms[nbr].Hindex, &hbond_list );

        for ( pk = nbrstart; pk < nbrend; pk++ )
        {
            jhbond = &hbond_list.hbond_list[pk];

            if ( jhbond->nbr == i )
            {
                ihbond->sym_index = pk;
                jhbond->sym_index = pj;
                break;
            }
        }

        pj += warpSize;
    }
}
#endif


#if defined(DEBUG_FOCUS)
CUDA_GLOBAL void k_print_forces( reax_atom *my_atoms, rvec *f, int n )
{
    int i; 

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    printf( "%8d: %24.15f, %24.15f, %24.15f\n",
            my_atoms[i].orig_id, f[i][0], f[i][1], f[i][2] );
}


CUDA_GLOBAL void k_print_hbonds( reax_atom *my_atoms, reax_list hbond_list, int n, int rank, int step )
{
    int i, k, pj, start, end; 
    hbond_data *hbond_jk;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    start = Start_Index( my_atoms[i].Hindex, &hbond_list );
    end = End_Index( my_atoms[i].Hindex, &hbond_list );

    for ( pj = start; pj < end; ++pj )
    {
        k = hbond_list.hbond_list[pj].nbr;
        hbond_jk = &hbond_list.hbond_list[pj];

#if !defined(CUDA_ACCUM_ATOMIC)
        printf( "p%03d, step %05d: %8d: %8d, %24.15f, %24.15f, %24.15f\n",
                rank, step, my_atoms[i].Hindex, k,
                hbond_jk->hb_f[0],
                hbond_jk->hb_f[1],
                hbond_jk->hb_f[2] );
#else
        printf( "p%03d, step %05d: %8d: %8d\n",
                rank, step, my_atoms[i].Hindex, k );
#endif
    }
}
#endif


#if defined(DEBUG_FOCUS)
static void Print_Forces( reax_system *system, control_params *control )
{
    int blocks;
    
    blocks = (system->n) / DEF_BLOCK_SIZE
        + (((system->n % DEF_BLOCK_SIZE) == 0) ? 0 : 1);

    k_print_forces <<< blocks, DEF_BLOCK_SIZE, 0, control->streams[0] >>>
        ( system->d_my_atoms, workspace->d_workspace->f, system->n );
    cudaCheckError( );
}


static void Print_HBonds( reax_system *system, control_params *control,
        int step )
{
    int blocks;
    
    blocks = (system->n) / DEF_BLOCK_SIZE
        + (((system->n % DEF_BLOCK_SIZE) == 0) ? 0 : 1);

    k_print_hbonds <<< blocks, DEF_BLOCK_SIZE, 0, control->streams[0] >>>
        ( system->d_my_atoms, *(lists[HBONDS]), system->n, system->my_rank, step );
    cudaCheckError( );
}
#endif


/* Initialize indices for far neighbors list post reallocation
 *
 * system: atomic system info. */
void Cuda_Init_Neighbor_Indices( reax_system *system, control_params *control,
        reax_list *far_nbr_list )
{
    int blocks;

    blocks = far_nbr_list->n / DEF_BLOCK_SIZE
        + (far_nbr_list->n % DEF_BLOCK_SIZE == 0 ? 0 : 1);

    /* init indices */
    Cuda_Scan_Excl_Sum( system->d_max_far_nbrs, far_nbr_list->index,
            far_nbr_list->n, 0, control->streams[0] );

    /* init end_indices */
    k_init_end_index <<< blocks, DEF_BLOCK_SIZE, 0, control->streams[0] >>>
        ( system->d_far_nbrs, far_nbr_list->index, far_nbr_list->end_index,
          far_nbr_list->n );
    cudaCheckError( );
}


/* Initialize indices for far hydrogen bonds list post reallocation
 *
 * system: atomic system info. */
void Cuda_Init_HBond_Indices( reax_system *system, storage *workspace,
        reax_list *hbond_list, hipStream_t s )
{
    int blocks, *temp;

    blocks = system->total_cap / DEF_BLOCK_SIZE
        + (system->total_cap % DEF_BLOCK_SIZE == 0 ? 0 : 1);

    sCudaCheckMalloc( &workspace->scratch[2], &workspace->scratch_size[2],
            sizeof(int) * system->total_cap, __FILE__, __LINE__ );
    temp = (int *) workspace->scratch[2];

    /* init indices and end_indices */
    Cuda_Scan_Excl_Sum( system->d_max_hbonds, temp, system->total_cap, 2, s );

    k_init_hbond_indices <<< blocks, DEF_BLOCK_SIZE, 0, s >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, system->d_hbonds, temp, 
          hbond_list->index, hbond_list->end_index, system->total_cap );
    cudaCheckError( );
}


/* Initialize indices for far bonds list post reallocation
 *
 * system: atomic system info. */
void Cuda_Init_Bond_Indices( reax_system *system, reax_list * bond_list,
        hipStream_t s )
{
    int blocks;

    blocks = system->total_cap / DEF_BLOCK_SIZE + 
        (system->total_cap % DEF_BLOCK_SIZE == 0 ? 0 : 1);

    /* init indices */
    Cuda_Scan_Excl_Sum( system->d_max_bonds, bond_list->index,
            system->total_cap, 1, s );

    /* init end_indices */
    k_init_end_index <<< blocks, DEF_BLOCK_SIZE, 0, s >>>
        ( system->d_bonds, bond_list->index, bond_list->end_index, system->total_cap );
    cudaCheckError( );
}


/* Initialize indices for charge matrix post reallocation
 *
 * system: atomic system info.
 * H: charge matrix */
void Cuda_Init_Sparse_Matrix_Indices( reax_system *system, sparse_matrix *H,
        hipStream_t s )
{
    int blocks;

    blocks = H->n_max / DEF_BLOCK_SIZE
        + (H->n_max % DEF_BLOCK_SIZE == 0 ? 0 : 1);

    /* init indices */
    Cuda_Scan_Excl_Sum( system->d_max_cm_entries, H->start, H->n_max, 5, s );

    //TODO: not needed for full format (Init_Forces sets H->end)
    /* init end_indices */
    k_init_end_index <<< blocks, DEF_BLOCK_SIZE, 0, s >>>
        ( system->d_cm_entries, H->start, H->end, H->n_max );
    cudaCheckError( );
}


void Cuda_Estimate_Storages( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, reax_list **lists,
        int realloc_cm, int realloc_bonds, int realloc_hbonds, int step )
{
    int blocks;
#if defined(LOG_PERFORMANCE)
    float time_elapsed;
#endif

    if ( realloc_cm == TRUE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_CM_START], control->streams[5] );
#endif

        blocks = workspace->d_workspace->H.n_max / DEF_BLOCK_SIZE
            + (workspace->d_workspace->H.n_max % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        if ( workspace->d_workspace->H.format == SYM_HALF_MATRIX )
        {
            k_estimate_storages_cm_half <<< blocks, DEF_BLOCK_SIZE, 0,
                                        control->streams[5] >>>
                ( system->d_my_atoms, (control_params *) control->d_control_params,
                  *(lists[FAR_NBRS]), workspace->d_workspace->H.n,
                  workspace->d_workspace->H.n_max,
                  system->d_cm_entries, system->d_max_cm_entries );
        }
        else
        {
            k_estimate_storages_cm_full <<< blocks, DEF_BLOCK_SIZE, 0,
                                        control->streams[5] >>>
                ( (control_params *) control->d_control_params,
                  *(lists[FAR_NBRS]), workspace->d_workspace->H.n,
                  workspace->d_workspace->H.n_max,
                  system->d_cm_entries, system->d_max_cm_entries );
        }
        cudaCheckError( );

        Cuda_Reduction_Sum( system->d_max_cm_entries, system->d_total_cm_entries,
                workspace->d_workspace->H.n_max, 5, control->streams[5] );
        sCudaMemcpyAsync( &system->total_cm_entries, system->d_total_cm_entries,
                sizeof(int), hipMemcpyDeviceToHost, control->streams[5], __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_CM_STOP], control->streams[5] );
#endif
    }

    if ( realloc_bonds == TRUE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_BOND_START], control->streams[1] );
#endif

        blocks = system->total_cap / DEF_BLOCK_SIZE
            + (system->total_cap % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        k_estimate_storage_bonds <<< blocks, DEF_BLOCK_SIZE, 0,
                                 control->streams[1] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp, 
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
              system->n, system->N, system->total_cap,
              system->d_bonds, system->d_max_bonds );
        cudaCheckError( );

        Cuda_Reduction_Sum( system->d_max_bonds, system->d_total_bonds,
                system->total_cap, 1, control->streams[1] );
        sCudaMemcpyAsync( &system->total_bonds, system->d_total_bonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->streams[1], __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_BOND_STOP], control->streams[1] );
#endif
    }

    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && realloc_hbonds == TRUE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_HBOND_START], control->streams[2] );
#endif

        blocks = system->total_cap / DEF_BLOCK_SIZE
            + (system->total_cap % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        k_estimate_storage_hbonds <<< blocks, DEF_BLOCK_SIZE, 0,
                                  control->streams[2] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
              system->n, system->N, system->total_cap,
              system->d_hbonds, system->d_max_hbonds );
        cudaCheckError( );

        Cuda_Reduction_Sum( system->d_max_hbonds, system->d_total_hbonds,
                system->total_cap, 2, control->streams[2] );
        sCudaMemcpyAsync( &system->total_hbonds, system->d_total_hbonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->streams[2], __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_HBOND_STOP], control->streams[2] );
#endif
    }

    if ( realloc_cm == TRUE )
    {
        hipStreamSynchronize( control->streams[5] );

#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_CM_START],
                control->time_events[TE_INIT_CM_STOP] ); 
        data->timing.init_cm += (real) (time_elapsed / 1000.0);
#endif
    }
    if ( realloc_bonds == TRUE )
    {
        hipStreamSynchronize( control->streams[1] );

#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_BOND_START],
                control->time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);
#endif
    }
    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && realloc_hbonds == TRUE )
    {
        hipStreamSynchronize( control->streams[2] );

#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_HBOND_START],
                control->time_events[TE_INIT_HBOND_STOP] ); 
        data->timing.init_hbond += (real) (time_elapsed / 1000.0);
#endif
    }
}


/* Initialize the bond list, hydrogen bond list, and charge matrix
 * data structures along with updating the pairwise distances in the
 * far neighbor (Verlet) list if required.
 *
 * NOTE: the control flow of the code follows a
 * try-compute-else-reallocate-and-retry logic which requires that
 * the initialization kernels be atomic transactions. Locks are used
 * to mark if the transaction succeeds (and thus should not be repeated).
 */
int Cuda_Init_Forces( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    int renbr, blocks, ret, realloc_cm, realloc_bonds, realloc_hbonds;
    static int dist_done = FALSE, cm_done = FALSE, bonds_done = FALSE, hbonds_done = FALSE;
#if defined(LOG_PERFORMANCE)
    float time_elapsed;
#endif

    renbr = (data->step - data->prev_steps) % control->reneighbor == 0 ? TRUE : FALSE;

    /* reset reallocation flags on device */
    if ( cm_done == FALSE )
    {
        sCudaMemsetAsync( system->d_realloc_cm_entries, FALSE, sizeof(int), 
                control->streams[5], __FILE__, __LINE__ );
    }
    if ( bonds_done == FALSE )
    {
        sCudaMemsetAsync( system->d_realloc_bonds, FALSE, sizeof(int), 
                control->streams[1], __FILE__, __LINE__ );
    }
    if ( hbonds_done == FALSE )
    {
        sCudaMemsetAsync( system->d_realloc_hbonds, FALSE, sizeof(int), 
                control->streams[2], __FILE__, __LINE__ );
    }

    if ( renbr == FALSE && dist_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_DIST_START], control->streams[0] );
#endif

//        k_init_dist <<< control->blocks_n, control->block_size_n, 0, control->streams[0] >>>
//            ( system->d_my_atoms, *(lists[FAR_NBRS]), system->N );

        blocks = system->N * 32 / DEF_BLOCK_SIZE
            + (system->N * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        k_init_dist_opt <<< blocks, DEF_BLOCK_SIZE, 0, control->streams[0] >>>
            ( system->d_my_atoms, *(lists[FAR_NBRS]), system->N );
        cudaCheckError( );

        hipEventRecord( control->stream_events[SE_INIT_DIST_DONE], control->streams[0] );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_DIST_STOP], control->streams[0] );
#endif

        dist_done = TRUE;
    }

    if ( cm_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_CM_START], control->streams[5] );
#endif

        blocks = workspace->d_workspace->H.n_max / DEF_BLOCK_SIZE
            + (workspace->d_workspace->H.n_max % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        /* update num. rows in matrix for this GPU */
        workspace->d_workspace->H.n = system->n;

        Cuda_Init_Sparse_Matrix_Indices( system, &workspace->d_workspace->H,
                control->streams[5] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->streams[5], control->stream_events[SE_INIT_DIST_DONE], 0 );
        }

        if ( workspace->d_workspace->H.format == SYM_HALF_MATRIX )
        {
            if ( control->tabulate <= 0 )
            {
                k_init_cm_half_fs <<< blocks, DEF_BLOCK_SIZE, 0, control->streams[5] >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp,
                      *(workspace->d_workspace), (control_params *) control->d_control_params,
                      *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
                      system->d_max_cm_entries, system->d_realloc_cm_entries );
            }
            else
            {
                k_init_cm_half_fs_tab <<< blocks, DEF_BLOCK_SIZE, 0, control->streams[5] >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp,
                      *(workspace->d_workspace), (control_params *) control->d_control_params,
                      *(lists[FAR_NBRS]), workspace->d_LR, system->reax_param.num_atom_types,
                      system->d_max_cm_entries, system->d_realloc_cm_entries );
            }
        }
        else
        {
            if ( control->tabulate <= 0 )
            {
//                k_init_cm_full_fs <<< blocks, DEF_BLOCK_SIZE, 0, control->streams[5] >>>
//                    ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp,
//                      *(workspace->d_workspace), (control_params *) control->d_control_params,
//                      *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
//                      system->d_max_cm_entries, system->d_realloc_cm_entries );

                blocks = workspace->d_workspace->H.n_max * 32 / DEF_BLOCK_SIZE
                    + (workspace->d_workspace->H.n_max * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

                k_init_cm_full_fs_opt <<< blocks, DEF_BLOCK_SIZE,
                                      sizeof(hipcub::WarpScan<int>::TempStorage) * (DEF_BLOCK_SIZE / 32),
                                      control->streams[5] >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp,
                      *(workspace->d_workspace), (control_params *) control->d_control_params,
                      *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
                      system->d_max_cm_entries, system->d_realloc_cm_entries );
            }
            else
            {
                k_init_cm_full_fs_tab <<< blocks, DEF_BLOCK_SIZE, 0,
                                      control->streams[5] >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp,
                      *(workspace->d_workspace), (control_params *) control->d_control_params,
                      *(lists[FAR_NBRS]), workspace->d_LR, system->reax_param.num_atom_types,
                      system->d_max_cm_entries, system->d_realloc_cm_entries );
            }
        }
        cudaCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_CM_STOP], control->streams[5] );
#endif
    }

    if ( bonds_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_BOND_START], control->streams[1] );
#endif

        blocks = system->total_cap / DEF_BLOCK_SIZE
            + ((system->total_cap % DEF_BLOCK_SIZE == 0 ) ? 0 : 1);

        Cuda_Init_Bond_Indices( system, lists[BONDS], control->streams[1] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->streams[1], control->stream_events[SE_INIT_DIST_DONE], 0 );
        }

//        k_init_bonds <<< control->blocks_n, control->block_size_n, 0, control->streams[1] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              system->reax_param.d_tbp, *(workspace->d_workspace),
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), *(lists[BONDS]),
//              system->n, system->N, system->reax_param.num_atom_types,
//              system->d_max_bonds, system->d_realloc_bonds );
//        cudaCheckError( );

        blocks = system->N * 32 / DEF_BLOCK_SIZE
            + (system->N * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        k_init_bonds_opt <<< blocks, DEF_BLOCK_SIZE,
                     (sizeof(hipcub::WarpScan<int>::TempStorage)
                      + sizeof(hipcub::WarpReduce<double>::TempStorage)) * (DEF_BLOCK_SIZE / 32),
                     control->streams[1] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              system->reax_param.d_tbp, *(workspace->d_workspace),
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), *(lists[BONDS]),
              system->n, system->N, system->reax_param.num_atom_types,
              system->d_max_bonds, system->d_realloc_bonds );
        cudaCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_BOND_STOP], control->streams[1] );
#endif
    }

    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && hbonds_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_HBOND_START], control->streams[2] );
#endif

        Cuda_Init_HBond_Indices( system, workspace, lists[HBONDS],
                control->streams[2] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->streams[2], control->stream_events[SE_INIT_DIST_DONE], 0 );
        }

//        k_init_hbonds <<< control->blocks_n, control->block_size_n, 0, control->streams[2] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), *(lists[HBONDS]),
//              system->n, system->N, system->reax_param.num_atom_types,
//              system->d_max_hbonds, system->d_realloc_hbonds );
//        cudaCheckError( );

        blocks = system->N * 32 / DEF_BLOCK_SIZE
            + (system->N * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        k_init_hbonds_opt <<< blocks, DEF_BLOCK_SIZE,
                          sizeof(hipcub::WarpScan<int>::TempStorage) * (DEF_BLOCK_SIZE / 32),
                          control->streams[2] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), *(lists[HBONDS]),
              system->n, system->N, system->reax_param.num_atom_types,
              system->d_max_hbonds, system->d_realloc_hbonds );
        cudaCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_HBOND_STOP], control->streams[2] );
#endif
    }

    /* check reallocation flags on device */
    if ( cm_done == FALSE )
    {
        sCudaMemcpyAsync( &realloc_cm, system->d_realloc_cm_entries, sizeof(int), 
                hipMemcpyDeviceToHost, control->streams[5], __FILE__, __LINE__ );
    }
    else
    {
        realloc_cm = FALSE;
    }
    if ( bonds_done == FALSE )
    {
        sCudaMemcpyAsync( &realloc_bonds, system->d_realloc_bonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->streams[1], __FILE__, __LINE__ );
    }
    else
    {
        realloc_bonds = FALSE;
    }
    if ( hbonds_done == FALSE )
    {
        sCudaMemcpyAsync( &realloc_hbonds, system->d_realloc_hbonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->streams[2], __FILE__, __LINE__ );
    }
    else
    {
        realloc_hbonds = FALSE;
    }

    hipStreamSynchronize( control->streams[0] );
    hipStreamSynchronize( control->streams[5] );
    hipStreamSynchronize( control->streams[1] );
    hipStreamSynchronize( control->streams[2] );

    ret = (realloc_cm == FALSE && realloc_bonds == FALSE && realloc_hbonds == FALSE
            ? SUCCESS : FAILURE);

    if ( realloc_cm == FALSE )
    {
        cm_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else
    {
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_CM_START],
                control->time_events[TE_INIT_CM_STOP] ); 
        data->timing.init_cm += (real) (time_elapsed / 1000.0);
    }
#endif
    if ( realloc_bonds == FALSE )
    {
        bonds_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else
    {
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_BOND_START],
                control->time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);
    }
#endif
    if ( realloc_hbonds == FALSE )
    {
        hbonds_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else
    {
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_HBOND_START],
                control->time_events[TE_INIT_HBOND_STOP] ); 
        data->timing.init_hbond += (real) (time_elapsed / 1000.0);
    }
#endif

    if ( ret == SUCCESS )
    {
#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_BOND_START],
                control->time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);

        hipEventRecord( control->time_events[TE_INIT_BOND_START], control->streams[1] );
#endif

        k_update_sym_dbond_indices <<< control->blocks_n, control->block_size_n,
                                   0, control->streams[1] >>> 
            ( *(lists[BONDS]), system->N );
        cudaCheckError( );

        hipEventRecord( control->stream_events[SE_INIT_BOND_DONE], control->streams[1] );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_BOND_STOP], control->streams[1] );
#endif

#if !defined(CUDA_ACCUM_ATOMIC)
        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
#if defined(LOG_PERFORMANCE)
            hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_HBOND_START],
                    control->time_events[TE_INIT_HBOND_STOP] ); 
            data->timing.init_hbond += (real) (time_elapsed / 1000.0);

            hipEventRecord( control->time_events[TE_INIT_HBOND_START], control->streams[2] );
#endif

            blocks = system->N * 32 / DEF_BLOCK_SIZE
                + (system->N * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

            /* make hbond_list symmetric */
            k_update_sym_hbond_indices_opt <<< blocks, DEF_BLOCK_SIZE,
                                           0, control->streams[2] >>>
                ( system->d_my_atoms, *(lists[HBONDS]), system->N );
            cudaCheckError( );

#if defined(LOG_PERFORMANCE)
            hipEventRecord( control->time_events[TE_INIT_HBOND_STOP], control->streams[2] );
#endif
        }
#endif

        dist_done = FALSE;
        cm_done = FALSE;
        bonds_done = FALSE;
        hbonds_done = FALSE;
    }
    else
    {
        Cuda_Estimate_Storages( system, control, data, workspace, lists,
               realloc_cm, realloc_bonds, realloc_hbonds,
               data->step - data->prev_steps );

        /* schedule reallocations after updating allocation sizes */
        workspace->d_workspace->realloc.cm = realloc_cm;
        workspace->d_workspace->realloc.bonds = realloc_bonds;
        workspace->d_workspace->realloc.hbonds = realloc_hbonds;
    }

    return ret;
}


int Cuda_Init_Forces_No_Charges( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    int renbr, blocks, ret, realloc_bonds, realloc_hbonds;
    static int dist_done = FALSE, bonds_done = FALSE, hbonds_done = FALSE;
#if defined(LOG_PERFORMANCE)
    float time_elapsed;
#endif

    renbr = (data->step - data->prev_steps) % control->reneighbor == 0 ? TRUE : FALSE;

    /* reset reallocation flags on device */
    if ( bonds_done == FALSE )
    {
        sCudaMemsetAsync( system->d_realloc_bonds, FALSE, sizeof(int), 
                control->streams[1], __FILE__, __LINE__ );
    }
    if ( hbonds_done == FALSE )
    {
        sCudaMemsetAsync( system->d_realloc_hbonds, FALSE, sizeof(int), 
                control->streams[2], __FILE__, __LINE__ );
    }

    if ( renbr == FALSE && dist_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_DIST_START], control->streams[0] );
#endif

//        k_init_dist <<< control->blocks_n, control->block_size_n, 0, control->streams[0] >>>
//            ( system->d_my_atoms, *(lists[FAR_NBRS]), system->N );

        blocks = system->N * 32 / DEF_BLOCK_SIZE
            + (system->N * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        k_init_dist_opt <<< blocks, DEF_BLOCK_SIZE, 0, control->streams[0] >>>
            ( system->d_my_atoms, *(lists[FAR_NBRS]), system->N );
        cudaCheckError( );

        hipEventRecord( control->stream_events[SE_INIT_DIST_DONE], control->streams[0] );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_DIST_STOP], control->streams[0] );
#endif

        dist_done = TRUE;
    }

    if ( bonds_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_BOND_START], control->streams[1] );
#endif

        blocks = system->total_cap / DEF_BLOCK_SIZE
            + ((system->total_cap % DEF_BLOCK_SIZE == 0 ) ? 0 : 1);

        Cuda_Init_Bond_Indices( system, lists[BONDS], control->streams[1] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->streams[1], control->stream_events[SE_INIT_DIST_DONE], 0 );
        }

//        k_init_bonds <<< control->blocks_n, control->block_size_n, 0, control->streams[1] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              system->reax_param.d_tbp, *(workspace->d_workspace),
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), *(lists[BONDS]),
//              system->n, system->N, system->reax_param.num_atom_types,
//              system->d_max_bonds, system->d_realloc_bonds );

        blocks = control->block_size_n * 32 / DEF_BLOCK_SIZE
            + (control->block_size_n * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        k_init_bonds_opt <<< blocks, DEF_BLOCK_SIZE,
                     sizeof(hipcub::WarpScan<int>::TempStorage) * (DEF_BLOCK_SIZE / 32),
                     control->streams[1] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              system->reax_param.d_tbp, *(workspace->d_workspace),
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), *(lists[BONDS]),
              system->n, system->N, system->reax_param.num_atom_types,
              system->d_max_bonds, system->d_realloc_bonds );
        cudaCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_BOND_STOP], control->streams[1] );
#endif
    }

    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && hbonds_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_HBOND_START], control->streams[2] );
#endif

        Cuda_Init_HBond_Indices( system, workspace, lists[HBONDS],
                control->streams[2] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->streams[2], control->stream_events[SE_INIT_DIST_DONE], 0 );
        }

//        k_init_hbonds <<< control->blocks_n, control->block_size_n, 0, control->streams[2] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), *(lists[HBONDS]),
//              system->n, system->N, system->reax_param.num_atom_types,
//              system->d_max_hbonds, system->d_realloc_hbonds );
//        cudaCheckError( );

        blocks = system->N * 32 / DEF_BLOCK_SIZE
            + (system->N * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        k_init_hbonds_opt <<< blocks, DEF_BLOCK_SIZE,
                          sizeof(hipcub::WarpScan<int>::TempStorage) * (DEF_BLOCK_SIZE / 32),
                          control->streams[2] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), *(lists[HBONDS]),
              system->n, system->N, system->reax_param.num_atom_types,
              system->d_max_hbonds, system->d_realloc_hbonds );
        cudaCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_HBOND_STOP], control->streams[2] );
#endif
    }

    /* check reallocation flags on device */
    if ( bonds_done == FALSE )
    {
        sCudaMemcpyAsync( &realloc_bonds, system->d_realloc_bonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->streams[1], __FILE__, __LINE__ );
    }
    else
    {
        realloc_bonds = FALSE;
    }
    if ( hbonds_done == FALSE )
    {
        sCudaMemcpyAsync( &realloc_hbonds, system->d_realloc_hbonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->streams[2], __FILE__, __LINE__ );
    }
    else
    {
        realloc_hbonds = FALSE;
    }

    hipStreamSynchronize( control->streams[0] );
    hipStreamSynchronize( control->streams[1] );
    hipStreamSynchronize( control->streams[2] );

    ret = (realloc_bonds == FALSE && realloc_hbonds == FALSE
            ? SUCCESS : FAILURE);

    if ( realloc_bonds == FALSE )
    {
        bonds_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else
    {
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_BOND_START],
                control->time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);
    }
#endif
    if ( realloc_hbonds == FALSE )
    {
        hbonds_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else
    {
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_HBOND_START],
                control->time_events[TE_INIT_HBOND_STOP] ); 
        data->timing.init_hbond += (real) (time_elapsed / 1000.0);
    }
#endif

    if ( ret == SUCCESS )
    {
#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_BOND_START],
                control->time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);

        hipEventRecord( control->time_events[TE_INIT_BOND_START], control->streams[1] );
#endif

        k_update_sym_dbond_indices <<< control->blocks_n, control->block_size_n,
                                   0, control->streams[1] >>> 
            ( *(lists[BONDS]), system->N );
        cudaCheckError( );

        hipEventRecord( control->stream_events[SE_INIT_BOND_DONE], control->streams[1] );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->time_events[TE_INIT_BOND_STOP], control->streams[1] );
#endif

#if !defined(CUDA_ACCUM_ATOMIC)
        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
#if defined(LOG_PERFORMANCE)
            hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_HBOND_START],
                    control->time_events[TE_INIT_HBOND_STOP] ); 
            data->timing.init_hbond += (real) (time_elapsed / 1000.0);

            hipEventRecord( control->time_events[TE_INIT_HBOND_START], control->streams[2] );
#endif

            blocks = system->N * 32 / DEF_BLOCK_SIZE
                + (system->N * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

            /* make hbond_list symmetric */
            k_update_sym_hbond_indices_opt <<< blocks, DEF_BLOCK_SIZE,
                                           0, control->streams[2] >>>
                ( system->d_my_atoms, *(lists[HBONDS]), system->N );
            cudaCheckError( );

#if defined(LOG_PERFORMANCE)
            hipEventRecord( control->time_events[TE_INIT_HBOND_STOP], control->streams[2] );
#endif
        }
#endif

        dist_done = FALSE;
        bonds_done = FALSE;
        hbonds_done = FALSE;
    }
    else
    {
        Cuda_Estimate_Storages( system, control, data, workspace, lists,
               FALSE, realloc_bonds, realloc_hbonds,
               data->step - data->prev_steps );

        /* schedule reallocations after updating allocation sizes */
        workspace->d_workspace->realloc.bonds = realloc_bonds;
        workspace->d_workspace->realloc.hbonds = realloc_hbonds;
    }

    return ret;
}


int Cuda_Compute_Bonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control )
{
    int ret;
    static int compute_bonded_part1 = FALSE;

    ret = SUCCESS;

    if ( compute_bonded_part1 == FALSE )
    {
        Cuda_Compute_Bond_Orders( system, control, data, workspace, lists,
                out_control );

        Cuda_Compute_Bonds( system, control, data, workspace, lists,
                out_control );

        Cuda_Compute_Atom_Energy( system, control, data, workspace, lists,
                out_control );

        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
            Cuda_Compute_Hydrogen_Bonds( system, control, data, workspace,
                    lists, out_control );
        }

        compute_bonded_part1 = TRUE;
    }

    ret = Cuda_Compute_Valence_Angles( system, control, data, workspace,
            lists, out_control );

    if ( ret == SUCCESS )
    {
        Cuda_Compute_Torsion_Angles( system, control, data, workspace, lists,
                out_control );

        compute_bonded_part1 = FALSE;
    }

    return ret;
}


static void Cuda_Compute_Total_Force( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, mpi_datatypes *mpi_data )
{
    rvec *f;

    smalloc_check( &workspace->host_scratch, &workspace->host_scratch_size,
            sizeof(rvec) * system->N, TRUE, SAFE_ZONE,
            __FILE__, __LINE__ );
    f = (rvec *) workspace->host_scratch;
    memset( f, 0, sizeof(rvec) * system->N );

    Cuda_Total_Forces_Part1( system, control, data, workspace, lists );

    /* now all forces are computed to their partially-final values
     * based on the neighbors information each processor has had.
     * final values of force on each atom needs to be computed by adding up
     * all partially-final pieces */
    sCudaMemcpyAsync( f, workspace->d_workspace->f, sizeof(rvec) * system->N,
            hipMemcpyDeviceToHost, control->streams[0], __FILE__, __LINE__ );
    hipStreamSynchronize( control->streams[0] );

    Coll( system, mpi_data, f, RVEC_PTR_TYPE, mpi_data->mpi_rvec );

    sCudaMemcpyAsync( workspace->d_workspace->f, f, sizeof(rvec) * system->N,
            hipMemcpyHostToDevice, control->streams[0], __FILE__, __LINE__ );

    Cuda_Total_Forces_Part2( system, control, workspace );
}


extern "C" int Cuda_Compute_Forces( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace, reax_list **lists,
        output_controls *out_control, mpi_datatypes *mpi_data )
{
    int i, charge_flag, ret;
    static int init_forces_done = FALSE, nonbonded_forces_part1_done = FALSE;
#if defined(LOG_PERFORMANCE)
    float time_elapsed, time_elapsed2, time_elapsed3;
    float time_elapsed4, time_elapsed5, time_elapsed6;
    float time_elapsed7, time_elapsed8, time_elapsed9;
#endif

    ret = SUCCESS;

    if ( control->charge_freq > 0
            && (data->step - data->prev_steps) % control->charge_freq == 0 )
    {
        charge_flag = TRUE;
    }
    else
    {
        charge_flag = FALSE;
    }

    if ( init_forces_done == FALSE )
    {
        if ( charge_flag == TRUE )
        {
            ret = Cuda_Init_Forces( system, control, data,
                    workspace, lists, out_control );
        }
        else
        {
            ret = Cuda_Init_Forces_No_Charges( system, control, data,
                    workspace, lists, out_control );
        }

        if ( ret == SUCCESS )
        {
            init_forces_done = TRUE;
        }
    }

    if ( nonbonded_forces_part1_done == FALSE )
    {
        Cuda_Compute_NonBonded_Forces_Part1( system, control, data, workspace,
                lists, out_control );

        nonbonded_forces_part1_done = TRUE;
    }

    if ( ret == SUCCESS )
    {
        ret = Cuda_Compute_Bonded_Forces( system, control, data,
                workspace, lists, out_control );
    }

    if ( ret == SUCCESS )
    {
#if defined(LOG_PERFORMANCE)
        real time;

        time = Get_Time( );
#endif

        if ( charge_flag == TRUE )
        {
            Cuda_Compute_Charges( system, control, data,
                    workspace, out_control, mpi_data, control->streams[5] );
        }
    
#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm );
#endif

        Cuda_Compute_NonBonded_Forces_Part2( system, control, data, workspace,
                lists, out_control );

        for ( i = 0; i < MAX_CUDA_STREAMS; ++i )
        {
            hipStreamSynchronize( control->streams[i] );
        }

        Cuda_Compute_Total_Force( system, control, data, workspace, lists, mpi_data );

#if defined(LOG_PERFORMANCE)
        if ( (data->step - data->prev_steps) % control->reneighbor == 0 )
        {
            hipEventElapsedTime( &time_elapsed, control->time_events[TE_NBRS_START],
                    control->time_events[TE_NBRS_STOP] ); 
            data->timing.nbrs += (real) (time_elapsed / 1000.0);

            hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_CM_START],
                    control->time_events[TE_INIT_CM_STOP] ); 
            hipEventElapsedTime( &time_elapsed2, control->time_events[TE_INIT_CM_START],
                    control->time_events[TE_INIT_BOND_STOP] ); 
            if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
            {
                hipEventElapsedTime( &time_elapsed3, control->time_events[TE_INIT_CM_START],
                        control->time_events[TE_INIT_HBOND_STOP] ); 
            }
            else
            {
                time_elapsed4 = 0.0;
            }
            hipEventElapsedTime( &time_elapsed4, control->time_events[TE_INIT_BOND_START],
                    control->time_events[TE_INIT_CM_STOP] ); 
            hipEventElapsedTime( &time_elapsed5, control->time_events[TE_INIT_BOND_START],
                    control->time_events[TE_INIT_BOND_STOP] ); 
            if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
            {
                hipEventElapsedTime( &time_elapsed6, control->time_events[TE_INIT_BOND_START],
                        control->time_events[TE_INIT_HBOND_STOP] ); 
                hipEventElapsedTime( &time_elapsed7, control->time_events[TE_INIT_HBOND_START],
                        control->time_events[TE_INIT_CM_STOP] ); 
                hipEventElapsedTime( &time_elapsed8, control->time_events[TE_INIT_HBOND_START],
                        control->time_events[TE_INIT_BOND_STOP] ); 
                hipEventElapsedTime( &time_elapsed9, control->time_events[TE_INIT_HBOND_START],
                        control->time_events[TE_INIT_HBOND_STOP] ); 
            }
            else
            {
                time_elapsed6 = 0.0;
                time_elapsed7 = 0.0;
                time_elapsed8 = 0.0;
                time_elapsed9 = 0.0;
            }
            data->timing.init_forces += (real) MAX3( 
                MAX3(time_elapsed / 1000.0, time_elapsed2 / 1000.0, time_elapsed3 / 1000.0),
                MAX3(time_elapsed4 / 1000.0, time_elapsed5 / 1000.0, time_elapsed6 / 1000.0),
                MAX3(time_elapsed7 / 1000.0, time_elapsed8 / 1000.0, time_elapsed9 / 1000.0) );
        }
        else
        {
            hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_DIST_START],
                    control->time_events[TE_INIT_CM_STOP] ); 
            hipEventElapsedTime( &time_elapsed2, control->time_events[TE_INIT_DIST_START],
                    control->time_events[TE_INIT_BOND_STOP] ); 
            if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
            {
                hipEventElapsedTime( &time_elapsed3, control->time_events[TE_INIT_DIST_START],
                        control->time_events[TE_INIT_HBOND_STOP] ); 
            }
            else
            {
                time_elapsed3 = 0.0;
            }
            data->timing.init_forces += (real) MAX3(time_elapsed / 1000.0, time_elapsed2 / 1000.0,
                    time_elapsed3 / 1000.0);

            hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_DIST_START],
                    control->time_events[TE_INIT_DIST_STOP] ); 
            data->timing.init_dist += (real) (time_elapsed / 1000.0);
        }

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_CM_START],
                control->time_events[TE_INIT_CM_STOP] ); 
        data->timing.init_cm += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_BOND_START],
                control->time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);

        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
            hipEventElapsedTime( &time_elapsed, control->time_events[TE_INIT_HBOND_START],
                    control->time_events[TE_INIT_HBOND_STOP] ); 
            data->timing.init_hbond += (real) (time_elapsed / 1000.0);
        }

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_BOND_ORDER_START],
                control->time_events[TE_LPOVUN_STOP] ); 
        hipEventElapsedTime( &time_elapsed2, control->time_events[TE_BOND_ORDER_START],
                control->time_events[TE_BONDS_STOP] ); 
        hipEventElapsedTime( &time_elapsed3, control->time_events[TE_BOND_ORDER_START],
                control->time_events[TE_TORSION_STOP] ); 
        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
            hipEventElapsedTime( &time_elapsed4, control->time_events[TE_BOND_ORDER_START],
                    control->time_events[TE_HBONDS_STOP] ); 
        }
        else
        {
            time_elapsed4 = 0.0;
        }
        data->timing.bonded += (real) MAX(MAX3(time_elapsed / 1000.0, time_elapsed2 / 1000.0,
                time_elapsed3 / 1000.0), time_elapsed4 / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_BOND_ORDER_START],
                control->time_events[TE_BOND_ORDER_STOP] ); 
        data->timing.bond_order += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_BONDS_START],
                control->time_events[TE_BONDS_STOP] ); 
        data->timing.bonds += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_LPOVUN_START],
                control->time_events[TE_LPOVUN_STOP] ); 
        data->timing.lpovun += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_VALENCE_START],
                control->time_events[TE_VALENCE_STOP] ); 
        data->timing.valence += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_TORSION_START],
                control->time_events[TE_TORSION_STOP] ); 
        data->timing.torsion += (real) (time_elapsed / 1000.0);

        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
            hipEventElapsedTime( &time_elapsed, control->time_events[TE_HBONDS_START],
                    control->time_events[TE_HBONDS_STOP] ); 
            data->timing.hbonds += (real) (time_elapsed / 1000.0);
        }

#if !defined(USE_FUSED_VDW_COULOMB)
        hipEventElapsedTime( &time_elapsed, control->time_events[TE_VDW_START],
                control->time_events[TE_VDW_STOP] ); 
        data->timing.nonb += (real) (time_elapsed / 1000.0);
#endif

        hipEventElapsedTime( &time_elapsed, control->time_events[TE_COULOMB_START],
                control->time_events[TE_COULOMB_STOP] ); 
        data->timing.nonb += (real) (time_elapsed / 1000.0);
#endif

        init_forces_done = FALSE;
        nonbonded_forces_part1_done = FALSE;
    }

    return ret;
}

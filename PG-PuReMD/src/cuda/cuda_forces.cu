#include "hip/hip_runtime.h"

#include "cuda_forces.h"

#include "cuda_bonds.h"
#include "cuda_bond_orders.h"
#include "cuda_charges.h"
#include "cuda_helpers.h"
#include "cuda_hydrogen_bonds.h"
#include "cuda_lin_alg.h"
#include "cuda_list.h"
#include "cuda_multi_body.h"
#include "cuda_neighbors.h"
#include "cuda_nonbonded.h"
#include "cuda_reduction.h"
#include "cuda_torsion_angles.h"
#include "cuda_utils.h"
#include "cuda_valence_angles.h"
#include "cuda_validation.h"

#include "../basic_comm.h"
#include "../forces.h"
#include "../index_utils.h"
#include "../tool_box.h"
#include "../vector.h"


CUDA_GLOBAL void k_disable_hydrogen_bonding( control_params *control )
{
    control->hbond_cut = 0.0;
}


CUDA_GLOBAL void k_estimate_storages( reax_atom *my_atoms, 
        single_body_parameters *sbp, two_body_parameters *tbp,
        control_params *control, reax_list far_nbrs, 
        int num_atom_types, int n, int N, int Hcap, int total_cap,
        int *cm_entries, int *max_cm_entries, int *realloc_cm_entries,
        int *bonds, int *max_bonds, int *realloc_bonds,
        int *hbonds, int *max_hbonds, int *realloc_hbonds )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb;
    int local;
    real cutoff;
    real r_ij, r2; 
    real C12, C34, C56;
    real BO, BO_s, BO_pi, BO_pi2;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= total_cap )
    {
        return;
    }

    if ( i < N )
    {
        atom_i = &(my_atoms[i]);
        type_i = atom_i->type;
        start_i = Dev_Start_Index( i, &far_nbrs );
        end_i = Dev_End_Index( i, &far_nbrs );
        sbp_i = &(sbp[type_i]);

        if ( i < n )
        { 
            local = TRUE;
            cutoff = control->nonb_cut;
            ++cm_entries[i];
//            ihb = sbp_i->p_hbond;
        }   
        else
        {
            local = FALSE;
            cutoff = control->bond_cut;
//            ihb = NON_H_BONDING_ATOM; 
        } 

        ihb = NON_H_BONDING_ATOM; 

        for ( pj = start_i; pj < end_i; ++pj )
        { 
            nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
            j = nbr_pj->nbr;
            atom_j = &(my_atoms[j]);

            if ( nbr_pj->d <= control->nonb_cut )
            {
                type_j = my_atoms[j].type;
                sbp_j = &(sbp[type_j]);
                ihb = sbp_i->p_hbond;
                jhb = sbp_j->p_hbond;

                if ( local == TRUE )
                {
                    if ( i < j && (j < n || atom_i->orig_id < atom_j->orig_id) )
                    {
                        ++cm_entries[i];
                    }
                    else if ( i > j && (j < n || atom_j->orig_id > atom_i->orig_id) )
                    {
                        ++cm_entries[i];
                    }
                }
                else
                {
                    if ( i > j && j < n && atom_j->orig_id < atom_i->orig_id )
                    {
                        ++cm_entries[i];
                    }
                }

                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( control->hbond_cut > 0.0 && nbr_pj->d <= control->hbond_cut 
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM && i >= n && j < n )
                {
                    ++hbonds[i];
                }

//                if ( i >= n )
//                {
//                    ihb = NON_H_BONDING_ATOM;
//                }
            }

            if ( nbr_pj->d <= cutoff )
            {
                type_j = my_atoms[j].type;
                r_ij = nbr_pj->d;
                sbp_j = &(sbp[type_j]);
                twbp = &(tbp[ index_tbp(type_i ,type_j, num_atom_types) ]);

                if ( local == TRUE )
                {
                    /* atom i: H atom OR H bonding atom, native */
                    if ( control->hbond_cut > 0.0 && (ihb == H_ATOM || ihb == H_BONDING_ATOM) &&
                            nbr_pj->d <= control->hbond_cut )
                    {
                        jhb = sbp_j->p_hbond;

                        /* atom i: H atom, native
                         * atom j: H bonding atom */
                        if( ihb == H_ATOM && jhb == H_BONDING_ATOM )
                        {
                            ++hbonds[i];
                        }
                        /* atom i: H bonding atom, native
                         * atom j: H atom, native */
                        else if( ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                        {
                            ++hbonds[i];
                        }
                    }
                }

                /* uncorrected bond orders */
                if ( nbr_pj->d <= control->bond_cut )
                {
                    r2 = SQR( r_ij );

                    if ( sbp_i->r_s > 0.0 && sbp_j->r_s > 0.0 )
                    {
                        C12 = twbp->p_bo1 * POW( r_ij / twbp->r_s, twbp->p_bo2 );
                        BO_s = (1.0 + control->bo_cut) * EXP( C12 );
                    }
                    else
                    {
                        BO_s = C12 = 0.0;
                    }

                    if ( sbp_i->r_pi > 0.0 && sbp_j->r_pi > 0.0 )
                    {
                        C34 = twbp->p_bo3 * POW( r_ij / twbp->r_p, twbp->p_bo4 );
                        BO_pi = EXP( C34 );
                    }
                    else
                    {
                        BO_pi = C34 = 0.0;
                    }

                    if ( sbp_i->r_pi_pi > 0.0 && sbp_j->r_pi_pi > 0.0 )
                    {
                        C56 = twbp->p_bo5 * POW( r_ij / twbp->r_pp, twbp->p_bo6 );
                        BO_pi2= EXP( C56 );
                    }
                    else
                    {
                        BO_pi2 = C56 = 0.0;
                    }

                    /* initially BO values are the uncorrected ones, page 1 */
                    BO = BO_s + BO_pi + BO_pi2;

                    if ( BO >= control->bo_cut )
                    {
                        ++bonds[i];
//                        atomicAdd( bonds + j, 1 );
                    }
                }
            }
        }
    }
    else
    {
        bonds[i] = MIN_BONDS;
        hbonds[i] = 0;
    }

    if ( bonds[i] > max_bonds[i] )
    {
        max_bonds[i] = MAX( (int)(bonds[i] * 2), MIN_BONDS );
        *realloc_bonds = TRUE;
    }

    if ( hbonds[i] > max_hbonds[i] )
    {
        max_hbonds[i] = MAX( (int)(hbonds[i] * SAFE_ZONE), MIN_HBONDS );
        *realloc_hbonds = TRUE;
    }

    if ( cm_entries[i] > max_cm_entries[i] )
    {
        max_cm_entries[i] = MAX( (int)(cm_entries[i] * SAFE_ZONE), MIN_CM_ENTRIES );
        *realloc_cm_entries = TRUE;
    }
}


/* Copy num. of bonds/hbonds per atom into atom structs (used for MPI messaging)
 *
 * my_atoms: atom structs
 * N: num. of atoms (native + ghost)
 * hbonds: num. of hydrogen bonds per atom
 * bonds: num. of bonds per atom */
CUDA_GLOBAL void k_init_system_atoms( reax_atom *my_atoms, int N, 
        int *hbonds, int *bonds )
{
    int i;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    my_atoms[i].num_bonds = bonds[i];
    my_atoms[i].num_hbonds = hbonds[i];
}


int Cuda_Estimate_Storages( reax_system *system, control_params *control, 
        reax_list **lists, sparse_matrix *H, int step )
{
    int i, ret, ret_bonds, ret_hbonds, ret_cm;
    int blocks = 0;

    ret = SUCCESS;

    /* careful: this wrapper around hipMemset(...) performs a byte-wide assignment
     * to the provided literal */
    cuda_memset( system->d_realloc_bonds, FALSE, sizeof(int), 
            "Cuda_Estimate_Storages::d_realloc_bonds" );
    cuda_memset( system->d_realloc_hbonds, FALSE, sizeof(int), 
            "Cuda_Estimate_Storages::d_realloc_hbonds" );
    cuda_memset( system->d_realloc_cm_entries, FALSE, sizeof(int), 
            "Cuda_Estimate_Storages::d_realloc_cm_entries" );
    cuda_memset( system->d_bonds, 0, system->total_cap * sizeof(int), 
            "Cuda_Estimate_Storages::d_bonds" );
    cuda_memset( system->d_hbonds, 0, system->total_cap * sizeof(int), 
            "Cuda_Estimate_Storages::d_hbonds" );
    cuda_memset( system->d_cm_entries, 0, system->total_cap * sizeof(int), 
            "Cuda_Estimate_Storages::d_cm_entries" );
 
    blocks = (int)CEIL( (real)system->total_cap / ST_BLOCK_SIZE );

    k_estimate_storages <<< blocks, ST_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp, 
          (control_params *)control->d_control_params,
          *(*dev_lists + FAR_NBRS), system->reax_param.num_atom_types,
          system->n, system->N, system->Hcap, system->total_cap,
          system->d_cm_entries, system->d_max_cm_entries, system->d_realloc_cm_entries,
          system->d_bonds, system->d_max_bonds, system->d_realloc_bonds,
          system->d_hbonds, system->d_max_hbonds, system->d_realloc_hbonds );
    hipDeviceSynchronize( );
    cudaCheckError( );

    /* check reallocation flags on device */
    copy_host_device( &ret_bonds, system->d_realloc_bonds, sizeof(int), 
            hipMemcpyDeviceToHost, "Cuda_Estimate_Storages::d_realloc_bonds" );
    copy_host_device( &ret_hbonds, system->d_realloc_hbonds, sizeof(int), 
            hipMemcpyDeviceToHost, "Cuda_Estimate_Storages::d_realloc_hbonds" );
    copy_host_device( &ret_cm, system->d_realloc_cm_entries, sizeof(int), 
            hipMemcpyDeviceToHost, "Cuda_Estimate_Storages::d_realloc_cm_entries" );

    if ( ret_bonds == TRUE )
    {
        Cuda_Reduction_Sum( system->d_max_bonds, system->d_total_bonds,
                system->total_cap );

        copy_host_device( &(system->total_bonds), system->d_total_bonds, sizeof(int), 
                hipMemcpyDeviceToHost, "Cuda_Estimate_Storages::d_total_bonds" );

        if ( step > 0 )
        {
            dev_workspace->realloc.bonds = TRUE;
        }
        ret = FAILURE;
    }

    if ( system->numH > 0 && control->hbond_cut > 0.0 && ret_hbonds == TRUE )
    {
        Cuda_Reduction_Sum( system->d_max_hbonds, system->d_total_hbonds,
                system->total_cap );

        copy_host_device( &(system->total_hbonds), system->d_total_hbonds, sizeof(int), 
                hipMemcpyDeviceToHost, "Cuda_Estimate_Storages::d_total_hbonds" );

        if ( step > 0 )
        {
            dev_workspace->realloc.hbonds = TRUE;
        }
        ret = FAILURE;
    }
    else
    {
        /* if number of hydrogen atoms is 0, disable hydrogen bond functionality */
        if ( system->numH == 0 && step == 0 )
        {
            fprintf( stderr, "[WARNING] DISABLING HYDROGEN BONDS\n" );
            control->hbond_cut = 0.0;
            k_disable_hydrogen_bonding <<< 1, 1 >>> ( (control_params *)control->d_control_params );
        }
    }

    if ( ret_cm == TRUE )
    {
        Cuda_Reduction_Sum( system->d_max_cm_entries, system->d_total_cm_entries, system->total_cap );

        copy_host_device( &(system->total_cm_entries), system->d_total_cm_entries, sizeof(int),
                hipMemcpyDeviceToHost, "d_total_cm_entries" );

        if ( step > 0 )
        {
            dev_workspace->realloc.cm = TRUE;
        }
        ret = FAILURE;
    }

#if defined(DEBUG)
    fprintf( stderr, "p:%d -->\n", system->my_rank );
    fprintf( stderr, " TOTAL DEVICE BOND COUNT: %d \n", system->total_bonds );
    fprintf( stderr, " TOTAL DEVICE HBOND COUNT: %d \n", system->total_hbonds );
    fprintf( stderr, " TOTAL DEVICE SPARSE COUNT: %d \n", system->total_cm_entries );
#endif

    k_init_system_atoms <<< blocks, ST_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->N, system->d_hbonds, system->d_bonds );
    hipDeviceSynchronize( );
    cudaCheckError( );

    return ret;
}


int Cuda_Estimate_Storage_Three_Body( reax_system *system, control_params *control, 
        int step, reax_list **lists, int *thbody )
{
    int ret;

    ret = SUCCESS;
    cuda_memset( thbody, 0, system->total_bonds * sizeof(int), "scratch::thbody" );

    Estimate_Cuda_Valence_Angles <<< BLOCKS_N, BLOCK_SIZE >>>
        ( system->d_my_atoms, (control_params *)control->d_control_params, 
          *(*dev_lists + BONDS), system->n, system->N, thbody );
    hipDeviceSynchronize( );
    cudaCheckError( );

    Cuda_Reduction_Sum( thbody, system->d_total_thbodies, system->total_bonds );

    copy_host_device( &(system->total_thbodies), system->d_total_thbodies, sizeof(int),
            hipMemcpyDeviceToHost, "Cuda_Estimate_Storage_Three_Body::d_total_thbodies" );

    if ( step == 0 )
    {
        /* create Three-body list */
        Dev_Make_List( system->total_bonds, system->total_thbodies,
                TYP_THREE_BODY, *dev_lists + THREE_BODIES );
    }

    if ( system->total_thbodies > (*dev_lists + THREE_BODIES)->num_intrs ||
            (*dev_lists + THREE_BODIES)->n < system->total_bonds )
    {
        system->total_thbodies = (*dev_lists + THREE_BODIES)->num_intrs * SAFE_ZONE;
        dev_workspace->realloc.num_3body = system->total_thbodies;
        ret = FAILURE;
    }

    return ret;
}


CUDA_DEVICE real Compute_H( real r, real gamma, real *ctap )
{
    real taper, dr3gamij_1, dr3gamij_3;

    taper = ctap[7] * r + ctap[6];
    taper = taper * r + ctap[5];
    taper = taper * r + ctap[4];
    taper = taper * r + ctap[3];
    taper = taper * r + ctap[2];
    taper = taper * r + ctap[1];
    taper = taper * r + ctap[0];    

    dr3gamij_1 = r * r * r + gamma;
    dr3gamij_3 = POW( dr3gamij_1 , 1.0 / 3.0 );

    return taper * EV_to_KCALpMOL / dr3gamij_3;
}


CUDA_DEVICE real Compute_tabH( LR_lookup_table *t_LR, real r_ij, int ti, int tj, int num_atom_types )
{
    int r, tmin, tmax;
    real val, dif, base;
    LR_lookup_table *t; 

    tmin = MIN( ti, tj );
    tmax = MAX( ti, tj );
    t = &( t_LR[ index_lr(tmin,tmax, num_atom_types) ] );    

    /* cubic spline interpolation */
    r = (int)(r_ij * t->inv_dx);
    if ( r == 0 )
    {
        ++r;
    }
    base = (real)(r + 1) * t->dx;
    dif = r_ij - base;
    val = ((t->ele[r].d * dif + t->ele[r].c) * dif + t->ele[r].b) * dif
        + t->ele[r].a;
    val *= EV_to_KCALpMOL / C_ele;

    return val;
}


CUDA_GLOBAL void k_print_hbond_info( reax_atom *my_atoms, single_body_parameters *sbp, 
        control_params *control, reax_list hbonds, int N )
{
    int i;
    int type_i;
    int ihb, ihb_top;
    single_body_parameters *sbp_i;
    reax_atom *atom_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    atom_i = &(my_atoms[i]);
    type_i = atom_i->type;
    sbp_i = &(sbp[type_i]);

    if ( control->hbond_cut > 0.0 )
    {
        ihb = sbp_i->p_hbond;
        if ( ihb == H_ATOM  || ihb == H_BONDING_ATOM )
        {
            ihb_top = Dev_Start_Index( atom_i->Hindex, &hbonds );
        }
        else
        {
            ihb_top = -1;
        }
    }

    printf( "atom %6d: ihb = %2d, ihb_top = %2d\n", i, ihb, ihb_top );
}


CUDA_GLOBAL void k_init_forces( reax_atom *my_atoms, single_body_parameters *sbp, 
        two_body_parameters *tbp, storage workspace, control_params *control, 
        reax_list far_nbrs, reax_list bonds, reax_list hbonds, 
        LR_lookup_table *t_LR, int n, int N, int num_atom_types, int renbr )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    int Htop;
    int btop_i, ihb, jhb, ihb_top;
    int local, flag, flag2, flag3;
    real r_ij, cutoff;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    H = &(workspace.H);
    Htop = H->start[i];

    atom_i = &(my_atoms[i]);
    type_i = atom_i->type;
    start_i = Dev_Start_Index( i, &far_nbrs );
    end_i = Dev_End_Index( i, &far_nbrs );
    btop_i = Dev_Start_Index( i, &bonds );
    sbp_i = &(sbp[type_i]);

    if ( i < n )
    {
        local = TRUE;
        cutoff = control->nonb_cut;

        //update bond mark here
        workspace.bond_mark[i] = 0;
    }
    else
    {
        local = FALSE;
        cutoff = control->bond_cut;

        //update bond mark here
        workspace.bond_mark[i] = 1000;
    }

    ihb = NON_H_BONDING_ATOM;
    ihb_top = -1;

    if ( local == TRUE )
    {
        H->entries[Htop].j = i;
        H->entries[Htop].val = sbp_i->eta;
        ++Htop;
    }

    if ( control->hbond_cut > 0.0 )
    {
        ihb = sbp_i->p_hbond;

        if ( ihb == H_ATOM || ihb == H_BONDING_ATOM )
        {
            ihb_top = Dev_Start_Index( atom_i->Hindex, &hbonds );
        }
        else
        {
            ihb_top = -1;
        }
    }

    /* update i-j distance - check if j is within cutoff */
    for ( pj = start_i; pj < end_i; ++pj )
    {
        nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
        j = nbr_pj->nbr;
        atom_j = &(my_atoms[j]);

        if ( renbr )
        {
            if ( nbr_pj->d <= cutoff )
            {
                flag = TRUE;
            }
            else
            {
                flag = FALSE;
            }

            if ( nbr_pj->d <= control->nonb_cut )
            {
                flag2 = TRUE;
            }
            else
            {
                flag2 = FALSE;
            }

        }
        else
        {
            if ( i < j )
            {
                nbr_pj->dvec[0] = atom_j->x[0] - atom_i->x[0];
                nbr_pj->dvec[1] = atom_j->x[1] - atom_i->x[1];
                nbr_pj->dvec[2] = atom_j->x[2] - atom_i->x[2];
                nbr_pj->d = rvec_Norm_Sqr( nbr_pj->dvec );
            }
            else
            {
                nbr_pj->dvec[0] = atom_i->x[0] - atom_j->x[0];
                nbr_pj->dvec[1] = atom_i->x[1] - atom_j->x[1];
                nbr_pj->dvec[2] = atom_i->x[2] - atom_j->x[2];
                nbr_pj->d = rvec_Norm_Sqr( nbr_pj->dvec );
            }

            if ( nbr_pj->d <= SQR( control->nonb_cut ) )
            {
                flag2 = TRUE;
            }
            else
            {
                flag2 = FALSE;
            }

            if ( nbr_pj->d <= SQR( control->nonb_cut ) )
            {
                nbr_pj->d = SQRT( nbr_pj->d );
                flag = TRUE;
            }
            else
            {
                flag = FALSE;
            }
        }
        if ( flag2 == TRUE )
        {
            type_j = atom_j->type;
            sbp_j = &(sbp[type_j]);
            ihb = sbp_i->p_hbond;
            jhb = sbp_j->p_hbond;

            /* atom i: H bonding, ghost
             * atom j: H atom, native */
            if ( control->hbond_cut > 0.0 && nbr_pj->d <= control->hbond_cut
                    && ihb == H_BONDING_ATOM && jhb == H_ATOM && i >= n && j < n ) 
            {
                hbonds.select.hbond_list[ihb_top].nbr = j;
                hbonds.select.hbond_list[ihb_top].scl = -1;
                hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                //CUDA SPECIFIC
                hbonds.select.hbond_list[ihb_top].sym_index = -1;
                rvec_MakeZero( hbonds.select.hbond_list[ihb_top].hb_f );

                ++ihb_top;
            }

            //if ((i < n) || (j < n))
            //if (local == TRUE || ((i >= n) &&(j < n)))

            flag3 = FALSE;
            if ( i < j && i < n && (j < n || atom_i->orig_id < atom_j->orig_id) )
            {
                flag3 = TRUE;
            }
            else if ( i > j && i >= n && j < n && atom_j->orig_id < atom_i->orig_id )
            {
                flag3 = TRUE;
            }
            else if ( i > j && i < n && (j < n || atom_j->orig_id < atom_i->orig_id ) )
            {
                flag3 = TRUE;
            }

            if ( flag3 == TRUE )
            {
                twbp = &(tbp[ index_tbp(type_i,type_j,num_atom_types) ]);
                r_ij = nbr_pj->d;

                //if (renbr) {
                H->entries[Htop].j = j;
                if ( control->tabulate == 0 )
                {
                    H->entries[Htop].val = Compute_H( r_ij,twbp->gamma,workspace.Tap );
                }
                else
                {
                    H->entries[Htop].val = Compute_tabH( t_LR, r_ij, type_i, type_j,num_atom_types );
                }
                //}
                ++Htop;
            }
        }

        if ( flag == TRUE )
        {
            type_j = atom_j->type;
            r_ij = nbr_pj->d;
            sbp_j = &(sbp[type_j]);
            twbp = &(tbp[ index_tbp(type_i, type_j, num_atom_types) ]);

            if ( local == TRUE )
            {
                /* H matrix entry */
//                if( j < n || atom_i->orig_id < atom_j->orig_id ) {//tryQEq||1
//                    H->entries[Htop].j = j;
//                    if( control->tabulate == 0 )
//                        H->entries[Htop].val = Compute_H(r_ij,twbp->gamma,workspace.Tap);
//                    else
//                        H->entries[Htop].val = Compute_tabH(t_LR, r_ij, type_i, type_j,num_atom_types);
//                    ++Htop;
//                } 
//                else if( j < n || atom_i->orig_id > atom_j->orig_id ) {//tryQEq||1
//                    H->entries[Htop].j = j;
//                    if( control->tabulate == 0 )
//                        H->entries[Htop].val = Compute_H(r_ij,twbp->gamma,workspace.Tap);
//                    else
//                        H->entries[Htop].val = Compute_tabH(t_LR, r_ij, type_i, type_j,num_atom_types);
//                    ++Htop;
//                } 
                //bool condition = !((i >= n) && (j >= n));

                /* hydrogen bond lists */
                if ( control->hbond_cut > 0.0 && (ihb == H_ATOM || ihb == H_BONDING_ATOM) &&
                        nbr_pj->d <= control->hbond_cut )
                {
                    jhb = sbp_j->p_hbond;

                    /* atom i: H atom, native
                     * atom j: H bonding atom */
                    if ( ihb == H_ATOM && jhb == H_BONDING_ATOM )
                    {
                        hbonds.select.hbond_list[ihb_top].nbr = j;

                        if ( i < j )
                        {
                            hbonds.select.hbond_list[ihb_top].scl = 1;
                        }
                        else
                        {
                            hbonds.select.hbond_list[ihb_top].scl = -1;
                        }
                        hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                        //CUDA SPECIFIC
                        hbonds.select.hbond_list[ihb_top].sym_index = -1;
                        rvec_MakeZero( hbonds.select.hbond_list[ihb_top].hb_f );

                        ++ihb_top;
                    }
                    /* atom i: H bonding atom, native
                     * atom j: H atom, native */
                    else if ( ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                    {
                        //jhb_top = End_Index( atom_j->Hindex, hbonds );
                        hbonds.select.hbond_list[ihb_top].nbr = j;
                        hbonds.select.hbond_list[ihb_top].scl = -1;
                        hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                        //CUDA SPECIFIC
                        hbonds.select.hbond_list[ihb_top].sym_index = -1;
                        rvec_MakeZero( hbonds.select.hbond_list[ihb_top].hb_f );

                        ++ihb_top;
                    }
                }
            }

            /* uncorrected bond orders */
            if ( nbr_pj->d <= control->bond_cut &&
                    Dev_BOp( bonds, control->bo_cut, i, btop_i, nbr_pj,
                        sbp_i, sbp_j, twbp, workspace.dDeltap_self,
                        workspace.total_bond_order ) == TRUE )
            {
                //num_bonds += 2;
                ++btop_i;

                /* TODO: Need to do later... since i and j are parallel */
//                if( workspace->bond_mark[j] > workspace->bond_mark[i] + 1 )
//                {
//                    workspace->bond_mark[j] = workspace->bond_mark[i] + 1;
//                }
//                else if( workspace->bond_mark[i] > workspace->bond_mark[j] + 1 )
//                {
//                    workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
//                }
            }
        }
    }

    Dev_Set_End_Index( i, btop_i, &bonds );
    H->end[i] = Htop;
//    if( local == TRUE )
//    {
        if ( control->hbond_cut > 0.0 && ihb_top > 0 && (ihb == H_ATOM || ihb == H_BONDING_ATOM) )
        {
            Dev_Set_End_Index( atom_i->Hindex, ihb_top, &hbonds );
        }
//    }
}


CUDA_GLOBAL void k_init_bond_mark( int offset, int n, int *bond_mark )
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
    {
        return;
    }

    bond_mark[offset + threadIdx.x] = 1000;
}


CUDA_GLOBAL void New_fix_sym_dbond_indices( reax_list pbonds, int N )
{
    int i, j, k, nbr;
    bond_data *ibond, *jbond;
    int atom_j;
    reax_list *bonds;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    bonds = &pbonds;

    for ( j = Dev_Start_Index(i, bonds); j < Dev_End_Index(i, bonds); j++ )
    {
        ibond = &( bonds->select.bond_list[j] );
        nbr = ibond->nbr;

        for ( k = Dev_Start_Index(nbr, bonds); k < Dev_End_Index(nbr, bonds); k++ )
        {
            jbond = &( bonds->select.bond_list[k] );
            atom_j = jbond->nbr;

            if ( atom_j == i )
            {
                if ( i > nbr )
                {
                    ibond->dbond_index = j;
                    jbond->dbond_index = j;

                    ibond->sym_index = k;
                    jbond->sym_index = j;
                }
            }
        }
    }
}


CUDA_GLOBAL void New_fix_sym_hbond_indices( reax_atom *my_atoms, reax_list hbonds, int N )
{
    int i, j, k;
    int nbr, nbrstart, nbrend;
    int start, end;
    hbond_data *ihbond, *jhbond;
    int __THREADS_PER_ATOM__ = HB_KER_SYM_THREADS_PER_ATOM;
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = thread_id / __THREADS_PER_ATOM__;
    int lane_id = thread_id & (__THREADS_PER_ATOM__ - 1);
    int my_bucket = threadIdx.x / __THREADS_PER_ATOM__;

    if ( warp_id > N )
    {
        return;
    }

    i = warp_id;
    start = Dev_Start_Index( my_atoms[i].Hindex, &hbonds );
    end = Dev_End_Index( my_atoms[i].Hindex, &hbonds );
    j = start + lane_id;

    while ( j < end )
    {
        ihbond = &( hbonds.select.hbond_list[j] );
        nbr = ihbond->nbr;

        nbrstart = Dev_Start_Index( my_atoms[nbr].Hindex, &hbonds );
        nbrend = Dev_End_Index( my_atoms[nbr].Hindex, &hbonds );

        for ( k = nbrstart; k < nbrend; k++ )
        {
            jhbond = &( hbonds.select.hbond_list[k] );

            if ( jhbond->nbr == i )
            {
                ihbond->sym_index = k;
                jhbond->sym_index = j;
                break;
            }
        }

        j += __THREADS_PER_ATOM__;
    }
}


CUDA_GLOBAL void k_update_bonds( reax_atom *my_atoms, reax_list bonds, int n )
{
    int i;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    my_atoms[i].num_bonds = Dev_Num_Entries( i, &bonds );
}


CUDA_GLOBAL void k_update_hbonds( reax_atom *my_atoms, reax_list hbonds, int n )
{
    int Hindex;
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    Hindex = my_atoms[i].Hindex;
    my_atoms[i].num_hbonds = Dev_Num_Entries( Hindex, &hbonds );
}


int Cuda_Validate_Lists( reax_system *system, storage *workspace,
        reax_list **lists, control_params *control, 
        int step, int numH )
{
    int blocks;
    int ret;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    ret = SUCCESS;

    k_update_bonds <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, *(*lists + BONDS), system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );

    if ( control->hbond_cut > 0.0 && system->numH > 0 )
    {
        k_update_hbonds <<< blocks, DEF_BLOCK_SIZE >>>
            ( system->d_my_atoms, *(*lists + HBONDS), system->n );
        hipDeviceSynchronize( );
        cudaCheckError( );
    }

    /* 3bodies list: since a more accurate estimate of the num.
     * of three body interactions requires that bond orders have
     * been computed, delay validation until for computation */

    return ret;
}


CUDA_GLOBAL void k_print_forces( reax_atom *my_atoms, rvec *f, int n )
{
    int i; 

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n)
    {
        return;
    }

    printf( "%8d: %24.15f, %24.15f, %24.15f\n",
            my_atoms[i].orig_id,
            f[i][0],
            f[i][1],
            f[i][2] );
}


static void Print_Forces( reax_system *system )
{
    int blocks;
    
    blocks = (system->n) / DEF_BLOCK_SIZE + 
        (((system->n % DEF_BLOCK_SIZE) == 0) ? 0 : 1);

    k_print_forces <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, dev_workspace->f, system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


CUDA_GLOBAL void k_init_bond_orders( reax_atom *my_atoms, reax_list far_nbrs, 
        reax_list bonds, real *total_bond_order, int N )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= N)
    {
        return;
    }

    atom_i = &(my_atoms[i]);
    start_i = Dev_Start_Index(i, &far_nbrs);
    end_i = Dev_End_Index(i, &far_nbrs);

    for( pj = start_i; pj < end_i; ++pj )
    { 
//        nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
//        j = nbr_pj->nbr;
//        atom_j = &(my_atoms[j]);
//
//        total_bond_order[i]++;
//        atom_i->Hindex++;
    }
}


CUDA_GLOBAL void k_bond_mark( reax_list p_bonds, storage p_workspace, int N )
{
    int i, j, k;
    reax_list *bonds = &( p_bonds );
    storage *workspace = &( p_workspace );

//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    if ( i >= N )
//    {
//        return;
//    }

    for ( i = 0; i < N; i++ )
    {
        for (k = Dev_Start_Index(i, bonds); k < Dev_End_Index(i, bonds); k++)
        {
            bond_data *bdata = &( bonds->select.bond_list[k] );
            j = bdata->nbr;

            if (i < j )
            {
                if ( workspace->bond_mark[j] > (workspace->bond_mark[i] + 1) )
                {
                    workspace->bond_mark[j] = workspace->bond_mark[i] + 1;    
                }
                else if ( workspace->bond_mark[i] > (workspace->bond_mark[j] + 1) )
                {
                    workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
                }
            }
        }
    }
}


int Cuda_Init_Forces( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    int i, ret, Htop;
    int blocks, hblocks;

    ret = Cuda_Estimate_Storages( system, control, dev_lists, &(dev_workspace->H), data->step );

    if ( ret == SUCCESS )
    {
//        /* init the workspace (bond_mark) */
//        cuda_memset( dev_workspace->bond_mark, 0, sizeof(int) * system->n, "bond_mark" );
//
//        blocks = (system->N - system->n) / DEF_BLOCK_SIZE + 
//           (((system->N - system->n) % DEF_BLOCK_SIZE == 0) ? 0 : 1);
//        k_init_bond_mark <<< blocks, DEF_BLOCK_SIZE >>>
//           ( system->n, (system->N - system->n), dev_workspace->bond_mark );
//        hipDeviceSynchronize( );
//        cudaCheckError( );

        /* main kernel */
        blocks = (system->N) / DEF_BLOCK_SIZE + 
            (((system->N % DEF_BLOCK_SIZE) == 0) ? 0 : 1);

//        k_init_bond_orders <<< blocks, DEF_BLOCK_SIZE >>>
//            ( system->d_my_atoms, *(*dev_lists + FAR_NBRS), *(*dev_lists + BONDS),
//              dev_workspace->total_bond_order, system->N );
//        hipDeviceSynchronize( );
//        cudaCheckError( );

//        k_print_hbond_info <<< blocks, DEF_BLOCK_SIZE >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              (control_params *)control->d_control_params,
//              *(*dev_lists + HBONDS), system->N );
//        hipDeviceSynchronize( );
//        cudaCheckError( );

        k_init_forces <<< blocks, DEF_BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              system->reax_param.d_tbp, *dev_workspace,
              (control_params *)control->d_control_params,
              *(*dev_lists + FAR_NBRS), *(*dev_lists + BONDS),
              *(*dev_lists + HBONDS), d_LR, system->n,
              system->N, system->reax_param.num_atom_types,
              (((data->step-data->prev_steps) % control->reneighbor) == 0) );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* fix sym_index and dbond_index */
        New_fix_sym_dbond_indices <<< blocks, BLOCK_SIZE >>> 
            ( *(*dev_lists + BONDS), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        if ( control->hbond_cut > 0 && system->numH > 0 )
        {
            /* make hbond_list symmetric */
            hblocks = (system->N * HB_KER_SYM_THREADS_PER_ATOM / HB_SYM_BLOCK_SIZE) + 
                ((((system->N * HB_KER_SYM_THREADS_PER_ATOM) % HB_SYM_BLOCK_SIZE) == 0) ? 0 : 1);

            New_fix_sym_hbond_indices <<< hblocks, HB_BLOCK_SIZE >>>
                ( system->d_my_atoms, *(*dev_lists + HBONDS), system->N );
            hipDeviceSynchronize( );
            cudaCheckError( );
        }

        /* update bond_mark */
//        k_bond_mark <<< blocks, DEF_BLOCK_SIZE >>>
//        k_bond_mark <<< 1, 1 >>>
//            ( *(*dev_lists + BONDS), *dev_workspace, system->N );
//        hipDeviceSynchronize( );
//        cudaCheckError( );

        /* validate lists */
        ret = Cuda_Validate_Lists( system, workspace, dev_lists, control,
                data->step, system->numH );
    }

    return ret;
}


int Cuda_Init_Forces_No_Charges( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    //TODO: implement later when figure out bond_mark usage
    return FAILURE;
}


int Cuda_Compute_Bonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control )
{
    int i, hbs, hnbrs_blocks, update_energy, ret;
    int *thbody;
    static int compute_bonded_part1 = FALSE;
    real t_start, t_elapsed;
    real *spad = (real *) scratch;
    rvec *rvec_spad;

    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;
    ret = SUCCESS;

    if ( compute_bonded_part1 == FALSE )
    {
        /* 1. Bond Order Interactions */
        t_start = Get_Time( );

#if defined(DEBUG)
        fprintf( stderr, " Begin Bonded Forces ... %d x %d\n",
                BLOCKS_N, BLOCK_SIZE );
#endif

        Cuda_Calculate_BO_init <<< BLOCKS_N, BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_sbp, 
              *dev_workspace, system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        Cuda_Calculate_BO <<< BLOCKS_N, BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, 
              system->reax_param.d_tbp, *dev_workspace, 
              *(*dev_lists + BONDS),
              system->reax_param.num_atom_types, system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        Cuda_Update_Uncorrected_BO <<< BLOCKS_N, BLOCK_SIZE >>>
            ( *dev_workspace, *(*dev_lists + BONDS), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        Cuda_Update_Workspace_After_BO <<< BLOCKS_N, BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, 
             *dev_workspace, system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        t_elapsed = Get_Timing_Info( t_start );

#if defined(DEBUG)
        fprintf( stderr, "Bond Orders... return value --> %d --- Timing %lf \n",
                hipGetLastError( ), t_elapsed );
        fprintf( stderr, "Cuda_Calculate_Bond_Orders Done... \n" );
#endif

        /* 2. Bond Energy Interactions */
        t_start = Get_Time( );
        cuda_memset( spad, 0, system->N * (2 * sizeof(real)) , "scratch" );

        Cuda_Bonds <<< BLOCKS, BLOCK_SIZE, sizeof(real)* BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, system->reax_param.d_tbp,
              *dev_workspace, *(*dev_lists + BONDS), 
              system->n, system->reax_param.num_atom_types, spad );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* reduction for E_BE */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_bond,
                    system->n );
        }

        t_elapsed = Get_Timing_Info( t_start );

#if defined(DEBUG)
        fprintf( stderr, "Cuda_Bond_Energy ... return value --> %d --- Timing %lf \n",
                hipGetLastError( ), t_elapsed );
        fprintf( stderr, "Cuda_Bond_Energy Done... \n" );
#endif

        /* 3. Atom Energy Interactions */
        t_start = Get_Time( );
        cuda_memset( spad, 0, ( 6 * sizeof(real) * system->n ), "scratch" );

        Cuda_Atom_Energy <<< BLOCKS, BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_gp,
              system->reax_param.d_sbp, system->reax_param.d_tbp, *dev_workspace,
              *(*dev_lists + BONDS), system->n, system->reax_param.num_atom_types,
              spad, spad + 2 * system->n, spad + 4 * system->n);
        hipDeviceSynchronize( );
        cudaCheckError( );

//        Cuda_Atom_Energy_PostProcess <<< BLOCKS, BLOCK_SIZE >>>
//            ( *(*dev_lists + BONDS), *dev_workspace, system->n );
        Cuda_Atom_Energy_PostProcess <<< BLOCKS_N, BLOCK_SIZE >>>
            ( *(*dev_lists + BONDS), *dev_workspace, system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* reduction for E_Lp */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_lp,
                    system->n );
        }

        /* reduction for E_Ov */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad + 2 * system->n,
                    &((simulation_data *)data->d_simulation_data)->my_en.e_ov,
                    system->n );
        }

        /* reduction for E_Un */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad + 4 * system->n,
                    &((simulation_data *)data->d_simulation_data)->my_en.e_ov,
                    system->n );
        }

        t_elapsed = Get_Timing_Info( t_start );

#if defined(DEBUG)
        fprintf( stderr, "test_LonePair_postprocess ... return value --> %d --- Timing %lf \n",
                hipGetLastError( ), t_elapsed );
        fprintf( stderr, "test_LonePair_postprocess Done... \n");
#endif

        compute_bonded_part1 = TRUE;
    }

    /* 4. Valence Angles Interactions */
    t_start = Get_Time( );

    thbody = (int *) scratch;
    ret = Cuda_Estimate_Storage_Three_Body( system, control, data->step,
            dev_lists, thbody );

#if defined(DEBUG)
    fprintf( stderr, "system->total_thbodies = %d, lists:THREE_BODIES->num_intrs = %d,\n",
            system->total_thbodies, (*lists + THREE_BODIES)->num_intrs );
    fprintf( stderr, "lists:THREE_BODIES->n = %d, lists:BONDS->num_intrs = %d,\n",
            (*lists + THREE_BODIES)->n, (*lists + BONDS)->num_intrs );
    fprintf( stderr, "system->total_thbodies = %d\n", system->total_thbodies );
#endif

    if ( ret == SUCCESS )
    {
        Cuda_Init_Three_Body_Indices( thbody, system->total_bonds );

        cuda_memset( spad, 0, 6 * sizeof(real) * system->N + sizeof(rvec) * system->N * 2, "scratch" );
        Cuda_Valence_Angles <<< BLOCKS_N, BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_gp, 
              system->reax_param.d_sbp, system->reax_param.d_thbp, 
              (control_params *)control->d_control_params,
              *dev_workspace, *(*dev_lists + BONDS), *(*dev_lists + THREE_BODIES),
              system->n, system->N, system->reax_param.num_atom_types, 
              spad, spad + 2 * system->N, spad + 4 * system->N, (rvec *)(spad + 6 * system->N) );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* reduction for E_Ang */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_ang,
                    system->N );
        }

        /* reduction for E_Pen */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad + 2 * system->N,
                    &((simulation_data *)data->d_simulation_data)->my_en.e_pen,
                    system->N );
        }

        /* reduction for E_Coa */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad + 4 * system->N,
                    &((simulation_data *)data->d_simulation_data)->my_en.e_coa,
                    system->N );
        }

        /* reduction for ext_pres */
        rvec_spad = (rvec *) (spad + 6 * system->N);
        k_reduction_rvec <<< BLOCKS_N, BLOCK_SIZE, sizeof(rvec) * BLOCK_SIZE >>>
            ( rvec_spad, rvec_spad + system->N,  system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        k_reduction_rvec <<< 1, BLOCKS_POW_2_N, sizeof(rvec) * BLOCKS_POW_2_N >>>
            ( rvec_spad + system->N, &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS_N );
        hipDeviceSynchronize ();
        cudaCheckError( );
//        Cuda_Reduction_Sum( rvec_spad,
//                &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                system->N );

        Cuda_Valence_Angles_PostProcess <<< BLOCKS_N, BLOCK_SIZE >>>
            ( system->d_my_atoms, (control_params *)control->d_control_params,
              *dev_workspace, *(*dev_lists + BONDS), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        t_elapsed = Get_Timing_Info( t_start );

#if defined(DEBUG)
        fprintf( stderr, "Three_Body_Interactions ...  Timing %lf \n",
                t_elapsed );
        fprintf( stderr, "Three_Body_Interactions Done... \n" );
#endif

        /* 5. Torsion Angles Interactions */
        t_start = Get_Time( );
        cuda_memset( spad, 0,
                4 * sizeof(real) * system->n + sizeof(rvec) * system->n * 2,
                "scratch" );
        Cuda_Torsion_Angles <<< BLOCKS, BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_fbp,
              (control_params *) control->d_control_params, *(*dev_lists + BONDS),
              *(*dev_lists + THREE_BODIES), *dev_workspace, system->n,
              system->reax_param.num_atom_types, 
              spad, spad + 2 * system->n, (rvec *) (spad + 4 * system->n) );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* reduction for E_Tor */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_tor,
                    system->n );
        }

        /* reduction for E_Con */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad + 2 * system->n,
                    &((simulation_data *)data->d_simulation_data)->my_en.e_con,
                    system->n );
        }

        /* reduction for ext_pres */
        rvec_spad = (rvec *) (spad + 4 * system->n);
        k_reduction_rvec <<< BLOCKS, BLOCK_SIZE, sizeof(rvec) * BLOCK_SIZE >>>
            ( rvec_spad, rvec_spad + system->n,  system->n );
        hipDeviceSynchronize( );
        cudaCheckError( );

        k_reduction_rvec <<< 1, BLOCKS_POW_2, sizeof(rvec) * BLOCKS_POW_2 >>>
                ( rvec_spad + system->n,
                  &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS );
        hipDeviceSynchronize( );
        cudaCheckError( );
//        Cuda_Reduction_Sum( rvec_spad,
//                &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                system->n );

        Cuda_Torsion_Angles_PostProcess <<< BLOCKS_N, BLOCK_SIZE >>>
                ( system->d_my_atoms, *dev_workspace, *(*dev_lists + BONDS),
                  system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        t_elapsed = Get_Timing_Info( t_start );

#if defined(DEBUG)
        fprintf( stderr, "Four_Body_post process return value --> %d --- Four body Timing %lf \n",
                hipGetLastError( ), t_elapsed );
        fprintf( stderr, " Four_Body_ Done... \n");
#endif

        /* 6. Hydrogen Bonds Interactions */
        if ( control->hbond_cut > 0.0 && system->numH > 0 )
        {
            t_start = Get_Time( );
            cuda_memset( spad, 0,
                    2 * sizeof(real) * system->n + sizeof(rvec) * system->n * 2, "scratch" );

            hbs = (system->n * HB_KER_THREADS_PER_ATOM / HB_BLOCK_SIZE) + 
                (((system->n * HB_KER_THREADS_PER_ATOM) % HB_BLOCK_SIZE) == 0 ? 0 : 1);

            Cuda_Hydrogen_Bonds <<< BLOCKS, BLOCK_SIZE >>>
//            Cuda_Hydrogen_Bonds_MT <<< hbs, HB_BLOCK_SIZE, 
//                    HB_BLOCK_SIZE * (2 * sizeof(real) + 2 * sizeof(rvec)) >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp,
                      system->reax_param.d_hbp, system->reax_param.d_gp,
                      (control_params *) control->d_control_params,
                      *dev_workspace, *(*dev_lists + BONDS), *(*dev_lists + HBONDS),
                      system->n, system->reax_param.num_atom_types,
                      spad, (rvec *) (spad + 2 * system->n) );
            hipDeviceSynchronize( );
            cudaCheckError( );

            /* reduction for E_HB */
            if ( update_energy == TRUE )
            {
                Cuda_Reduction_Sum( spad,
                        &((simulation_data *)data->d_simulation_data)->my_en.e_hb,
                        system->n );
            }

            /* reduction for ext_pres */
            rvec_spad = (rvec *) (spad + 2 * system->n);
            k_reduction_rvec <<< BLOCKS, BLOCK_SIZE, sizeof(rvec) * BLOCK_SIZE >>>
                (rvec_spad, rvec_spad + system->n,  system->n);
            hipDeviceSynchronize( );
            cudaCheckError( );

            k_reduction_rvec <<< 1, BLOCKS_POW_2, sizeof(rvec) * BLOCKS_POW_2 >>>
                (rvec_spad + system->n, &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS);
            hipDeviceSynchronize( );
            cudaCheckError( );
//            Cuda_Reduction_Sum( rvec_spad,
//                    &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                    system->n );

            /* post process step1 */
            Cuda_Hydrogen_Bonds_PostProcess <<< BLOCKS_N, BLOCK_SIZE, BLOCK_SIZE * sizeof(rvec) >>>
                ( system->d_my_atoms, *dev_workspace,
                  *(*dev_lists + BONDS), system->N );
            hipDeviceSynchronize( );
            cudaCheckError( );

            /* post process step2 */
            hnbrs_blocks = (system->N * HB_POST_PROC_KER_THREADS_PER_ATOM / HB_POST_PROC_BLOCK_SIZE) +
                (((system->N * HB_POST_PROC_KER_THREADS_PER_ATOM) % HB_POST_PROC_BLOCK_SIZE) == 0 ? 0 : 1);

            Cuda_Hydrogen_Bonds_HNbrs <<< system->N, 32, 32 * sizeof(rvec) >>>
                ( system->d_my_atoms, *dev_workspace, *(*dev_lists + HBONDS) );
//            Cuda_Hydrogen_Bonds_HNbrs_BL <<< hnbrs_blocks, HB_POST_PROC_BLOCK_SIZE, 
//                    HB_POST_PROC_BLOCK_SIZE * sizeof(rvec) >>>
//                ( system->d_my_atoms, *dev_workspace, *(*dev_lists + HBONDS), system->N );
            hipDeviceSynchronize( );
            cudaCheckError( );

            t_elapsed = Get_Timing_Info( t_start );

#if defined(DEBUG)
            fprintf( stderr,
                    "Hydrogen bonds return value --> %d --- HydrogenBonds Timing %lf \n",
                    hipGetLastError( ), t_elapsed );
            fprintf( stderr, "Hydrogen_Bond Done... \n" );
#endif
        }

        compute_bonded_part1 = FALSE;
    }

    return ret;
}


void Cuda_Compute_NonBonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control,
        mpi_datatypes *mpi_data )
{
    /* van der Waals and Coulomb interactions */
    Cuda_NonBonded_Energy( system, control, workspace, data,
            lists, out_control, (control->tabulate == 0) ? false: true );
}


void Cuda_Compute_Total_Force( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, mpi_datatypes *mpi_data )
{
    rvec *f;

    f = (rvec *) host_scratch;
    memset( f, 0, sizeof(rvec) * system->N );

    Cuda_Total_Forces( system, control, data, workspace );

#if defined(PURE_REAX)
    /* now all forces are computed to their partially-final values
     * based on the neighbors information each processor has had.
     * final values of force on each atom needs to be computed by adding up
     * all partially-final pieces */

    //MVAPICH2
    copy_host_device( f, dev_workspace->f, sizeof(rvec) * system->N ,
            hipMemcpyDeviceToHost, "total_force:f:get" );

    Coll( system, mpi_data, f, mpi_data->mpi_rvec,
          sizeof(rvec) / sizeof(void), rvec_unpacker );

    copy_host_device( f, dev_workspace->f, sizeof(rvec) * system->N,
            hipMemcpyHostToDevice, "total_force:f:put" );

    Cuda_Total_Forces_PURE( system, dev_workspace );
#endif

}


int Cuda_Compute_Forces( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace, reax_list **lists,
        output_controls *out_control, mpi_datatypes *mpi_data )
{
    int charge_flag, retVal;

#if defined(LOG_PERFORMANCE)
    real t_start = 0;

    //MPI_Barrier( MPI_COMM_WORLD );
    if ( system->my_rank == MASTER_NODE )
    {
        t_start = Get_Time( );
    }
#endif

    retVal = SUCCESS;

    /********* init forces ************/
    if ( control->charge_freq && (data->step - data->prev_steps) % control->charge_freq == 0 )
    {
        charge_flag = TRUE;
    }
    else
    {
        charge_flag = FALSE;
    }

    if ( charge_flag == TRUE )
    {
        retVal = Cuda_Init_Forces( system, control, data, workspace, lists, out_control );
    }
    else
    {
        retVal = Cuda_Init_Forces_No_Charges( system, control, data, workspace, lists, out_control );
    }

    if ( retVal == SUCCESS )
    {
        //validate_sparse_matrix( system, workspace );

#if defined(LOG_PERFORMANCE)
        //MPI_Barrier( MPI_COMM_WORLD );
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &(data->timing.init_forces) );
        }
#endif

        /********* bonded interactions ************/
        retVal = Cuda_Compute_Bonded_Forces( system, control, data, workspace, lists, out_control );

#if defined(LOG_PERFORMANCE)
        //MPI_Barrier( MPI_COMM_WORLD );
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &(data->timing.bonded) );
        }
#endif

#if defined(DEBUG_FOCUS)
        fprintf( stderr, "p%d @ step%d: completed bonded\n",
                 system->my_rank, data->step );
        MPI_Barrier( MPI_COMM_WORLD );
#endif
    }

    if ( retVal == SUCCESS )
    {
    /**************** charges ************************/
#if defined(PURE_REAX)
        if ( charge_flag == TRUE )
        {
            Cuda_QEq( system, control, data, workspace, out_control, mpi_data );
        }

#if defined(LOG_PERFORMANCE)
        //MPI_Barrier( MPI_COMM_WORLD );
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &(data->timing.cm) );
        }
#endif

#if defined(DEBUG_FOCUS)
        fprintf(stderr, "p%d @ step%d: qeq completed\n", system->my_rank, data->step);
        MPI_Barrier( MPI_COMM_WORLD );
#endif
#endif //PURE_REAX

        /********* nonbonded interactions ************/
        Cuda_Compute_NonBonded_Forces( system, control, data, workspace,
                lists, out_control, mpi_data );

#if defined(LOG_PERFORMANCE)
        //MPI_Barrier( MPI_COMM_WORLD );
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &(data->timing.nonb) );
        }
#endif
#if defined(DEBUG_FOCUS)
        fprintf( stderr, "p%d @ step%d: nonbonded forces completed\n",
                system->my_rank, data->step );
        MPI_Barrier( MPI_COMM_WORLD );
#endif

        /*********** total force ***************/
        Cuda_Compute_Total_Force( system, control, data, workspace, lists, mpi_data );

#if defined(LOG_PERFORMANCE)
        //MPI_Barrier( MPI_COMM_WORLD );
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &(data->timing.bonded) );
        }
#endif
#if defined(DEBUG_FOCUS)
        fprintf( stderr, "p%d @ step%d: total forces computed\n",
                system->my_rank, data->step );
        //Print_Total_Force( system, data, workspace );
        MPI_Barrier( MPI_COMM_WORLD );

#endif
    }

    return retVal;
}


#include "cuda_forces.h"

#include "cuda_bonds.h"
#include "cuda_bond_orders.h"
#include "cuda_charges.h"
#include "cuda_helpers.h"
#include "cuda_hydrogen_bonds.h"
#include "cuda_lin_alg.h"
#include "cuda_list.h"
#include "cuda_multi_body.h"
#include "cuda_neighbors.h"
#include "cuda_nonbonded.h"
#include "cuda_reduction.h"
#include "cuda_torsion_angles.h"
#include "cuda_utils.h"
#include "cuda_valence_angles.h"

#include "../basic_comm.h"
#include "../forces.h"
#include "../index_utils.h"
#include "../tool_box.h"
#include "../vector.h"


typedef enum
{
    DIAGONAL = 0,
    OFF_DIAGONAL = 1,
} MATRIX_ENTRY_POSITION;


CUDA_DEVICE real Init_Charge_Matrix_Entry( single_body_parameters *sbp_i, real *ctap,
        control_params *control, int i, int j, real r_ij, real gamma, MATRIX_ENTRY_POSITION pos )
{
    real taper, dr3gamij_1, dr3gamij_3, ret;

    ret = 0.0;

    switch ( control->charge_method )
    {
    case QEQ_CM:
    case EE_CM:
    case ACKS2_CM:
        switch ( pos )
        {
            case OFF_DIAGONAL:
                taper = ctap[7] * r_ij + ctap[6];
                taper = taper * r_ij + ctap[5];
                taper = taper * r_ij + ctap[4];
                taper = taper * r_ij + ctap[3];
                taper = taper * r_ij + ctap[2];
                taper = taper * r_ij + ctap[1];
                taper = taper * r_ij + ctap[0];    

                /* shielding */
                dr3gamij_1 = r_ij * r_ij * r_ij + gamma;
                dr3gamij_3 = POW( dr3gamij_1 , 1.0 / 3.0 );

                //TODO: investigate why conditional is excluded (OpenMP code below)
//                ret = ((i == j) ? 0.5 : 1.0) * Tap * EV_to_KCALpMOL / dr3gamij_3;
                ret = taper * EV_to_KCALpMOL / dr3gamij_3;
                break;

            case DIAGONAL:
                ret = sbp_i->eta;
                break;

            default:
//                fprintf( stderr, "[ERROR] Invalid matrix position. Terminating...\n" );
//                exit( INVALID_INPUT );
                break;
        }
        break;


    default:
//        fprintf( stderr, "[ERROR] Invalid charge method. Terminating...\n" );
//        exit( INVALID_INPUT );
        break;
    }

    return ret;
}


CUDA_DEVICE real Init_Charge_Matrix_Entry_Tab( LR_lookup_table *t_LR, real r_ij,
        int ti, int tj, int num_atom_types )
{
    int r, tmin, tmax;
    real val, dif, base;
    LR_lookup_table *t; 

    tmin = MIN( ti, tj );
    tmax = MAX( ti, tj );
    t = &t_LR[ index_lr(tmin,tmax, num_atom_types) ];

    /* cubic spline interpolation */
    r = (int)(r_ij * t->inv_dx);
    if ( r == 0 )
    {
        ++r;
    }
    base = (real)(r + 1) * t->dx;
    dif = r_ij - base;
    val = ((t->ele[r].d * dif + t->ele[r].c) * dif + t->ele[r].b) * dif
        + t->ele[r].a;
    val *= EV_to_KCALpMOL / C_ELE;

    return val;
}


CUDA_GLOBAL void k_disable_hydrogen_bonding( control_params *control )
{
    control->hbond_cut = 0.0;
}


CUDA_GLOBAL void k_init_end_index( int * intr_cnt, int *indices, int *end_indices, int N )
{
    int i;

    i = blockIdx.x * blockDim.x  + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    end_indices[i] = indices[i] + intr_cnt[i];
}


CUDA_GLOBAL void k_setup_hindex( reax_atom *my_atoms, int N )
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    my_atoms[i].Hindex = i;
}


CUDA_GLOBAL void k_init_hbond_indices( reax_atom * atoms, single_body_parameters *sbp,
        int *hbonds, int *max_hbonds, int *indices, int *end_indices, int N )
{
    int i, hindex, my_hbonds;

    i = blockIdx.x * blockDim.x  + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    hindex = atoms[i].Hindex;

    if ( sbp[ atoms[i].type ].p_hbond == H_ATOM
            || sbp[ atoms[i].type ].p_hbond == H_BONDING_ATOM )
    {
        my_hbonds = hbonds[i];
        indices[hindex] = max_hbonds[i];
        end_indices[hindex] = indices[hindex] + hbonds[i];
    }
    else
    {
        my_hbonds = 0;
        indices[hindex] = 0;
        end_indices[hindex] = 0;
    }
    atoms[i].num_hbonds = my_hbonds;
}


CUDA_GLOBAL void k_print_hbond_info( reax_atom *my_atoms, single_body_parameters *sbp, 
        control_params *control, reax_list hbond_list, int N )
{
    int i;
    int type_i;
    int ihb, ihb_top;
    single_body_parameters *sbp_i;
    reax_atom *atom_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    atom_i = &my_atoms[i];
    type_i = atom_i->type;
    sbp_i = &sbp[type_i];

    if ( control->hbond_cut > 0.0 )
    {
        ihb = sbp_i->p_hbond;

        if ( ihb == H_ATOM  || ihb == H_BONDING_ATOM )
        {
            ihb_top = Start_Index( atom_i->Hindex, &hbond_list );
        }
        else
        {
            ihb_top = -1;
        }
    }

    printf( "atom %6d: ihb = %2d, ihb_top = %2d\n", i, ihb, ihb_top );
}


CUDA_GLOBAL void k_init_forces( reax_atom *my_atoms, single_body_parameters *sbp, 
        two_body_parameters *tbp, storage workspace, control_params *control, 
        reax_list far_nbr_list, reax_list bond_list, reax_list hbond_list, 
        LR_lookup_table *t_LR, int n, int N, int num_atom_types, int renbr,
        int *max_cm_entries, int *realloc_cm_entries,
        int *max_bonds, int *realloc_bonds,
        int *max_hbonds, int *realloc_hbonds )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    int Htop, btop_i, ihb, jhb, ihb_top;
    int num_bonds, num_hbonds, num_cm_entries;
    int local, flag, flag2, flag3;
    real r_ij, cutoff;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    reax_atom *atom_i, *atom_j;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    H = &workspace.H;
    Htop = H->start[i];

    atom_i = &my_atoms[i];
    type_i = atom_i->type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    btop_i = Start_Index( i, &bond_list );
    sbp_i = &sbp[type_i];

    if ( i < n )
    {
        local = TRUE;
        cutoff = control->nonb_cut;

        //update bond mark here
        workspace.bond_mark[i] = 0;
    }
    else
    {
        local = FALSE;
        cutoff = control->bond_cut;

        //update bond mark here
        workspace.bond_mark[i] = 1000;
    }

    ihb = NON_H_BONDING_ATOM;
    ihb_top = -1;

    if ( local == TRUE )
    {
        H->entries[Htop].j = i;
        H->entries[Htop].val = Init_Charge_Matrix_Entry( sbp_i, workspace.Tap, control,
                i, H->entries[Htop].j, 0.0, 0.0, DIAGONAL );
        ++Htop;
    }

    if ( control->hbond_cut > 0.0 )
    {
        ihb = sbp_i->p_hbond;

        if ( ihb == H_ATOM || ihb == H_BONDING_ATOM )
        {
            ihb_top = Start_Index( atom_i->Hindex, &hbond_list );
        }
        else
        {
            ihb_top = -1;
        }
    }

    /* update i-j distance - check if j is within cutoff */
    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        atom_j = &my_atoms[j];

        if ( renbr )
        {
            if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff )
            {
                flag = TRUE;
            }
            else
            {
                flag = FALSE;
            }

            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                flag2 = TRUE;
            }
            else
            {
                flag2 = FALSE;
            }

        }
        else
        {
            if ( i < j )
            {
                far_nbr_list.far_nbr_list.dvec[pj][0] = atom_j->x[0] - atom_i->x[0];
                far_nbr_list.far_nbr_list.dvec[pj][1] = atom_j->x[1] - atom_i->x[1];
                far_nbr_list.far_nbr_list.dvec[pj][2] = atom_j->x[2] - atom_i->x[2];
            }
            else
            {
                far_nbr_list.far_nbr_list.dvec[pj][0] = atom_i->x[0] - atom_j->x[0];
                far_nbr_list.far_nbr_list.dvec[pj][1] = atom_i->x[1] - atom_j->x[1];
                far_nbr_list.far_nbr_list.dvec[pj][2] = atom_i->x[2] - atom_j->x[2];
            }
            far_nbr_list.far_nbr_list.d[pj] = rvec_Norm_Sqr( far_nbr_list.far_nbr_list.dvec[pj] );

            if ( far_nbr_list.far_nbr_list.d[pj] <= SQR( control->nonb_cut ) )
            {
                flag2 = TRUE;
            }
            else
            {
                flag2 = FALSE;
            }

            if ( far_nbr_list.far_nbr_list.d[pj] <= SQR( control->nonb_cut ) )
            {
                far_nbr_list.far_nbr_list.d[pj] = SQRT( far_nbr_list.far_nbr_list.d[pj] );
                flag = TRUE;
            }
            else
            {
                flag = FALSE;
            }
        }
        if ( flag2 == TRUE )
        {
            type_j = atom_j->type;
            sbp_j = &sbp[type_j];
            ihb = sbp_i->p_hbond;
            jhb = sbp_j->p_hbond;

            /* atom i: H bonding, ghost
             * atom j: H atom, native */
            if ( control->hbond_cut > 0.0
                    && far_nbr_list.far_nbr_list.d[pj] <= control->hbond_cut
                    && ihb == H_BONDING_ATOM && jhb == H_ATOM && i >= n && j < n ) 
            {
                hbond_list.hbond_list[ihb_top].nbr = j;
                hbond_list.hbond_list[ihb_top].scl = -1;
                hbond_list.hbond_list[ihb_top].ptr = pj;

                /* CUDA-specific */
                hbond_list.hbond_list[ihb_top].sym_index = -1;
                rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );

                ++ihb_top;
            }

            //if ((i < n) || (j < n))
            //if (local == TRUE || ((i >= n) &&(j < n)))

            flag3 = FALSE;
            if ( i < j && i < n && (j < n || atom_i->orig_id < atom_j->orig_id) )
            {
                flag3 = TRUE;
            }
            else if ( i > j && i >= n && j < n && atom_j->orig_id < atom_i->orig_id )
            {
                flag3 = TRUE;
            }
            else if ( i > j && i < n && (j < n || atom_j->orig_id < atom_i->orig_id ) )
            {
                flag3 = TRUE;
            }

            if ( flag3 == TRUE )
            {
                twbp = &tbp[ index_tbp(type_i,type_j,num_atom_types) ];
                r_ij = far_nbr_list.far_nbr_list.d[pj];

                //if (renbr) {
                H->entries[Htop].j = j;
                if ( control->tabulate == 0 )
                {
                    H->entries[Htop].val = Init_Charge_Matrix_Entry( sbp_i, workspace.Tap,
                            control, i, H->entries[Htop].j, r_ij, twbp->gamma, OFF_DIAGONAL );
                }
                else
                {
                    H->entries[Htop].val = Init_Charge_Matrix_Entry_Tab( t_LR, r_ij, type_i, type_j,num_atom_types );
                }
                //}
                ++Htop;
            }
        }

        if ( flag == TRUE )
        {
            type_j = atom_j->type;
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            sbp_j = &sbp[type_j];
            twbp = &tbp[ index_tbp(type_i, type_j, num_atom_types) ];

            if ( local == TRUE )
            {
                /* H matrix entry */
//                if( j < n || atom_i->orig_id < atom_j->orig_id ) {//tryQEq||1
//                    H->entries[Htop].j = j;
//                    if( control->tabulate == 0 )
//                            H->entries[Htop].val = Init_Charge_Matrix_Entry( sbp_i, workspace.Tap,
//                                    control, i, H->entries[Htop].j, r_ij, twbp->gamma, OFF_DIAGONAL );
//                    else
//                        H->entries[Htop].val = Init_Charge_Matrix_Entry_Tab(t_LR, r_ij, type_i, type_j,num_atom_types);
//                    ++Htop;
//                } 
//                else if( j < n || atom_i->orig_id > atom_j->orig_id ) {//tryQEq||1
//                    H->entries[Htop].j = j;
//                    if( control->tabulate == 0 )
//                            H->entries[Htop].val = Init_Charge_Matrix_Entry( sbp_i, workspace.Tap,
//                                    control, i, H->entries[Htop].j, r_ij, twbp->gamma, OFF_DIAGONAL );
//                    else
//                        H->entries[Htop].val = Init_Charge_Matrix_Entry_Tab(t_LR, r_ij, type_i, type_j,num_atom_types);
//                    ++Htop;
//                } 

                /* hydrogen bond lists */
                if ( control->hbond_cut > 0.0
                        && (ihb == H_ATOM || ihb == H_BONDING_ATOM)
                        && far_nbr_list.far_nbr_list.d[pj] <= control->hbond_cut )
                {
                    jhb = sbp_j->p_hbond;

                    /* atom i: H atom, native
                     * atom j: H bonding atom */
                    if ( ihb == H_ATOM && jhb == H_BONDING_ATOM )
                    {
                        hbond_list.hbond_list[ihb_top].nbr = j;

                        if ( i < j )
                        {
                            hbond_list.hbond_list[ihb_top].scl = 1;
                        }
                        else
                        {
                            hbond_list.hbond_list[ihb_top].scl = -1;
                        }
                        hbond_list.hbond_list[ihb_top].ptr = pj;

                        /* CUDA-specific */
                        hbond_list.hbond_list[ihb_top].sym_index = -1;
                        rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );

                        ++ihb_top;
                    }
                    /* atom i: H bonding atom, native
                     * atom j: H atom, native */
                    else if ( ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                    {
                        //jhb_top = End_Index( atom_j->Hindex, &hbond_list );
                        hbond_list.hbond_list[ihb_top].nbr = j;
                        hbond_list.hbond_list[ihb_top].scl = -1;
                        hbond_list.hbond_list[ihb_top].ptr = pj;

                        //CUDA SPECIFIC
                        hbond_list.hbond_list[ihb_top].sym_index = -1;
                        rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );

                        ++ihb_top;
                    }
                }
            }

            /* uncorrected bond orders */
            if ( far_nbr_list.far_nbr_list.d[pj] <= control->bond_cut
                    && Cuda_BOp( bond_list, control->bo_cut,
                        i, btop_i, far_nbr_list.far_nbr_list.nbr[pj],
                        &far_nbr_list.far_nbr_list.rel_box[pj], far_nbr_list.far_nbr_list.d[pj],
                        &far_nbr_list.far_nbr_list.dvec[pj], far_nbr_list.format,
                        sbp_i, sbp_j, twbp, workspace.dDeltap_self,
                        workspace.total_bond_order ) == TRUE )
            {
                ++btop_i;

                /* TODO: Need to do later... since i and j are parallel */
//                if( workspace->bond_mark[j] > workspace->bond_mark[i] + 1 )
//                {
//                    workspace->bond_mark[j] = workspace->bond_mark[i] + 1;
//                }
//                else if( workspace->bond_mark[i] > workspace->bond_mark[j] + 1 )
//                {
//                    workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
//                }
            }
        }
    }

    Set_End_Index( i, btop_i, &bond_list );
    H->end[i] = Htop;
//    if( local == TRUE )
//    {
        if ( control->hbond_cut > 0.0 && ihb_top > 0 && (ihb == H_ATOM || ihb == H_BONDING_ATOM) )
        {
            Set_End_Index( atom_i->Hindex, ihb_top, &hbond_list );
        }
//    }

    num_bonds = btop_i - Start_Index( i, &bond_list );
    num_hbonds = ihb_top - Start_Index( atom_i->Hindex, &hbond_list );
    num_cm_entries = Htop - H->start[i];

    /* copy (h)bond info to atom structure
     * (needed for atom ownership transfer via MPI) */
    my_atoms[i].num_bonds = num_bonds;
    my_atoms[i].num_hbonds = num_hbonds;

    /* reallocation checks */
    if ( num_bonds > max_bonds[i] )
    {
        *realloc_bonds = TRUE;
    }

    if ( num_hbonds > max_hbonds[i] )
    {
        *realloc_hbonds = TRUE;
    }

    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


CUDA_GLOBAL void k_estimate_storages( reax_atom *my_atoms, 
        single_body_parameters *sbp, two_body_parameters *tbp,
        control_params *control, reax_list p_far_nbr_list, 
        int num_atom_types, int n, int N, int total_cap,
        int *cm_entries, int *max_cm_entries,
        int *bonds, int *max_bonds,
        int *hbonds, int *max_hbonds )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb;
    int local;
    int num_bonds, num_hbonds, num_cm_entries;
    real cutoff;
    real r_ij; 
    real C12, C34, C56;
    real BO, BO_s, BO_pi, BO_pi2;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    reax_atom *atom_i, *atom_j;
    reax_list *far_nbr_list;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= total_cap )
    {
        return;
    }

    far_nbr_list = &p_far_nbr_list;
    num_bonds = 0;
    num_hbonds = 0;
    num_cm_entries = 0;

    if ( i < N )
    {
        atom_i = &my_atoms[i];
        type_i = atom_i->type;
        start_i = Start_Index( i, far_nbr_list );
        end_i = End_Index( i, far_nbr_list );
        sbp_i = &sbp[type_i];

        if ( i < n )
        { 
            local = TRUE;
            cutoff = control->nonb_cut;
            ++num_cm_entries;
//            ihb = sbp_i->p_hbond;
        }   
        else
        {
            local = FALSE;
            cutoff = control->bond_cut;
//            ihb = NON_H_BONDING_ATOM; 
        } 

        ihb = NON_H_BONDING_ATOM; 

        for ( pj = start_i; pj < end_i; ++pj )
        { 
            j = far_nbr_list->far_nbr_list.nbr[pj];
            atom_j = &my_atoms[j];

            if ( far_nbr_list->far_nbr_list.d[pj] <= control->nonb_cut )
            {
                type_j = my_atoms[j].type;
                sbp_j = &sbp[type_j];
                ihb = sbp_i->p_hbond;
                jhb = sbp_j->p_hbond;

                if ( local == TRUE )
                {
                    if ( i < j && (j < n || atom_i->orig_id < atom_j->orig_id) )
                    {
                        ++num_cm_entries;
                    }
                    else if ( i > j && (j < n || atom_j->orig_id > atom_i->orig_id) )
                    {
                        ++num_cm_entries;
                    }
                }
                else
                {
                    if ( i > j && j < n && atom_j->orig_id < atom_i->orig_id )
                    {
                        ++num_cm_entries;
                    }
                }

                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( control->hbond_cut > 0.0
                        && far_nbr_list->far_nbr_list.d[pj] <= control->hbond_cut 
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM && i >= n && j < n )
                {
                    ++num_hbonds;
                }

//                if ( i >= n )
//                {
//                    ihb = NON_H_BONDING_ATOM;
//                }
            }

            if ( far_nbr_list->far_nbr_list.d[pj] <= cutoff )
            {
                type_j = my_atoms[j].type;
                r_ij = far_nbr_list->far_nbr_list.d[pj];
                sbp_j = &sbp[type_j];
                twbp = &tbp[ index_tbp(type_i ,type_j, num_atom_types) ];

                if ( local == TRUE )
                {
                    /* atom i: H atom OR H bonding atom, native */
                    if ( control->hbond_cut > 0.0
                            && (ihb == H_ATOM || ihb == H_BONDING_ATOM)
                            && far_nbr_list->far_nbr_list.d[pj] <= control->hbond_cut )
                    {
                        jhb = sbp_j->p_hbond;

                        /* atom i: H atom, native
                         * atom j: H bonding atom */
                        if( ihb == H_ATOM && jhb == H_BONDING_ATOM )
                        {
                            ++num_hbonds;
                        }
                        /* atom i: H bonding atom, native
                         * atom j: H atom, native */
                        else if( ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                        {
                            ++num_hbonds;
                        }
                    }
                }

                /* uncorrected bond orders */
                if ( far_nbr_list->far_nbr_list.d[pj] <= control->bond_cut )
                {
                    if ( sbp_i->r_s > 0.0 && sbp_j->r_s > 0.0 )
                    {
                        C12 = twbp->p_bo1 * POW( r_ij / twbp->r_s, twbp->p_bo2 );
                        BO_s = (1.0 + control->bo_cut) * EXP( C12 );
                    }
                    else
                    {
                        C12 = 0.0;
                        BO_s = 0.0;
                    }

                    if ( sbp_i->r_pi > 0.0 && sbp_j->r_pi > 0.0 )
                    {
                        C34 = twbp->p_bo3 * POW( r_ij / twbp->r_p, twbp->p_bo4 );
                        BO_pi = EXP( C34 );
                    }
                    else
                    {
                        C34 = 0.0;
                        BO_pi = 0.0;
                    }

                    if ( sbp_i->r_pi_pi > 0.0 && sbp_j->r_pi_pi > 0.0 )
                    {
                        C56 = twbp->p_bo5 * POW( r_ij / twbp->r_pp, twbp->p_bo6 );
                        BO_pi2= EXP( C56 );
                    }
                    else
                    {
                        C56 = 0.0;
                        BO_pi2 = 0.0;
                    }

                    /* initially BO values are the uncorrected ones, page 1 */
                    BO = BO_s + BO_pi + BO_pi2;

                    if ( BO >= control->bo_cut )
                    {
                        ++num_bonds;
                    }
                }
            }
        }
    }

    bonds[i] = num_bonds;
    max_bonds[i] = MAX( (int)(num_bonds * 2), MIN_BONDS );

    hbonds[i] = num_hbonds;
    max_hbonds[i] = MAX( (int)(num_hbonds * SAFE_ZONE), MIN_HBONDS );

    cm_entries[i] = num_cm_entries;
    max_cm_entries[i] = MAX( (int)(num_cm_entries * SAFE_ZONE), MIN_CM_ENTRIES );
}


CUDA_GLOBAL void k_init_bond_mark( int offset, int n, int *bond_mark )
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    bond_mark[offset + threadIdx.x] = 1000;
}


CUDA_GLOBAL void k_update_sym_dbond_indices( reax_list bond_list, int N )
{
    int i, pj, pk, nbr_ij, nbr_jk;
    bond_data *ibond, *jbond;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    /* i-j bonds */
    for ( pj = Start_Index(i, &bond_list); pj < End_Index(i, &bond_list); ++pj )
    {
        ibond = &bond_list.bond_list[pj];
        nbr_ij = ibond->nbr;

        /* j-k bonds */
        for ( pk = Start_Index(nbr_ij, &bond_list); pk < End_Index(nbr_ij, &bond_list); ++pk )
        {
            jbond = &bond_list.bond_list[pk];
            nbr_jk = jbond->nbr;

            if ( i == nbr_jk && i > nbr_ij )
            {
                ibond->dbond_index = pj;
                jbond->dbond_index = pj;

                ibond->sym_index = pk;
                jbond->sym_index = pj;
            }
        }
    }
}


CUDA_GLOBAL void k_update_sym_hbond_indices( reax_atom *my_atoms, reax_list hbond_list, int N )
{
    int i, j, k;
    int nbr, nbrstart, nbrend;
    int start, end;
    hbond_data *ihbond, *jhbond;
    int __THREADS_PER_ATOM__;
    int thread_id;
    int warp_id;
    int lane_id;

    __THREADS_PER_ATOM__ = HB_KER_SYM_THREADS_PER_ATOM;
    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id / __THREADS_PER_ATOM__;

    if ( warp_id > N )
    {
        return;
    }

    lane_id = thread_id & (__THREADS_PER_ATOM__ - 1);
    i = warp_id;
    start = Start_Index( my_atoms[i].Hindex, &hbond_list );
    end = End_Index( my_atoms[i].Hindex, &hbond_list );
    j = start + lane_id;

    while ( j < end )
    {
        ihbond = &hbond_list.hbond_list[j];
        nbr = ihbond->nbr;

        nbrstart = Start_Index( my_atoms[nbr].Hindex, &hbond_list );
        nbrend = End_Index( my_atoms[nbr].Hindex, &hbond_list );

        for ( k = nbrstart; k < nbrend; k++ )
        {
            jhbond = &hbond_list.hbond_list[k];

            if ( jhbond->nbr == i )
            {
                ihbond->sym_index = k;
                jhbond->sym_index = j;
                break;
            }
        }

        j += __THREADS_PER_ATOM__;
    }
}


#if defined(DEBUG_FOCUS)
CUDA_GLOBAL void k_print_forces( reax_atom *my_atoms, rvec *f, int n )
{
    int i; 

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    printf( "%8d: %24.15f, %24.15f, %24.15f\n",
            my_atoms[i].orig_id,
            f[i][0],
            f[i][1],
            f[i][2] );
}


CUDA_GLOBAL void k_print_hbonds( reax_atom *my_atoms, reax_list hbond_list, int n, int rank, int step )
{
    int i, k, pj, start, end; 
    hbond_data *hbond_jk;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    start = Start_Index( my_atoms[i].Hindex, &hbond_list );
    end = End_Index( my_atoms[i].Hindex, &hbond_list );

    for ( pj = start; pj < end; ++pj )
    {
        k = hbond_list.hbond_list[pj].nbr;
        hbond_jk = &hbond_list.hbond_list[pj];

        printf( "p%03d, step %05d: %8d: %8d, %24.15f, %24.15f, %24.15f\n",
                rank, step, my_atoms[i].Hindex, k,
                hbond_jk->hb_f[0],
                hbond_jk->hb_f[1],
                hbond_jk->hb_f[2] );
    }
}
#endif


CUDA_GLOBAL void k_bond_mark( reax_list p_bond_list, storage p_workspace, int N )
{
    int i, j, k;
    reax_list *bond_list;
    storage *workspace;

//    i = blockIdx.x * blockDim.x + threadIdx.x;
//    if ( i >= N )
//    {
//        return;
//    }

    bond_list = &p_bond_list;
    workspace = &p_workspace;

    for ( i = 0; i < N; i++ )
    {
        for ( k = Start_Index( i, bond_list ); k < End_Index( i, bond_list ); k++ )
        {
            j = bond_list->bond_list[k].nbr;

            if ( i < j )
            {
                if ( workspace->bond_mark[j] > (workspace->bond_mark[i] + 1) )
                {
                    workspace->bond_mark[j] = workspace->bond_mark[i] + 1;    
                }
                else if ( workspace->bond_mark[i] > (workspace->bond_mark[j] + 1) )
                {
                    workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
                }
            }
        }
    }
}


static int Cuda_Estimate_Storage_Three_Body( reax_system *system, control_params *control, 
        storage *workspace, int step, reax_list **lists, int *thbody )
{
    int ret;

    ret = SUCCESS;

    cuda_memset( thbody, 0, system->total_bonds * sizeof(int),
            "Cuda_Estimate_Storage_Three_Body::thbody" );

    Estimate_Cuda_Valence_Angles <<< control->blocks_n, control->block_size >>>
        ( system->d_my_atoms, (control_params *)control->d_control_params, 
          *(lists[BONDS]), system->n, system->N, thbody );
    hipDeviceSynchronize( );
    cudaCheckError( );

    Cuda_Reduction_Sum( thbody, system->d_total_thbodies, system->total_bonds );

    copy_host_device( &system->total_thbodies, system->d_total_thbodies, sizeof(int),
            hipMemcpyDeviceToHost, "Cuda_Estimate_Storage_Three_Body::d_total_thbodies" );

    if ( step == 0 )
    {
        system->total_thbodies = MAX( (int)(system->total_thbodies * SAFE_ZONE), MIN_3BODIES );
        system->total_thbodies_indices = system->total_bonds;

        /* create Three-body list */
        Cuda_Make_List( system->total_thbodies_indices, system->total_thbodies,
                TYP_THREE_BODY, lists[THREE_BODIES] );
    }

    if ( system->total_thbodies > lists[THREE_BODIES]->max_intrs ||
            system->total_bonds > lists[THREE_BODIES]->n )
    {
        if ( system->total_thbodies > lists[THREE_BODIES]->max_intrs )
        {
            system->total_thbodies = MAX( (int)(lists[THREE_BODIES]->max_intrs * SAFE_ZONE),
                    system->total_thbodies );
        }
        if ( system->total_bonds > lists[THREE_BODIES]->n )
        {
            system->total_thbodies_indices = MAX( (int)(lists[THREE_BODIES]->n * SAFE_ZONE),
                    system->total_bonds );
        }

        workspace->d_workspace->realloc.thbody = TRUE;
        ret = FAILURE;
    }

    return ret;
}


#if defined(DEBUG_FOCUS)
static void Print_Forces( reax_system *system )
{
    int blocks;
    
    blocks = (system->n) / DEF_BLOCK_SIZE + 
        (((system->n % DEF_BLOCK_SIZE) == 0) ? 0 : 1);

    k_print_forces <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, workspace->d_workspace->f, system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


static void Print_HBonds( reax_system *system, int step )
{
    int blocks;
    
    blocks = (system->n) / DEF_BLOCK_SIZE + 
        (((system->n % DEF_BLOCK_SIZE) == 0) ? 0 : 1);

    k_print_hbonds <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, *(lists[HBONDS]), system->n, system->my_rank, step );
    hipDeviceSynchronize( );
    cudaCheckError( );
}
#endif


/* Initialize indices for far neighbors list post reallocation
 *
 * system: atomic system info. */
void Cuda_Init_Neighbor_Indices( reax_system *system, reax_list **lists )
{
    int blocks;
    reax_list *far_nbr_list = lists[FAR_NBRS];

    /* init indices */
    Cuda_Scan_Excl_Sum( system->d_max_far_nbrs, far_nbr_list->index, system->total_cap );

    /* init end_indices */
    blocks = system->total_cap / DEF_BLOCK_SIZE + 
        ((system->total_cap % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_init_end_index <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_far_nbrs, far_nbr_list->index, far_nbr_list->end_index, system->total_cap );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


/* Initialize indices for far hydrogen bonds list post reallocation
 *
 * system: atomic system info. */
void Cuda_Init_HBond_Indices( reax_system *system, storage *workspace,
        reax_list **lists )
{
    int blocks, *temp;
    reax_list *hbond_list;

    hbond_list = lists[HBONDS];
    temp = (int *) workspace->scratch;

    /* init Hindices */
    blocks = system->N / DEF_BLOCK_SIZE + 
        ((system->N % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_setup_hindex <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->N );
    hipDeviceSynchronize( );
    cudaCheckError( );

    /* init indices and end_indices */
    Cuda_Scan_Excl_Sum( system->d_max_hbonds, temp, system->total_cap );

    k_init_hbond_indices <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, system->d_hbonds, temp, 
          hbond_list->index, hbond_list->end_index, system->N );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


/* Initialize indices for far bonds list post reallocation
 *
 * system: atomic system info. */
void Cuda_Init_Bond_Indices( reax_system *system, reax_list **lists )
{
    int blocks;
    reax_list *bond_list;

    bond_list = lists[BONDS];

    /* init indices */
    Cuda_Scan_Excl_Sum( system->d_max_bonds, bond_list->index, system->total_cap );

    /* init end_indices */
    blocks = system->N / DEF_BLOCK_SIZE + 
        ((system->N % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_init_end_index <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_bonds, bond_list->index, bond_list->end_index, system->N );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


/* Initialize indices for charge matrix post reallocation
 *
 * system: atomic system info.
 * H: charge matrix */
void Cuda_Init_Sparse_Matrix_Indices( reax_system *system, sparse_matrix *H )
{
    int blocks;

    /* init indices */
    Cuda_Scan_Excl_Sum( system->d_max_cm_entries, H->start, system->total_cap );

    /* init end_indices */
    blocks = system->total_cap / DEF_BLOCK_SIZE
        + ((system->total_cap % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    k_init_end_index <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_cm_entries, H->start, H->end, system->total_cap );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


/* Initialize indices for three body list post reallocation
 *
 * indices: list indices
 * entries: num. of entries in list */
void Cuda_Init_Three_Body_Indices( int *indices, int entries, reax_list **lists )
{
    reax_list *thbody;

    thbody = lists[THREE_BODIES];

    Cuda_Scan_Excl_Sum( indices, thbody->index, entries );
}


void Cuda_Estimate_Storages( reax_system *system, control_params *control, 
        reax_list **lists, int realloc_bonds, int realloc_hbonds, int realloc_cm,
        int step )
{
    int blocks;

    blocks = system->total_cap / ST_BLOCK_SIZE + 
        (((system->total_cap % ST_BLOCK_SIZE == 0)) ? 0 : 1);

    k_estimate_storages <<< blocks, ST_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp, 
          (control_params *)control->d_control_params,
          *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
          system->n, system->N, system->total_cap,
          system->d_cm_entries, system->d_max_cm_entries,
          system->d_bonds, system->d_max_bonds,
          system->d_hbonds, system->d_max_hbonds );
    hipDeviceSynchronize( );
    cudaCheckError( );

    if ( realloc_bonds == TRUE )
    {
        Cuda_Reduction_Sum( system->d_max_bonds, system->d_total_bonds,
                system->total_cap );
        copy_host_device( &system->total_bonds, system->d_total_bonds, sizeof(int), 
                hipMemcpyDeviceToHost, "Cuda_Estimate_Storages::d_total_bonds" );
    }

    if ( system->numH > 0 && control->hbond_cut > 0.0 )
    {
        if ( realloc_hbonds == TRUE )
        {
            Cuda_Reduction_Sum( system->d_max_hbonds, system->d_total_hbonds,
                    system->total_cap );
            copy_host_device( &system->total_hbonds, system->d_total_hbonds, sizeof(int), 
                    hipMemcpyDeviceToHost, "Cuda_Estimate_Storages::d_total_hbonds" );
        }
    }
    else
    {
        if ( step == 0 )
        {
#if defined(DEBUG_FOCUS)
            if ( system->numH == 0 )
            {
                fprintf( stderr, "[INFO] DISABLING HYDROGEN BOND COMPUTATION: NO HYDROGEN ATOMS FOUND\n" );
            }
#endif

#if defined(DEBUG_FOCUS)
            if ( control->hbond_cut <= 0.0 )
            {
                fprintf( stderr, "[INFO] DISABLING HYDROGEN BOND COMPUTATION: BOND CUTOFF LENGTH IS ZERO\n" );
            }
#endif

            control->hbond_cut = 0.0;
            k_disable_hydrogen_bonding <<< 1, 1 >>> ( (control_params *)control->d_control_params );
        }
    }

    if ( realloc_cm == TRUE )
    {
        Cuda_Reduction_Sum( system->d_max_cm_entries, system->d_total_cm_entries, system->total_cap );
        copy_host_device( &system->total_cm_entries, system->d_total_cm_entries, sizeof(int),
                hipMemcpyDeviceToHost, "Cuda_Estimate_Storages::d_total_cm_entries" );
    }
}


int Cuda_Init_Forces( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    int ret, ret_bonds, ret_hbonds, ret_cm;
    int blocks, hblocks;

    /* init the workspace (bond_mark) */
//    cuda_memset( workspace->d_workspace->bond_mark, 0, sizeof(int) * system->n, "bond_mark" );
//
//    blocks = (system->N - system->n) / DEF_BLOCK_SIZE + 
//       (((system->N - system->n) % DEF_BLOCK_SIZE == 0) ? 0 : 1);
//    k_init_bond_mark <<< blocks, DEF_BLOCK_SIZE >>>
//       ( system->n, (system->N - system->n), workspace->d_workspace->bond_mark );
//    hipDeviceSynchronize( );
//    cudaCheckError( );

    blocks = (system->N) / DEF_BLOCK_SIZE + 
        (((system->N % DEF_BLOCK_SIZE) == 0) ? 0 : 1);

//    k_print_hbond_info <<< blocks, DEF_BLOCK_SIZE >>>
//        ( system->d_my_atoms, system->reax_param.d_sbp,
//          (control_params *)control->d_control_params,
//          *(lists[HBONDS]), system->N );
//    hipDeviceSynchronize( );
//    cudaCheckError( );

    /* reset reallocation flags on device */
    cuda_memset( system->d_realloc_bonds, FALSE, sizeof(int), 
            "Cuda_Init_Forces::d_realloc_bonds" );
    cuda_memset( system->d_realloc_hbonds, FALSE, sizeof(int), 
            "Cuda_Init_Forces::d_realloc_hbonds" );
    cuda_memset( system->d_realloc_cm_entries, FALSE, sizeof(int), 
            "Cuda_Init_Forces::d_realloc_cm_entries" );

    k_init_forces <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp,
          system->reax_param.d_tbp, *(workspace->d_workspace), (control_params *)control->d_control_params,
          *(lists[FAR_NBRS]), *(lists[BONDS]), *(lists[HBONDS]),
          workspace->d_LR, system->n, system->N, system->reax_param.num_atom_types,
          (((data->step-data->prev_steps) % control->reneighbor) == 0),
          system->d_max_cm_entries, system->d_realloc_cm_entries,
          system->d_max_bonds, system->d_realloc_bonds,
          system->d_max_hbonds, system->d_realloc_hbonds );
    hipDeviceSynchronize( );
    cudaCheckError( );

    /* check reallocation flags on device */
    copy_host_device( &ret_bonds, system->d_realloc_bonds, sizeof(int), 
            hipMemcpyDeviceToHost, "Cuda_Init_Forces::d_realloc_bonds" );
    copy_host_device( &ret_hbonds, system->d_realloc_hbonds, sizeof(int), 
            hipMemcpyDeviceToHost, "Cuda_Init_Forces::d_realloc_hbonds" );
    copy_host_device( &ret_cm, system->d_realloc_cm_entries, sizeof(int), 
            hipMemcpyDeviceToHost, "Cuda_Init_Forces::d_realloc_cm_entries" );

    ret = (ret_bonds == FALSE && ret_hbonds == FALSE && ret_cm == FALSE) ? SUCCESS : FAILURE;

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "[INFO] p%d, step %d: ret = %d, ret_bonds = %d, ret_hbonds = %d, ret_cm = %d\n",
            system->my_rank, data->step, ret, ret_bonds, ret_hbonds, ret_cm );
#endif

    if ( ret == SUCCESS )
    {
        k_update_sym_dbond_indices <<< blocks, control->block_size >>> 
            ( *(lists[BONDS]), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        if ( control->hbond_cut > 0.0 && system->numH > 0 )
        {
            /* make hbond_list symmetric */
            hblocks = (system->N * HB_KER_SYM_THREADS_PER_ATOM / HB_SYM_BLOCK_SIZE) + 
                ((((system->N * HB_KER_SYM_THREADS_PER_ATOM) % HB_SYM_BLOCK_SIZE) == 0) ? 0 : 1);

            k_update_sym_hbond_indices <<< hblocks, HB_BLOCK_SIZE >>>
                ( system->d_my_atoms, *(lists[HBONDS]), system->N );
            hipDeviceSynchronize( );
            cudaCheckError( );
        }

        /* update bond_mark */
//        k_bond_mark <<< blocks, DEF_BLOCK_SIZE >>>
//        k_bond_mark <<< 1, 1 >>>
//            ( *(lists[BONDS]), *(workspace->d_workspace), system->N );
//        hipDeviceSynchronize( );
//        cudaCheckError( );
    }
    else
    {
        Cuda_Estimate_Storages( system, control, lists,
               ret_bonds, ret_hbonds, ret_cm, data->step );

        workspace->d_workspace->realloc.bonds = ret_bonds;
        workspace->d_workspace->realloc.hbonds = ret_hbonds;
        workspace->d_workspace->realloc.cm = ret_cm;
    }

    return ret;
}


int Cuda_Init_Forces_No_Charges( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    //TODO: implement later when figure out bond_mark usage
    return FAILURE;
}


int Cuda_Compute_Bonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control )
{
    int update_energy, ret;
//    int hbs, hnbrs_blocks;
    int *thbody;
    static int compute_bonded_part1 = FALSE;
    real *spad;
    rvec *rvec_spad;
#if defined(DEBUG_FOCUS)
    real t_start, t_elapsed;
#endif

    spad = (real *) workspace->scratch;
    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;
    ret = SUCCESS;

    if ( compute_bonded_part1 == FALSE )
    {
        /* 1. Bond Order Interactions */
#if defined(DEBUG_FOCUS)
        t_start = Get_Time( );

        fprintf( stderr, " Begin Bonded Forces ... %d x %d\n",
                control->blocks_n, control->block_size );
#endif

        Cuda_BO_Part1 <<< control->blocks_n, control->block_size >>>
            ( system->d_my_atoms, system->reax_param.d_sbp, 
              *(workspace->d_workspace), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        Cuda_BO_Part2 <<< control->blocks_n, control->block_size >>>
            ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, 
              system->reax_param.d_tbp, *(workspace->d_workspace), 
              *(lists[BONDS]),
              system->reax_param.num_atom_types, system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        Cuda_BO_Part3 <<< control->blocks_n, control->block_size >>>
            ( *(workspace->d_workspace), *(lists[BONDS]), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        Cuda_BO_Part4 <<< control->blocks_n, control->block_size >>>
            ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, 
             *(workspace->d_workspace), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

#if defined(DEBUG_FOCUS)
        t_elapsed = Get_Timing_Info( t_start );

        fprintf( stderr, "Bond Orders... return value --> %d --- Timing %lf \n",
                hipGetLastError( ), t_elapsed );
        fprintf( stderr, "Cuda_Calculate_Bond_Orders Done... \n" );
#endif

        /* 2. Bond Energy Interactions */
#if defined(DEBUG_FOCUS)
        t_start = Get_Time( );
#endif

        cuda_memset( spad, 0, sizeof(real) * 2 * system->N,
                "Compute_Bonded_Forces::spad" );

        Cuda_Bonds <<< control->blocks, control->block_size, sizeof(real) * control->block_size >>>
            ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, system->reax_param.d_tbp,
              *(workspace->d_workspace), *(lists[BONDS]), 
              system->n, system->reax_param.num_atom_types, spad );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* reduction for E_BE */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_bond,
                    system->n );
        }

#if defined(DEBUG_FOCUS)
        t_elapsed = Get_Timing_Info( t_start );

        fprintf( stderr, "Cuda_Bond_Energy ... return value --> %d --- Timing %lf \n",
                hipGetLastError( ), t_elapsed );
        fprintf( stderr, "Cuda_Bond_Energy Done... \n" );
#endif

        /* 3. Atom Energy Interactions */
#if defined(DEBUG_FOCUS)
        t_start = Get_Time( );
#endif

        cuda_memset( spad, 0, sizeof(real) * 3 * system->n,
                "Compute_Bonded_Forces::spad" );

        Cuda_Atom_Energy_Part1 <<< control->blocks, control->block_size >>>
            ( system->d_my_atoms, system->reax_param.d_gp,
              system->reax_param.d_sbp, system->reax_param.d_tbp, *(workspace->d_workspace),
              *(lists[BONDS]), system->n, system->reax_param.num_atom_types,
              spad, &spad[system->n], &spad[2 * system->n] );
        hipDeviceSynchronize( );
        cudaCheckError( );

        Cuda_Atom_Energy_Part2 <<< control->blocks, control->block_size >>>
            ( *(lists[BONDS]), *(workspace->d_workspace), system->n );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* reduction for E_Lp */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_lp,
                    system->n );
        }

        /* reduction for E_Ov */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( &spad[system->n],
                    &((simulation_data *)data->d_simulation_data)->my_en.e_ov,
                    system->n );
        }

        /* reduction for E_Un */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( &spad[2 * system->n],
                    &((simulation_data *)data->d_simulation_data)->my_en.e_un,
                    system->n );
        }

#if defined(DEBUG_FOCUS)
        t_elapsed = Get_Timing_Info( t_start );

        fprintf( stderr, "test_LonePair_postprocess ... return value --> %d --- Timing %lf \n",
                hipGetLastError( ), t_elapsed );
        fprintf( stderr, "test_LonePair_postprocess Done... \n");
#endif

        compute_bonded_part1 = TRUE;
    }

    /* 4. Valence Angles Interactions */
#if defined(DEBUG_FOCUS)
    t_start = Get_Time( );
#endif

    thbody = (int *) workspace->scratch;
    ret = Cuda_Estimate_Storage_Three_Body( system, control, workspace,
            data->step, lists, thbody );

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "system->total_thbodies = %d, lists:THREE_BODIES->max_intrs = %d,\n",
            system->total_thbodies, lists[THREE_BODIES]->max_intrs );
    fprintf( stderr, "lists:THREE_BODIES->n = %d, lists:BONDS->max_intrs = %d,\n",
            lists[THREE_BODIES]->n, lists[BONDS]->max_intrs );
    fprintf( stderr, "system->total_thbodies = %d\n", system->total_thbodies );
#endif

    if ( ret == SUCCESS )
    {
        Cuda_Init_Three_Body_Indices( thbody, system->total_thbodies_indices, lists );

        cuda_memset( spad, 0, 6 * sizeof(real) * system->N + sizeof(rvec) * system->N * 2,
                "Cuda_Compute_Bonded_Forces::spad" );

        Cuda_Valence_Angles_Part1 <<< control->blocks_n, control->block_size >>>
            ( system->d_my_atoms, system->reax_param.d_gp, 
              system->reax_param.d_sbp, system->reax_param.d_thbp, 
              (control_params *)control->d_control_params,
              *(workspace->d_workspace), *(lists[BONDS]), *(lists[THREE_BODIES]),
              system->n, system->N, system->reax_param.num_atom_types, 
              spad, &spad[2 * system->N], &spad[4 * system->N], (rvec *)(&spad[6 * system->N]) );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* reduction for E_Ang */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_ang,
                    system->N );
        }

        /* reduction for E_Pen */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( &spad[2 * system->N],
                    &((simulation_data *)data->d_simulation_data)->my_en.e_pen,
                    system->N );
        }

        /* reduction for E_Coa */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( &spad[4 * system->N],
                    &((simulation_data *)data->d_simulation_data)->my_en.e_coa,
                    system->N );
        }

        /* reduction for ext_pres */
        rvec_spad = (rvec *) (&spad[6 * system->N]);
        k_reduction_rvec <<< control->blocks_n, control->block_size, sizeof(rvec) * control->block_size >>>
            ( rvec_spad, rvec_spad + system->N,  system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        k_reduction_rvec <<< 1, control->blocks_pow_2_n, sizeof(rvec) * control->blocks_pow_2_n >>>
            ( rvec_spad + system->N, &((simulation_data *)data->d_simulation_data)->my_ext_press, control->blocks_n );
        hipDeviceSynchronize ();
        cudaCheckError( );
//        Cuda_Reduction_Sum( rvec_spad,
//                &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                system->N );

        Cuda_Valence_Angles_Part2 <<< control->blocks_n, control->block_size >>>
            ( system->d_my_atoms, (control_params *)control->d_control_params,
              *(workspace->d_workspace), *(lists[BONDS]), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

#if defined(DEBUG_FOCUS)
        t_elapsed = Get_Timing_Info( t_start );

        fprintf( stderr, "Three_Body_Interactions ...  Timing %lf \n",
                t_elapsed );
        fprintf( stderr, "Three_Body_Interactions Done... \n" );
#endif

        /* 5. Torsion Angles Interactions */
#if defined(DEBUG_FOCUS)
        t_start = Get_Time( );
#endif

        cuda_memset( spad, 0, 4 * sizeof(real) * system->n + sizeof(rvec) * system->n * 2,
                "Cuda_Compute_Bonded_Forces::spad" );

        Cuda_Torsion_Angles_Part1 <<< control->blocks, control->block_size >>>
            ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_fbp,
              (control_params *) control->d_control_params, *(lists[BONDS]),
              *(lists[THREE_BODIES]), *(workspace->d_workspace), system->n,
              system->reax_param.num_atom_types, 
              spad, &spad[2 * system->n], (rvec *) (&spad[4 * system->n]) );
        hipDeviceSynchronize( );
        cudaCheckError( );

        /* reduction for E_Tor */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_tor,
                    system->n );
        }

        /* reduction for E_Con */
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( &spad[2 * system->n],
                    &((simulation_data *)data->d_simulation_data)->my_en.e_con,
                    system->n );
        }

        /* reduction for ext_pres */
        rvec_spad = (rvec *) (&spad[4 * system->n]);
        k_reduction_rvec <<< control->blocks, control->block_size, sizeof(rvec) * control->block_size >>>
            ( rvec_spad, rvec_spad + system->n,  system->n );
        hipDeviceSynchronize( );
        cudaCheckError( );

        k_reduction_rvec <<< 1, control->blocks_pow_2, sizeof(rvec) * control->blocks_pow_2 >>>
                ( rvec_spad + system->n,
                  &((simulation_data *)data->d_simulation_data)->my_ext_press, control->blocks );
        hipDeviceSynchronize( );
        cudaCheckError( );
//        Cuda_Reduction_Sum( rvec_spad,
//                &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                system->n );

        Cuda_Torsion_Angles_Part2 <<< control->blocks_n, control->block_size >>>
                ( system->d_my_atoms, *(workspace->d_workspace), *(lists[BONDS]),
                  system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

#if defined(DEBUG_FOCUS)
        t_elapsed = Get_Timing_Info( t_start );

        fprintf( stderr, "Four_Body_post process return value --> %d --- Four body Timing %lf \n",
                hipGetLastError( ), t_elapsed );
        fprintf( stderr, " Four_Body_ Done... \n");
#endif

        /* 6. Hydrogen Bonds Interactions */
        if ( control->hbond_cut > 0.0 && system->numH > 0 )
        {
#if defined(DEBUG_FOCUS)
            t_start = Get_Time( );
#endif

            cuda_memset( spad, 0,
                    2 * sizeof(real) * system->n + sizeof(rvec) * system->n * 2,
                    "Cuda_Compute_Bonded_Forces::spad" );

//            hbs = (system->n * HB_KER_THREADS_PER_ATOM / HB_BLOCK_SIZE) + 
//                (((system->n * HB_KER_THREADS_PER_ATOM) % HB_BLOCK_SIZE) == 0 ? 0 : 1);

            Cuda_Hydrogen_Bonds <<< control->blocks, control->block_size >>>
//            Cuda_Hydrogen_Bonds_MT <<< hbs, HB_BLOCK_SIZE, 
//                    HB_BLOCK_SIZE * (2 * sizeof(real) + 2 * sizeof(rvec)) >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp,
                      system->reax_param.d_hbp, system->reax_param.d_gp,
                      (control_params *) control->d_control_params,
                      *(workspace->d_workspace),
                      *(lists[FAR_NBRS]), *(lists[BONDS]), *(lists[HBONDS]),
                      system->n, system->reax_param.num_atom_types,
                      spad, (rvec *) (&spad[2 * system->n]), system->my_rank, data->step );
            hipDeviceSynchronize( );
            cudaCheckError( );

//            if ( data->step == 10 )
//            {
//                Print_HBonds( system, data->step );
//            }

            /* reduction for E_HB */
            if ( update_energy == TRUE )
            {
                Cuda_Reduction_Sum( spad,
                        &((simulation_data *)data->d_simulation_data)->my_en.e_hb,
                        system->n );
            }

            /* reduction for ext_pres */
            rvec_spad = (rvec *) (&spad[2 * system->n]);
            k_reduction_rvec <<< control->blocks, control->block_size, sizeof(rvec) * control->block_size >>>
                (rvec_spad, rvec_spad + system->n,  system->n);
            hipDeviceSynchronize( );
            cudaCheckError( );

            k_reduction_rvec <<< 1, control->blocks_pow_2, sizeof(rvec) * control->blocks_pow_2 >>>
                ( rvec_spad + system->n,
                  &((simulation_data *)data->d_simulation_data)->my_ext_press,
                  control->blocks );
            hipDeviceSynchronize( );
            cudaCheckError( );
//            Cuda_Reduction_Sum( rvec_spad,
//                    &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                    system->n );

            /* post process step1 */
            Cuda_Hydrogen_Bonds_PostProcess <<< control->blocks_n, control->block_size, control->block_size * sizeof(rvec) >>>
                ( system->d_my_atoms, *(workspace->d_workspace),
                  *(lists[BONDS]), system->N );
            hipDeviceSynchronize( );
            cudaCheckError( );

            /* post process step2 */
//            hnbrs_blocks = (system->N * HB_POST_PROC_KER_THREADS_PER_ATOM / HB_POST_PROC_BLOCK_SIZE) +
//                (((system->N * HB_POST_PROC_KER_THREADS_PER_ATOM) % HB_POST_PROC_BLOCK_SIZE) == 0 ? 0 : 1);

            Cuda_Hydrogen_Bonds_HNbrs <<< system->N, 32, 32 * sizeof(rvec) >>>
                ( system->d_my_atoms, *(workspace->d_workspace), *(lists[HBONDS]) );
//            Cuda_Hydrogen_Bonds_HNbrs_BL <<< hnbrs_blocks, HB_POST_PROC_BLOCK_SIZE, 
//                    HB_POST_PROC_BLOCK_SIZE * sizeof(rvec) >>>
//                ( system->d_my_atoms, *(workspace->d_workspace), *(lists[HBONDS]), system->N );
            hipDeviceSynchronize( );
            cudaCheckError( );

#if defined(DEBUG_FOCUS)
            t_elapsed = Get_Timing_Info( t_start );

            fprintf( stderr,
                    "Hydrogen bonds return value --> %d --- HydrogenBonds Timing %lf \n",
                    hipGetLastError( ), t_elapsed );
            fprintf( stderr, "Hydrogen_Bond Done... \n" );
#endif
        }

        compute_bonded_part1 = FALSE;
    }

    return ret;
}


void Cuda_Compute_NonBonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control,
        mpi_datatypes *mpi_data )
{
    Cuda_NonBonded_Energy( system, control, workspace, data,
            lists, out_control, (control->tabulate == 0) ? false: true );
}


void Cuda_Compute_Total_Force( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, mpi_datatypes *mpi_data )
{
    rvec *f;

    f = (rvec *) workspace->host_scratch;
    memset( f, 0, sizeof(rvec) * system->N );

    Cuda_Total_Forces( system, control, data, workspace, lists );

#if defined(PURE_REAX)
    /* now all forces are computed to their partially-final values
     * based on the neighbors information each processor has had.
     * final values of force on each atom needs to be computed by adding up
     * all partially-final pieces */
    copy_host_device( f, workspace->d_workspace->f, sizeof(rvec) * system->N ,
            hipMemcpyDeviceToHost, "Cuda_Compute_Total_Force::workspace->d_workspace->f" );

    Coll( system, mpi_data, f, RVEC_PTR_TYPE, mpi_data->mpi_rvec );

    copy_host_device( f, workspace->d_workspace->f, sizeof(rvec) * system->N,
            hipMemcpyHostToDevice, "Cuda_Compute_Total_Force::workspace->d_workspace->f" );

    Cuda_Total_Forces_PURE( system, workspace );
#endif

}


int Cuda_Compute_Forces( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace, reax_list **lists,
        output_controls *out_control, mpi_datatypes *mpi_data )
{
    int charge_flag, retVal;
    static int init_forces_done = FALSE;

#if defined(LOG_PERFORMANCE)
    real t_start = 0;

    if ( system->my_rank == MASTER_NODE )
    {
        t_start = Get_Time( );
    }
#endif

    retVal = SUCCESS;

    if ( control->charge_freq && (data->step - data->prev_steps) % control->charge_freq == 0 )
    {
        charge_flag = TRUE;
    }
    else
    {
        charge_flag = FALSE;
    }

    if ( init_forces_done == FALSE )
    {
        if ( charge_flag == TRUE )
        {
            retVal = Cuda_Init_Forces( system, control, data, workspace, lists, out_control );
        }
        else
        {
            retVal = Cuda_Init_Forces_No_Charges( system, control, data, workspace, lists, out_control );
        }

        if ( retVal == SUCCESS )
        {
            init_forces_done = TRUE;
        }
    }

    if ( retVal == SUCCESS )
    {
#if defined(LOG_PERFORMANCE)
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &data->timing.init_forces );
        }
#endif

        retVal = Cuda_Compute_Bonded_Forces( system, control, data,
                workspace, lists, out_control );

#if defined(LOG_PERFORMANCE)
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &data->timing.bonded );
        }
#endif
    }

    if ( retVal == SUCCESS )
    {
#if defined(PURE_REAX)
        if ( charge_flag == TRUE )
        {
            Cuda_Compute_Charges( system, control, data, workspace, out_control, mpi_data );
        }

#if defined(LOG_PERFORMANCE)
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &data->timing.cm );
        }
#endif

#endif //PURE_REAX

        Cuda_Compute_NonBonded_Forces( system, control, data, workspace,
                lists, out_control, mpi_data );

#if defined(LOG_PERFORMANCE)
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &data->timing.nonb );
        }
#endif

        Cuda_Compute_Total_Force( system, control, data, workspace, lists, mpi_data );

#if defined(LOG_PERFORMANCE)
        if ( system->my_rank == MASTER_NODE )
        {
            Update_Timing_Info( &t_start, &data->timing.bonded );
        }
#endif

        init_forces_done = FALSE;
    }

    return retVal;
}

#include "cuda_environment.h"

#include "cuda_utils.h"


static void compute_blocks( int *blocks, int *block_size, int threads )
{
    *block_size = DEF_BLOCK_SIZE; // threads per block
    *blocks = (threads + (DEF_BLOCK_SIZE - 1)) / DEF_BLOCK_SIZE; // blocks per grid
}


static void compute_nearest_multiple_32( int blocks, int *result )
{
    *result = ((blocks + 31) / 32) * 32;
}


extern "C" void Cuda_Setup_Environment( reax_system const * const system,
        control_params * const control )
{

    int i, least_priority, greatest_priority, deviceCount;
    hipError_t ret;
    
    ret = hipGetDeviceCount( &deviceCount );

    if ( ret != hipSuccess || deviceCount < 1 )
    {
        fprintf( stderr, "[ERROR] no CUDA capable device(s) found. Terminating...\n" );
        exit( CANNOT_INITIALIZE );
    }
    else if ( deviceCount < control->gpus_per_node || control->gpus_per_node < 1 )
    {
        fprintf( stderr, "[ERROR] invalid number of CUDA capable devices requested (gpus_per_node = %d). Terminating...\n",
                control->gpus_per_node );
        exit( INVALID_INPUT );
    }

    /* assign the GPU for each process */
    //TODO: handle condition where # CPU procs > # GPUs
    ret = hipSetDevice( system->my_rank % control->gpus_per_node );

    if ( ret == hipErrorInvalidDevice )
    {
        fprintf( stderr, "[ERROR] invalid CUDA device ID set (%d). Terminating...\n",
              system->my_rank % control->gpus_per_node );
        exit( CANNOT_INITIALIZE );
    }
    else if ( ret == hipErrorContextAlreadyInUse )
    {
        fprintf( stderr, "[ERROR] CUDA device with specified ID already in use (%d). Terminating...\n",
                system->my_rank % control->gpus_per_node );
        exit( CANNOT_INITIALIZE );
    }

    ret = hipDeviceGetStreamPriorityRange( &least_priority, &greatest_priority );

    if ( ret != hipSuccess )
    {
        fprintf( stderr, "[ERROR] CUDA strema priority query failed. Terminating...\n" );
        exit( CANNOT_INITIALIZE );
    }

    /* stream assignment (default to 0 for any kernel not listed):
     * 0: init dist, init CM, bond order (uncorrected/corrected), lone pair/over coord/under coord
     * 1: (after init dist) init bonds, (after bond order) bonds, valence angles, torsions
     * 2: (after init dist) init hbonds, (after bonds) hbonds
     * 3: (after init dist) van der Waals
     * 4: (after init CM) CM, Coulomb
     */
    for ( i = 0; i < MAX_CUDA_STREAMS; ++i )
    {
        /* all non-CM streams of equal priority */
        if ( i < MAX_CUDA_STREAMS - 1 )
        {
            ret = hipStreamCreateWithPriority( &control->streams[i], hipStreamNonBlocking, least_priority );
        }
        /* CM gets highest priority due to MPI comms and hipMemcpy's */
        else
        {
            ret = hipStreamCreateWithPriority( &control->streams[i], hipStreamNonBlocking, greatest_priority );
        }

        if ( ret != hipSuccess )
        {
            fprintf( stderr, "[ERROR] CUDA stream creation failed (%d). Terminating...\n",
                    i );
            exit( CANNOT_INITIALIZE );
        }
    }

    /* stream event assignment:
     * 0: init dist done (stream 0)
     * 1: init CM done (stream 4)
     * 2: init bonds done (stream 1)
     * 3: init hbonds done (stream 2)
     * 4: bond orders done (stream 0)
     * 5: bonds done (stream 1)
     */
    for ( i = 0; i < MAX_CUDA_STREAM_EVENTS; ++i )
    {
        ret = hipEventCreateWithFlags( &control->stream_events[i], hipEventDisableTiming );

        if ( ret != hipSuccess )
        {
            fprintf( stderr, "[ERROR] CUDA event creation failed (%d). Terminating...\n",
                    i );
            exit( CANNOT_INITIALIZE );
        }
    }

    //TODO: revisit additional device configurations
//    hipDeviceSetLimit( hipLimitStackSize, 8192 );
//    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
}


extern "C" void Cuda_Init_Block_Sizes( reax_system *system,
        control_params *control )
{
    compute_blocks( &control->blocks, &control->block_size, system->n );
    compute_nearest_multiple_32( control->blocks, &control->blocks_pow_2 );

    compute_blocks( &control->blocks_n, &control->block_size_n, system->N );
    compute_nearest_multiple_32( control->blocks_n, &control->blocks_pow_2_n );
}


extern "C" void Cuda_Cleanup_Environment( control_params const * const control )
{
    int i;
    hipError_t ret;

    for ( i = 0; i < MAX_CUDA_STREAMS; ++i )
    {
        ret = hipStreamDestroy( control->streams[i] );

        if ( ret != hipSuccess )
        {
            fprintf( stderr, "[ERROR] CUDA strema destruction failed (%d). Terminating...\n",
                    i );
            exit( CANNOT_INITIALIZE );
        }
    }

    hipDeviceReset( );
    hipDeviceSynchronize( );
}

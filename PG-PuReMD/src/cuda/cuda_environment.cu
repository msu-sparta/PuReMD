#include "cuda_environment.h"

#include "cuda_utils.h"


static void compute_blocks( int *blocks, int *block_size, int threads )
{
    *block_size = DEF_BLOCK_SIZE; // threads per block
    *blocks = (threads + (DEF_BLOCK_SIZE - 1)) / DEF_BLOCK_SIZE; // blocks per grid
}


static void compute_nearest_multiple_32( int blocks, int *result )
{
    *result = ((blocks + 31) / 32) * 32;
}


extern "C" void Cuda_Setup_Environment( reax_system const * const system,
        control_params * const control )
{
    int i, least_priority, greatest_priority, is_stream_priority_supported;
    int deviceCount;
    hipError_t ret;
    
    ret = hipGetDeviceCount( &deviceCount );

    if ( ret != hipSuccess || deviceCount < 1 )
    {
        fprintf( stderr, "[ERROR] no CUDA capable device(s) found. Terminating...\n" );
        exit( CANNOT_INITIALIZE );
    }
    else if ( deviceCount < control->gpus_per_node || control->gpus_per_node < 1 )
    {
        fprintf( stderr, "[ERROR] invalid number of CUDA capable devices requested (gpus_per_node = %d). Terminating...\n",
                control->gpus_per_node );
        exit( INVALID_INPUT );
    }

    /* assign the GPU for each process */
    //TODO: handle condition where # CPU procs > # GPUs
    ret = hipSetDevice( system->my_rank % control->gpus_per_node );

    if ( ret == hipErrorInvalidDevice )
    {
        fprintf( stderr, "[ERROR] invalid CUDA device ID set (%d). Terminating...\n",
              system->my_rank % control->gpus_per_node );
        exit( CANNOT_INITIALIZE );
    }
    else if ( ret == hipErrorContextAlreadyInUse )
    {
        fprintf( stderr, "[ERROR] CUDA device with specified ID already in use (%d). Terminating...\n",
                system->my_rank % control->gpus_per_node );
        exit( CANNOT_INITIALIZE );
    }

    ret = hipDeviceGetAttribute( &is_stream_priority_supported,
            hipDeviceAttributeStreamPrioritiesSupported,
            system->my_rank % control->gpus_per_node );

    if ( ret != hipSuccess )
    {
        fprintf( stderr, "[ERROR] hipDeviceGetAttribute failure. Terminating...\n" );
        exit( CANNOT_INITIALIZE );
    }

    if ( is_stream_priority_supported == 1 )
    {
        ret = hipDeviceGetStreamPriorityRange( &least_priority, &greatest_priority );
    
        if ( ret != hipSuccess )
        {
            fprintf( stderr, "[ERROR] CUDA stream priority query failed. Terminating...\n" );
            exit( CANNOT_INITIALIZE );
        }
    
        /* stream assignment (default to 0 for any kernel not listed):
         * 0: init dist, (after init bonds) bond order (uncorrected/corrected), lone pair/over coord/under coord
         * 1: (after init dist) init bonds, (after bond order) bonds, valence angles, torsions
         * 2: (after init dist) init hbonds, (after bonds) hbonds
         * 3: (after init dist) van der Waals
         * 4: init CM, CM, Coulomb
         */
        for ( i = MAX_CUDA_STREAMS - 1; i >= 0; --i )
        {
            if ( MAX_CUDA_STREAMS - 1 - i < control->gpu_streams )
            {
                /* all non-CM streams of equal priority */
                if ( i != MAX_CUDA_STREAMS - 1 )
                {
                    ret = hipStreamCreateWithPriority( &control->streams[i], hipStreamNonBlocking, least_priority );
                }
                /* CM gets highest priority due to MPI comms and hipMemcpy's */
                else
                {
                    ret = hipStreamCreateWithPriority( &control->streams[i], hipStreamNonBlocking, greatest_priority );
                }
        
                if ( ret != hipSuccess )
                {
                    fprintf( stderr, "[ERROR] hipStreamCreateWithPriority failure (%d). Terminating...\n",
                            i );
                    exit( CANNOT_INITIALIZE );
                }
            }
            else
            {
                control->streams[i] = control->streams[MAX_CUDA_STREAMS - 1 - ((MAX_CUDA_STREAMS - 1 - i) % control->gpu_streams)];
            }
        }
    }
    else
    {
        /* stream assignment (default to 0 for any kernel not listed):
         * 0: init dist, bond order (uncorrected/corrected), lone pair/over coord/under coord
         * 1: (after init dist) init bonds, (after bond order) bonds
         * 2: (after init dist) init hbonds, (after bond order) hbonds
         * 3: (after bond order) valence angles, torsions
         * 4: (after init dist) van der Waals
         * 5: (after init dist) init CM, CM, Coulomb
         */
        for ( i = MAX_CUDA_STREAMS - 1; i >= 0; --i )
        {
            if ( MAX_CUDA_STREAMS - 1 - i < control->gpu_streams )
            {
                ret = hipStreamCreateWithFlags( &control->streams[i], hipStreamNonBlocking );
        
                if ( ret != hipSuccess )
                {
                    fprintf( stderr, "[ERROR] hipStreamCreateWithFlags failure (%d). Terminating...\n",
                            i );
                    exit( CANNOT_INITIALIZE );
                }
            }
            else
            {
                control->streams[i] = control->streams[MAX_CUDA_STREAMS - 1 - ((MAX_CUDA_STREAMS - 1 - i) % control->gpu_streams)];
            }
       }
    }

    for ( i = 0; i < CUDA_STREAM_SYNC_EVENT_N; ++i )
    {
        ret = hipEventCreateWithFlags( &control->stream_events[i], hipEventDisableTiming );

        if ( ret != hipSuccess )
        {
            fprintf( stderr, "[ERROR] hipEventCreateWithFlags failure (%d). Terminating...\n",
                    i );
            exit( CANNOT_INITIALIZE );
        }
    }

#if defined(LOG_PERFORMANCE)
    for ( i = 0; i < CUDA_TIMING_EVENT_N; ++i )
    {
        ret = hipEventCreate( &control->time_events[i] );

        if ( ret != hipSuccess )
        {
            fprintf( stderr, "[ERROR] hipEventCreate failure (%d). Terminating...\n", i );
            exit( CANNOT_INITIALIZE );
        }
    }
#endif

    //TODO: revisit additional device configurations
//    hipDeviceSetLimit( hipLimitStackSize, 8192 );
//    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
}


extern "C" void Cuda_Init_Block_Sizes( reax_system *system,
        control_params *control )
{
    compute_blocks( &control->blocks, &control->block_size, system->n );
    compute_nearest_multiple_32( control->blocks, &control->blocks_pow_2 );

    compute_blocks( &control->blocks_n, &control->block_size_n, system->N );
    compute_nearest_multiple_32( control->blocks_n, &control->blocks_pow_2_n );
}


extern "C" void Cuda_Cleanup_Environment( control_params const * const control )
{
    int i;
    hipError_t ret;

    for ( i = MAX_CUDA_STREAMS - 1; i >= 0; --i )
    {
        if ( MAX_CUDA_STREAMS - 1 - i < control->gpu_streams )
        {
            ret = hipStreamDestroy( control->streams[i] );
    
            if ( ret != hipSuccess )
            {
                fprintf( stderr, "[ERROR] CUDA stream destruction failed (%d). Terminating...\n",
                        i );
                exit( CANNOT_INITIALIZE );
            }
        }
    }

    for ( i = 0; i < CUDA_STREAM_SYNC_EVENT_N; ++i )
    {
        ret = hipEventDestroy( control->stream_events[i] );

        if ( ret != hipSuccess )
        {
            fprintf( stderr, "[ERROR] CUDA event destruction failure (%d). Terminating...\n",
                    i );
            exit( RUNTIME_ERROR );
        }
    }

#if defined(LOG_PERFORMANCE)
    for ( i = 0; i < CUDA_TIMING_EVENT_N; ++i )
    {
        ret = hipEventDestroy( control->time_events[i] );

        if ( ret != hipSuccess )
        {
            fprintf( stderr, "[ERROR] CUDA event destruction failure (%d). Terminating...\n", i );
            exit( RUNTIME_ERROR );
        }
    }
#endif
}


extern "C" void Cuda_Print_Mem_Usage( simulation_data const * const data )
{
    int rank;
    size_t total, free;
    hipError_t ret;

    MPI_Comm_rank( MPI_COMM_WORLD, &rank );
    ret = hipMemGetInfo( &free, &total );

    if ( ret != hipSuccess )
    {
        fprintf( stderr,
                "[WARNING] could not get message usage info from device\n"
                "    [INFO] CUDA API error code: %d\n",
                ret );
        return;
    }

    fprintf( stderr, "[INFO] step %d on MPI processor %d, Total: %zu bytes (%7.2f MB) Free %zu bytes (%7.2f MB)\n", 
            data->step, rank,
            total, (long long int) total / (1024.0 * 1024.0),
            free, (long long int) free / (1024.0 * 1024.0) );
    fflush( stderr );
}

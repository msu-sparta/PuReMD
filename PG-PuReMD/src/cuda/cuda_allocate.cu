#include "hip/hip_runtime.h"

#include "cuda_allocate.h"

#include "cuda_allocate.h"
#include "cuda_forces.h"
#include "cuda_list.h"
#include "cuda_neighbors.h"
#include "cuda_utils.h"

#include "../allocate.h"
#include "../index_utils.h"
#include "../tool_box.h"
#include "../vector.h"

extern "C"
{


void Cuda_Allocate_Control( control_params *control )
{
    cuda_malloc( (void **)&control->d_control_params,
            sizeof(control_params), TRUE, "control_params" );
    copy_host_device( control, control->d_control_params,
            sizeof(control_params), hipMemcpyHostToDevice, "control_params" );
}


CUDA_GLOBAL void Init_Nbrs( ivec *nbrs, int N )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if ( index >= N )
    {
        return;
    }

    nbrs[index][0] = -1; 
    nbrs[index][1] = -1; 
    nbrs[index][2] = -1; 
}


void Cuda_Allocate_Grid( reax_system *system )
{
    int total;
//    grid_cell local_cell;
    grid *host = &system->my_grid;
    grid *device = &system->d_my_grid;
//    ivec *nbrs_x = (ivec *) workspace->scratch;

    total = host->ncells[0] * host->ncells[1] * host->ncells[2];
    ivec_Copy( device->ncells, host->ncells );
    rvec_Copy( device->cell_len, host->cell_len );
    rvec_Copy( device->inv_len, host->inv_len );

    ivec_Copy( device->bond_span, host->bond_span );
    ivec_Copy( device->nonb_span, host->nonb_span );
    ivec_Copy( device->vlist_span, host->vlist_span );

    ivec_Copy( device->native_cells, host->native_cells );
    ivec_Copy( device->native_str, host->native_str );
    ivec_Copy( device->native_end, host->native_end );

    device->ghost_cut = host->ghost_cut;
    ivec_Copy( device->ghost_span, host->ghost_span );
    ivec_Copy( device->ghost_nonb_span, host->ghost_nonb_span );
    ivec_Copy( device->ghost_hbond_span, host->ghost_hbond_span );
    ivec_Copy( device->ghost_bond_span, host->ghost_bond_span );

    cuda_malloc( (void **) &device->str, sizeof(int) * total, TRUE,
            "Cuda_Allocate_Grid::grid->str" );
    cuda_malloc( (void **) &device->end, sizeof(int) * total, TRUE,
            "Cuda_Allocate_Grid::grid->end" );
    cuda_malloc( (void **) &device->cutoff, sizeof(real) * total, TRUE,
            "Cuda_Allocate_Grid::grid->cutoff" );

    cuda_malloc( (void **) &device->nbrs_x, sizeof(ivec) * total * host->max_nbrs,
            TRUE, "Cuda_Allocate_Grid::grid->nbrs_x" );
    cuda_malloc( (void **) &device->nbrs_cp, sizeof(rvec) * total * host->max_nbrs,
            TRUE, "Cuda_Allocate_Grid::grid->nbrs_cp" );
    cuda_malloc( (void **) &device->rel_box, sizeof(ivec) * total,
            TRUE, "Cuda_Allocate_Grid::grid->rel_box" );

//    int block_size = 512;
//    int blocks = (host->max_nbrs) / block_size + ((host->max_nbrs) % block_size == 0 ? 0 : 1); 
//
//    Init_Nbrs <<< blocks, block_size >>>
//        ( nbrs_x, host->max_nbrs );
//    hipDeviceSynchronize( );
//    cudaCheckError( );
//
//    cuda_malloc( (void **)& device->cells, sizeof(grid_cell) * total,
//            TRUE, "grid:cells");
//    fprintf( stderr, " Device cells address --> %ld \n", device->cells );
//    cuda_malloc( (void **) &device->order,
//            sizeof(ivec) * (host->total + 1), TRUE, "grid:order" );
//
//    local_cell.top = local_cell.mark = local_cell.str = local_cell.end = 0;
//    fprintf( stderr, "Total cells to be allocated -- > %d \n", total );
//    for (int i = 0; i < total; i++)
//    {
//        //fprintf( stderr, "Address of the local atom -> %ld  \n", &local_cell );
//
//        cuda_malloc( (void **) &local_cell.atoms, sizeof(int) * host->max_atoms,
//                TRUE, "alloc:grid:cells:atoms" );
//        //fprintf( stderr, "Allocated address of the atoms --> %ld  (%d)\n", local_cell.atoms, host->max_atoms );
//
//        cuda_malloc( (void **) &local_cell.nbrs_x, sizeof(ivec) * host->max_nbrs,
//                TRUE, "alloc:grid:cells:nbrs_x" );
//        copy_device( local_cell.nbrs_x, nbrs_x, host->max_nbrs * sizeof(ivec), "grid:nbrs_x" );
//        //fprintf( stderr, "Allocated address of the nbrs_x--> %ld \n", local_cell.nbrs_x );
//
//        cuda_malloc( (void **) &local_cell.nbrs_cp, sizeof(rvec) * host->max_nbrs,
//                TRUE, "alloc:grid:cells:nbrs_cp" );
//        //fprintf( stderr, "Allocated address of the nbrs_cp--> %ld \n", local_cell.nbrs_cp );
//
//        //cuda_malloc( (void **) &local_cell.nbrs, sizeof(grid_cell *) * host->max_nbrs,
//        //                TRUE, "alloc:grid:cells:nbrs" );
//        //fprintf( stderr, "Allocated address of the nbrs--> %ld \n", local_cell.nbrs );
//
//        copy_host_device( &local_cell, &device->cells[i], sizeof(grid_cell),
//                hipMemcpyHostToDevice, "grid:cell-alloc" );
//    }
}


void Cuda_Deallocate_Grid_Cell_Atoms( reax_system *system )
{
    int i, total;
    grid_cell local_cell;
    grid *host, *device;

    host = &system->my_grid;
    device = &system->d_my_grid;
    total = host->ncells[0] * host->ncells[1] * host->ncells[2];

    for ( i = 0; i < total; ++i )
    {
        copy_host_device( &local_cell, &device->cells[i], 
                sizeof(grid_cell), hipMemcpyDeviceToHost,
                "Cuda_Deallocate_Grid_Cell_Atoms::grid" );

        cuda_free( local_cell.atoms,
                "Cuda_Deallocate_Grid_Cell_Atoms::grid_cell.atoms" );
    }
}


void Cuda_Allocate_Grid_Cell_Atoms( reax_system *system, int cap )
{
    int i, total;
    grid_cell local_cell;
    grid *host, *device;

    host = &system->my_grid;
    device = &system->d_my_grid;
    total = host->ncells[0] * host->ncells[1] * host->ncells[2];

    for ( i = 0; i < total; i++ )
    {
        copy_host_device( &local_cell, &device->cells[i], 
                sizeof(grid_cell), hipMemcpyDeviceToHost, "grid:cell-dealloc" );
        cuda_malloc( (void **)&local_cell.atoms, sizeof(int) * cap, 
                TRUE, "realloc:grid:cells:atoms" );
        copy_host_device( &local_cell, &device->cells[i], 
                sizeof(grid_cell), hipMemcpyHostToDevice, "grid:cell-realloc" );
    }
}


void Cuda_Allocate_System( reax_system *system )
{
    /* atoms */
    cuda_malloc( (void **) &system->d_my_atoms,
            system->total_cap * sizeof(reax_atom),
            TRUE, "system:d_my_atoms" );
    cuda_malloc( (void **) &system->d_numH, sizeof(int), TRUE, "system:d_numH" );

    /* list management */
    cuda_malloc( (void **) &system->d_far_nbrs,
            system->total_cap * sizeof(int), TRUE, "system:d_far_nbrs" );
    cuda_malloc( (void **) &system->d_max_far_nbrs,
            system->total_cap * sizeof(int), TRUE, "system:d_max_far_nbrs" );
    cuda_malloc( (void **) &system->d_total_far_nbrs,
            sizeof(int), TRUE, "system:d_total_far_nbrs" );
    cuda_malloc( (void **) &system->d_realloc_far_nbrs,
            sizeof(int), TRUE, "system:d_realloc_far_nbrs" );

    cuda_malloc( (void **) &system->d_bonds,
            system->total_cap * sizeof(int), TRUE, "system:d_bonds" );
    cuda_malloc( (void **) &system->d_max_bonds,
            system->total_cap * sizeof(int), TRUE, "system:d_max_bonds" );
    cuda_malloc( (void **) &system->d_total_bonds,
            sizeof(int), TRUE, "system:d_total_bonds" );
    cuda_malloc( (void **) &system->d_realloc_bonds,
            sizeof(int), TRUE, "system:d_realloc_bonds" );

    cuda_malloc( (void **) &system->d_hbonds,
            system->total_cap * sizeof(int), TRUE, "system:d_hbonds" );
    cuda_malloc( (void **) &system->d_max_hbonds,
            system->total_cap * sizeof(int), TRUE, "system:d_max_hbonds" );
    cuda_malloc( (void **) &system->d_total_hbonds,
            sizeof(int), TRUE, "system:d_total_hbonds" );
    cuda_malloc( (void **) &system->d_realloc_hbonds,
            sizeof(int), TRUE, "system:d_realloc_hbonds" );

    cuda_malloc( (void **) &system->d_cm_entries,
            system->total_cap * sizeof(int), TRUE, "system:d_cm_entries" );
    cuda_malloc( (void **) &system->d_max_cm_entries,
            system->total_cap * sizeof(int), TRUE, "system:d_max_cm_entries" );
    cuda_malloc( (void **) &system->d_total_cm_entries,
            sizeof(int), TRUE, "system:d_total_cm_entries" );
    cuda_malloc( (void **) &system->d_realloc_cm_entries,
            sizeof(int), TRUE, "system:d_realloc_cm_entries" );

    cuda_malloc( (void **) &system->d_total_thbodies,
            sizeof(int), TRUE, "system:d_total_thbodies" );

    /* simulation boxes */
    cuda_malloc( (void **) &system->d_big_box,
            sizeof(simulation_box), TRUE, "system:d_big_box" );
    cuda_malloc( (void **) &system->d_my_box,
            sizeof(simulation_box), TRUE, "system:d_my_box" );
    cuda_malloc( (void **) &system->d_my_ext_box,
            sizeof(simulation_box), TRUE, "d_my_ext_box" );

    /* interaction parameters */
    cuda_malloc( (void **) &system->reax_param.d_sbp,
            system->reax_param.num_atom_types * sizeof(single_body_parameters),
            TRUE, "system:d_sbp" );

    cuda_malloc( (void **) &system->reax_param.d_tbp,
            POW( system->reax_param.num_atom_types, 2.0 ) * sizeof(two_body_parameters), 
            TRUE, "system:d_tbp" );

    cuda_malloc( (void **) &system->reax_param.d_thbp,
            POW( system->reax_param.num_atom_types, 3.0 ) * sizeof(three_body_header),
            TRUE, "system:d_thbp" );

    cuda_malloc( (void **) &system->reax_param.d_hbp,
            POW( system->reax_param.num_atom_types, 3.0 ) * sizeof(hbond_parameters),
            TRUE, "system:d_hbp" );

    cuda_malloc( (void **) &system->reax_param.d_fbp,
            POW( system->reax_param.num_atom_types, 4.0 ) * sizeof(four_body_header),
            TRUE, "system:d_fbp" );

    cuda_malloc( (void **) &system->reax_param.d_gp.l,
            system->reax_param.gp.n_global * sizeof(real), TRUE, "system:d_gp.l" );

    system->reax_param.d_gp.n_global = 0;
    system->reax_param.d_gp.vdw_type = 0;
}


static void Cuda_Reallocate_System( reax_system *system, storage *workspace,
        int old_total_cap, int total_cap )
{
    int *temp;
    reax_atom *temp_atom;

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
            MAX( sizeof(reax_atom), sizeof(int) ) * old_total_cap,
            "Cuda_Reallocate_System::workspace->scratch" );
    temp = (int *) workspace->scratch;
    temp_atom = (reax_atom *) workspace->scratch;

    /* free the existing storage for atoms, leave other info allocated */
    copy_device( temp_atom, system->d_my_atoms, sizeof(reax_atom) * old_total_cap,
            "Cuda_Reallocate_System::temp_atom" );
    cuda_free( system->d_my_atoms, "system::d_my_atoms" );
    cuda_malloc( (void **) &system->d_my_atoms,
            sizeof(reax_atom) * total_cap, TRUE, "Cuda_Reallocate_System::d_my_atoms" );
    copy_device( system->d_my_atoms, temp_atom, sizeof(reax_atom) * old_total_cap,
            "Cuda_Reallocate_System::temp_atom" );

    /* list management */
    copy_device( temp, system->d_far_nbrs, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
    cuda_free( system->d_far_nbrs, "Cuda_Reallocate_System::d_far_nbrs" );
    cuda_malloc( (void **) &system->d_far_nbrs,
            sizeof(int) * system->total_cap, TRUE, "Cuda_Reallocate_System::d_far_nbrs" );
    copy_device( system->d_far_nbrs, temp, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );

    copy_device( temp, system->d_max_far_nbrs, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
    cuda_free( system->d_max_far_nbrs, "Cuda_Reallocate_System::d_max_far_nbrs" );
    cuda_malloc( (void **) &system->d_max_far_nbrs,
            sizeof(int) * system->total_cap, TRUE, "Cuda_Reallocate_System::d_max_far_nbrs" );
    copy_device( system->d_max_far_nbrs, temp, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );

    copy_device( temp, system->d_bonds, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
    cuda_free( system->d_bonds, "Cuda_Reallocate_System::d_bonds" );
    cuda_malloc( (void **) &system->d_bonds,
            sizeof(int) * system->total_cap, TRUE, "Cuda_Reallocate_System::d_bonds" );
    copy_device( system->d_bonds, temp, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );

    copy_device( temp, system->d_max_bonds, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
    cuda_free( system->d_max_bonds, "Cuda_Reallocate_System::d_max_bonds" );
    cuda_malloc( (void **) &system->d_max_bonds,
            sizeof(int) * system->total_cap, TRUE, "Cuda_Reallocate_System::d_max_bonds" );
    copy_device( system->d_max_bonds, temp, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );

    copy_device( temp, system->d_hbonds, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
    cuda_free( system->d_hbonds, "system::d_hbonds" );
    cuda_malloc( (void **) &system->d_hbonds,
            sizeof(int) * system->total_cap, TRUE, "Cuda_Reallocate_System::d_hbonds" );
    copy_device( system->d_hbonds, temp, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );

    copy_device( temp, system->d_max_hbonds, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
    cuda_free( system->d_max_hbonds, "system::d_max_hbonds" );
    cuda_malloc( (void **) &system->d_max_hbonds,
            sizeof(int) * system->total_cap, TRUE, "Cuda_Reallocate_System::d_max_hbonds" );
    copy_device( system->d_max_hbonds, temp, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );

    copy_device( temp, system->d_cm_entries, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
    cuda_free( system->d_cm_entries, "Cuda_Reallocate_System::d_cm_entries" );
    cuda_malloc( (void **) &system->d_cm_entries,
            sizeof(int) * system->total_cap, TRUE, "Cuda_Reallocate_System::d_cm_entries" );
    copy_device( system->d_cm_entries, temp, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );

    copy_device( temp, system->d_max_cm_entries, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
    cuda_free( system->d_max_cm_entries, "Cuda_Reallocate_System::d_max_cm_entries" );
    cuda_malloc( (void **) &system->d_max_cm_entries,
            sizeof(int) * system->total_cap, TRUE, "Cuda_Reallocate_System::d_max_cm_entries" );
    copy_device( system->d_max_cm_entries, temp, sizeof(int) * old_total_cap,
            "Cuda_Reallocate_System::temp" );
}


void Cuda_Allocate_Simulation_Data( simulation_data *data )
{
    cuda_malloc( (void **) &data->d_simulation_data,
            sizeof(simulation_data), TRUE, "simulation_data" );
}


void Cuda_Allocate_Workspace( reax_system *system, control_params *control, 
        storage *workspace, int local_cap, int total_cap )
{
    int total_real, total_rvec, local_rvec;

    workspace->allocated = TRUE;

    total_real = sizeof(real) * total_cap;
    total_rvec = sizeof(rvec) * total_cap;
    local_rvec = sizeof(rvec) * local_cap;

    /* scratch space */
    workspace->scratch = NULL;
    workspace->scratch_size = 0;
    workspace->host_scratch = NULL;
    workspace->host_scratch_size = 0;

    /* bond order related storage  */
    cuda_malloc( (void **) &workspace->total_bond_order, total_real, TRUE, "total_bo" );
    cuda_malloc( (void **) &workspace->Deltap, total_real, TRUE, "Deltap" );
    cuda_malloc( (void **) &workspace->Deltap_boc, total_real, TRUE, "Deltap_boc" );
    cuda_malloc( (void **) &workspace->dDeltap_self, total_rvec, TRUE, "dDeltap_self" );
    cuda_malloc( (void **) &workspace->Delta, total_real, TRUE, "Delta" );
    cuda_malloc( (void **) &workspace->Delta_lp, total_real, TRUE, "Delta_lp" );
    cuda_malloc( (void **) &workspace->Delta_lp_temp, total_real, TRUE, "Delta_lp_temp" );
    cuda_malloc( (void **) &workspace->dDelta_lp, total_real, TRUE, "Delta_lp_temp" );
    cuda_malloc( (void **) &workspace->dDelta_lp_temp, total_real, TRUE, "dDelta_lp_temp" );
    cuda_malloc( (void **) &workspace->Delta_e, total_real, TRUE, "Delta_e" );
    cuda_malloc( (void **) &workspace->Delta_boc, total_real, TRUE, "Delta_boc" );
    cuda_malloc( (void **) &workspace->nlp, total_real, TRUE, "nlp" );
    cuda_malloc( (void **) &workspace->nlp_temp, total_real, TRUE, "nlp_temp" );
    cuda_malloc( (void **) &workspace->Clp, total_real, TRUE, "Clp" );
    cuda_malloc( (void **) &workspace->vlpex, total_real, TRUE, "vlpex" );
    cuda_malloc( (void **) &workspace->bond_mark, total_real, TRUE, "bond_mark" );

    /* charge matrix storage */
    if ( control->cm_solver_pre_comp_type == JACOBI_PC )
    {
        cuda_malloc( (void **) &workspace->Hdia_inv, sizeof(real) * total_cap, TRUE, "Hdia_inv" );
    }
    cuda_malloc( (void **) &workspace->b_s, sizeof(real) * total_cap, TRUE, "b_s" );
    cuda_malloc( (void **) &workspace->b_t, sizeof(real) * total_cap, TRUE, "b_t" );
    cuda_malloc( (void **) &workspace->s, sizeof(real) * total_cap, TRUE, "s" );
    cuda_malloc( (void **) &workspace->t, sizeof(real) * total_cap, TRUE, "t" );
    if ( control->cm_solver_pre_comp_type == ICHOLT_PC
            || control->cm_solver_pre_comp_type == ILUT_PC
            || control->cm_solver_pre_comp_type == ILUTP_PC
            || control->cm_solver_pre_comp_type == FG_ILUT_PC )
    {
        cuda_malloc( (void **) &workspace->droptol, sizeof(real) * total_cap, TRUE, "droptol" );
    }
#if defined(DUAL_SOLVER)
    cuda_malloc( (void **) &workspace->b, sizeof(rvec2) * total_cap, TRUE, "b" );
    cuda_malloc( (void **) &workspace->x, sizeof(rvec2) * total_cap, TRUE, "x" );
#endif

    switch ( control->cm_solver_type )
    {
    case GMRES_S:
    case GMRES_H_S:
        cuda_malloc( (void **) &workspace->b_prc,
                sizeof(real) * total_cap, TRUE, "b_prc" );
        cuda_malloc( (void **) &workspace->b_prm,
                sizeof(real) * total_cap, TRUE, "b_prm" );
        cuda_malloc( (void **) &workspace->y,
                (control->cm_solver_restart + 1) * sizeof(real), TRUE, "y" );
        cuda_malloc( (void **) &workspace->z,
                (control->cm_solver_restart + 1) * sizeof(real), TRUE, "z" );
        cuda_malloc( (void **) &workspace->g,
                (control->cm_solver_restart + 1) * sizeof(real), TRUE, "g" );
        cuda_malloc( (void **) &workspace->h,
                SQR(control->cm_solver_restart + 1) * sizeof(real), TRUE, "h" );
        cuda_malloc( (void **) &workspace->hs,
                (control->cm_solver_restart + 1) * sizeof(real), TRUE, "hs" );
        cuda_malloc( (void **) &workspace->hc,
                (control->cm_solver_restart + 1) * sizeof(real), TRUE, "hc" );
        cuda_malloc( (void **) &workspace->v,
                SQR(control->cm_solver_restart + 1) * sizeof(real), TRUE, "v" );
        break;

    case SDM_S:
        break;

    case CG_S:
        cuda_malloc( (void **) &workspace->r, sizeof(real) * total_cap, TRUE, "r" );
        cuda_malloc( (void **) &workspace->d, sizeof(real) * total_cap, TRUE, "d" );
        cuda_malloc( (void **) &workspace->q, sizeof(real) * total_cap, TRUE, "q" );
        cuda_malloc( (void **) &workspace->p, sizeof(real) * total_cap, TRUE, "p" );
#if defined(DUAL_SOLVER)
        cuda_malloc( (void **) &workspace->r2, sizeof(rvec2) * total_cap, TRUE, "r2" );
        cuda_malloc( (void **) &workspace->d2, sizeof(rvec2) * total_cap, TRUE, "d2" );
        cuda_malloc( (void **) &workspace->q2, sizeof(rvec2) * total_cap, TRUE, "q2" );
        cuda_malloc( (void **) &workspace->p2, sizeof(rvec2) * total_cap, TRUE, "p2" );
#endif
        break;

    default:
        fprintf( stderr, "[ERROR] Unknown charge method linear solver type. Terminating...\n" );
        exit( INVALID_INPUT );
        break;
    }

    /* integrator storage */
    if ( control->ensemble == nhNVT )
    {
        cuda_malloc( (void **) &workspace->v_const, local_rvec, TRUE, "v_const" );
    }

    /* storage for analysis */
    if ( control->molecular_analysis || control->diffusion_coef )
    {
        cuda_malloc( (void **) &workspace->mark, local_cap * sizeof(int), TRUE, "mark" );
        cuda_malloc( (void **) &workspace->old_mark, local_cap * sizeof(int), TRUE, "old_mark" );
    }
    else
    {
        workspace->mark = NULL;
        workspace->old_mark = NULL;
    }

    if( control->diffusion_coef )
    {
        cuda_malloc( (void **) &workspace->x_old, local_cap * sizeof(rvec), TRUE, "x_old" );
    }
    else
    {
        workspace->x_old = NULL;
    }

    /* force related storage */
    cuda_malloc( (void **) &workspace->f, sizeof(rvec) * total_cap, TRUE, "f" );
    cuda_malloc( (void **) &workspace->CdDelta, sizeof(rvec) * total_cap, TRUE, "CdDelta" );

    /* Taper params */
    cuda_malloc( (void **) &workspace->Tap, sizeof(real) * 8, TRUE, "Tap" );
}


void Cuda_Deallocate_Workspace( control_params *control, storage *workspace )
{
    if ( workspace->allocated == FALSE )
    {
        return;
    }

    workspace->allocated = FALSE;

    /* bond order related storage  */
    cuda_free( workspace->total_bond_order, "total_bo" );
    cuda_free( workspace->Deltap, "Deltap" );
    cuda_free( workspace->Deltap_boc, "Deltap_boc" );
    cuda_free( workspace->dDeltap_self, "dDeltap_self" );
    cuda_free( workspace->Delta, "Delta" );
    cuda_free( workspace->Delta_lp, "Delta_lp" );
    cuda_free( workspace->Delta_lp_temp, "Delta_lp_temp" );
    cuda_free( workspace->dDelta_lp, "Delta_lp_temp" );
    cuda_free( workspace->dDelta_lp_temp, "dDelta_lp_temp" );
    cuda_free( workspace->Delta_e, "Delta_e" );
    cuda_free( workspace->Delta_boc, "Delta_boc" );
    cuda_free( workspace->nlp, "nlp" );
    cuda_free( workspace->nlp_temp, "nlp_temp" );
    cuda_free( workspace->Clp, "Clp" );
    cuda_free( workspace->vlpex, "vlpex" );
    cuda_free( workspace->bond_mark, "bond_mark" );

    /* charge matrix storage */
    if ( control->cm_solver_pre_comp_type == JACOBI_PC )
    {
        cuda_free( workspace->Hdia_inv, "Hdia_inv" );
    }
    if ( control->cm_solver_pre_comp_type == ICHOLT_PC
            || control->cm_solver_pre_comp_type == ILUT_PC
            || control->cm_solver_pre_comp_type == ILUTP_PC
            || control->cm_solver_pre_comp_type == FG_ILUT_PC )
    {
        cuda_free( workspace->droptol, "droptol" );
    }
    cuda_free( workspace->b_s, "b_s" );
    cuda_free( workspace->b_t, "b_t" );
    cuda_free( workspace->s, "s" );
    cuda_free( workspace->t, "t" );
#if defined(DUAL_SOLVER)
    cuda_free( workspace->b, "b" );
    cuda_free( workspace->x, "x" );
#endif

    switch ( control->cm_solver_type )
    {
        case GMRES_S:
        case GMRES_H_S:
            cuda_free( workspace->b_prc, "b_prc" );
            cuda_free( workspace->b_prm, "b_prm" );
            cuda_free( workspace->y, "y" );
            cuda_free( workspace->z, "z" );
            cuda_free( workspace->g, "g" );
            cuda_free( workspace->h, "h" );
            cuda_free( workspace->hs, "hs" );
            cuda_free( workspace->hc, "hc" );
            cuda_free( workspace->v, "v" );
            break;

        case CG_S:
            cuda_free( workspace->r, "r" );
            cuda_free( workspace->d, "d" );
            cuda_free( workspace->q, "q" );
            cuda_free( workspace->p, "p" );
#if defined(DUAL_SOLVER)
            cuda_free( workspace->r2, "r2" );
            cuda_free( workspace->d2, "d2" );
            cuda_free( workspace->q2, "q2" );
            cuda_free( workspace->p2, "p2" );
#endif
            break;

        case SDM_S:
            break;

        default:
            fprintf( stderr, "[ERROR] Unknown charge method linear solver type. Terminating...\n" );
            exit( INVALID_INPUT );
            break;
    }

    /* Nose-Hoover integrator */
    if ( control->ensemble == nhNVT )
    {
        cuda_free( workspace->v_const, "v_const" );
    }

    /* storage for analysis */
    if ( control->molecular_analysis || control->diffusion_coef )
    {
        cuda_free( workspace->mark, "mark" );
        cuda_free( workspace->old_mark, "old_mark" );
    }
    else
    {
        workspace->mark = NULL;
        workspace->old_mark = NULL;
    }

    if ( control->diffusion_coef )
    {
        cuda_free( workspace->x_old, "x_old" );
    }
    else
    {
        workspace->x_old = NULL;
    }

    /* force related storage */
    cuda_free( workspace->f, "f" );
    cuda_free( workspace->CdDelta, "CdDelta" );

    /* Taper params */
    cuda_free( workspace->Tap, "Tap" );
}


/* Allocate sparse matrix struc
 *
 * H: pointer to struct
 * n: currently utilized number of rows
 * n_max: max number of rows allocated
 * m: max number of entries allocated
 * format: sparse matrix format
 */
void Cuda_Allocate_Matrix( sparse_matrix * const H, int n, int n_max, int m,
       int format )
{
    H->allocated = TRUE;
    H->n = n;
    H->n_max = n_max;
    H->m = m;
    H->format = format;

    cuda_malloc( (void **) &H->start, sizeof(int) * n_max, TRUE,
            "Cuda_Allocate_Matrix::H->start" );
    cuda_malloc( (void **) &H->end, sizeof(int) * n_max, TRUE,
            "Cuda_Allocate_Matrix::H->end" );
    cuda_malloc( (void **) &H->j, sizeof(int) * m, TRUE,
            "Cuda_Allocate_Matrix::H->j" );
    cuda_malloc( (void **) &H->val, sizeof(real) * m, TRUE,
            "Cuda_Allocate_Matrix::H->val" );
}


void Cuda_Deallocate_Matrix( sparse_matrix *H )
{
    H->allocated = FALSE;
    H->n = 0;
    H->n_max = 0;
    H->m = 0;

    cuda_free( H->start, "Cuda_Deallocate_Matrix::start" );
    cuda_free( H->end, "Cuda_Deallocate_Matrix::end" );
    cuda_free( H->j, "Cuda_Deallocate_Matrix::j" );
    cuda_free( H->val, "Cuda_Deallocate_Matrix::val" );
}


void Cuda_Reallocate_Neighbor_List( reax_list *far_nbrs, size_t n, size_t max_intrs )
{
    Cuda_Delete_List( far_nbrs );
    Cuda_Make_List( n, max_intrs, TYP_FAR_NEIGHBOR, far_nbrs );
}


void Cuda_Reallocate_HBonds_List( reax_list *hbonds, size_t n, size_t max_intrs )
{
    Cuda_Delete_List( hbonds );
    Cuda_Make_List( n, max_intrs, TYP_HBOND, hbonds );
}


void Cuda_Reallocate_Bonds_List( reax_list *bonds, size_t n, size_t max_intrs )
{
    Cuda_Delete_List( bonds );
    Cuda_Make_List( n, max_intrs, TYP_BOND, bonds );
}


void Cuda_Reallocate_Thbodies_List( reax_list *thbodies, size_t n, size_t max_intrs )
{
    Cuda_Delete_List( thbodies );
    Cuda_Make_List( n, max_intrs, TYP_THREE_BODY, thbodies );

}


void Cuda_ReAllocate( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace, reax_list **lists,
        mpi_datatypes *mpi_data )
{
    int i, j, k;
    int nflag, Nflag, old_total_cap; 
    int renbr, format;
    reallocate_data *realloc;
    reax_list *far_nbrs;
    sparse_matrix *H;
    grid *g;

    realloc = &workspace->d_workspace->realloc;
    g = &system->my_grid;
    H = &workspace->d_workspace->H;

    /* IMPORTANT: LOOSE ZONES CHECKS ARE DISABLED FOR NOW BY &&'ing with FALSE!!! */
    nflag = FALSE;
    if ( system->n >= (int) CEIL( DANGER_ZONE * system->local_cap )
            || (FALSE && system->n <= (int) CEIL( LOOSE_ZONE * system->local_cap )) )
    {
        nflag = TRUE;
        system->local_cap = (int) CEIL( system->n * SAFE_ZONE );
    }

    Nflag = FALSE;
    if ( system->N >= (int) CEIL( DANGER_ZONE * system->total_cap )
            || (FALSE && system->N <= (int) CEIL( LOOSE_ZONE * system->total_cap )) )
    {
        Nflag = TRUE;
        old_total_cap = system->total_cap;
        system->total_cap = (int) CEIL( system->N * SAFE_ZONE );
    }

    if ( Nflag == TRUE )
    {
        /* system */
        Cuda_Reallocate_System( system, workspace, old_total_cap,
                system->total_cap );

        /* workspace */
        Cuda_Deallocate_Workspace( control, workspace );
        Cuda_Allocate_Workspace( system, control, workspace, system->local_cap,
                system->total_cap );
    }

    renbr = (data->step - data->prev_steps) % control->reneighbor == 0 ? TRUE : FALSE;
    /* far neighbors */
    if ( renbr == TRUE && (Nflag == TRUE || realloc->far_nbrs == TRUE) )
    {
        far_nbrs = lists[FAR_NBRS];

        Cuda_Reallocate_Neighbor_List( far_nbrs, system->total_cap, system->total_far_nbrs );
        Cuda_Init_Neighbor_Indices( system, lists );
        realloc->far_nbrs = FALSE;
    }

    /* charge matrix */
    if ( nflag == TRUE || realloc->cm == TRUE )
    {
        format = H->format;

        Cuda_Deallocate_Matrix( H );
        Cuda_Allocate_Matrix( H, system->n, system->local_cap,
                system->total_cm_entries, format );

        Cuda_Init_Sparse_Matrix_Indices( system, H );

        //Deallocate_Matrix( workspace->L );
        //Deallocate_Matrix( workspace->U );
        //workspace->L = NULL;
        //workspace->U = NULL;

        realloc->cm = FALSE;
    }

    /* hydrogen bonds list */
    if ( control->hbond_cut > 0.0 && system->numH > 0
            && (Nflag == TRUE || realloc->hbonds == TRUE) )
    {
        Cuda_Reallocate_HBonds_List( lists[HBONDS], system->total_cap, system->total_hbonds );
        Cuda_Init_HBond_Indices( system, workspace, lists );
        realloc->hbonds = FALSE;
    }

    /* bonds list */
    if ( Nflag == TRUE || realloc->bonds == TRUE )
    {
        Cuda_Reallocate_Bonds_List( lists[BONDS], system->total_cap, system->total_bonds );
        Cuda_Init_Bond_Indices( system, lists );
        realloc->bonds = FALSE;
    }

    /* 3-body list */
    if ( Nflag == TRUE || realloc->thbody == TRUE )
    {
        Cuda_Reallocate_Thbodies_List( lists[THREE_BODIES],
                system->total_thbodies_indices, system->total_thbodies );
        realloc->thbody = FALSE;
    }

    /* grid */
    if ( renbr && realloc->gcell_atoms > -1 )
    {
        for ( i = g->native_str[0]; i < g->native_end[0]; i++ )
        {
            for ( j = g->native_str[1]; j < g->native_end[1]; j++ )
            {
                for ( k = g->native_str[2]; k < g->native_end[2]; k++ )
                {
                    // reallocate g->atoms
                    sfree( g->cells[ index_grid_3d(i,j,k,g) ].atoms, "g:atoms" );
                    g->cells[ index_grid_3d(i,j,k,g) ].atoms = (int*)
                            scalloc( realloc->gcell_atoms, sizeof(int), "g:atoms" );
                }
            }
        }

        //TODO
        //do the same thing for the device here.
        fprintf( stderr, "p:%d - *** Reallocating Grid Cell Atoms *** Step:%d\n", system->my_rank, data->step );
        //MPI_Abort( MPI_COMM_WORLD, INSUFFICIENT_MEMORY );

        //FIX - 1 - Tested the reallocation logic
        //Cuda_Deallocate_Grid_Cell_Atoms( system );
        //Cuda_Allocate_Grid_Cell_Atoms( system, realloc->gcell_atoms );
        realloc->gcell_atoms = -1;
    }
}


}

#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_nonbonded.h"

#include "cuda_helpers.h"
#include "cuda_list.h"
#include "cuda_utils.h"
#include "cuda_reduction.h"

#include "../index_utils.h"
#include "../vector.h"


/* mask used to determine which threads within a warp participate in operations */
#define FULL_MASK (0xFFFFFFFF)


CUDA_GLOBAL void k_compute_polarization_energy( reax_atom *my_atoms, 
        single_body_parameters *sbp, int n, real *data_e_pol )
{
    int i, type_i;
    real q;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    q = my_atoms[i].q;
    type_i = my_atoms[i].type;

    data_e_pol[i] = KCALpMOL_to_EV * (sbp[type_i].chi * q
            + (sbp[type_i].eta / 2.0) * SQR(q));
}


/* one thread per atom implementation */
CUDA_GLOBAL void k_vdW_coulomb_energy( reax_atom *my_atoms, 
        two_body_parameters *tbp, global_parameters gp, control_params *control, 
        storage workspace, reax_list far_nbr_list, int n, int num_atom_types, 
        real *data_e_vdW, real *data_e_ele, rvec *data_ext_press )
{
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j;
    real self_coef;
    real p_vdW1, p_vdW1i;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real Tap, dTap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_ele, e_vdW, e_core, de_core, e_clb, de_clb;
    rvec temp, ext_press;
    two_body_parameters *twbp;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    p_vdW1 = gp.l[28];
    p_vdW1i = 1.0 / p_vdW1;
    e_vdW = 0.0;
    e_ele = 0.0;
    rvec_MakeZero( data_ext_press[i] );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        //TODO: assuming far_nbr_list in FULL_LIST, add conditions for HALF_LIST
        if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut 
                && orig_i < orig_j )
        {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            twbp = &tbp[
                index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types) ];

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            Tap = workspace.Tap[7] * r_ij
                + workspace.Tap[6];
            Tap = Tap * r_ij + workspace.Tap[5];
            Tap = Tap * r_ij + workspace.Tap[4];
            Tap = Tap * r_ij + workspace.Tap[3];
            Tap = Tap * r_ij + workspace.Tap[2];
            Tap = Tap * r_ij + workspace.Tap[1];
            Tap = Tap * r_ij + workspace.Tap[0];

            dTap = 7.0 * workspace.Tap[7] * r_ij
                + 6.0 * workspace.Tap[6];
            dTap = dTap * r_ij + 5.0 * workspace.Tap[5];
            dTap = dTap * r_ij + 4.0 * workspace.Tap[4];
            dTap = dTap * r_ij + 3.0 * workspace.Tap[3];
            dTap = dTap * r_ij + 2.0 * workspace.Tap[2];
            dTap = dTap * r_ij + workspace.Tap[1];

            /* vdWaals Calculations */
            if ( gp.vdw_type == 1 || gp.vdw_type == 3 )
            {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / twbp->gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW += self_coef * (e_base * Tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else
            {
                exp1 = EXP( twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW += self_coef * (e_base * Tap);

                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( gp.vdw_type == 2 || gp.vdw_type == 3 )
            {
                e_core = twbp->ecore * EXP( twbp->acore * (1.0 - (r_ij / twbp->rcore)) );
                e_vdW += self_coef * (e_core * Tap);

                de_core = -(twbp->acore / twbp->rcore) * e_core;
            }
            else
            {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * Tap
                    + (e_base + e_core) * dTap );

#if defined(DEBUG_FOCUS)
            printf( "%6d%6d%24.12f%24.12f%24.12f%24.12f\n",
                    i + 1, j + 1, 
                    e_base, de_base, e_core, de_core );
#endif

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij
                + POW( twbp->gamma, -3.0 );
            dr3gamij_3 = POW( dr3gamij_1 , 1.0 / 3.0 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) / dr3gamij_3;
            e_ele += self_coef * (e_clb * Tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * Tap + e_clb * dTap);

#if defined(DEBUG_FOCUS)
            printf( "%6d%6d%24.12f%24.12f\n",
                    i + 1, j + 1, e_clb, de_clb );
#endif

            if ( control->virial == 0 )
            {
                if ( i < j ) 
                {
                    rvec_Scale( temp, -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                    atomic_rvecAdd( workspace.f[i], temp );
                    rvec_Scale( temp, -1.0, temp );
                    atomic_rvecAdd( workspace.f[j], temp );
                }
                else 
                {
                    rvec_Scale( temp, (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                    atomic_rvecAdd( workspace.f[i], temp );
                    rvec_Scale( temp, -1.0, temp );
                    atomic_rvecAdd( workspace.f[j], temp );
                }
            }
            /* NPT, iNPT or sNPT */
            else
            {
                /* for pressure coupling, terms not related to bond order 
                   derivatives are added directly into pressure vector/tensor */
                if ( i < j ) 
                {
                    rvec_Scale( temp,
                            -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                    atomic_rvecAdd( workspace.f[i], temp );
                    rvec_Scale( temp, -1.0, temp );
                    atomic_rvecAdd( workspace.f[j], temp );
                }
                else 
                {
                    rvec_Scale( temp,
                            (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                    atomic_rvecAdd( workspace.f[i], temp );
                    rvec_Scale( temp, -1.0, temp );
                    atomic_rvecAdd( workspace.f[j], temp );
                }

                rvec_iMultiply( ext_press,
                        far_nbr_list.far_nbr_list.rel_box[pj], temp );
                rvec_Add( data_ext_press[i], ext_press );
            }

#if defined(TEST_ENERGY)
            // fprintf( out_control->evdw, 
            // "%12.9f%12.9f%12.9f%12.9f%12.9f%12.9f%12.9f%12.9f\n", 
            // workspace.Tap[7], workspace.Tap[6], workspace.Tap[5],
            // workspace.Tap[4], workspace.Tap[3], workspace.Tap[2], 
            // workspace.Tap[1], Tap );
            //fprintf( out_control->evdw, "%6d%6d%24.15e%24.15e%24.15e\n",
            fprintf( out_control->evdw, "%6d%6d%12.4f%12.4f%12.4f\n",
                    my_atoms[i].orig_id, my_atoms[j].orig_id, 
                    r_ij, e_vdW, data->my_en.e_vdW );
            //fprintf(out_control->ecou,"%6d%6d%24.15e%24.15e%24.15e%24.15e%24.15e\n",
            fprintf( out_control->ecou, "%6d%6d%12.4f%12.4f%12.4f%12.4f%12.4f\n",
                    my_atoms[i].orig_id, my_atoms[j].orig_id,
                    r_ij, my_atoms[i].q, my_atoms[j].q, 
                    e_ele, data->my_en.e_ele );
#endif

#if defined(TEST_FORCES)
            rvec_ScaledAdd( workspace.f_vdw[i], -CEvd,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_vdw[j], +CEvd,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_ele[i], -CEclmb,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_ele[j], +CEclmb,
                    far_nbr_list.far_nbr_list.dvec[pj] );
#endif
        }
    }

    __syncthreads( );
    data_e_vdW[i] = e_vdW;
    data_e_ele[i] = e_ele;
}


//TODO: fix issue with atomic forces not being correctly accumulated
/* one warp of threads (32) per atom implementation */
CUDA_GLOBAL void k_vdW_coulomb_energy_opt( reax_atom *my_atoms, 
        two_body_parameters *tbp, global_parameters gp, control_params *control, 
        storage workspace, reax_list far_nbr_list, int n, int num_atom_types, 
        real *data_e_vdW, real *data_e_ele, rvec *data_ext_press )
{
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j;
    real self_coef;
    real p_vdW1, p_vdW1i;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real Tap, dTap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_ele, e_vdW, e_core, de_core, e_clb, de_clb;
    rvec temp, ext_press;
    two_body_parameters *twbp;
    int thread_id, warp_id, lane_id, offset;
    unsigned int mask;
    real e_vdW_s, e_ele_s;
    rvec f_s;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id >> 5;
    lane_id = thread_id & 0x0000001F; 
    mask = __ballot_sync( FULL_MASK, i < n );

    if ( warp_id >= n )
    {
        return;
    }

    i = warp_id;
    e_vdW_s = 0.0;
    e_ele_s = 0.0;
    rvec_MakeZero( f_s );

    p_vdW1 = gp.l[28];
    p_vdW1i = 1.0 / p_vdW1;

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    pj = start_i + lane_id;
    while ( pj < end_i )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        //TODO: assuming far_nbr_list in FULL_LIST, add conditions for HALF_LIST
        if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut 
                && orig_i < orig_j )
        {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            twbp = &tbp[
                index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types) ];

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            Tap = workspace.Tap[7] * r_ij
                + workspace.Tap[6];
            Tap = Tap * r_ij + workspace.Tap[5];
            Tap = Tap * r_ij + workspace.Tap[4];
            Tap = Tap * r_ij + workspace.Tap[3];
            Tap = Tap * r_ij + workspace.Tap[2];
            Tap = Tap * r_ij + workspace.Tap[1];
            Tap = Tap * r_ij + workspace.Tap[0];

            dTap = 7.0 * workspace.Tap[7] * r_ij
                + 6.0 * workspace.Tap[6];
            dTap = dTap * r_ij + 5.0 * workspace.Tap[5];
            dTap = dTap * r_ij + 4.0 * workspace.Tap[4];
            dTap = dTap * r_ij + 3.0 * workspace.Tap[3];
            dTap = dTap * r_ij + 2.0 * workspace.Tap[2];
            dTap = dTap * r_ij + workspace.Tap[1];

            /* vdWaals Calculations */
            if ( gp.vdw_type == 1 || gp.vdw_type == 3 )
            {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / twbp->gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW = self_coef * (e_base * Tap);
                e_vdW_s += e_vdW;

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else
            {
                exp1 = EXP( twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW = self_coef * (e_base * Tap);
                e_vdW_s += e_vdW;

                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( gp.vdw_type == 2 || gp.vdw_type == 3 )
            {
                e_core = twbp->ecore * EXP( twbp->acore * (1.0 - (r_ij / twbp->rcore)) );
                e_vdW += self_coef * (e_core * Tap);
                e_vdW_s += (self_coef * (e_core * Tap));

                de_core = -(twbp->acore / twbp->rcore) * e_core;
            }
            else
            {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * Tap
                    + (e_base + e_core) * dTap );

#if defined(DEBUG_FOCUS)
            if ( i < j )
            {
                printf( "%6d%6d%24.12f%24.12f%24.12f%24.12f\n",
                        i + 1, j + 1, 
                        e_base, de_base, e_core, de_core );
            }
            else
            {
                printf( "%6d%6d%24.12f%24.12f%24.12f%24.12f\n",
                        j + 1, i + 1, 
                        e_base, de_base, e_core, de_core );
            }
#endif

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij
                + POW( twbp->gamma, -3.0 );
            dr3gamij_3 = POW( dr3gamij_1 , 1.0 / 3.0 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) / dr3gamij_3;
            e_ele = self_coef * (e_clb * Tap);
            e_ele_s += e_ele;

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * Tap + e_clb * dTap);

#if defined(DEBUG_FOCUS)
            printf( "%6d%6d%24.12f%24.12f\n",
                    i + 1, j + 1, e_clb, de_clb );
#endif

            if ( control->virial == 0 )
            {
                if ( i < j ) 
                {
                    rvec_ScaledAdd( f_s,
                            -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                }
                else 
                {
                    rvec_ScaledAdd( f_s,
                            (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                }
            }
            /* NPT, iNPT or sNPT */
            else
            {
                /* for pressure coupling, terms not related to bond order 
                   derivatives are added directly into pressure vector/tensor */
                rvec_Scale( temp,
                        (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );

                if ( i < j ) 
                {
                    rvec_ScaledAdd( f_s, -1.0, temp );
                }
                else 
                {
                    rvec_Add( f_s, temp );
                }

                rvec_iMultiply( ext_press,
                        far_nbr_list.far_nbr_list.rel_box[pj], temp );
                rvec_Add( data_ext_press[i], ext_press );
            }

#if defined(TEST_ENERGY)
            // fprintf( out_control->evdw, 
            // "%12.9f%12.9f%12.9f%12.9f%12.9f%12.9f%12.9f%12.9f\n", 
            // workspace.Tap[7], workspace.Tap[6], workspace.Tap[5],
            // workspace.Tap[4], workspace.Tap[3], workspace.Tap[2], 
            // workspace.Tap[1], Tap );
            //fprintf( out_control->evdw, "%6d%6d%24.15e%24.15e%24.15e\n",
            fprintf( out_control->evdw, "%6d%6d%12.4f%12.4f%12.4f\n",
                    my_atoms[i].orig_id, my_atoms[j].orig_id, 
                    r_ij, e_vdW, data->my_en.e_vdW );
            //fprintf(out_control->ecou,"%6d%6d%24.15e%24.15e%24.15e%24.15e%24.15e\n",
            fprintf( out_control->ecou, "%6d%6d%12.4f%12.4f%12.4f%12.4f%12.4f\n",
                    my_atoms[i].orig_id, my_atoms[j].orig_id,
                    r_ij, my_atoms[i].q, my_atoms[j].q, 
                    e_ele, data->my_en.e_ele );
#endif

#if defined(TEST_FORCES)
            rvec_ScaledAdd( workspace.f_vdw[i], -CEvd,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_vdw[j], +CEvd,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_ele[i], -CEclmb,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_ele[j], +CEclmb,
                    far_nbr_list.far_nbr_list.dvec[pj] );
#endif
        }

        pj += 32;
    }

    /* warp-level sum using registers within a warp */
    for ( offset = 16; offset > 0; offset /= 2 )
    {
        e_vdW_s += __shfl_down_sync( mask, e_vdW_s, offset );
        e_ele_s += __shfl_down_sync( mask, e_ele_s, offset );
        f_s[0] += __shfl_down_sync( mask, f_s[0], offset );
        f_s[1] += __shfl_down_sync( mask, f_s[1], offset );
        f_s[2] += __shfl_down_sync( mask, f_s[2], offset );
    }

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 )
    {
        data_e_vdW[i] = e_vdW_s;
        data_e_ele[i] = e_ele_s;
        rvec_Add( workspace.f[i], f_s );
    }
}


/* one thread per atom implementation */
CUDA_GLOBAL void k_tabulated_vdW_coulomb_energy( reax_atom *my_atoms, 
        global_parameters gp, control_params *control, 
        storage workspace, reax_list far_nbr_list, 
        LR_lookup_table *t_LR, int n, int num_atom_types, 
        int step, int prev_steps, int energy_update_freq, 
        real *data_e_vdW, real *data_e_ele, rvec *data_ext_press )
{
    int i, j, pj, r, steps, update_freq, update_energies;
    int type_i, type_j, tmin, tmax;
    int start_i, end_i, orig_i, orig_j;
    real r_ij, self_coef, base, dif;
    real e_vdW, e_ele;
    real CEvd, CEclmb;
    rvec temp, ext_press;
    LR_lookup_table *t;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    steps = step - prev_steps;
    update_freq = energy_update_freq;
    update_energies = update_freq > 0 && steps % update_freq == 0;
    e_ele = 0.0;
    e_vdW = 0.0;
    data_e_vdW[i] = 0.0;
    data_e_ele[i] = 0.0;

    type_i = my_atoms[i].type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        //TODO: assuming far_nbr_list in FULL_LIST, add conditions for HALF_LIST
        if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut
                && orig_i < orig_j )
        {
            type_j = my_atoms[j].type;
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            self_coef = (i == j) ? 0.5 : 1.0;
            tmin = MIN( type_i, type_j );
            tmax = MAX( type_i, type_j );
            t = &t_LR[ index_lr(tmin, tmax, num_atom_types) ];

            /* Cubic Spline Interpolation */
            r = (int)(r_ij * t->inv_dx);
            if ( r == 0 )
            {
                ++r;
            }
            base = (real)(r + 1) * t->dx;
            dif = r_ij - base;

            if ( update_energies )
            {
                e_vdW = ((t->vdW[r].d * dif + t->vdW[r].c) * dif + t->vdW[r].b)
                    * dif + t->vdW[r].a;
                e_vdW *= self_coef;

                e_ele = ((t->ele[r].d * dif + t->ele[r].c) * dif + t->ele[r].b)
                    * dif + t->ele[r].a;
                e_ele *= self_coef * my_atoms[i].q * my_atoms[j].q;

                data_e_vdW[i] += e_vdW;
                data_e_ele[i] += e_ele;
            }    

            CEvd = ((t->CEvd[r].d * dif + t->CEvd[r].c) * dif + t->CEvd[r].b)
                * dif + t->CEvd[r].a;
            CEvd *= self_coef;

            CEclmb = ((t->CEclmb[r].d * dif + t->CEclmb[r].c) * dif + t->CEclmb[r].b)
                * dif + t->CEclmb[r].a;
            CEclmb *= self_coef * my_atoms[i].q * my_atoms[j].q;

            if ( control->virial == 0 )
            {
                if ( i < j ) 
                {
                    rvec_ScaledAdd( workspace.f[i],
                            -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                }
                else 
                {
                    rvec_ScaledAdd( workspace.f[i],
                            (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                }
            }
            /* NPT, iNPT or sNPT */
            else
            {
                /* for pressure coupling, terms not related to bond order derivatives
                   are added directly into pressure vector/tensor */
                rvec_Scale( temp,
                        (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );

                rvec_ScaledAdd( workspace.f[i], -1.0, temp );
                rvec_Add( workspace.f[j], temp );

                rvec_iMultiply( ext_press, far_nbr_list.far_nbr_list.rel_box[pj], temp );
                rvec_Add( data_ext_press[i], ext_press );
            }

#if defined(TEST_ENERGY)
            //fprintf( out_control->evdw, "%6d%6d%24.15e%24.15e%24.15e\n",
            fprintf( out_control->evdw, "%6d%6d%12.4f%12.4f%12.4f\n",
                    my_atoms[i].orig_id, my_atoms[j].orig_id, 
                    r_ij, e_vdW, data->my_en.e_vdW );
            //fprintf(out_control->ecou,"%6d%6d%24.15e%24.15e%24.15e%24.15e%24.15e\n",
            fprintf( out_control->ecou, "%6d%6d%12.4f%12.4f%12.4f%12.4f%12.4f\n",
                    my_atoms[i].orig_id, my_atoms[j].orig_id,
                    r_ij, my_atoms[i].q, my_atoms[j].q, 
                    e_ele, data->my_en.e_ele );
#endif

#if defined(TEST_FORCES)
            rvec_ScaledAdd( workspace.f_vdw[i], -CEvd,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_vdw[j], +CEvd,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_ele[i], -CEclmb,
                    far_nbr_list.far_nbr_list.dvec[pj] );
            rvec_ScaledAdd( workspace.f_ele[j], +CEclmb,
                    far_nbr_list.far_nbr_list.dvec[pj] );
#endif
        }
    }
}


static void Cuda_Compute_Polarization_Energy( reax_system *system, storage *workspace,
        simulation_data *data )
{
    int blocks;
    real *spad;

    blocks = system->n / DEF_BLOCK_SIZE
        + ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    spad = (real *) workspace->scratch;

    k_compute_polarization_energy <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, 
          system->n, spad );
    hipDeviceSynchronize( );
    cudaCheckError( );

    Cuda_Reduction_Sum( spad,
            &((simulation_data *)data->d_simulation_data)->my_en.e_pol,
            system->n );
}


void Cuda_NonBonded_Energy( reax_system *system, control_params *control, 
        storage *workspace, simulation_data *data, reax_list **lists,
        output_controls *out_control )
{
    int blocks, rblocks, update_energy;
    rvec *spad_rvec;
    real *spad;

    spad = (real *) workspace->scratch;
    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;
    rblocks = system->n / DEF_BLOCK_SIZE
        + ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    blocks = (system->n * VDW_KER_THREADS_PER_ATOM / DEF_BLOCK_SIZE) 
        + ((system->n * VDW_KER_THREADS_PER_ATOM % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    if ( control->tabulate == 0 )
    {
        k_vdW_coulomb_energy <<< rblocks, DEF_BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_tbp, 
              system->reax_param.d_gp, (control_params *)control->d_control_params, 
              *(workspace->d_workspace), *(lists[FAR_NBRS]), 
              system->n, system->reax_param.num_atom_types, 
              spad, &spad[system->n], (rvec *)(&spad[2 * system->n]) );
//        k_vdW_coulomb_energy_opt <<< blocks, DEF_BLOCK_SIZE,
//                             (2 * sizeof(real) + sizeof(rvec)) * DEF_BLOCK_SIZE >>>
//            ( system->d_my_atoms, system->reax_param.d_tbp, 
//              system->reax_param.d_gp, (control_params *)control->d_control_params, 
//              *(workspace->d_workspace), *(lists[FAR_NBRS]), 
//              system->n, system->reax_param.num_atom_types, 
//              spad, &spad[system->n], (rvec *)(&spad[2 * system->n]) );
        hipDeviceSynchronize( );
        cudaCheckError( );
    }
    else
    {
        k_tabulated_vdW_coulomb_energy <<< blocks, DEF_BLOCK_SIZE >>>
            ( system->d_my_atoms, system->reax_param.d_gp, 
              (control_params *)control->d_control_params, 
              *(workspace->d_workspace), *(lists[FAR_NBRS]), 
              workspace->d_LR, system->n,
              system->reax_param.num_atom_types, 
              data->step, data->prev_steps, 
              out_control->energy_update_freq,
              spad, &spad[system->n], (rvec *)(&spad[2 * system->n]));
        hipDeviceSynchronize( );
        cudaCheckError( );
    }

    if ( update_energy == TRUE )
    {
        /* reduction for vdw */
        Cuda_Reduction_Sum( spad, &((simulation_data *)data->d_simulation_data)->my_en.e_vdW,
                system->n );

        /* reduction for ele */
        Cuda_Reduction_Sum( &spad[system->n], &((simulation_data *)data->d_simulation_data)->my_en.e_ele,
                system->n );
    }

    /* reduction for ext_press */
    spad_rvec = (rvec *) (&spad[2 * system->n]);
    k_reduction_rvec <<< rblocks, DEF_BLOCK_SIZE, sizeof(rvec) * DEF_BLOCK_SIZE >>>
        ( spad_rvec, &spad_rvec[system->n], system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );

    k_reduction_rvec <<< 1, control->blocks_pow_2_n, sizeof(rvec) * control->blocks_pow_2_n>>>
        ( &spad_rvec[system->n], &((simulation_data *)data->d_simulation_data)->my_ext_press, rblocks );
    hipDeviceSynchronize( );
    cudaCheckError( );

    if ( update_energy == TRUE )
    {
        Cuda_Compute_Polarization_Energy( system, workspace, data );
    }
}

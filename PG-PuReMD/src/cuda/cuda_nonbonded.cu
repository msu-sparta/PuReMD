#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_nonbonded.h"

#include "cuda_helpers.h"
#include "cuda_list.h"
#include "cuda_reduction.h"
#include "cuda_utils.h"

#include "../index_utils.h"
#include "../vector.h"

#include "../cub/cub/warp/warp_reduce.cuh"
//#include <cub/warp/warp_reduce.cuh>


CUDA_GLOBAL void k_compute_polarization_energy( reax_atom *my_atoms, 
        single_body_parameters *sbp, int n, real *e_pol_g )
{
    int i, type_i;
    real q;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    q = my_atoms[i].q;
    type_i = my_atoms[i].type;

#if !defined(CUDA_ACCUM_ATOMIC)
    e_pol_g[i] = KCALpMOL_to_EV * (sbp[type_i].chi * q
            + (sbp[type_i].eta / 2.0) * SQR(q));
#else
    atomicAdd( (double *) e_pol_g, (double) (KCALpMOL_to_EV * (sbp[type_i].chi
                    * q + (sbp[type_i].eta / 2.0) * SQR(q))) );
#endif
}


/* one thread per atom implementation */
CUDA_GLOBAL void k_vdW_coulomb_energy( reax_atom *my_atoms, 
        two_body_parameters *tbp, global_parameters gp, control_params *control, 
        storage workspace, reax_list far_nbr_list, int n, int num_atom_types, 
        real *e_vdW_g, real *e_ele_g )
{
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j;
    real self_coef;
    real p_vdW1, p_vdW1i;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real Tap, dTap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_ele_l, e_vdW_l, e_core, de_core, e_clb, de_clb;
    rvec temp, f_i_l;
    two_body_parameters *twbp;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    p_vdW1 = gp.l[28];
    p_vdW1i = 1.0 / p_vdW1;
    e_vdW_l = 0.0;
    e_ele_l = 0.0;
    rvec_MakeZero( f_i_l );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        //TODO: assuming far_nbr_list in FULL_LIST, add conditions for HALF_LIST
        if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut 
                && orig_i < orig_j )
        {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            twbp = &tbp[
                index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types) ];

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            Tap = workspace.Tap[7] * r_ij
                + workspace.Tap[6];
            Tap = Tap * r_ij + workspace.Tap[5];
            Tap = Tap * r_ij + workspace.Tap[4];
            Tap = Tap * r_ij + workspace.Tap[3];
            Tap = Tap * r_ij + workspace.Tap[2];
            Tap = Tap * r_ij + workspace.Tap[1];
            Tap = Tap * r_ij + workspace.Tap[0];

            dTap = 7.0 * workspace.Tap[7] * r_ij
                + 6.0 * workspace.Tap[6];
            dTap = dTap * r_ij + 5.0 * workspace.Tap[5];
            dTap = dTap * r_ij + 4.0 * workspace.Tap[4];
            dTap = dTap * r_ij + 3.0 * workspace.Tap[3];
            dTap = dTap * r_ij + 2.0 * workspace.Tap[2];
            dTap = dTap * r_ij + workspace.Tap[1];

            /* vdWaals Calculations */
            if ( gp.vdw_type == 1 || gp.vdw_type == 3 )
            {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / twbp->gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW_l += self_coef * (e_base * Tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else
            {
                exp1 = EXP( twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW_l += self_coef * (e_base * Tap);

                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( gp.vdw_type == 2 || gp.vdw_type == 3 )
            {
                e_core = twbp->ecore * EXP( twbp->acore * (1.0 - (r_ij / twbp->rcore)) );
                e_vdW_l += self_coef * (e_core * Tap);

                de_core = -(twbp->acore / twbp->rcore) * e_core;
            }
            else
            {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * Tap
                    + (e_base + e_core) * dTap );

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij
                + POW( twbp->gamma, -3.0 );
            dr3gamij_3 = POW( dr3gamij_1, 1.0 / 3.0 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) / dr3gamij_3;
            e_ele_l += self_coef * (e_clb * Tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * Tap + e_clb * dTap);

            if ( i < j ) 
            {
                rvec_Scale( temp, -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            }
            else 
            {
                rvec_Scale( temp, (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            }
            rvec_Add( f_i_l, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( workspace.f[j], temp );
        }
    }

    atomic_rvecAdd( workspace.f[i], f_i_l );
#if !defined(CUDA_ACCUM_ATOMIC)
    e_vdW_g[i] = e_vdW_l;
    e_ele_g[i] = e_ele_l;
#else
    atomicAdd( (double *) e_vdW_g, (double) e_vdW_l );
    atomicAdd( (double *) e_ele_g, (double) e_ele_l );
#endif
}


/* one thread per atom implementation */
CUDA_GLOBAL void k_vdW_coulomb_virial_energy( reax_atom *my_atoms, 
        two_body_parameters *tbp, global_parameters gp, control_params *control, 
        storage workspace, reax_list far_nbr_list, int n, int num_atom_types, 
        real *e_vdW_g, real *e_ele_g, rvec *ext_press_g )
{
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j;
    real self_coef;
    real p_vdW1, p_vdW1i;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real Tap, dTap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_ele_l, e_vdW_l, e_core, de_core, e_clb, de_clb;
    rvec temp, f_i_l, ext_press_l;
    two_body_parameters *twbp;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    p_vdW1 = gp.l[28];
    p_vdW1i = 1.0 / p_vdW1;
    e_vdW_l = 0.0;
    e_ele_l = 0.0;
    rvec_MakeZero( f_i_l );
    rvec_MakeZero( ext_press_l );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        //TODO: assuming far_nbr_list in FULL_LIST, add conditions for HALF_LIST
        if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut 
                && orig_i < orig_j )
        {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            twbp = &tbp[
                index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types) ];

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            Tap = workspace.Tap[7] * r_ij
                + workspace.Tap[6];
            Tap = Tap * r_ij + workspace.Tap[5];
            Tap = Tap * r_ij + workspace.Tap[4];
            Tap = Tap * r_ij + workspace.Tap[3];
            Tap = Tap * r_ij + workspace.Tap[2];
            Tap = Tap * r_ij + workspace.Tap[1];
            Tap = Tap * r_ij + workspace.Tap[0];

            dTap = 7.0 * workspace.Tap[7] * r_ij
                + 6.0 * workspace.Tap[6];
            dTap = dTap * r_ij + 5.0 * workspace.Tap[5];
            dTap = dTap * r_ij + 4.0 * workspace.Tap[4];
            dTap = dTap * r_ij + 3.0 * workspace.Tap[3];
            dTap = dTap * r_ij + 2.0 * workspace.Tap[2];
            dTap = dTap * r_ij + workspace.Tap[1];

            /* vdWaals Calculations */
            if ( gp.vdw_type == 1 || gp.vdw_type == 3 )
            {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / twbp->gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW_l += self_coef * (e_base * Tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else
            {
                exp1 = EXP( twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW_l += self_coef * (e_base * Tap);

                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( gp.vdw_type == 2 || gp.vdw_type == 3 )
            {
                e_core = twbp->ecore * EXP( twbp->acore * (1.0 - (r_ij / twbp->rcore)) );
                e_vdW_l += self_coef * (e_core * Tap);

                de_core = -(twbp->acore / twbp->rcore) * e_core;
            }
            else
            {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * Tap
                    + (e_base + e_core) * dTap );

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij
                + POW( twbp->gamma, -3.0 );
            dr3gamij_3 = POW( dr3gamij_1, 1.0 / 3.0 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) / dr3gamij_3;
            e_ele_l += self_coef * (e_clb * Tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * Tap + e_clb * dTap);

            /* for pressure coupling, terms not related to bond order 
               derivatives are added directly into pressure vector/tensor */
            if ( i < j ) 
            {
                rvec_Scale( temp,
                        -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            }
            else 
            {
                rvec_Scale( temp,
                        (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            }
            rvec_Add( f_i_l, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( workspace.f[j], temp );

            rvec_iMultiply( temp,
                    far_nbr_list.far_nbr_list.rel_box[pj], temp );
            rvec_Add( ext_press_l, temp );
        }
    }

    atomic_rvecAdd( workspace.f[i], f_i_l );
#if !defined(CUDA_ACCUM_ATOMIC)
    e_vdW_g[i] = e_vdW_l;
    e_ele_g[i] = e_ele_l;
    rvec_Copy( ext_press_g[j], ext_press_l );
#else
    atomicAdd( (double *) e_vdW_g, (double) e_vdW_l );
    atomicAdd( (double *) e_ele_g, (double) e_ele_l );
    atomic_rvecAdd( *ext_press_g, ext_press_l );
#endif
}


/* one warp of threads per atom implementation */
CUDA_GLOBAL void k_vdW_coulomb_energy_opt( reax_atom *my_atoms, 
        two_body_parameters *tbp, global_parameters gp, control_params *control, 
        storage workspace, reax_list far_nbr_list, int n, int num_atom_types, 
        real *e_vdW_g, real *e_ele_g )
{
    typedef hipcub::WarpReduce<double> WarpReduce;
    extern __shared__ typename WarpReduce::TempStorage temp_storage[];
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j;
    real self_coef;
    real p_vdW1, p_vdW1i;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real Tap, dTap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_vdW_l, e_ele_l, e_core, de_core, e_clb, de_clb;
    rvec temp, f_i_l;
    two_body_parameters *twbp;
    int thread_id, warp_id, lane_id;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id >> 5;

    if ( warp_id >= n )
    {
        return;
    }

    lane_id = thread_id & 0x0000001F; 
    i = warp_id;
    p_vdW1 = gp.l[28];
    p_vdW1i = 1.0 / p_vdW1;
    e_vdW_l = 0.0;
    e_ele_l = 0.0;
    rvec_MakeZero( f_i_l );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    pj = start_i + lane_id;
    while ( pj < end_i )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        //TODO: assuming far_nbr_list in FULL_LIST, add conditions for HALF_LIST
        if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut 
                && orig_i < orig_j )
        {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            twbp = &tbp[
                index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types) ];

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            Tap = workspace.Tap[7] * r_ij
                + workspace.Tap[6];
            Tap = Tap * r_ij + workspace.Tap[5];
            Tap = Tap * r_ij + workspace.Tap[4];
            Tap = Tap * r_ij + workspace.Tap[3];
            Tap = Tap * r_ij + workspace.Tap[2];
            Tap = Tap * r_ij + workspace.Tap[1];
            Tap = Tap * r_ij + workspace.Tap[0];

            dTap = 7.0 * workspace.Tap[7] * r_ij
                + 6.0 * workspace.Tap[6];
            dTap = dTap * r_ij + 5.0 * workspace.Tap[5];
            dTap = dTap * r_ij + 4.0 * workspace.Tap[4];
            dTap = dTap * r_ij + 3.0 * workspace.Tap[3];
            dTap = dTap * r_ij + 2.0 * workspace.Tap[2];
            dTap = dTap * r_ij + workspace.Tap[1];

            /* vdWaals Calculations */
            if ( gp.vdw_type == 1 || gp.vdw_type == 3 )
            {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / twbp->gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW_l += self_coef * (e_base * Tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else
            {
                exp1 = EXP( twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW_l += self_coef * (e_base * Tap);

                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( gp.vdw_type == 2 || gp.vdw_type == 3 )
            {
                e_core = twbp->ecore * EXP( twbp->acore * (1.0 - (r_ij / twbp->rcore)) );
                e_vdW_l += self_coef * (e_core * Tap);

                de_core = -(twbp->acore / twbp->rcore) * e_core;
            }
            else
            {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * Tap
                    + (e_base + e_core) * dTap );

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij
                + POW( twbp->gamma, -3.0 );
            dr3gamij_3 = POW( dr3gamij_1, 1.0 / 3.0 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) / dr3gamij_3;
            e_ele_l += self_coef * (e_clb * Tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * Tap + e_clb * dTap);

            if ( i < j ) 
            {
                rvec_Scale( temp, -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            }
            else 
            {
                rvec_Scale( temp, (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            }
            rvec_Add( f_i_l, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( workspace.f[j], temp );
        }

        pj += warpSize;
    }

    e_vdW_l = WarpReduce(temp_storage[warp_id]).Sum(e_vdW_l);
    e_ele_l = WarpReduce(temp_storage[warp_id]).Sum(e_ele_l);
    f_i_l[0] = WarpReduce(temp_storage[warp_id]).Sum(f_i_l[0]);
    f_i_l[1] = WarpReduce(temp_storage[warp_id]).Sum(f_i_l[1]);
    f_i_l[2] = WarpReduce(temp_storage[warp_id]).Sum(f_i_l[2]);

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 )
    {
        atomic_rvecAdd( workspace.f[i], f_i_l );
#if !defined(CUDA_ACCUM_ATOMIC)
        e_vdW_g[i] = e_vdW_l;
        e_ele_g[i] = e_ele_l;
#else
        atomicAdd( (double *) e_vdW_g, (double) e_vdW_l );
        atomicAdd( (double *) e_ele_g, (double) e_ele_l );
#endif
    }
}


/* one warp of threads per atom implementation */
CUDA_GLOBAL void k_vdW_coulomb_energy_virial_opt( reax_atom *my_atoms, 
        two_body_parameters *tbp, global_parameters gp, control_params *control, 
        storage workspace, reax_list far_nbr_list, int n, int num_atom_types, 
        real *e_vdW_g, real *e_ele_g, rvec *ext_press_g )
{
    typedef hipcub::WarpReduce<double> WarpReduce;
    extern __shared__ typename WarpReduce::TempStorage temp_storage[];
    int i, j, pj;
    int start_i, end_i, orig_i, orig_j;
    real self_coef;
    real p_vdW1, p_vdW1i;
    real powr_vdW1, powgi_vdW1;
    real r_ij, fn13, exp1, exp2, e_base, de_base;
    real Tap, dTap, dfn13, CEvd, CEclmb;
    real dr3gamij_1, dr3gamij_3;
    real e_vdW_l, e_ele_l, e_core, de_core, e_clb, de_clb;
    rvec temp, f_i_l, ext_press_l;
    two_body_parameters *twbp;
    int thread_id, warp_id, lane_id;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id >> 5;

    if ( warp_id >= n )
    {
        return;
    }

    lane_id = thread_id & 0x0000001F; 
    i = warp_id;
    p_vdW1 = gp.l[28];
    p_vdW1i = 1.0 / p_vdW1;
    e_vdW_l = 0.0;
    e_ele_l = 0.0;
    rvec_MakeZero( f_i_l );
    rvec_MakeZero( ext_press_l );

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    pj = start_i + lane_id;
    while ( pj < end_i )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        //TODO: assuming far_nbr_list in FULL_LIST, add conditions for HALF_LIST
        if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut 
                && orig_i < orig_j )
        {
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            twbp = &tbp[
                index_tbp(my_atoms[i].type, my_atoms[j].type, num_atom_types) ];

            /* i == j: self-interaction from periodic image,
             * important for supporting small boxes! */
            self_coef = (orig_i == orig_j) ? 0.5 : 1.0;

            /* Calculate Taper and its derivative */
            Tap = workspace.Tap[7] * r_ij
                + workspace.Tap[6];
            Tap = Tap * r_ij + workspace.Tap[5];
            Tap = Tap * r_ij + workspace.Tap[4];
            Tap = Tap * r_ij + workspace.Tap[3];
            Tap = Tap * r_ij + workspace.Tap[2];
            Tap = Tap * r_ij + workspace.Tap[1];
            Tap = Tap * r_ij + workspace.Tap[0];

            dTap = 7.0 * workspace.Tap[7] * r_ij
                + 6.0 * workspace.Tap[6];
            dTap = dTap * r_ij + 5.0 * workspace.Tap[5];
            dTap = dTap * r_ij + 4.0 * workspace.Tap[4];
            dTap = dTap * r_ij + 3.0 * workspace.Tap[3];
            dTap = dTap * r_ij + 2.0 * workspace.Tap[2];
            dTap = dTap * r_ij + workspace.Tap[1];

            /* vdWaals Calculations */
            if ( gp.vdw_type == 1 || gp.vdw_type == 3 )
            {
                /* shielding */
                powr_vdW1 = POW( r_ij, p_vdW1 );
                powgi_vdW1 = POW( 1.0 / twbp->gamma_w, p_vdW1 );

                fn13 = POW( powr_vdW1 + powgi_vdW1, p_vdW1i );
                exp1 = EXP( twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - fn13 / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW_l += self_coef * (e_base * Tap);

                dfn13 = POW( r_ij, p_vdW1 - 1.0 )
                    * POW( powr_vdW1 + powgi_vdW1, p_vdW1i - 1.0 );
                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1) * dfn13;
            }
            /* no shielding */
            else
            {
                exp1 = EXP( twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                exp2 = EXP( 0.5 * twbp->alpha * (1.0 - r_ij / twbp->r_vdW) );
                e_base = twbp->D * (exp1 - 2.0 * exp2);

                e_vdW_l += self_coef * (e_base * Tap);

                de_base = (twbp->D * twbp->alpha / twbp->r_vdW) * (exp2 - exp1);
            }

            /* calculate inner core repulsion */
            if ( gp.vdw_type == 2 || gp.vdw_type == 3 )
            {
                e_core = twbp->ecore * EXP( twbp->acore * (1.0 - (r_ij / twbp->rcore)) );
                e_vdW_l += self_coef * (e_core * Tap);

                de_core = -(twbp->acore / twbp->rcore) * e_core;
            }
            else
            {
                e_core = 0.0;
                de_core = 0.0;
            }

            CEvd = self_coef * ( (de_base + de_core) * Tap
                    + (e_base + e_core) * dTap );

            /* Coulomb Calculations */
            dr3gamij_1 = r_ij * r_ij * r_ij
                + POW( twbp->gamma, -3.0 );
            dr3gamij_3 = POW( dr3gamij_1, 1.0 / 3.0 );
            e_clb = C_ELE * (my_atoms[i].q * my_atoms[j].q) / dr3gamij_3;
            e_ele_l += self_coef * (e_clb * Tap);

            de_clb = -C_ELE * (my_atoms[i].q * my_atoms[j].q)
                    * (r_ij * r_ij) / POW( dr3gamij_1, 4.0 / 3.0 );
            CEclmb = self_coef * (de_clb * Tap + e_clb * dTap);

            /* for pressure coupling, terms not related to bond order 
               derivatives are added directly into pressure vector/tensor */
            if ( i < j ) 
            {
                rvec_Scale( temp,
                        -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            }
            else 
            {
                rvec_Scale( temp,
                        (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
            }
            rvec_Add( f_i_l, temp );
            rvec_Scale( temp, -1.0, temp );
            atomic_rvecAdd( workspace.f[j], temp );

            rvec_iMultiply( temp,
                    far_nbr_list.far_nbr_list.rel_box[pj], temp );
            rvec_Add( ext_press_l, temp );
        }

        pj += warpSize;
    }

    e_vdW_l = WarpReduce(temp_storage[warp_id]).Sum(e_vdW_l);
    e_ele_l = WarpReduce(temp_storage[warp_id]).Sum(e_ele_l);
    f_i_l[0] = WarpReduce(temp_storage[warp_id]).Sum(f_i_l[0]);
    f_i_l[1] = WarpReduce(temp_storage[warp_id]).Sum(f_i_l[1]);
    f_i_l[2] = WarpReduce(temp_storage[warp_id]).Sum(f_i_l[2]);

    /* first thread within a warp writes warp-level sum to global memory */
    if ( lane_id == 0 )
    {
        atomic_rvecAdd( workspace.f[i], f_i_l );
#if !defined(CUDA_ACCUM_ATOMIC)
        e_vdW_g[i] = e_vdW_l;
        e_ele_g[i] = e_ele_l;
        rvec_Copy( ext_press_g[j], ext_press_l );
#else
        atomicAdd( (double *) e_vdW_g, (double) e_vdW_l );
        atomicAdd( (double *) e_ele_g, (double) e_ele_l );
        atomic_rvecAdd( *ext_press_g, ext_press_l );
#endif
    }
}


/* one thread per atom implementation */
CUDA_GLOBAL void k_vdW_coulomb_energy_tab( reax_atom *my_atoms, 
        global_parameters gp, control_params *control, 
        storage workspace, reax_list far_nbr_list, 
        LR_lookup_table *t_LR, int n, int num_atom_types, 
        int step, int prev_steps, int energy_update_freq, 
        real *e_vdW_g, real *e_ele_g, rvec *ext_press_g )
{
    int i, j, pj, r, steps, update_freq, update_energies;
    int type_i, type_j, tmin, tmax;
    int start_i, end_i, orig_i, orig_j;
    real r_ij, self_coef, base, dif;
    real e_vdW_l, e_ele_l;
    real CEvd, CEclmb;
    rvec temp, f_i_l, ext_press_l;
    LR_lookup_table *t;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    steps = step - prev_steps;
    update_freq = energy_update_freq;
    update_energies = update_freq > 0 && steps % update_freq == 0;
    e_ele_l = 0.0;
    e_vdW_l = 0.0;
    rvec_MakeZero( f_i_l );
    rvec_MakeZero( ext_press_l );

    type_i = my_atoms[i].type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    orig_i = my_atoms[i].orig_id;

    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];
        orig_j = my_atoms[j].orig_id;

        //TODO: assuming far_nbr_list in FULL_LIST, add conditions for HALF_LIST
        if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut
                && orig_i < orig_j )
        {
            type_j = my_atoms[j].type;
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            self_coef = (i == j) ? 0.5 : 1.0;
            tmin = MIN( type_i, type_j );
            tmax = MAX( type_i, type_j );
            t = &t_LR[ index_lr(tmin, tmax, num_atom_types) ];

            /* Cubic Spline Interpolation */
            r = (int)(r_ij * t->inv_dx);
            if ( r == 0 )
            {
                ++r;
            }
            base = (real)(r + 1) * t->dx;
            dif = r_ij - base;

            if ( update_energies )
            {
                e_vdW_l += self_coef * (((t->vdW[r].d * dif + t->vdW[r].c) * dif + t->vdW[r].b)
                    * dif + t->vdW[r].a);

                e_ele_l += (((t->ele[r].d * dif + t->ele[r].c) * dif + t->ele[r].b)
                    * dif + t->ele[r].a) * self_coef * my_atoms[i].q * my_atoms[j].q;
            }    

            CEvd = ((t->CEvd[r].d * dif + t->CEvd[r].c) * dif + t->CEvd[r].b)
                * dif + t->CEvd[r].a;
            CEvd *= self_coef;

            CEclmb = ((t->CEclmb[r].d * dif + t->CEclmb[r].c) * dif + t->CEclmb[r].b)
                * dif + t->CEclmb[r].a;
            CEclmb *= self_coef * my_atoms[i].q * my_atoms[j].q;

            if ( control->virial == 0 )
            {
                if ( i < j ) 
                {
                    rvec_ScaledAdd( temp,
                            -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                }
                else 
                {
                    rvec_ScaledAdd( temp,
                            (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                }
                rvec_Add( f_i_l, temp );
                rvec_Scale( temp, -1.0, temp );
                atomic_rvecAdd( workspace.f[j], temp );
            }
            /* NPT, iNPT or sNPT */
            else
            {
                /* for pressure coupling, terms not related to bond order derivatives
                   are added directly into pressure vector/tensor */
                if ( i < j ) 
                {
                    rvec_Scale( temp,
                            -(CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                }
                else 
                {
                    rvec_Scale( temp,
                            (CEvd + CEclmb) / r_ij, far_nbr_list.far_nbr_list.dvec[pj] );
                }
                rvec_Add( f_i_l, temp );
                rvec_ScaledAdd( temp, -1.0, temp );
                atomic_rvecAdd( workspace.f[j], temp );

                rvec_iMultiply( temp, far_nbr_list.far_nbr_list.rel_box[pj], temp );
                rvec_Add( ext_press_l, temp );
            }
        }
    }

    atomic_rvecAdd( workspace.f[i], f_i_l );
#if !defined(CUDA_ACCUM_ATOMIC)
    __syncthreads( );
    e_vdW_g[i] = e_vdW_l;
    e_ele_g[i] = e_ele_l;
    if ( control->virial == 1 )
        rvec_Copy( ext_press_g[j], ext_press_l );
#else
    atomicAdd( (double *) e_vdW_g, (double) e_vdW_l );
    atomicAdd( (double *) e_ele_g, (double) e_ele_l );
    if ( control->virial == 1 )
        atomic_rvecAdd( *ext_press_g, ext_press_l );
#endif
}


static void Cuda_Compute_Polarization_Energy( reax_system *system, storage *workspace,
        simulation_data *data )
{
    int blocks;
#if !defined(CUDA_ACCUM_ATOMIC)
    real *spad;

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
            sizeof(real) * system->n,
            "Cuda_Compute_Polarization_Energy::workspace->scratch" );
    spad = (real *) workspace->scratch;
#else
    cuda_memset( &((simulation_data *)data->d_simulation_data)->my_en.e_pol,
            0, sizeof(real), "Cuda_Compute_Bonded_Forces::e_pol" );
#endif

    blocks = system->n / DEF_BLOCK_SIZE
        + ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_compute_polarization_energy <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, 
          system->n,
#if !defined(CUDA_ACCUM_ATOMIC)
          spad
#else
          &((simulation_data *)data->d_simulation_data)->my_en.e_pol
#endif
        );
    cudaCheckError( );

#if !defined(CUDA_ACCUM_ATOMIC)
    Cuda_Reduction_Sum( spad,
            &((simulation_data *)data->d_simulation_data)->my_en.e_pol,
            system->n );
#endif
}


void Cuda_Compute_NonBonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, reax_list **lists,
        output_controls *out_control )
{
    int update_energy, blocks;
#if !defined(CUDA_ACCUM_ATOMIC)
    size_t s;
    real *spad;
    rvec *spad_rvec;
#endif

    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;

#if !defined(CUDA_ACCUM_ATOMIC)
    if ( control->virial == 1 )
        s = (sizeof(real) * 2 + sizeof(rvec)) * system->n + sizeof(rvec) * control->blocks;
    else
        s = sizeof(real) * 2 * system->n;
    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
            s, "Cuda_Compute_NonBonded_Forces::workspace->scratch" );
    spad = (real *) workspace->scratch;
#endif

#if defined(CUDA_ACCUM_ATOMIC)
        cuda_memset( &((simulation_data *)data->d_simulation_data)->my_en.e_vdW,
                0, sizeof(real), "Cuda_Compute_Bonded_Forces::e_vdW" );
        cuda_memset( &((simulation_data *)data->d_simulation_data)->my_en.e_ele,
                0, sizeof(real), "Cuda_Compute_Bonded_Forces::e_ele" );
        if ( control->virial == 1 )
        {
            cuda_memset( &((simulation_data *)data->d_simulation_data)->my_ext_press,
                    0, sizeof(rvec), "Cuda_Compute_Bonded_Forces::my_ext_press" );
        }
#endif

    blocks = system->n * 32 / DEF_BLOCK_SIZE
        + (system->n * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

    if ( control->tabulate == 0 )
    {
        if ( control->virial == 1 )
        {
//            k_vdW_coulomb_energy_virial <<< control->blocks, control->block_size >>>
//                ( system->d_my_atoms, system->reax_param.d_tbp, 
//                  system->reax_param.d_gp, (control_params *) control->d_control_params, 
//                  *(workspace->d_workspace), *(lists[FAR_NBRS]), 
//                  system->n, system->reax_param.num_atom_types, 
//#if !defined(CUDA_ACCUM_ATOMIC)
//                  spad, &spad[system->n], (rvec *) (&spad[2 * system->n])
//#else
//                  &((simulation_data *)data->d_simulation_data)->my_en.e_vdW,
//                  &((simulation_data *)data->d_simulation_data)->my_en.e_ele,
//                  &((simulation_data *)data->d_simulation_data)->my_ext_press
//#endif
//            );

        k_vdW_coulomb_energy_virial_opt <<< blocks, DEF_BLOCK_SIZE,
                                 sizeof(real) * (DEF_BLOCK_SIZE / 32) >>>
            ( system->d_my_atoms, system->reax_param.d_tbp, 
              system->reax_param.d_gp, (control_params *) control->d_control_params, 
              *(workspace->d_workspace), *(lists[FAR_NBRS]), 
              system->n, system->reax_param.num_atom_types, 
#if !defined(CUDA_ACCUM_ATOMIC)
              spad, &spad[system->n], (rvec *) (&spad[2 * system->n])
#else
              &((simulation_data *)data->d_simulation_data)->my_en.e_vdW,
              &((simulation_data *)data->d_simulation_data)->my_en.e_ele,
              &((simulation_data *)data->d_simulation_data)->my_ext_press
#endif
            );
        }
        else
        {
//            k_vdW_coulomb_energy <<< control->blocks, control->block_size >>>
//                ( system->d_my_atoms, system->reax_param.d_tbp, 
//                  system->reax_param.d_gp, (control_params *) control->d_control_params, 
//                  *(workspace->d_workspace), *(lists[FAR_NBRS]), 
//                  system->n, system->reax_param.num_atom_types, 
//#if !defined(CUDA_ACCUM_ATOMIC)
//                  spad, &spad[system->n]
//#else
//                  &((simulation_data *)data->d_simulation_data)->my_en.e_vdW,
//                  &((simulation_data *)data->d_simulation_data)->my_en.e_ele
//#endif
//                );

        k_vdW_coulomb_energy_opt <<< blocks, DEF_BLOCK_SIZE,
                                 sizeof(real) * (DEF_BLOCK_SIZE / 32) >>>
            ( system->d_my_atoms, system->reax_param.d_tbp, 
              system->reax_param.d_gp, (control_params *) control->d_control_params, 
              *(workspace->d_workspace), *(lists[FAR_NBRS]), 
              system->n, system->reax_param.num_atom_types, 
#if !defined(CUDA_ACCUM_ATOMIC)
              spad, &spad[system->n]
#else
              &((simulation_data *)data->d_simulation_data)->my_en.e_vdW,
              &((simulation_data *)data->d_simulation_data)->my_en.e_ele
#endif
            );
        }
        cudaCheckError( );
    }
    else
    {
        k_vdW_coulomb_energy_tab <<< control->blocks, control->block_size >>>
            ( system->d_my_atoms, system->reax_param.d_gp, 
              (control_params *) control->d_control_params, 
              *(workspace->d_workspace), *(lists[FAR_NBRS]), 
              workspace->d_LR, system->n,
              system->reax_param.num_atom_types, 
              data->step, data->prev_steps, 
              out_control->energy_update_freq,
#if !defined(CUDA_ACCUM_ATOMIC)
              spad, &spad[system->n], (rvec *) (&spad[2 * system->n])
#else
              &((simulation_data *)data->d_simulation_data)->my_en.e_vdW,
              &((simulation_data *)data->d_simulation_data)->my_en.e_ele,
              &((simulation_data *)data->d_simulation_data)->my_ext_press
#endif
            );
        cudaCheckError( );
    }

#if !defined(CUDA_ACCUM_ATOMIC)
    if ( update_energy == TRUE )
    {
        /* reduction for vdw */
        Cuda_Reduction_Sum( spad,
                &((simulation_data *)data->d_simulation_data)->my_en.e_vdW,
                system->n );

        /* reduction for ele */
        Cuda_Reduction_Sum( &spad[system->n],
                &((simulation_data *)data->d_simulation_data)->my_en.e_ele,
                system->n );
    }

    if ( control->virial == 1 )
    {
        spad_rvec = (rvec *) (&spad[2 * system->n]);

        /* reduction for ext_press */
        k_reduction_rvec <<< control->blocks, control->block_size,
                         sizeof(rvec) * (control->block_size / 32) >>>
            ( spad_rvec, &spad_rvec[system->n], system->n );
        cudaCheckError( );

        k_reduction_rvec <<< 1, control->blocks_pow_2,
                         sizeof(rvec) * (control->blocks_pow_2 / 32) >>>
            ( &spad_rvec[system->n],
              &((simulation_data *)data->d_simulation_data)->my_ext_press,
              control->blocks );
        cudaCheckError( );
    }
#endif

    if ( update_energy == TRUE )
    {
        Cuda_Compute_Polarization_Energy( system, workspace, data );
    }
}

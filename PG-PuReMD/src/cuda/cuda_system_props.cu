#include "hip/hip_runtime.h"

#include "cuda_system_props.h"

#include "cuda_copy.h"
#include "cuda_utils.h"
#include "cuda_random.h"
#include "cuda_reduction.h"
#include "cuda_vector.h"

#include "../comm_tools.h"
#include "../tool_box.h"
#include "../vector.h"


/* mask used to determine which threads within a warp participate in operations */
#define FULL_MASK (0xFFFFFFFF)


CUDA_GLOBAL void k_center_of_mass_blocks_xcm( single_body_parameters *sbp,
        reax_atom *atoms, rvec *xcm_g, size_t n )
{
    extern __shared__ rvec xcm_s[];
    unsigned int i, mask;
    int offset;
    rvec xcm;
    real m;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    mask = __ballot_sync( FULL_MASK, i < n );

    if ( i < n )
    {
        m = sbp[ atoms[i].type ].mass;
        rvec_Scale( xcm, m, atoms[i].x );

        /* warp-level sum using registers within a warp */
        for ( offset = warpSize >> 1; offset > 0; offset /= 2 )
        {
            xcm[0] += __shfl_down_sync( mask, xcm[0], offset );
            xcm[1] += __shfl_down_sync( mask, xcm[1], offset );
            xcm[2] += __shfl_down_sync( mask, xcm[2], offset );
        }

        /* first thread within a warp writes warp-level sum to shared memory */
        if ( threadIdx.x % warpSize == 0 )
        {
            rvec_Copy( xcm_s[ threadIdx.x >> 5 ], xcm );
        }
    }
    __syncthreads( );

    /* block-level sum using shared memory */
    for ( offset = blockDim.x >> 6; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            rvec_Add( xcm_s[threadIdx.x], xcm_s[threadIdx.x + offset] );
        }

        __syncthreads( );
    }

    /* one thread writes the block-level partial sum
     * of the reduction back to global memory */
    if ( threadIdx.x == 0 )
    {
        rvec_Copy( xcm_g[blockIdx.x], xcm_s[0] );
    }
}


CUDA_GLOBAL void k_center_of_mass_blocks_vcm( single_body_parameters *sbp,
        reax_atom *atoms, rvec *vcm_g, size_t n )
{
    extern __shared__ rvec vcm_s[];
    unsigned int i, mask;
    int offset;
    real m;
    rvec vcm;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    mask = __ballot_sync( FULL_MASK, i < n );

    if ( i < n )
    {
        m = sbp[ atoms[i].type ].mass;
        rvec_Scale( vcm, m, atoms[i].v );

        /* warp-level sum using registers within a warp */
        for ( offset = warpSize >> 1; offset > 0; offset /= 2 )
        {
            vcm[0] += __shfl_down_sync( mask, vcm[0], offset );
            vcm[1] += __shfl_down_sync( mask, vcm[1], offset );
            vcm[2] += __shfl_down_sync( mask, vcm[2], offset );
        }

        /* first thread within a warp writes warp-level sum to shared memory */
        if ( threadIdx.x % warpSize == 0 )
        {
            rvec_Copy( vcm_s[ threadIdx.x >> 5 ], vcm );
        }
    }
    __syncthreads( );

    /* block-level sum using shared memory */
    for ( offset = blockDim.x >> 6; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            rvec_Add( vcm_s[threadIdx.x], vcm_s[threadIdx.x + offset] );
        }
        __syncthreads( );
    }

    /* one thread writes the block-level partial sum
     * of the reduction back to global memory */
    if ( threadIdx.x == 0 )
    {
        rvec_Copy( vcm_g[blockIdx.x], vcm_s[0] );
    }
}


CUDA_GLOBAL void k_center_of_mass_blocks_amcm( single_body_parameters *sbp,
        reax_atom *atoms, rvec *amcm_g, size_t n )
{
    extern __shared__ rvec amcm_s[];
    unsigned int i, mask;
    int offset;
    real m;
    rvec amcm;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    mask = __ballot_sync( FULL_MASK, i < n );

    if ( i < n )
    {
        m = sbp[ atoms[i].type ].mass;
        rvec_Cross( amcm, atoms[i].x, atoms [i].v );
        rvec_Scale( amcm, m, amcm );

        /* warp-level sum using registers within a warp */
        for ( offset = warpSize >> 1; offset > 0; offset /= 2 )
        {
            amcm[0] += __shfl_down_sync( mask, amcm[0], offset );
            amcm[1] += __shfl_down_sync( mask, amcm[1], offset );
            amcm[2] += __shfl_down_sync( mask, amcm[2], offset );
        }

        /* first thread within a warp writes warp-level sum to shared memory */
        if ( threadIdx.x % warpSize == 0 )
        {
            rvec_Copy( amcm_s[ threadIdx.x >> 5 ], amcm );
        }
    }
    __syncthreads( );

    /* block-level sum using shared memory */
    for ( offset = blockDim.x >> 6; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            rvec_Add( amcm_s[threadIdx.x], amcm_s[threadIdx.x + offset] );
        }
        __syncthreads( );
    }

    /* one thread writes the block-level partial sum
     * of the reduction back to global memory */
    if ( threadIdx.x == 0 )
    {
        rvec_Copy( amcm_g[blockIdx.x], amcm_s[0] );
    }
}


CUDA_GLOBAL void k_compute_inertial_tensor_blocks( real *input, real *output, size_t n )
{
    extern __shared__ real t_s[];
    unsigned int i, index;
    int offset;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i < n )
    {
        t_s[ 6 * i ] = input[ i * 6 ];
        t_s[ 6 * i + 1 ] = input[ i * 6 + 1 ];
        t_s[ 6 * i + 2 ] = input[ i * 6 + 2 ];
        t_s[ 6 * i + 3 ] = input[ i * 6 + 3 ];
        t_s[ 6 * i + 4 ] = input[ i * 6 + 4 ];
        t_s[ 6 * i + 5 ] = input[ i * 6 + 5 ];
    }
    else
    {
        t_s[ 6 * i ] = 0.0;
        t_s[ 6 * i + 1 ] = 0.0;
        t_s[ 6 * i + 2 ] = 0.0;
        t_s[ 6 * i + 3 ] = 0.0;
        t_s[ 6 * i + 4 ] = 0.0;
        t_s[ 6 * i + 5 ] = 0.0;
    }
    __syncthreads( );

    for ( offset = blockDim.x / 2; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            index = 6 * (threadIdx.x + offset);
            t_s[ 6 * threadIdx.x ] += t_s[ index ];
            t_s[ 6 * threadIdx.x + 1 ] += t_s[ index + 1 ];
            t_s[ 6 * threadIdx.x + 2 ] += t_s[ index + 2 ];
            t_s[ 6 * threadIdx.x + 3 ] += t_s[ index + 3 ];
            t_s[ 6 * threadIdx.x + 4 ] += t_s[ index + 4 ];
            t_s[ 6 * threadIdx.x + 5 ] += t_s[ index + 5 ];
        }
        __syncthreads( );
    }

    if ( threadIdx.x == 0 )
    {
        output[0] = t_s[0];
        output[1] = t_s[1];
        output[2] = t_s[2];
        output[3] = t_s[3];
        output[4] = t_s[4];
        output[5] = t_s[5];
    }
}


CUDA_GLOBAL void k_compute_inertial_tensor_xx_xy( single_body_parameters *sbp,
        reax_atom *atoms, real *t_g, real xcm0, real xcm1, real xcm2, size_t n )
{
    extern __shared__ real xx_xy_s[];
    unsigned int i, index, mask;
    int offset;
    real xx, xy, m;
    rvec diff, xcm;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    mask = __ballot_sync( FULL_MASK, i < n );
    xcm[0] = xcm0;
    xcm[1] = xcm1;
    xcm[2] = xcm2;

    if ( i < n )
    {
        m = sbp[ atoms[i].type ].mass;
        rvec_ScaledSum( diff, 1.0, atoms[i].x, -1.0, xcm );
        xx = diff[0] * diff[0] * m;
        xy = diff[0] * diff[1] * m;

        /* warp-level sum using registers within a warp */
        for ( offset = warpSize >> 1; offset > 0; offset /= 2 )
        {
            xx += __shfl_down_sync( mask, xx, offset );
            xy += __shfl_down_sync( mask, xy, offset );
        }

        /* first thread within a warp writes warp-level sum to shared memory */
        if ( threadIdx.x % warpSize == 0 )
        {
            xx_xy_s[2 * (threadIdx.x >> 5)] = xx;    
            xx_xy_s[2 * (threadIdx.x >> 5) + 1] = xy;    
        }
    }
    __syncthreads( );

    /* block-level sum using shared memory */
    for ( offset = blockDim.x >> 6; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            index = 2 * (threadIdx.x + offset);
            xx_xy_s[ 2 * threadIdx.x ] += xx_xy_s[ index ];
            xx_xy_s[ 2 * threadIdx.x + 1 ] += xx_xy_s[ index + 1 ];
        }
        __syncthreads( );
    }

    /* one thread writes the block-level partial sum
     * of the reduction back to global memory */
    if ( threadIdx.x == 0 )
    {
        t_g[ blockIdx.x * 6 ] = xx_xy_s[ 0 ];
        t_g[ blockIdx.x * 6 + 1 ] = xx_xy_s[ 1 ];
    }
}


CUDA_GLOBAL void k_compute_inertial_tensor_xz_yy( single_body_parameters *sbp,
        reax_atom *atoms, real *t_g, real xcm0, real xcm1, real xcm2, size_t n )
{
    extern __shared__ real xz_yy_s[];
    unsigned int i, index, mask;
    int offset;
    real xz, yy, m;
    rvec diff, xcm;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    mask = __ballot_sync( FULL_MASK, i < n );
    xcm[0] = xcm0;
    xcm[1] = xcm1;
    xcm[2] = xcm2;

    if ( i < n )
    {
        m = sbp[ atoms[i].type ].mass;
        rvec_ScaledSum( diff, 1.0, atoms[i].x, -1.0, xcm );
        xz = diff[0] * diff[2] * m;
        yy = diff[1] * diff[1] * m;

        /* warp-level sum using registers within a warp */
        for ( offset = warpSize >> 1; offset > 0; offset /= 2 )
        {
            xz += __shfl_down_sync( mask, xz, offset );
            yy += __shfl_down_sync( mask, yy, offset );
        }

        /* first thread within a warp writes warp-level sum to shared memory */
        if ( threadIdx.x % warpSize == 0 )
        {
            xz_yy_s[2 * (threadIdx.x >> 5)] = xz;    
            xz_yy_s[2 * (threadIdx.x >> 5) + 1] = yy;    
        }
    }
    __syncthreads( );

    /* block-level sum using shared memory */
    for ( offset = blockDim.x >> 6; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            index = 2 * (threadIdx.x + offset);
            xz_yy_s[ 2 * threadIdx.x ] += xz_yy_s[ index ];
            xz_yy_s[ 2 * threadIdx.x + 1 ] += xz_yy_s[ index + 1 ];
        }
        __syncthreads( );
    }

    /* one thread writes the block-level partial sum
     * of the reduction back to global memory */
    if ( threadIdx.x == 0 )
    {
        t_g[ blockIdx.x * 6 + 2 ] = xz_yy_s[ 0 ];
        t_g[ blockIdx.x * 6 + 3 ] = xz_yy_s[ 1 ];
    }
}


CUDA_GLOBAL void k_compute_inertial_tensor_yz_zz( single_body_parameters *sbp,
        reax_atom *atoms, real *t_g, real xcm0, real xcm1, real xcm2, size_t n )
{
    extern __shared__ real yz_zz_s[];
    unsigned int i, index, mask;
    int offset;
    real yz, zz, m;
    rvec diff, xcm;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    mask = __ballot_sync( FULL_MASK, i < n );
    xcm[0] = xcm0;
    xcm[1] = xcm1;
    xcm[2] = xcm2;

    if ( i < n )
    {
        m = sbp[ atoms[i].type ].mass;
        rvec_ScaledSum( diff, 1.0, atoms[i].x, -1.0, xcm );
        yz = diff[1] * diff[2] * m;
        zz = diff[2] * diff[2] * m;

        /* warp-level sum using registers within a warp */
        for ( offset = warpSize >> 1; offset > 0; offset /= 2 )
        {
            yz += __shfl_down_sync( mask, yz, offset );
            zz += __shfl_down_sync( mask, zz, offset );
        }

        /* first thread within a warp writes warp-level sum to shared memory */
        if ( threadIdx.x % warpSize == 0 )
        {
            yz_zz_s[2 * (threadIdx.x >> 5)] = yz;    
            yz_zz_s[2 * (threadIdx.x >> 5)] = zz;    
        }
    }
    __syncthreads( );

    /* block-level sum using shared memory */
    for ( offset = blockDim.x >> 6; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            index = 2 * (threadIdx.x + offset);
            yz_zz_s[ 2 * threadIdx.x ] += yz_zz_s[ index ];
            yz_zz_s[ 2 * threadIdx.x + 1 ] += yz_zz_s[ index + 1 ];
        }
        __syncthreads( );
    }

    /* one thread writes the block-level partial sum
     * of the reduction back to global memory */
    if ( threadIdx.x == 0 )
    {
        t_g[ blockIdx.x * 6 + 4 ] = yz_zz_s[ 0 ];
        t_g[ blockIdx.x * 6 + 5 ] = yz_zz_s[ 1 ];
    }
}


CUDA_GLOBAL void k_compute_total_mass( single_body_parameters *sbp, reax_atom *my_atoms, 
        real *results, int n )
{
    extern __shared__ real M_s[];
    unsigned int i, mask;
    int offset;
    real M;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    mask = __ballot_sync( FULL_MASK, i < n );

    if ( i < n )
    {
        M = sbp[ my_atoms[i].type ].mass;

        for ( offset = warpSize >> 1; offset > 0; offset /= 2 )
        {
            M += __shfl_down_sync( mask, M, offset );
        }

        if ( threadIdx.x % warpSize == 0 )
        {
            M_s[threadIdx.x >> 5] = M;
        }
    }
    __syncthreads( );

    for ( offset = blockDim.x >> 6; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            M_s[threadIdx.x] += M_s[threadIdx.x + offset];
        }

        __syncthreads( );
    }

    if ( threadIdx.x == 0 )
    {
        results[blockIdx.x] = M_s[0];
    }
}


CUDA_GLOBAL void k_compute_kinetic_energy( single_body_parameters *sbp, reax_atom *my_atoms, 
        real *results, int n )
{
    extern __shared__ real e_kin_s[];
    unsigned int i, mask;
    int offset;
    real e_kin;
    rvec p;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    mask = __ballot_sync( FULL_MASK, i < n );

    if ( i < n )
    {
        rvec_Scale( p, sbp[ my_atoms[i].type ].mass, my_atoms[i].v );
        e_kin = 0.5 * rvec_Dot( p, my_atoms[i].v );

        /* warp-level sum using registers within a warp */
        for ( offset = warpSize >> 1; offset > 0; offset /= 2 )
        {
            e_kin += __shfl_down_sync( mask, e_kin, offset );
        }

        /* first thread within a warp writes warp-level sum to shared memory */
        if ( threadIdx.x % warpSize == 0 )
        {
            e_kin_s[threadIdx.x >> 5] = e_kin;
        }
    }
    __syncthreads( );

    /* block-level sum using shared memory */
    for ( offset = blockDim.x >> 6; offset > 0; offset >>= 1 )
    {
        if ( threadIdx.x < offset )
        {
            e_kin_s[threadIdx.x] += e_kin_s[threadIdx.x + offset];
        }

        __syncthreads( );
    }

    /* one thread writes the block-level partial sum
     * of the reduction back to global memory */
    if ( threadIdx.x == 0 )
    {
        results[blockIdx.x] = e_kin_s[0];
    }
}


CUDA_GLOBAL void k_generate_initial_velocities( single_body_parameters *sbp,
        reax_atom *my_atoms, real T, int n )
{
    int i;
    real m, scale, norm;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    if ( T <= 0.1 )
    {
        rvec_MakeZero( my_atoms[i].v );
    }
    else
    {
        cuda_rvec_Random( my_atoms[i].v );

        norm = rvec_Norm_Sqr( my_atoms[i].v );
        m = sbp[ my_atoms[i].type ].mass;
        scale = SQRT( m * norm / (3.0 * K_B * T) );

        rvec_Scale( my_atoms[i].v, 1.0 / scale, my_atoms[i].v );
    }
}


CUDA_GLOBAL void k_compute_pressure( reax_atom *my_atoms, simulation_box *big_box,
        rvec *int_press, int n )
{
    reax_atom *p_atom;
    rvec tx;
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    p_atom = &my_atoms[i];
    rvec_MakeZero( int_press[i] );

    /* transform x into unit box coordinates, store in tx */
    Transform_to_UnitBox( p_atom->x, big_box, 1, tx );

    /* this atom's contribution to internal pressure */
    rvec_Multiply( int_press[i], p_atom->f, tx );
}


static void Cuda_Compute_Momentum( reax_system *system, control_params *control,
        storage *workspace, rvec xcm, rvec vcm, rvec amcm )
{
    rvec *spad;

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
            sizeof(rvec) * (control->blocks + 1),
            "Cuda_Compute_Momentum::workspace->scratch" );
    spad = (rvec *) workspace->scratch;

    // xcm
    cuda_memset( spad, 0, sizeof(rvec) * (control->blocks + 1),
            "Cuda_Compute_Momentum::spad" );
    
    k_center_of_mass_blocks_xcm <<< control->blocks, control->block_size,
                                sizeof(rvec) * (control->block_size / 32) >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, spad, system->n );
    cudaCheckError( );
    
    k_reduction_rvec <<< 1, control->blocks_pow_2,
                     sizeof(rvec) * (control->blocks_pow_2 / 32) >>>
            ( spad, &spad[control->blocks], control->blocks );
    cudaCheckError( );

    copy_host_device( xcm, &spad[control->blocks], sizeof(rvec),
            hipMemcpyDeviceToHost, "Cuda_Compute_Momentum::xcm" );
    
    // vcm
    cuda_memset( spad, 0, sizeof(rvec) * (control->blocks + 1),
            "Cuda_Compute_Momentum::spad" );
    
    k_center_of_mass_blocks_vcm <<< control->blocks, control->block_size,
                                sizeof(rvec) * (control->block_size / 32) >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, spad, system->n );
    cudaCheckError( );
    
    k_reduction_rvec <<< 1, control->blocks_pow_2,
                     sizeof(rvec) * (control->blocks_pow_2 / 32) >>>
        ( spad, &spad[control->blocks], control->blocks );
    cudaCheckError( );

    copy_host_device( vcm, &spad[control->blocks], sizeof(rvec),
        hipMemcpyDeviceToHost, "Cuda_Compute_Momentum::vcm" );
    
    // amcm
    cuda_memset( spad, 0,  sizeof(rvec) * (control->blocks + 1),
            "Cuda_Compute_Momentum::spad");
    
    k_center_of_mass_blocks_amcm <<< control->blocks, control->block_size,
                                 sizeof(rvec) * (control->block_size / 32) >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, spad, system->n );
    cudaCheckError( );
    
    k_reduction_rvec <<< 1, control->blocks_pow_2,
                     sizeof(rvec) * (control->blocks_pow_2 / 32) >>>
        ( spad, &spad[control->blocks], control->blocks );
    cudaCheckError( );

    copy_host_device( amcm, &spad[control->blocks], sizeof(rvec),
        hipMemcpyDeviceToHost,"Cuda_Compute_Momentum::amcm" );
}


static void Cuda_Compute_Inertial_Tensor( reax_system *system, control_params *control,
        storage *workspace, real *t, rvec my_xcm )
{
    real *spad;

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
            sizeof(real) * 6 * (control->blocks + 1),
            "Cuda_Compute_Inertial_Tensor::workspace->scratch" );
    spad = (real *) workspace->scratch;
    cuda_memset( spad, 0, sizeof(real) * 6 * (control->blocks + 1),
            "Cuda_Compute_Intertial_Tensor::tmp" );

    k_compute_inertial_tensor_xx_xy <<< control->blocks, control->block_size,
                                sizeof(real) * 2 * (control->block_size / 32) >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, spad,
          my_xcm[0], my_xcm[1], my_xcm[2], system->n );
    cudaCheckError( );

    k_compute_inertial_tensor_xz_yy <<< control->blocks, control->block_size,
                                sizeof(real) * 2 * (control->block_size / 32) >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, spad,
          my_xcm[0], my_xcm[1], my_xcm[2], system->n );
    cudaCheckError( );

    k_compute_inertial_tensor_yz_zz <<< control->blocks, control->block_size,
                                sizeof(real) * 2 * (control->block_size / 32) >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, spad,
          my_xcm[0], my_xcm[1], my_xcm[2], system->n );
    cudaCheckError( );

    /* reduction of block-level partial sums for inertial tensor */
    k_compute_inertial_tensor_blocks <<< 1, control->blocks_pow_2,
                              sizeof(real) * 6 * control->blocks_pow_2 >>>
        ( spad, &spad[6 * control->blocks], control->blocks );
    cudaCheckError( );

    copy_host_device( t, &spad[6 * control->blocks],
        sizeof(real) * 6, hipMemcpyDeviceToHost,
        "Cuda_Compute_Intertial_Tensor::t" );
}


void Cuda_Generate_Initial_Velocities( reax_system *system, real T )
{
    int blocks;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    if ( T > 0.1 )
    {
        Cuda_Randomize( );
    }

    k_generate_initial_velocities <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, T, system->n );
}


extern "C" void Cuda_Compute_Kinetic_Energy( reax_system *system,
        control_params *control, storage *workspace, simulation_data *data,
        MPI_Comm comm )
{
    int ret;
    real *block_energy;

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
            sizeof(real) * (control->blocks + 1),
            "Cuda_Compute_Kinetic_Energy::workspace->scratch" );
    block_energy = (real *) workspace->scratch;
    cuda_memset( block_energy, 0, sizeof(real) * (control->blocks + 1),
            "Cuda_Compute_Kinetic_Energy::tmp" );

    data->my_en.e_kin = 0.0;

    k_compute_kinetic_energy <<< control->blocks, control->block_size,
                             sizeof(real) * (control->block_size / 32) >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, block_energy, system->n );
    cudaCheckError( );

    /* note: above kernel sums the kinetic energy contribution within blocks,
     * and this call finishes the global reduction across all blocks */
    Cuda_Reduction_Sum( block_energy, &block_energy[control->blocks], control->blocks );

    copy_host_device( &data->my_en.e_kin, &block_energy[control->blocks],
            sizeof(real), hipMemcpyDeviceToHost,
            "Cuda_Compute_Kinetic_Energy::tmp" );

    ret = MPI_Allreduce( &data->my_en.e_kin, &data->sys_en.e_kin,
            1, MPI_DOUBLE, MPI_SUM, comm );
    Check_MPI_Error( ret, __FILE__, __LINE__ );

    data->therm.T = 2.0 * data->sys_en.e_kin / (data->N_f * K_B);

    /* avoid T being an absolute zero, might cause F.P.E! */
    if ( FABS(data->therm.T) < ALMOST_ZERO )
    {
        data->therm.T = ALMOST_ZERO;
    }
}


void Cuda_Compute_Total_Mass( reax_system *system, control_params *control,
        storage *workspace, simulation_data *data, MPI_Comm comm  )
{
    int ret;
    real M_l, *spad_real;

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
            sizeof(real) * (control->blocks + 1),
            "Cuda_Compute_Total_Mass::workspace->scratch" );
    spad_real = (real *) workspace->scratch;
    cuda_memset( spad_real, 0, sizeof(real) * (control->blocks + 1),
            "Cuda_Compute_Total_Mass::spad_real" );

    k_compute_total_mass <<< control->blocks, control->block_size,
                         sizeof(real) * (control->block_size / 32) >>>
        ( system->reax_param.d_sbp, system->d_my_atoms, spad_real, system->n );
    cudaCheckError( );

    /* note: above kernel sums the mass contribution within blocks,
     * and this call finishes the global reduction across all blocks */
    Cuda_Reduction_Sum( spad_real, &spad_real[control->blocks], control->blocks );

    copy_host_device( &M_l, &spad_real[control->blocks], sizeof(real), 
            hipMemcpyDeviceToHost, "total_mass:M_l" );

    ret = MPI_Allreduce( &M_l, &data->M, 1, MPI_DOUBLE, MPI_SUM, comm );
    Check_MPI_Error( ret, __FILE__, __LINE__ );

    data->inv_M = 1.0 / data->M;
}


extern "C" void Cuda_Compute_Center_of_Mass( reax_system *system,
        control_params *control, storage *workspace, simulation_data *data,
        mpi_datatypes *mpi_data, MPI_Comm comm )
{
    int ret;
    real det; //xx, xy, xz, yy, yz, zz;
    real tmp_mat[6], tot_mat[6];
    rvec my_xcm, my_vcm, my_amcm, my_avcm;
    rvec tvec;
    rtensor mat, inv;

    rvec_MakeZero( my_xcm );  // position of CoM
    rvec_MakeZero( my_vcm );  // velocity of CoM
    rvec_MakeZero( my_amcm ); // angular momentum of CoM
    rvec_MakeZero( my_avcm ); // angular velocity of CoM

    /* Compute the position, vel. and ang. momentum about the center of mass */
    Cuda_Compute_Momentum( system, control, workspace, my_xcm, my_vcm, my_amcm );

    ret = MPI_Allreduce( my_xcm, data->xcm, 3, MPI_DOUBLE, MPI_SUM, comm );
    Check_MPI_Error( ret, __FILE__, __LINE__ );
    ret = MPI_Allreduce( my_vcm, data->vcm, 3, MPI_DOUBLE, MPI_SUM, comm );
    Check_MPI_Error( ret, __FILE__, __LINE__ );
    ret = MPI_Allreduce( my_amcm, data->amcm, 3, MPI_DOUBLE, MPI_SUM, comm );
    Check_MPI_Error( ret, __FILE__, __LINE__ );

    rvec_Scale( data->xcm, data->inv_M, data->xcm );
    rvec_Scale( data->vcm, data->inv_M, data->vcm );
    rvec_Cross( tvec, data->xcm, data->vcm );
    rvec_ScaledAdd( data->amcm, -data->M, tvec );
    data->etran_cm = 0.5 * data->M * rvec_Norm_Sqr( data->vcm );

    /* Calculate and then invert the inertial tensor */
    Cuda_Compute_Inertial_Tensor( system, control, workspace, tmp_mat, data->xcm );

    ret = MPI_Reduce( tmp_mat, tot_mat, 6, MPI_DOUBLE, MPI_SUM, MASTER_NODE, comm );
    Check_MPI_Error( ret, __FILE__, __LINE__ );

    if ( system->my_rank == MASTER_NODE )
    {
        mat[0][0] = tot_mat[3] + tot_mat[5];  // yy + zz;
        mat[0][1] = mat[1][0] = -tot_mat[1];  // -xy;
        mat[0][2] = mat[2][0] = -tot_mat[2];  // -xz;
        mat[1][1] = tot_mat[0] + tot_mat[5];  // xx + zz;
        mat[2][1] = mat[1][2] = -tot_mat[4];  // -yz;
        mat[2][2] = tot_mat[0] + tot_mat[3];  // xx + yy;

        /* invert the inertial tensor */
        det = ( mat[0][0] * mat[1][1] * mat[2][2] +
                mat[0][1] * mat[1][2] * mat[2][0] +
                mat[0][2] * mat[1][0] * mat[2][1] ) -
              ( mat[0][0] * mat[1][2] * mat[2][1] +
                mat[0][1] * mat[1][0] * mat[2][2] +
                mat[0][2] * mat[1][1] * mat[2][0] );

        inv[0][0] = mat[1][1] * mat[2][2] - mat[1][2] * mat[2][1];
        inv[0][1] = mat[0][2] * mat[2][1] - mat[0][1] * mat[2][2];
        inv[0][2] = mat[0][1] * mat[1][2] - mat[0][2] * mat[1][1];
        inv[1][0] = mat[1][2] * mat[2][0] - mat[1][0] * mat[2][2];
        inv[1][1] = mat[0][0] * mat[2][2] - mat[0][2] * mat[2][0];
        inv[1][2] = mat[0][2] * mat[1][0] - mat[0][0] * mat[1][2];
        inv[2][0] = mat[1][0] * mat[2][1] - mat[2][0] * mat[1][1];
        inv[2][1] = mat[2][0] * mat[0][1] - mat[0][0] * mat[2][1];
        inv[2][2] = mat[0][0] * mat[1][1] - mat[1][0] * mat[0][1];

        if ( det > ALMOST_ZERO )
        {
            rtensor_Scale( inv, 1.0 / det, inv );
        }
        else
        {
            rtensor_MakeZero( inv );
        }

        /* Compute the angular velocity about the centre of mass */
        rtensor_MatVec( data->avcm, inv, data->amcm );
    }

    ret = MPI_Bcast( data->avcm, 3, MPI_DOUBLE, MASTER_NODE, comm );
    Check_MPI_Error( ret, __FILE__, __LINE__ );

    /* Compute the rotational energy */
    data->erot_cm = 0.5 * E_CONV * rvec_Dot( data->avcm, data->amcm );

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "xcm:  %24.15e %24.15e %24.15e\n",
             data->xcm[0], data->xcm[1], data->xcm[2] );
    fprintf( stderr, "vcm:  %24.15e %24.15e %24.15e\n",
             data->vcm[0], data->vcm[1], data->vcm[2] );
    fprintf( stderr, "amcm: %24.15e %24.15e %24.15e\n",
             data->amcm[0], data->amcm[1], data->amcm[2] );
    fprintf( stderr, "mat:  %f %f %f\n     %f %f %f\n     %f %f %f\n",
       mat[0][0], mat[0][1], mat[0][2],
       mat[1][0], mat[1][1], mat[1][2],
       mat[2][0], mat[2][1], mat[2][2] );
    fprintf( stderr, "inv:  %g %g %g\n     %g %g %g\n     %g %g %g\n",
       inv[0][0], inv[0][1], inv[0][2],
       inv[1][0], inv[1][1], inv[1][2],
       inv[2][0], inv[2][1], inv[2][2] );
    fprintf( stderr, "avcm: %24.15e %24.15e %24.15e\n",
             data->avcm[0], data->avcm[1], data->avcm[2] );
#endif
}


/* IMPORTANT: This function assumes that current kinetic energy
 * the system is already computed
 *
 * IMPORTANT: In Klein's paper, it is stated that a dU/dV term needs
 *  to be added when there are long-range interactions or long-range
 *  corrections to short-range interactions present.
 *  We may want to add that for more accuracy.
 */
void Cuda_Compute_Pressure( reax_system* system, control_params *control,
        storage *workspace, simulation_data* data, mpi_datatypes *mpi_data )
{
    int ret;
    rvec *rvec_spad, int_press;
    simulation_box *big_box;
    
    big_box = &system->big_box;

    /* 0: both int and ext, 1: ext only, 2: int only */
    if ( control->press_mode == 0 || control->press_mode == 2 )
    {
        cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
                sizeof(rvec) * (system->n + control->blocks + 1),
                "Cuda_Compute_Pressure::workspace->scratch" );
        rvec_spad = (rvec *) workspace->scratch;

        k_compute_pressure <<< control->blocks, control->block_size >>>
            ( system->d_my_atoms, system->d_big_box, rvec_spad,
              system->n );

        k_reduction_rvec <<< control->blocks, control->block_size,
                         sizeof(rvec) * (control->block_size / 32) >>>
            ( rvec_spad, &rvec_spad[system->n],  system->n );
        cudaCheckError( );

        k_reduction_rvec <<< 1, control->blocks_pow_2,
                         sizeof(rvec) * (control->blocks_pow_2 / 32) >>>
            ( &rvec_spad[system->n], &rvec_spad[system->n + control->blocks],
              control->blocks );
        cudaCheckError( );

        copy_host_device( &int_press, &rvec_spad[system->n + control->blocks],
                sizeof(rvec), hipMemcpyDeviceToHost,
                "Cuda_Compute_Pressure::int_press" );
    }

    /* sum up internal and external pressure */
    ret = MPI_Allreduce( int_press, data->int_press,
            3, MPI_DOUBLE, MPI_SUM, mpi_data->comm_mesh3D );
    Check_MPI_Error( ret, __FILE__, __LINE__ );
    ret = MPI_Allreduce( data->my_ext_press, data->ext_press,
            3, MPI_DOUBLE, MPI_SUM, mpi_data->comm_mesh3D );
    Check_MPI_Error( ret, __FILE__, __LINE__ );

    /* kinetic contribution */
    data->kin_press = 2.0 * (E_CONV * data->sys_en.e_kin)
        / (3.0 * big_box->V * P_CONV);

    /* Calculate total pressure in each direction */
    data->tot_press[0] = data->kin_press -
        (( data->int_press[0] + data->ext_press[0] ) /
         ( big_box->box_norms[1] * big_box->box_norms[2] * P_CONV ));

    data->tot_press[1] = data->kin_press -
        (( data->int_press[1] + data->ext_press[1] ) /
         ( big_box->box_norms[0] * big_box->box_norms[2] * P_CONV ));

    data->tot_press[2] = data->kin_press -
        (( data->int_press[2] + data->ext_press[2] ) /
         ( big_box->box_norms[0] * big_box->box_norms[1] * P_CONV ));

    /* Average pressure for the whole box */
    data->iso_bar.P =
        ( data->tot_press[0] + data->tot_press[1] + data->tot_press[2] ) / 3.0;
}

#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_valence_angles.h"

#if defined(CUDA_ACCUM_ATOMIC)
#include "cuda_helpers.h"
#endif
#include "cuda_list.h"
#include "cuda_reduction.h"
#include "cuda_utils.h"

#include "../index_utils.h"
#include "../vector.h"


/* Compute 3-body interactions, in which the main role is played by
   atom j, which sits in the middle of the other two atoms i and k. */
CUDA_GLOBAL void k_valence_angles_part1( reax_atom *my_atoms,
        global_parameters gp, single_body_parameters *sbp, three_body_header *d_thbh,
        control_params *control, storage workspace, reax_list bond_list,
        reax_list thb_list, int n, int N, int num_atom_types,
        real *e_ang_g, real *e_pen_g, real *e_coa_g )
{
    int i, j, pi, k, pk, t;
    int type_i, type_j, type_k;
    int start_j, end_j;
//    int start_pk, end_pk;
    int cnt, num_thb_intrs;
    real temp, temp_bo_jt, pBOjt7;
    real p_val1, p_val2, p_val3, p_val4, p_val5;
    real p_val6, p_val7, p_val8, p_val9, p_val10;
    real p_pen1, p_pen2, p_pen3, p_pen4;
    real p_coa1, p_coa2, p_coa3, p_coa4;
    real trm8, expval6, expval7, expval2theta, expval12theta, exp3ij, exp3jk;
    real exp_pen2ij, exp_pen2jk, exp_pen3, exp_pen4, trm_pen34, exp_coa2;
    real dSBO1, dSBO2, SBO, SBO2, CSBO2, SBOp, prod_SBO, vlpadj;
    real CEval1, CEval2, CEval3, CEval4, CEval5, CEval6, CEval7, CEval8;
    real CEpen1, CEpen2, CEpen3;
    real e_ang_l, e_coa, e_coa_l, e_pen, e_pen_l;
    real CEcoa1, CEcoa2, CEcoa3, CEcoa4, CEcoa5;
    real Cf7ij, Cf7jk, Cf8j, Cf9j;
    real f7_ij, f7_jk, f8_Dj, f9_Dj;
    real Ctheta_0, theta_0, theta_00, theta, cos_theta, sin_theta;
    real BOA_ij, BOA_jk;
    rvec f_j_l;
    three_body_header *thbh;
    three_body_parameters *thbp;
    three_body_interaction_data *p_ijk;
//    three_body_interaction_data *p_kji;
    bond_data *pbond_ij, *pbond_jk, *pbond_jt;
    bond_order_data *bo_ij, *bo_jk, *bo_jt;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= N )
    {
        return;
    }

    p_pen2 = gp.l[19];
    p_pen3 = gp.l[20];
    p_pen4 = gp.l[21];
    p_coa2 = gp.l[2];
    p_coa3 = gp.l[38];
    p_coa4 = gp.l[30];
    p_val6 = gp.l[14];
    p_val8 = gp.l[33];
    p_val9 = gp.l[16];
    p_val10 = gp.l[17];
    //num_thb_intrs = j * THREE_BODY_OFFSET;
    e_ang_l = 0.0;
    e_coa_l = 0.0;
    e_pen_l = 0.0;
    rvec_MakeZero( f_j_l );

    type_j = my_atoms[j].type;
    start_j = Start_Index( j, &bond_list );
    end_j = End_Index( j, &bond_list );
    p_val3 = sbp[ type_j ].p_val3;
    p_val5 = sbp[ type_j ].p_val5;

    /* sum of pi and pi-pi BO terms for all neighbors of atom j,
     * used in determining the equilibrium angle between i-j-k */
    SBOp = 0.0;
    /* product of e^{-BO_j^8} terms for all neighbors of atom j,
     * used in determining the equilibrium angle between i-j-k */
    prod_SBO = 1.0;

    for ( t = start_j; t < end_j; ++t )
    {
        bo_jt = &bond_list.bond_list[t].bo_data;
        SBOp += bo_jt->BO_pi + bo_jt->BO_pi2;
        temp = SQR( bo_jt->BO );
        temp *= temp;
        temp *= temp;
        prod_SBO *= EXP( -temp );
    }

    /* modifications to match Adri's code - 09/01/09 */
    if ( workspace.vlpex[j] >= 0.0 )
    {
        vlpadj = 0.0;
        dSBO2 = prod_SBO - 1.0;
    }
    else
    {
        vlpadj = workspace.nlp[j];
        dSBO2 = (prod_SBO - 1.0) * (1.0 - p_val8 * workspace.dDelta_lp[j]);
    }

    SBO = SBOp + (1.0 - prod_SBO) * (-workspace.Delta_boc[j] - p_val8 * vlpadj);
    dSBO1 = -8.0 * prod_SBO * ( workspace.Delta_boc[j] + p_val8 * vlpadj );

    if ( SBO <= 0.0 )
    {
        SBO2 = 0.0;
        CSBO2 = 0.0;
    }
    else if ( SBO > 0.0 && SBO <= 1.0 )
    {
        SBO2 = POW( SBO, p_val9 );
        CSBO2 = p_val9 * POW( SBO, p_val9 - 1.0 );
    }
    else if( SBO > 1.0 && SBO < 2.0 )
    {
        SBO2 = 2.0 - POW( 2.0 - SBO, p_val9 );
        CSBO2 = p_val9 * POW( 2.0 - SBO, p_val9 - 1.0 );
    }
    else
    {
        SBO2 = 2.0;
        CSBO2 = 0.0;
    }

    expval6 = EXP( p_val6 * workspace.Delta_boc[j] );

    for ( pi = start_j; pi < end_j; ++pi )
    {
        num_thb_intrs = Start_Index( pi, &thb_list );
        pbond_ij = &bond_list.bond_list[pi];
        bo_ij = &pbond_ij->bo_data;
        BOA_ij = bo_ij->BO - control->thb_cut;

        if ( BOA_ij >= 0.0 && (j < n || pbond_ij->nbr < n) )
        {
            i = pbond_ij->nbr;
            type_i = my_atoms[i].type;

            /* first copy 3-body intrs from previously computed ones where i > k.
             * IMPORTANT: if it is less costly to compute theta and its
             * derivative, we should definitely re-compute them,
             * instead of copying!
             * in the second for-loop below, we compute only new 3-body intrs
             * where i < k */
            // The copy loop commented out because strange asynchronous issues started to surface
            // Each kernel now manually generates everything
//            for( pk = start_j; pk < pi; ++pk )
//            {
//                start_pk = Start_Index( pk, &thb_list );
//                end_pk = End_Index( pk, &thb_list );
//
//                for( t = start_pk; t < end_pk; ++t )
//                {
//                    if( thb_list.three_body_list[t].thb == i )
//                    {
//                        p_ijk = &thb_list.three_body_list[num_thb_intrs];
//                        p_kji = &thb_list.three_body_list[t];
//
//                        p_ijk->thb = bond_list.bond_list[pk].nbr;
//                        p_ijk->pthb  = pk;
//                        p_ijk->theta = p_kji->theta;
//                        rvec_Copy( p_ijk->dcos_di, p_kji->dcos_dk );
//                        rvec_Copy( p_ijk->dcos_dj, p_kji->dcos_dj );
//                        rvec_Copy( p_ijk->dcos_dk, p_kji->dcos_di );
//
//                        ++num_thb_intrs;
//                        break;
//                    }
//                }
//            }

            /* and this is the second for loop mentioned above */
            // Except that now the loop goes all the way from start_j to end_j
            for ( pk = start_j; pk < end_j; ++pk )
            {
                if ( pk == pi )
                {
                    continue;
                }

                pbond_jk = &bond_list.bond_list[pk];
                bo_jk = &pbond_jk->bo_data;
                BOA_jk = bo_jk->BO - control->thb_cut;

                if ( BOA_jk < 0.0 )
                {
                    continue;
                }

                k = pbond_jk->nbr;
                type_k = my_atoms[k].type;
                p_ijk = &thb_list.three_body_list[num_thb_intrs];

                Calculate_Theta( pbond_ij->dvec, pbond_ij->d,
                        pbond_jk->dvec, pbond_jk->d,
                        &theta, &cos_theta );

                Calculate_dCos_Theta( pbond_ij->dvec, pbond_ij->d,
                        pbond_jk->dvec, pbond_jk->d,
                        &p_ijk->dcos_di, &p_ijk->dcos_dj,
                        &p_ijk->dcos_dk );

                p_ijk->thb = k;
                p_ijk->pthb = pk;
                p_ijk->theta = theta;

                sin_theta = SIN( theta );
                if ( sin_theta < 1.0e-5 )
                {
                    sin_theta = 1.0e-5;
                }

                ++num_thb_intrs;

                /* Fortran ReaxFF code hard-codes the constant below
                 * as of 2019-02-27, so use that for now */
                if ( j >= n || BOA_jk < 0.0 || (bo_ij->BO * bo_jk->BO) < 0.00001 )
//                if ( j >= n || BOA_jk < 0.0 || (bo_ij->BO * bo_jk->BO) < SQR(control->thb_cut) )
                {
                    continue;
                }

                thbh = &d_thbh[
                    index_thbp(type_i, type_j, type_k, num_atom_types) ];

                for ( cnt = 0; cnt < thbh->cnt; ++cnt )
                {
                    /* valence angle does not exist in the force field */
                    if ( FABS(thbh->prm[cnt].p_val1) < 0.001 )
                    {
                        continue;
                    }

                    thbp = &thbh->prm[cnt];

                    /* calculate valence angle energy */
                    p_val1 = thbp->p_val1;
                    p_val2 = thbp->p_val2;
                    p_val4 = thbp->p_val4;
                    p_val7 = thbp->p_val7;
                    theta_00 = thbp->theta_00;

                    exp3ij = EXP( -p_val3 * POW( BOA_ij, p_val4 ) );
                    f7_ij = 1.0 - exp3ij;
                    Cf7ij = p_val3 * p_val4
                        * POW( BOA_ij, p_val4 - 1.0 ) * exp3ij;

                    exp3jk = EXP( -p_val3 * POW( BOA_jk, p_val4 ) );
                    f7_jk = 1.0 - exp3jk;
                    Cf7jk = p_val3 * p_val4
                        * POW( BOA_jk, p_val4 - 1.0 ) * exp3jk;

                    expval7 = EXP( -p_val7 * workspace.Delta_boc[j] );
                    trm8 = 1.0 + expval6 + expval7;
                    f8_Dj = p_val5 - (p_val5 - 1.0) * (2.0 + expval6) / trm8;
                    Cf8j = ( (1.0 - p_val5) / SQR(trm8) )
                        * (p_val6 * expval6 * trm8
                                - (2.0 + expval6) * ( p_val6 * expval6 - p_val7 * expval7) );

                    theta_0 = 180.0 - theta_00 * (1.0 - EXP(-p_val10 * (2.0 - SBO2)));
                    theta_0 = DEG2RAD( theta_0 );

                    expval2theta = p_val1 * EXP(-p_val2 * SQR(theta_0 - theta));
                    if ( p_val1 >= 0.0 )
                    {
                        expval12theta = p_val1 - expval2theta;
                    }
                    /* to avoid linear Me-H-Me angles (6/6/06) */
                    else
                    {
                        expval12theta = -expval2theta;
                    }

                    CEval1 = Cf7ij * f7_jk * f8_Dj * expval12theta;
                    CEval2 = Cf7jk * f7_ij * f8_Dj * expval12theta;
                    CEval3 = Cf8j * f7_ij * f7_jk * expval12theta;
                    CEval4 = 2.0 * p_val2 * f7_ij * f7_jk * f8_Dj
                        * expval2theta * (theta_0 - theta);

                    Ctheta_0 = p_val10 * DEG2RAD(theta_00)
                        * EXP( -p_val10 * (2.0 - SBO2) );

                    CEval5 = CEval4 * Ctheta_0 * CSBO2;
                    CEval6 = CEval5 * dSBO1;
                    CEval7 = CEval5 * dSBO2;
                    CEval8 = CEval4 / sin_theta;

                    if ( pk < pi )
                    {
                        e_ang_l += f7_ij * f7_jk * f8_Dj * expval12theta;
                    }

                    /* calculate penalty for double bonds in valency angles */
                    p_pen1 = thbp->p_pen1;

                    exp_pen2ij = EXP( -p_pen2 * SQR( BOA_ij - 2.0 ) );
                    exp_pen2jk = EXP( -p_pen2 * SQR( BOA_jk - 2.0 ) );
                    exp_pen3 = EXP( -p_pen3 * workspace.Delta[j] );
                    exp_pen4 = EXP(  p_pen4 * workspace.Delta[j] );
                    trm_pen34 = 1.0 + exp_pen3 + exp_pen4;
                    f9_Dj = ( 2.0 + exp_pen3 ) / trm_pen34;
                    Cf9j = (-p_pen3 * exp_pen3 * trm_pen34
                            - (2.0 + exp_pen3) * ( -p_pen3 * exp_pen3
                                + p_pen4 * exp_pen4 )) / SQR( trm_pen34 );

                    /* very important: since each kernel generates all interactions,
                     * need to prevent all energies becoming duplicates */
                    if ( pk < pi )
                    {
                        e_pen = p_pen1 * f9_Dj * exp_pen2ij * exp_pen2jk;
                        e_pen_l += e_pen;
                    }

                    CEpen1 = e_pen * Cf9j / f9_Dj;
                    temp = -2.0 * p_pen2 * e_pen;
                    CEpen2 = temp * (BOA_ij - 2.0);
                    CEpen3 = temp * (BOA_jk - 2.0);

                    /* calculate valency angle conjugation energy */
                    p_coa1 = thbp->p_coa1;

                    exp_coa2 = EXP( p_coa2 * workspace.Delta_boc[j] );
                    e_coa = p_coa1
                        * EXP( -p_coa4 * SQR(BOA_ij - 1.5) )
                        * EXP( -p_coa4 * SQR(BOA_jk - 1.5) )
                        * EXP( -p_coa3 * SQR(workspace.total_bond_order[i] - BOA_ij) )
                        * EXP( -p_coa3 * SQR(workspace.total_bond_order[k] - BOA_jk) )
                        / (1.0 + exp_coa2);
                    /* similar to above comment regarding if statement */
                    if ( pk < pi )
                    {
                        e_coa_l += e_coa;
                    }

                    CEcoa1 = -2.0 * p_coa4 * (BOA_ij - 1.5) * e_coa;
                    CEcoa2 = -2.0 * p_coa4 * (BOA_jk - 1.5) * e_coa;
                    CEcoa3 = -p_coa2 * exp_coa2 * e_coa / (1.0 + exp_coa2);
                    CEcoa4 = -2.0 * p_coa3 * (workspace.total_bond_order[i] - BOA_ij) * e_coa;
                    CEcoa5 = -2.0 * p_coa3 * (workspace.total_bond_order[k] - BOA_jk) * e_coa;

                    /* calculate force contributions */
                    /* we must again check for pk < pi for entire forces part */
                    if ( pk < pi )
                    {
#if !defined(CUDA_ACCUM_ATOMIC)
                        bo_ij->Cdbo += CEval1 + CEpen2 + (CEcoa1 - CEcoa4);
                        bo_jk->Cdbo += CEval2 + CEpen3 + (CEcoa2 - CEcoa5);
                        workspace.CdDelta[j] += (CEval3 + CEval7) + CEpen1 + CEcoa3;
                        pbond_ij->va_CdDelta += CEcoa4;
                        pbond_jk->va_CdDelta += CEcoa5;
#else
                        atomicAdd( &bo_ij->Cdbo, CEval1 + CEpen2 + (CEcoa1 - CEcoa4) );
                        atomicAdd( &bo_jk->Cdbo, CEval2 + CEpen3 + (CEcoa2 - CEcoa5) );
                        atomicAdd( &workspace.CdDelta[j], (CEval3 + CEval7) + CEpen1 + CEcoa3 );
                        atomicAdd( &workspace.CdDelta[i], CEcoa4 );
                        atomicAdd( &workspace.CdDelta[k], CEcoa5 );
#endif

                        for ( t = start_j; t < end_j; ++t )
                        {
                            pbond_jt = &bond_list.bond_list[t];
                            bo_jt = &pbond_jt->bo_data;
                            temp_bo_jt = bo_jt->BO;
                            temp = CUBE( temp_bo_jt );
                            pBOjt7 = temp * temp * temp_bo_jt;

#if !defined(CUDA_ACCUM_ATOMIC)
                            bo_jt->Cdbo += (CEval6 * pBOjt7);
                            bo_jt->Cdbopi += CEval5;
                            bo_jt->Cdbopi2 += CEval5;
#else
                            atomicAdd( &bo_jt->Cdbo, CEval6 * pBOjt7 );
                            atomicAdd( &bo_jt->Cdbopi, CEval5 );
                            atomicAdd( &bo_jt->Cdbopi2, CEval5 );
#endif
                        }

#if !defined(CUDA_ACCUM_ATOMIC)
                        rvec_ScaledAdd( pbond_ij->va_f, CEval8, p_ijk->dcos_di );
                        rvec_ScaledAdd( f_j_l, CEval8, p_ijk->dcos_dj );
                        rvec_ScaledAdd( pbond_jk->va_f, CEval8, p_ijk->dcos_dk );
#else
                        atomic_rvecScaledAdd( workspace.f[i], CEval8, p_ijk->dcos_di );
                        rvec_ScaledAdd( f_j_l, CEval8, p_ijk->dcos_dj );
                        atomic_rvecScaledAdd( workspace.f[k], CEval8, p_ijk->dcos_dk );
#endif
                    }
                }
            }
        }

        Set_End_Index( pi, num_thb_intrs, &thb_list );
    }

#if !defined(CUDA_ACCUM_ATOMIC)
    rvec_Add( workspace.f[j], f_j_l );
    e_ang_g[j] = e_ang_l;
    e_coa_g[j] = e_coa_l;
    e_pen_g[j] = e_pen_l;
#else
    atomic_rvecAdd( workspace.f[j], f_j_l );
    atomicAdd( (double *) e_ang_g, (double) e_ang_l );
    atomicAdd( (double *) e_coa_g, (double) e_coa_l );
    atomicAdd( (double *) e_pen_g, (double) e_pen_l );
#endif
}


/* Compute 3-body interactions, in which the main role is played by
   atom j, which sits in the middle of the other two atoms i and k. */
CUDA_GLOBAL void k_valence_angles_virial_part1( reax_atom *my_atoms,
        global_parameters gp, single_body_parameters *sbp, three_body_header *d_thbh,
        control_params *control, storage workspace, reax_list bond_list,
        reax_list thb_list, int n, int N, int num_atom_types,
        real *e_ang_g, real *e_pen_g, real *e_coa_g, rvec *ext_press_g )
{
    int i, j, pi, k, pk, t;
    int type_i, type_j, type_k;
    int start_j, end_j;
//    int start_pk, end_pk;
    int cnt, num_thb_intrs;
    real temp, temp_bo_jt, pBOjt7;
    real p_val1, p_val2, p_val3, p_val4, p_val5;
    real p_val6, p_val7, p_val8, p_val9, p_val10;
    real p_pen1, p_pen2, p_pen3, p_pen4;
    real p_coa1, p_coa2, p_coa3, p_coa4;
    real trm8, expval6, expval7, expval2theta, expval12theta, exp3ij, exp3jk;
    real exp_pen2ij, exp_pen2jk, exp_pen3, exp_pen4, trm_pen34, exp_coa2;
    real dSBO1, dSBO2, SBO, SBO2, CSBO2, SBOp, prod_SBO, vlpadj;
    real CEval1, CEval2, CEval3, CEval4, CEval5, CEval6, CEval7, CEval8;
    real CEpen1, CEpen2, CEpen3;
    real e_ang_l, e_coa, e_coa_l, e_pen, e_pen_l;
    real CEcoa1, CEcoa2, CEcoa3, CEcoa4, CEcoa5;
    real Cf7ij, Cf7jk, Cf8j, Cf9j;
    real f7_ij, f7_jk, f8_Dj, f9_Dj;
    real Ctheta_0, theta_0, theta_00, theta, cos_theta, sin_theta;
    real BOA_ij, BOA_jk;
    rvec rvec_temp, f_j_l, ext_press_l;
    three_body_header *thbh;
    three_body_parameters *thbp;
    three_body_interaction_data *p_ijk;
//    three_body_interaction_data *p_kji;
    bond_data *pbond_ij, *pbond_jk, *pbond_jt;
    bond_order_data *bo_ij, *bo_jk, *bo_jt;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= N )
    {
        return;
    }

    p_pen2 = gp.l[19];
    p_pen3 = gp.l[20];
    p_pen4 = gp.l[21];
    p_coa2 = gp.l[2];
    p_coa3 = gp.l[38];
    p_coa4 = gp.l[30];
    p_val6 = gp.l[14];
    p_val8 = gp.l[33];
    p_val9 = gp.l[16];
    p_val10 = gp.l[17];
    //num_thb_intrs = j * THREE_BODY_OFFSET;
    e_ang_l = 0.0;
    e_coa_l = 0.0;
    e_pen_l = 0.0;
    rvec_MakeZero( f_j_l );
    rvec_MakeZero( ext_press_l );

    type_j = my_atoms[j].type;
    start_j = Start_Index( j, &bond_list );
    end_j = End_Index( j, &bond_list );
    p_val3 = sbp[ type_j ].p_val3;
    p_val5 = sbp[ type_j ].p_val5;

    /* sum of pi and pi-pi BO terms for all neighbors of atom j,
     * used in determining the equilibrium angle between i-j-k */
    SBOp = 0.0;
    /* product of e^{-BO_j^8} terms for all neighbors of atom j,
     * used in determining the equilibrium angle between i-j-k */
    prod_SBO = 1.0;

    for ( t = start_j; t < end_j; ++t )
    {
        bo_jt = &bond_list.bond_list[t].bo_data;
        SBOp += bo_jt->BO_pi + bo_jt->BO_pi2;
        temp = SQR( bo_jt->BO );
        temp *= temp;
        temp *= temp;
        prod_SBO *= EXP( -temp );
    }

    /* modifications to match Adri's code - 09/01/09 */
    if ( workspace.vlpex[j] >= 0.0 )
    {
        vlpadj = 0.0;
        dSBO2 = prod_SBO - 1.0;
    }
    else
    {
        vlpadj = workspace.nlp[j];
        dSBO2 = (prod_SBO - 1.0) * (1.0 - p_val8 * workspace.dDelta_lp[j]);
    }

    SBO = SBOp + (1.0 - prod_SBO) * (-workspace.Delta_boc[j] - p_val8 * vlpadj);
    dSBO1 = -8.0 * prod_SBO * ( workspace.Delta_boc[j] + p_val8 * vlpadj );

    if ( SBO <= 0.0 )
    {
        SBO2 = 0.0;
        CSBO2 = 0.0;
    }
    else if ( SBO > 0.0 && SBO <= 1.0 )
    {
        SBO2 = POW( SBO, p_val9 );
        CSBO2 = p_val9 * POW( SBO, p_val9 - 1.0 );
    }
    else if( SBO > 1.0 && SBO < 2.0 )
    {
        SBO2 = 2.0 - POW( 2.0 - SBO, p_val9 );
        CSBO2 = p_val9 * POW( 2.0 - SBO, p_val9 - 1.0 );
    }
    else
    {
        SBO2 = 2.0;
        CSBO2 = 0.0;
    }

    expval6 = EXP( p_val6 * workspace.Delta_boc[j] );

    for ( pi = start_j; pi < end_j; ++pi )
    {
        num_thb_intrs = Start_Index( pi, &thb_list );
        pbond_ij = &bond_list.bond_list[pi];
        bo_ij = &pbond_ij->bo_data;
        BOA_ij = bo_ij->BO - control->thb_cut;

        if ( BOA_ij >= 0.0 && (j < n || pbond_ij->nbr < n) )
        {
            i = pbond_ij->nbr;
            type_i = my_atoms[i].type;

            /* first copy 3-body intrs from previously computed ones where i > k.
             * IMPORTANT: if it is less costly to compute theta and its
             * derivative, we should definitely re-compute them,
             * instead of copying!
             * in the second for-loop below, we compute only new 3-body intrs
             * where i < k */
            // The copy loop commented out because strange asynchronous issues started to surface
            // Each kernel now manually generates everything
//            for( pk = start_j; pk < pi; ++pk )
//            {
//                start_pk = Start_Index( pk, &thb_list );
//                end_pk = End_Index( pk, &thb_list );
//
//                for( t = start_pk; t < end_pk; ++t )
//                {
//                    if( thb_list.three_body_list[t].thb == i )
//                    {
//                        p_ijk = &thb_list.three_body_list[num_thb_intrs];
//                        p_kji = &thb_list.three_body_list[t];
//
//                        p_ijk->thb = bond_list.bond_list[pk].nbr;
//                        p_ijk->pthb  = pk;
//                        p_ijk->theta = p_kji->theta;
//                        rvec_Copy( p_ijk->dcos_di, p_kji->dcos_dk );
//                        rvec_Copy( p_ijk->dcos_dj, p_kji->dcos_dj );
//                        rvec_Copy( p_ijk->dcos_dk, p_kji->dcos_di );
//
//                        ++num_thb_intrs;
//                        break;
//                    }
//                }
//            }

            /* and this is the second for loop mentioned above */
            // Except that now the loop goes all the way from start_j to end_j
            for ( pk = start_j; pk < end_j; ++pk )
            {
                if ( pk == pi )
                {
                    continue;
                }

                pbond_jk = &bond_list.bond_list[pk];
                bo_jk = &pbond_jk->bo_data;
                BOA_jk = bo_jk->BO - control->thb_cut;

                if ( BOA_jk < 0.0 )
                {
                    continue;
                }

                k = pbond_jk->nbr;
                type_k = my_atoms[k].type;
                p_ijk = &thb_list.three_body_list[num_thb_intrs];

                Calculate_Theta( pbond_ij->dvec, pbond_ij->d,
                        pbond_jk->dvec, pbond_jk->d,
                        &theta, &cos_theta );

                Calculate_dCos_Theta( pbond_ij->dvec, pbond_ij->d,
                        pbond_jk->dvec, pbond_jk->d,
                        &p_ijk->dcos_di, &p_ijk->dcos_dj,
                        &p_ijk->dcos_dk );

                p_ijk->thb = k;
                p_ijk->pthb = pk;
                p_ijk->theta = theta;

                sin_theta = SIN( theta );
                if ( sin_theta < 1.0e-5 )
                {
                    sin_theta = 1.0e-5;
                }

                ++num_thb_intrs;

                /* Fortran ReaxFF code hard-codes the constant below
                 * as of 2019-02-27, so use that for now */
                if ( j >= n || BOA_jk < 0.0 || (bo_ij->BO * bo_jk->BO) < 0.00001 )
//                if ( j >= n || BOA_jk < 0.0 || (bo_ij->BO * bo_jk->BO) < SQR(control->thb_cut) )
                {
                    continue;
                }

                thbh = &d_thbh[
                    index_thbp(type_i, type_j, type_k, num_atom_types) ];

                for ( cnt = 0; cnt < thbh->cnt; ++cnt )
                {
                    /* valence angle does not exist in the force field */
                    if ( FABS(thbh->prm[cnt].p_val1) < 0.001 )
                    {
                        continue;
                    }

                    thbp = &thbh->prm[cnt];

                    /* calculate valence angle energy */
                    p_val1 = thbp->p_val1;
                    p_val2 = thbp->p_val2;
                    p_val4 = thbp->p_val4;
                    p_val7 = thbp->p_val7;
                    theta_00 = thbp->theta_00;

                    exp3ij = EXP( -p_val3 * POW( BOA_ij, p_val4 ) );
                    f7_ij = 1.0 - exp3ij;
                    Cf7ij = p_val3 * p_val4
                        * POW( BOA_ij, p_val4 - 1.0 ) * exp3ij;

                    exp3jk = EXP( -p_val3 * POW( BOA_jk, p_val4 ) );
                    f7_jk = 1.0 - exp3jk;
                    Cf7jk = p_val3 * p_val4
                        * POW( BOA_jk, p_val4 - 1.0 ) * exp3jk;

                    expval7 = EXP( -p_val7 * workspace.Delta_boc[j] );
                    trm8 = 1.0 + expval6 + expval7;
                    f8_Dj = p_val5 - (p_val5 - 1.0) * (2.0 + expval6) / trm8;
                    Cf8j = ( (1.0 - p_val5) / SQR(trm8) )
                        * (p_val6 * expval6 * trm8
                                - (2.0 + expval6) * ( p_val6 * expval6 - p_val7 * expval7) );

                    theta_0 = 180.0 - theta_00 * (1.0 - EXP(-p_val10 * (2.0 - SBO2)));
                    theta_0 = DEG2RAD( theta_0 );

                    expval2theta = p_val1 * EXP(-p_val2 * SQR(theta_0 - theta));
                    if ( p_val1 >= 0.0 )
                    {
                        expval12theta = p_val1 - expval2theta;
                    }
                    /* to avoid linear Me-H-Me angles (6/6/06) */
                    else
                    {
                        expval12theta = -expval2theta;
                    }

                    CEval1 = Cf7ij * f7_jk * f8_Dj * expval12theta;
                    CEval2 = Cf7jk * f7_ij * f8_Dj * expval12theta;
                    CEval3 = Cf8j * f7_ij * f7_jk * expval12theta;
                    CEval4 = 2.0 * p_val2 * f7_ij * f7_jk * f8_Dj
                        * expval2theta * (theta_0 - theta);

                    Ctheta_0 = p_val10 * DEG2RAD(theta_00)
                        * EXP( -p_val10 * (2.0 - SBO2) );

                    CEval5 = CEval4 * Ctheta_0 * CSBO2;
                    CEval6 = CEval5 * dSBO1;
                    CEval7 = CEval5 * dSBO2;
                    CEval8 = CEval4 / sin_theta;

                    if ( pk < pi )
                    {
                        e_ang_l += f7_ij * f7_jk * f8_Dj * expval12theta;
                    }

                    /* calculate penalty for double bonds in valency angles */
                    p_pen1 = thbp->p_pen1;

                    exp_pen2ij = EXP( -p_pen2 * SQR( BOA_ij - 2.0 ) );
                    exp_pen2jk = EXP( -p_pen2 * SQR( BOA_jk - 2.0 ) );
                    exp_pen3 = EXP( -p_pen3 * workspace.Delta[j] );
                    exp_pen4 = EXP(  p_pen4 * workspace.Delta[j] );
                    trm_pen34 = 1.0 + exp_pen3 + exp_pen4;
                    f9_Dj = ( 2.0 + exp_pen3 ) / trm_pen34;
                    Cf9j = (-p_pen3 * exp_pen3 * trm_pen34
                            - (2.0 + exp_pen3) * ( -p_pen3 * exp_pen3
                                + p_pen4 * exp_pen4 )) / SQR( trm_pen34 );

                    /* very important: since each kernel generates all interactions,
                     * need to prevent all energies becoming duplicates */
                    if ( pk < pi )
                    {
                        e_pen = p_pen1 * f9_Dj * exp_pen2ij * exp_pen2jk;
                        e_pen_l += e_pen;
                    }

                    CEpen1 = e_pen * Cf9j / f9_Dj;
                    temp = -2.0 * p_pen2 * e_pen;
                    CEpen2 = temp * (BOA_ij - 2.0);
                    CEpen3 = temp * (BOA_jk - 2.0);

                    /* calculate valency angle conjugation energy */
                    p_coa1 = thbp->p_coa1;

                    exp_coa2 = EXP( p_coa2 * workspace.Delta_boc[j] );
                    e_coa = p_coa1
                        * EXP( -p_coa4 * SQR(BOA_ij - 1.5) )
                        * EXP( -p_coa4 * SQR(BOA_jk - 1.5) )
                        * EXP( -p_coa3 * SQR(workspace.total_bond_order[i] - BOA_ij) )
                        * EXP( -p_coa3 * SQR(workspace.total_bond_order[k] - BOA_jk) )
                        / (1.0 + exp_coa2);
                    /* similar to above comment regarding if statement */
                    if ( pk < pi )
                    {
                        e_coa_l += e_coa;
                    }

                    CEcoa1 = -2.0 * p_coa4 * (BOA_ij - 1.5) * e_coa;
                    CEcoa2 = -2.0 * p_coa4 * (BOA_jk - 1.5) * e_coa;
                    CEcoa3 = -p_coa2 * exp_coa2 * e_coa / (1.0 + exp_coa2);
                    CEcoa4 = -2.0 * p_coa3 * (workspace.total_bond_order[i] - BOA_ij) * e_coa;
                    CEcoa5 = -2.0 * p_coa3 * (workspace.total_bond_order[k] - BOA_jk) * e_coa;

                    /* calculate force contributions */
                    /* we must again check for pk < pi for entire forces part */
                    if ( pk < pi )
                    {
#if !defined(CUDA_ACCUM_ATOMIC)
                        bo_ij->Cdbo += CEval1 + CEpen2 + (CEcoa1 - CEcoa4);
                        bo_jk->Cdbo += CEval2 + CEpen3 + (CEcoa2 - CEcoa5);
                        workspace.CdDelta[j] += (CEval3 + CEval7) + CEpen1 + CEcoa3;
                        pbond_ij->va_CdDelta += CEcoa4;
                        pbond_jk->va_CdDelta += CEcoa5;
#else
                        atomicAdd( &bo_ij->Cdbo, CEval1 + CEpen2 + (CEcoa1 - CEcoa4) );
                        atomicAdd( &bo_jk->Cdbo, CEval2 + CEpen3 + (CEcoa2 - CEcoa5) );
                        atomicAdd( &workspace.CdDelta[j], (CEval3 + CEval7) + CEpen1 + CEcoa3 );
                        atomicAdd( &workspace.CdDelta[i], CEcoa4 );
                        atomicAdd( &workspace.CdDelta[k], CEcoa5 );
#endif

                        for ( t = start_j; t < end_j; ++t )
                        {
                            pbond_jt = &bond_list.bond_list[t];
                            bo_jt = &pbond_jt->bo_data;
                            temp_bo_jt = bo_jt->BO;
                            temp = CUBE( temp_bo_jt );
                            pBOjt7 = temp * temp * temp_bo_jt;

#if !defined(CUDA_ACCUM_ATOMIC)
                            bo_jt->Cdbo += (CEval6 * pBOjt7);
                            bo_jt->Cdbopi += CEval5;
                            bo_jt->Cdbopi2 += CEval5;
#else
                            atomicAdd( &bo_jt->Cdbo, CEval6 * pBOjt7 );
                            atomicAdd( &bo_jt->Cdbopi, CEval5 );
                            atomicAdd( &bo_jt->Cdbopi2, CEval5 );
#endif
                        }

#if !defined(CUDA_ACCUM_ATOMIC)
                        /* terms not related to bond order derivatives are
                         * added directly into forces and pressure vector/tensor */
                        rvec_Scale( rvec_temp, CEval8, p_ijk->dcos_di );
                        rvec_Add( pbond_ij->va_f, rvec_temp );
                        rvec_iMultiply( rvec_temp, pbond_ij->rel_box, rvec_temp );
                        rvec_Add( ext_press_l, rvec_temp );

                        rvec_ScaledAdd( f_j_l, CEval8, p_ijk->dcos_dj );

                        rvec_Scale( rvec_temp, CEval8, p_ijk->dcos_dk );
                        rvec_Add( pbond_jk->va_f, rvec_temp );
                        rvec_iMultiply( rvec_temp, pbond_jk->rel_box, rvec_temp );
                        rvec_Add( ext_press_l, rvec_temp );
#else
                        /* terms not related to bond order derivatives are
                         * added directly into forces and pressure vector/tensor */
                        rvec_Scale( rvec_temp, CEval8, p_ijk->dcos_di );
                        atomic_rvecAdd( workspace.f[i], rvec_temp );
                        rvec_iMultiply( rvec_temp, pbond_ij->rel_box, rvec_temp );
                        rvec_Add( ext_press_l, rvec_temp );

                        rvec_ScaledAdd( f_j_l, CEval8, p_ijk->dcos_dj );

                        rvec_Scale( rvec_temp, CEval8, p_ijk->dcos_dk );
                        atomic_rvecAdd( workspace.f[k], rvec_temp );
                        rvec_iMultiply( rvec_temp, pbond_jk->rel_box, rvec_temp );
                        rvec_Add( ext_press_l, rvec_temp );
#endif
                    }
                }
            }
        }

        Set_End_Index( pi, num_thb_intrs, &thb_list );
    }

#if !defined(CUDA_ACCUM_ATOMIC)
    rvec_Add( workspace.f[j], f_j_l );
    e_ang_g[j] = e_ang_l;
    e_coa_g[j] = e_coa_l;
    e_pen_g[j] = e_pen_l;
    rvec_Copy( ext_press_g[j], ext_press_l );
#else
    atomic_rvecAdd( workspace.f[j], f_j_l );
    atomicAdd( (double *) e_ang_g, (double) e_ang_l );
    atomicAdd( (double *) e_coa_g, (double) e_coa_l );
    atomicAdd( (double *) e_pen_g, (double) e_pen_l );
    atomic_rvecAdd( *ext_press_g, ext_press_l );
#endif
}


#if !defined(CUDA_ACCUM_ATOMIC)
CUDA_GLOBAL void k_valence_angles_part2( reax_atom *atoms,
        control_params *control, storage workspace,
        reax_list bond_list, int N )
{
    int i, pj;
    bond_data *pbond;
    bond_data *sym_index_bond;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    for ( pj = Start_Index(i, &bond_list); pj < End_Index(i, &bond_list); ++pj )
    {
        pbond = &bond_list.bond_list[pj];
        sym_index_bond = &bond_list.bond_list[ pbond->sym_index ];

        workspace.CdDelta[i] += sym_index_bond->va_CdDelta;

        //rvec_Add( atoms[i].f, sym_index_bond->va_f );
        rvec_Add( workspace.f[i], sym_index_bond->va_f );
    }
}
#endif


/* Estimate the num. of three-body interactions */
CUDA_GLOBAL void k_estimate_valence_angles( reax_atom *my_atoms,
        control_params *control, reax_list bond_list, int n, int N, int *count )
{
    int j, pi, pk, start_j, end_j, num_thb_intrs;
    real BOA_ij, BOA_jk;
    bond_data *pbond_ij, *pbond_jk;
    bond_order_data *bo_ij, *bo_jk;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= N )
    {
        return;
    }

    start_j = Start_Index( j, &bond_list );
    end_j = End_Index( j, &bond_list );

    for ( pi = start_j; pi < end_j; ++pi )
    {
        num_thb_intrs = 0;
        count[ pi ] = 0;

        pbond_ij = &bond_list.bond_list[pi];
        bo_ij = &pbond_ij->bo_data;
        BOA_ij = bo_ij->BO - control->thb_cut;

        if ( BOA_ij >= 0.0 && (j < n || pbond_ij->nbr < n) )
        {
            for ( pk = start_j; pk < end_j; ++pk )
            {
                if ( pk == pi )
                {
                    continue;
                }

                pbond_jk = &bond_list.bond_list[pk];
                bo_jk = &pbond_jk->bo_data;
                BOA_jk = bo_jk->BO - control->thb_cut;

                if ( BOA_jk < 0.0 )
                {
                    continue;
                }

                ++num_thb_intrs;
            }
        }

        count[ pi ] = num_thb_intrs;
    }
}


static int Cuda_Estimate_Storage_Three_Body( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, reax_list **lists, int *thbody )
{
    int ret;

    ret = SUCCESS;

    cuda_memset( thbody, 0, system->total_bonds * sizeof(int),
            "Cuda_Estimate_Storage_Three_Body::thbody" );

    k_estimate_valence_angles <<< control->blocks_n, control->block_size_n >>>
        ( system->d_my_atoms, (control_params *)control->d_control_params, 
          *(lists[BONDS]), system->n, system->N, thbody );
    cudaCheckError( );

    Cuda_Reduction_Sum( thbody, system->d_total_thbodies, system->total_bonds );

    copy_host_device( &system->total_thbodies, system->d_total_thbodies, sizeof(int),
            hipMemcpyDeviceToHost, "Cuda_Estimate_Storage_Three_Body::d_total_thbodies" );

    if ( data->step - data->prev_steps == 0 )
    {
        system->total_thbodies = MAX( (int) (system->total_thbodies * SAFE_ZONE), MIN_3BODIES );
        system->total_thbodies_indices = system->total_bonds;

        Cuda_Make_List( system->total_thbodies_indices, system->total_thbodies,
                TYP_THREE_BODY, lists[THREE_BODIES] );
    }

    if ( system->total_thbodies > lists[THREE_BODIES]->max_intrs
            || system->total_bonds > lists[THREE_BODIES]->n )
    {
        if ( system->total_thbodies > lists[THREE_BODIES]->max_intrs )
        {
            system->total_thbodies = MAX( (int) (lists[THREE_BODIES]->max_intrs * SAFE_ZONE),
                    system->total_thbodies );
        }
        if ( system->total_bonds > lists[THREE_BODIES]->n )
        {
            system->total_thbodies_indices = MAX( (int) (lists[THREE_BODIES]->n * SAFE_ZONE),
                    system->total_bonds );
        }

        workspace->d_workspace->realloc.thbody = TRUE;
        ret = FAILURE;
    }

    return ret;
}


/* Initialize indices for three body list post reallocation
 *
 * indices: list indices
 * entries: num. of entries in list */
void Cuda_Init_Three_Body_Indices( int *indices, int entries, reax_list **lists )
{
    reax_list *thbody;

    thbody = lists[THREE_BODIES];

    Cuda_Scan_Excl_Sum( indices, thbody->index, entries );
}


int Cuda_Compute_Valence_Angles( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control )
{
    int ret, *thbody;
    size_t s;
#if !defined(CUDA_ACCUM_ATOMIC)
    int update_energy;
    real *spad;
    rvec *rvec_spad;
#endif

#if !defined(CUDA_ACCUM_ATOMIC)
    s = MAX( sizeof(int) * system->total_bonds,
            (sizeof(real) * 3 + sizeof(rvec)) * system->N + sizeof(rvec) * control->blocks_n ),
#else
    s = sizeof(int) * system->total_bonds;
#endif

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
            s, "Cuda_Compute_Valence_Angles::workspace->scratch" );

    thbody = (int *) workspace->scratch;
#if !defined(CUDA_ACCUM_ATOMIC)
    spad = (real *) workspace->scratch;
    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;
#endif

    ret = Cuda_Estimate_Storage_Three_Body( system, control, data, workspace,
            lists, thbody );

    if ( ret == SUCCESS )
    {
        Cuda_Init_Three_Body_Indices( thbody, system->total_thbodies_indices, lists );

#if defined(CUDA_ACCUM_ATOMIC)
        cuda_memset( &((simulation_data *)data->d_simulation_data)->my_en.e_ang,
                0, sizeof(real), "Cuda_Compute_Valence_Angles::e_ang" );
        cuda_memset( &((simulation_data *)data->d_simulation_data)->my_en.e_pen,
                0, sizeof(real), "Cuda_Compute_Valence_Angles::e_pen" );
        cuda_memset( &((simulation_data *)data->d_simulation_data)->my_en.e_coa,
                0, sizeof(real), "Cuda_Compute_Valence_Angles::e_coa" );
        cuda_memset( &((simulation_data *)data->d_simulation_data)->my_ext_press,
                0, sizeof(rvec), "Cuda_Compute_Valence_Angles::my_ext_press" );
#endif

        if ( control->virial == 1 )
        {
            k_valence_angles_virial_part1 <<< control->blocks_n, control->block_size_n >>>
                ( system->d_my_atoms, system->reax_param.d_gp,
                  system->reax_param.d_sbp, system->reax_param.d_thbp, 
                  (control_params *) control->d_control_params,
                  *(workspace->d_workspace), *(lists[BONDS]), *(lists[THREE_BODIES]),
                  system->n, system->N, system->reax_param.num_atom_types, 
#if !defined(CUDA_ACCUM_ATOMIC)
                  spad, &spad[system->N], &spad[2 * system->N], (rvec *) (&spad[3 * system->N])
#else
                  &((simulation_data *)data->d_simulation_data)->my_en.e_ang,
                  &((simulation_data *)data->d_simulation_data)->my_en.e_pen,
                  &((simulation_data *)data->d_simulation_data)->my_en.e_coa,
                  &((simulation_data *)data->d_simulation_data)->my_ext_press
#endif
                );
        }
        else
        {
            k_valence_angles_part1 <<< control->blocks_n, control->block_size_n >>>
                ( system->d_my_atoms, system->reax_param.d_gp,
                  system->reax_param.d_sbp, system->reax_param.d_thbp, 
                  (control_params *) control->d_control_params,
                  *(workspace->d_workspace), *(lists[BONDS]), *(lists[THREE_BODIES]),
                  system->n, system->N, system->reax_param.num_atom_types, 
#if !defined(CUDA_ACCUM_ATOMIC)
                  spad, &spad[system->N], &spad[2 * system->N]
#else
                  &((simulation_data *)data->d_simulation_data)->my_en.e_ang,
                  &((simulation_data *)data->d_simulation_data)->my_en.e_pen,
                  &((simulation_data *)data->d_simulation_data)->my_en.e_coa
#endif
                );
        }
        cudaCheckError( );

#if !defined(CUDA_ACCUM_ATOMIC)
        if ( update_energy == TRUE )
        {
            Cuda_Reduction_Sum( spad,
                    &((simulation_data *)data->d_simulation_data)->my_en.e_ang,
                    system->N );

            Cuda_Reduction_Sum( &spad[system->N],
                    &((simulation_data *)data->d_simulation_data)->my_en.e_pen,
                    system->N );

            Cuda_Reduction_Sum( &spad[2 * system->N],
                    &((simulation_data *)data->d_simulation_data)->my_en.e_coa,
                    system->N );
        }

        if ( control->virial == 1 )
        {
            rvec_spad = (rvec *) (&spad[3 * system->N]);

            k_reduction_rvec <<< control->blocks_n, control->block_size_n,
                             sizeof(rvec) * (control->block_size_n / 32) >>>
                ( rvec_spad, &rvec_spad[system->N], system->N );
            cudaCheckError( );

            k_reduction_rvec <<< 1, control->blocks_pow_2_n,
                             sizeof(rvec) * (control->blocks_pow_2_n / 32) >>>
                ( &rvec_spad[system->N],
                  &((simulation_data *)data->d_simulation_data)->my_ext_press,
                  control->blocks_n );
            cudaCheckError( );
//            Cuda_Reduction_Sum( rvec_spad,
//                    &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                    system->N );
        }
#endif

#if !defined(CUDA_ACCUM_ATOMIC)
        k_valence_angles_part2 <<< control->blocks_n, control->block_size_n >>>
            ( system->d_my_atoms, (control_params *) control->d_control_params,
              *(workspace->d_workspace), *(lists[BONDS]), system->N );
        cudaCheckError( );
#endif
    }

    return ret;
}

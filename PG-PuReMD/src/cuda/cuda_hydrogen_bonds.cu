#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_hydrogen_bonds.h"

#include "cuda_valence_angles.h"
#include "cuda_helpers.h"
#include "cuda_list.h"
#include "cuda_reduction.h"
#include "cuda_utils.h"

#include "../index_utils.h"
#include "../vector.h"

#include "../cub/cub/warp/warp_reduce.cuh"
//#include <cub/warp/warp_reduce.cuh>


/* one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part1( reax_atom *my_atoms, single_body_parameters *sbp, 
        hbond_parameters *d_hbp, global_parameters gp,
        control_params *control, storage workspace,
        reax_list far_nbr_list, reax_list bond_list, reax_list hbond_list, int n, 
        int num_atom_types, real *e_hb_g )
{
    int i, j, k, pi, pk;
    int type_i, type_j, type_k;
    int start_j, end_j, hb_start_j, hb_end_j;
    int *hblist, hblist_size;
    int itr, top;
    int nbr_jk;
    real r_ij, r_jk, theta, cos_theta, sin_xhz4, cos_xhz1, sin_theta2;
    real e_hb, e_hb_l, exp_hb2, exp_hb3, CEhb1, CEhb2, CEhb3;
    rvec dcos_theta_di, dcos_theta_dj, dcos_theta_dk;
    rvec dvec_jk;
    rvec f_j_l, f_k_l;
    hbond_parameters *hbp;
    bond_order_data *bo_ij;
    bond_data *pbond_ij;
    hbond_data *phbond_jk;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= n )
    {
        return;
    }

    /* discover the Hydrogen bonds between i-j-k triplets.
     * here j is H atom and there has to be some bond between i and j.
     * Hydrogen bond is between j and k.
     * so in this function i->X, j->H, k->Z when we map 
     * variables onto the ones in the handout. */

    /* j must be a hydrogen atom */
    if ( sbp[ my_atoms[j].type ].p_hbond == H_ATOM )
    {
        hblist = NULL;
        hblist_size = 0;
        type_j = my_atoms[j].type;
        start_j = Start_Index( j, &bond_list );
        end_j = End_Index( j, &bond_list );
        hb_start_j = Start_Index( my_atoms[j].Hindex, &hbond_list );
        hb_end_j = End_Index( my_atoms[j].Hindex, &hbond_list );
        top = 0;
        e_hb_l = 0.0;
        rvec_MakeZero( f_j_l );

        if ( Num_Entries( j, &bond_list ) > hblist_size )
        {
            hblist_size = Num_Entries( j, &bond_list );
            hblist = (int *) malloc( sizeof(int) * hblist_size );
        }

        /* search bonded atoms i to atom j (hydrogen atom) for potential hydrogen bonding */
        for ( pi = start_j; pi < end_j; ++pi )
        {
            pbond_ij = &bond_list.bond_list[pi];
            i = pbond_ij->nbr;
            bo_ij = &pbond_ij->bo_data;
            type_i = my_atoms[i].type;

            if ( sbp[type_i].p_hbond == H_BONDING_ATOM
                    && bo_ij->BO >= HB_THRESHOLD )
            {
                hblist[top] = pi;
                ++top;
            }
        }

        /* for each hbond of atom j */
        for ( pk = hb_start_j; pk < hb_end_j; ++pk )
        {
            phbond_jk = &hbond_list.hbond_list[pk];
            k = phbond_jk->nbr;
            type_k = my_atoms[k].type;
            nbr_jk = phbond_jk->ptr;
            r_jk = far_nbr_list.far_nbr_list.d[nbr_jk];

            rvec_MakeZero( f_k_l );

            rvec_Scale( dvec_jk, hbond_list.hbond_list[pk].scl,
                    far_nbr_list.far_nbr_list.dvec[nbr_jk] );

            /* find matching hbond to atoms j and k */
            for ( itr = 0; itr < top; ++itr )
            {
                pi = hblist[itr];
                pbond_ij = &bond_list.bond_list[pi];
                i = pbond_ij->nbr;

                if ( my_atoms[i].orig_id != my_atoms[k].orig_id )
                {
                    bo_ij = &pbond_ij->bo_data;
                    type_i = my_atoms[i].type;
                    r_ij = pbond_ij->d;
                    hbp = &d_hbp[ index_hbp(type_i, type_j, type_k, num_atom_types) ];

                    Calculate_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
                            &theta, &cos_theta );

                    /* the derivative of cos(theta) */
                    Calculate_dCos_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
                            &dcos_theta_di, &dcos_theta_dj, &dcos_theta_dk );

                    /* hydrogen bond energy */
                    sin_theta2 = SIN( theta / 2.0 );
                    sin_xhz4 = SQR( sin_theta2 );
                    sin_xhz4 *= sin_xhz4;
                    cos_xhz1 = ( 1.0 - cos_theta );
                    exp_hb2 = EXP( -1.0 * hbp->p_hb2 * bo_ij->BO );
                    exp_hb3 = EXP( -1.0 * hbp->p_hb3 * ( hbp->r0_hb / r_jk
                                + r_jk / hbp->r0_hb - 2.0 ) );

                    e_hb = hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * sin_xhz4;
                    e_hb_l += e_hb;

                    CEhb1 = hbp->p_hb1 * hbp->p_hb2 * exp_hb2 * exp_hb3 * sin_xhz4;
                    CEhb2 = -0.5 * hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * cos_xhz1;
                    CEhb3 = hbp->p_hb3 * e_hb * (hbp->r0_hb / SQR( r_jk )
                            + -1.0 / hbp->r0_hb);

                    /* hydrogen bond forces */
                    /* dbo term */
                    bo_ij->Cdbo += CEhb1;

                    /* dcos terms */
#if !defined(CUDA_ACCUM_ATOMIC)
                    rvec_ScaledAdd( pbond_ij->hb_f, CEhb2, dcos_theta_di ); 
#else
                    atomic_rvecScaledAdd( workspace.f[i], CEhb2, dcos_theta_di );
#endif
                    rvec_ScaledAdd( f_j_l, CEhb2, dcos_theta_dj );
                    rvec_ScaledAdd( f_k_l, CEhb2, dcos_theta_dk );

                    /* dr terms */
                    rvec_ScaledAdd( f_j_l, -1.0 * CEhb3 / r_jk, dvec_jk ); 
                    rvec_ScaledAdd( f_k_l, CEhb3 / r_jk, dvec_jk );
                }
            }

#if !defined(CUDA_ACCUM_ATOMIC)
            rvec_Copy( phbond_jk->hb_f, f_k_l );
#else
            atomic_rvecAdd( workspace.f[k], f_k_l );
#endif
        }

        if ( hblist != NULL )
        {
            free( hblist );
        }

#if !defined(CUDA_ACCUM_ATOMIC)
        /* write conflicts for accumulating partial forces resolved by subsequent kernels */
        rvecCopy( workspace.f[j], f_j_l );
        e_hb_g[j] = e_hb_l;
#else
        atomic_rvecAdd( workspace.f[j], f_j_l );
        atomicAdd( (double *) e_hb_g, (double) e_hb_l );
#endif
    }
}


/* one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part1_opt( reax_atom *my_atoms, single_body_parameters *sbp, 
        hbond_parameters *d_hbp, global_parameters gp,
        control_params *control, storage workspace,
        reax_list far_nbr_list, reax_list bond_list, reax_list hbond_list, int n, 
        int num_atom_types, real *e_hb_g )
{
    extern __shared__ hipcub::WarpReduce<double>::TempStorage temp_d[];
    int i, j, k, pi, pk, thread_id, lane_id, itr;
    int type_i, type_j, type_k;
    int start_j, end_j, hb_start_j, hb_end_j;
    int nbr_jk;
    real r_jk, theta, cos_theta, sin_xhz4, cos_xhz1, sin_theta2;
    real e_hb, e_hb_l, exp_hb2, exp_hb3, CEhb1, CEhb2, CEhb3;
    rvec dcos_theta_di, dcos_theta_dj, dcos_theta_dk;
    rvec dvec_jk;
    rvec f_j_l, f_k_l;
    hbond_parameters *hbp;
    bond_order_data *bo_ij;
    bond_data *pbond_ij;
    hbond_data *phbond_jk;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    j = thread_id / warpSize;

    if ( j >= n )
    {
        return;
    }

    lane_id = thread_id % warpSize; 

    /* discover the Hydrogen bonds between i-j-k triplets.
     * here j is H atom and there has to be some bond between i and j.
     * Hydrogen bond is between j and k.
     * so in this function i->X, j->H, k->Z when we map 
     * variables onto the ones in the handout. */

    /* j must be a hydrogen atom */
    if ( sbp[ my_atoms[j].type ].p_hbond == H_ATOM )
    {
        type_j = my_atoms[j].type;
        start_j = Start_Index( j, &bond_list );
        end_j = End_Index( j, &bond_list );
        hb_start_j = Start_Index( my_atoms[j].Hindex, &hbond_list );
        hb_end_j = End_Index( my_atoms[j].Hindex, &hbond_list );
        e_hb_l = 0.0;
        rvec_MakeZero( f_j_l );

        /* for each hbond of atom j */
        for ( pk = hb_start_j; pk < hb_end_j; ++pk )
        {
            phbond_jk = &hbond_list.hbond_list[pk];
            k = phbond_jk->nbr;
            type_k = my_atoms[k].type;
            nbr_jk = phbond_jk->ptr;
            r_jk = far_nbr_list.far_nbr_list.d[nbr_jk];

            rvec_MakeZero( f_k_l );

            rvec_Scale( dvec_jk, hbond_list.hbond_list[pk].scl,
                    far_nbr_list.far_nbr_list.dvec[nbr_jk] );

            /* search bonded atoms i to atom j (hydrogen atom) for potential hydrogen bonding */
            for ( itr = 0, pi = start_j + lane_id; itr < (end_j - start_j + warpSize - 1) / warpSize; ++itr )
            {
                if ( pi < end_j )
                {
                    pbond_ij = &bond_list.bond_list[pi];
                    i = pbond_ij->nbr;
                    bo_ij = &pbond_ij->bo_data;
                    type_i = my_atoms[i].type;

                    if ( sbp[type_i].p_hbond == H_BONDING_ATOM
                            && bo_ij->BO >= HB_THRESHOLD
                            && my_atoms[i].orig_id != my_atoms[k].orig_id )
                    {
                        hbp = &d_hbp[ index_hbp(type_i, type_j, type_k, num_atom_types) ];

                        Calculate_Theta( pbond_ij->dvec, pbond_ij->d, dvec_jk, r_jk,
                                &theta, &cos_theta );

                        /* the derivative of cos(theta) */
                        Calculate_dCos_Theta( pbond_ij->dvec, pbond_ij->d, dvec_jk, r_jk,
                                &dcos_theta_di, &dcos_theta_dj, &dcos_theta_dk );

                        /* hydrogen bond energy */
                        sin_theta2 = SIN( theta / 2.0 );
                        sin_xhz4 = SQR( sin_theta2 );
                        sin_xhz4 *= sin_xhz4;
                        cos_xhz1 = ( 1.0 - cos_theta );
                        exp_hb2 = EXP( -1.0 * hbp->p_hb2 * bo_ij->BO );
                        exp_hb3 = EXP( -1.0 * hbp->p_hb3 * ( hbp->r0_hb / r_jk
                                    + r_jk / hbp->r0_hb - 2.0 ) );

                        e_hb = hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * sin_xhz4;
                        e_hb_l += e_hb;

                        CEhb1 = hbp->p_hb1 * hbp->p_hb2 * exp_hb2 * exp_hb3 * sin_xhz4;
                        CEhb2 = -0.5 * hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * cos_xhz1;
                        CEhb3 = hbp->p_hb3 * e_hb * (hbp->r0_hb / SQR( r_jk )
                                + -1.0 / hbp->r0_hb);

                        /* hydrogen bond forces */
                        /* dbo term */
                        bo_ij->Cdbo += CEhb1;

                        /* dcos terms */
#if !defined(CUDA_ACCUM_ATOMIC)
                        rvec_ScaledAdd( pbond_ij->hb_f, CEhb2, dcos_theta_di ); 
#else
                        atomic_rvecScaledAdd( workspace.f[i], CEhb2, dcos_theta_di );
#endif
                        rvec_ScaledAdd( f_j_l, CEhb2, dcos_theta_dj );
                        rvec_ScaledAdd( f_k_l, CEhb2, dcos_theta_dk );

                        /* dr terms */
                        rvec_ScaledAdd( f_j_l, -1.0 * CEhb3 / r_jk, dvec_jk ); 
                        rvec_ScaledAdd( f_k_l, CEhb3 / r_jk, dvec_jk );
                    }
                }

                pi += warpSize;
            }

            f_k_l[0] = hipcub::WarpReduce<double>(temp_d[j % (blockDim.x / warpSize)]).Sum(f_k_l[0]);
            f_k_l[1] = hipcub::WarpReduce<double>(temp_d[j % (blockDim.x / warpSize)]).Sum(f_k_l[1]);
            f_k_l[2] = hipcub::WarpReduce<double>(temp_d[j % (blockDim.x / warpSize)]).Sum(f_k_l[2]);

            if ( lane_id == 0 )
            {
#if !defined(CUDA_ACCUM_ATOMIC)
                rvec_Copy( phbond_jk->hb_f, f_k_l );
#else
                atomic_rvecAdd( workspace.f[k], f_k_l );
#endif
            }
        }

        f_j_l[0] = hipcub::WarpReduce<double>(temp_d[j % (blockDim.x / warpSize)]).Sum(f_j_l[0]);
        f_j_l[1] = hipcub::WarpReduce<double>(temp_d[j % (blockDim.x / warpSize)]).Sum(f_j_l[1]);
        f_j_l[2] = hipcub::WarpReduce<double>(temp_d[j % (blockDim.x / warpSize)]).Sum(f_j_l[2]);
        e_hb_l = hipcub::WarpReduce<double>(temp_d[j % (blockDim.x / warpSize)]).Sum(e_hb_l);

        if ( lane_id == 0 )
        {
#if !defined(CUDA_ACCUM_ATOMIC)
            /* write conflicts for accumulating partial forces resolved by subsequent kernels */
            rvecCopy( workspace.f[j], f_j_l );
            e_hb_g[j] = e_hb_l;
#else
            atomic_rvecAdd( workspace.f[j], f_j_l );
            atomicAdd( (double *) e_hb_g, (double) e_hb_l );
#endif
        }
    }
}


/* one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_virial_part1( reax_atom *my_atoms, single_body_parameters *sbp, 
        hbond_parameters *d_hbp, global_parameters gp,
        control_params *control, storage workspace,
        reax_list far_nbr_list, reax_list bond_list, reax_list hbond_list, int n, 
        int num_atom_types, real *e_hb_g, rvec *ext_press_g )
{
    int i, j, k, pi, pk;
    int type_i, type_j, type_k;
    int start_j, end_j, hb_start_j, hb_end_j;
    int *hblist, hblist_size;
    int itr, top;
    int nbr_jk;
    ivec rel_jk;
    real r_ij, r_jk, theta, cos_theta, sin_xhz4, cos_xhz1, sin_theta2;
    real e_hb, e_hb_l, exp_hb2, exp_hb3, CEhb1, CEhb2, CEhb3;
    rvec dcos_theta_di, dcos_theta_dj, dcos_theta_dk;
    rvec dvec_jk, temp, ext_press_l;
#if defined(CUDA_ACCUM_ATOMIC)
    rvec f_j_l;
#endif
    hbond_parameters *hbp;
    bond_order_data *bo_ij;
    bond_data *pbond_ij;
    hbond_data *phbond_jk;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= n )
    {
        return;
    }

    e_hb_l = 0.0;
    rvec_MakeZero( ext_press_l );
#if defined(CUDA_ACCUM_ATOMIC)
    rvec_MakeZero( f_j_l );
#endif

    /* discover the Hydrogen bonds between i-j-k triplets.
     * here j is H atom and there has to be some bond between i and j.
     * Hydrogen bond is between j and k.
     * so in this function i->X, j->H, k->Z when we map 
     * variables onto the ones in the handout. */

    /* j must be a hydrogen atom */
    if ( sbp[ my_atoms[j].type ].p_hbond == H_ATOM )
    {
        hblist = NULL;
        hblist_size = 0;
        type_j = my_atoms[j].type;
        start_j = Start_Index( j, &bond_list );
        end_j = End_Index( j, &bond_list );
        hb_start_j = Start_Index( my_atoms[j].Hindex, &hbond_list );
        hb_end_j = End_Index( my_atoms[j].Hindex, &hbond_list );
        top = 0;

        if ( Num_Entries( j, &bond_list ) > hblist_size )
        {
            hblist_size = Num_Entries( j, &bond_list );
            hblist = (int *) malloc( sizeof(int) * hblist_size );
        }

        /* search bonded atoms i to atom j (hydrogen atom) for potential hydrogen bonding */
        for ( pi = start_j; pi < end_j; ++pi )
        {
            pbond_ij = &bond_list.bond_list[pi];
            i = pbond_ij->nbr;
            bo_ij = &pbond_ij->bo_data;
            type_i = my_atoms[i].type;

            if ( sbp[type_i].p_hbond == H_BONDING_ATOM
                    && bo_ij->BO >= HB_THRESHOLD )
            {
                hblist[top] = pi;
                ++top;
            }
        }

        /* for each hbond of atom j */
        for ( pk = hb_start_j; pk < hb_end_j; ++pk )
        {
            phbond_jk = &hbond_list.hbond_list[pk];
            k = phbond_jk->nbr;
            type_k = my_atoms[k].type;
            nbr_jk = phbond_jk->ptr;
            r_jk = far_nbr_list.far_nbr_list.d[nbr_jk];

            rvec_Scale( dvec_jk, hbond_list.hbond_list[pk].scl,
                    far_nbr_list.far_nbr_list.dvec[nbr_jk] );

#if !defined(CUDA_ACCUM_ATOMIC)
            rvec_MakeZero( phbond_jk->hb_f );
#endif

            /* find matching hbond to atoms j and k */
            for ( itr = 0; itr < top; ++itr )
            {
                pi = hblist[itr];
                pbond_ij = &bond_list.bond_list[pi];
                i = pbond_ij->nbr;

                if ( my_atoms[i].orig_id != my_atoms[k].orig_id )
                {
                    bo_ij = &pbond_ij->bo_data;
                    type_i = my_atoms[i].type;
                    r_ij = pbond_ij->d;
                    hbp = &d_hbp[ index_hbp(type_i, type_j, type_k, num_atom_types) ];

                    Calculate_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
                            &theta, &cos_theta );

                    /* the derivative of cos(theta) */
                    Calculate_dCos_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
                            &dcos_theta_di, &dcos_theta_dj, &dcos_theta_dk );

                    /* hydrogen bond energy */
                    sin_theta2 = SIN( theta / 2.0 );
                    sin_xhz4 = SQR( sin_theta2 );
                    sin_xhz4 *= sin_xhz4;
                    cos_xhz1 = ( 1.0 - cos_theta );
                    exp_hb2 = EXP( -1.0 * hbp->p_hb2 * bo_ij->BO );
                    exp_hb3 = EXP( -1.0 * hbp->p_hb3 * ( hbp->r0_hb / r_jk
                                + r_jk / hbp->r0_hb - 2.0 ) );

                    e_hb = hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * sin_xhz4;
                    e_hb_l += e_hb;

                    CEhb1 = hbp->p_hb1 * hbp->p_hb2 * exp_hb2 * exp_hb3 * sin_xhz4;
                    CEhb2 = -0.5 * hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * cos_xhz1;
                    CEhb3 = hbp->p_hb3 * e_hb * (hbp->r0_hb / SQR( r_jk )
                            + -1.0 / hbp->r0_hb);

                    /* hydrogen bond forces */
                    /* dbo term */
                    bo_ij->Cdbo += CEhb1;

#if !defined(CUDA_ACCUM_ATOMIC)
                    /* for pressure coupling, terms that are not related to bond order
                     * derivatives are added directly into pressure vector/tensor */
                    /* dcos terms */
                    rvec_Scale( temp, CEhb2, dcos_theta_di );
                    rvec_Add( pbond_ij->hb_f, temp );
                    rvec_iMultiply( temp, pbond_ij->rel_box, temp );
                    rvec_Add( ext_press_l, temp );

                    rvec_ScaledAdd( workspace.f[j], CEhb2, dcos_theta_dj );

                    ivec_Scale( rel_jk, hbond_list.hbond_list[pk].scl,
                            far_nbr_list.far_nbr_list.rel_box[nbr_jk] );
                    rvec_Scale( temp, CEhb2, dcos_theta_dk );
                    rvec_Add( phbond_jk->hb_f, temp );
                    rvec_iMultiply( temp, rel_jk, temp );
                    rvec_Add( ext_press_l, temp );

                    /* dr terms */
                    rvec_ScaledAdd( workspace.f[j], -1.0 * CEhb3 / r_jk, dvec_jk ); 

                    rvec_Scale( temp, CEhb3 / r_jk, dvec_jk );
                    rvec_Add( phbond_jk->hb_f, temp );
                    rvec_iMultiply( temp, rel_jk, temp );
                    rvec_Add( ext_press_l, temp );
#else
                    /* for pressure coupling, terms that are not related to bond order
                     * derivatives are added directly into pressure vector/tensor */
                    /* dcos terms */
                    rvec_Scale( temp, CEhb2, dcos_theta_di );
                    atomic_rvecAdd( workspace.f[i], temp );
                    rvec_iMultiply( temp, pbond_ij->rel_box, temp );
                    rvec_Add( ext_press_l, temp );

                    rvec_ScaledAdd( f_j_l, CEhb2, dcos_theta_dj );

                    ivec_Scale( rel_jk, hbond_list.hbond_list[pk].scl,
                            far_nbr_list.far_nbr_list.rel_box[nbr_jk] );
                    rvec_Scale( temp, CEhb2, dcos_theta_dk );
                    atomic_rvecAdd( workspace.f[k], temp );
                    rvec_iMultiply( temp, rel_jk, temp );
                    rvec_Add( ext_press_l, temp );

                    /* dr terms */
                    rvec_ScaledAdd( f_j_l, -1.0 * CEhb3 / r_jk, dvec_jk ); 

                    rvec_Scale( temp, CEhb3 / r_jk, dvec_jk );
                    atomic_rvecAdd( workspace.f[k], temp );
                    rvec_iMultiply( temp, rel_jk, temp );
                    rvec_Add( ext_press_l, temp );
#endif
                }
            }
        }

        if ( hblist != NULL )
        {
            free( hblist );
        }
    }

#if !defined(CUDA_ACCUM_ATOMIC)
    /* write conflicts for accumulating partial forces resolved by subsequent kernels */
    rvecCopy( workspace.f[j], f_j_l );
    e_hb_g[j] = e_hb_l;
    rvecCopy( ext_press_g[j], ext_press_l );
#else
    atomic_rvecAdd( workspace.f[j], f_j_l );
    atomicAdd( (double *) e_hb_g, (double) e_hb_l );
    atomic_rvecAdd( *ext_press_g, ext_press_l );
#endif
}


#if !defined(CUDA_ACCUM_ATOMIC)
/* Accumulate forces stored in the bond list
 * using a one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part2( reax_atom *atoms,
        storage workspace, reax_list bond_list, int n )
{
    int j, pj;
    bond_data *pbond, *sym_index_bond;
    rvec hb_f;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= n )
    {
        return;
    }

    rvec_MakeZero( hb_f );

    for ( pj = Start_Index(j, &bond_list); pj < End_Index(j, &bond_list); ++pj )
    {
        pbond = &bond_list.bond_list[pj];
        sym_index_bond = &bond_list.bond_list[pbond->sym_index];

        rvec_Add( hb_f, sym_index_bond->hb_f );
    }

    rvec_Add( workspace.f[j], hb_f );
}


/* Accumulate forces stored in the bond list
 * using a one warp threads per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part2_opt( reax_atom *atoms,
        storage workspace, reax_list bond_list, int n )
{
    typedef hipcub::WarpReduce<double> WarpReduce;
    extern __shared__ typename WarpReduce::TempStorage temp_storage[];
    int j, pj, start, end;
    bond_data *pbond, *sym_index_bond;
    /* thread-local variables */
    int thread_id, warp_id, lane_id;
    rvec hb_f;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id >> 5;
    lane_id = thread_id & 0x0000001F; 

    if ( warp_id >= n )
    {
        return;
    }

    j = warp_id;
    start = Start_Index( j, &bond_list );
    end = End_Index( j, &bond_list );
    pj = start + lane_id;
    rvec_MakeZero( hb_f );

    while ( pj < end )
    {
        pbond = &bond_list.bond_list[pj];
        sym_index_bond = &bond_list.bond_list[pbond->sym_index];

        rvec_Add( hb_f, sym_index_bond->hb_f );

        pj += warpSize;
    }

    __syncthreads( );

    hb_f[0] = WarpReduce(temp_storage[warp_id]).Sum(hb_f[0]);
    hb_f[1] = WarpReduce(temp_storage[warp_id]).Sum(hb_f[1]);
    hb_f[2] = WarpReduce(temp_storage[warp_id]).Sum(hb_f[2]);

    /* first thread within a warp writes warp-level sums to global memory */
    if ( lane_id == 0 )
    {
        rvec_Add( workspace.f[j], hb_f );
    }
}


/* Accumulate forces stored in the hbond list
 * using a one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part3( reax_atom *atoms,
        storage workspace, reax_list hbond_list, int n )
{
    int j, pj;
    hbond_data *nbr_pj, *sym_index_nbr;
    rvec hb_f;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= n )
    {
        return;
    }

    rvec_MakeZero( hb_f );

    for ( pj = Start_Index(atoms[j].Hindex, &hbond_list); pj < End_Index(atoms[j].Hindex, &hbond_list); ++pj )
    {
        nbr_pj = &hbond_list.hbond_list[pj];
        sym_index_nbr = &hbond_list.hbond_list[ nbr_pj->sym_index ];

        rvec_Add( hb_f, sym_index_nbr->hb_f );
    }

    rvec_Add( workspace.f[j], hb_f );
}


/* Accumulate forces stored in the hbond list
 * using a one warp of threads per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part3_opt( reax_atom *atoms,
        storage workspace, reax_list hbond_list, int n )
{
    typedef hipcub::WarpReduce<double> WarpReduce;
    extern __shared__ typename WarpReduce::TempStorage temp_storage[];
    int j, pj, start, end;
    hbond_data *nbr_pj, *sym_index_nbr;
    /* thread-local variables */
    int thread_id, warp_id, lane_id, offset;
    rvec hb_f_l;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id >> 5;
    lane_id = thread_id & 0x0000001F; 

    if ( warp_id >= n )
    {
        return;
    }

    j = warp_id;
    start = Start_Index( atoms[j].Hindex, &hbond_list );
    end = End_Index( atoms[j].Hindex, &hbond_list );
    pj = start + lane_id;
    rvec_MakeZero( hb_f_l );

    while ( pj < end )
    {
        nbr_pj = &hbond_list.hbond_list[pj];
        sym_index_nbr = &hbond_list.hbond_list[ nbr_pj->sym_index ];

        rvec_Add( hb_f_l, sym_index_nbr->hb_f );

        pj += warpSize;
    }

    __syncthreads( );

    hb_f_l[0] = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[0]);
    hb_f_l[1] = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[1]);
    hb_f_l[2] = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[2]);

    /* first thread within a warp writes warp-level sums to global memory */
    if ( lane_id == 0 )
    {
        rvec_Add( workspace.f[j], hb_f_l );
    }
}
#endif



void Cuda_Compute_Hydrogen_Bonds( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control )
{
    int blocks;
//    int hbs, hnbrs_blocks;
#if !defined(CUDA_ACCUM_ATOMIC)
    int update_energy;
    real *spad;
    rvec *rvec_spad;

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
                (sizeof(real) * 3 + sizeof(rvec)) * system->N + sizeof(rvec) * control->blocks_n,
            "Cuda_Compute_Hydrogen_Bonds::workspace->scratch" );
    spad = (real *) workspace->scratch;
    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;
#else
    cuda_memset( &((simulation_data *)data->d_simulation_data)->my_en.e_hb,
            0, sizeof(real), "Cuda_Compute_Hydrogen_Bonds::e_hb" );
    if ( control->virial == 1 )
    {
        cuda_memset( &((simulation_data *)data->d_simulation_data)->my_ext_press,
                0, sizeof(rvec), "Cuda_Compute_Hydrogen_Bonds::my_ext_press" );
    }
#endif

    if ( control->virial == 1 )
    {
        k_hydrogen_bonds_virial_part1 <<< control->blocks, control->block_size,
                                      0, control->streams[0] >>>
                ( system->d_my_atoms, system->reax_param.d_sbp,
                  system->reax_param.d_hbp, system->reax_param.d_gp,
                  (control_params *) control->d_control_params,
                  *(workspace->d_workspace),
                  *(lists[FAR_NBRS]), *(lists[BONDS]), *(lists[HBONDS]),
                  system->n, system->reax_param.num_atom_types,
#if !defined(CUDA_ACCUM_ATOMIC)
                  spad, (rvec *) (&spad[system->n])
#else
                  &((simulation_data *)data->d_simulation_data)->my_en.e_hb,
                  &((simulation_data *)data->d_simulation_data)->my_ext_press
#endif
                );
        cudaCheckError( );
    }
    else
    {
//        k_hydrogen_bonds_part1 <<< control->blocks, control->block_size, 0, control->streams[0] >>>
//                ( system->d_my_atoms, system->reax_param.d_sbp,
//                  system->reax_param.d_hbp, system->reax_param.d_gp,
//                  (control_params *) control->d_control_params,
//                  *(workspace->d_workspace),
//                  *(lists[FAR_NBRS]), *(lists[BONDS]), *(lists[HBONDS]),
//                  system->n, system->reax_param.num_atom_types,
//#if !defined(CUDA_ACCUM_ATOMIC)
//                  spad
//#else
//                  &((simulation_data *)data->d_simulation_data)->my_en.e_hb
//#endif
//                );
//        cudaCheckError( );

        blocks = system->n * 32 / DEF_BLOCK_SIZE
            + (system->n * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);
        
        k_hydrogen_bonds_part1_opt <<< blocks, DEF_BLOCK_SIZE,
                                   sizeof(hipcub::WarpReduce<double>::TempStorage) * (DEF_BLOCK_SIZE / 32),
                                   control->streams[0] >>>
                ( system->d_my_atoms, system->reax_param.d_sbp,
                  system->reax_param.d_hbp, system->reax_param.d_gp,
                  (control_params *) control->d_control_params,
                  *(workspace->d_workspace),
                  *(lists[FAR_NBRS]), *(lists[BONDS]), *(lists[HBONDS]),
                  system->n, system->reax_param.num_atom_types,
#if !defined(CUDA_ACCUM_ATOMIC)
                  spad
#else
                  &((simulation_data *)data->d_simulation_data)->my_en.e_hb
#endif
                );
        cudaCheckError( );
    }

#if !defined(CUDA_ACCUM_ATOMIC)
    if ( update_energy == TRUE )
    {
        Cuda_Reduction_Sum( spad,
                &((simulation_data *)data->d_simulation_data)->my_en.e_hb,
                system->n );
    }

    if ( control->virial == 1 )
    {
        rvec_spad = (rvec *) (&spad[system->n]);

        k_reduction_rvec <<< control->blocks, control->block_size,
                         sizeof(rvec) * (control->block_size / 32),
                         control->streams[0] >>>
            ( rvec_spad, &rvec_spad[system->n], system->n );
        cudaCheckError( );

        k_reduction_rvec <<< 1, control->blocks_pow_2,
                         sizeof(rvec) * (control->blocks_pow_2 / 32),
                         control->streams[0] >>>
            ( &rvec_spad[system->n],
              &((simulation_data *)data->d_simulation_data)->my_ext_press,
              control->blocks );
        cudaCheckError( );
//        Cuda_Reduction_Sum( rvec_spad,
//                &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                system->n );
    }
#endif

#if !defined(CUDA_ACCUM_ATOMIC)
    k_hydrogen_bonds_part2 <<< control->blocks, control->block_size, 0,
                           control->streams[0] >>>
        ( system->d_my_atoms, *(workspace->d_workspace),
          *(lists[BONDS]), system->n );
    cudaCheckError( );

//    hnbrs_blocks = (system->n * HB_POST_PROC_KER_THREADS_PER_ATOM / HB_POST_PROC_BLOCK_SIZE) +
//        (((system->n * HB_POST_PROC_KER_THREADS_PER_ATOM) % HB_POST_PROC_BLOCK_SIZE) == 0 ? 0 : 1);

    k_hydrogen_bonds_part3 <<< control->blocks, control->block_size, 0,
                           control->streams[0] >>>
        ( system->d_my_atoms, *(workspace->d_workspace), *(lists[HBONDS]), system->n );
//    k_hydrogen_bonds_part3_opt <<< hnbrs_blocks, HB_POST_PROC_BLOCK_SIZE, 
//            sizeof(rvec) * HB_POST_PROC_BLOCK_SIZE, control->streams[0] >>>
//        ( system->d_my_atoms, *(workspace->d_workspace), *(lists[HBONDS]), system->n );
    cudaCheckError( );
#endif
}

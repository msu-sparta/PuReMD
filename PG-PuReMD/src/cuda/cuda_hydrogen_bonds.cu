#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_hydrogen_bonds.h"

#include "cuda_valence_angles.h"
#include "cuda_helpers.h"
#include "cuda_list.h"
#include "cuda_reduction.h"
#include "cuda_utils.h"

#include "../index_utils.h"
#include "../vector.h"

#include "../cub/cub/warp/warp_reduce.cuh"
//#include <cub/warp/warp_reduce.cuh>


/* one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part1( reax_atom *my_atoms, single_body_parameters *sbp, 
        hbond_parameters *d_hbp, global_parameters gp,
        control_params *control, storage workspace,
        reax_list far_nbr_list, reax_list bond_list, reax_list hbond_list, int n, 
        int num_atom_types, real *e_hb_g )
{
    int i, j, k, pi, pk;
    int type_i, type_j, type_k;
    int start_j, end_j, hb_start_j, hb_end_j;
    int *hblist, hblist_size;
    int itr, top;
    int nbr_jk;
    real r_ij, r_jk, theta, cos_theta, sin_xhz4, cos_xhz1, sin_theta2;
    real e_hb, e_hb_l, exp_hb2, exp_hb3, CEhb1, CEhb2, CEhb3;
    rvec dcos_theta_di, dcos_theta_dj, dcos_theta_dk;
    rvec dvec_jk;
#if defined(CUDA_ACCUM_ATOMIC)
    rvec f_j_l;
#endif
    hbond_parameters *hbp;
    bond_order_data *bo_ij;
    bond_data *pbond_ij;
    hbond_data *phbond_jk;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= n )
    {
        return;
    }

    e_hb_l = 0.0;
#if defined(CUDA_ACCUM_ATOMIC)
    rvec_MakeZero( f_j_l );
#endif

    /* discover the Hydrogen bonds between i-j-k triplets.
     * here j is H atom and there has to be some bond between i and j.
     * Hydrogen bond is between j and k.
     * so in this function i->X, j->H, k->Z when we map 
     * variables onto the ones in the handout. */

    /* j must be a hydrogen atom */
    if ( sbp[ my_atoms[j].type ].p_hbond == H_ATOM )
    {
        hblist = NULL;
        hblist_size = 0;
        type_j = my_atoms[j].type;
        start_j = Start_Index( j, &bond_list );
        end_j = End_Index( j, &bond_list );
        hb_start_j = Start_Index( my_atoms[j].Hindex, &hbond_list );
        hb_end_j = End_Index( my_atoms[j].Hindex, &hbond_list );
        top = 0;

        if ( Num_Entries( j, &bond_list ) > hblist_size )
        {
            hblist_size = Num_Entries( j, &bond_list );
            hblist = (int *) malloc( sizeof(int) * hblist_size );
        }

        /* search bonded atoms i to atom j (hydrogen atom) for potential hydrogen bonding */
        for ( pi = start_j; pi < end_j; ++pi )
        {
            pbond_ij = &bond_list.bond_list[pi];
            i = pbond_ij->nbr;
            bo_ij = &pbond_ij->bo_data;
            type_i = my_atoms[i].type;

            if ( sbp[type_i].p_hbond == H_BONDING_ATOM
                    && bo_ij->BO >= HB_THRESHOLD )
            {
                hblist[top] = pi;
                ++top;
            }
        }

        /* for each hbond of atom j */
        for ( pk = hb_start_j; pk < hb_end_j; ++pk )
        {
            phbond_jk = &hbond_list.hbond_list[pk];
            k = phbond_jk->nbr;
            type_k = my_atoms[k].type;
            nbr_jk = phbond_jk->ptr;
            r_jk = far_nbr_list.far_nbr_list.d[nbr_jk];

            rvec_Scale( dvec_jk, hbond_list.hbond_list[pk].scl,
                    far_nbr_list.far_nbr_list.dvec[nbr_jk] );

#if !defined(CUDA_ACCUM_ATOMIC)
            rvec_MakeZero( phbond_jk->hb_f );
#endif

            /* find matching hbond to atoms j and k */
            for ( itr = 0; itr < top; ++itr )
            {
                pi = hblist[itr];
                pbond_ij = &bond_list.bond_list[pi];
                i = pbond_ij->nbr;

                if ( my_atoms[i].orig_id != my_atoms[k].orig_id )
                {
                    bo_ij = &pbond_ij->bo_data;
                    type_i = my_atoms[i].type;
                    r_ij = pbond_ij->d;
                    hbp = &d_hbp[ index_hbp(type_i, type_j, type_k, num_atom_types) ];

                    Calculate_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
                            &theta, &cos_theta );

                    /* the derivative of cos(theta) */
                    Calculate_dCos_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
                            &dcos_theta_di, &dcos_theta_dj, &dcos_theta_dk );

                    /* hydrogen bond energy */
                    sin_theta2 = SIN( theta / 2.0 );
                    sin_xhz4 = SQR( sin_theta2 );
                    sin_xhz4 *= sin_xhz4;
                    cos_xhz1 = ( 1.0 - cos_theta );
                    exp_hb2 = EXP( -1.0 * hbp->p_hb2 * bo_ij->BO );
                    exp_hb3 = EXP( -1.0 * hbp->p_hb3 * ( hbp->r0_hb / r_jk
                                + r_jk / hbp->r0_hb - 2.0 ) );

                    e_hb = hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * sin_xhz4;
                    e_hb_l += e_hb;

                    CEhb1 = hbp->p_hb1 * hbp->p_hb2 * exp_hb2 * exp_hb3 * sin_xhz4;
                    CEhb2 = -0.5 * hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * cos_xhz1;
                    CEhb3 = hbp->p_hb3 * e_hb * (hbp->r0_hb / SQR( r_jk )
                            + -1.0 / hbp->r0_hb);

                    /* hydrogen bond forces */
                    /* dbo term */
                    bo_ij->Cdbo += CEhb1;

#if !defined(CUDA_ACCUM_ATOMIC)
                    /* dcos terms */
                    rvec_ScaledAdd( pbond_ij->hb_f, CEhb2, dcos_theta_di ); 
                    rvec_ScaledAdd( workspace.f[j], CEhb2, dcos_theta_dj );
                    rvec_ScaledAdd( phbond_jk->hb_f, CEhb2, dcos_theta_dk );

                    /* dr terms */
                    rvec_ScaledAdd( workspace.f[j], -1.0 * CEhb3 / r_jk, dvec_jk ); 
                    rvec_ScaledAdd( phbond_jk->hb_f, CEhb3 / r_jk, dvec_jk );
#else
                    /* dcos terms */
                    atomic_rvecScaledAdd( workspace.f[i], CEhb2, dcos_theta_di );
                    rvec_ScaledAdd( f_j_l, CEhb2, dcos_theta_dj );
                    atomic_rvecScaledAdd( workspace.f[k], CEhb2, dcos_theta_dk );

                    /* dr terms */
                    rvec_ScaledAdd( f_j_l, -1.0 * CEhb3 / r_jk, dvec_jk );
                    atomic_rvecScaledAdd( workspace.f[k], CEhb3 / r_jk, dvec_jk );
#endif
                }
            }
        }

        if ( hblist != NULL )
        {
            free( hblist );
        }
    }

#if !defined(CUDA_ACCUM_ATOMIC)
    /* write conflicts for accumulating partial forces resolved by subsequent kernels */
    rvecCopy( workspace.f[j], f_j_l );
    e_hb_g[j] = e_hb_l;
#else
    atomic_rvecAdd( workspace.f[j], f_j_l );
    atomicAdd( (double *) e_hb_g, (double) e_hb_l );
#endif
}


/* one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_virial_part1( reax_atom *my_atoms, single_body_parameters *sbp, 
        hbond_parameters *d_hbp, global_parameters gp,
        control_params *control, storage workspace,
        reax_list far_nbr_list, reax_list bond_list, reax_list hbond_list, int n, 
        int num_atom_types, real *e_hb_g, rvec *ext_press_g )
{
    int i, j, k, pi, pk;
    int type_i, type_j, type_k;
    int start_j, end_j, hb_start_j, hb_end_j;
    int *hblist, hblist_size;
    int itr, top;
    int nbr_jk;
    ivec rel_jk;
    real r_ij, r_jk, theta, cos_theta, sin_xhz4, cos_xhz1, sin_theta2;
    real e_hb, e_hb_l, exp_hb2, exp_hb3, CEhb1, CEhb2, CEhb3;
    rvec dcos_theta_di, dcos_theta_dj, dcos_theta_dk;
    rvec dvec_jk, temp, ext_press_l;
#if defined(CUDA_ACCUM_ATOMIC)
    rvec f_j_l;
#endif
    hbond_parameters *hbp;
    bond_order_data *bo_ij;
    bond_data *pbond_ij;
    hbond_data *phbond_jk;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= n )
    {
        return;
    }

    e_hb_l = 0.0;
    rvec_MakeZero( ext_press_l );
#if defined(CUDA_ACCUM_ATOMIC)
    rvec_MakeZero( f_j_l );
#endif

    /* discover the Hydrogen bonds between i-j-k triplets.
     * here j is H atom and there has to be some bond between i and j.
     * Hydrogen bond is between j and k.
     * so in this function i->X, j->H, k->Z when we map 
     * variables onto the ones in the handout. */

    /* j must be a hydrogen atom */
    if ( sbp[ my_atoms[j].type ].p_hbond == H_ATOM )
    {
        hblist = NULL;
        hblist_size = 0;
        type_j = my_atoms[j].type;
        start_j = Start_Index( j, &bond_list );
        end_j = End_Index( j, &bond_list );
        hb_start_j = Start_Index( my_atoms[j].Hindex, &hbond_list );
        hb_end_j = End_Index( my_atoms[j].Hindex, &hbond_list );
        top = 0;

        if ( Num_Entries( j, &bond_list ) > hblist_size )
        {
            hblist_size = Num_Entries( j, &bond_list );
            hblist = (int *) malloc( sizeof(int) * hblist_size );
        }

        /* search bonded atoms i to atom j (hydrogen atom) for potential hydrogen bonding */
        for ( pi = start_j; pi < end_j; ++pi )
        {
            pbond_ij = &bond_list.bond_list[pi];
            i = pbond_ij->nbr;
            bo_ij = &pbond_ij->bo_data;
            type_i = my_atoms[i].type;

            if ( sbp[type_i].p_hbond == H_BONDING_ATOM
                    && bo_ij->BO >= HB_THRESHOLD )
            {
                hblist[top] = pi;
                ++top;
            }
        }

        /* for each hbond of atom j */
        for ( pk = hb_start_j; pk < hb_end_j; ++pk )
        {
            phbond_jk = &hbond_list.hbond_list[pk];
            k = phbond_jk->nbr;
            type_k = my_atoms[k].type;
            nbr_jk = phbond_jk->ptr;
            r_jk = far_nbr_list.far_nbr_list.d[nbr_jk];

            rvec_Scale( dvec_jk, hbond_list.hbond_list[pk].scl,
                    far_nbr_list.far_nbr_list.dvec[nbr_jk] );

#if !defined(CUDA_ACCUM_ATOMIC)
            rvec_MakeZero( phbond_jk->hb_f );
#endif

            /* find matching hbond to atoms j and k */
            for ( itr = 0; itr < top; ++itr )
            {
                pi = hblist[itr];
                pbond_ij = &bond_list.bond_list[pi];
                i = pbond_ij->nbr;

                if ( my_atoms[i].orig_id != my_atoms[k].orig_id )
                {
                    bo_ij = &pbond_ij->bo_data;
                    type_i = my_atoms[i].type;
                    r_ij = pbond_ij->d;
                    hbp = &d_hbp[ index_hbp(type_i, type_j, type_k, num_atom_types) ];

                    Calculate_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
                            &theta, &cos_theta );

                    /* the derivative of cos(theta) */
                    Calculate_dCos_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
                            &dcos_theta_di, &dcos_theta_dj, &dcos_theta_dk );

                    /* hydrogen bond energy */
                    sin_theta2 = SIN( theta / 2.0 );
                    sin_xhz4 = SQR( sin_theta2 );
                    sin_xhz4 *= sin_xhz4;
                    cos_xhz1 = ( 1.0 - cos_theta );
                    exp_hb2 = EXP( -1.0 * hbp->p_hb2 * bo_ij->BO );
                    exp_hb3 = EXP( -1.0 * hbp->p_hb3 * ( hbp->r0_hb / r_jk
                                + r_jk / hbp->r0_hb - 2.0 ) );

                    e_hb = hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * sin_xhz4;
                    e_hb_l += e_hb;

                    CEhb1 = hbp->p_hb1 * hbp->p_hb2 * exp_hb2 * exp_hb3 * sin_xhz4;
                    CEhb2 = -0.5 * hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * cos_xhz1;
                    CEhb3 = hbp->p_hb3 * e_hb * (hbp->r0_hb / SQR( r_jk )
                            + -1.0 / hbp->r0_hb);

                    /* hydrogen bond forces */
                    /* dbo term */
                    bo_ij->Cdbo += CEhb1;

#if !defined(CUDA_ACCUM_ATOMIC)
                    /* for pressure coupling, terms that are not related to bond order
                     * derivatives are added directly into pressure vector/tensor */
                    /* dcos terms */
                    rvec_Scale( temp, CEhb2, dcos_theta_di );
                    rvec_Add( pbond_ij->hb_f, temp );
                    rvec_iMultiply( temp, pbond_ij->rel_box, temp );
                    rvec_Add( ext_press_l, temp );

                    rvec_ScaledAdd( workspace.f[j], CEhb2, dcos_theta_dj );

                    ivec_Scale( rel_jk, hbond_list.hbond_list[pk].scl,
                            far_nbr_list.far_nbr_list.rel_box[nbr_jk] );
                    rvec_Scale( temp, CEhb2, dcos_theta_dk );
                    rvec_Add( phbond_jk->hb_f, temp );
                    rvec_iMultiply( temp, rel_jk, temp );
                    rvec_Add( ext_press_l, temp );

                    /* dr terms */
                    rvec_ScaledAdd( workspace.f[j], -1.0 * CEhb3 / r_jk, dvec_jk ); 

                    rvec_Scale( temp, CEhb3 / r_jk, dvec_jk );
                    rvec_Add( phbond_jk->hb_f, temp );
                    rvec_iMultiply( temp, rel_jk, temp );
                    rvec_Add( ext_press_l, temp );
#else
                    /* for pressure coupling, terms that are not related to bond order
                     * derivatives are added directly into pressure vector/tensor */
                    /* dcos terms */
                    rvec_Scale( temp, CEhb2, dcos_theta_di );
                    atomic_rvecAdd( workspace.f[i], temp );
                    rvec_iMultiply( temp, pbond_ij->rel_box, temp );
                    rvec_Add( ext_press_l, temp );

                    rvec_ScaledAdd( f_j_l, CEhb2, dcos_theta_dj );

                    ivec_Scale( rel_jk, hbond_list.hbond_list[pk].scl,
                            far_nbr_list.far_nbr_list.rel_box[nbr_jk] );
                    rvec_Scale( temp, CEhb2, dcos_theta_dk );
                    atomic_rvecAdd( workspace.f[k], temp );
                    rvec_iMultiply( temp, rel_jk, temp );
                    rvec_Add( ext_press_l, temp );

                    /* dr terms */
                    rvec_ScaledAdd( f_j_l, -1.0 * CEhb3 / r_jk, dvec_jk ); 

                    rvec_Scale( temp, CEhb3 / r_jk, dvec_jk );
                    atomic_rvecAdd( workspace.f[k], temp );
                    rvec_iMultiply( temp, rel_jk, temp );
                    rvec_Add( ext_press_l, temp );
#endif
                }
            }
        }

        if ( hblist != NULL )
        {
            free( hblist );
        }
    }

#if !defined(CUDA_ACCUM_ATOMIC)
    /* write conflicts for accumulating partial forces resolved by subsequent kernels */
    rvecCopy( workspace.f[j], f_j_l );
    e_hb_g[j] = e_hb_l;
    rvecCopy( ext_press_g[j], ext_press_l );
#else
    atomic_rvecAdd( workspace.f[j], f_j_l );
    atomicAdd( (double *) e_hb_g, (double) e_hb_l );
    atomic_rvecAdd( *ext_press_g, ext_press_l );
#endif
}


/* one warp of threads per atom implementation */
//CUDA_GLOBAL void k_hydrogen_bonds_part1_opt( reax_atom *my_atoms, single_body_parameters *sbp, 
//        hbond_parameters *d_hbp, global_parameters gp, control_params *control, storage workspace,
//        reax_list far_nbr_list, reax_list bond_list, reax_list hbond_list, int n, 
//        int num_atom_types, real *e_hb_g, rvec *ext_press_g )
//{
//    typedef hipcub::WarpReduce<double> WarpReduce;
//    extern __shared__ typename hipcub::WarpReduce::TempStorage temp_storage[];
//    int i, j, k, pi, pk;
//    int type_i, type_j, type_k;
//    int start_j, end_j, hb_start_j, hb_end_j;
//    //TODO: re-write and remove
//    int hblist[30];
//    int itr, top;
//    int loopcount, count;
//    ivec rel_jk;
//    real r_ij, r_jk, theta, cos_theta, sin_xhz4, cos_xhz1, sin_theta2;
//    real e_hb, e_hb_l, exp_hb2, exp_hb3, CEhb1, CEhb1_l, CEhb2, CEhb3;
//    rvec dcos_theta_di, dcos_theta_dj, dcos_theta_dk;
//    rvec dvec_jk, temp, f_j_l, ext_press_l;
//#if !defined(CUDA_ACCUM_ATOMIC)
//    rvec hb_f_l;
//#endif
//    double dtemp;
//    hbond_parameters *hbp;
//    bond_order_data *bo_ij;
//    int nbr_jk;
//    bond_data *pbond_ij;
//    hbond_data *phbond_jk;
//    /* thread-local variables */
//    int thread_id, warp_id, lane_id;
//
//    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
//    warp_id = thread_id >> 5;
//    lane_id = thread_id & 0x0000001F; 
//
//    if ( warp_id >= n )
//    {
//        return;
//    }
//
//    j = warp_id; // group of threads assigned to atom j
//
//    /* discover the Hydrogen bonds between i-j-k triplets.
//     * here j is H atom and there has to be some bond between i and j.
//     * Hydrogen bond is between j and k.
//     * so in this function i->X, j->H, k->Z when we map 
//     * variables onto the ones in the handout.*/
//    e_hb_l = 0.0;
//    rvec_MakeZero( f_j_l );
//    rvec_MakeZero( ext_press_l );
//
//    /* j has to be of type H */
//    if ( sbp[ my_atoms[j].type ].p_hbond == H_ATOM )
//    {
//        type_j = my_atoms[j].type;
//        start_j = Start_Index( j, &bond_list );
//        end_j = End_Index( j, &bond_list );
//        hb_start_j = Start_Index( my_atoms[j].Hindex, &hbond_list );
//        hb_end_j = End_Index( my_atoms[j].Hindex, &hbond_list );
//        top = 0;
//
//        /* search bonded atoms i to atom j (hydrogen atom) for potential hydrogen bonding */
//        for ( pi = start_j; pi < end_j; ++pi ) 
//        {
//            pbond_ij = &bond_list.bond_list[pi];
//            i = pbond_ij->nbr;
//            bo_ij = &pbond_ij->bo_data;
//            type_i = my_atoms[i].type;
//
//            if ( sbp[type_i].p_hbond == H_BONDING_ATOM
//                    && bo_ij->BO >= HB_THRESHOLD )
//            {
//                hblist[top] = pi;
//                ++top;
//            }
//        }
//
//        /* find matching hbond to atoms j and k */
//        for ( itr = 0; itr < top; ++itr )
//        {
//            pi = hblist[itr];
//            pbond_ij = &bond_list.bond_list[pi];
//            i = pbond_ij->nbr;
//#if !defined(CUDA_ACCUM_ATOMIC)
//            rvec_MakeZero( hb_f_l );
//#endif
//            CEhb1_l = 0.0;
//
//            //for( pk = hb_start_j; pk < hb_end_j; ++pk ) {
//            loopcount = (hb_end_j - hb_start_j) / warpSize + 
//                (((hb_end_j - hb_start_j) % warpSize == 0) ? 0 : 1);
//
//            count = 0;
//            pk = hb_start_j + lane_id;
//            while ( count < loopcount )
//            {
//                /* only allow threads with an actual hbond */
//                if ( pk < hb_end_j )
//                {
//                    phbond_jk = &hbond_list.hbond_list[pk];
//
//                    /* set k's varibles */
//                    k = hbond_list.hbond_list[pk].nbr;
//                    type_k = my_atoms[k].type;
//                    nbr_jk = hbond_list.hbond_list[pk].ptr;
//                    r_jk = far_nbr_list.far_nbr_list.d[nbr_jk];
//
//                    rvec_Scale( dvec_jk, phbond_jk->scl,
//                            far_nbr_list.far_nbr_list.dvec[nbr_jk] );
//                }
//                else
//                {
//                    k = -1;
//                }
//
//                if ( my_atoms[i].orig_id != my_atoms[k].orig_id && k != -1 )
//                {
//                    bo_ij = &pbond_ij->bo_data;
//                    type_i = my_atoms[i].type;
//                    r_ij = pbond_ij->d;
//                    hbp = &d_hbp[ index_hbp(type_i,type_j,type_k,num_atom_types) ];
//
//                    Calculate_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
//                            &theta, &cos_theta );
//
//                    /* the derivative of cos(theta) */
//                    Calculate_dCos_Theta( pbond_ij->dvec, r_ij, dvec_jk, r_jk,
//                            &dcos_theta_di, &dcos_theta_dj, &dcos_theta_dk );
//
//                    /* hydrogen bond energy */
//                    sin_theta2 = SIN( theta / 2.0 );
//                    sin_xhz4 = SQR( sin_theta2 );
//                    sin_xhz4 *= sin_xhz4;
//                    cos_xhz1 = ( 1.0 - cos_theta );
//                    exp_hb2 = EXP( -1.0 * hbp->p_hb2 * bo_ij->BO );
//                    exp_hb3 = EXP( -1.0 * hbp->p_hb3 * ( hbp->r0_hb / r_jk
//                                + r_jk / hbp->r0_hb - 2.0 ) );
//
//                    e_hb = hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * sin_xhz4;
//                    e_hb_l += e_hb;
//
//                    CEhb1 = hbp->p_hb1 * hbp->p_hb2 * exp_hb2 * exp_hb3 * sin_xhz4;
//                    CEhb2 = -0.5 * hbp->p_hb1 * (1.0 - exp_hb2) * exp_hb3 * cos_xhz1;
//                    CEhb3 = hbp->p_hb3 * e_hb * (hbp->r0_hb / SQR( r_jk )
//                            + -1.0 / hbp->r0_hb);
//
//                    /* hydrogen bond forces */
//                    /* dbo term */
//                    CEhb1_l += CEhb1;
//
//                    if ( control->virial == 0 )
//                    {
//#if !defined(CUDA_ACCUM_ATOMIC)
//                        /* dcos terms */
//                        rvec_ScaledAdd( hb_f_l, CEhb2, dcos_theta_di ); 
//                        rvec_ScaledAdd( f_j_l, CEhb2, dcos_theta_dj );
//                        rvec_ScaledAdd( phbond_jk->hb_f, CEhb2, dcos_theta_dk );
//
//                        /* dr terms */
//                        rvec_ScaledAdd( f_j_l, -1.0 * CEhb3 / r_jk, dvec_jk ); 
//                        rvec_ScaledAdd( phbond_jk->hb_f, CEhb3 / r_jk, dvec_jk );
//#else
//                        /* dcos terms */
//                        atomic_rvecScaledAdd( workspace.f[i], CEhb2, dcos_theta_di ); 
//                        rvec_ScaledAdd( f_j_l, CEhb2, dcos_theta_dj );
//                        atomic_rvecScaledAdd( workspace.f[k], CEhb2, dcos_theta_dk );
//
//                        /* dr terms */
//                        rvec_ScaledAdd( f_j_l, -1.0 * CEhb3 / r_jk, dvec_jk ); 
//                        atomic_rvecScaledAdd( workspace.f[k], CEhb3 / r_jk, dvec_jk );
//#endif
//                    }
//                    else
//                    {
//#if !defined(CUDA_ACCUM_ATOMIC)
//                        /* for pressure coupling, terms that are not related to bond order
//                         * derivatives are added directly into pressure vector/tensor */
//                        /* dcos terms */
//                        rvec_Scale( temp, CEhb2, dcos_theta_di );
//                        rvec_Add( pbond_ij->hb_f, temp );
//                        rvec_iMultiply( temp, pbond_ij->rel_box, temp );
//                        rvec_Add( ext_press_l, temp );
//
//                        rvec_ScaledAdd( workspace.f[j], CEhb2, dcos_theta_dj );
//
//                        ivec_Scale( rel_jk, hbond_list.hbond_list[pk].scl,
//                                far_nbr_list.far_nbr_list.rel_box[nbr_jk] );
//                        rvec_Scale( temp, CEhb2, dcos_theta_dk );
//                        rvec_Add( phbond_jk->hb_f, temp );
//                        rvec_iMultiply( temp, rel_jk, temp );
//                        rvec_Add( ext_press_l, temp );
//
//                        /* dr terms */
//                        rvec_ScaledAdd( workspace.f[j], -1.0 * CEhb3 / r_jk, dvec_jk ); 
//
//                        rvec_Scale( temp, CEhb3 / r_jk, dvec_jk );
//                        rvec_Add( phbond_jk->hb_f, temp );
//                        rvec_iMultiply( temp, rel_jk, temp );
//                        rvec_Add( ext_press_l, temp );
//#else
//                        /* for pressure coupling, terms that are not related to bond order
//                         * derivatives are added directly into pressure vector/tensor */
//                        /* dcos terms */
//                        rvec_Scale( temp, CEhb2, dcos_theta_di );
//                        atomic_rvecAdd( workspace.f[i], temp );
//                        rvec_iMultiply( temp, pbond_ij->rel_box, temp );
//                        rvec_Add( ext_press_l, temp );
//
//                        rvec_ScaledAdd( f_j_l, CEhb2, dcos_theta_dj );
//
//                        ivec_Scale( rel_jk, hbond_list.hbond_list[pk].scl,
//                                far_nbr_list.far_nbr_list.rel_box[nbr_jk] );
//                        rvec_Scale( temp, CEhb2, dcos_theta_dk );
//                        atomic_rvecAdd( workspace.f[k], temp );
//                        rvec_iMultiply( temp, rel_jk, temp );
//                        rvec_Add( ext_press_l, temp );
//
//                        /* dr terms */
//                        rvec_ScaledAdd( f_j_l, -1.0 * CEhb3 / r_jk, dvec_jk ); 
//
//                        rvec_Scale( temp, CEhb3 / r_jk, dvec_jk );
//                        atomic_rvecAdd( workspace.f[k], temp );
//                        rvec_iMultiply( temp, rel_jk, temp );
//                        rvec_Add( ext_press_l, temp );
//#endif
//                    }
//
//                } //orid id end
//
//                pk += warpSize;
//                count++;
//
//            } //for itr loop end
//
//            CEhb1_l = WarpReduce(temp_storage[warp_id]).Sum(CEhb1_l);
//#if !defined(CUDA_ACCUM_ATOMIC)
//            hb_f_l[0] = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[0]);
//            hb_f_l[1] = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[1]);
//            hb_f_l[2]  = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[2]);
//#endif
//            }
//
//            /* first thread within a warp writes warp-level sum to shared memory */
//            if ( lane_id == 0 )
//            {
//                bo_ij->Cdbo += CEhb1_l ;
//#if !defined(CUDA_ACCUM_ATOMIC)
//                rvec_Add( pbond_ij->hb_f, hb_f_l );
//#endif
//            }
//        } // for loop hbonds end
//    } //if Hbond check end
//
//    __syncthreads( );
//
//    f_j_l[0] = WarpReduce(temp_storage[warp_id]).Sum(f_j_l[0]);
//    f_j_l[1] = WarpReduce(temp_storage[warp_id]).Sum(f_j_l[1]);
//    f_j_l[2] = WarpReduce(temp_storage[warp_id]).Sum(f_j_l[2]);
//    e_hb_l = WarpReduce(temp_storage[warp_id]).Sum(e_hb_l);
//    ext_press_l[0] = WarpReduce(temp_storage[warp_id]).Sum(ext_press_l[0]);
//    ext_press_l[1] = WarpReduce(temp_storage[warp_id]).Sum(ext_press_l[1]);
//    ext_press_l[2] = WarpReduce(temp_storage[warp_id]).Sum(ext_press_l[2]);
//
//    /* first thread within a warp writes warp-level sums to global memory */
//    if ( lane_id == 0 )
//    {
//#if !defined(CUDA_ACCUM_ATOMIC)
//        rvec_Add( workspace.f[j], f_j_l );
//        e_hb_g[j] = e_hb_l;
//        rvecCopy( ext_press_g[j], ext_press_l );
//#else
//        atomic_rvecAdd( workspace.f[j], f_j_l );
//        atomicAdd( (double *) e_hb_g, (double) e_hb_l );
//        atomic_rvecAdd( *ext_press_g, ext_press_l );
//#endif
//    }
//}


#if !defined(CUDA_ACCUM_ATOMIC)
/* Accumulate forces stored in the bond list
 * using a one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part2( reax_atom *atoms,
        storage workspace, reax_list bond_list, int n )
{
    int j, pj;
    bond_data *pbond, *sym_index_bond;
    rvec hb_f;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= n )
    {
        return;
    }

    rvec_MakeZero( hb_f );

    for ( pj = Start_Index(j, &bond_list); pj < End_Index(j, &bond_list); ++pj )
    {
        pbond = &bond_list.bond_list[pj];
        sym_index_bond = &bond_list.bond_list[pbond->sym_index];

        rvec_Add( hb_f, sym_index_bond->hb_f );
    }

    rvec_Add( workspace.f[j], hb_f );
}


/* Accumulate forces stored in the bond list
 * using a one warp threads per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part2_opt( reax_atom *atoms,
        storage workspace, reax_list bond_list, int n )
{
    typedef hipcub::WarpReduce<double> WarpReduce;
    extern __shared__ typename WarpReduce::TempStorage temp_storage[];
    int j, pj, start, end;
    bond_data *pbond, *sym_index_bond;
    /* thread-local variables */
    int thread_id, warp_id, lane_id;
    rvec hb_f;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id >> 5;
    lane_id = thread_id & 0x0000001F; 

    if ( warp_id >= n )
    {
        return;
    }

    j = warp_id;
    start = Start_Index( j, &bond_list );
    end = End_Index( j, &bond_list );
    pj = start + lane_id;
    rvec_MakeZero( hb_f );

    while ( pj < end )
    {
        pbond = &bond_list.bond_list[pj];
        sym_index_bond = &bond_list.bond_list[pbond->sym_index];

        rvec_Add( hb_f, sym_index_bond->hb_f );

        pj += warpSize;
    }

    __syncthreads( );

    hb_f[0] = WarpReduce(temp_storage[warp_id]).Sum(hb_f[0]);
    hb_f[1] = WarpReduce(temp_storage[warp_id]).Sum(hb_f[1]);
    hb_f[2] = WarpReduce(temp_storage[warp_id]).Sum(hb_f[2]);

    /* first thread within a warp writes warp-level sums to global memory */
    if ( lane_id == 0 )
    {
        rvec_Add( workspace.f[j], hb_f );
    }
}


/* Accumulate forces stored in the hbond list
 * using a one thread per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part3( reax_atom *atoms,
        storage workspace, reax_list hbond_list, int n )
{
    int j, pj;
    hbond_data *nbr_pj, *sym_index_nbr;
    rvec hb_f;

    j = blockIdx.x * blockDim.x + threadIdx.x;

    if ( j >= n )
    {
        return;
    }

    rvec_MakeZero( hb_f );

    for ( pj = Start_Index(atoms[j].Hindex, &hbond_list); pj < End_Index(atoms[j].Hindex, &hbond_list); ++pj )
    {
        nbr_pj = &hbond_list.hbond_list[pj];
        sym_index_nbr = &hbond_list.hbond_list[ nbr_pj->sym_index ];

        rvec_Add( hb_f, sym_index_nbr->hb_f );
    }

    rvec_Add( workspace.f[j], hb_f );
}


/* Accumulate forces stored in the hbond list
 * using a one warp of threads per atom implementation */
CUDA_GLOBAL void k_hydrogen_bonds_part3_opt( reax_atom *atoms,
        storage workspace, reax_list hbond_list, int n )
{
    typedef hipcub::WarpReduce<double> WarpReduce;
    extern __shared__ typename WarpReduce::TempStorage temp_storage[];
    int j, pj, start, end;
    hbond_data *nbr_pj, *sym_index_nbr;
    /* thread-local variables */
    int thread_id, warp_id, lane_id, offset;
    rvec hb_f_l;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    warp_id = thread_id >> 5;
    lane_id = thread_id & 0x0000001F; 

    if ( warp_id >= n )
    {
        return;
    }

    j = warp_id;
    start = Start_Index( atoms[j].Hindex, &hbond_list );
    end = End_Index( atoms[j].Hindex, &hbond_list );
    pj = start + lane_id;
    rvec_MakeZero( hb_f_l );

    while ( pj < end )
    {
        nbr_pj = &hbond_list.hbond_list[pj];
        sym_index_nbr = &hbond_list.hbond_list[ nbr_pj->sym_index ];

        rvec_Add( hb_f_l, sym_index_nbr->hb_f );

        pj += warpSize;
    }

    __syncthreads( );

    hb_f_l[0] = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[0]);
    hb_f_l[1] = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[1]);
    hb_f_l[2] = WarpReduce(temp_storage[warp_id]).Sum(hb_f_l[2]);

    /* first thread within a warp writes warp-level sums to global memory */
    if ( lane_id == 0 )
    {
        rvec_Add( workspace.f[j], hb_f_l );
    }
}
#endif



void Cuda_Compute_Hydrogen_Bonds( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control )
{
//    int hbs, hnbrs_blocks;
#if !defined(CUDA_ACCUM_ATOMIC)
    int update_energy;
    real *spad;
    rvec *rvec_spad;

    cuda_check_malloc( &workspace->scratch, &workspace->scratch_size,
                (sizeof(real) * 3 + sizeof(rvec)) * system->N + sizeof(rvec) * control->blocks_n,
            "Cuda_Compute_Hydrogen_Bonds::workspace->scratch" );
    spad = (real *) workspace->scratch;
    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;
#else
    cuda_memset( &((simulation_data *)data->d_simulation_data)->my_en.e_hb,
            0, sizeof(real), "Cuda_Compute_Hydrogen_Bonds::e_hb" );
    if ( control->virial == 1 )
    {
        cuda_memset( &((simulation_data *)data->d_simulation_data)->my_ext_press,
                0, sizeof(rvec), "Cuda_Compute_Hydrogen_Bonds::my_ext_press" );
    }
#endif

//    hbs = (system->n * HB_KER_THREADS_PER_ATOM / HB_BLOCK_SIZE) + 
//        (((system->n * HB_KER_THREADS_PER_ATOM) % HB_BLOCK_SIZE) == 0 ? 0 : 1);

    if ( control->virial == 1 )
    {
        k_hydrogen_bonds_virial_part1 <<< control->blocks, control->block_size >>>
//        k_hydrogen_bonds_virial_part1_opt <<< hbs, HB_BLOCK_SIZE, 
//                sizeof(real) * (hbs / warpSize) >>>
                ( system->d_my_atoms, system->reax_param.d_sbp,
                  system->reax_param.d_hbp, system->reax_param.d_gp,
                  (control_params *) control->d_control_params,
                  *(workspace->d_workspace),
                  *(lists[FAR_NBRS]), *(lists[BONDS]), *(lists[HBONDS]),
                  system->n, system->reax_param.num_atom_types,
#if !defined(CUDA_ACCUM_ATOMIC)
                  spad, (rvec *) (&spad[system->n])
#else
                  &((simulation_data *)data->d_simulation_data)->my_en.e_hb,
                  &((simulation_data *)data->d_simulation_data)->my_ext_press
#endif
                );
    }
    else
    {
        k_hydrogen_bonds_part1 <<< control->blocks, control->block_size >>>
//        k_hydrogen_bonds_part1_opt <<< hbs, HB_BLOCK_SIZE, 
//                sizeof(real) * (hbs / warpSize) >>>
                ( system->d_my_atoms, system->reax_param.d_sbp,
                  system->reax_param.d_hbp, system->reax_param.d_gp,
                  (control_params *) control->d_control_params,
                  *(workspace->d_workspace),
                  *(lists[FAR_NBRS]), *(lists[BONDS]), *(lists[HBONDS]),
                  system->n, system->reax_param.num_atom_types,
#if !defined(CUDA_ACCUM_ATOMIC)
                  spad
#else
                  &((simulation_data *)data->d_simulation_data)->my_en.e_hb
#endif
                );
    }
    cudaCheckError( );

#if !defined(CUDA_ACCUM_ATOMIC)
    if ( update_energy == TRUE )
    {
        Cuda_Reduction_Sum( spad,
                &((simulation_data *)data->d_simulation_data)->my_en.e_hb,
                system->n );
    }

    if ( control->virial == 1 )
    {
        rvec_spad = (rvec *) (&spad[system->n]);

        k_reduction_rvec <<< control->blocks, control->block_size,
                         sizeof(rvec) * (control->block_size / 32) >>>
            ( rvec_spad, &rvec_spad[system->n], system->n );
        cudaCheckError( );

        k_reduction_rvec <<< 1, control->blocks_pow_2,
                         sizeof(rvec) * (control->blocks_pow_2 / 32) >>>
            ( &rvec_spad[system->n],
              &((simulation_data *)data->d_simulation_data)->my_ext_press,
              control->blocks );
        cudaCheckError( );
//        Cuda_Reduction_Sum( rvec_spad,
//                &((simulation_data *)data->d_simulation_data)->my_ext_press,
//                system->n );
    }
#endif

#if !defined(CUDA_ACCUM_ATOMIC)
    k_hydrogen_bonds_part2 <<< control->blocks, control->block_size >>>
        ( system->d_my_atoms, *(workspace->d_workspace),
          *(lists[BONDS]), system->n );
    cudaCheckError( );

//    hnbrs_blocks = (system->n * HB_POST_PROC_KER_THREADS_PER_ATOM / HB_POST_PROC_BLOCK_SIZE) +
//        (((system->n * HB_POST_PROC_KER_THREADS_PER_ATOM) % HB_POST_PROC_BLOCK_SIZE) == 0 ? 0 : 1);

    k_hydrogen_bonds_part3 <<< control->blocks, control->block_size >>>
        ( system->d_my_atoms, *(workspace->d_workspace), *(lists[HBONDS]), system->n );
//    k_hydrogen_bonds_part3_opt <<< hnbrs_blocks, HB_POST_PROC_BLOCK_SIZE, 
//            HB_POST_PROC_BLOCK_SIZE * sizeof(rvec) >>>
//        ( system->d_my_atoms, *(workspace->d_workspace), *(lists[HBONDS]), system->n );
    cudaCheckError( );
#endif
}

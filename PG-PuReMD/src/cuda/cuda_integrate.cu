
#include "cuda_integrate.h"

#include "cuda_allocate.h"
#include "cuda_box.h"
#include "cuda_forces.h"
#include "cuda_integrate.h"
#include "cuda_copy.h"
#include "cuda_neighbors.h"
#include "cuda_reduction.h"
#include "cuda_reset_tools.h"
#include "cuda_system_props.h"
#include "cuda_utils.h"

#include "../comm_tools.h"
#include "../grid.h"
#include "../vector.h"


CUDA_GLOBAL void k_update_velocity_1( reax_atom *my_atoms, 
        single_body_parameters *sbp, real dt, int n )
{
    real inv_m;
    rvec dx;
    reax_atom *atom;
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    /* velocity verlet, 1st part */
    atom = &(my_atoms[i]);
    inv_m = 1.0 / sbp[atom->type].mass;
    /* Compute x(t + dt) */
    rvec_ScaledSum( dx, dt, atom->v, 0.5 * -F_CONV * inv_m * SQR(dt), atom->f );
    rvec_Add( atom->x, dx );
    /* Compute v(t + dt/2) */
    rvec_ScaledAdd( atom->v, 0.5 * -F_CONV * inv_m * dt, atom->f );
}


void update_velocity_part1( reax_system *system, real dt )
{
    int blocks;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_update_velocity_1 <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, dt, system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


CUDA_GLOBAL void k_update_velocity_2( reax_atom *my_atoms, 
        single_body_parameters *sbp, real dt, int n )
{
    reax_atom *atom;
    real inv_m;
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    /* velocity verlet, 2nd part */
    atom = &(my_atoms[i]);
    inv_m = 1.0 / sbp[atom->type].mass;
    /* Compute v(t + dt) */
    rvec_ScaledAdd( atom->v, 0.5 * dt * -F_CONV * inv_m, atom->f );
}


void update_velocity_part2( reax_system *system, real dt )
{
    int blocks;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_update_velocity_2 <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, dt, system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


CUDA_GLOBAL void k_nhNVT_update_velocity_1( reax_atom *my_atoms, 
        single_body_parameters *sbp, real dt, int n )
{
    real inv_m;
    rvec dx;
    reax_atom *atom;
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    /* velocity verlet, 1st part */
    atom = &(my_atoms[i]);
    inv_m = 1.0 / sbp[atom->type].mass;
    rvec_ScaledSum( dx, dt, atom->v, 0.5 * -F_CONV * inv_m * SQR(dt), atom->f );
    rvec_Add( atom->x, dx );
    rvec_Copy( atom->f_old, atom->f );
}


void nhNVT_update_velocity_part1( reax_system *system, real dt )
{
    int blocks;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_nhNVT_update_velocity_1 <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, dt, system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


CUDA_GLOBAL void k_nhNVT_update_velocity_2( reax_atom *my_atoms, rvec * v_const,
        single_body_parameters *sbp, real dt, real v_xi, int n )
{
    reax_atom *atom;
    real inv_m;
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    /* velocity verlet, 2nd part */
    atom = &(my_atoms[i]);
    inv_m = 1.0 / sbp[atom->type].mass;
    /* Compute v(t + dt) */
    rvec_Scale( v_const[i], 1.0 - 0.5 * dt * v_xi, atom->v );
    rvec_ScaledAdd( v_const[i], 0.5 * dt * inv_m * -F_CONV, atom->f_old );
    rvec_ScaledAdd( v_const[i], 0.5 * dt * inv_m * -F_CONV, atom->f );
}


void nhNVT_update_velocity_part2( reax_system *system, storage *workspace, real dt, real v_xi )
{
    int blocks;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_nhNVT_update_velocity_2 <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, workspace->v_const, system->reax_param.d_sbp, dt, v_xi, system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


CUDA_GLOBAL void k_nhNVT_update_velocity_3( reax_atom *my_atoms, rvec *v_const,
        single_body_parameters *sbp, real dt, real v_xi_old, real * my_ekin, int n )
{
    reax_atom *atom;
    real coef_v;
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    atom = &(my_atoms[i]);
    coef_v = 1.0 / (1.0 + 0.5 * dt * v_xi_old);
    rvec_Scale( atom->v, coef_v, v_const[i] );
    my_ekin[i] = (0.5 * sbp[atom->type].mass * rvec_Dot(atom->v, atom->v));
}


int nhNVT_update_velocity_part3( reax_system *system, storage *workspace,
       real dt, real v_xi_old, real * d_my_ekin, real * d_total_my_ekin )
{
    int blocks, my_ekin;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_nhNVT_update_velocity_3 <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, workspace->v_const, system->reax_param.d_sbp, dt, v_xi_old, d_my_ekin, system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );

    Cuda_Reduction_Sum( d_my_ekin, d_total_my_ekin, system->n );

    copy_host_device( &my_ekin, d_total_my_ekin, sizeof(int), 
            hipMemcpyDeviceToHost, "nhNVT_update_velocity_part3::d_total_my_ekin" );

    return my_ekin;
}


CUDA_GLOBAL void k_bNVT_scale_velocities( reax_atom *my_atoms, real lambda, int n )
{
    reax_atom *atom;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    /* Scale velocities and positions at t+dt */
    atom = &(my_atoms[i]);
    rvec_Scale( atom->v, lambda, atom->v );
}


void bNVT_scale_velocities( reax_system *system, real lambda )
{
    int blocks;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_bNVT_scale_velocities <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, lambda, system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


CUDA_GLOBAL void k_bNVP_scale_velocities( reax_atom *my_atoms, real lambda,
        real mu0, real mu1, real mu2, int n )
{
    reax_atom *atom;
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    /* Scale velocities and positions at t+dt */
    atom = &(my_atoms[i]);
    rvec_Scale( atom->v, lambda, atom->v );
//    rvec_Multiply( atom->x, mu, atom->x );
    atom->x[0] = mu0 * atom->x[0];
    atom->x[1] = mu1 * atom->x[1];
    atom->x[2] = mu2 * atom->x[2];
}


void bNVP_scale_velocities( reax_system *system, real lambda, rvec mu )
{
    int blocks;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_bNVP_scale_velocities <<< blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, lambda, mu[0], mu[1], mu[2], system->n );
    hipDeviceSynchronize( );
    cudaCheckError( );
}


int Cuda_Velocity_Verlet_NVE( reax_system* system, control_params* control,
        simulation_data *data, storage *workspace, reax_list **lists,
        output_controls *out_control, mpi_datatypes *mpi_data )
{
    int steps, renbr, ret;
    static int verlet_part1_done = FALSE, far_nbrs_done = FALSE;
    real dt;
#if defined(DEBUG)
    real t_over_start, t_over_elapsed;
#endif

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "p%d @ step %d\n", system->my_rank, data->step );
    MPI_Barrier( MPI_COMM_WORLD );
#endif

    dt = control->dt;
    steps = data->step - data->prev_steps;
    renbr = steps % control->reneighbor == 0 ? TRUE : FALSE;
    ret = SUCCESS;

    if ( verlet_part1_done == FALSE )
    {
        update_velocity_part1( system, dt );

        verlet_part1_done = TRUE;

#if defined(DEBUG_FOCUS)
        fprintf( stderr, "p%d @ step%d: verlet1 done\n", system->my_rank, data->step );
        MPI_Barrier( MPI_COMM_WORLD );
#endif

        if ( renbr )
        {
            Update_Grid( system, control, mpi_data->world );
        }

        Output_Sync_Atoms( system );
        Comm_Atoms( system, control, data, workspace, lists, mpi_data, renbr );
        Sync_Atoms( system );

        /* sync grid to device */
        Sync_Grid( &system->my_grid, &system->d_my_grid );

        init_blocks( system );
    }

    Cuda_ReAllocate( system, control, data, workspace, lists, mpi_data );

    Cuda_Reset( system, control, data, workspace, lists );

    if ( renbr && far_nbrs_done == FALSE )
    {
#if defined(DEBUG)
        t_over_start  = Get_Time( );
#endif

        ret = Cuda_Generate_Neighbor_Lists( system, data, workspace, lists );

        if ( ret != SUCCESS )
        {
            Cuda_Estimate_Neighbors( system );
        }
        if ( ret == SUCCESS )
        {
            far_nbrs_done = TRUE;
        }
    
#if defined(DEBUG)
        t_over_elapsed = Get_Timing_Info( t_over_start );
        fprintf( stderr, "p%d --> Overhead (Step-%d) %f \n",
                system->my_rank, data->step, t_over_elapsed );
#endif
    }

    if ( ret == SUCCESS )
    {
        ret = Cuda_Compute_Forces( system, control, data, workspace,
                lists, out_control, mpi_data );
    }

    if ( ret == SUCCESS )
    {
        update_velocity_part2( system, dt );

        verlet_part1_done = FALSE;
        far_nbrs_done = FALSE;
    }
    
#if defined(DEBUG_FOCUS)
    fprintf( stderr, "p%d @ step%d: verlet2 done\n", system->my_rank, data->step );
    MPI_Barrier( MPI_COMM_WORLD );
#endif

    return ret;
}


int Cuda_Velocity_Verlet_Nose_Hoover_NVT_Klein( reax_system* system,
        control_params* control, simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control, mpi_datatypes *mpi_data )
{
    int itr, steps, renbr, ret;
    real *d_my_ekin, *d_total_my_ekin;
    static int verlet_part1_done = FALSE, far_nbrs_done = FALSE;
    real dt, dt_sqr;
    real my_ekin, new_ekin;
    real G_xi_new, v_xi_new, v_xi_old;
    thermostat *therm;

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "p%d @ step%d\n", system->my_rank, data->step );
    MPI_Barrier( MPI_COMM_WORLD );
#endif

    dt = control->dt;
    dt_sqr = SQR(dt);
    therm = &( data->therm );
    steps = data->step - data->prev_steps;
    renbr = steps % control->reneighbor == 0 ? TRUE : FALSE;

    if ( verlet_part1_done == FALSE )
    {
        nhNVT_update_velocity_part1( system, dt );
    
        /* Compute xi(t + dt) */
        therm->xi += ( therm->v_xi * dt + 0.5 * dt_sqr * therm->G_xi );

        verlet_part1_done = TRUE;

#if defined(DEBUG_FOCUS)
        fprintf( stderr, "p%d @ step%d: verlet1 done\n", system->my_rank, data->step );
        MPI_Barrier( MPI_COMM_WORLD );
#endif

        if ( renbr )
        {
            Update_Grid( system, control, mpi_data->world );
        }

        Output_Sync_Atoms( system );
        Comm_Atoms( system, control, data, workspace, lists, mpi_data, renbr );
        Sync_Atoms( system );

        /* sync grid to device */
        Sync_Grid( &system->my_grid, &system->d_my_grid );

        init_blocks( system );
    }

    Cuda_ReAllocate( system, control, data, workspace, lists, mpi_data );

    Cuda_Reset( system, control, data, workspace, lists );

    if ( renbr && far_nbrs_done == FALSE )
    {
#if defined(DEBUG)
        t_over_start  = Get_Time( );
#endif

        ret = Cuda_Generate_Neighbor_Lists( system, data, workspace, lists );

        if ( ret != SUCCESS )
        {
            Cuda_Estimate_Neighbors( system );
        }
        if ( ret == SUCCESS )
        {
            far_nbrs_done = TRUE;
        }

#if defined(DEBUG)
        t_over_elapsed = Get_Timing_Info( t_over_start );
        fprintf( stderr, "p%d --> Overhead (Step-%d) %f \n",
                system->my_rank, data->step, t_over_elapsed );
#endif
    }

    if ( ret == SUCCESS )
    {
        ret = Cuda_Compute_Forces( system, control, data, workspace,
                lists, out_control, mpi_data );
    }

    if ( ret == SUCCESS )
    {
        /* Compute iteration constants for each atom's velocity */
        nhNVT_update_velocity_part2( system, dev_workspace, dt, therm->v_xi );
    
        v_xi_new = therm->v_xi_old + 2.0 * dt * therm->G_xi;
        my_ekin = G_xi_new = v_xi_old = 0;
        itr = 0;

        cuda_malloc( (void **) &d_my_ekin, sizeof(real) * system->n, FALSE,
                "Cuda_Velocity_Verlet_Nose_Hoover_NVT_Klein::d_my_ekin" );
        cuda_malloc( (void **) &d_total_my_ekin, sizeof(real), FALSE,
                "Cuda_Velocity_Verlet_Nose_Hoover_NVT_Klein::d_total_my_ekin" );

        do
        {
            itr++;
    
            /* new values become old in this iteration */
            v_xi_old = v_xi_new;
    
            my_ekin = nhNVT_update_velocity_part3( system, dev_workspace, dt, v_xi_old,
                    d_my_ekin, d_total_my_ekin );
    
            MPI_Allreduce( &my_ekin, &new_ekin, 1, MPI_DOUBLE, MPI_SUM,
                    mpi_data->comm_mesh3D  );
    
            G_xi_new = control->Tau_T * ( 2.0 * new_ekin - data->N_f * K_B * control->T );
            v_xi_new = therm->v_xi + 0.5 * dt * ( therm->G_xi + G_xi_new );
        }
        while ( FABS(v_xi_new - v_xi_old) > 1e-5 );
        therm->v_xi_old = therm->v_xi;
        therm->v_xi = v_xi_new;
        therm->G_xi = G_xi_new;

        cuda_free( d_total_my_ekin,
                "Cuda_Velocity_Verlet_Nose_Hoover_NVT_Klein::d_total_my_ekin" );
        cuda_free( d_my_ekin,
                "Cuda_Velocity_Verlet_Nose_Hoover_NVT_Klein::d_my_ekin" );

        verlet_part1_done = FALSE;
        far_nbrs_done = FALSE;
    }
    
#if defined(DEBUG_FOCUS)
    fprintf( stderr, "p%d @ step%d: verlet2 done\n", system->my_rank, data->step );
    MPI_Barrier( MPI_COMM_WORLD );
#endif

    return ret;
}


/* uses Berendsen-type coupling for both T and P.
   All box dimensions are scaled by the same amount,
   there is no change in the angles between axes. */
int Cuda_Velocity_Verlet_Berendsen_NVT( reax_system* system, control_params* control,
        simulation_data *data, storage *workspace, reax_list **lists,
        output_controls *out_control, mpi_datatypes *mpi_data )
{
    int steps, renbr, ret;
    static int verlet_part1_done = FALSE, far_nbrs_done = FALSE;
    real dt, lambda;
#if defined(DEBUG)
    real t_over_start, t_over_elapsed;
#endif

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "p%d @ step%d\n", system->my_rank, data->step );
    MPI_Barrier( MPI_COMM_WORLD );
#endif

    dt = control->dt;
    steps = data->step - data->prev_steps;
    renbr = steps % control->reneighbor == 0 ? TRUE : FALSE;
    ret = SUCCESS;

    if ( verlet_part1_done == FALSE )
    {
        /* velocity verlet, 1st part */
        update_velocity_part1( system, dt );

        verlet_part1_done = TRUE;

#if defined(DEBUG_FOCUS)
        fprintf( stderr, "p%d @ step%d: verlet1 done\n", system->my_rank, data->step );
        MPI_Barrier( MPI_COMM_WORLD );
#endif

        Cuda_ReAllocate( system, control, data, workspace, lists, mpi_data );

        if ( renbr )
        {
            Update_Grid( system, control, mpi_data->world );
        }

        Output_Sync_Atoms( system );
        Comm_Atoms( system, control, data, workspace, lists, mpi_data, renbr );
        Sync_Atoms( system );

        /* sync grid to device */
        Sync_Grid( &system->my_grid, &system->d_my_grid );

        init_blocks( system );
    
        Cuda_Reset( system, control, data, workspace, lists );
    }
    else
    {
        Cuda_ReAllocate( system, control, data, workspace, lists, mpi_data );
    
        Cuda_Reset( system, control, data, workspace, lists );
    }

    if ( renbr && far_nbrs_done == FALSE )
    {
#if defined(DEBUG)
        t_over_start  = Get_Time( );
#endif

        ret = Cuda_Generate_Neighbor_Lists( system, data, workspace, lists );

        if ( ret != SUCCESS )
        {
            Cuda_Estimate_Neighbors( system );
        }
        if ( ret == SUCCESS )
        {
            far_nbrs_done = TRUE;
        }
        
#if defined(DEBUG)
        t_over_elapsed  = Get_Timing_Info( t_over_start );
        fprintf( stderr, "p%d --> Overhead (Step-%d) %f \n",
                system->my_rank, data->step, t_over_elapsed );
#endif
    }

    if ( ret == SUCCESS )
    {
        ret = Cuda_Compute_Forces( system, control, data, workspace,
                lists, out_control, mpi_data );
    }

    if ( ret == SUCCESS )
    {
        /* velocity verlet, 2nd part */
        update_velocity_part2( system, dt );

#if defined(DEBUG_FOCUS)
        fprintf(stderr, "p%d @ step%d: verlet2 done\n", system->my_rank, data->step);
        MPI_Barrier( MPI_COMM_WORLD );
#endif

        /* temperature scaler */
        Cuda_Compute_Kinetic_Energy( system, data, mpi_data->comm_mesh3D );

        lambda = 1.0 + (dt / control->Tau_T) * (control->T / data->therm.T - 1.0);
        if ( lambda < MIN_dT )
        {
            lambda = MIN_dT;
        }
        else if (lambda > MAX_dT )
        {
            lambda = MAX_dT;
        }
        lambda = SQRT( lambda );

        /* Scale velocities and positions at t+dt */
        bNVT_scale_velocities( system, lambda );

        Cuda_Compute_Kinetic_Energy( system, data, mpi_data->comm_mesh3D );

#if defined(DEBUG_FOCUS)
        fprintf( stderr, "p%d @ step%d: scaled velocities\n",
                 system->my_rank, data->step );
        MPI_Barrier( MPI_COMM_WORLD );
#endif

        verlet_part1_done = FALSE;
        far_nbrs_done = FALSE;
    }

    return ret;
}


/* uses Berendsen-type coupling for both T and P.
 * All box dimensions are scaled by the same amount,
 * there is no change in the angles between axes. */
int Cuda_Velocity_Verlet_Berendsen_NPT( reax_system* system, control_params* control,
        simulation_data *data, storage *workspace, reax_list **lists,
        output_controls *out_control, mpi_datatypes *mpi_data )
{
    int steps, renbr, ret;
    static int verlet_part1_done = FALSE, far_nbrs_done = FALSE;
    real dt;
#if defined(DEBUG)
    real t_over_start, t_over_elapsed;
#endif

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "p%d @ step %d\n", system->my_rank, data->step );
    MPI_Barrier( MPI_COMM_WORLD );
#endif

    dt = control->dt;
    steps = data->step - data->prev_steps;
    renbr = steps % control->reneighbor == 0 ? TRUE : FALSE;
    ret = SUCCESS;

    if ( verlet_part1_done == FALSE )
    {
        update_velocity_part1( system, dt );

        verlet_part1_done = TRUE;

#if defined(DEBUG_FOCUS)
        fprintf( stderr, "p%d @ step%d: verlet1 done\n", system->my_rank, data->step );
        MPI_Barrier( MPI_COMM_WORLD );
#endif

        if ( renbr )
        {
            Update_Grid( system, control, mpi_data->world );
        }

        Output_Sync_Atoms( system );
        Comm_Atoms( system, control, data, workspace, lists, mpi_data, renbr );
        Sync_Atoms( system );

        /* sync grid to device */
        Sync_Grid( &system->my_grid, &system->d_my_grid );

        init_blocks( system );
    }

    Cuda_ReAllocate( system, control, data, workspace, lists, mpi_data );

    Cuda_Reset( system, control, data, workspace, lists );

    if ( renbr && far_nbrs_done == FALSE )
    {
#if defined(DEBUG)
        t_over_start  = Get_Time( );
#endif

        ret = Cuda_Generate_Neighbor_Lists( system, data, workspace, lists );

        if ( ret != SUCCESS )
        {
            Cuda_Estimate_Neighbors( system );
        }
        if ( ret == SUCCESS )
        {
            far_nbrs_done = TRUE;
        }
    
#if defined(DEBUG)
        t_over_elapsed = Get_Timing_Info( t_over_start );
        fprintf( stderr, "p%d --> Overhead (Step-%d) %f \n",
                system->my_rank, data->step, t_over_elapsed );
#endif
    }

    if ( ret == SUCCESS )
    {
        ret = Cuda_Compute_Forces( system, control, data, workspace,
                lists, out_control, mpi_data );
    }

    if ( ret == SUCCESS )
    {
        update_velocity_part2( system, dt );

        Cuda_Compute_Kinetic_Energy( system, data, mpi_data->comm_mesh3D );
        Cuda_Compute_Pressure( system, control, data, mpi_data );
        Cuda_Scale_Box( system, control, data, mpi_data );

        verlet_part1_done = FALSE;
        far_nbrs_done = FALSE;
    }
    
#if defined(DEBUG_FOCUS)
    fprintf( stderr, "p%d @ step%d: verlet2 done\n", system->my_rank, data->step );
    MPI_Barrier( MPI_COMM_WORLD );
#endif

    return ret;
}

#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "cuda_spar_lin_alg.h"

#if defined(CUDA_DEVICE_PACK)
  #include "cuda_basic_comm.h"
#endif
#include "cuda_dense_lin_alg.h"
#include "cuda_helpers.h"
#include "cuda_utils.h"
#include "cuda_reduction.h"

#if !defined(CUDA_DEVICE_PACK)
  #include "../basic_comm.h"
#endif
#include "../comm_tools.h"
#include "../tool_box.h"


/* mask used to determine which threads within a warp participate in operations */
#define FULL_MASK (0xFFFFFFFF)


/* Jacobi preconditioner computation */
CUDA_GLOBAL void k_jacobi_cm_half( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        real * const Hdia_inv, int N )
{
    int i;
    real diag;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    if ( FABS( vals[row_ptr_end[i]] ) >= 1.0e-12 )
    {
        diag = 1.0 / vals[row_ptr_end[i]];
    }
    else
    {
        diag = 1.0;
    }

    Hdia_inv[i] = diag;
}


/* Jacobi preconditioner computation */
CUDA_GLOBAL void k_jacobi_cm_full( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        real * const Hdia_inv, int N )
{
    int i, pj;
    real diag;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    for ( pj = row_ptr_start[i]; pj < row_ptr_end[i]; ++pj )
    {
        if ( col_ind[pj] == i )
        {
            if ( FABS( vals[pj] ) >= 1.0e-12 )
            {
                diag = 1.0 / vals[pj];
            }
            else
            {
                diag = 1.0;
            }

            break;
        }
    }

    __syncthreads( );

    Hdia_inv[i] = diag;
}


CUDA_GLOBAL void k_dual_jacobi_apply( real const * const Hdia_inv, rvec2 const * const y,
        rvec2 * const x, int n )
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    x[i][0] = Hdia_inv[i] * y[i][0];
    x[i][1] = Hdia_inv[i] * y[i][1];
}


CUDA_GLOBAL void k_jacobi_apply( real const * const Hdia_inv, real const * const y,
        real * const x, int n )
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    x[i] = Hdia_inv[i] * y[i];
}


/* sparse matrix, dense vector multiplication Ax = b,
 * where one GPU thread multiplies a row
 *
 * A: symmetric (upper triangular portion only stored) matrix,
 *    stored in CSR format
 * x: dense vector, size equal to num. columns in A
 * b (output): dense vector, size equal to num. columns in A
 * N: number of rows in A */
CUDA_GLOBAL void k_sparse_matvec_half_csr( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        const real * const x, real * const b, int N )
{
    int i, pj, si, ei;
    real sum;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    si = row_ptr_start[i];
    ei = row_ptr_end[i];

    /* A symmetric, upper triangular portion stored
     * => diagonal only contributes once */
    sum = vals[si] * x[i];

    for ( pj = si + 1; pj < ei; ++pj )
    {
        sum += vals[pj] * x[col_ind[pj]];
        /* symmetric contribution to row j */
        atomicAdd( (double *) &b[col_ind[pj]], (double) (vals[pj] * x[i]) );
    }

    /* local contribution to row i for this thread */
    atomicAdd( (double *) &b[i], (double) sum );
}


/* sparse matrix, dense vector multiplication Ax = b,
 * where warps of 32 threads collaborate to multiply each row
 *
 * A: symmetric (upper triangular portion only stored) matrix,
 *    stored in CSR format
 * x: dense vector, size equal to num. columns in A
 * b (output): dense vector, size equal to num. columns in A
 * N: number of rows in A */
CUDA_GLOBAL void k_sparse_matvec_half_opt_csr( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        const real * const x, real * const b, int N )
{
    int pj, si, ei, thread_id, warp_id, lane_id, offset, itr, col_ind_l;
    real vals_l, sum;

    thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    warp_id = thread_id >> 5;

    if ( warp_id >= N )
    {
        return;
    }

    lane_id = thread_id & 0x0000001F; 
    si = row_ptr_start[warp_id];
    ei = row_ptr_end[warp_id];
    sum = 0.0;

    /* partial sums per thread */
    for ( itr = 0, pj = si + lane_id; itr < (ei - si + 0x0000001F) >> 5; ++itr )
    {
        /* coaleseced 128-bit aligned reads from global memory */
        vals_l = vals[pj];
        col_ind_l = col_ind[pj];

        /* only threads with value non-zero positions accumulate the result */
        if ( pj < ei )
        {
            /* gather on x from global memory and compute partial sum for this non-zero entry */
            sum += vals_l * x[col_ind_l];

            /* A symmetric, upper triangular portion stored
             * => diagonal only contributes once */
            if ( pj > si )
            {
                /* symmetric contribution to row j */
                atomicAdd( (double *) &b[col_ind[pj]], (double) (vals_l * x[warp_id]) );
            }
        }

        pj += warpSize;
    }

    /* warp-level reduction of partial sums
     * using registers within a warp */
    for ( offset = warpSize >> 1; offset > 0; offset >>= 1 )
    {
        sum += __shfl_down_sync( FULL_MASK, sum, offset );
    }

    /* local contribution to row i for this warp */
    if ( lane_id == 0 )
    {
        atomicAdd( (double *) &b[warp_id], (double) sum );
    }
}


/* sparse matrix, dense vector multiplication Ax = b,
 * where one GPU thread multiplies a row
 *
 * A: symmetric matrix,
 *    stored in CSR format
 * x: dense vector, size equal to num. columns in A
 * b (output): dense vector, size equal to num. columns in A
 * N: number of rows in A */
CUDA_GLOBAL void k_sparse_matvec_full_csr( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        const real * const x, real * const b, int n )
{
    int i, pj, si, ei;
    real sum;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    sum = 0.0;
    si = row_ptr_start[i];
    ei = row_ptr_end[i];

    for ( pj = si; pj < ei; ++pj )
    {
        sum += vals[pj] * x[col_ind[pj]];
    }

    __syncthreads( );

    b[i] = sum;
}


/* sparse matrix, dense vector multiplication Ax = b,
 * where warps of 32 threads collaborate to multiply each row
 *
 * A: symmetric matrix,
 *    stored in CSR format
 * x: dense vector, size equal to num. columns in A
 * b (output): dense vector, size equal to num. columns in A
 * N: number of rows in A */
CUDA_GLOBAL void k_sparse_matvec_full_opt_csr( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        const real * const x, real * const b, int n )
{
    int pj, si, ei, thread_id, warp_id, lane_id, offset, itr, col_ind_l;
    real vals_l, sum;

    thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    warp_id = thread_id >> 5;

    if ( warp_id >= n )
    {
        return;
    }

    lane_id = thread_id & 0x0000001F; 
    si = row_ptr_start[warp_id];
    ei = row_ptr_end[warp_id];
    sum = 0.0;

    /* partial sums per thread */
    for ( itr = 0, pj = si + lane_id; itr < (ei - si + 0x0000001F) >> 5; ++itr )
    {
        /* coaleseced 128-bit aligned reads from global memory */
        vals_l = vals[pj];
        col_ind_l = col_ind[pj];

        /* only threads with value non-zero positions accumulate the result */
        if ( pj < ei )
        {
            /* gather on x from global memory and compute partial sum for this non-zero entry */
            sum += vals_l * x[col_ind_l];
        }

        pj += warpSize;
    }

    /* warp-level reduction of partial sums
     * using registers within a warp */
    for ( offset = warpSize >> 1; offset > 0; offset >>= 1 )
    {
        sum += __shfl_down_sync( FULL_MASK, sum, offset );
    }

    __syncthreads( );

    /* first thread within a warp writes sum to global memory */
    if ( lane_id == 0 )
    {
        b[warp_id] = sum;
    }
}


/* sparse matrix, dense vector multiplication Ax = b,
 * where one GPU thread multiplies a row
 *
 * A: symmetric (upper triangular portion only stored) matrix,
 *    stored in CSR format
 * X: 2 dense vectors, size equal to num. columns in A
 * B (output): 2 dense vectors, size equal to num. columns in A
 * N: number of rows in A */
CUDA_GLOBAL void k_dual_sparse_matvec_half_csr( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        const rvec2 * const x, rvec2 * const b, int N )
{
    int i, pj, si, ei;
    rvec2 sum;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    si = row_ptr_start[i];
    ei = row_ptr_end[i];

    /* A symmetric, upper triangular portion stored
     * => diagonal only contributes once */
    sum[0] = vals[si] * x[i][0];
    sum[1] = vals[si] * x[i][1];

    for ( pj = si + 1; pj < ei; ++pj )
    {
        sum[0] += vals[pj] * x[col_ind[pj]][0];
        sum[1] += vals[pj] * x[col_ind[pj]][1];
        /* symmetric contribution to row j */
        atomicAdd( (double *) &b[col_ind[pj]][0], (double) (vals[pj] * x[i][0]) );
        atomicAdd( (double *) &b[col_ind[pj]][1], (double) (vals[pj] * x[i][1]) );
    }

    /* local contribution to row i for this thread */
    atomicAdd( (double *) &b[i][0], (double) sum[0] );
    atomicAdd( (double *) &b[i][1], (double) sum[1] );
}


/* sparse matrix, dense vector multiplication Ax = b,
 * where warps of 32 threads collaborate to multiply each row
 *
 * A: symmetric (upper triangular portion only stored) matrix,
 *    stored in CSR format
 * X: 2 dense vectors, size equal to num. columns in A
 * B (output): 2 dense vectors, size equal to num. columns in A
 * N: number of rows in A */
CUDA_GLOBAL void k_dual_sparse_matvec_half_opt_csr( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        const rvec2 * const x, rvec2 * const b, int N )
{
    int pj, si, ei, thread_id, warp_id, lane_id, offset;
    rvec2 sum;

    thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    warp_id = thread_id >> 5;

    if ( warp_id >= N )
    {
        return;
    }

    lane_id = thread_id & 0x0000001F; 
    si = row_ptr_start[warp_id];
    ei = row_ptr_end[warp_id];

    /* A symmetric, upper triangular portion stored
     * => diagonal only contributes once */
    if ( lane_id == 0 )
    {
        sum[0] = vals[si] * x[warp_id][0];
        sum[1] = vals[si] * x[warp_id][1];
    }
    else
    {
        sum[0] = 0.0;
        sum[1] = 0.0;
    }

    /* partial sums per thread */
    for ( pj = si + lane_id + 1; pj < ei; pj += warpSize )
    {
        sum[0] += vals[pj] * x[col_ind[pj]][0];
        sum[1] += vals[pj] * x[col_ind[pj]][1];
        /* symmetric contribution to row j */
        atomicAdd( (double *) &b[col_ind[pj]][0], (double) (vals[pj] * x[warp_id][0]) );
        atomicAdd( (double *) &b[col_ind[pj]][1], (double) (vals[pj] * x[warp_id][1]) );
    }

    /* warp-level reduction of partial sums
     * using registers within a warp */
    for ( offset = warpSize >> 1; offset > 0; offset >>= 1 )
    {
        sum[0] += __shfl_down_sync( FULL_MASK, sum[0], offset );
        sum[1] += __shfl_down_sync( FULL_MASK, sum[1], offset );
    }

    /* local contribution to row i for this warp */
    if ( lane_id == 0 )
    {
        atomicAdd( (double *) &b[warp_id][0], (double) sum[0] );
        atomicAdd( (double *) &b[warp_id][1], (double) sum[1] );
    }
}


/* 1 thread per row implementation */
CUDA_GLOBAL void k_dual_sparse_matvec_full_csr( sparse_matrix A,
        rvec2 const * const x, rvec2 * const b, int n )
{
    int i, pj, si, ei;
    rvec2 sum;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    sum[0] = 0.0;
    sum[1] = 0.0;
    si = A.start[i];
    ei = A.end[i];

    for ( pj = si; pj < ei; ++pj )
    {
        sum[0] += A.val[pj] * x[A.j[pj]][0];
        sum[1] += A.val[pj] * x[A.j[pj]][1];
    }

    b[i][0] = sum[0];
    b[i][1] = sum[1];
}


/* sparse matrix, dense vector multiplication AX = B,
 * where warps of 32 threads
 * collaborate to multiply each row
 *
 * A: symmetric matrix,
 *    stored in CSR format
 * X: 2 dense vectors, size equal to num. columns in A
 * B (output): 2 dense vectors, size equal to num. columns in A
 * n: number of rows in A */
CUDA_GLOBAL void k_dual_sparse_matvec_full_opt_csr( int *row_ptr_start,
        int *row_ptr_end, int *col_ind, real *vals,
        rvec2 const * const x, rvec2 * const b, int n )
{
    int pj, si, ei, thread_id, warp_id, lane_id, offset;
    rvec2 sum;

    thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    warp_id = thread_id >> 5;

    if ( warp_id >= n )
    {
        return;
    }

    lane_id = thread_id & 0x0000001F; 
    si = row_ptr_start[warp_id];
    ei = row_ptr_end[warp_id];
    sum[0] = 0.0;
    sum[1] = 0.0;

    /* partial sums per thread */
    for ( pj = si + lane_id; pj < ei; pj += warpSize )
    {
        sum[0] += vals[pj] * x[col_ind[pj]][0];
        sum[1] += vals[pj] * x[col_ind[pj]][1];
    }

    /* warp-level reduction of partial sums
     * using registers within a warp */
    for ( offset = warpSize >> 1; offset > 0; offset >>= 1 )
    {
        sum[0] += __shfl_down_sync( FULL_MASK, sum[0], offset );
        sum[1] += __shfl_down_sync( FULL_MASK, sum[1], offset );
    }

    __syncthreads( );

    /* first thread within a warp writes sum to global memory */
    if ( lane_id == 0 )
    {
        b[warp_id][0] = sum[0];
        b[warp_id][1] = sum[1];
    }
}


void dual_jacobi_apply( real const * const Hdia_inv, rvec2 const * const y,
        rvec2 * const x, int n )
{
    int blocks;

    blocks = (n / DEF_BLOCK_SIZE)
        + ((n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_dual_jacobi_apply <<< blocks, DEF_BLOCK_SIZE >>>
        ( Hdia_inv, y, x, n );
    cudaCheckError( );
}


void jacobi_apply( real const * const Hdia_inv, real const * const y,
        real * const x, int n )
{
    int blocks;

    blocks = (n / DEF_BLOCK_SIZE)
        + ((n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_jacobi_apply <<< blocks, DEF_BLOCK_SIZE >>>
        ( Hdia_inv, y, x, n );
    cudaCheckError( );
}


/* Communications for sparse matrix-dense vector multiplication AX = B
 *
 * system:
 * control: 
 * mpi_data:
 * x: dense vector (device)
 * n: number of entries in x
 * buf_type: data structure type for x
 * mpi_type: MPI_Datatype struct for communications
 *
 * returns: communication time
 */
static void Dual_Sparse_MatVec_Comm_Part1( const reax_system * const system,
        const control_params * const control, storage * const workspace,
        mpi_datatypes * const mpi_data, void const * const x, int n,
        int buf_type, MPI_Datatype mpi_type )
{
    rvec2 *spad;

#if defined(CUDA_DEVICE_PACK)
    /* exploit 3D domain decomposition of simulation space with 3-stage communication pattern */
    Cuda_Dist( system, mpi_data, x, buf_type, mpi_type );
#else
    check_smalloc( &workspace->host_scratch, &workspace->host_scratch_size,
            sizeof(rvec2) * n, TRUE, SAFE_ZONE,
            "Dual_Sparse_MatVec_Comm_Part1::workspace->host_scratch" );
    spad = (rvec2 *) workspace->host_scratch;

    copy_host_device( spad, (void *) x, sizeof(rvec2) * n,
            hipMemcpyDeviceToHost, "Dual_Sparse_MatVec_Comm_Part1::x (d-to-h)" );

    /* exploit 3D domain decomposition of simulation space with 3-stage communication pattern */
    Dist( system, mpi_data, spad, buf_type, mpi_type );

    copy_host_device( spad, (void *) x, sizeof(rvec2) * n,
            hipMemcpyHostToDevice, "Dual_Sparse_MatVec_Comm_Part1::x (h-to-d)" );
#endif
}


/* Local arithmetic portion of sparse matrix-dense vector multiplication AX = B
 *
 * control:
 * A: sparse matrix, 1D partitioned row-wise
 * x: dense vector
 * b (output): dense vector
 * n: number of entries in b
 */
static void Dual_Sparse_MatVec_local( control_params const * const control,
        sparse_matrix const * const A, rvec2 const * const x,
        rvec2 * const b, int n )
{
    int blocks;

    if ( A->format == SYM_HALF_MATRIX )
    {
        /* half-format requires entries of b be initialized to zero */
        cuda_memset( b, 0, sizeof(rvec2) * n, "Dual_Sparse_MatVec_local::b" );

        /* 1 thread per row implementation */
//        k_dual_sparse_matvec_half_csr <<< control->blocks, control->block_size >>>
//            ( A->start, A->end, A->j, A->val, x, b, A->n );

        blocks = A->n * 32 / DEF_BLOCK_SIZE
            + (A->n * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);
        
        /* 32 threads per row implementation
         * using registers to accumulate partial row sums */
        k_dual_sparse_matvec_half_opt_csr <<< blocks, DEF_BLOCK_SIZE >>>
             ( A->start, A->end, A->j, A->val, x, b, A->n );
    }
    else if ( A->format == SYM_FULL_MATRIX )
    {
        /* 1 thread per row implementation */
//        k_dual_sparse_matvec_full_csr <<< control->blocks_n, control->blocks_size_n >>>
//             ( *A, x, b, A->n );

        blocks = ((A->n * 32) / DEF_BLOCK_SIZE)
            + (((A->n * 32) % DEF_BLOCK_SIZE) == 0 ? 0 : 1);
        
        /* 32 threads per row implementation
         * using registers to accumulate partial row sums */
        k_dual_sparse_matvec_full_opt_csr <<< blocks, DEF_BLOCK_SIZE >>>
                ( A->start, A->end, A->j, A->val, x, b, A->n );
    }
    cudaCheckError( );
}


/* Communications for collecting the distributed partial sums
 * in the sparse matrix-dense vector multiplication AX = B.
 * Specifically, B contains the distributed partial sums
 * (and hence has the same number of entries as X).
 *
 * system:
 * control:
 * mpi_data:
 * mat_format: storage type of sparse matrix A
 * b: dense vector (device)
 * n1: number of entries in x
 * n2: number of entries in b (at output)
 * buf_type: data structure type for b
 * mpi_type: MPI_Datatype struct for communications
 *
 * returns: communication time
 */
static void Dual_Sparse_MatVec_Comm_Part2( const reax_system * const system,
        const control_params * const control, storage * const workspace,
        mpi_datatypes * const mpi_data, int mat_format,
        void * const b, int n1, int n2, int buf_type, MPI_Datatype mpi_type )
{
    rvec2 *spad;

    /* reduction required for symmetric half matrix */
    if ( mat_format == SYM_HALF_MATRIX )
    {
#if defined(CUDA_DEVICE_PACK)
        Cuda_Coll( system, mpi_data, b, buf_type, mpi_type );
#else
        check_smalloc( &workspace->host_scratch, &workspace->host_scratch_size,
                sizeof(rvec2) * n1, TRUE, SAFE_ZONE,
                "Dual_Sparse_MatVec_Comm_Part2::workspace->host_scratch" );
        spad = (rvec2 *) workspace->host_scratch;
        copy_host_device( spad, b, sizeof(rvec2) * n1,
                hipMemcpyDeviceToHost, "Dual_Sparse_MatVec_Comm_Part2::b" );

        Coll( system, mpi_data, spad, buf_type, mpi_type );

        copy_host_device( spad, b, sizeof(rvec2) * n2,
                hipMemcpyHostToDevice, "Dual_Sparse_MatVec_Comm_Part2::b" );
#endif
    }
}


/* sparse matrix, dense vector multiplication AX = B
 *
 * system:
 * control:
 * data:
 * workspace: storage container for workspace structures
 * A: symmetric matrix,
 *    stored in CSR format
 * X: dense vector, size equal to num. columns in A
 * n: number of rows in X
 * B (output): dense vector */
static void Dual_Sparse_MatVec( const reax_system * const system,
        control_params const * const control, simulation_data * const data,
        storage * const workspace, mpi_datatypes * const mpi_data,
        sparse_matrix const * const A, rvec2 * const x,
        int n, rvec2 * const b )
{
#if defined(LOG_PERFORMANCE)
    real time;

    time = Get_Time( );
#endif

    Dual_Sparse_MatVec_Comm_Part1( system, control, workspace, mpi_data,
            x, n, RVEC2_PTR_TYPE, mpi_data->mpi_rvec2 );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_comm );
#endif

    Dual_Sparse_MatVec_local( control, A, x, b, n );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_spmv );
#endif

    Dual_Sparse_MatVec_Comm_Part2( system, control, workspace, mpi_data,
            A->format, b, n, A->n, RVEC2_PTR_TYPE, mpi_data->mpi_rvec2 );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_comm );
#endif
}


/* Communications for sparse matrix-dense vector multiplication Ax = b
 *
 * system:
 * control: 
 * mpi_data:
 * x: dense vector (device)
 * n: number of entries in x
 * buf_type: data structure type for x
 * mpi_type: MPI_Datatype struct for communications
 */
static void Sparse_MatVec_Comm_Part1( const reax_system * const system,
        const control_params * const control, storage * const workspace,
        mpi_datatypes * const mpi_data, void const * const x, int n,
        int buf_type, MPI_Datatype mpi_type )
{
    real *spad;

#if defined(CUDA_DEVICE_PACK)
    /* exploit 3D domain decomposition of simulation space with 3-stage communication pattern */
    Cuda_Dist( system, mpi_data, x, buf_type, mpi_type );
#else
    check_smalloc( &workspace->host_scratch, &workspace->host_scratch_size,
            sizeof(real) * n, TRUE, SAFE_ZONE,
            "Sparse_MatVec_Comm_Part1::workspace->host_scratch" );
    spad = (real *) workspace->host_scratch;
    copy_host_device( spad, (void *) x, sizeof(real) * n,
            hipMemcpyDeviceToHost, "Sparse_MatVec_Comm_Part1::x" );

    /* exploit 3D domain decomposition of simulation space with 3-stage communication pattern */
    Dist( system, mpi_data, spad, buf_type, mpi_type );

    copy_host_device( spad, (void *) x, sizeof(real) * n,
            hipMemcpyHostToDevice, "Sparse_MatVec_Comm_Part1::x" );
#endif
}


/* Local arithmetic portion of sparse matrix-dense vector multiplication Ax = b
 *
 * control:
 * A: sparse matrix, 1D partitioned row-wise
 * x: dense vector
 * b (output): dense vector
 * n: number of entries in b
 */
static void Sparse_MatVec_local( control_params const * const control,
        sparse_matrix const * const A, real const * const x,
        real * const b, int n )
{
    int blocks;

    if ( A->format == SYM_HALF_MATRIX )
    {
        /* half-format requires entries of b be initialized to zero */
        cuda_memset( b, 0, sizeof(real) * n, "Sparse_MatVec_local::b" );

        /* 1 thread per row implementation */
//        k_sparse_matvec_half_csr <<< control->blocks, control->block_size >>>
//            ( A->start, A->end, A->j, A->val, x, b, A->n );

        blocks = (A->n * 32 / DEF_BLOCK_SIZE)
            + (A->n * 32 % DEF_BLOCK_SIZE == 0 ? 0 : 1);

        /* 32 threads per row implementation
         * using registers to accumulate partial row sums */
        k_sparse_matvec_half_opt_csr <<< blocks, DEF_BLOCK_SIZE >>>
             ( A->start, A->end, A->j, A->val, x, b, A->n );
    }
    else if ( A->format == SYM_FULL_MATRIX )
    {
        /* 1 thread per row implementation */
//        k_sparse_matvec_full_csr <<< control->blocks, control->blocks_size >>>
//             ( A->start, A->end, A->j, A->val, x, b, A->n );

        blocks = ((A->n * 32) / DEF_BLOCK_SIZE)
            + (((A->n * 32) % DEF_BLOCK_SIZE) == 0 ? 0 : 1);

        /* 32 threads per row implementation
         * using registers to accumulate partial row sums */
        k_sparse_matvec_full_opt_csr <<< blocks, DEF_BLOCK_SIZE >>>
             ( A->start, A->end, A->j, A->val, x, b, A->n );
    }
    cudaCheckError( );
}


/* Communications for collecting the distributed partial sums
 * in the sparse matrix-dense vector multiplication Ax = b.
 * Specifically, b contains the distributed partial sums
 * (and hence has the same number of entries as x).
 *
 * system:
 * control:
 * mpi_data:
 * mat_format: storage type of sparse matrix A
 * b: dense vector (device)
 * n1: number of entries in x
 * n2: number of entries in b (at output)
 * buf_type: data structure type for b
 * mpi_type: MPI_Datatype struct for communications
 */
static void Sparse_MatVec_Comm_Part2( const reax_system * const system,
        const control_params * const control, storage * const workspace,
        mpi_datatypes * const mpi_data, int mat_format,
        void * const b, int n1, int n2, int buf_type, MPI_Datatype mpi_type )
{
    real *spad;

    /* reduction required for symmetric half matrix */
    if ( mat_format == SYM_HALF_MATRIX )
    {
#if defined(CUDA_DEVICE_PACK)
        Cuda_Coll( system, mpi_data, b, buf_type, mpi_type );
#else
        check_smalloc( &workspace->host_scratch, &workspace->host_scratch_size,
                sizeof(real) * n1, TRUE, SAFE_ZONE,
                "Sparse_MatVec_Comm_Part2::workspace->host_scratch" );
        spad = (real *) workspace->host_scratch;
        copy_host_device( spad, b, sizeof(real) * n1,
                hipMemcpyDeviceToHost, "Sparse_MatVec_Comm_Part2::b" );

        Coll( system, mpi_data, spad, buf_type, mpi_type );

        copy_host_device( spad, b, sizeof(real) * n2,
                hipMemcpyHostToDevice, "Sparse_MatVec_Comm_Part2::b" );
#endif
    }
}


/* sparse matrix, dense vector multiplication Ax = b
 *
 * system:
 * control:
 * data:
 * workspace: storage container for workspace structures
 * A: symmetric matrix,
 *    stored in CSR format
 * x: dense vector
 * n: number of entries in x
 * b (output): dense vector */
static void Sparse_MatVec( reax_system const * const system,
        control_params const * const control, simulation_data * const data,
        storage * const workspace, mpi_datatypes * const mpi_data,
        sparse_matrix const * const A, real const * const x,
        int n, real * const b )
{
#if defined(LOG_PERFORMANCE)
    real time;

    time = Get_Time( );
#endif

    Sparse_MatVec_Comm_Part1( system, control, workspace, mpi_data,
            x, n, REAL_PTR_TYPE, MPI_DOUBLE );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_comm );
#endif

    Sparse_MatVec_local( control, A, x, b, n );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_spmv );
#endif

    Sparse_MatVec_Comm_Part2( system, control, workspace, mpi_data,
            A->format, b, n, A->n, REAL_PTR_TYPE, MPI_DOUBLE );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_comm );
#endif
}


/* Preconditioned Conjugate Gradient Method
 * Note: this version is for the dual QEq solver */
int Cuda_dual_CG( reax_system const * const system,
        control_params const * const control,
        simulation_data * const data, storage * const workspace,
        sparse_matrix const * const H, rvec2 const * const b, real tol,
        rvec2 * const x, mpi_datatypes * const mpi_data )
{
    unsigned int i, matvecs;
    int ret;
    rvec2 tmp, alpha, beta, r_norm, b_norm, sig_old, sig_new;
    real redux[6];
#if defined(LOG_PERFORMANCE)
    real time;

    time = Get_Time( );
#endif

//    fprintf( stderr, "[INFO] Dual_Sparse_MatVec: p%d, i = %d\n", system->my_rank, 0 );
//    fflush( stderr );
    Dual_Sparse_MatVec( system, control, data, workspace, mpi_data,
            H, x, system->N, workspace->d_workspace->q2 );

    Vector_Sum_rvec2( workspace->d_workspace->r2, 1.0, 1.0, b,
            -1.0, -1.0, workspace->d_workspace->q2, system->n );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

    dual_jacobi_apply( workspace->d_workspace->Hdia_inv, workspace->d_workspace->r2,
            workspace->d_workspace->d2, system->n );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_pre_app );
#endif

    Dot_local_rvec2( control, workspace, workspace->d_workspace->r2,
            workspace->d_workspace->d2, system->n, &redux[0], &redux[1] );
    Dot_local_rvec2( control, workspace, workspace->d_workspace->d2,
            workspace->d_workspace->d2, system->n, &redux[2], &redux[3] );
    Dot_local_rvec2( control, workspace, b, b, system->n, &redux[4], &redux[5] );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

    ret = MPI_Allreduce( MPI_IN_PLACE, redux, 6, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD );
    Check_MPI_Error( ret, __FILE__, __LINE__ );
    sig_new[0] = redux[0];
    sig_new[1] = redux[1];
    r_norm[0] = SQRT( redux[2] );
    r_norm[1] = SQRT( redux[3] );
    b_norm[0] = SQRT( redux[4] );
    b_norm[1] = SQRT( redux[5] );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

    for ( i = 0; i < control->cm_solver_max_iters; ++i )
    {
        if ( r_norm[0] / b_norm[0] <= tol || r_norm[1] / b_norm[1] <= tol )
        {
            break;
        }

//        fprintf( stderr, "[INFO] Dual_Sparse_MatVec: p%d, i = %d\n", system->my_rank, i );
//        fflush( stderr );
        Dual_Sparse_MatVec( system, control, data, workspace, mpi_data,
                H, workspace->d_workspace->d2, system->N, workspace->d_workspace->q2 );

        Dot_local_rvec2( control, workspace, workspace->d_workspace->d2,
                workspace->d_workspace->q2, system->n, &redux[0], &redux[1] );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 2, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
        tmp[0] = redux[0];
        tmp[1] = redux[1];

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        alpha[0] = sig_new[0] / tmp[0];
        alpha[1] = sig_new[1] / tmp[1];
        Vector_Add_rvec2( x, alpha[0], alpha[1],
                workspace->d_workspace->d2, system->n );
        Vector_Add_rvec2( workspace->d_workspace->r2, -1.0 * alpha[0], -1.0 * alpha[1],
                workspace->d_workspace->q2, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        dual_jacobi_apply( workspace->d_workspace->Hdia_inv, workspace->d_workspace->r2,
                workspace->d_workspace->p2, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_pre_app );
#endif

        Dot_local_rvec2( control, workspace, workspace->d_workspace->r2,
                workspace->d_workspace->p2, system->n, &redux[0], &redux[1] );
        Dot_local_rvec2( control, workspace, workspace->d_workspace->p2,
                workspace->d_workspace->p2, system->n, &redux[2], &redux[3] );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 4, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        sig_old[0] = sig_new[0];
        sig_old[1] = sig_new[1];
        sig_new[0] = redux[0];
        sig_new[1] = redux[1];
        r_norm[0] = SQRT( redux[2] );
        r_norm[1] = SQRT( redux[3] );
        beta[0] = sig_new[0] / sig_old[0];
        beta[1] = sig_new[1] / sig_old[1];
        /* d = p + beta * d */
        Vector_Sum_rvec2( workspace->d_workspace->d2,
                1.0, 1.0, workspace->d_workspace->p2,
                beta[0], beta[1], workspace->d_workspace->d2, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif
    }

    if ( r_norm[0] / b_norm[0] <= tol
            && r_norm[1] / b_norm[1] > tol )
    {
        Vector_Copy_From_rvec2( workspace->d_workspace->t,
                workspace->d_workspace->x, 1, system->n );

        matvecs = Cuda_CG( system, control, data, workspace, H,
                workspace->d_workspace->b_t, tol,
                workspace->d_workspace->t, mpi_data );

        Vector_Copy_To_rvec2( workspace->d_workspace->x,
                workspace->d_workspace->t, 1, system->n );
    }
    else if ( r_norm[1] / b_norm[1] <= tol
            && r_norm[0] / b_norm[0] > tol )
    {
        Vector_Copy_From_rvec2( workspace->d_workspace->s,
                workspace->d_workspace->x, 0, system->n );

        matvecs = Cuda_CG( system, control, data, workspace, H,
                workspace->d_workspace->b_s, tol,
                workspace->d_workspace->s, mpi_data );

        Vector_Copy_To_rvec2( workspace->d_workspace->x,
                workspace->d_workspace->s, 0, system->n );
    }
    else
    {
        matvecs = 0;
    }

    if ( i >= control->cm_solver_max_iters )
    {
        fprintf( stderr, "[WARNING] p%d: dual CG convergence failed (%d iters)\n",
                system->my_rank, i );
        fprintf( stderr, "    [INFO] Rel. residual error for s solve: %e\n", r_norm[0] / b_norm[0] );
        fprintf( stderr, "    [INFO] Rel. residual error for t solve: %e\n", r_norm[1] / b_norm[1] );
    }

    return (i + 1) + matvecs;
}


/* Preconditioned Conjugate Gradient Method */
int Cuda_CG( reax_system const * const system, control_params const * const control,
        simulation_data * const data, storage * const workspace,
        sparse_matrix const * const H, real const * const b, real tol,
        real * const x, mpi_datatypes * const mpi_data )
{
    unsigned int i;
    int ret;
    real tmp, alpha, beta, r_norm, b_norm;
    real sig_old, sig_new;
    real redux[3];
#if defined(LOG_PERFORMANCE)
    real time;

    time = Get_Time( );
#endif

    Sparse_MatVec( system, control, data, workspace, mpi_data,
            H, x, system->N, workspace->d_workspace->q );

    Vector_Sum( workspace->d_workspace->r, 1.0, b,
            -1.0, workspace->d_workspace->q, system->n );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

    jacobi_apply( workspace->d_workspace->Hdia_inv, workspace->d_workspace->r,
            workspace->d_workspace->d, system->n );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_pre_app );
#endif

    redux[0] = Dot_local( workspace, workspace->d_workspace->r,
            workspace->d_workspace->d, system->n );
    redux[1] = Dot_local( workspace, workspace->d_workspace->d,
            workspace->d_workspace->d, system->n );
    redux[2] = Dot_local( workspace, b, b, system->n );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

    ret = MPI_Allreduce( MPI_IN_PLACE, redux, 3, MPI_DOUBLE,
            MPI_SUM, MPI_COMM_WORLD );
    Check_MPI_Error( ret, __FILE__, __LINE__ );
    sig_new = redux[0];
    r_norm = SQRT( redux[1] );
    b_norm = SQRT( redux[2] );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

    for ( i = 0; i < control->cm_solver_max_iters && r_norm / b_norm > tol; ++i )
    {
        Sparse_MatVec( system, control, data, workspace, mpi_data,
                H, workspace->d_workspace->d, system->N, workspace->d_workspace->q );

        tmp = Dot( workspace, workspace->d_workspace->d, workspace->d_workspace->q,
                system->n, MPI_COMM_WORLD );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        alpha = sig_new / tmp;
        Vector_Add( x, alpha, workspace->d_workspace->d, system->n );
        Vector_Add( workspace->d_workspace->r, -1.0 * alpha,
                workspace->d_workspace->q, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        jacobi_apply( workspace->d_workspace->Hdia_inv, workspace->d_workspace->r,
                workspace->d_workspace->p, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_pre_app );
#endif

        redux[0] = Dot_local( workspace, workspace->d_workspace->r,
                workspace->d_workspace->p, system->n );
        redux[1] = Dot_local( workspace, workspace->d_workspace->p,
                workspace->d_workspace->p, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 2, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        sig_old = sig_new;
        sig_new = redux[0];
        r_norm = SQRT( redux[1] );
        beta = sig_new / sig_old;
        Vector_Sum( workspace->d_workspace->d, 1.0, workspace->d_workspace->p,
                beta, workspace->d_workspace->d, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif
    }

    if ( i >= control->cm_solver_max_iters )
    {
        fprintf( stderr, "[WARNING] p%d: CG convergence failed (%d iters)\n",
                system->my_rank, i );
        fprintf( stderr, "    [INFO] Rel. residual error: %e\n", r_norm / b_norm );
    }

    return i;
}


/* Bi-conjugate gradient stabalized method with left preconditioning for
 * solving nonsymmetric linear systems
 * Note: this version is for the dual QEq solver
 *
 * system: 
 * workspace: struct containing storage for workspace for the linear solver
 * control: struct containing parameters governing the simulation and numeric methods
 * data: struct containing simulation data (e.g., atom info)
 * H: sparse, symmetric matrix in CSR format
 * b: right-hand side of the linear system
 * tol: tolerence compared against the relative residual for determining convergence
 * x: inital guess
 * mpi_data: 
 *
 * Reference: Netlib (in MATLAB)
 *  http://www.netlib.org/templates/matlab/bicgstab.m
 * */
int Cuda_dual_BiCGStab( reax_system const * const system, control_params const * const control,
        simulation_data * const data, storage * const workspace,
        sparse_matrix const * const H, rvec2 const * const b, real tol,
        rvec2 * const x, mpi_datatypes * const mpi_data )
{
    unsigned int i, matvecs;
    int ret;
    rvec2 tmp, alpha, beta, omega, sigma, rho, rho_old, r_norm, b_norm;
    real redux[4];
#if defined(LOG_PERFORMANCE)
    real time;

    time = Get_Time( );
#endif

    Dual_Sparse_MatVec( system, control, data, workspace, mpi_data,
            H, x, system->N, workspace->d_workspace->d2 );

    Vector_Sum_rvec2( workspace->d_workspace->r2, 1.0, 1.0, b,
            -1.0, -1.0, workspace->d_workspace->d2, system->n );
    Dot_local_rvec2( control, workspace, b,
            b, system->n, &redux[0], &redux[1] );
    Dot_local_rvec2( control, workspace, workspace->d_workspace->r2,
            workspace->d_workspace->r2, system->n, &redux[2], &redux[3] );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

    ret = MPI_Allreduce( MPI_IN_PLACE, redux, 4, MPI_DOUBLE,
            MPI_SUM, MPI_COMM_WORLD );
    Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

    b_norm[0] = SQRT( redux[0] );
    b_norm[1] = SQRT( redux[1] );
    r_norm[0] = SQRT( redux[2] );
    r_norm[1] = SQRT( redux[3] );
    if ( b_norm[0] == 0.0 )
    {
        b_norm[0] = 1.0;
    }
    if ( b_norm[1] == 0.0 )
    {
        b_norm[1] = 1.0;
    }
    Vector_Copy_rvec2( workspace->d_workspace->r_hat2,
            workspace->d_workspace->r2, system->n );
    omega[0] = 1.0;
    omega[1] = 1.0;
    rho[0] = 1.0;
    rho[1] = 1.0;

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

    for ( i = 0; i < control->cm_solver_max_iters; ++i )
    {
        if ( r_norm[0] / b_norm[0] <= tol || r_norm[1] / b_norm[1] <= tol )
        {
            break;
        }

        Dot_local_rvec2( control, workspace, workspace->d_workspace->r_hat2,
                workspace->d_workspace->r2, system->n, &redux[0], &redux[1] );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 2, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        rho[0] = redux[0];
        rho[1] = redux[1];
        if ( rho[0] == 0.0 || rho[1] == 0.0 )
        {
            break;
        }
        if ( i > 0 )
        {
            beta[0] = (rho[0] / rho_old[0]) * (alpha[0] / omega[0]);
            beta[1] = (rho[1] / rho_old[1]) * (alpha[1] / omega[1]);
            Vector_Sum_rvec2( workspace->d_workspace->q2,
                    1.0, 1.0, workspace->d_workspace->p2,
                    -1.0 * omega[0], -1.0 * omega[1], workspace->d_workspace->z2, system->n );
            Vector_Sum_rvec2( workspace->d_workspace->p2,
                    1.0, 1.0, workspace->d_workspace->r2,
                    beta[0], beta[1], workspace->d_workspace->q2, system->n );
        }
        else
        {
            Vector_Copy_rvec2( workspace->d_workspace->p2,
                    workspace->d_workspace->r2, system->n );
        }

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        dual_jacobi_apply( workspace->d_workspace->Hdia_inv, workspace->d_workspace->p2,
                workspace->d_workspace->d2, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_pre_app );
#endif

        Dual_Sparse_MatVec( system, control, data, workspace, mpi_data,
                H, workspace->d_workspace->d2, system->N, workspace->d_workspace->z2 );

        Dot_local_rvec2( control, workspace, workspace->d_workspace->r_hat2,
                workspace->d_workspace->z2, system->n, &redux[0], &redux[1] );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 2, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        tmp[0] = redux[0];
        tmp[1] = redux[1];
        alpha[0] = rho[0] / tmp[0];
        alpha[1] = rho[1] / tmp[1];
        Vector_Sum_rvec2( workspace->d_workspace->q2,
                1.0, 1.0, workspace->d_workspace->r2,
                -1.0 * alpha[0], -1.0 * alpha[1], workspace->d_workspace->z2, system->n );
        Dot_local_rvec2( control, workspace, workspace->d_workspace->q2,
                workspace->d_workspace->q2, system->n, &redux[0], &redux[1] );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 2, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        tmp[0] = redux[0];
        tmp[1] = redux[1];
        /* early convergence check */
        if ( tmp[0] < tol || tmp[1] < tol )
        {
            Vector_Add_rvec2( x, alpha[0], alpha[1], workspace->d_workspace->d2, system->n );
            break;
        }

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        dual_jacobi_apply( workspace->d_workspace->Hdia_inv, workspace->d_workspace->q2,
                workspace->d_workspace->q_hat2, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_pre_app );
#endif

        Dual_Sparse_MatVec( system, control, data, workspace, mpi_data,
                H, workspace->d_workspace->q_hat2, system->N, workspace->d_workspace->y2 );

        Dot_local_rvec2( control, workspace, workspace->d_workspace->y2,
                workspace->d_workspace->q2, system->n, &redux[0], &redux[1] );
        Dot_local_rvec2( control, workspace, workspace->d_workspace->y2,
                workspace->d_workspace->y2, system->n, &redux[2], &redux[3] );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 4, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        sigma[0] = redux[0];
        sigma[1] = redux[1];
        tmp[0] = redux[2];
        tmp[1] = redux[3];
        omega[0] = sigma[0] / tmp[0];
        omega[1] = sigma[1] / tmp[1];
        Vector_Sum_rvec2( workspace->d_workspace->g2,
                alpha[0], alpha[1], workspace->d_workspace->d2,
                omega[0], omega[1], workspace->d_workspace->q_hat2, system->n );
        Vector_Add_rvec2( x, 1.0, 1.0, workspace->d_workspace->g2, system->n );
        Vector_Sum_rvec2( workspace->d_workspace->r2,
                1.0, 1.0, workspace->d_workspace->q2,
                -1.0 * omega[0], -1.0 * omega[1], workspace->d_workspace->y2, system->n );
        Dot_local_rvec2( control, workspace, workspace->d_workspace->r2,
                workspace->d_workspace->r2, system->n, &redux[0], &redux[1] );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 2, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        r_norm[0] = SQRT( redux[0] );
        r_norm[1] = SQRT( redux[1] );
        if ( omega[0] == 0.0 || omega[1] == 0.0 )
        {
            break;
        }
        rho_old[0] = rho[0];
        rho_old[1] = rho[1];

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif
    }

    if ( r_norm[0] / b_norm[0] <= tol
            && r_norm[1] / b_norm[1] > tol )
    {
        Vector_Copy_From_rvec2( workspace->d_workspace->t,
                workspace->d_workspace->x, 1, system->n );

        matvecs = Cuda_BiCGStab( system, control, data, workspace, H,
                workspace->d_workspace->b_t, tol,
                workspace->d_workspace->t, mpi_data );

        Vector_Copy_To_rvec2( workspace->d_workspace->x,
                workspace->d_workspace->t, 1, system->n );
    }
    else if ( r_norm[1] / b_norm[1] <= tol
            && r_norm[0] / b_norm[0] > tol )
    {
        Vector_Copy_From_rvec2( workspace->d_workspace->s,
                workspace->d_workspace->x, 0, system->n );

        matvecs = Cuda_BiCGStab( system, control, data, workspace, H,
                workspace->d_workspace->b_s, tol,
                workspace->d_workspace->s, mpi_data );

        Vector_Copy_To_rvec2( workspace->d_workspace->x,
                workspace->d_workspace->s, 0, system->n );
    }
    else
    {
        matvecs = 0;
    }

    if ( i >= control->cm_solver_max_iters )
    {
        fprintf( stderr, "[WARNING] p%d: dual BiCGStab convergence failed (%d iters)\n",
                system->my_rank, i );
        fprintf( stderr, "    [INFO] Rel. residual error for s solve: %e\n", r_norm[0] / b_norm[0] );
        fprintf( stderr, "    [INFO] Rel. residual error for t solve: %e\n", r_norm[1] / b_norm[1] );
    }

    return (i + 1) + matvecs;
}


/* Bi-conjugate gradient stabalized method with left preconditioning for
 * solving nonsymmetric linear systems
 *
 * system: 
 * workspace: struct containing storage for workspace for the linear solver
 * control: struct containing parameters governing the simulation and numeric methods
 * data: struct containing simulation data (e.g., atom info)
 * H: sparse, symmetric matrix in CSR format
 * b: right-hand side of the linear system
 * tol: tolerence compared against the relative residual for determining convergence
 * x: inital guess
 * mpi_data: 
 *
 * Reference: Netlib (in MATLAB)
 *  http://www.netlib.org/templates/matlab/bicgstab.m
 * */
int Cuda_BiCGStab( reax_system const * const system, control_params const * const control,
        simulation_data * const data, storage * const workspace,
        sparse_matrix const * const H, real const * const b, real tol,
        real * const x, mpi_datatypes * const mpi_data )
{
    unsigned int i;
    int ret;
    real tmp, alpha, beta, omega, sigma, rho, rho_old, r_norm, b_norm;
    real redux[2];
#if defined(LOG_PERFORMANCE)
    real time;

    time = Get_Time( );
#endif

    Sparse_MatVec( system, control, data, workspace, mpi_data,
            H, x, system->N, workspace->d_workspace->d );

    Vector_Sum( workspace->d_workspace->r, 1.0, b,
            -1.0, workspace->d_workspace->d, system->n );
    redux[0] = Dot_local( workspace, b, b, system->n );
    redux[1] = Dot_local( workspace, workspace->d_workspace->r,
            workspace->d_workspace->r, system->n );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

    ret = MPI_Allreduce( MPI_IN_PLACE, redux, 2, MPI_DOUBLE,
            MPI_SUM, MPI_COMM_WORLD );
    Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

    b_norm = SQRT( redux[0] );
    r_norm = SQRT( redux[1] );
    if ( b_norm == 0.0 )
    {
        b_norm = 1.0;
    }
    Vector_Copy( workspace->d_workspace->r_hat,
            workspace->d_workspace->r, system->n );
    omega = 1.0;
    rho = 1.0;

#if defined(LOG_PERFORMANCE)
    Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

    for ( i = 0; i < control->cm_solver_max_iters && r_norm / b_norm > tol; ++i )
    {
        redux[0] = Dot_local( workspace, workspace->d_workspace->r_hat,
                workspace->d_workspace->r, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 1, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        rho = redux[0];
        if ( rho == 0.0 )
        {
            break;
        }
        if ( i > 0 )
        {
            beta = (rho / rho_old) * (alpha / omega);
            Vector_Sum( workspace->d_workspace->q,
                    1.0, workspace->d_workspace->p,
                    -1.0 * omega, workspace->d_workspace->z, system->n );
            Vector_Sum( workspace->d_workspace->p,
                    1.0, workspace->d_workspace->r,
                    beta, workspace->d_workspace->q, system->n );
        }
        else
        {
            Vector_Copy( workspace->d_workspace->p,
                    workspace->d_workspace->r, system->n );
        }

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        jacobi_apply( workspace->d_workspace->Hdia_inv, workspace->d_workspace->p,
                workspace->d_workspace->d, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_pre_app );
#endif

        Sparse_MatVec( system, control, data, workspace, mpi_data,
                H, workspace->d_workspace->d, system->N, workspace->d_workspace->z );

        redux[0] = Dot_local( workspace, workspace->d_workspace->r_hat,
                workspace->d_workspace->z, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 1, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        tmp = redux[0];
        alpha = rho / tmp;
        Vector_Sum( workspace->d_workspace->q,
                1.0, workspace->d_workspace->r,
                -1.0 * alpha, workspace->d_workspace->z, system->n );
        redux[0] = Dot_local( workspace, workspace->d_workspace->q,
                workspace->d_workspace->q, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 1, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        tmp = redux[0];
        /* early convergence check */
        if ( tmp < tol )
        {
            Vector_Add( x, alpha, workspace->d_workspace->d, system->n );
            break;
        }

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        jacobi_apply( workspace->d_workspace->Hdia_inv, workspace->d_workspace->q,
                workspace->d_workspace->q_hat, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_pre_app );
#endif

        Sparse_MatVec( system, control, data, workspace, mpi_data,
                H, workspace->d_workspace->q_hat, system->N, workspace->d_workspace->y );

        redux[0] = Dot_local( workspace, workspace->d_workspace->y,
                workspace->d_workspace->q, system->n );
        redux[1] = Dot_local( workspace, workspace->d_workspace->y,
                workspace->d_workspace->y, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 2, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        sigma = redux[0];
        tmp = redux[1];
        omega = sigma / tmp;
        Vector_Sum( workspace->d_workspace->g,
                alpha, workspace->d_workspace->d,
                omega, workspace->d_workspace->q_hat, system->n );
        Vector_Add( x, 1.0, workspace->d_workspace->g, system->n );
        Vector_Sum( workspace->d_workspace->r,
                1.0, workspace->d_workspace->q,
                -1.0 * omega, workspace->d_workspace->y, system->n );
        redux[0] = Dot_local( workspace, workspace->d_workspace->r,
                workspace->d_workspace->r, system->n );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif

        ret = MPI_Allreduce( MPI_IN_PLACE, redux, 1, MPI_DOUBLE,
                MPI_SUM, MPI_COMM_WORLD );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_allreduce );
#endif

        r_norm = SQRT( redux[0] );
        if ( omega == 0.0 )
        {
            break;
        }
        rho_old = rho;

#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm_solver_vector_ops );
#endif
    }

    if ( i >= control->cm_solver_max_iters )
    {
        fprintf( stderr, "[WARNING] p%d: BiCGStab convergence failed (%d iters)\n",
                system->my_rank, i );
        fprintf( stderr, "    [INFO] Rel. residual error: %e\n", r_norm / b_norm );
    }

    return i;
}

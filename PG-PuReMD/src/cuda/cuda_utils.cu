#include "cuda_utils.h"


void cuda_malloc( void **ptr, size_t size, int mem_set, const char *msg )
{

    hipError_t retVal = hipSuccess;

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "[INFO] requesting %zu bytes for %s\n",
            size, msg );
    fflush( stderr );
#endif

    retVal = hipMalloc( ptr, size );

    if ( retVal != hipSuccess )
    {
        fprintf( stderr, "[ERROR] failed to allocate memory on device for resouce %s\n", msg );
        fprintf( stderr, "    [INFO] CUDA API error code: %d, requested memory size (in bytes): %lu\n", 
                retVal, size );
        exit( INSUFFICIENT_MEMORY );
    }  

#if defined(DEBUG_FOCUS)
    fprintf( stderr, "[INFO] granted memory at address: %p\n", *ptr );
    fflush( stderr );
#endif

    if ( mem_set == TRUE )
    {
        retVal = hipMemset( *ptr, 0, size );

        if( retVal != hipSuccess )
        {
            fprintf( stderr, "[ERROR] failed to memset memory on device for resource %s\n", msg );
            fprintf( stderr, "    [INFO] CUDA API error code: %d, requested memory size (in bytes): %lu\n", 
                    retVal, size );
            exit( INSUFFICIENT_MEMORY );
        }
    }  
}


void cuda_free( void *ptr, const char *msg )
{

    hipError_t retVal = hipSuccess;

    if ( !ptr )
    {
        return;
    }  

    retVal = hipFree( ptr );

    if( retVal != hipSuccess )
    {
        fprintf( stderr, "[WARNING] failed to release memory on device for resource %s\n",
                msg );
        fprintf( stderr, "    [INFO] CUDA API error code: %d, memory address: %ld\n", 
                retVal, (long int) ptr );
        return;
    }  
}


void cuda_memset( void *ptr, int data, size_t count, const char *msg )
{
    hipError_t retVal = hipSuccess;

    retVal = hipMemset( ptr, data, count );

    if( retVal != hipSuccess )
    {
        fprintf( stderr, "[ERROR] failed to memset memory on device for resource %s\n", msg );
        fprintf( stderr, "    [INFO] CUDA API error code: %d\n", retVal );
        exit( RUNTIME_ERROR );
    }
}


/* Checks if the amount of space currently allocated to ptr is sufficient,
 * and, if not, frees any space allocated to ptr before allocating the
 * requested amount of space */
void cuda_check_malloc( void **ptr, size_t *cur_size, size_t new_size, const char *msg )
{
#if defined(DEBUG_FOCUS)
    fprintf( stderr, "[INFO] requesting %zu bytes for %s (%zu currently allocated)\n",
            new_size, msg, *cur_size );
    fflush( stderr );
#endif

    assert( new_size > 0 );

    if ( new_size > *cur_size )
    {
        if ( *cur_size > 0 || *ptr != NULL )
        {
            cuda_free( *ptr, msg );
        }

        //TODO: look into using aligned alloc's
        /* intentionally over-allocate to reduce the number of allocation operations,
         * and record the new allocation size */
        *cur_size = (size_t) CEIL( new_size * SAFE_ZONE );
        cuda_malloc( ptr, *cur_size, 0, msg );
    }
}


void copy_host_device( void *host, void *dev, size_t size,
        hipMemcpyKind dir, const char *msg )
{
    hipError_t retVal = hipErrorNotReady;

    if ( dir == hipMemcpyHostToDevice )
    {
        retVal = hipMemcpy( dev, host, size, hipMemcpyHostToDevice );
    }
    else
    {
        retVal = hipMemcpy( host, dev, size, hipMemcpyDeviceToHost );
    }

    if ( retVal != hipSuccess )
    {
        fprintf( stderr,
                "[ERROR] could not copy resource %s from host to device\n    [INFO] CUDA API error code: %d\n",
                msg, retVal );
        exit( INSUFFICIENT_MEMORY );
    }
}


void copy_device( void *dest, void *src, size_t size, const char *msg )
{
    hipError_t retVal;

    retVal = hipMemcpy( dest, src, size, hipMemcpyDeviceToDevice );

    if ( retVal != hipSuccess )
    {
        fprintf( stderr,
                "[ERROR] could not copy resource %s from device to device\n    [INFO] CUDA API error code: %d\n",
                msg, retVal );
        exit( INSUFFICIENT_MEMORY );
    }
}


void Cuda_Print_Mem_Usage( )
{
    size_t total, free;
    hipError_t retVal;

    retVal = hipMemGetInfo( &free, &total );

    if ( retVal != hipSuccess )
    {
        fprintf( stderr,
                "[WARNING] could not get message usage info from device\n"
                "    [INFO] CUDA API error code: %d\n",
                retVal );
        return;
    }

    fprintf( stderr, "Total: %zu bytes (%7.2f MB)\nFree %zu bytes (%7.2f MB)\n", 
            total, (long long int) total / (1024.0 * 1024.0),
            free, (long long int) free / (1024.0 * 1024.0) );
}

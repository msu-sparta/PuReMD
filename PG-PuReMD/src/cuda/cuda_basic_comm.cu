
#include "cuda_basic_comm.h"

#include "cuda_utils.h"

#include "../comm_tools.h"
#include "../tool_box.h"
#include "../vector.h"


typedef void (*cuda_dist_packer)( void const * const, mpi_out_data * const,
        int, hipStream_t );
typedef void (*cuda_coll_unpacker)( void const * const, void * const,
        mpi_out_data * const, int, hipStream_t );


/* copy integer entries from buffer to MPI egress buffer
 *
 * arguments:
 *  src: buffer containing data to be copied
 *  dest: MPI egress buffer
 *  index: indices for buffer to be copied into the MPI egress buffer
 *  k: number of entries in buffer
 */
GPU_GLOBAL void k_int_packer( int const * const src, int * const dest,
        int const * const index, int k )
{
    unsigned int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= k )
    {
        return;
    }

    dest[i] = src[index[i]];
}


/* copy double precision entries from buffer to MPI egress buffer
 *
 * arguments:
 *  src: buffer containing data to be copied
 *  dest: MPI egress buffer
 *  index: indices for buffer to be copied into the MPI egress buffer
 *  k: number of entries in buffer
 */
GPU_GLOBAL void k_real_packer( real const * const src, real * const dest,
        int const * const index, int k )
{
    unsigned int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= k )
    {
        return;
    }

    dest[i] = src[index[i]];
}


/* copy rvec entries from buffer to MPI egress buffer
 *
 * arguments:
 *  src: buffer containing data to be copied
 *  dest: MPI egress buffer
 *  index: indices for buffer to be copied into the MPI egress buffer
 *  k: number of entries in buffer
 */
GPU_GLOBAL void k_rvec_packer( rvec const * const src, rvec * const dest,
        int const * const index, int k )
{
    unsigned int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= k )
    {
        return;
    }

    rvec_Copy( dest[i], src[index[i]] );
}


/* copy rvec2 entries from buffer to MPI egress buffer
 *
 * arguments:
 *  src: buffer containing data to be copied
 *  dest: MPI egress buffer
 *  index: indices for buffer to be copied into the MPI egress buffer
 *  k: number of entries in buffer
 */
GPU_GLOBAL void k_rvec2_packer( rvec2 const * const src, rvec2 * const dest,
        int const * const index, int k )
{
    unsigned int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= k )
    {
        return;
    }

    dest[i][0] = src[index[i]][0];
    dest[i][1] = src[index[i]][1];
}


/* copy integer entries from MPI ingress buffer to buffer
 *
 * arguments:
 *  src: MPI ingress buffer containing data to be copied
 *  dest: buffer to be copied into
 *  index: indices for buffer to be copied into
 *  k: number of entries in buffer
 */
GPU_GLOBAL void k_int_unpacker( int const * const src, int * const dest,
        int const * const index, int k )
{
    unsigned int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= k )
    {
        return;
    }

    dest[index[i]] = src[i];
}


/* copy double precision entries from MPI ingress buffer to buffer
 *
 * arguments:
 *  src: MPI ingress buffer containing data to be copied
 *  dest: buffer to be copied into
 *  index: indices for buffer to be copied into
 *  k: number of entries in buffer
 */
GPU_GLOBAL void k_real_unpacker( real const * const src, real * const dest,
        int const * const index, int k )
{
    unsigned int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= k )
    {
        return;
    }

    dest[index[i]] += src[i];
}


/* copy rvec entries from MPI ingress buffer to buffer
 *
 * arguments:
 *  src: MPI ingress buffer containing data to be copied
 *  dest: buffer to be copied into
 *  index: indices for buffer to be copied into
 *  k: number of entries in buffer
 */
GPU_GLOBAL void k_rvec_unpacker( rvec const * const src, rvec * const dest,
        int const * const index, int k )
{
    unsigned int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= k )
    {
        return;
    }

    rvec_Add( dest[index[i]], src[i] );
}


/* copy rvec2 entries from MPI ingress buffer to buffer
 *
 * arguments:
 *  src: MPI ingress buffer containing data to be copied
 *  dest: buffer to be copied into
 *  index: indices for buffer to be copied into
 *  k: number of entries in buffer
 */
GPU_GLOBAL void k_rvec2_unpacker( rvec2 const * const src, rvec2 * const dest,
        int const * const index, int k )
{
    unsigned int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= k )
    {
        return;
    }

    dest[index[i]][0] += src[i][0];
    dest[index[i]][1] += src[i][1];
}


static void int_packer( void const * const src, mpi_out_data * const out_buf,
        int block_size, hipStream_t s )
{
    int blocks;

    blocks = (out_buf->cnt / block_size)
        + ((out_buf->cnt % block_size == 0) ? 0 : 1);

    k_int_packer <<< blocks, block_size, 0, s >>>
        ( (int *) src, (int *) out_buf->out_atoms, out_buf->index, out_buf->cnt );
    cudaCheckError( );

    hipStreamSynchronize( s );
}


static void real_packer( void const * const src, mpi_out_data * const out_buf,
        int block_size, hipStream_t s )
{
    int blocks;

    blocks = (out_buf->cnt / block_size)
        + ((out_buf->cnt % block_size == 0) ? 0 : 1);

    k_real_packer <<< blocks, block_size, 0, s >>>
        ( (real *) src, (real *) out_buf->out_atoms, out_buf->index, out_buf->cnt );
    cudaCheckError( );

    hipStreamSynchronize( s );
}


static void rvec_packer( void const * const src, mpi_out_data * const out_buf,
        int block_size, hipStream_t s )
{
    int blocks;

    blocks = (out_buf->cnt / block_size)
        + ((out_buf->cnt % block_size == 0) ? 0 : 1);

    k_rvec_packer <<< blocks, block_size, 0, s >>>
        ( (rvec *) src, (rvec *) out_buf->out_atoms, out_buf->index, out_buf->cnt );
    cudaCheckError( );

    hipStreamSynchronize( s );
}


static void rvec2_packer( void const * const src, mpi_out_data * const out_buf,
        int block_size, hipStream_t s )
{
    int blocks;

    blocks = (out_buf->cnt / block_size)
        + ((out_buf->cnt % block_size == 0) ? 0 : 1);

    k_rvec2_packer <<< blocks, block_size, 0, s >>>
        ( (rvec2 *) src, (rvec2 *) out_buf->out_atoms, out_buf->index, out_buf->cnt );
    cudaCheckError( );

    hipStreamSynchronize( s );
}


static void int_unpacker( void const * const dummy_in, void * const dummy_buf,
        mpi_out_data * const out_buf, int block_size, hipStream_t s )
{
    int blocks;

    blocks = (out_buf->cnt / block_size)
        + ((out_buf->cnt % block_size == 0) ? 0 : 1);

    k_int_unpacker <<< blocks, block_size, 0, s >>>
        ( (int *) dummy_in, (int *) dummy_buf, out_buf->index, out_buf->cnt );
    cudaCheckError( );

    hipStreamSynchronize( s );
}


static void real_unpacker( void const * const dummy_in, void * const dummy_buf,
        mpi_out_data * const out_buf, int block_size, hipStream_t s )
{
    int blocks;

    blocks = (out_buf->cnt / block_size)
        + ((out_buf->cnt % block_size == 0) ? 0 : 1);

    k_real_unpacker <<< blocks, block_size, 0, s >>>
        ( (real *) dummy_in, (real *) dummy_buf, out_buf->index, out_buf->cnt );
    cudaCheckError( );

    hipStreamSynchronize( s );
}


static void rvec_unpacker( void const * const dummy_in, void * const dummy_buf,
        mpi_out_data * const out_buf, int block_size, hipStream_t s )
{
    int blocks;

    blocks = (out_buf->cnt / block_size)
        + ((out_buf->cnt % block_size == 0) ? 0 : 1);

    k_rvec_unpacker <<< blocks, block_size, 0, s >>>
        ( (rvec *) dummy_in, (rvec *) dummy_buf, out_buf->index, out_buf->cnt );
    cudaCheckError( );

    hipStreamSynchronize( s );
}


static void rvec2_unpacker( void const * const dummy_in, void * const dummy_buf,
        mpi_out_data * const out_buf, int block_size, hipStream_t s )
{
    int blocks;

    blocks = (out_buf->cnt / block_size)
        + ((out_buf->cnt % block_size == 0) ? 0 : 1);

    k_rvec2_unpacker <<< blocks, block_size, 0, s >>>
        ( (rvec2 *) dummy_in, (rvec2 *) dummy_buf, out_buf->index, out_buf->cnt );
    cudaCheckError( );

    hipStreamSynchronize( s );
}


static void * Get_Buffer_Offset( void const * const buffer,
        int offset, int type )
{
    void * ptr;

    switch ( type )
    {
        case INT_PTR_TYPE:
            ptr = &((int *) buffer)[offset];
            break;

        case REAL_PTR_TYPE:
            ptr = &((real *) buffer)[offset];
            break;

        case RVEC_PTR_TYPE:
            ptr = &((rvec *) buffer)[offset];
            break;

        case RVEC2_PTR_TYPE:
            ptr = &((rvec2 *) buffer)[offset];
            break;

        default:
            fprintf( stderr, "[ERROR] unknown pointer type. Terminating...\n" );
            exit( UNKNOWN_OPTION );
            break;
    }

    return ptr;
}


static cuda_dist_packer Get_Packer( int type )
{
    cuda_dist_packer func_ptr;

    switch ( type )
    {
        case INT_PTR_TYPE:
            func_ptr = &int_packer;
            break;

        case REAL_PTR_TYPE:
            func_ptr = &real_packer;
            break;

        case RVEC_PTR_TYPE:
            func_ptr = &rvec_packer;
            break;

        case RVEC2_PTR_TYPE:
            func_ptr = &rvec2_packer;
            break;

        default:
            fprintf( stderr, "[ERROR] unknown pointer type. Terminating...\n" );
            exit( UNKNOWN_OPTION );
            break;
    }

    return func_ptr;
}


static cuda_coll_unpacker Get_Unpacker( int type )
{
    cuda_coll_unpacker func_ptr;

    switch ( type )
    {
        case INT_PTR_TYPE:
            func_ptr = &int_unpacker;
            break;

        case REAL_PTR_TYPE:
            func_ptr = &real_unpacker;
            break;

        case RVEC_PTR_TYPE:
            func_ptr = &rvec_unpacker;
            break;

        case RVEC2_PTR_TYPE:
            func_ptr = &rvec2_unpacker;
            break;

        default:
            fprintf( stderr, "[ERROR] unknown pointer type. Terminating...\n" );
            exit( UNKNOWN_OPTION );
            break;
    }

    return func_ptr;
}


void Cuda_Dist( reax_system const * const system, storage * const workspace,
        mpi_datatypes * const mpi_data, void const * const buf,
        int buf_type, MPI_Datatype type, int block_size, hipStream_t s )
{
    int d, cnt1, cnt2, ret;
    mpi_out_data *out_bufs;
    MPI_Comm comm;
    MPI_Request req1, req2;
    MPI_Status stat1, stat2;
    const neighbor_proc *nbr1, *nbr2;
    cuda_dist_packer pack;
    MPI_Aint extent, lower_bound;
    size_t type_size;

    ret = MPI_Type_get_extent( type, &lower_bound, &extent );
    Check_MPI_Error( ret, __FILE__, __LINE__ );
//    type_size = MPI_Aint_add( lower_bound, extent );
    type_size = extent;

    comm = mpi_data->comm_mesh3D;
    out_bufs = mpi_data->d_out_buffers;
    pack = Get_Packer( buf_type );

    for ( d = 0; d < 3; ++d )
    {
        nbr1 = &system->my_nbrs[2 * d];
        nbr2 = &system->my_nbrs[2 * d + 1];

        /* pack MPI buffers and initiate sends */
        sCudaCheckMalloc( &out_bufs[2 * d].out_atoms,
                &out_bufs[2 * d].out_atoms_size,
                type_size * out_bufs[2 * d].cnt, __FILE__, __LINE__ );
	if ( out_bufs[2 * d].index_size < sizeof(int) * out_bufs[2 * d].cnt )
	{
            sCudaCheckMalloc( &workspace->d_workspace->scratch[3],
                    &workspace->d_workspace->scratch_size[3],
                    out_bufs[2 * d].index_size, __FILE__, __LINE__ );

            sCudaMemcpyAsync( workspace->d_workspace->scratch[3],
                    out_bufs[2 * d].index, out_bufs[2 * d].index_size,
                    hipMemcpyDeviceToDevice, s, __FILE__, __LINE__ );
            sCudaFree( out_bufs[2 * d].index, __FILE__, __LINE__ );
            sCudaMalloc( (void **) &out_bufs[2 * d].index,
                    (size_t) CEIL( (sizeof(int) * out_bufs[2 * d].cnt) * SAFE_ZONE ),
                    __FILE__, __LINE__ );
            sCudaMemcpyAsync( out_bufs[2 * d].index, workspace->d_workspace->scratch[3],
                    out_bufs[2 * d].index_size,
                    hipMemcpyDeviceToDevice, s, __FILE__, __LINE__ );

            out_bufs[2 * d].index_size = (size_t) CEIL( (sizeof(int) * out_bufs[2 * d].cnt) * SAFE_ZONE );
	}

        pack( buf, &out_bufs[2 * d], block_size, s );

        ret = MPI_Isend( out_bufs[2 * d].out_atoms, out_bufs[2 * d].cnt,
                type, nbr1->rank, 2 * d, comm, &req1 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        sCudaCheckMalloc( &out_bufs[2 * d + 1].out_atoms,
                &out_bufs[2 * d + 1].out_atoms_size,
                type_size * out_bufs[2 * d + 1].cnt, __FILE__, __LINE__ );
	if ( out_bufs[2 * d + 1].index_size < sizeof(int) * out_bufs[2 * d + 1].cnt )
	{
            sCudaCheckMalloc( &workspace->d_workspace->scratch[3],
                    &workspace->d_workspace->scratch_size[3],
                    out_bufs[2 * d + 1].index_size, __FILE__, __LINE__ );

            sCudaMemcpyAsync( workspace->d_workspace->scratch[3], out_bufs[2 * d + 1].index,
                    out_bufs[2 * d + 1].index_size,
                    hipMemcpyDeviceToDevice, s, __FILE__, __LINE__ );
            sCudaFree( out_bufs[2 * d + 1].index, __FILE__, __LINE__ );
            sCudaMalloc( (void **) &out_bufs[2 * d + 1].index,
                    (size_t) CEIL( (sizeof(int) * out_bufs[2 * d + 1].cnt) * SAFE_ZONE ),
                    __FILE__, __LINE__ );
            sCudaMemcpyAsync( out_bufs[2 * d + 1].index, workspace->d_workspace->scratch[3],
                    out_bufs[2 * d + 1].index_size,
                    hipMemcpyDeviceToDevice, s, __FILE__, __LINE__ );

            out_bufs[2 * d + 1].index_size = (size_t) CEIL( (sizeof(int) * out_bufs[2 * d + 1].cnt) * SAFE_ZONE );
	}

        pack( buf, &out_bufs[2 * d + 1], block_size, s );

        ret = MPI_Isend( out_bufs[2 * d + 1].out_atoms, out_bufs[2 * d + 1].cnt,
                type, nbr2->rank, 2 * d + 1, comm, &req2 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        /* recv both messages in dimension d */
        ret = MPI_Probe( nbr1->rank, 2 * d + 1, comm, &stat1 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
        ret = MPI_Get_count( &stat1, type, &cnt1 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        if ( cnt1 == MPI_UNDEFINED )
        {
            fprintf( stderr, "[ERROR] MPI_Get_count returned MPI_UNDEFINED\n" );
            MPI_Abort( MPI_COMM_WORLD, RUNTIME_ERROR );
        }
        else if ( cnt1 + nbr1->atoms_str > system->total_cap )
        {
            fprintf( stderr, "[ERROR] Cuda_Dist: not enough space in recv buffer for nbr1 (dim = %d)\n", d );
            MPI_Abort( MPI_COMM_WORLD, RUNTIME_ERROR );
        }

        ret = MPI_Recv( Get_Buffer_Offset( buf, nbr1->atoms_str, buf_type ),
                cnt1, type, nbr1->rank, 2 * d + 1, comm, MPI_STATUS_IGNORE );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        ret = MPI_Probe( nbr2->rank, 2 * d, comm, &stat2 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
        ret = MPI_Get_count( &stat2, type, &cnt2 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        if ( cnt2 == MPI_UNDEFINED )
        {
            fprintf( stderr, "[ERROR] MPI_Get_count returned MPI_UNDEFINED\n" );
            MPI_Abort( MPI_COMM_WORLD, RUNTIME_ERROR );
        }
        else if ( cnt2 + nbr2->atoms_str > system->total_cap )
        {
            fprintf( stderr, "[ERROR] Cuda_Dist: not enough space in recv buffer for nbr2 (dim = %d)\n", d );
            MPI_Abort( MPI_COMM_WORLD, RUNTIME_ERROR );
        }

        ret = MPI_Recv( Get_Buffer_Offset( buf, nbr2->atoms_str, buf_type ),
                cnt2, type, nbr2->rank, 2 * d, comm, MPI_STATUS_IGNORE );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        ret = MPI_Wait( &req1, MPI_STATUS_IGNORE );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
        ret = MPI_Wait( &req2, MPI_STATUS_IGNORE );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
    }
}


void Cuda_Coll( reax_system const * const system, mpi_datatypes * const mpi_data,
        void * const buf, int buf_type, MPI_Datatype type, int block_size, hipStream_t s )
{   
    int d, cnt1, cnt2, ret;
    mpi_out_data *out_bufs;
    MPI_Comm comm;
    MPI_Request req1, req2;
    MPI_Status stat1, stat2;
    const neighbor_proc *nbr1, *nbr2;
    cuda_coll_unpacker unpack;
    MPI_Aint extent, lower_bound;
    size_t type_size;

    ret = MPI_Type_get_extent( type, &lower_bound, &extent );
    Check_MPI_Error( ret, __FILE__, __LINE__ );
//    type_size = MPI_Aint_add( lower_bound, extent );
    type_size = extent;

    comm = mpi_data->comm_mesh3D;
    out_bufs = mpi_data->d_out_buffers;
    unpack = Get_Unpacker( buf_type );

    for ( d = 2; d >= 0; --d )
    {
        nbr1 = &system->my_nbrs[2 * d];
        nbr2 = &system->my_nbrs[2 * d + 1];
        
        /* send both messages in dimension d */
        ret = MPI_Isend( Get_Buffer_Offset( buf, nbr1->atoms_str, buf_type ),
                nbr1->atoms_cnt, type, nbr1->rank, 2 * d, comm, &req1 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
    
        ret = MPI_Isend( Get_Buffer_Offset( buf, nbr2->atoms_str, buf_type ),
                nbr2->atoms_cnt, type, nbr2->rank, 2 * d + 1, comm, &req2 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        /* recvs and unpack messages */
        ret = MPI_Probe( nbr1->rank, 2 * d + 1, comm, &stat1 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
        ret = MPI_Get_count( &stat1, type, &cnt1 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        if ( cnt1 == MPI_UNDEFINED )
        {
            fprintf( stderr, "[ERROR] MPI_Get_count returned MPI_UNDEFINED\n" );
            MPI_Abort( MPI_COMM_WORLD, RUNTIME_ERROR );
        }

        sCudaCheckMalloc( &mpi_data->d_in1_buffer, &mpi_data->d_in1_buffer_size,
                type_size * cnt1, __FILE__, __LINE__ );

        ret = MPI_Recv( mpi_data->d_in1_buffer, cnt1,
                type, nbr1->rank, 2 * d + 1, comm, MPI_STATUS_IGNORE );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        ret = MPI_Probe( nbr2->rank, 2 * d, comm, &stat2 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
        ret = MPI_Get_count( &stat2, type, &cnt2 );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        if ( cnt2 == MPI_UNDEFINED )
        {
            fprintf( stderr, "[ERROR] MPI_Get_count returned MPI_UNDEFINED\n" );
            MPI_Abort( MPI_COMM_WORLD, RUNTIME_ERROR );
        }

        sCudaCheckMalloc( &mpi_data->d_in2_buffer, &mpi_data->d_in2_buffer_size,
                type_size * cnt2, __FILE__, __LINE__ );

        ret = MPI_Recv( mpi_data->d_in2_buffer, cnt2,
                type, nbr2->rank, 2 * d, comm, MPI_STATUS_IGNORE );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        ret = MPI_Wait( &req1, MPI_STATUS_IGNORE );
        Check_MPI_Error( ret, __FILE__, __LINE__ );
        ret = MPI_Wait( &req2, MPI_STATUS_IGNORE );
        Check_MPI_Error( ret, __FILE__, __LINE__ );

        unpack( mpi_data->d_in1_buffer, buf, &out_bufs[2 * d], block_size, s );
        unpack( mpi_data->d_in2_buffer, buf, &out_bufs[2 * d + 1], block_size, s );
    }
}

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD - Purdue ReaxFF Molecular Dynamics Program

  Copyright (2010) Purdue University
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Joseph Fogarty, jcfogart@mail.usf.edu
  Sagar Pandit, pandit@usf.edu
  Ananth Y Grama, ayg@cs.purdue.edu

  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "hip_bonds.h"

#include "hip_list.h"
#include "hip_helpers.h"
#include "hip_reduction.h"
#include "hip_utils.h"

#include "../index_utils.h"

#include <hipcub/warp/warp_reduce.hpp>


GPU_GLOBAL void k_bonds( reax_atom *my_atoms, global_parameters gp, 
        single_body_parameters *sbp, two_body_parameters *tbp, 
        storage p_workspace, reax_list p_bond_list, int n, int num_atom_types, 
        real *e_bond_g )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    real pow_BOs_be2, exp_be12, CEbo, e_bond_;
    real gp3, gp4, gp7, gp10;
    real exphu, exphua1, exphub1, exphuov, hulpov;
    real decobdbo, decobdboua, decobdboub;
    real CdDelta_i;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    bond_order_data *bo_ij;
    reax_list *bond_list;
    storage *workspace;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    bond_list = &p_bond_list;
    workspace = &p_workspace;
    gp3 = gp.l[3];
    gp4 = gp.l[4];
    gp7 = gp.l[7];
    gp10 = gp.l[10];
    e_bond_ = 0.0;
    CdDelta_i = 0.0;

    start_i = Start_Index( i, bond_list );
    end_i = End_Index( i, bond_list );

    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = bond_list->bond_list[pj].nbr;

        if ( my_atoms[i].orig_id <= my_atoms[j].orig_id )
        {
            type_i = my_atoms[i].type;
            type_j = my_atoms[j].type;
            sbp_i = &sbp[type_i];
            sbp_j = &sbp[type_j];
            twbp = &tbp[ index_tbp(type_i,type_j, num_atom_types) ];
            bo_ij = &bond_list->bond_list[pj].bo_data;

            pow_BOs_be2 = POW( bo_ij->BO_s, twbp->p_be2 );
            exp_be12 = EXP( twbp->p_be1 * ( 1.0 - pow_BOs_be2 ) );
            CEbo = -twbp->De_s * exp_be12
                * (1.0 - twbp->p_be1 * twbp->p_be2 * pow_BOs_be2);

            /* calculate bond energy */
            e_bond_ += -twbp->De_s * bo_ij->BO_s * exp_be12
                - twbp->De_p * bo_ij->BO_pi
                - twbp->De_pp * bo_ij->BO_pi2;

            /* calculate derivatives of bond orders */
            atomicAdd( &bo_ij->Cdbo, CEbo );
            atomicAdd( &bo_ij->Cdbopi, -1.0 * (CEbo + twbp->De_p) );
            atomicAdd( &bo_ij->Cdbopi2, -1.0 * (CEbo + twbp->De_pp) );

            /* Stabilisation terminal triple bond */
            if ( bo_ij->BO >= 1.00 )
            {
                if ( (Hip_strncmp( sbp_i->name, "C", sizeof(sbp_i->name) ) == 0
                            && Hip_strncmp( sbp_j->name, "O", sizeof(sbp_j->name) ) == 0)
                        || (Hip_strncmp( sbp_i->name, "O", sizeof(sbp_i->name) ) == 0
                            && Hip_strncmp( sbp_j->name, "C", sizeof(sbp_j->name) ) == 0) )
                {
                    //ba = SQR( bo_ij->BO - 2.5 );
                    exphu = EXP( -gp7 * SQR(bo_ij->BO - 2.5) );
                    //oboa = abo(j1) - boa;
                    //obob = abo(j2) - boa;
                    exphua1 = EXP(-gp3 * (workspace->total_bond_order[i] - bo_ij->BO));
                    exphub1 = EXP(-gp3 * (workspace->total_bond_order[j] - bo_ij->BO));
                    //ovoab = abo(j1) - aval(it1) + abo(j2) - aval(it2);
                    exphuov = EXP(gp4 * (workspace->Delta[i] + workspace->Delta[j]));
                    hulpov = 1.0 / (1.0 + 25.0 * exphuov);

                    e_bond_ += gp10 * exphu * hulpov * (exphua1 + exphub1);

                    decobdbo = gp10 * exphu * hulpov * (exphua1 + exphub1)
                        * ( gp3 - 2.0 * gp7 * (bo_ij->BO - 2.5) );
                    decobdboua = -gp10 * exphu * hulpov
                        * (gp3 * exphua1 + 25.0 * gp4 * exphuov * hulpov * (exphua1 + exphub1));
                    decobdboub = -gp10 * exphu * hulpov
                        * (gp3 * exphub1 + 25.0 * gp4 * exphuov * hulpov * (exphua1 + exphub1));

                    atomicAdd( &bo_ij->Cdbo, decobdbo );
                    CdDelta_i += decobdboua;
                    atomicAdd( &workspace->CdDelta[j], decobdboub );
                }
            }
        }
    }

    atomicAdd( &workspace->CdDelta[i], CdDelta_i );

#if !defined(GPU_ACCUM_ATOMIC)
    e_bond_g[i] = e_bond_;
#else
    atomicAdd( (double *) e_bond_g, (double) e_bond_ );
#endif
}


GPU_GLOBAL void k_bonds_opt( reax_atom *my_atoms, global_parameters gp, 
        single_body_parameters *sbp, two_body_parameters *tbp, 
        storage p_workspace, reax_list p_bond_list, int n, int num_atom_types, 
        real *e_bond_g )
{
    extern __shared__ hipcub::WarpReduce<double>::TempStorage temp_d[];
    int i, j, pj, thread_id, warp_id, lane_id, itr;;
    int start_i, end_i;
    int type_i, type_j;
    real pow_BOs_be2, exp_be12, CEbo, e_bond_;
    real gp3, gp4, gp7, gp10;
    real exphu, exphua1, exphub1, exphuov, hulpov;
    real decobdbo, decobdboua, decobdboub;
    real CdDelta_i;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    bond_order_data *bo_ij;
    reax_list *bond_list;
    storage *workspace;

    thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    /* all threads within a warp are assigned the interactions
     * for a unique atom */
    i = thread_id / warpSize;

    if ( i >= n )
    {
        return;
    }

    warp_id = threadIdx.x / warpSize;
    lane_id = thread_id % warpSize;
    bond_list = &p_bond_list;
    workspace = &p_workspace;
    gp3 = gp.l[3];
    gp4 = gp.l[4];
    gp7 = gp.l[7];
    gp10 = gp.l[10];
    e_bond_ = 0.0;
    CdDelta_i = 0.0;

    start_i = Start_Index( i, bond_list );
    end_i = End_Index( i, bond_list );

    for ( itr = 0, pj = start_i + lane_id; itr < (end_i - start_i + warpSize - 1) / warpSize; ++itr )
    {
        if ( pj < end_i )
        {
            j = bond_list->bond_list[pj].nbr;

            if ( my_atoms[i].orig_id <= my_atoms[j].orig_id )
            {
                type_i = my_atoms[i].type;
                type_j = my_atoms[j].type;
                sbp_i = &sbp[type_i];
                sbp_j = &sbp[type_j];
                twbp = &tbp[ index_tbp(type_i,type_j, num_atom_types) ];
                bo_ij = &bond_list->bond_list[pj].bo_data;

                pow_BOs_be2 = POW( bo_ij->BO_s, twbp->p_be2 );
                exp_be12 = EXP( twbp->p_be1 * ( 1.0 - pow_BOs_be2 ) );
                CEbo = -twbp->De_s * exp_be12
                    * (1.0 - twbp->p_be1 * twbp->p_be2 * pow_BOs_be2);

                /* calculate bond energy */
                e_bond_ += -twbp->De_s * bo_ij->BO_s * exp_be12
                    - twbp->De_p * bo_ij->BO_pi
                    - twbp->De_pp * bo_ij->BO_pi2;

                /* calculate derivatives of bond orders */
                atomicAdd( &bo_ij->Cdbo, CEbo );
                atomicAdd( &bo_ij->Cdbopi, -1.0 * (CEbo + twbp->De_p) );
                atomicAdd( &bo_ij->Cdbopi2, -1.0 * (CEbo + twbp->De_pp) );

                /* Stabilisation terminal triple bond */
                if ( bo_ij->BO >= 1.00 )
                {
                    if ( (Hip_strncmp( sbp_i->name, "C", sizeof(sbp_i->name) ) == 0
                                && Hip_strncmp( sbp_j->name, "O", sizeof(sbp_j->name) ) == 0)
                            || (Hip_strncmp( sbp_i->name, "O", sizeof(sbp_i->name) ) == 0
                                && Hip_strncmp( sbp_j->name, "C", sizeof(sbp_j->name) ) == 0) )
                    {
                        //ba = SQR( bo_ij->BO - 2.5 );
                        exphu = EXP( -gp7 * SQR(bo_ij->BO - 2.5) );
                        //oboa = abo(j1) - boa;
                        //obob = abo(j2) - boa;
                        exphua1 = EXP(-gp3 * (workspace->total_bond_order[i] - bo_ij->BO));
                        exphub1 = EXP(-gp3 * (workspace->total_bond_order[j] - bo_ij->BO));
                        //ovoab = abo(j1) - aval(it1) + abo(j2) - aval(it2);
                        exphuov = EXP(gp4 * (workspace->Delta[i] + workspace->Delta[j]));
                        hulpov = 1.0 / (1.0 + 25.0 * exphuov);

                        e_bond_ += gp10 * exphu * hulpov * (exphua1 + exphub1);

                        decobdbo = gp10 * exphu * hulpov * (exphua1 + exphub1)
                            * ( gp3 - 2.0 * gp7 * (bo_ij->BO - 2.5) );
                        decobdboua = -gp10 * exphu * hulpov
                            * (gp3 * exphua1 + 25.0 * gp4 * exphuov * hulpov * (exphua1 + exphub1));
                        decobdboub = -gp10 * exphu * hulpov
                            * (gp3 * exphub1 + 25.0 * gp4 * exphuov * hulpov * (exphua1 + exphub1));

                        atomicAdd( &bo_ij->Cdbo, decobdbo );
                        CdDelta_i += decobdboua;
                        atomicAdd( &workspace->CdDelta[j], decobdboub );
                    }
                }
            }
        }

        pj += warpSize;
    }

    CdDelta_i = hipcub::WarpReduce<double>(temp_d[warp_id]).Sum(CdDelta_i);
    e_bond_ = hipcub::WarpReduce<double>(temp_d[warp_id]).Sum(e_bond_);

    if ( lane_id == 0 )
    {
        atomicAdd( &workspace->CdDelta[i], CdDelta_i );

#if !defined(GPU_ACCUM_ATOMIC)
        e_bond_g[i] = e_bond_;
#else
        atomicAdd( (double *) e_bond_g, (double) e_bond_ );
#endif
    }
}


void Hip_Compute_Bonds( reax_system const * const system,
        control_params const * const control, simulation_data * const data,
        storage * const workspace, reax_list **lists,
        output_controls const * const out_control )
{
#if !defined(GPU_ACCUM_ATOMIC)
    int update_energy;
    real *spad;
#endif

#if defined(LOG_PERFORMANCE)
    hipEventRecord( control->hip_time_events[TE_BONDS_START], control->hip_streams[1] );
#endif

#if !defined(GPU_ACCUM_ATOMIC)
    sHipCheckMalloc( &workspace->scratch[1], &workspace->scratch_size[1],
            sizeof(real) * system->n, __FILE__, __LINE__ );

    spad = (real *) workspace->scratch[1];
    update_energy = (out_control->energy_update_freq > 0
            && data->step % out_control->energy_update_freq == 0) ? TRUE : FALSE;
#else
    sHipMemsetAsync( &data->d_my_en->e_bond,
            0, sizeof(real), control->hip_streams[1], __FILE__, __LINE__ );
#endif

    hipStreamWaitEvent( control->hip_streams[1], control->hip_stream_events[SE_BOND_ORDER_DONE], 0 );

//    k_bonds <<< control->blocks_n, control->gpu_block_size, 0, control->hip_streams[1] >>>
//        ( system->d_my_atoms, system->reax_param.d_gp,
//          system->reax_param.d_sbp, system->reax_param.d_tbp,
//          *(workspace->d_workspace), *(lists[BONDS]), 
//          system->n, system->reax_param.num_atom_types,
//#if !defined(GPU_ACCUM_ATOMIC)
//          spad
//#else
//          &data->d_my_en->e_bond
//#endif
//        );
//    hipCheckError( );

    k_bonds_opt <<< control->blocks_warp_n, control->gpu_block_size,
                sizeof(hipcub::WarpReduce<double>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                control->hip_streams[1] >>>
        ( system->d_my_atoms, system->reax_param.d_gp,
          system->reax_param.d_sbp, system->reax_param.d_tbp,
          *(workspace->d_workspace), *(lists[BONDS]), 
          system->n, system->reax_param.num_atom_types,
#if !defined(GPU_ACCUM_ATOMIC)
          spad
#else
          &data->d_my_en->e_bond
#endif
        );
    hipCheckError( );

#if !defined(GPU_ACCUM_ATOMIC)
    if ( update_energy == TRUE )
    {
        Hip_Reduction_Sum( spad, &data->d_my_en->e_bond,
                system->n, 1, control->hip_streams[1] );
    }
#endif

#if defined(LOG_PERFORMANCE)
    hipEventRecord( control->hip_time_events[TE_BONDS_STOP], control->hip_streams[1] );
#endif
}

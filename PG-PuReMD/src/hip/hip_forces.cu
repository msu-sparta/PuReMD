#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "hip_forces.h"

#include "hip_bonds.h"
#include "hip_bond_orders.h"
#include "hip_charges.h"
#include "hip_helpers.h"
#include "hip_hydrogen_bonds.h"
#include "hip_list.h"
#include "hip_multi_body.h"
#include "hip_neighbors.h"
#include "hip_nonbonded.h"
#include "hip_reduction.h"
#include "hip_spar_lin_alg.h"
#include "hip_torsion_angles.h"
#include "hip_utils.h"
#include "hip_valence_angles.h"

#include "../basic_comm.h"
#include "../forces.h"
#include "../index_utils.h"
#include "../tool_box.h"
#include "../vector.h"

#include <hipcub/util_ptx.hpp>
#include <hipcub/warp/warp_reduce.hpp>
#include <hipcub/warp/warp_scan.hpp>


#define FULL_WARP_MASK (0xFFFFFFFF)


typedef enum
{
    DIAGONAL = 0,
    OFF_DIAGONAL = 1,
} MATRIX_ENTRY_POSITION;


GPU_DEVICE real Init_Charge_Matrix_Entry_Tab( LR_lookup_table const * const t_LR, real r_ij,
        int ti, int tj, int num_atom_types )
{
    int r, tmin, tmax;
    real val, dif, base;

    tmin = MIN( ti, tj );
    tmax = MAX( ti, tj );
    LR_lookup_table const * const t = &t_LR[ index_lr(tmin,tmax, num_atom_types) ];

    /* cubic spline interpolation */
    r = (int)(r_ij * t->inv_dx);
    if ( r == 0 )
    {
        ++r;
    }
    base = (real)(r + 1) * t->dx;
    dif = r_ij - base;
    val = ((t->ele[r].d * dif + t->ele[r].c) * dif + t->ele[r].b) * dif
        + t->ele[r].a;
    val *= EV_to_KCALpMOL / C_ELE;

    return val;
}


GPU_GLOBAL void k_init_end_index( int const * const intr_cnt,
        int const * const indices, int * const end_indices, int N )
{
    int i;

    i = blockIdx.x * blockDim.x  + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    end_indices[i] = indices[i] + intr_cnt[i];
}


GPU_GLOBAL void k_init_hbond_indices( reax_atom * const atoms,
        single_body_parameters const * const sbp,
        int const * const hbonds, int const * const max_hbonds,
        int * const indices, int * const end_indices, int N )
{
    int i, hindex, flag;

    i = blockIdx.x * blockDim.x  + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    hindex = atoms[i].Hindex;

    flag = (sbp[atoms[i].type].p_hbond == H_ATOM
            || sbp[atoms[i].type].p_hbond == H_BONDING_ATOM ? TRUE : FALSE);

    indices[hindex] = (flag == TRUE ? max_hbonds[i] : 0);
    end_indices[hindex] = (flag == TRUE ? indices[hindex] + hbonds[i] : 0);
    atoms[i].num_hbonds = (flag == TRUE ? hbonds[i] : 0);
}


GPU_GLOBAL void k_print_hbond_info( reax_atom *my_atoms, single_body_parameters *sbp, 
        control_params *control, reax_list hbond_list, int N )
{
    int i;
    int type_i;
    single_body_parameters *sbp_i;
    reax_atom *atom_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    atom_i = &my_atoms[i];
    type_i = atom_i->type;
    sbp_i = &sbp[type_i];

    printf( "atom %6d: ihb = %2d, ihb_top = %2d\n", i, sbp_i->p_hbond,
            Start_Index( atom_i->Hindex, &hbond_list ) );
}


/* 1 thread computes the distances and displacement vectors of an atom for its neighbors
 * in the far neighbors list if it's a NOT re-neighboring step
 */
GPU_GLOBAL void k_init_dist( reax_atom const * const my_atoms,
        reax_list far_nbr_list, int N )
{
    int i, j, pj, start_i, end_i;
    rvec x_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    rvec_Copy( x_i, my_atoms[i].x );

    /* update distance and displacement vector between atoms i and j (i-j) */
    for ( pj = start_i; pj < end_i; ++pj )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];

        far_nbr_list.far_nbr_list.dvec[pj][0] = my_atoms[j].x[0] - x_i[0];
        far_nbr_list.far_nbr_list.dvec[pj][1] = my_atoms[j].x[1] - x_i[1];
        far_nbr_list.far_nbr_list.dvec[pj][2] = my_atoms[j].x[2] - x_i[2];
        far_nbr_list.far_nbr_list.d[pj] = rvec_Norm( far_nbr_list.far_nbr_list.dvec[pj] );
    }
}


/* 1 warp of threads computes the distances and displacement vectors of an atom for its neighbors
 * in the far neighbors list if it's a NOT re-neighboring step
 */
GPU_GLOBAL void k_init_dist_opt( reax_atom const * const my_atoms,
        reax_list far_nbr_list, int N )
{
    int j, pj, start_i, end_i, i, lane_id;
    rvec x_i, d;

    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i >= N )
    {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize; 
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    rvec_Copy( x_i, my_atoms[i].x );

    /* update distance and displacement vector between atoms i and j (i-j) */
    for ( pj = start_i + lane_id; pj < end_i; pj += warpSize )
    {
        j = far_nbr_list.far_nbr_list.nbr[pj];

        d[0] = my_atoms[j].x[0] - x_i[0];
        d[1] = my_atoms[j].x[1] - x_i[1];
        d[2] = my_atoms[j].x[2] - x_i[2];

        far_nbr_list.far_nbr_list.d[pj] = rvec_Norm( d );
        rvec_Copy( far_nbr_list.far_nbr_list.dvec[pj], d );
    }
}


/* Compute the charge matrix entries and store the matrix in half format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
GPU_GLOBAL void k_init_cm_qeq_half_fs( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    int i, j, pj, start_i, end_i, type_i, orig_id_i;
    int cm_top, num_cm_entries;
    real tap_coef[8], tap, dr3gamij_1, dr3gamij_3, r_ij;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        type_i = my_atoms[i].type;
        orig_id_i = my_atoms[i].orig_id;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        tap_coef[0] = workspace.tap_coef[0];
        tap_coef[1] = workspace.tap_coef[1];
        tap_coef[2] = workspace.tap_coef[2];
        tap_coef[3] = workspace.tap_coef[3];
        tap_coef[4] = workspace.tap_coef[4];
        tap_coef[5] = workspace.tap_coef[5];
        tap_coef[6] = workspace.tap_coef[6];
        tap_coef[7] = workspace.tap_coef[7];

        /* diagonal entry in the matrix */
        H->j[cm_top] = i;
        H->val[cm_top] = sbp[type_i].eta;
        ++cm_top;

        for ( pj = start_i; pj < end_i; ++pj )
        {
            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];

                /* if j is a local OR ghost atom in the upper triangular region of the matrix */
                if ( orig_id_i < my_atoms[j].orig_id )
                {
                    r_ij = far_nbr_list.far_nbr_list.d[pj];

                    H->j[cm_top] = j;

                    tap = tap_coef[7] * r_ij + tap_coef[6];
                    tap = tap * r_ij + tap_coef[5];
                    tap = tap * r_ij + tap_coef[4];
                    tap = tap * r_ij + tap_coef[3];
                    tap = tap * r_ij + tap_coef[2];
                    tap = tap * r_ij + tap_coef[1];
                    tap = tap * r_ij + tap_coef[0];    

                    /* shielding */
                    dr3gamij_1 = r_ij * r_ij * r_ij
                        + tbp[index_tbp(type_i, my_atoms[j].type, num_atom_types)].gamma;
                    dr3gamij_3 = CBRT( dr3gamij_1 );

                    /* i == j: periodic self-interaction term
                     * i != j: general interaction term */
                    H->val[cm_top] = ((i == j) ? 0.5 : 1.0) * tap * EV_to_KCALpMOL / dr3gamij_3;

                    ++cm_top;
                }
            }
        }
    }

    __syncthreads( );

    H->end[i] = cm_top;
    num_cm_entries = cm_top - H->start[i];

    /* reallocation check */
    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


/* Compute the tabulated charge matrix entries and store the matrix in half format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
GPU_GLOBAL void k_init_cm_qeq_half_fs_tab( reax_atom * const my_atoms,
        single_body_parameters const * const sbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, LR_lookup_table const * const t_LR, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    int cm_top;
    int num_cm_entries;
    real r_ij;
    reax_atom *atom_i, *atom_j;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        atom_i = &my_atoms[i];
        type_i = atom_i->type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry in the matrix */
        H->j[cm_top] = i;
        H->val[cm_top] = sbp[type_i].eta;
        ++cm_top;

        for ( pj = start_i; pj < end_i; ++pj )
        {
            j = far_nbr_list.far_nbr_list.nbr[pj];

            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                atom_j = &my_atoms[j];
                type_j = atom_j->type;

                /* if j is a local OR ghost atom in the upper triangular region of the matrix */
                if ( atom_i->orig_id < atom_j->orig_id )
                {
                    r_ij = far_nbr_list.far_nbr_list.d[pj];

                    H->j[cm_top] = j;
                    H->val[cm_top] = Init_Charge_Matrix_Entry_Tab( t_LR, r_ij,
                            type_i, type_j, num_atom_types );
                    ++cm_top;
                }
            }
        }
    }

    __syncthreads( );

    H->end[i] = cm_top;
    num_cm_entries = cm_top - H->start[i];

    /* reallocation check */
    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


/* Compute the charge matrix entries and store the matrix in full format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
GPU_GLOBAL void k_init_cm_qeq_full_fs( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    int i, j, pj, start_i, end_i, type_i;
    int cm_top, num_cm_entries;
    real tap_coef[8], tap, dr3gamij_1, dr3gamij_3, r_ij;
    reax_atom *atom_i;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        atom_i = &my_atoms[i];
        type_i = atom_i->type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        tap_coef[0] = workspace.tap_coef[0];
        tap_coef[1] = workspace.tap_coef[1];
        tap_coef[2] = workspace.tap_coef[2];
        tap_coef[3] = workspace.tap_coef[3];
        tap_coef[4] = workspace.tap_coef[4];
        tap_coef[5] = workspace.tap_coef[5];
        tap_coef[6] = workspace.tap_coef[6];
        tap_coef[7] = workspace.tap_coef[7];

        /* diagonal entry in the matrix */
        H->j[cm_top] = i;
        H->val[cm_top] = sbp[type_i].eta;
        ++cm_top;

        for ( pj = start_i; pj < end_i; ++pj )
        {
            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];

                H->j[cm_top] = j;

                r_ij = far_nbr_list.far_nbr_list.d[pj];
                tap = tap_coef[7] * r_ij + tap_coef[6];
                tap = tap * r_ij + tap_coef[5];
                tap = tap * r_ij + tap_coef[4];
                tap = tap * r_ij + tap_coef[3];
                tap = tap * r_ij + tap_coef[2];
                tap = tap * r_ij + tap_coef[1];
                tap = tap * r_ij + tap_coef[0];    

                /* shielding */
                dr3gamij_1 = r_ij * r_ij * r_ij
                    + tbp[index_tbp(type_i, my_atoms[j].type, num_atom_types)].gamma;
                dr3gamij_3 = CBRT( dr3gamij_1 );

                /* i == j: periodic self-interaction term
                 * i != j: general interaction term */
                H->val[cm_top] = ((i == j) ? 0.5 : 1.0) * tap * EV_to_KCALpMOL / dr3gamij_3;

                ++cm_top;
            }
        }
    }

    __syncthreads( );

    H->end[i] = cm_top;
    num_cm_entries = cm_top - H->start[i];

    /* reallocation check */
    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


/* Compute the charge matrix entries for QEq and store the matrix in full format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
GPU_GLOBAL void k_init_cm_qeq_full_fs_opt( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    extern __shared__ hipcub::WarpScan<int>::TempStorage temp1[];
    int i, j, pj, lane_id, itr;
    int start_i, end_i, type_i;
    int cm_top, num_cm_entries, offset, flag;
    real tap_coef[8], tap, dr3gamij_1, dr3gamij_3, r_ij;
    reax_atom *atom_i;
    sparse_matrix *H;

    /* all threads within a warp are assigned the same unique row 
     * in the charge matrix */
    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize;
    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        atom_i = &my_atoms[i];
        type_i = atom_i->type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        tap_coef[0] = workspace.tap_coef[0];
        tap_coef[1] = workspace.tap_coef[1];
        tap_coef[2] = workspace.tap_coef[2];
        tap_coef[3] = workspace.tap_coef[3];
        tap_coef[4] = workspace.tap_coef[4];
        tap_coef[5] = workspace.tap_coef[5];
        tap_coef[6] = workspace.tap_coef[6];
        tap_coef[7] = workspace.tap_coef[7];

        /* diagonal entry in the matrix */
        if ( lane_id == 0 )
        {
            H->j[cm_top] = i;
            H->val[cm_top] = sbp[type_i].eta; 
        }
        ++cm_top;

        for ( itr = 0, pj = start_i + lane_id; itr < (end_i - start_i + warpSize - 1) / warpSize; ++itr )
        {
            offset = (pj < end_i && far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut) ? 1 : 0;
            flag = (offset == 1) ? TRUE : FALSE;
            hipcub::WarpScan<int>(temp1[threadIdx.x / warpSize]).ExclusiveSum(offset, offset);

            if ( flag == TRUE )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];

                H->j[cm_top + offset] = j;

                r_ij = far_nbr_list.far_nbr_list.d[pj];
                tap = tap_coef[7] * r_ij + tap_coef[6];
                tap = tap * r_ij + tap_coef[5];
                tap = tap * r_ij + tap_coef[4];
                tap = tap * r_ij + tap_coef[3];
                tap = tap * r_ij + tap_coef[2];
                tap = tap * r_ij + tap_coef[1];
                tap = tap * r_ij + tap_coef[0];    

                /* shielding */
                dr3gamij_1 = r_ij * r_ij * r_ij
                    + tbp[index_tbp(type_i, my_atoms[j].type, num_atom_types)].gamma;
                dr3gamij_3 = CBRT( dr3gamij_1 );

                /* i == j: periodic self-interaction term
                 * i != j: general interaction term */
                H->val[cm_top + offset] = ((i == j) ? 0.5 : 1.0) * tap * EV_to_KCALpMOL / dr3gamij_3;
            }

            /* get cm_top from thread in last lane */
            cm_top = cm_top + offset + (flag == TRUE ? 1 : 0);
            cm_top = hipcub::ShuffleIndex<WARP_SIZE>( cm_top, warpSize - 1, FULL_WARP_MASK );

            pj += warpSize;
        }
    }

    if ( lane_id == 0 )
    {
        H->end[i] = cm_top;
        num_cm_entries = cm_top - H->start[i];

        /* reallocation check */
        if ( num_cm_entries > max_cm_entries[i] )
        {
            *realloc_cm_entries = TRUE;
        }
    }
}


/* Compute the tabulated charge matrix entries and store the matrix in full format
 * using the far neighbors list (stored in full format) and according to
 * the full shell communication method */
GPU_GLOBAL void k_init_cm_qeq_full_fs_tab( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, 
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, LR_lookup_table *t_LR, int num_atom_types,
        int * const max_cm_entries, int * const realloc_cm_entries )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i;
    int cm_top;
    int num_cm_entries;
    sparse_matrix *H;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= workspace.H.n_max )
    {
        return;
    }

    H = &workspace.H;
    cm_top = H->start[i];

    if ( i < H->n )
    {
        type_i = my_atoms[i].type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry in the matrix */
        H->j[cm_top] = i;
        H->val[cm_top] = sbp[type_i].eta;
        ++cm_top;

        for ( pj = start_i; pj < end_i; ++pj )
        {
            j = far_nbr_list.far_nbr_list.nbr[pj];

            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                H->j[cm_top] = j;
                H->val[cm_top] = Init_Charge_Matrix_Entry_Tab( t_LR,
                        far_nbr_list.far_nbr_list.d[pj],
                        type_i, my_atoms[j].type, num_atom_types );
                ++cm_top;
            }
        }
    }

    __syncthreads( );

    H->end[i] = cm_top;
    num_cm_entries = cm_top - H->start[i];

    /* reallocation check */
    if ( num_cm_entries > max_cm_entries[i] )
    {
        *realloc_cm_entries = TRUE;
    }
}


GPU_GLOBAL void k_init_bonds( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, reax_list bond_list, int n, int N,
        int num_atom_types, int * const max_bonds, int * const realloc_bonds )
{
    int i, j, pj, start_i, end_i;
    int type_i, type_j, tbp_ij;
    int btop_i, num_bonds;
    real total_bond_order_i;
    rvec dDeltap_self_i;
    real cutoff, r_ij;
    real C12, C34, C56;
    real BO_s, BO_pi, BO_pi2, BO;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    type_i = my_atoms[i].type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    btop_i = Start_Index( i, &bond_list );
    total_bond_order_i = 0.0;
    rvec_MakeZero( dDeltap_self_i );

    if ( i < n )
    {
        cutoff = MIN( control->nonb_cut, control->bond_cut );
//        workspace.bond_mark[i] = 0;
    }
    else
    {
        cutoff = control->bond_cut;
        /* put ghost atoms to an infinite distance (i.e., 1000) */
//        workspace.bond_mark[i] = 1000;
    }

    /* check if j is within cutoff */
    for ( pj = start_i; pj < end_i; ++pj )
    {
        /* uncorrected bond orders */
        if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff )
        {
            j = far_nbr_list.far_nbr_list.nbr[pj];
            type_j = my_atoms[j].type;
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(type_i, type_j, num_atom_types);

            /* uncorrected bond orders */
            if ( sbp[type_i].r_s > 0.0 && sbp[type_j].r_s > 0.0 )
            {
                C12 = tbp[tbp_ij].p_bo1 * POW( r_ij / tbp[tbp_ij].r_s, tbp[tbp_ij].p_bo2 );
                BO_s = (1.0 + control->bo_cut) * EXP( C12 );
            }
            else
            {
                C12 = 0.0;
                BO_s = 0.0;
            }

            if ( sbp[type_i].r_pi > 0.0 && sbp[type_j].r_pi > 0.0 )
            {
                C34 = tbp[tbp_ij].p_bo3 * POW( r_ij / tbp[tbp_ij].r_p, tbp[tbp_ij].p_bo4 );
                BO_pi = EXP( C34 );
            }
            else
            {
                C34 = 0.0;
                BO_pi = 0.0;
            }

            if ( sbp[type_i].r_pi_pi > 0.0 && sbp[type_j].r_pi_pi > 0.0 )
            {
                C56 = tbp[tbp_ij].p_bo5 * POW( r_ij / tbp[tbp_ij].r_pp, tbp[tbp_ij].p_bo6 );
                BO_pi2 = EXP( C56 );
            }
            else
            {
                C56 = 0.0;
                BO_pi2 = 0.0;
            }

            /* initially BO values are the uncorrected ones, page 1 */
            BO = BO_s + BO_pi + BO_pi2;

            if ( BO >= control->bo_cut )
            {
                /* compute and append bond info to list */
                Hip_Compute_BOp( bond_list, control->bo_cut, i, btop_i,
                        far_nbr_list.far_nbr_list.nbr[pj],
                        C12, C34, C56, BO_s, BO_pi, BO_pi2, BO,
                        &far_nbr_list.far_nbr_list.rel_box[pj],
                        far_nbr_list.far_nbr_list.d[pj],
                        &far_nbr_list.far_nbr_list.dvec[pj], far_nbr_list.format,
                        &tbp[tbp_ij], dDeltap_self_i, &total_bond_order_i );

                ++btop_i;

                /* TODO: future optimization if bond_mark implemented */
//                if ( workspace->bond_mark[j] > workspace->bond_mark[i] + 1 )
//                {
//                    workspace->bond_mark[j] = workspace->bond_mark[i] + 1;
//                }
//                else if ( workspace->bond_mark[i] > workspace->bond_mark[j] + 1 )
//                {
//                    workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
//                }
            }
        }
    }

    Set_End_Index( i, btop_i, &bond_list );

    num_bonds = btop_i - Start_Index( i, &bond_list );

    /* copy bond info to atom structure
     * (needed for atom ownership transfer via MPI) */
    my_atoms[i].num_bonds = num_bonds;

    workspace.total_bond_order[i] = total_bond_order_i;
    rvec_Copy( workspace.dDeltap_self[i], dDeltap_self_i );

    /* reallocation check */
    if ( num_bonds > max_bonds[i] )
    {
        *realloc_bonds = TRUE;
    }
}


GPU_GLOBAL void k_init_bonds_opt( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        storage workspace, control_params const * const control, 
        reax_list far_nbr_list, reax_list bond_list, int n, int N,
        int num_atom_types, int * const max_bonds, int * const realloc_bonds )
{
    extern __shared__ hipcub::WarpScan<int>::TempStorage temp21[];
    hipcub::WarpReduce<double>::TempStorage *temp22;
    int i, j, pj, warp_id, lane_id, itr;
    int start_i, end_i, tbp_ij;
    int type_i, type_j;
    int btop_i, offset, flag, num_bonds;
    real cutoff, r_ij;
    real C12, C34, C56;
    real BO_s, BO_pi, BO_pi2, BO;
    real total_bond_order_i;
    rvec dDeltap_self_i;

    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i >= N )
    {
        return;
    }

    warp_id = threadIdx.x / warpSize;
    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize;
    temp22 = (hipcub::WarpReduce<double>::TempStorage *) &temp21[warp_id];
    type_i = my_atoms[i].type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    btop_i = Start_Index( i, &bond_list );
    total_bond_order_i = 0.0;
    rvec_MakeZero( dDeltap_self_i );

    if ( i < n )
    {
        cutoff = MIN( control->nonb_cut, control->bond_cut );
//        workspace.bond_mark[i] = 0;
    }
    else
    {
        cutoff = control->bond_cut;
        /* put ghost atoms to an infinite distance (i.e., 1000) */
//        workspace.bond_mark[i] = 1000;
    }

    for ( itr = 0, pj = start_i + lane_id; itr < (end_i - start_i + warpSize - 1) / warpSize; ++itr )
    {
        /* uncorrected bond orders */
        if ( pj < end_i && far_nbr_list.far_nbr_list.d[pj] <= cutoff )
        {
            j = far_nbr_list.far_nbr_list.nbr[pj];
            type_j = my_atoms[j].type;
            r_ij = far_nbr_list.far_nbr_list.d[pj];
            tbp_ij = index_tbp(type_i, type_j, num_atom_types);

            /* uncorrected bond orders */
            if ( sbp[type_i].r_s > 0.0 && sbp[type_j].r_s > 0.0 )
            {
                C12 = tbp[tbp_ij].p_bo1 * POW( r_ij / tbp[tbp_ij].r_s, tbp[tbp_ij].p_bo2 );
                BO_s = (1.0 + control->bo_cut) * EXP( C12 );
            }
            else
            {
                C12 = 0.0;
                BO_s = 0.0;
            }

            if ( sbp[type_i].r_pi > 0.0 && sbp[type_j].r_pi > 0.0 )
            {
                C34 = tbp[tbp_ij].p_bo3 * POW( r_ij / tbp[tbp_ij].r_p, tbp[tbp_ij].p_bo4 );
                BO_pi = EXP( C34 );
            }
            else
            {
                C34 = 0.0;
                BO_pi = 0.0;
            }

            if ( sbp[type_i].r_pi_pi > 0.0 && sbp[type_j].r_pi_pi > 0.0 )
            {
                C56 = tbp[tbp_ij].p_bo5 * POW( r_ij / tbp[tbp_ij].r_pp, tbp[tbp_ij].p_bo6 );
                BO_pi2 = EXP( C56 );
            }
            else
            {
                C56 = 0.0;
                BO_pi2 = 0.0;
            }
        }
        else
        {
            BO_s = 0.0;
            BO_pi = 0.0;
            BO_pi2 = 0.0;
        }

        /* initially BO values are the uncorrected ones, page 1 */
        BO = BO_s + BO_pi + BO_pi2;

        offset = (pj < end_i && far_nbr_list.far_nbr_list.d[pj] <= cutoff && BO >= control->bo_cut) ? 1 : 0;
        flag = (offset == 1) ? TRUE : FALSE;
        hipcub::WarpScan<int>(temp21[warp_id]).ExclusiveSum(offset, offset);

        if ( flag == TRUE )
        {
            /* compute and append bond info to list */
            Hip_Compute_BOp( bond_list, control->bo_cut, i, btop_i + offset,
                    far_nbr_list.far_nbr_list.nbr[pj],
                    C12, C34, C56, BO_s, BO_pi, BO_pi2, BO,
                    &far_nbr_list.far_nbr_list.rel_box[pj],
                    far_nbr_list.far_nbr_list.d[pj],
                    &far_nbr_list.far_nbr_list.dvec[pj], far_nbr_list.format,
                    &tbp[tbp_ij], dDeltap_self_i, &total_bond_order_i );

            /* TODO: future optimization if bond_mark implemented */
//            if ( workspace->bond_mark[j] > workspace->bond_mark[i] + 1 )
//            {
//                workspace->bond_mark[j] = workspace->bond_mark[i] + 1;
//            }
//            else if ( workspace->bond_mark[i] > workspace->bond_mark[j] + 1 )
//            {
//                workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
//            }
        }

        /* get btop_i from thread in last lane */
        btop_i = btop_i + offset + (flag == TRUE ? 1 : 0);
        btop_i = hipcub::ShuffleIndex<WARP_SIZE>( btop_i, warpSize - 1, FULL_WARP_MASK );

        pj += warpSize;
    }

    total_bond_order_i = hipcub::WarpReduce<double>(temp22[warp_id]).Sum(total_bond_order_i);
    dDeltap_self_i[0] = hipcub::WarpReduce<double>(temp22[warp_id]).Sum(dDeltap_self_i[0]);
    dDeltap_self_i[1] = hipcub::WarpReduce<double>(temp22[warp_id]).Sum(dDeltap_self_i[1]);
    dDeltap_self_i[2] = hipcub::WarpReduce<double>(temp22[warp_id]).Sum(dDeltap_self_i[2]);

    if ( lane_id == 0 )
    {
        Set_End_Index( i, btop_i, &bond_list );

        num_bonds = btop_i - Start_Index( i, &bond_list );

        /* copy bond info to atom structure
         * (needed for atom ownership transfer via MPI) */
        my_atoms[i].num_bonds = num_bonds;

        workspace.total_bond_order[i] = total_bond_order_i;
        rvec_Copy( workspace.dDeltap_self[i], dDeltap_self_i );

        /* reallocation check */
        if ( num_bonds > max_bonds[i] )
        {
            *realloc_bonds = TRUE;
        }
    }
}


/* Construct the interaction list for hydrogen bonds */
GPU_GLOBAL void k_init_hbonds( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, control_params const * const control,
        reax_list far_nbr_list, reax_list hbond_list,
        int n, int N, int num_atom_types, int * const max_hbonds, int * const realloc_hbonds )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb, ihb_top;
    int num_hbonds;
    real cutoff;
    reax_atom *atom_i;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    atom_i = &my_atoms[i];
    type_i = atom_i->type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    ihb = sbp[type_i].p_hbond;

    cutoff = MIN( control->nonb_cut, control->hbond_cut );

    ihb_top = Start_Index( atom_i->Hindex, &hbond_list );

    if ( (i < n && ihb == H_ATOM) || ihb == H_BONDING_ATOM )
    {
        /* check if j is within cutoff */
        for ( pj = start_i; pj < end_i; ++pj )
        {
            if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                jhb = sbp[type_j].p_hbond;

                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( i >= n && j < n
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM )
                {
                    hbond_list.hbond_list[ihb_top].nbr = j;
                    hbond_list.hbond_list[ihb_top].scl = -1;
                    hbond_list.hbond_list[ihb_top].ptr = pj;

#if !defined(GPU_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );
#endif

                    ++ihb_top;
                }
                /* atom i: H atom, native
                 * atom j: H bonding atom */
                else if ( i < n
                        && ihb == H_ATOM && jhb == H_BONDING_ATOM )
                {
                    hbond_list.hbond_list[ihb_top].nbr = j;
                    hbond_list.hbond_list[ihb_top].scl = 1;
                    hbond_list.hbond_list[ihb_top].ptr = pj;

#if !defined(GPU_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );
#endif

                    ++ihb_top;
                }
                /* atom i: H bonding atom, native
                 * atom j: H atom, native */
                else if ( i < n
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                {
                    hbond_list.hbond_list[ihb_top].nbr = j;
                    hbond_list.hbond_list[ihb_top].scl = -1;
                    hbond_list.hbond_list[ihb_top].ptr = pj;

#if !defined(GPU_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top].hb_f );
#endif

                    ++ihb_top;
                }
            }
        }
    }

    Set_End_Index( atom_i->Hindex, ihb_top, &hbond_list );

    num_hbonds = ihb_top - Start_Index( atom_i->Hindex, &hbond_list );

    /* copy hbond info to atom structure
     * (needed for atom ownership transfer via MPI) */
    my_atoms[i].num_hbonds = num_hbonds;

    /* reallocation check */
    if ( num_hbonds > max_hbonds[i] )
    {
        *realloc_hbonds = TRUE;
    }
}


/* Construct the interaction list for hydrogen bonds */
GPU_GLOBAL void k_init_hbonds_opt( reax_atom * const my_atoms,
        single_body_parameters const * const sbp, control_params const * const control,
        reax_list far_nbr_list, reax_list hbond_list,
        int n, int N, int num_atom_types, int * const max_hbonds, int * const realloc_hbonds )
{
    extern __shared__ hipcub::WarpScan<int>::TempStorage temp3[];
    int i, j, pj, lane_id, itr;
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb, ihb_top, offset, flag;
    int num_hbonds;
    real cutoff;
    reax_atom *atom_i;

    /* all threads within a warp are assigned the bonds
     * for a unique atom */
    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i >= N )
    {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize;
    atom_i = &my_atoms[i];
    type_i = atom_i->type;
    start_i = Start_Index( i, &far_nbr_list );
    end_i = End_Index( i, &far_nbr_list );
    ihb = sbp[type_i].p_hbond;

    cutoff = MIN( control->nonb_cut, control->hbond_cut );

    ihb_top = Start_Index( atom_i->Hindex, &hbond_list );

    if ( (i < n && ihb == H_ATOM) || ihb == H_BONDING_ATOM )
    {
        for ( itr = 0, pj = start_i + lane_id; itr < (end_i - start_i + warpSize - 1) / warpSize; ++itr )
        {
            if ( pj < end_i )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                jhb = sbp[type_j].p_hbond;

                offset = (pj < end_i && far_nbr_list.far_nbr_list.d[pj] <= cutoff
                        && ((i >= n && j < n && ihb == H_BONDING_ATOM && jhb == H_ATOM)
                            || (i < n && ihb == H_ATOM && jhb == H_BONDING_ATOM)
                            || (i < n && ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n))) ? 1 : 0;
            }
            else
            {
                offset = 0;
            }

            flag = (offset == 1) ? TRUE : FALSE;
            hipcub::WarpScan<int>(temp3[threadIdx.x / warpSize]).ExclusiveSum(offset, offset);

            if ( flag == TRUE )
            {
                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( i >= n && j < n
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM )
                {
                    hbond_list.hbond_list[ihb_top + offset].nbr = j;
                    hbond_list.hbond_list[ihb_top + offset].scl = -1;
                    hbond_list.hbond_list[ihb_top + offset].ptr = pj;

#if !defined(GPU_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top + offset].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top + offset].hb_f );
#endif
                }
                /* atom i: H atom, native
                 * atom j: H bonding atom */
                else if ( i < n
                        && ihb == H_ATOM && jhb == H_BONDING_ATOM )
                {
                    hbond_list.hbond_list[ihb_top + offset].nbr = j;
                    hbond_list.hbond_list[ihb_top + offset].scl = 1;
                    hbond_list.hbond_list[ihb_top + offset].ptr = pj;

#if !defined(GPU_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top + offset].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top + offset].hb_f );
#endif
                }
                /* atom i: H bonding atom, native
                 * atom j: H atom, native */
                else if ( i < n
                        && ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                {
                    hbond_list.hbond_list[ihb_top + offset].nbr = j;
                    hbond_list.hbond_list[ihb_top + offset].scl = -1;
                    hbond_list.hbond_list[ihb_top + offset].ptr = pj;

#if !defined(GPU_ACCUM_ATOMIC)
                    hbond_list.hbond_list[ihb_top + offset].sym_index = -1;
                    rvec_MakeZero( hbond_list.hbond_list[ihb_top + offset].hb_f );
#endif
                }
            }

            /* get ihb_top from thread in last lane */
            ihb_top = ihb_top + offset + (flag == TRUE ? 1 : 0);
            ihb_top = hipcub::ShuffleIndex<WARP_SIZE>( ihb_top, warpSize - 1, FULL_WARP_MASK );

            pj += warpSize;
        }
    }

    if ( lane_id == 0 )
    {
        Set_End_Index( atom_i->Hindex, ihb_top, &hbond_list );

        num_hbonds = ihb_top - Start_Index( atom_i->Hindex, &hbond_list );

        /* copy hbond info to atom structure
         * (needed for atom ownership transfer via MPI) */
        my_atoms[i].num_hbonds = num_hbonds;

        /* reallocation check */
        if ( num_hbonds > max_hbonds[i] )
        {
            *realloc_hbonds = TRUE;
        }
    }
}


/* Construct the interaction list for bonds */
GPU_GLOBAL void k_estimate_storages_cm_half( reax_atom const * const my_atoms,
        control_params const * const control, reax_list far_nbr_list,
        int cm_n, int cm_n_max, int * const cm_entries, int * const max_cm_entries )
{
    int i, j, pj; 
    int start_i, end_i;
    int num_cm_entries;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= cm_n_max )
    {
        return;
    }

    num_cm_entries = 0;

    if ( i < cm_n )
    {
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry */
        ++num_cm_entries;

        for ( pj = start_i; pj < end_i; ++pj )
        { 
            j = far_nbr_list.far_nbr_list.nbr[pj];

            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut
                    && (j < cm_n || my_atoms[i].orig_id < my_atoms[j].orig_id) )
            {
                ++num_cm_entries;
            }
        }
    }

    __syncthreads( );

    cm_entries[i] = num_cm_entries;
    /* round up to the nearest multiple of warp size to ensure that reads along
     * rows can be coalesced */
    max_cm_entries[i] = MAX( ((int) CEIL( num_cm_entries * SAFE_ZONE )
                + warpSize - 1) / warpSize * warpSize, MIN_CM_ENTRIES );
}


GPU_GLOBAL void k_estimate_storages_cm_full( control_params const * const control,
        reax_list far_nbr_list, int cm_n, int cm_n_max,
        int * const cm_entries, int * const max_cm_entries )
{
    int i, pj; 
    int start_i, end_i;
    int num_cm_entries;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= cm_n_max )
    {
        return;
    }

    num_cm_entries = 0;

    if ( i < cm_n )
    {
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        /* diagonal entry */
        ++num_cm_entries;

        for ( pj = start_i; pj < end_i; ++pj )
        { 
            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                ++num_cm_entries;
            }
        }
    }

    __syncthreads( );

    cm_entries[i] = num_cm_entries;
    /* round up to the nearest multiple of warp size to ensure that reads along
     * rows can be coalesced */
    max_cm_entries[i] = MAX( ((int) CEIL( num_cm_entries * SAFE_ZONE )
                + warpSize - 1) / warpSize * warpSize, MIN_CM_ENTRIES );
}


GPU_GLOBAL void k_estimate_storages_cm_full_opt( control_params const * const control,
        reax_list far_nbr_list, int cm_n, int cm_n_max,
        int * const cm_entries, int * const max_cm_entries )
{
    extern __shared__ hipcub::WarpReduce<int>::TempStorage temp4[];
    int i, pj, start_i, end_i, lane_id, num_cm_entries;

    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i >= cm_n_max )
    {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize; 
    num_cm_entries = 0;

    if ( i < cm_n )
    {
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        for ( pj = start_i + lane_id; pj < end_i; pj += warpSize )
        { 
            if ( far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut )
            {
                ++num_cm_entries;
            }
        }

        num_cm_entries = hipcub::WarpReduce<int>(temp4[threadIdx.x / warpSize]).Sum(num_cm_entries);

        /* diagonal entry -- only matters for thread in lane 0 (add once) */
        ++num_cm_entries;
    }

    if ( lane_id == 0 )
    {
        cm_entries[i] = num_cm_entries;
        /* round up to the nearest multiple of warp size to ensure that reads along
         * rows can be coalesced */
        max_cm_entries[i] = MAX( ((int) CEIL( num_cm_entries * SAFE_ZONE )
                    + warpSize - 1) / warpSize * warpSize, MIN_CM_ENTRIES );
    }
}


GPU_GLOBAL void k_estimate_storage_bonds( reax_atom const * const my_atoms, 
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        control_params *control, reax_list far_nbr_list, 
        int num_atom_types, int n, int N, int total_cap,
        int * const bonds, int * const max_bonds )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j, tbp_ij;
    int num_bonds;
    real cutoff, r_ij; 
    real C12, C34, C56;
    real BO_s, BO_pi, BO_pi2;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= total_cap )
    {
        return;
    }

    num_bonds = 0;

    if ( i < N )
    {
        type_i = my_atoms[i].type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );

        if ( i < n )
        {
            cutoff = MIN( control->nonb_cut, control->bond_cut );
        }
        else
        {
            cutoff = control->bond_cut;
        }

        for ( pj = start_i; pj < end_i; ++pj )
        { 
            if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                r_ij = far_nbr_list.far_nbr_list.d[pj];
                tbp_ij = index_tbp(type_i ,type_j, num_atom_types);

                /* uncorrected bond orders */
                if ( sbp[type_i].r_s > 0.0 && sbp[type_j].r_s > 0.0 )
                {
                    C12 = tbp[tbp_ij].p_bo1 * POW( r_ij / tbp[tbp_ij].r_s, tbp[tbp_ij].p_bo2 );
                    BO_s = (1.0 + control->bo_cut) * EXP( C12 );
                }
                else
                {
                    C12 = 0.0;
                    BO_s = 0.0;
                }

                if ( sbp[type_i].r_pi > 0.0 && sbp[type_j].r_pi > 0.0 )
                {
                    C34 = tbp[tbp_ij].p_bo3 * POW( r_ij / tbp[tbp_ij].r_p, tbp[tbp_ij].p_bo4 );
                    BO_pi = EXP( C34 );
                }
                else
                {
                    C34 = 0.0;
                    BO_pi = 0.0;
                }

                if ( sbp[type_i].r_pi_pi > 0.0 && sbp[type_j].r_pi_pi > 0.0 )
                {
                    C56 = tbp[tbp_ij].p_bo5 * POW( r_ij / tbp[tbp_ij].r_pp, tbp[tbp_ij].p_bo6 );
                    BO_pi2= EXP( C56 );
                }
                else
                {
                    C56 = 0.0;
                    BO_pi2 = 0.0;
                }

                /* initially BO values are the uncorrected ones, page 1 */
                if ( BO_s + BO_pi + BO_pi2 >= control->bo_cut )
                {
                    ++num_bonds;
                }
            }
        }
    }

    __syncthreads( );

    bonds[i] = num_bonds;
    /* round up to the nearest multiple of warp size to ensure that reads along
     * rows can be coalesced */
    max_bonds[i] = MAX( ((int) CEIL(2 * num_bonds * SAFE_ZONE)
                + warpSize - 1) / warpSize * warpSize, MIN_BONDS );
}


GPU_GLOBAL void k_estimate_storage_bonds_opt( reax_atom const * const my_atoms, 
        single_body_parameters const * const sbp, two_body_parameters const * const tbp,
        control_params *control, reax_list far_nbr_list, 
        int num_atom_types, int n, int N, int total_cap,
        int * const bonds, int * const max_bonds )
{
    extern __shared__ hipcub::WarpReduce<int>::TempStorage temp5[];
    int i, j, pj, lane_id; 
    int start_i, end_i;
    int type_i, type_j, tbp_ij;
    int num_bonds;
    real cutoff, r_ij; 
    real C12, C34, C56;
    real BO_s, BO_pi, BO_pi2;
    real r_s, r_pi, r_pi_pi;

    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i >= total_cap )
    {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize;
    num_bonds = 0;

    if ( i < N )
    {
        type_i = my_atoms[i].type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );
        r_s = sbp[type_i].r_s;
        r_pi = sbp[type_i].r_pi;
        r_pi_pi = sbp[type_i].r_pi_pi;

        if ( i < n )
        {
            cutoff = MIN( control->nonb_cut, control->bond_cut );
        }
        else
        {
            cutoff = control->bond_cut;
        }

        for ( pj = start_i + lane_id; pj < end_i; pj += warpSize )
        { 
            if ( far_nbr_list.far_nbr_list.d[pj] <= cutoff )
            {
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                r_ij = far_nbr_list.far_nbr_list.d[pj];
                tbp_ij = index_tbp(type_i ,type_j, num_atom_types);

                /* uncorrected bond orders */
                if ( r_s > 0.0 && sbp[type_j].r_s > 0.0 )
                {
                    C12 = tbp[tbp_ij].p_bo1 * POW( r_ij / tbp[tbp_ij].r_s, tbp[tbp_ij].p_bo2 );
                    BO_s = (1.0 + control->bo_cut) * EXP( C12 );
                }
                else
                {
                    C12 = 0.0;
                    BO_s = 0.0;
                }

                if ( r_pi > 0.0 && sbp[type_j].r_pi > 0.0 )
                {
                    C34 = tbp[tbp_ij].p_bo3 * POW( r_ij / tbp[tbp_ij].r_p, tbp[tbp_ij].p_bo4 );
                    BO_pi = EXP( C34 );
                }
                else
                {
                    C34 = 0.0;
                    BO_pi = 0.0;
                }

                if ( r_pi_pi > 0.0 && sbp[type_j].r_pi_pi > 0.0 )
                {
                    C56 = tbp[tbp_ij].p_bo5 * POW( r_ij / tbp[tbp_ij].r_pp, tbp[tbp_ij].p_bo6 );
                    BO_pi2= EXP( C56 );
                }
                else
                {
                    C56 = 0.0;
                    BO_pi2 = 0.0;
                }

                /* initially BO values are the uncorrected ones, page 1 */
                if ( BO_s + BO_pi + BO_pi2 >= control->bo_cut )
                {
                    ++num_bonds;
                }
            }
        }

        num_bonds = hipcub::WarpReduce<int>(temp5[threadIdx.x / warpSize]).Sum(num_bonds);
    }

    if ( lane_id == 0 )
    {
        bonds[i] = num_bonds;
        /* round up to the nearest multiple of warp size to ensure that reads along
         * rows can be coalesced */
        max_bonds[i] = MAX( ((int) CEIL(2 * num_bonds * SAFE_ZONE)
                    + warpSize - 1) / warpSize * warpSize, MIN_BONDS );
    }
}


GPU_GLOBAL void k_estimate_storage_hbonds( reax_atom const * const my_atoms, 
        single_body_parameters const * const sbp, control_params const * const control,
        reax_list far_nbr_list, int num_atom_types, int n, int N,
        int total_cap, int * const hbonds, int * const max_hbonds )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb;
    int num_hbonds;
    real cutoff;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= total_cap )
    {
        return;
    }

    num_hbonds = 0;

    if ( i < N )
    {
        type_i = my_atoms[i].type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );
        ihb = sbp[type_i].p_hbond;

        if ( i < n )
        { 
            cutoff = control->nonb_cut;
        }   
        else
        {
            cutoff = control->bond_cut;
        } 

        if ( (i < n && ihb == H_ATOM) || ihb == H_BONDING_ATOM )
        {
            for ( pj = start_i; pj < end_i; ++pj )
            { 
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                jhb = sbp[type_j].p_hbond;

                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( i >= n && j < n && ihb == H_BONDING_ATOM && jhb == H_ATOM
                        && far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut
                        && far_nbr_list.far_nbr_list.d[pj] <= control->hbond_cut )
                {
                    ++num_hbonds;
                }
                else if ( i < n && far_nbr_list.far_nbr_list.d[pj] <= cutoff
                        && far_nbr_list.far_nbr_list.d[pj] <= control->hbond_cut )
                {
                    /* atom i: H atom, native
                     * atom j: H bonding atom */
                    if ( ihb == H_ATOM && jhb == H_BONDING_ATOM )
                    {
                        ++num_hbonds;
                    }
                    /* atom i: H bonding atom, native
                     * atom j: H atom, native */
                    else if ( ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                    {
                        ++num_hbonds;
                    }
                }
            }
        }
    }

    __syncthreads( );

    hbonds[i] = num_hbonds;
    /* round up to the nearest multiple of warp size to ensure that reads along
     * rows can be coalesced */
    max_hbonds[i] = MAX( ((int) CEIL(num_hbonds * SAFE_ZONE)
                + warpSize - 1) / warpSize * warpSize, MIN_HBONDS );
}


GPU_GLOBAL void k_estimate_storage_hbonds_opt( reax_atom const * const my_atoms, 
        single_body_parameters const * const sbp, control_params const * const control,
        reax_list far_nbr_list, int num_atom_types, int n, int N,
        int total_cap, int * const hbonds, int * const max_hbonds )
{
    extern __shared__ hipcub::WarpReduce<int>::TempStorage temp6[];
    int i, j, pj, lane_id;
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb;
    int num_hbonds;
    real cutoff;

    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i >= total_cap )
    {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize; 
    num_hbonds = 0;

    if ( i < N )
    {
        type_i = my_atoms[i].type;
        start_i = Start_Index( i, &far_nbr_list );
        end_i = End_Index( i, &far_nbr_list );
        ihb = sbp[type_i].p_hbond;

        if ( i < n )
        { 
            cutoff = control->nonb_cut;
        }   
        else
        {
            cutoff = control->bond_cut;
        } 

        if ( (i < n && ihb == H_ATOM) || ihb == H_BONDING_ATOM )
        {
            for ( pj = start_i + lane_id; pj < end_i; pj += warpSize )
            { 
                j = far_nbr_list.far_nbr_list.nbr[pj];
                type_j = my_atoms[j].type;
                jhb = sbp[type_j].p_hbond;

                /* atom i: H bonding, ghost
                 * atom j: H atom, native */
                if ( i >= n && j < n && ihb == H_BONDING_ATOM && jhb == H_ATOM
                        && far_nbr_list.far_nbr_list.d[pj] <= control->nonb_cut
                        && far_nbr_list.far_nbr_list.d[pj] <= control->hbond_cut )
                {
                    ++num_hbonds;
                }
                else if ( i < n && far_nbr_list.far_nbr_list.d[pj] <= cutoff
                        && far_nbr_list.far_nbr_list.d[pj] <= control->hbond_cut )
                {
                    /* atom i: H atom, native
                     * atom j: H bonding atom */
                    if ( ihb == H_ATOM && jhb == H_BONDING_ATOM )
                    {
                        ++num_hbonds;
                    }
                    /* atom i: H bonding atom, native
                     * atom j: H atom, native */
                    else if ( ihb == H_BONDING_ATOM && jhb == H_ATOM && j < n )
                    {
                        ++num_hbonds;
                    }
                }
            }

            num_hbonds = hipcub::WarpReduce<int>(temp6[threadIdx.x / warpSize]).Sum(num_hbonds);
        }
    }

    if ( lane_id == 0 )
    {
        hbonds[i] = num_hbonds;
        /* round up to the nearest multiple of warp size to ensure that reads along
         * rows can be coalesced */
        max_hbonds[i] = MAX( ((int) CEIL(num_hbonds * SAFE_ZONE)
                    + warpSize - 1) / warpSize * warpSize, MIN_HBONDS );
    }
}


GPU_GLOBAL void k_update_sym_dbond_indices( reax_list bond_list, int N )
{
    int i, pj, pk, nbr_ij, nbr_jk;
    bond_data *ibond, *jbond;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= N )
    {
        return;
    }

    /* i-j bonds */
    for ( pj = Start_Index(i, &bond_list); pj < End_Index(i, &bond_list); ++pj )
    {
        ibond = &bond_list.bond_list[pj];
        nbr_ij = ibond->nbr;

        /* j-k bonds */
        for ( pk = Start_Index(nbr_ij, &bond_list); pk < End_Index(nbr_ij, &bond_list); ++pk )
        {
            jbond = &bond_list.bond_list[pk];
            nbr_jk = jbond->nbr;

            if ( i == nbr_jk && i > nbr_ij )
            {
                ibond->sym_index = pk;
                jbond->sym_index = pj;
                break;
            }
        }
    }
}


GPU_GLOBAL void k_update_sym_dbond_indices_opt( reax_list bond_list, int N )
{
    int i, pj, pk, start_i, end_i, nbr_ij, nbr_jk, flag, lane_id;
    bond_data *ibond, *jbond;

    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i >= N )
    {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize; 
    start_i = Start_Index(i, &bond_list);
    end_i = End_Index(i, &bond_list);

    /* i-j bonds */
    for ( pj = start_i + lane_id; pj < end_i; pj += warpSize )
    {
        ibond = &bond_list.bond_list[pj];
        nbr_ij = ibond->nbr;
        flag = FALSE;

        /* j-k bonds */
        for ( pk = Start_Index(nbr_ij, &bond_list); pk < End_Index(nbr_ij, &bond_list); ++pk )
        {
            jbond = &bond_list.bond_list[pk];
            nbr_jk = jbond->nbr;

            if ( i == nbr_jk && i > nbr_ij )
            {
                flag = TRUE;
                break;
            }
        }

        if ( flag == TRUE )
        {
            ibond->sym_index = pk;
            jbond->sym_index = pj;
        }
    }
}


#if !defined(GPU_ACCUM_ATOMIC)
GPU_GLOBAL void k_update_sym_hbond_indices_opt( reax_atom *my_atoms,
        reax_list hbond_list, int N )
{
    int i, pj, pk;
    int nbr, nbrstart, nbrend;
    int start, end, flag, lane_id;
    hbond_data *ihbond, *jhbond;

    i = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    if ( i > N )
    {
        return;
    }

    lane_id = (blockIdx.x * blockDim.x + threadIdx.x) % warpSize; 
    start = Start_Index( my_atoms[i].Hindex, &hbond_list );
    end = End_Index( my_atoms[i].Hindex, &hbond_list );
    pj = start + lane_id;

    while ( pj < end )
    {
        ihbond = &hbond_list.hbond_list[pj];
        nbr = ihbond->nbr;
        flag = FALSE;
        nbrstart = Start_Index( my_atoms[nbr].Hindex, &hbond_list );
        nbrend = End_Index( my_atoms[nbr].Hindex, &hbond_list );

        for ( pk = nbrstart; pk < nbrend; pk++ )
        {
            jhbond = &hbond_list.hbond_list[pk];

            if ( jhbond->nbr == i )
            {
                flag = TRUE;
                break;
            }
        }

        if ( flag == TRUE )
        {
            ihbond->sym_index = pk;
            jhbond->sym_index = pj;
        }

        pj += warpSize;
    }
}
#endif


#if defined(DEBUG_FOCUS)
GPU_GLOBAL void k_print_forces( reax_atom *my_atoms, rvec *f, int n )
{
    int i; 

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    printf( "%8d: %24.15f, %24.15f, %24.15f\n",
            my_atoms[i].orig_id, f[i][0], f[i][1], f[i][2] );
}


GPU_GLOBAL void k_print_hbonds( reax_atom *my_atoms, reax_list hbond_list, int n, int rank, int step )
{
    int i, k, pj, start, end; 
    hbond_data *hbond_jk;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if ( i >= n )
    {
        return;
    }

    start = Start_Index( my_atoms[i].Hindex, &hbond_list );
    end = End_Index( my_atoms[i].Hindex, &hbond_list );

    for ( pj = start; pj < end; ++pj )
    {
        k = hbond_list.hbond_list[pj].nbr;
        hbond_jk = &hbond_list.hbond_list[pj];

#if !defined(GPU_ACCUM_ATOMIC)
        printf( "p%03d, step %05d: %8d: %8d, %24.15f, %24.15f, %24.15f\n",
                rank, step, my_atoms[i].Hindex, k,
                hbond_jk->hb_f[0],
                hbond_jk->hb_f[1],
                hbond_jk->hb_f[2] );
#else
        printf( "p%03d, step %05d: %8d: %8d\n",
                rank, step, my_atoms[i].Hindex, k );
#endif
    }
}
#endif


#if defined(DEBUG_FOCUS)
static void Print_Forces( reax_system const * const system,
        control_params const * const control )
{
    k_print_forces <<< control->blocks_n, control->gpu_block_size,
                   0, control->hip_streams[0] >>>
        ( system->d_my_atoms, workspace->d_workspace->f, system->n );
    hipCheckError( );
}


static void Print_HBonds( reax_system const * const system,
        control_params const * const control, int step )
{
    k_print_hbonds <<< control->blocks_n, control->gpu_block_size,
                   0, control->hip_streams[0] >>>
        ( system->d_my_atoms, *(lists[HBONDS]), system->n, system->my_rank, step );
    hipCheckError( );
}
#endif


/* Initialize indices for far neighbors list post reallocation
 *
 * system: atomic system info. */
void Hip_Init_Neighbor_Indices( reax_system * const system,
        control_params const * const control,
        reax_list * const far_nbr_list )

{
    int blocks;

    blocks = far_nbr_list->n / control->gpu_block_size
        + (far_nbr_list->n % control->gpu_block_size == 0 ? 0 : 1);

    /* init indices */
    Hip_Scan_Excl_Sum( system->d_max_far_nbrs, far_nbr_list->index,
            far_nbr_list->n, 0, control->hip_streams[0] );

    /* init end_indices */
    k_init_end_index <<< blocks, control->gpu_block_size,
                     0, control->hip_streams[0] >>>
        ( system->d_far_nbrs, far_nbr_list->index, far_nbr_list->end_index,
          far_nbr_list->n );
    hipCheckError( );
}


/* Initialize indices for far hydrogen bonds list post reallocation
 *
 * system: atomic system info. */
void Hip_Init_HBond_Indices( reax_system * const system, storage * const workspace,
        reax_list * const hbond_list, int block_size, hipStream_t s )
{
    int blocks, *temp;

    blocks = system->total_cap / block_size
        + (system->total_cap % block_size == 0 ? 0 : 1);

    sHipCheckMalloc( &workspace->scratch[2], &workspace->scratch_size[2],
            sizeof(int) * system->total_cap, __FILE__, __LINE__ );
    temp = (int *) workspace->scratch[2];

    /* init indices and end_indices */
    Hip_Scan_Excl_Sum( system->d_max_hbonds, temp, system->total_cap, 2, s );

    k_init_hbond_indices <<< blocks, block_size, 0, s >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, system->d_hbonds, temp, 
          hbond_list->index, hbond_list->end_index, system->total_cap );
    hipCheckError( );
}


/* Initialize indices for far bonds list post reallocation
 *
 * system: atomic system info. */
void Hip_Init_Bond_Indices( reax_system * const system, reax_list * const bond_list,
        int block_size, hipStream_t s )
{
    int blocks;

    blocks = system->total_cap / block_size + 
        (system->total_cap % block_size == 0 ? 0 : 1);

    /* init indices */
    Hip_Scan_Excl_Sum( system->d_max_bonds, bond_list->index,
            system->total_cap, 1, s );

    /* init end_indices */
    k_init_end_index <<< blocks, block_size, 0, s >>>
        ( system->d_bonds, bond_list->index, bond_list->end_index, system->total_cap );
    hipCheckError( );
}


/* Initialize indices for charge matrix post reallocation
 *
 * system: atomic system info.
 * H: charge matrix */
void Hip_Init_Sparse_Matrix_Indices( reax_system * const system, sparse_matrix * const H,
        int block_size, hipStream_t s )
{
    int blocks;

    blocks = H->n_max / block_size
        + (H->n_max % block_size == 0 ? 0 : 1);

    /* init indices */
    Hip_Scan_Excl_Sum( system->d_max_cm_entries, H->start, H->n_max, 5, s );

    //TODO: not needed for full format (Init_Forces sets H->end)
    /* init end_indices */
    k_init_end_index <<< blocks, block_size, 0, s >>>
        ( system->d_cm_entries, H->start, H->end, H->n_max );
    hipCheckError( );
}


void Hip_Estimate_Storages( reax_system * const system, control_params * const control,
        simulation_data * const data, storage * const workspace, reax_list ** const lists,
        int realloc_cm, int realloc_bonds, int realloc_hbonds, int step )
{
    int blocks;
#if defined(LOG_PERFORMANCE)
    float time_elapsed;
#endif

    if ( realloc_cm == TRUE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_CM_START], control->hip_streams[5] );
#endif

//        blocks = workspace->d_workspace->H.n_max / control->gpu_block_size
//            + (workspace->d_workspace->H.n_max % control->gpu_block_size == 0 ? 0 : 1);
        blocks = workspace->d_workspace->H.n_max * WARP_SIZE / control->gpu_block_size
            + (workspace->d_workspace->H.n_max * WARP_SIZE % control->gpu_block_size == 0 ? 0 : 1);

        if ( workspace->d_workspace->H.format == SYM_HALF_MATRIX )
        {
            k_estimate_storages_cm_half <<< blocks, control->gpu_block_size, 0,
                                        control->hip_streams[5] >>>
                ( system->d_my_atoms, (control_params *) control->d_control_params,
                  *(lists[FAR_NBRS]), workspace->d_workspace->H.n,
                  workspace->d_workspace->H.n_max,
                  system->d_cm_entries, system->d_max_cm_entries );
        }
        else
        {
//            k_estimate_storages_cm_full <<< blocks, control->gpu_block_size, 0,
//                                        control->hip_streams[5] >>>
//                ( (control_params *) control->d_control_params,
//                  *(lists[FAR_NBRS]), workspace->d_workspace->H.n,
//                  workspace->d_workspace->H.n_max,
//                  system->d_cm_entries, system->d_max_cm_entries );

            k_estimate_storages_cm_full_opt <<< blocks, control->gpu_block_size,
                                            sizeof(hipcub::WarpReduce<int>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                            control->hip_streams[5] >>>
                ( (control_params *) control->d_control_params,
                  *(lists[FAR_NBRS]), workspace->d_workspace->H.n,
                  workspace->d_workspace->H.n_max,
                  system->d_cm_entries, system->d_max_cm_entries );
        }
        hipCheckError( );

        Hip_Reduction_Sum( system->d_max_cm_entries, system->d_total_cm_entries,
                workspace->d_workspace->H.n_max, 5, control->hip_streams[5] );
        sHipMemcpyAsync( &system->total_cm_entries, system->d_total_cm_entries,
                sizeof(int), hipMemcpyDeviceToHost, control->hip_streams[5], __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_CM_STOP], control->hip_streams[5] );
#endif
    }

    if ( realloc_bonds == TRUE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_BOND_START], control->hip_streams[1] );
#endif

//        blocks = system->total_cap / control->gpu_block_size
//            + (system->total_cap % control->gpu_block_size == 0 ? 0 : 1);
        blocks = system->total_cap * WARP_SIZE / control->gpu_block_size
            + (system->total_cap * WARP_SIZE % control->gpu_block_size == 0 ? 0 : 1);

//        k_estimate_storage_bonds <<< blocks, control->gpu_block_size, 0,
//                                 control->hip_streams[1] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp, 
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
//              system->n, system->N, system->total_cap,
//              system->d_bonds, system->d_max_bonds );
        k_estimate_storage_bonds_opt <<< blocks, control->gpu_block_size,
                                     sizeof(hipcub::WarpReduce<int>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                     control->hip_streams[1] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp, 
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
              system->n, system->N, system->total_cap,
              system->d_bonds, system->d_max_bonds );
        hipCheckError( );

        Hip_Reduction_Sum( system->d_max_bonds, system->d_total_bonds,
                system->total_cap, 1, control->hip_streams[1] );
        sHipMemcpyAsync( &system->total_bonds, system->d_total_bonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->hip_streams[1], __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_BOND_STOP], control->hip_streams[1] );
#endif
    }

    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && realloc_hbonds == TRUE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_HBOND_START], control->hip_streams[2] );
#endif

//        blocks = system->total_cap / control->gpu_block_size
//            + (system->total_cap % control->gpu_block_size == 0 ? 0 : 1);
        blocks = system->total_cap * WARP_SIZE / control->gpu_block_size
            + (system->total_cap * WARP_SIZE % control->gpu_block_size == 0 ? 0 : 1);

//        k_estimate_storage_hbonds <<< blocks, control->gpu_block_size, 0,
//                                  control->hip_streams[2] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
//              system->n, system->N, system->total_cap,
//              system->d_hbonds, system->d_max_hbonds );
        k_estimate_storage_hbonds_opt <<< blocks, control->gpu_block_size,
                                      sizeof(hipcub::WarpReduce<int>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                      control->hip_streams[2] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
              system->n, system->N, system->total_cap,
              system->d_hbonds, system->d_max_hbonds );
        hipCheckError( );

        Hip_Reduction_Sum( system->d_max_hbonds, system->d_total_hbonds,
                system->total_cap, 2, control->hip_streams[2] );
        sHipMemcpyAsync( &system->total_hbonds, system->d_total_hbonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->hip_streams[2], __FILE__, __LINE__ );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_HBOND_STOP], control->hip_streams[2] );
#endif
    }

    if ( realloc_cm == TRUE )
    {
        hipStreamSynchronize( control->hip_streams[5] );

#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_CM_START],
                control->hip_time_events[TE_INIT_CM_STOP] ); 
        data->timing.init_cm += (real) (time_elapsed / 1000.0);
#endif
    }
    if ( realloc_bonds == TRUE )
    {
        hipStreamSynchronize( control->hip_streams[1] );

#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_BOND_START],
                control->hip_time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);
#endif
    }
    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && realloc_hbonds == TRUE )
    {
        hipStreamSynchronize( control->hip_streams[2] );

#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_HBOND_START],
                control->hip_time_events[TE_INIT_HBOND_STOP] ); 
        data->timing.init_hbond += (real) (time_elapsed / 1000.0);
#endif
    }
}


/* Initialize the bond list, hydrogen bond list, and charge matrix
 * data structures along with updating the pairwise distances in the
 * far neighbor (Verlet) list if required.
 *
 * NOTE: the control flow of the code follows a
 * try-compute-else-reallocate-and-retry logic which requires that
 * the initialization kernels be atomic transactions. Locks are used
 * to mark if the transaction succeeds (and thus should not be repeated).
 */
int Hip_Init_Forces( reax_system * const system, control_params * const control,
        simulation_data * const data, storage * const workspace,
        reax_list ** const lists, output_controls * const out_control ) 
{
    int renbr, blocks, ret;
    static int dist_done = FALSE, cm_done = FALSE, bonds_done = FALSE, hbonds_done = FALSE;
#if defined(LOG_PERFORMANCE)
    float time_elapsed;
#endif

    renbr = (data->step - data->prev_steps) % control->reneighbor == 0 ? TRUE : FALSE;

    /* reset reallocation flags on device */
    if ( cm_done == FALSE )
    {
        sHipMemsetAsync( system->d_realloc_cm_entries, FALSE, sizeof(int), 
                control->hip_streams[5], __FILE__, __LINE__ );
    }
    if ( bonds_done == FALSE )
    {
        sHipMemsetAsync( system->d_realloc_bonds, FALSE, sizeof(int), 
                control->hip_streams[1], __FILE__, __LINE__ );
    }
    if ( hbonds_done == FALSE )
    {
        sHipMemsetAsync( system->d_realloc_hbonds, FALSE, sizeof(int), 
                control->hip_streams[2], __FILE__, __LINE__ );
    }

    if ( renbr == FALSE && dist_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_DIST_START], control->hip_streams[0] );
#endif

//        k_init_dist <<< control->blocks_N, control->gpu_block_size,
//                    0, control->hip_streams[0] >>>
//            ( system->d_my_atoms, *(lists[FAR_NBRS]), system->N );

        k_init_dist_opt <<< control->blocks_warp_N, control->gpu_block_size,
                        0, control->hip_streams[0] >>>
            ( system->d_my_atoms, *(lists[FAR_NBRS]), system->N );
        hipCheckError( );

        hipEventRecord( control->hip_stream_events[SE_INIT_DIST_DONE], control->hip_streams[0] );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_DIST_STOP], control->hip_streams[0] );
#endif

        dist_done = TRUE;
    }

    if ( cm_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_CM_START], control->hip_streams[5] );
#endif

        blocks = workspace->d_workspace->H.n_max / control->gpu_block_size
            + (workspace->d_workspace->H.n_max % control->gpu_block_size == 0 ? 0 : 1);

        /* update num. rows in matrix for this GPU */
        workspace->d_workspace->H.n = system->n;

        Hip_Init_Sparse_Matrix_Indices( system, &workspace->d_workspace->H,
                control->gpu_block_size, control->hip_streams[5] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->hip_streams[5], control->hip_stream_events[SE_INIT_DIST_DONE], 0 );
        }

        if ( workspace->d_workspace->H.format == SYM_HALF_MATRIX )
        {
            if ( control->tabulate <= 0 )
            {
                k_init_cm_qeq_half_fs <<< blocks, control->gpu_block_size,
                                      0, control->hip_streams[5] >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp,
                      *(workspace->d_workspace), (control_params *) control->d_control_params,
                      *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
                      system->d_max_cm_entries, system->d_realloc_cm_entries );
            }
            else
            {
                k_init_cm_qeq_half_fs_tab <<< blocks, control->gpu_block_size,
                                          0, control->hip_streams[5] >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp,
                      *(workspace->d_workspace), (control_params *) control->d_control_params,
                      *(lists[FAR_NBRS]), workspace->d_LR, system->reax_param.num_atom_types,
                      system->d_max_cm_entries, system->d_realloc_cm_entries );
            }
        }
        else
        {
            if ( control->tabulate <= 0 )
            {
//                k_init_cm_qeq_full_fs <<< blocks, control->gpu_block_size, 0, control->hip_streams[5] >>>
//                    ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp,
//                      *(workspace->d_workspace), (control_params *) control->d_control_params,
//                      *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
//                      system->d_max_cm_entries, system->d_realloc_cm_entries );

                blocks = workspace->d_workspace->H.n_max * WARP_SIZE / control->gpu_block_size
                    + (workspace->d_workspace->H.n_max * WARP_SIZE % control->gpu_block_size == 0 ? 0 : 1);

                k_init_cm_qeq_full_fs_opt <<< blocks, control->gpu_block_size,
                                      sizeof(hipcub::WarpScan<int>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                                      control->hip_streams[5] >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp,
                      *(workspace->d_workspace), (control_params *) control->d_control_params,
                      *(lists[FAR_NBRS]), system->reax_param.num_atom_types,
                      system->d_max_cm_entries, system->d_realloc_cm_entries );
            }
            else
            {
                k_init_cm_qeq_full_fs_tab <<< blocks, control->gpu_block_size, 0,
                                      control->hip_streams[5] >>>
                    ( system->d_my_atoms, system->reax_param.d_sbp,
                      *(workspace->d_workspace), (control_params *) control->d_control_params,
                      *(lists[FAR_NBRS]), workspace->d_LR, system->reax_param.num_atom_types,
                      system->d_max_cm_entries, system->d_realloc_cm_entries );
            }
        }
        hipCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_CM_STOP], control->hip_streams[5] );
#endif
    }

    if ( bonds_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_BOND_START], control->hip_streams[1] );
#endif

        blocks = system->total_cap / control->gpu_block_size
            + ((system->total_cap % control->gpu_block_size == 0 ) ? 0 : 1);

        Hip_Init_Bond_Indices( system, lists[BONDS],  control->gpu_block_size,
                control->hip_streams[1] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->hip_streams[1], control->hip_stream_events[SE_INIT_DIST_DONE], 0 );
        }

//        k_init_bonds <<< control->blocks_N, control->gpu_block_size, 0, control->hip_streams[1] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              system->reax_param.d_tbp, *(workspace->d_workspace),
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), *(lists[BONDS]),
//              system->n, system->N, system->reax_param.num_atom_types,
//              system->d_max_bonds, system->d_realloc_bonds );
//        hipCheckError( );

        k_init_bonds_opt <<< control->blocks_warp_N, control->gpu_block_size,
                     (sizeof(hipcub::WarpScan<int>::TempStorage)
                      + sizeof(hipcub::WarpReduce<double>::TempStorage)) * (control->gpu_block_size / WARP_SIZE),
                     control->hip_streams[1] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              system->reax_param.d_tbp, *(workspace->d_workspace),
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), *(lists[BONDS]),
              system->n, system->N, system->reax_param.num_atom_types,
              system->d_max_bonds, system->d_realloc_bonds );
        hipCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_BOND_STOP], control->hip_streams[1] );
#endif
    }

    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && hbonds_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_HBOND_START], control->hip_streams[2] );
#endif

        Hip_Init_HBond_Indices( system, workspace, lists[HBONDS], control->gpu_block_size,
                control->hip_streams[2] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->hip_streams[2], control->hip_stream_events[SE_INIT_DIST_DONE], 0 );
        }

//        k_init_hbonds <<< control->blocks_N, control->gpu_block_size, 0, control->hip_streams[2] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), *(lists[HBONDS]),
//              system->n, system->N, system->reax_param.num_atom_types,
//              system->d_max_hbonds, system->d_realloc_hbonds );
//        hipCheckError( );

        k_init_hbonds_opt <<< control->blocks_warp_N, control->gpu_block_size,
                          sizeof(hipcub::WarpScan<int>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                          control->hip_streams[2] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), *(lists[HBONDS]),
              system->n, system->N, system->reax_param.num_atom_types,
              system->d_max_hbonds, system->d_realloc_hbonds );
        hipCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_HBOND_STOP], control->hip_streams[2] );
#endif
    }

    /* check reallocation flags on device */
    if ( cm_done == FALSE )
    {
        sHipMemcpyAsync( &workspace->d_workspace->realloc->cm,
                system->d_realloc_cm_entries, sizeof(int), 
                hipMemcpyDeviceToHost, control->hip_streams[5], __FILE__, __LINE__ );
    }
    else
    {
        workspace->d_workspace->realloc->cm = FALSE;
    }
    if ( bonds_done == FALSE )
    {
        sHipMemcpyAsync( &workspace->d_workspace->realloc->bonds,
                system->d_realloc_bonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->hip_streams[1], __FILE__, __LINE__ );
    }
    else
    {
        workspace->d_workspace->realloc->bonds = FALSE;
    }
    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && hbonds_done == FALSE )
    {
        sHipMemcpyAsync( &workspace->d_workspace->realloc->hbonds,
                system->d_realloc_hbonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->hip_streams[2], __FILE__, __LINE__ );
    }
    else
    {
        workspace->d_workspace->realloc->hbonds = FALSE;
    }

    hipStreamSynchronize( control->hip_streams[0] );
    hipStreamSynchronize( control->hip_streams[5] );
    hipStreamSynchronize( control->hip_streams[1] );
    hipStreamSynchronize( control->hip_streams[2] );

    ret = (workspace->d_workspace->realloc->cm == FALSE
            && workspace->d_workspace->realloc->bonds == FALSE
            && workspace->d_workspace->realloc->hbonds == FALSE
            ? SUCCESS : FAILURE);

    if ( workspace->d_workspace->realloc->cm == FALSE )
    {
        cm_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else
    {
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_CM_START],
                control->hip_time_events[TE_INIT_CM_STOP] ); 
        data->timing.init_cm += (real) (time_elapsed / 1000.0);
    }
#endif
    if ( workspace->d_workspace->realloc->bonds == FALSE )
    {
        bonds_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else
    {
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_BOND_START],
                control->hip_time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);
    }
#endif
    if ( workspace->d_workspace->realloc->hbonds == FALSE )
    {
        hbonds_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
    {
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_HBOND_START],
                control->hip_time_events[TE_INIT_HBOND_STOP] ); 
        data->timing.init_hbond += (real) (time_elapsed / 1000.0);
    }
#endif

    if ( ret == SUCCESS )
    {
#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_BOND_START],
                control->hip_time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);

        hipEventRecord( control->hip_time_events[TE_INIT_BOND_START], control->hip_streams[1] );
#endif

//        k_update_sym_dbond_indices <<< control->blocks_N, control->gpu_block_size,
//                                   0, control->hip_streams[1] >>> 
//            ( *(lists[BONDS]), system->N );
        k_update_sym_dbond_indices_opt <<< control->blocks_warp_N, control->gpu_block_size,
                                       0, control->hip_streams[1] >>>
            ( *(lists[BONDS]), system->N );
        hipCheckError( );

        hipEventRecord( control->hip_stream_events[SE_INIT_BOND_DONE], control->hip_streams[1] );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_BOND_STOP], control->hip_streams[1] );
#endif

#if !defined(GPU_ACCUM_ATOMIC)
        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
#if defined(LOG_PERFORMANCE)
            hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_HBOND_START],
                    control->hip_time_events[TE_INIT_HBOND_STOP] ); 
            data->timing.init_hbond += (real) (time_elapsed / 1000.0);

            hipEventRecord( control->hip_time_events[TE_INIT_HBOND_START], control->hip_streams[2] );
#endif

            /* make hbond_list symmetric */
            k_update_sym_hbond_indices_opt <<< control->blocks_warp_N, control->gpu_block_size,
                                           0, control->hip_streams[2] >>>
                ( system->d_my_atoms, *(lists[HBONDS]), system->N );
            hipCheckError( );

#if defined(LOG_PERFORMANCE)
            hipEventRecord( control->hip_time_events[TE_INIT_HBOND_STOP], control->hip_streams[2] );
#endif
        }
#endif

        dist_done = FALSE;
        cm_done = FALSE;
        bonds_done = FALSE;
        hbonds_done = FALSE;
    }
    else
    {
        Hip_Estimate_Storages( system, control, data, workspace, lists,
               workspace->d_workspace->realloc->cm,
               workspace->d_workspace->realloc->bonds,
               workspace->d_workspace->realloc->hbonds,
               data->step - data->prev_steps );
    }

    return ret;
}


int Hip_Init_Forces_No_Charges( reax_system * const system, control_params * const control,
        simulation_data * const data, storage * const workspace,
        reax_list ** const lists, output_controls * const out_control ) 
{
    int renbr, ret;
    static int dist_done = FALSE, bonds_done = FALSE, hbonds_done = FALSE;
#if defined(LOG_PERFORMANCE)
    float time_elapsed;
#endif

    renbr = (data->step - data->prev_steps) % control->reneighbor == 0 ? TRUE : FALSE;

    /* reset reallocation flags on device */
    if ( bonds_done == FALSE )
    {
        sHipMemsetAsync( system->d_realloc_bonds, FALSE, sizeof(int), 
                control->hip_streams[1], __FILE__, __LINE__ );
    }
    if ( hbonds_done == FALSE )
    {
        sHipMemsetAsync( system->d_realloc_hbonds, FALSE, sizeof(int), 
                control->hip_streams[2], __FILE__, __LINE__ );
    }

    if ( renbr == FALSE && dist_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_DIST_START], control->hip_streams[0] );
#endif

//        k_init_dist <<< control->blocks_N, control->gpu_block_size,
//                    0, control->hip_streams[0] >>>
//            ( system->d_my_atoms, *(lists[FAR_NBRS]), system->N );

        k_init_dist_opt <<< control->blocks_warp_N, control->gpu_block_size,
                        0, control->hip_streams[0] >>>
            ( system->d_my_atoms, *(lists[FAR_NBRS]), system->N );
        hipCheckError( );

        hipEventRecord( control->hip_stream_events[SE_INIT_DIST_DONE], control->hip_streams[0] );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_DIST_STOP], control->hip_streams[0] );
#endif

        dist_done = TRUE;
    }

    if ( bonds_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_BOND_START], control->hip_streams[1] );
#endif

        Hip_Init_Bond_Indices( system, lists[BONDS], control->gpu_block_size,
                control->hip_streams[1] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->hip_streams[1], control->hip_stream_events[SE_INIT_DIST_DONE], 0 );
        }

//        k_init_bonds <<< control->blocks_N, control->gpu_block_size, 0, control->hip_streams[1] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              system->reax_param.d_tbp, *(workspace->d_workspace),
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), *(lists[BONDS]),
//              system->n, system->N, system->reax_param.num_atom_types,
//              system->d_max_bonds, system->d_realloc_bonds );

        k_init_bonds_opt <<< control->blocks_warp_N, control->gpu_block_size,
                     sizeof(hipcub::WarpScan<int>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                     control->hip_streams[1] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              system->reax_param.d_tbp, *(workspace->d_workspace),
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), *(lists[BONDS]),
              system->n, system->N, system->reax_param.num_atom_types,
              system->d_max_bonds, system->d_realloc_bonds );
        hipCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_BOND_STOP], control->hip_streams[1] );
#endif
    }

    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && hbonds_done == FALSE )
    {
#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_HBOND_START], control->hip_streams[2] );
#endif

        Hip_Init_HBond_Indices( system, workspace, lists[HBONDS],
                control->gpu_block_size, control->hip_streams[2] );

        if ( renbr == FALSE )
        {
            hipStreamWaitEvent( control->hip_streams[2], control->hip_stream_events[SE_INIT_DIST_DONE], 0 );
        }

//        k_init_hbonds <<< control->blocks_N, control->gpu_block_size, 0, control->hip_streams[2] >>>
//            ( system->d_my_atoms, system->reax_param.d_sbp,
//              (control_params *) control->d_control_params,
//              *(lists[FAR_NBRS]), *(lists[HBONDS]),
//              system->n, system->N, system->reax_param.num_atom_types,
//              system->d_max_hbonds, system->d_realloc_hbonds );
//        hipCheckError( );

        k_init_hbonds_opt <<< control->blocks_N, control->gpu_block_size,
                          sizeof(hipcub::WarpScan<int>::TempStorage) * (control->gpu_block_size / WARP_SIZE),
                          control->hip_streams[2] >>>
            ( system->d_my_atoms, system->reax_param.d_sbp,
              (control_params *) control->d_control_params,
              *(lists[FAR_NBRS]), *(lists[HBONDS]),
              system->n, system->N, system->reax_param.num_atom_types,
              system->d_max_hbonds, system->d_realloc_hbonds );
        hipCheckError( );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_HBOND_STOP], control->hip_streams[2] );
#endif
    }

    /* check reallocation flags on device */
    if ( bonds_done == FALSE )
    {
        sHipMemcpyAsync( &workspace->d_workspace->realloc->bonds,
                system->d_realloc_bonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->hip_streams[1], __FILE__, __LINE__ );
    }
    else
    {
        workspace->d_workspace->realloc->bonds = FALSE;
    }
    if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 && hbonds_done == FALSE )
    {
        sHipMemcpyAsync( &workspace->d_workspace->realloc->hbonds,
                system->d_realloc_hbonds, sizeof(int), 
                hipMemcpyDeviceToHost, control->hip_streams[2], __FILE__, __LINE__ );
    }
    else
    {
        workspace->d_workspace->realloc->hbonds = FALSE;
    }

    hipStreamSynchronize( control->hip_streams[0] );
    hipStreamSynchronize( control->hip_streams[1] );
    hipStreamSynchronize( control->hip_streams[2] );

    ret = (workspace->d_workspace->realloc->bonds == FALSE
            && workspace->d_workspace->realloc->hbonds == FALSE
            ? SUCCESS : FAILURE);

    if ( workspace->d_workspace->realloc->bonds == FALSE )
    {
        bonds_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else
    {
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_BOND_START],
                control->hip_time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);
    }
#endif
    if ( workspace->d_workspace->realloc->hbonds == FALSE )
    {
        hbonds_done = TRUE;
    }
#if defined(LOG_PERFORMANCE)
    else if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
    {
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_HBOND_START],
                control->hip_time_events[TE_INIT_HBOND_STOP] ); 
        data->timing.init_hbond += (real) (time_elapsed / 1000.0);
    }
#endif

    if ( ret == SUCCESS )
    {
#if defined(LOG_PERFORMANCE)
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_BOND_START],
                control->hip_time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);

        hipEventRecord( control->hip_time_events[TE_INIT_BOND_START], control->hip_streams[1] );
#endif

//        k_update_sym_dbond_indices <<< control->blocks_N, control->gpu_block_size,
//                                   0, control->hip_streams[1] >>>
//            ( *(lists[BONDS]), system->N );
        k_update_sym_dbond_indices_opt <<< control->blocks_warp_N, control->gpu_block_size,
                                       0, control->hip_streams[1] >>>
            ( *(lists[BONDS]), system->N );
        hipCheckError( );

        hipEventRecord( control->hip_stream_events[SE_INIT_BOND_DONE], control->hip_streams[1] );

#if defined(LOG_PERFORMANCE)
        hipEventRecord( control->hip_time_events[TE_INIT_BOND_STOP], control->hip_streams[1] );
#endif

#if !defined(GPU_ACCUM_ATOMIC)
        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
#if defined(LOG_PERFORMANCE)
            hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_HBOND_START],
                    control->hip_time_events[TE_INIT_HBOND_STOP] ); 
            data->timing.init_hbond += (real) (time_elapsed / 1000.0);

            hipEventRecord( control->hip_time_events[TE_INIT_HBOND_START], control->hip_streams[2] );
#endif

            /* make hbond_list symmetric */
            k_update_sym_hbond_indices_opt <<< control->blocks_warp_N, control->gpu_block_size,
                                           0, control->hip_streams[2] >>>
                ( system->d_my_atoms, *(lists[HBONDS]), system->N );
            hipCheckError( );

#if defined(LOG_PERFORMANCE)
            hipEventRecord( control->hip_time_events[TE_INIT_HBOND_STOP], control->hip_streams[2] );
#endif
        }
#endif

        dist_done = FALSE;
        bonds_done = FALSE;
        hbonds_done = FALSE;
    }
    else
    {
        Hip_Estimate_Storages( system, control, data, workspace, lists,
               FALSE, workspace->d_workspace->realloc->bonds,
               workspace->d_workspace->realloc->hbonds,
               data->step - data->prev_steps );
    }

    return ret;
}


int Hip_Compute_Bonded_Forces( reax_system * const system, control_params * const control, 
        simulation_data * const data, storage * const workspace, 
        reax_list ** const lists, output_controls * const out_control )
{
    int ret;
    static int compute_bonded_part1 = FALSE;

    ret = SUCCESS;

    if ( compute_bonded_part1 == FALSE )
    {
        Hip_Compute_Bond_Orders( system, control, data, workspace, lists,
                out_control );

        Hip_Compute_Bonds( system, control, data, workspace, lists,
                out_control );

        Hip_Compute_Atom_Energy( system, control, data, workspace, lists,
                out_control );

        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
            Hip_Compute_Hydrogen_Bonds( system, control, data, workspace,
                    lists, out_control );
        }

        compute_bonded_part1 = TRUE;
    }

    ret = Hip_Compute_Valence_Angles( system, control, data, workspace,
            lists, out_control );

    if ( ret == SUCCESS )
    {
        Hip_Compute_Torsion_Angles( system, control, data, workspace, lists,
                out_control );

        compute_bonded_part1 = FALSE;
    }

    return ret;
}


static void Hip_Compute_Total_Force( reax_system * const system, control_params * const control,
        simulation_data * const data, storage * const workspace,
        reax_list ** const lists, mpi_datatypes * const mpi_data )
{
    sHipHostMallocCheck( &workspace->host_scratch, &workspace->host_scratch_size,
            sizeof(rvec) * system->N, hipHostMallocNumaUser | hipHostMallocPortable, TRUE, SAFE_ZONE,
            __FILE__, __LINE__ );
    memset( workspace->host_scratch, 0, sizeof(rvec) * system->N );

    Hip_Total_Forces_Part1( system, control, data, workspace, lists );

    /* now all forces are computed to their partially-final values
     * based on the neighbors information each processor has had.
     * final values of force on each atom needs to be computed by adding up
     * all partially-final pieces */
    sHipMemcpyAsync( workspace->host_scratch, workspace->d_workspace->f,
            sizeof(rvec) * system->N, hipMemcpyDeviceToHost,
            control->hip_streams[0], __FILE__, __LINE__ );
    hipStreamSynchronize( control->hip_streams[0] );

    Coll( system, mpi_data, workspace->host_scratch, RVEC_PTR_TYPE,
            mpi_data->mpi_rvec );

    sHipMemcpyAsync( workspace->d_workspace->f, workspace->host_scratch,
            sizeof(rvec) * system->N, hipMemcpyHostToDevice,
            control->hip_streams[0], __FILE__, __LINE__ );

    Hip_Total_Forces_Part2( system, control, workspace );
}


extern "C" int Hip_Compute_Forces( reax_system * const system, control_params * const control,
        simulation_data * const data, storage * const workspace, reax_list ** const lists,
        output_controls * const out_control, mpi_datatypes * const mpi_data )
{
    int i, charge_flag, ret;
    static int init_forces_done = FALSE, nonbonded_forces_part1_done = FALSE;
#if defined(LOG_PERFORMANCE)
    float time_elapsed, time_elapsed2, time_elapsed3;
    float time_elapsed4, time_elapsed5, time_elapsed6;
    float time_elapsed7, time_elapsed8, time_elapsed9;
#endif

    ret = SUCCESS;

    if ( control->charge_freq > 0
            && (data->step - data->prev_steps) % control->charge_freq == 0 )
    {
        charge_flag = TRUE;
    }
    else
    {
        charge_flag = FALSE;
    }

    if ( init_forces_done == FALSE )
    {
        if ( charge_flag == TRUE )
        {
            ret = Hip_Init_Forces( system, control, data,
                    workspace, lists, out_control );
        }
        else
        {
            ret = Hip_Init_Forces_No_Charges( system, control, data,
                    workspace, lists, out_control );
        }

        if ( ret == SUCCESS )
        {
            init_forces_done = TRUE;
        }
    }

    if ( nonbonded_forces_part1_done == FALSE )
    {
        Hip_Compute_NonBonded_Forces_Part1( system, control, data, workspace,
                lists, out_control );

        nonbonded_forces_part1_done = TRUE;
    }

    if ( ret == SUCCESS )
    {
        ret = Hip_Compute_Bonded_Forces( system, control, data,
                workspace, lists, out_control );
    }

    if ( ret == SUCCESS )
    {
#if defined(LOG_PERFORMANCE)
        real time;

        time = Get_Time( );
#endif

        if ( charge_flag == TRUE )
        {
            Hip_Compute_Charges( system, control, data,
                    workspace, out_control, mpi_data, control->hip_streams[5] );
        }
    
#if defined(LOG_PERFORMANCE)
        Update_Timing_Info( &time, &data->timing.cm );
#endif

        Hip_Compute_NonBonded_Forces_Part2( system, control, data, workspace,
                lists, out_control );

        for ( i = 0; i < MAX_GPU_STREAMS; ++i )
        {
            hipStreamSynchronize( control->hip_streams[i] );
        }

        Hip_Compute_Total_Force( system, control, data, workspace, lists, mpi_data );

#if defined(LOG_PERFORMANCE)
        if ( (data->step - data->prev_steps) % control->reneighbor == 0 )
        {
            hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_NBRS_START],
                    control->hip_time_events[TE_NBRS_STOP] ); 
            data->timing.nbrs += (real) (time_elapsed / 1000.0);

            hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_CM_START],
                    control->hip_time_events[TE_INIT_CM_STOP] ); 
            hipEventElapsedTime( &time_elapsed2, control->hip_time_events[TE_INIT_CM_START],
                    control->hip_time_events[TE_INIT_BOND_STOP] ); 
            if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
            {
                hipEventElapsedTime( &time_elapsed3, control->hip_time_events[TE_INIT_CM_START],
                        control->hip_time_events[TE_INIT_HBOND_STOP] ); 
            }
            else
            {
                time_elapsed3 = 0.0;
            }
            hipEventElapsedTime( &time_elapsed4, control->hip_time_events[TE_INIT_BOND_START],
                    control->hip_time_events[TE_INIT_CM_STOP] ); 
            hipEventElapsedTime( &time_elapsed5, control->hip_time_events[TE_INIT_BOND_START],
                    control->hip_time_events[TE_INIT_BOND_STOP] ); 
            if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
            {
                hipEventElapsedTime( &time_elapsed6, control->hip_time_events[TE_INIT_BOND_START],
                        control->hip_time_events[TE_INIT_HBOND_STOP] ); 
                hipEventElapsedTime( &time_elapsed7, control->hip_time_events[TE_INIT_HBOND_START],
                        control->hip_time_events[TE_INIT_CM_STOP] ); 
                hipEventElapsedTime( &time_elapsed8, control->hip_time_events[TE_INIT_HBOND_START],
                        control->hip_time_events[TE_INIT_BOND_STOP] ); 
                hipEventElapsedTime( &time_elapsed9, control->hip_time_events[TE_INIT_HBOND_START],
                        control->hip_time_events[TE_INIT_HBOND_STOP] ); 
            }
            else
            {
                time_elapsed6 = 0.0;
                time_elapsed7 = 0.0;
                time_elapsed8 = 0.0;
                time_elapsed9 = 0.0;
            }
            data->timing.init_forces += (real) MAX3( 
                MAX3(time_elapsed / 1000.0, time_elapsed2 / 1000.0, time_elapsed3 / 1000.0),
                MAX3(time_elapsed4 / 1000.0, time_elapsed5 / 1000.0, time_elapsed6 / 1000.0),
                MAX3(time_elapsed7 / 1000.0, time_elapsed8 / 1000.0, time_elapsed9 / 1000.0) );
        }
        else
        {
            hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_DIST_START],
                    control->hip_time_events[TE_INIT_CM_STOP] ); 
            hipEventElapsedTime( &time_elapsed2, control->hip_time_events[TE_INIT_DIST_START],
                    control->hip_time_events[TE_INIT_BOND_STOP] ); 
            if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
            {
                hipEventElapsedTime( &time_elapsed3, control->hip_time_events[TE_INIT_DIST_START],
                        control->hip_time_events[TE_INIT_HBOND_STOP] ); 
            }
            else
            {
                time_elapsed3 = 0.0;
            }
            data->timing.init_forces += (real) MAX3(time_elapsed / 1000.0, time_elapsed2 / 1000.0,
                    time_elapsed3 / 1000.0);

            hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_DIST_START],
                    control->hip_time_events[TE_INIT_DIST_STOP] ); 
            data->timing.init_dist += (real) (time_elapsed / 1000.0);
        }

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_CM_START],
                control->hip_time_events[TE_INIT_CM_STOP] ); 
        data->timing.init_cm += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_BOND_START],
                control->hip_time_events[TE_INIT_BOND_STOP] ); 
        data->timing.init_bond += (real) (time_elapsed / 1000.0);

        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
            hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_INIT_HBOND_START],
                    control->hip_time_events[TE_INIT_HBOND_STOP] ); 
            data->timing.init_hbond += (real) (time_elapsed / 1000.0);
        }

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_BOND_ORDER_START],
                control->hip_time_events[TE_LPOVUN_STOP] ); 
        hipEventElapsedTime( &time_elapsed2, control->hip_time_events[TE_BOND_ORDER_START],
                control->hip_time_events[TE_BONDS_STOP] ); 
        hipEventElapsedTime( &time_elapsed3, control->hip_time_events[TE_BOND_ORDER_START],
                control->hip_time_events[TE_TORSION_STOP] ); 
        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
            hipEventElapsedTime( &time_elapsed4, control->hip_time_events[TE_BOND_ORDER_START],
                    control->hip_time_events[TE_HBONDS_STOP] ); 
        }
        else
        {
            time_elapsed4 = 0.0;
        }
        data->timing.bonded += (real) MAX(MAX3(time_elapsed / 1000.0, time_elapsed2 / 1000.0,
                time_elapsed3 / 1000.0), time_elapsed4 / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_BOND_ORDER_START],
                control->hip_time_events[TE_BOND_ORDER_STOP] ); 
        data->timing.bond_order += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_BONDS_START],
                control->hip_time_events[TE_BONDS_STOP] ); 
        data->timing.bonds += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_LPOVUN_START],
                control->hip_time_events[TE_LPOVUN_STOP] ); 
        data->timing.lpovun += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_VALENCE_START],
                control->hip_time_events[TE_VALENCE_STOP] ); 
        data->timing.valence += (real) (time_elapsed / 1000.0);

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_TORSION_START],
                control->hip_time_events[TE_TORSION_STOP] ); 
        data->timing.torsion += (real) (time_elapsed / 1000.0);

        if ( system->total_H_atoms > 0 && control->hbond_cut > 0.0 )
        {
            hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_HBONDS_START],
                    control->hip_time_events[TE_HBONDS_STOP] ); 
            data->timing.hbonds += (real) (time_elapsed / 1000.0);
        }

#if !defined(USE_FUSED_VDW_COULOMB)
        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_VDW_START],
                control->hip_time_events[TE_VDW_STOP] ); 
        data->timing.nonb += (real) (time_elapsed / 1000.0);
#endif

        hipEventElapsedTime( &time_elapsed, control->hip_time_events[TE_COULOMB_START],
                control->hip_time_events[TE_COULOMB_STOP] ); 
        data->timing.nonb += (real) (time_elapsed / 1000.0);
#endif

        init_forces_done = FALSE;
        nonbonded_forces_part1_done = FALSE;
    }

    return ret;
}

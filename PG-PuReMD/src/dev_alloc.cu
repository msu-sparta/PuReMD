#include "hip/hip_runtime.h"

#include "dev_alloc.h"
#include "cuda_utils.h"

#include "vector.h"

extern "C"
{

    int dev_alloc_control (control_params *control)
    {
        cuda_malloc ((void **)&control->d_control_params, sizeof (control_params), 1, "control_params");
        copy_host_device (control, control->d_control_params, sizeof (control_params), hipMemcpyHostToDevice, "control_params");
    }

    CUDA_GLOBAL void Init_Nbrs(ivec *nbrs, int N)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if (index >= N) return;

        nbrs[index][0] = -1; 
        nbrs[index][1] = -1; 
        nbrs[index][2] = -1; 
    }


    int dev_alloc_grid (reax_system *system)
    {
        int total;
        grid_cell local_cell;
        grid *host = &system->my_grid;
        grid *device = &system->d_my_grid;
        ivec *nbrs_x = (ivec *) scratch;

        total = host->ncells[0] * host->ncells[1] * host->ncells[2];
        ivec_Copy (device->ncells, host->ncells);
        rvec_Copy (device->cell_len, host->cell_len);
        rvec_Copy (device->inv_len, host->inv_len);

        ivec_Copy (device->bond_span, host->bond_span );
        ivec_Copy (device->nonb_span, host->nonb_span );
        ivec_Copy (device->vlist_span, host->vlist_span );

        ivec_Copy (device->native_cells, host->native_cells );
        ivec_Copy (device->native_str, host->native_str );
        ivec_Copy (device->native_end, host->native_end );

        device->ghost_cut = host->ghost_cut;
        ivec_Copy (device->ghost_span, host->ghost_span );
        ivec_Copy (device->ghost_nonb_span, host->ghost_nonb_span );
        ivec_Copy (device->ghost_hbond_span, host->ghost_hbond_span );
        ivec_Copy (device->ghost_bond_span, host->ghost_bond_span );

        cuda_malloc ((void **) &device->str, sizeof (int) * total, 1, "grid:str");
        cuda_malloc ((void **) &device->end, sizeof (int) * total, 1, "grid:end");
        cuda_malloc ((void **) &device->cutoff, sizeof (real) * total, 1, "grid:cutoff");
        cuda_malloc ((void **) &device->nbrs_x, sizeof (ivec) * total * host->max_nbrs, 1, "grid:nbrs_x");
        cuda_malloc ((void **) &device->nbrs_cp, sizeof (rvec) * total * host->max_nbrs, 1, "grid:nbrs_cp");
        cuda_malloc ((void **) &device->rel_box, sizeof (ivec) * total, 1, "grid:rel_box");

        /*
           int block_size = 512;
           int blocks = (host->max_nbrs) / block_size + ((host->max_nbrs) % block_size == 0 ? 0 : 1); 

           Init_Nbrs <<<blocks, block_size>>>
           (nbrs_x, host->max_nbrs );
           hipDeviceSynchronize (); 
           cudaCheckError ();

           cuda_malloc ((void **)& device->cells, 
           sizeof (grid_cell) * total, 
           1, "grid:cells");
           fprintf (stderr, " Device cells address --> %ld \n", device->cells );
           cuda_malloc ((void **) &device->order, sizeof (ivec) * (host->total + 1), 1, "grid:order");

           local_cell.top = local_cell.mark = local_cell.str = local_cell.end = 0;
           fprintf (stderr, "Total cells to be allocated -- > %d \n", total );
           for (int i = 0; i < total; i++) {
        //fprintf (stderr, "Address of the local atom -> %ld  \n", &local_cell);

        cuda_malloc ((void **) &local_cell.atoms, sizeof (int) * host->max_atoms, 
        1, "alloc:grid:cells:atoms");
        //fprintf (stderr, "Allocated address of the atoms --> %ld  (%d)\n", local_cell.atoms, host->max_atoms );

        cuda_malloc ((void **) &local_cell.nbrs_x, sizeof (ivec) * host->max_nbrs, 
        1, "alloc:grid:cells:nbrs_x" );
        copy_device (local_cell.nbrs_x, nbrs_x, host->max_nbrs * sizeof (ivec), "grid:nbrs_x");    
        //fprintf (stderr, "Allocated address of the nbrs_x--> %ld \n", local_cell.nbrs_x);

        cuda_malloc ((void **) &local_cell.nbrs_cp, sizeof (rvec) * host->max_nbrs, 
        1, "alloc:grid:cells:nbrs_cp" );
        //fprintf (stderr, "Allocated address of the nbrs_cp--> %ld \n", local_cell.nbrs_cp);

        //cuda_malloc ((void **) &local_cell.nbrs, sizeof (grid_cell *) * host->max_nbrs , 
        //                1, "alloc:grid:cells:nbrs" );
        //fprintf (stderr, "Allocated address of the nbrs--> %ld \n", local_cell.nbrs);

        copy_host_device (&local_cell, &device->cells[i], sizeof (grid_cell), hipMemcpyHostToDevice, "grid:cell-alloc");
        }
         */

        return SUCCESS;
    }

    int dev_dealloc_grid_cell_atoms (reax_system *system)
    {
        int total;
        grid_cell local_cell;
        grid *host = &system->my_grid;
        grid *device = &system->d_my_grid;

        total = host->ncells[0] * host->ncells[1] * host->ncells[2];


        for (int i = 0; i < total; i++) {
            copy_host_device (&local_cell, &device->cells[i], 
                    sizeof (grid_cell), hipMemcpyDeviceToHost, "grid:cell-dealloc");
            cuda_free (local_cell.atoms, "grid_cell:atoms" );
        }
    }

    int dev_alloc_grid_cell_atoms (reax_system *system, int cap)
    {
        int total;
        grid_cell local_cell;
        grid *host = &system->my_grid;
        grid *device = &system->d_my_grid;

        total = host->ncells[0] * host->ncells[1] * host->ncells[2];

        for (int i = 0; i < total; i++) {
            copy_host_device (&local_cell, &device->cells[i], 
                    sizeof (grid_cell), hipMemcpyDeviceToHost, "grid:cell-dealloc");
            cuda_malloc ((void **) &local_cell.atoms, sizeof (int) * cap, 
                    1, "realloc:grid:cells:atoms");
            copy_host_device (&local_cell, &device->cells[i], 
                    sizeof (grid_cell), hipMemcpyHostToDevice, "grid:cell-realloc");
        }
    }


    int dev_alloc_system (reax_system *system)
    {
        cuda_malloc ( (void **) &system->d_my_atoms, system->total_cap * sizeof (reax_atom), 1, "system:d_my_atoms");  
        //fprintf (stderr, "p:%d - allocated atoms : %d (%ld, %ld) \n", system->my_rank, system->total_cap, 
        //                                                                                    system->my_atoms, system->d_my_atoms);

        //simulation boxes
        cuda_malloc ( (void **) &system->d_big_box, sizeof (simulation_box), 1, "system:d_big_box");
        cuda_malloc ( (void **) &system->d_my_box, sizeof (simulation_box), 1, "system:d_my_box");
        cuda_malloc ( (void **) &system->d_my_ext_box, sizeof (simulation_box), 1, "d_my_ext_box");

        //interaction parameters
        cuda_malloc ((void **) &system->reax_param.d_sbp, system->reax_param.num_atom_types * sizeof (single_body_parameters),
                1, "system:d_sbp");

        cuda_malloc ((void **) &system->reax_param.d_tbp, pow (system->reax_param.num_atom_types, 2) * sizeof (two_body_parameters), 
                1, "system:d_tbp");

        cuda_malloc ((void **) &system->reax_param.d_thbp, pow (system->reax_param.num_atom_types, 3) * sizeof (three_body_header),
                1, "system:d_thbp");

        cuda_malloc ((void **) &system->reax_param.d_hbp, pow (system->reax_param.num_atom_types, 3) * sizeof (hbond_parameters),
                1, "system:d_hbp");

        cuda_malloc ((void **) &system->reax_param.d_fbp, pow (system->reax_param.num_atom_types, 4) * sizeof (four_body_header),
                1, "system:d_fbp");

        cuda_malloc ((void **) &system->reax_param.d_gp.l, system->reax_param.gp.n_global * sizeof (real), 1, "system:d_gp.l");

        system->reax_param.d_gp.n_global = 0;
        system->reax_param.d_gp.vdw_type = 0;

        return SUCCESS;
    }

    int dev_realloc_system (reax_system *system, int local_cap, int total_cap, char *msg)
    {
        //free the existing storage for atoms
        cuda_free (system->d_my_atoms, "system:d_my_atoms");

        cuda_malloc ((void **) &system->d_my_atoms, sizeof (reax_atom) * total_cap, 
                1, "system:d_my_atoms");
        return FAILURE;
    }


    int dev_alloc_simulation_data(simulation_data *data)
    {
        cuda_malloc ((void **) &(data->d_simulation_data), sizeof (simulation_data), 1, "simulation_data");
        return SUCCESS;
    }

    int dev_alloc_workspace (reax_system *system, control_params *control, 
            storage *workspace, int local_cap, int total_cap, 
            char *msg)
    {
        int i, total_real, total_rvec, local_int, local_real, local_rvec;

        workspace->allocated = 1;
        total_real = total_cap * sizeof(real);
        total_rvec = total_cap * sizeof(rvec);
        local_int = local_cap * sizeof(int);
        local_real = local_cap * sizeof(real);
        local_rvec = local_cap * sizeof(rvec);

        /* communication storage */  
        /*
           workspace->tmp_dbl = NULL;
           workspace->tmp_rvec = NULL;
           workspace->tmp_rvec2 = NULL;
         */

        //fprintf (stderr, "Deltap and TOTAL BOND ORDER size --> %d \n", total_cap );

        /* bond order related storage  */
        cuda_malloc ((void **) &workspace->within_bond_box, total_cap * sizeof (int), 1, "skin");
        cuda_malloc ((void **) &workspace->total_bond_order, total_real, 1, "total_bo");
        cuda_malloc ((void **) &workspace->Deltap, total_real, 1, "Deltap");
        cuda_malloc ((void **) &workspace->Deltap_boc, total_real, 1, "Deltap_boc");
        cuda_malloc ((void **) &workspace->dDeltap_self, total_rvec, 1, "dDeltap_self");
        cuda_malloc ((void **) &workspace->Delta, total_real, 1, "Delta" );
        cuda_malloc ((void **) &workspace->Delta_lp, total_real, 1, "Delta_lp" );
        cuda_malloc ((void **) &workspace->Delta_lp_temp, total_real, 1, "Delta_lp_temp" );
        cuda_malloc ((void **) &workspace->dDelta_lp, total_real, 1, "Delta_lp_temp" );
        cuda_malloc ((void **) &workspace->dDelta_lp_temp, total_real, 1, "dDelta_lp_temp" );
        cuda_malloc ((void **) &workspace->Delta_e, total_real, 1, "Delta_e" );
        cuda_malloc ((void **) &workspace->Delta_boc, total_real, 1, "Delta_boc");
        cuda_malloc ((void **) &workspace->nlp, total_real, 1, "nlp");
        cuda_malloc ((void **) &workspace->nlp_temp, total_real, 1, "nlp_temp");
        cuda_malloc ((void **) &workspace->Clp, total_real, 1, "Clp");
        cuda_malloc ((void **) &workspace->vlpex, total_real, 1, "vlpex");
        cuda_malloc ((void **) &workspace->bond_mark, total_real, 1, "bond_mark");
        cuda_malloc ((void **) &workspace->done_after, total_real, 1, "done_after");


        /* QEq storage */
        cuda_malloc ((void **) &workspace->Hdia_inv, total_cap * sizeof (real), 1, "Hdia_inv");
        cuda_malloc ((void **) &workspace->b_s, total_cap * sizeof (real), 1, "b_s");
        cuda_malloc ((void **) &workspace->b_t, total_cap * sizeof (real), 1, "b_t");
        cuda_malloc ((void **) &workspace->b_prc, total_cap * sizeof (real), 1, "b_prc");
        cuda_malloc ((void **) &workspace->b_prm, total_cap * sizeof (real), 1, "b_prm");
        cuda_malloc ((void **) &workspace->s, total_cap * sizeof (real), 1, "s");
        cuda_malloc ((void **) &workspace->t, total_cap * sizeof (real), 1, "t");
        cuda_malloc ((void **) &workspace->droptol, total_cap * sizeof (real), 1, "droptol");
        cuda_malloc ((void **) &workspace->b, total_cap * sizeof (rvec2), 1, "b");
        cuda_malloc ((void **) &workspace->x, total_cap * sizeof (rvec2), 1, "x");

        /* GMRES storage */
        cuda_malloc ((void **) &workspace->y, (RESTART+1)*sizeof (real), 1, "y");
        cuda_malloc ((void **) &workspace->z, (RESTART+1)*sizeof (real), 1, "z");
        cuda_malloc ((void **) &workspace->g, (RESTART+1)*sizeof (real), 1, "g");
        cuda_malloc ((void **) &workspace->h, (RESTART+1)*(RESTART+1)*sizeof (real), 1, "h");
        cuda_malloc ((void **) &workspace->hs, (RESTART+1)*sizeof (real), 1, "hs");
        cuda_malloc ((void **) &workspace->hc, (RESTART+1)*sizeof (real), 1, "hc");
        cuda_malloc ((void **) &workspace->v, (RESTART+1)*(RESTART+1)*sizeof (real), 1, "v");

        /* CG storage */
        cuda_malloc ((void **) &workspace->r, total_cap * sizeof (real), 1,  "r");
        cuda_malloc ((void **) &workspace->d, total_cap * sizeof (real), 1, "d");
        cuda_malloc ((void **) &workspace->q, total_cap * sizeof (real), 1, "q");
        cuda_malloc ((void **) &workspace->p, total_cap * sizeof (real), 1, "p");
        cuda_malloc ((void **) &workspace->r2, total_cap * sizeof (rvec2), 1, "r2");
        cuda_malloc ((void **) &workspace->d2, total_cap * sizeof (rvec2), 1, "d2");
        cuda_malloc ((void **) &workspace->q2, total_cap * sizeof (rvec2), 1, "q2");
        cuda_malloc ((void **) &workspace->p2, total_cap * sizeof (rvec2), 1, "p2");

        /* integrator storage */
        cuda_malloc ((void **) &workspace->v_const, local_rvec, 1, "v_const");

        /* storage for analysis */
        if( control->molecular_analysis || control->diffusion_coef ) {
            cuda_malloc ((void **) &workspace->mark, local_cap * sizeof (int), 1, "mark");
            cuda_malloc ((void **) &workspace->old_mark, local_cap * sizeof (int), 1, "old_mark");
        }
        else
            workspace->mark = workspace->old_mark = NULL;

        if( control->diffusion_coef )
            cuda_malloc ((void **) &workspace->x_old, local_cap * sizeof (rvec), 1, "x_old");
        else
            workspace->x_old = NULL;

        /* force related storage */
        cuda_malloc ((void **) &workspace->f, total_cap * sizeof (rvec), 1, "f");
        cuda_malloc ((void **) &workspace->CdDelta, total_cap * sizeof (rvec), 1, "CdDelta");

        /* Taper params */
        cuda_malloc ((void **) &workspace->Tap, 8 * sizeof (real), 1, "Tap");

        return SUCCESS;
    }

    int dev_dealloc_workspace (reax_system *system, control_params *control, 
            storage *workspace, int local_cap, int total_cap, 
            char *msg)
    {
        /* communication storage */  
        /*
           workspace->tmp_dbl = NULL;
           workspace->tmp_rvec = NULL;
           workspace->tmp_rvec2 = NULL;
         */

        /* bond order related storage  */
        cuda_free (workspace->within_bond_box, "skin");
        cuda_free (workspace->total_bond_order, "total_bo");
        cuda_free (workspace->Deltap, "Deltap");
        cuda_free (workspace->Deltap_boc, "Deltap_boc");
        cuda_free (workspace->dDeltap_self, "dDeltap_self");
        cuda_free (workspace->Delta, "Delta" );
        cuda_free (workspace->Delta_lp, "Delta_lp" );
        cuda_free (workspace->Delta_lp_temp, "Delta_lp_temp" );
        cuda_free (workspace->dDelta_lp, "Delta_lp_temp" );
        cuda_free (workspace->dDelta_lp_temp, "dDelta_lp_temp" );
        cuda_free (workspace->Delta_e, "Delta_e" );
        cuda_free (workspace->Delta_boc, "Delta_boc");
        cuda_free (workspace->nlp, "nlp");
        cuda_free (workspace->nlp_temp, "nlp_temp");
        cuda_free (workspace->Clp, "Clp");
        cuda_free (workspace->vlpex, "vlpex");
        cuda_free (workspace->bond_mark, "bond_mark");
        cuda_free (workspace->done_after, "done_after");

        /* QEq storage */
        cuda_free (workspace->Hdia_inv, "Hdia_inv");
        cuda_free (workspace->b_s, "b_s");
        cuda_free (workspace->b_t, "b_t");
        cuda_free (workspace->b_prc, "b_prc");
        cuda_free (workspace->b_prm, "b_prm");
        cuda_free (workspace->s, "s");
        cuda_free (workspace->t, "t");
        cuda_free (workspace->droptol, "droptol");
        cuda_free (workspace->b, "b");
        cuda_free (workspace->x, "x");

        /* GMRES storage */
        cuda_free (workspace->y, "y");
        cuda_free (workspace->z, "z");
        cuda_free (workspace->g, "g");
        cuda_free (workspace->h, "h");
        cuda_free (workspace->hs, "hs");
        cuda_free (workspace->hc, "hc");
        cuda_free (workspace->v, "v");

        /* CG storage */
        cuda_free (workspace->r, "r");
        cuda_free (workspace->d, "d");
        cuda_free (workspace->q, "q");
        cuda_free (workspace->p, "p");
        cuda_free (workspace->r2, "r2");
        cuda_free (workspace->d2, "d2");
        cuda_free (workspace->q2, "q2");
        cuda_free (workspace->p2, "p2");

        /* integrator storage */
        cuda_free (workspace->v_const, "v_const");

        /* storage for analysis */
        if( control->molecular_analysis || control->diffusion_coef ) {
            cuda_free (workspace->mark, "mark");
            cuda_free (workspace->old_mark, "old_mark");
        }
        else
            workspace->mark = workspace->old_mark = NULL;

        if( control->diffusion_coef )
            cuda_free (workspace->x_old, "x_old");
        else
            workspace->x_old = NULL;

        /* force related storage */
        cuda_free (workspace->f, "f");
        cuda_free (workspace->CdDelta, "CdDelta");

        /* Taper params */
        cuda_free (workspace->Tap, "Tap");

        return FAILURE;
    }




    int dev_alloc_matrix (sparse_matrix *H, int cap, int m)
    {
        //sparse_matrix *H;
        //H = *pH;

        H->cap = cap;
        H->m = m;
        cuda_malloc ((void **) &H->start, sizeof (int) * cap, 1, "matrix_start");
        cuda_malloc ((void **) &H->end, sizeof (int) * cap, 1, "matrix_end");
        cuda_malloc ((void **) &H->entries, sizeof (sparse_matrix_entry) * m, 1, "matrix_entries");

        return SUCCESS;
    }

    int dev_dealloc_matrix (sparse_matrix *H)
    {
        cuda_free (H->start, "matrix_start");
        cuda_free (H->end, "matrix_end");
        cuda_free (H->entries, "matrix_entries");

        return SUCCESS;
    }


}


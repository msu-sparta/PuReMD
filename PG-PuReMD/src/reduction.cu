#include "hip/hip_runtime.h"

#include "reduction.h"
#include "vector.h"

#include "cuda_shuffle.h"

CUDA_GLOBAL void k_reduction(const real *input, real *per_block_results, const size_t n)
{
#if defined(__SM_35__)
    extern __shared__ real my_results[];
    real sdata;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    real x = 0;

    if(i < n)
        x = input[i];

    sdata = x;
    __syncthreads();

    for(int z = 16; z >=1; z/=2)
        sdata+= shfl ( sdata, z);

    if (threadIdx.x % 32 == 0)
        my_results[threadIdx.x >> 5] = sdata;

    __syncthreads ();

    for(int offset = blockDim.x >> 6; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset)
            my_results[threadIdx.x] += my_results[threadIdx.x + offset];

        __syncthreads();
    }

    if(threadIdx.x == 0)
        per_block_results[blockIdx.x] = my_results[0];

#else

    extern __shared__ real sdata[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    real x = 0;

    if(i < n)
    {
        x = input[i];
    }
    sdata[threadIdx.x] = x;
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0)
    {
        per_block_results[blockIdx.x] = sdata[0];
    }
#endif
}

CUDA_GLOBAL void k_reduction_rvec (rvec *input, rvec *results, size_t n)
{
#if defined(__SM_35__)


    extern __shared__ rvec my_rvec[];
    rvec sdata;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    rvec_MakeZero( sdata );

    if(i < n)
        rvec_Copy (sdata, input[i]);

    __syncthreads();

    for(int z = 16; z >=1; z/=2){
        sdata[0] += shfl ( sdata[0], z);
        sdata[1] += shfl ( sdata[1], z);
        sdata[2] += shfl ( sdata[2], z);
    }

    if (threadIdx.x % 32 == 0)
        rvec_Copy( my_rvec[threadIdx.x >> 5] , sdata );

    __syncthreads ();

    for(int offset = blockDim.x >> 6; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset)
            rvec_Add( my_rvec[threadIdx.x], my_rvec[threadIdx.x + offset] );

        __syncthreads();
    }

    if(threadIdx.x == 0)
        rvec_Add (results[blockIdx.x], my_rvec[0]);


#else


    extern __shared__ rvec svec_data[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    rvec x;

    rvec_MakeZero (x);

    if(i < n)
    {
        rvec_Copy (x, input[i]);
    }

    rvec_Copy (svec_data[threadIdx.x], x);
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            rvec_Add (svec_data[threadIdx.x], svec_data[threadIdx.x + offset]);
        }

        __syncthreads();
    }

    if(threadIdx.x == 0)
    {
        //rvec_Copy (results[blockIdx.x], svec_data[0]);
        rvec_Add (results[blockIdx.x], svec_data[0]);
    }
#endif


}

CUDA_GLOBAL void k_reduction_rvec2 (rvec2 *input, rvec2 *results, size_t n)
{
#if defined(__SM_35__)

    extern __shared__ rvec2 my_rvec2[];
    rvec2 sdata;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[0] = 0.0;
    sdata[1] = 0.0;

    if(i < n){
        sdata[0] = input[i][0];
        sdata[1] = input[i][1];
    }

    __syncthreads();

    for(int z = 16; z >=1; z/=2){
        sdata[0] += shfl ( sdata[0], z);
        sdata[1] += shfl ( sdata[1], z);
    }

    if (threadIdx.x % 32 == 0){
        my_rvec2[threadIdx.x >> 5][0] = sdata[0];
        my_rvec2[threadIdx.x >> 5][1] = sdata[1];
    }

    __syncthreads ();

    for(int offset = blockDim.x >> 6; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset){
            my_rvec2[threadIdx.x][0] += my_rvec2[threadIdx.x + offset][0];
            my_rvec2[threadIdx.x][1] += my_rvec2[threadIdx.x + offset][1];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0){
        results[blockIdx.x][0] = my_rvec2[0][0];
        results[blockIdx.x][1] = my_rvec2[0][1];
    }

#else
    extern __shared__ rvec2 svec2_data[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    rvec2 x;

    x[0] = 0.0;
    x[1] = 0.0;

    if(i < n)
    {
        x[0] += input[i][0];
        x[1] += input[i][1];
    }

    svec2_data [threadIdx.x][0] = x[0];
    svec2_data [threadIdx.x][1] = x[1];
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            svec2_data [threadIdx.x][0] += svec2_data [threadIdx.x + offset][0];
            svec2_data [threadIdx.x][1] += svec2_data [threadIdx.x + offset][1];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0)
    {
        //rvec_Copy (results[blockIdx.x], svec_data[0]);
        results [blockIdx.x][0] += svec2_data [0][0];
        results [blockIdx.x][1] += svec2_data [0][1];
    }
#endif
}

CUDA_GLOBAL void k_dot (const real *a, const real *b, real *per_block_results, const size_t n )
{
#if defined(__SM_35__)

    extern __shared__ real my_dot[];
    real sdot;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdot = 0.0;
    if(i < n)
        sdot = a[i] * b[i];

    __syncthreads();

    for(int z = 16; z >=1; z/=2)
        sdot += shfl ( sdot, z);

    if (threadIdx.x % 32 == 0)
        my_dot[threadIdx.x >> 5] = sdot;

    __syncthreads ();

    for(int offset = blockDim.x >> 6; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset)
            my_dot[threadIdx.x] += my_dot[threadIdx.x + offset];

        __syncthreads();
    }

    if(threadIdx.x == 0)
        per_block_results[blockIdx.x] = my_dot[0];

#else

    extern __shared__ real sdot[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    real x = 0;

    if(i < n)
    {
        x = a[i] * b[i];
    }
    sdot[threadIdx.x] = x;
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            sdot[threadIdx.x] += sdot[threadIdx.x + offset];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0)
    {
        per_block_results[blockIdx.x] = sdot[0];
    }

#endif

}

CUDA_GLOBAL void k_norm (const real *input, real *per_block_results, const size_t n, int pass)
{
#if defined(__SM_35__)

    extern __shared__ real my_norm[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    real snorm = 0.0;

    if(i < n)
        snorm = SQR (input[i]);

    __syncthreads();

    for(int z = 16; z >=1; z/=2)
        snorm += shfl ( snorm, z);

    if (threadIdx.x % 32 == 0)
        my_norm[threadIdx.x >> 5] = snorm;

    __syncthreads ();

    for(int offset = blockDim.x >> 6; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset)
            my_norm[threadIdx.x] += my_norm[threadIdx.x + offset];

        __syncthreads();
    }

    if(threadIdx.x == 0)
        per_block_results[blockIdx.x] = my_norm[0];

#else
    extern __shared__ real snorm[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    real x = 0;

    if(i < n)
        x = SQR (input[i]);

    snorm[threadIdx.x] = x;
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            snorm[threadIdx.x] += snorm[threadIdx.x + offset];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0)
        per_block_results[blockIdx.x] = snorm[0];


#endif


}

CUDA_GLOBAL void k_norm_rvec2 (const rvec2 *input, rvec2 *per_block_results, const size_t n, int pass)
{
#if defined(__SM_35__)

    extern __shared__ rvec2 my_norm2[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    rvec2 snorm2;
    snorm2[0] = snorm2[1] = 0;

    if(i < n) {
        if (pass == INITIAL) {    
            snorm2[0] = SQR (input[i][0]);
            snorm2[1] = SQR (input[i][1]);
        } else {
            snorm2[0] = input[i][0];
            snorm2[1] = input[i][1];
        }
    }
    __syncthreads();

    for(int z = 16; z >=1; z/=2){
        snorm2[0] += shfl ( snorm2[0], z);
        snorm2[1] += shfl ( snorm2[1], z);
    }

    if (threadIdx.x % 32 == 0){
        my_norm2[threadIdx.x >> 5][0] = snorm2[0];
        my_norm2[threadIdx.x >> 5][1] = snorm2[1];
    }

    __syncthreads ();

    for(int offset = blockDim.x >> 6; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset){
            my_norm2[threadIdx.x][0] += my_norm2[threadIdx.x + offset][0];
            my_norm2[threadIdx.x][1] += my_norm2[threadIdx.x + offset][1];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0) {
        per_block_results[blockIdx.x][0] = my_norm2[0][0];
        per_block_results[blockIdx.x][1] = my_norm2[0][1];
    }

#else

    extern __shared__ rvec2 snorm2[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    rvec2 x;
    x[0] = x[1] = 0;

    if(i < n) {
        if (pass == INITIAL) {    
            x[0] = SQR (input[i][0]);
            x[1] = SQR (input[i][1]);
        } else {
            x[0] = input[i][0];
            x[1] = input[i][1];
        }
    }

    snorm2[threadIdx.x][0] = x[0];
    snorm2[threadIdx.x][1] = x[1];
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            snorm2[threadIdx.x][0] += snorm2[threadIdx.x + offset][0];
            snorm2[threadIdx.x][1] += snorm2[threadIdx.x + offset][1];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0) {
        per_block_results[blockIdx.x][0] = snorm2[0][0];
        per_block_results[blockIdx.x][1] = snorm2[0][1];
    }
#endif
}

CUDA_GLOBAL void k_dot_rvec2 (const rvec2 *a, rvec2 *b, rvec2 *res, const size_t n)
{
#if defined(__SM_35__)

    extern __shared__ rvec2 my_dot2[];
    rvec2 sdot2;

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdot2[0] = sdot2[1] = 0;

    if(i < n) {
        sdot2[0] = a[i][0] * b[i][0];
        sdot2[1] = a[i][1] * b[i][1];
    }

    __syncthreads();

    for(int z = 16; z >=1; z/=2){
        sdot2[0] += shfl ( sdot2[0], z);
        sdot2[1] += shfl ( sdot2[1], z);
    }

    if (threadIdx.x % 32 == 0){
        my_dot2[threadIdx.x >> 5][0] = sdot2[0];
        my_dot2[threadIdx.x >> 5][1] = sdot2[1];
    }

    __syncthreads ();

    for(int offset = blockDim.x >> 6; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset){
            my_dot2[threadIdx.x][0] += my_dot2[threadIdx.x + offset][0];
            my_dot2[threadIdx.x][1] += my_dot2[threadIdx.x + offset][1];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0) {
        res[blockIdx.x][0] = my_dot2[0][0];
        res[blockIdx.x][1] = my_dot2[0][1];
    }


#else
    extern __shared__ rvec2 sdot2[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    rvec2 x;
    x[0] = x[1] = 0;

    if(i < n) {
        x[0] = a[i][0] * b[i][0];
        x[1] = a[i][1] * b[i][1];
    }

    sdot2[threadIdx.x][0] = x[0];
    sdot2[threadIdx.x][1] = x[1];
    __syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            sdot2[threadIdx.x][0] += sdot2[threadIdx.x + offset][0];
            sdot2[threadIdx.x][1] += sdot2[threadIdx.x + offset][1];
        }

        __syncthreads();
    }

    if(threadIdx.x == 0) {
        res[blockIdx.x][0] = sdot2[0][0];
        res[blockIdx.x][1] = sdot2[0][1];
    }
#endif
}

//////////////////////////////////////////////////
//vector functions
//////////////////////////////////////////////////

CUDA_GLOBAL void k_vector_sum( real* dest, real c, real* v, real d, real* y, int k )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= k) return;

    dest[i] = c * v[i] + d * y[i];
}


CUDA_GLOBAL void k_vector_mul( real* dest, real* v, real* y, int k )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= k) return;

    dest[i] = v[i] * y[i];
}

CUDA_GLOBAL void k_rvec2_mul( rvec2* dest, rvec2* v, rvec2* y, int k )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= k) return;

    dest[i][0] = v[i][0] * y[i][0];
    dest[i][1] = v[i][1] * y[i][1];
}

CUDA_GLOBAL void k_rvec2_pbetad (rvec2 *dest, rvec2 *a, 
        real beta0, real beta1, 
        rvec2 *b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i >= n) return;

    dest[i][0] = a[i][0] + beta0 * b[i][0];
    dest[i][1] = a[i][1] + beta1 * b[i][1];
}

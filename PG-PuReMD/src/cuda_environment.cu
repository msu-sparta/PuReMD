#include "cuda_environment.h"

#include "cuda_utils.h"


extern "C" void Setup_Cuda_Environment(int rank, int nprocs, int gpus_per_node)
{

    int deviceCount;
    hipError_t flag;
    
    flag = hipGetDeviceCount(&deviceCount);

    if ( flag != hipSuccess )
    {
        fprintf( stderr, "ERROR: no CUDA capable device(s) found. Terminating...\n" );
        exit( CANNOT_INITIALIZE );
    }

    //Calculate the # of GPUs per processor
    //and assign the GPU for each process
    //TODO: handle condition where # CPU procs > # GPUs
    hipSetDevice( (rank % (deviceCount)) );

#if defined(__CUDA_DEBUG__)
    fprintf( stderr, "p:%d is using GPU: %d \n", rank, (rank % deviceCount));
#endif

    //CHANGE ORIGINAL
    //hipDeviceSetLimit( hipLimitStackSize, 8192 );
    //hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
    //cudaCheckError();
}


extern "C" void Cleanup_Cuda_Environment()
{
    hipDeviceReset();
    hipDeviceSynchronize();
}

#include "hip/hip_runtime.h"

#include "cuda_forces.h"

#include "reax_types.h"
#include "reax_types.h"
#include "dev_list.h"
#include "cuda_utils.h"
#include "cuda_helpers.h"
#include "index_utils.h"
#include "vector.h"

#include "cuda_neighbors.h"

#include "forces.h"
#include "cuda_bond_orders.h"
#include "reduction.h"
#include "cuda_bonds.h"
#include "cuda_multi_body.h"
#include "cuda_valence_angles.h"
#include "cuda_torsion_angles.h"
#include "cuda_hydrogen_bonds.h"
#include "tool_box.h"
#include "cuda_nonbonded.h"


extern "C" void Make_List( int, int, int, reax_list* );
extern "C" void Delete_List( reax_list* );


CUDA_GLOBAL void k_disable_hydrogen_bonding( control_params *control )
{
    control->hbond_cut = 0.0;
}


CUDA_GLOBAL void k_estimate_storages( reax_atom *my_atoms, 
        single_body_parameters *sbp, two_body_parameters *tbp,
        control_params *control, reax_list far_nbrs, 
        int num_atom_types, int n, int N, int Hcap, int total_cap,
        int *Htop, int *num_3body, int *bond_top, int *hb_top )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb;
    int local;
    real cutoff;
    real r_ij, r2; 
    real C12, C34, C56;
    real BO, BO_s, BO_pi, BO_pi2;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    atom_i = &(my_atoms[i]);
    type_i  = atom_i->type;
    start_i = Dev_Start_Index(i, &far_nbrs);
    end_i = Dev_End_Index(i, &far_nbrs);
    sbp_i = &(sbp[type_i]);

    if ( i < n )
    { 
        local = 1;
        cutoff = control->nonb_cut;
        atomicAdd( Htop, 1 );
        ihb = sbp_i->p_hbond;
    }   
    else
    {
        local = 0;
        cutoff = control->bond_cut;
        ihb = -1; 
    } 

    for ( pj = start_i; pj < end_i; ++pj )
    { 
        nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
        j = nbr_pj->nbr;
        atom_j = &(my_atoms[j]);

        if ( nbr_pj->d <= control->nonb_cut )
        {
            type_j = my_atoms[j].type;
            sbp_j = &(sbp[type_j]);
            ihb = sbp_i->p_hbond;
            jhb = sbp_j->p_hbond;
            if ( (control->hbond_cut > 0.1) 
                    && (nbr_pj->d <= control->hbond_cut) 
                    && (ihb == 2) && (jhb == 1) && (j < n) && (i > n) )
            {
                atomicAdd( &hb_top[i], 1 );
            }

            if ( i >= n )
            {
                ihb = -1;
            }
        }

        if ( nbr_pj->d <= cutoff )
        {
            type_j = my_atoms[j].type;
            r_ij = nbr_pj->d;
            sbp_j = &(sbp[type_j]);
            twbp = &(tbp[index_tbp (type_i,type_j,num_atom_types)]);

            if ( local )
            {
                if ( j < n || atom_i->orig_id < atom_j->orig_id ) //tryQEq ||1
                {
                    atomicAdd( Htop, 1 );
                }
                else if ( j < n || atom_i->orig_id > atom_j->orig_id ) //tryQEq ||1
                {
                    atomicAdd( Htop, 1 );
                }

                if ( control->hbond_cut > 0.1 && (ihb==1 || ihb==2) &&
                        nbr_pj->d <= control->hbond_cut )
                {
                    jhb = sbp_j->p_hbond;
                    if( ihb == 1 && jhb == 2 )
                    {
                        atomicAdd( &hb_top[i], 1 );
                    }
                    else if( ihb == 2 && jhb == 1 && j < n )
                    {
                        atomicAdd( &hb_top[i], 1 );
                    }
                }
            }

            // uncorrected bond orders 
            if ( nbr_pj->d <= control->bond_cut )
            {
                r2 = SQR( r_ij );

                if ( sbp_i->r_s > 0.0 && sbp_j->r_s > 0.0 )
                {
                    C12 = twbp->p_bo1 * POW( r_ij / twbp->r_s, twbp->p_bo2 );
                    BO_s = (1.0 + control->bo_cut) * EXP( C12 );
                }
                else
                {
                    BO_s = C12 = 0.0;
                }

                if ( sbp_i->r_pi > 0.0 && sbp_j->r_pi > 0.0 )
                {
                    C34 = twbp->p_bo3 * POW( r_ij / twbp->r_p, twbp->p_bo4 );
                    BO_pi = EXP( C34 );
                }
                else
                {
                    BO_pi = C34 = 0.0;
                }

                if ( sbp_i->r_pi_pi > 0.0 && sbp_j->r_pi_pi > 0.0 )
                {
                    C56 = twbp->p_bo5 * POW( r_ij / twbp->r_pp, twbp->p_bo6 );
                    BO_pi2= EXP( C56 );
                }
                else
                {
                    BO_pi2 = C56 = 0.0;
                }

                // Initially BO values are the uncorrected ones, page 1 
                BO = BO_s + BO_pi + BO_pi2;

                if ( BO >= control->bo_cut )
                {
                    atomicAdd( &bond_top[i], 1 );
                    //atomicAdd( &bond_top[j], 1 );
                }
            }
        }
    }
}


CUDA_GLOBAL void k_init_system_atoms(reax_atom *my_atoms, int N, 
        int *hb_top, int *bond_top)
{
    int i;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    my_atoms[i].num_bonds = bond_top [i];
    my_atoms[i].num_hbonds = hb_top [i];
}


void Cuda_Estimate_Storages(reax_system *system, control_params *control, 
        reax_list **lists, int local_cap, int total_cap,
        int *Htop, int *hb_top, int *bond_top, int *num_3body)
{
    int i;
    int blocks = 0;
    int *d_Htop, *d_hb_top, *d_bond_top, *d_num_3body;
    int * tmp = (int*) scratch;
    int bond_count = 0;
    int hbond_count = 0;
    int max_bonds = 0, min_bonds = 999999;
    int max_hbonds = 0, min_hbonds = 999999;

    *Htop = 0;
    //memset( hb_top, 0, sizeof(int) * local_cap);
    memset( hb_top, 0, sizeof(int) * total_cap );
    memset( bond_top, 0, sizeof(int) * total_cap );
    *num_3body = 0;
	
//    cuda_memset( tmp, 0,
//            1 + 1 + sizeof(int) * (local_cap+ total_cap), "Cuda_Estimate_Storages" );
    cuda_memset( tmp, 0, sizeof(int) *
            (1 + 1 + total_cap + total_cap), "Cuda_Estimate_Storages" );
 
    d_Htop = tmp; 
    d_num_3body = d_Htop + 1;
    d_hb_top = d_num_3body + 1;
    //d_bond_top = d_hb_top + local_cap;
    d_bond_top = d_hb_top + total_cap;
   
    blocks = (int) CEIL((real)system->N / ST_BLOCK_SIZE);

    k_estimate_storages <<< blocks, ST_BLOCK_SIZE>>>
        (system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp, 
         (control_params *)control->d_control_params,
         *(*dev_lists + FAR_NBRS), system->reax_param.num_atom_types,
         system->n, system->N, system->Hcap, system->total_cap, 
         d_Htop, d_num_3body, d_bond_top, d_hb_top );

    hipDeviceSynchronize( );
    cudaCheckError( );

    copy_host_device( Htop, d_Htop, sizeof(int),
            hipMemcpyDeviceToHost, "Htop");
    copy_host_device( num_3body, d_num_3body, sizeof(int),
            hipMemcpyDeviceToHost, "num_3body");
//    copy_host_device( hb_top, d_hb_top, sizeof(int) * local_cap,
//            hipMemcpyDeviceToHost, "hb_top");
    copy_host_device( hb_top, d_hb_top, sizeof(int) * total_cap,
            hipMemcpyDeviceToHost, "hb_top");
    copy_host_device( bond_top, d_bond_top, sizeof(int) * total_cap,
            hipMemcpyDeviceToHost, "bond_top");

    //TODO: change
    for ( i = 0; i < system->N; i++ )
    {
        if ( bond_top[i] >= max_bonds )
        {
            max_bonds = bond_top[i];
        }
        if ( bond_top[i] <= min_bonds )
        {
            min_bonds = bond_top[i];
        }

        bond_count += bond_top[i];
    }
    system->max_bonds = max_bonds * SAFER_ZONE;

    for (i = 0; i < system->N; i++)
    {
        if (hb_top[i] >= max_hbonds)
        {
            max_hbonds = hb_top[i];
        }
        if (hb_top[i] <= min_hbonds)
        {
            min_hbonds = hb_top[i];
        }

        hbond_count += hb_top[i];
    }
    system->max_hbonds = max_hbonds * SAFER_ZONE;

#if defined(DEBUG)
    fprintf( stderr, " TOTAL DEVICE BOND COUNT: %d \n", bond_count );
    fprintf( stderr, " TOTAL DEVICE HBOND COUNT: %d \n", hbond_count );
    fprintf( stderr, " TOTAL DEVICE SPARSE COUNT: %d \n", *Htop );
    fprintf( stderr, "p:%d --> Bonds(%d, %d) HBonds (%d, %d) *******\n", 
            system->my_rank, min_bonds, max_bonds, min_hbonds, max_hbonds );
#endif

    /* if number of hydrogen atoms is 0, disable hydrogen bond functionality */
    if ( hbond_count == 0 )
    {
        control->hbond_cut = 0.0;
        k_disable_hydrogen_bonding <<<1,1>>> ( (control_params *)control->d_control_params );
    }

    k_init_system_atoms <<<blocks, ST_BLOCK_SIZE>>>
        (system->d_my_atoms, system->N, d_hb_top, d_bond_top );

    hipDeviceSynchronize();
    cudaCheckError();
}


CUDA_DEVICE real Compute_H( real r, real gamma, real *ctap )
{
    real taper, dr3gamij_1, dr3gamij_3;

    taper = ctap[7] * r + ctap[6];
    taper = taper * r + ctap[5];
    taper = taper * r + ctap[4];
    taper = taper * r + ctap[3];
    taper = taper * r + ctap[2];
    taper = taper * r + ctap[1];
    taper = taper * r + ctap[0];    

    dr3gamij_1 = ( r*r*r + gamma );
    dr3gamij_3 = POW( dr3gamij_1 , 0.33333333333333 );

    return taper * EV_to_KCALpMOL / dr3gamij_3;
}


CUDA_DEVICE real Compute_tabH( LR_lookup_table *t_LR, real r_ij, int ti, int tj, int num_atom_types )
{
    int r, tmin, tmax;
    real val, dif, base;
    LR_lookup_table *t; 

    tmin  = MIN( ti, tj );
    tmax  = MAX( ti, tj );
    t = &( t_LR[index_lr (tmin,tmax, num_atom_types)] );    

    /* cubic spline interpolation */
    r = (int)(r_ij * t->inv_dx);
    if( r == 0 )
    {
        ++r;
    }
    base = (real)(r+1) * t->dx;
    dif = r_ij - base;
    val = ((t->ele[r].d*dif + t->ele[r].c)*dif + t->ele[r].b)*dif
            + t->ele[r].a;
    val *= EV_to_KCALpMOL / C_ele;

    return val;
}


CUDA_GLOBAL void k_estimate_sparse_matrix (reax_atom *my_atoms, control_params *control, 
        reax_list p_far_nbrs, int n, int N, int renbr, int *indices)
{
    int i, j, pj;
    int start_i, end_i;
    int flag;
    real cutoff;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;
    reax_list *far_nbrs = &( p_far_nbrs );

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    atom_i = &(my_atoms[i]);
    start_i = Dev_Start_Index(i, far_nbrs);
    end_i   = Dev_End_Index(i, far_nbrs);

    cutoff = control->nonb_cut;

    //++Htop;
    if ( i < n )
    {
        indices[i]++;
    }

    /* update i-j distance - check if j is within cutoff */
    for( pj = start_i; pj < end_i; ++pj )
    {
        nbr_pj = &( far_nbrs->select.far_nbr_list[pj] );
        j = nbr_pj->nbr;
        atom_j = &(my_atoms[j]);
        if( renbr )
        {
            if(nbr_pj->d <= cutoff)
            {
                flag = 1;
            }
            else
            {
                flag = 0;
            }
        }
        else
        {
            if (i < j)
            {
                nbr_pj->dvec[0] = atom_j->x[0] - atom_i->x[0];
                nbr_pj->dvec[1] = atom_j->x[1] - atom_i->x[1];
                nbr_pj->dvec[2] = atom_j->x[2] - atom_i->x[2];
            }
            else
            {
                nbr_pj->dvec[0] = atom_i->x[0] - atom_j->x[0];
                nbr_pj->dvec[1] = atom_i->x[1] - atom_j->x[1];
                nbr_pj->dvec[2] = atom_i->x[2] - atom_j->x[2];
            }
            nbr_pj->d = rvec_Norm_Sqr( nbr_pj->dvec );
            //TODO
            //if( nbr_pj->d <= (cutoff) ) {
            if( nbr_pj->d <= SQR(cutoff) )
            {
                nbr_pj->d = sqrt(nbr_pj->d);
                flag = 1;
            }
            else
            {
                flag = 0;
            }
        }

        if( flag )
        {
            /* H matrix entry */
            //if( j < n || atom_i->orig_id < atom_j->orig_id )
            //++Htop;
            //    indices [i] ++;
            //else if (j < n || atom_i->orig_id > atom_j->orig_id )
            //    indices [i] ++;

            //if ((i < n) || (j < n))
            //    indices [i] ++;
            //if ((i < n) && (i < j) && ((j < n) || atom_i->orig_id < atom_j->orig_id))
            //    indices [i] ++;
            //if ( i >= n && j < n && atom_i->orig_id > atom_j->orig_id)
            //    indices [i] ++;
            //else if ((i >=n) && (i > j) && ((j < n) || (atom_i->orig_id > atom_j->orig_id)))
            //    indices [i] ++;
            //THIS IS THE HOST CONDITION
            //if (i < n && i < j && ( j < n || atom_i->orig_id < atom_j->orig_id ))
            //if (i < n && i < j && atom_i->orig_id < atom_j->orig_id && j >=n)
            //    indices [i] ++;
            //THIS IS THE DEVICE CONDITION
            //if ( i > j && i >= n && j < n && atom_j->orig_id < atom_i->orig_id)
            //    indices [i] ++;

            //this is the working condition
            if (i < j && i < n && ( j < n || atom_i->orig_id < atom_j->orig_id))
            {
                indices[i]++;
            }
            else if (i > j && i >= n && j < n && atom_j->orig_id < atom_i->orig_id)
            {
                indices[i]++;
            }
            else if (i > j && i < n && ( j < n || atom_j->orig_id < atom_i->orig_id ))
            {
                indices[i]++;
            }
        }
    }
}


int Cuda_Estimate_Sparse_Matrix( reax_system *system, control_params *control, 
        simulation_data *data, reax_list **lists )
{
    int blocks, max_sp_entries;
    int *indices = (int *) scratch;
    int *h_indices = (int *) host_scratch;
    int total_sparse = 0;

    cuda_memset( indices, 0, sizeof(int) * system->N, "sp_matrix:indices" );

    blocks = system->N / DEF_BLOCK_SIZE + 
        ((system->N % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    //TODO
    k_estimate_sparse_matrix  <<< blocks, DEF_BLOCK_SIZE >>>
        (system->d_my_atoms, (control_params *)control->d_control_params, 
         *(*dev_lists + FAR_NBRS), system->n, system->N, 
         (((data->step-data->prev_steps) % control->reneighbor) == 0), indices);
    hipDeviceSynchronize( );
    cudaCheckError( );

    copy_host_device( h_indices, indices, sizeof(int) * system->N, 
            hipMemcpyDeviceToHost, "sp_matrix:indices" );
    max_sp_entries = 0;    
    for (int i = 0; i < system->N; i++)
    {
        total_sparse += h_indices [i];
        if (max_sp_entries < h_indices[i])
        {
            max_sp_entries = h_indices[i];
        }
    }

    //fprintf (stderr, " TOTAL DEVICE SPARSE ENTRIES: %d \n", total_sparse );
    //fprintf (stderr, "p%d: Max sparse entries -> %d \n", system->my_rank, max_sp_entries );
    system->max_sparse_entries = max_sp_entries * SAFE_ZONE;

    return SUCCESS;
}


CUDA_GLOBAL void k_init_forces( reax_atom *my_atoms, single_body_parameters *sbp, 
        two_body_parameters *tbp, storage workspace, control_params *control, 
        reax_list far_nbrs, reax_list bonds, reax_list hbonds, 
        LR_lookup_table *t_LR, int n, int N, int num_atom_types, 
        int max_sparse_entries, int renbr, int max_bonds, int max_hbonds )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    int Htop;
    int btop_i, ihb, jhb, ihb_top;
    //int btop_j, jhb, jhb_top;
    int local, flag, flag2, flag3;
    real r_ij, cutoff;
    //reax_list *far_nbrs, *bonds, *hbonds;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;
    sparse_matrix *H = &(workspace.H);

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    Htop = i * max_sparse_entries;
    btop_i = 0;

    //Commented for CUDA KERNEL
    //for( i = 0; i < system->N; ++i ) {
    atom_i = &(my_atoms[i]);
    type_i = atom_i->type;
    start_i = Dev_Start_Index( i, &far_nbrs );
    end_i = Dev_End_Index( i, &far_nbrs );
    //CHANGE ORIGINAL
    //btop_i = Dev_Start_Index( i, &bonds );
    btop_i = i * max_bonds;
    Dev_Set_Start_Index( i, btop_i, &bonds );
    //CHANGE ORIGINAL

    sbp_i = &(sbp[type_i]);

    if( i < n )
    {
        local = 1;
        cutoff = control->nonb_cut;

        //update bond mark here
        workspace.bond_mark [i] = 0;

    }
    else
    {
        local = 0;
        cutoff = control->bond_cut;

        //update bond mark here
        workspace.bond_mark [i] = 1000;
    }

    ihb = -1;
    ihb_top = -1;
    //CHANGE ORIGINAL
    H->start[i] = Htop;

    if( local )
    {
        H->entries[Htop].j = i;
        H->entries[Htop].val = sbp_i->eta;
        ++Htop;
    }
    //CHANGE ORIGINAL

    if( control->hbond_cut > 0.0 )
    {
        ihb = sbp_i->p_hbond;
        //CHANGE ORIGINAL
        if( ihb == 1  || ihb == 2)
        {
            //CHANGE ORIGINAL
            //ihb_top = Dev_Start_Index( atom_i->Hindex, &hbonds );
            ihb_top = i * max_hbonds;
            Dev_Set_Start_Index( atom_i->Hindex, ihb_top, &hbonds );
        }
        else
        {
            ihb_top = -1;
        }
    }

    /* update i-j distance - check if j is within cutoff */
    for( pj = start_i; pj < end_i; ++pj )
    {
        nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
        j = nbr_pj->nbr;
        atom_j = &(my_atoms[j]);
        if( renbr )
        {
            if(nbr_pj->d <= cutoff)
            {
                flag = 1;
            }
            else
            {
                flag = 0;
            }

            if(nbr_pj->d <= control->nonb_cut)
            {
                flag2 = 1;
            }
            else
            {
                flag2 = 0;
            }

        }
        else
        {
            if (i < j)
            {
                nbr_pj->dvec[0] = atom_j->x[0] - atom_i->x[0];
                nbr_pj->dvec[1] = atom_j->x[1] - atom_i->x[1];
                nbr_pj->dvec[2] = atom_j->x[2] - atom_i->x[2];
                nbr_pj->d = rvec_Norm_Sqr( nbr_pj->dvec );
            }
            else
            {
                nbr_pj->dvec[0] = atom_i->x[0] - atom_j->x[0];
                nbr_pj->dvec[1] = atom_i->x[1] - atom_j->x[1];
                nbr_pj->dvec[2] = atom_i->x[2] - atom_j->x[2];
                nbr_pj->d = rvec_Norm_Sqr( nbr_pj->dvec );
            }

            if(nbr_pj->d <= SQR (control->nonb_cut))
            {
                flag2 = 1;
            }
            else
            {
                flag2 = 0;
            }

            //if( nbr_pj->d <= SQR(cutoff) ) {
            if( nbr_pj->d <= SQR(control->nonb_cut) )
            {
                nbr_pj->d = sqrt(nbr_pj->d);
                flag = 1;
            }
            else
            {
                flag = 0;
            }
        }
        if (flag2)
        {
            ihb = sbp_i->p_hbond;
            type_j = atom_j->type;
            sbp_j = &(sbp[type_j]);
            jhb = sbp_j->p_hbond;
            if( control->hbond_cut > 0 && nbr_pj->d <= control->hbond_cut
                    && (ihb == 2) && (jhb == 1) && (i >= n) && (j < n) ) 
            {
                hbonds.select.hbond_list[ihb_top].nbr = j;
                hbonds.select.hbond_list[ihb_top].scl = -1;
                hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                //CUDA SPECIFIC
                hbonds.select.hbond_list[ihb_top].sym_index = -1;
                rvec_MakeZero( hbonds.select.hbond_list[ihb_top].hb_f );

                ++ihb_top;
            }

            //if ((i < n) || (j < n))
            //if (local || ((i >= n) &&(j < n)))

            flag3 = false;
            if (i < j && i < n && ( j < n || atom_i->orig_id < atom_j->orig_id))
            {
                flag3 = true;
            }
            else if (i > j && i >= n && j < n && atom_j->orig_id < atom_i->orig_id)
            {
                flag3 = true;
            }
            else if (i > j && i < n && ( j < n || atom_j->orig_id < atom_i->orig_id ))
            {
                flag3 = true;
            }

            if ( flag3 )
            {
                twbp = &(tbp[ index_tbp (type_i,type_j,num_atom_types)]);
                r_ij = nbr_pj->d;

                //if (renbr) {
                H->entries[Htop].j = j;
                if( control->tabulate == 0 )
                {
                    H->entries[Htop].val = Compute_H(r_ij,twbp->gamma,workspace.Tap);
                }
                else
                {
                    H->entries[Htop].val = Compute_tabH(t_LR, r_ij, type_i, type_j,num_atom_types);
                }
                //}
                ++Htop;
            }
        }

        if( flag )
        {
            type_j = atom_j->type;
            r_ij = nbr_pj->d;
            sbp_j = &(sbp[type_j]);
            twbp = &(tbp[ index_tbp (type_i,type_j,num_atom_types)]);

            if ( local )
            {
                /* H matrix entry */
                /*
                   if( j < n || atom_i->orig_id < atom_j->orig_id ) {//tryQEq||1
                   H->entries[Htop].j = j;
                   if( control->tabulate == 0 )
                   H->entries[Htop].val = Compute_H(r_ij,twbp->gamma,workspace.Tap);
                   else H->entries[Htop].val = Compute_tabH(t_LR, r_ij, type_i, type_j,num_atom_types);
                   ++Htop;
                   } 
                   else if( j < n || atom_i->orig_id > atom_j->orig_id ) {//tryQEq||1
                   H->entries[Htop].j = j;
                   if( control->tabulate == 0 )
                   H->entries[Htop].val = Compute_H(r_ij,twbp->gamma,workspace.Tap);
                   else H->entries[Htop].val = Compute_tabH(t_LR, r_ij, type_i, type_j,num_atom_types);
                   ++Htop;
                   } 
                 */

                //bool condition = !((i >= n) && (j >= n));
                /* hydrogen bond lists */
                if( control->hbond_cut > 0 && (ihb==1 || ihb==2) &&
                        nbr_pj->d <= control->hbond_cut // && i < j
                  )
                {
                    jhb = sbp_j->p_hbond;
                    if( ihb == 1 && jhb == 2 )
                    {
                        hbonds.select.hbond_list[ihb_top].nbr = j;
                        if (i < j) 
                        {
                            hbonds.select.hbond_list[ihb_top].scl = 1;
                        }
                        else
                        {
                            hbonds.select.hbond_list[ihb_top].scl = -1;
                        }
                        hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                        //CUDA SPECIFIC
                        hbonds.select.hbond_list[ihb_top].sym_index = -1;
                        rvec_MakeZero( hbonds.select.hbond_list[ihb_top].hb_f );

                        ++ihb_top;
                    }
                    //else if( j < n && ihb == 2 && jhb == 1 ) 
                    else if( ihb == 2 && jhb == 1 && j < n)
                    {
                        //jhb_top = End_Index( atom_j->Hindex, hbonds );
                        hbonds.select.hbond_list[ihb_top].nbr = j;
                        hbonds.select.hbond_list[ihb_top].scl = -1;
                        hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                        //CUDA SPECIFIC
                        hbonds.select.hbond_list[ihb_top].sym_index = -1;
                        rvec_MakeZero (hbonds.select.hbond_list[ihb_top].hb_f);

                        ++ihb_top;

                        //Set_End_Index( atom_j->Hindex, jhb_top+1, hbonds );
                        //++num_hbonds;
                    }
                }
            }

            /* uncorrected bond orders */
            if( nbr_pj->d <= control->bond_cut 
                    && Dev_BOp( bonds, control->bo_cut, 
                        i , btop_i, nbr_pj, sbp_i, sbp_j, twbp, 
                        workspace.dDeltap_self, workspace.total_bond_order) )
            {
                //num_bonds += 2;
                ++btop_i;

                /* Need to do later... since i and j are parallel
                   if( workspace->bond_mark[j] > workspace->bond_mark[i] + 1 )
                   workspace->bond_mark[j] = workspace->bond_mark[i] + 1;
                   else if( workspace->bond_mark[i] > workspace->bond_mark[j] + 1 ) {
                   workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
                   }
                 */
            }
        }
    }

    Dev_Set_End_Index( i, btop_i, &bonds );
    //    if( local ) {
    H->end[i] = Htop;
    //   }
    //CHANGE ORIGINAL
    if(( ihb == 1 || ihb == 2 ) && (ihb_top > 0) && (control->hbond_cut > 0))
    {
        Dev_Set_End_Index( atom_i->Hindex, ihb_top, &hbonds );
    }
    //} Commented for cuda kernel
}


CUDA_GLOBAL void k_init_bond_mark( int offset, int n, int *bond_mark )
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
    {
        return;
    }

    bond_mark[offset + threadIdx.x] = 1000;
}


CUDA_GLOBAL void New_fix_sym_dbond_indices( reax_list pbonds, int N )
{
    int i, nbr;
    bond_data *ibond, *jbond;
    int atom_j;

    reax_list *bonds = &pbonds;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    for (int j = Dev_Start_Index(i, bonds); j < Dev_End_Index(i, bonds); j++)
    {
        ibond = &( bonds->select.bond_list [j] );
        nbr = ibond->nbr;

        for (int k = Dev_Start_Index(nbr, bonds); k < Dev_End_Index(nbr, bonds); k++)
        {
            jbond = &( bonds->select.bond_list[k] );
            atom_j = jbond->nbr;

            if ( (atom_j == i) )
            {
                if (i > nbr)
                {
                    ibond->dbond_index = j;
                    jbond->dbond_index = j;

                    ibond->sym_index = k;
                    jbond->sym_index = j;
                }
            }
        }
    }
}


CUDA_GLOBAL void New_fix_sym_hbond_indices( reax_atom *my_atoms, reax_list hbonds, int N )
{
    hbond_data *ihbond, *jhbond;

    int __THREADS_PER_ATOM__ = HB_KER_SYM_THREADS_PER_ATOM;
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = thread_id / __THREADS_PER_ATOM__;
    int lane_id = thread_id & (__THREADS_PER_ATOM__ - 1);
    int my_bucket = threadIdx.x / __THREADS_PER_ATOM__;

    if (warp_id > N)
    {
        return;
    }

    int i = warp_id;
    int nbr;
    int k;
    int start = Dev_Start_Index( my_atoms[i].Hindex, &hbonds );
    int end = Dev_End_Index( my_atoms[i].Hindex, &hbonds );
    int j = start + lane_id;

    while (j < end)
    {
        ihbond = &( hbonds.select.hbond_list [j] );
        nbr = ihbond->nbr;

        int nbrstart = Dev_Start_Index (my_atoms[nbr].Hindex, &hbonds);
        int nbrend = Dev_End_Index (my_atoms[nbr].Hindex, &hbonds);

        for (k = nbrstart; k < nbrend; k++)
        {
            jhbond = &( hbonds.select.hbond_list [k] );

            if (jhbond->nbr == i)
            {
                ihbond->sym_index = k;
                jhbond->sym_index = j;
                break;
            }
        }

        j += __THREADS_PER_ATOM__;
    }
}


////////////////////////
// HBOND ISSUE
CUDA_GLOBAL void k_update_bonds( reax_atom *my_atoms, 
        reax_list bonds, int n )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n)
    {
        return;
    }

    my_atoms [i].num_bonds = 
        MAX( Dev_Num_Entries(i, &bonds) * 2, MIN_BONDS );
}


CUDA_GLOBAL void k_update_hbonds( reax_atom *my_atoms, 
        reax_list hbonds, int n )
{
    int Hindex;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n)
    {
        return;
    }

    Hindex = my_atoms[i].Hindex;
    my_atoms [i].num_hbonds = 
        MAX( Dev_Num_Entries(Hindex, &hbonds) * SAFER_ZONE, MIN_HBONDS );
}
////////////////////////
////////////////////////
////////////////////////


int Cuda_Validate_Lists( reax_system *system, storage *workspace,
        reax_list **lists, control_params *control, 
        int step, int n, int N, int numH )
{
    int blocks;
    int i, comp, Hindex;
    int *index, *end_index;
    reax_list *bonds, *hbonds;
    reax_atom *my_atoms;
    reallocate_data *realloc;
    realloc = &( dev_workspace->realloc );

    int max_sp_entries, num_hbonds, num_bonds;
    int total_sp_entries;
    int max_bonds, max_hbonds;

    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    k_update_bonds <<< blocks, DEF_BLOCK_SIZE >>>
        (system->d_my_atoms, *(*lists + BONDS), 
         system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );

    ////////////////////////
    // HBOND ISSUE
    //FIX - 4 - Added this check for hydrogen bond issue
    if ((control->hbond_cut > 0) && (system->numH > 0))
    {
        k_update_hbonds <<< blocks, DEF_BLOCK_SIZE >>>
            (system->d_my_atoms, *(*lists + HBONDS), 
             system->n);
        hipDeviceSynchronize( );
        cudaCheckError( );
    }

    //validate sparse matrix entries.
    memset( host_scratch, 0, 2 * system->N * sizeof (int) );
    index = (int *) host_scratch;
    end_index = index + system->N;
    copy_host_device (index, dev_workspace->H.start, system->N * sizeof (int), 
            hipMemcpyDeviceToHost, "sparse_matrix:start" );
    copy_host_device (end_index, dev_workspace->H.end, system->N * sizeof (int), 
            hipMemcpyDeviceToHost, "sparse_matrix:end" );
    max_sp_entries = total_sp_entries = 0;
    for (i = 0; i < N; i++ )
    {
        //if (i < N-1)
        //    comp = index [i+1];
        //else
        //    comp = dev_workspace->H.m;

        total_sp_entries += end_index [i] - index[i];
        if (end_index [i] - index[i] > system->max_sparse_entries)
        {
            fprintf( stderr, "step%d-sparsemat-chk failed: i=%d start(i)=%d end(i)=%d \n",
                    step, i, index[i], end_index[i] );
            return FAILURE;
        }
        else if (end_index[i] >= dev_workspace->H.m)
        {
            //SUDHIR_FIX_SPARSE_MATRIX
            //TODO move this carver
            fprintf( stderr, "p:%d - step%d-sparsemat-chk failed (exceed limits): i=%d start(i)=%d end(i)=%d \n", 
                    system->my_rank, step, i, index[i], end_index[i] );
            //TODO move this carver
            return FAILURE;
        }
        else
        {
            if ( max_sp_entries <= end_index[i] - index [i] )
            {
                max_sp_entries = end_index[i] - index [i];
            }
        }
    }
    //if (max_sp_entries <= end_index[i] - index [i])
    //    max_sp_entries = end_index[i] - index [i];

    //update the current step max_sp_entries;
    realloc->Htop = max_sp_entries;

#if defined(DEBUG)
    fprintf( stderr, "p:%d - Cuda_Reallocate: Total H matrix entries: %d, cap: %d, used: %d \n", 
            system->my_rank, dev_workspace->H.n, dev_workspace->H.m, total_sp_entries );
#endif

    if (total_sp_entries >= dev_workspace->H.m)
    {
        fprintf( stderr, "p:%d - **ran out of space for sparse matrix: step: %d, allocated: %d, used: %d \n", 
                system->my_rank, step, dev_workspace->H.m, total_sp_entries );

        return FAILURE;
    }

    //validate Bond list
    if ( N > 0 )
    {
        num_bonds = 0;

        bonds = *lists + BONDS;
        memset (host_scratch, 0, 2 * bonds->n * sizeof(int));
        index = (int *) host_scratch;
        end_index = index + bonds->n;

        copy_host_device( index, bonds->index, bonds->n * sizeof(int), 
                hipMemcpyDeviceToHost, "bonds:index" );
        copy_host_device( end_index, bonds->end_index, bonds->n * sizeof(int), 
                hipMemcpyDeviceToHost, "bonds:end_index" );

        /*
           for (i = 0; i < N; i++) {
           if (i < N-1)
           comp = index [i+1];
           else
           comp = bonds->num_intrs;

           if (end_index [i] > comp) {
           fprintf( stderr, "step%d-bondchk failed: i=%d start(i)=%d end(i)=%d str(i+1)=%d\n",
           step, i, index[i], end_index[i], comp );
           return FAILURE;
           }

           num_bonds += MAX( (end_index[i] - index[i]) * 4, MIN_BONDS);
           }

           if (end_index[N-1] >= bonds->num_intrs) {
           fprintf( stderr, "step%d-bondchk failed(end): i=N-1 start(i)=%d end(i)=%d num_intrs=%d\n",
           step, index[N-1], end_index[N-1], bonds->num_intrs);
           return FAILURE;
           }
           num_bonds = MAX( num_bonds, MIN_CAP*MIN_BONDS );
        //check the condition for reallocation
        realloc->num_bonds = num_bonds;
         */

        max_bonds = 0;
        for (i = 0; i < N; i++)
        {
            if ( end_index[i] - index[i] >= system->my_atoms[i].max_bonds )
            {
                fprintf( stderr, "step%d-bondchk failed: i=%d start(i)=%d end(i)=%d max_bonds=%d\n",
                        step, i, index[i], end_index[i], system->my_atoms[i].max_bonds );
                return FAILURE;
            }
            if ( end_index[i] - index[i] >= max_bonds )
            {
                max_bonds = end_index[i] - index[i];
            }
        }
        realloc->num_bonds = max_bonds;
    }

    //validate Hbonds list
    num_hbonds = 0;
    // FIX - 4 - added additional check here
    if ((numH > 0) && (control->hbond_cut > 0))
    {
        hbonds = *lists + HBONDS;
        memset( host_scratch, 0,
                2 * hbonds->n * sizeof(int) + sizeof(reax_atom) * system->N );
        index = (int *) host_scratch;
        end_index = index + hbonds->n;
        my_atoms = (reax_atom *)(end_index + hbonds->n);

        copy_host_device( index, hbonds->index, hbonds->n * sizeof(int), 
                hipMemcpyDeviceToHost, "hbonds:index" );
        copy_host_device( end_index, hbonds->end_index, hbonds->n * sizeof(int), 
                hipMemcpyDeviceToHost, "hbonds:end_index" );
        copy_host_device( my_atoms, system->d_my_atoms, system->N * sizeof(reax_atom), 
                hipMemcpyDeviceToHost, "system:d_my_atoms" );

        //fprintf (stderr, " Total local atoms: %d \n", n);

        /*
           for (i = 0; i < N-1; i++) {
           Hindex = my_atoms [i].Hindex;
           if (Hindex > -1) 
           comp = index [Hindex + 1];
           else
           comp = hbonds->num_intrs;

           if (end_index [Hindex] > comp) {
           fprintf(stderr,"step%d-atom:%d hbondchk failed: H=%d start(H)=%d end(H)=%d str(H+1)=%d\n",
           step, i, Hindex, index[Hindex], end_index[Hindex], comp );
           return FAILURE;
           }

           num_hbonds += MAX( (end_index [Hindex] - index [Hindex]) * 2, MIN_HBONDS * 2);
           }
           if (end_index [my_atoms[i].Hindex] > hbonds->num_intrs) {
           fprintf(stderr,"step%d-atom:%d hbondchk failed: H=%d start(H)=%d end(H)=%d num_intrs=%d\n",
           step, i, Hindex, index[Hindex], end_index[Hindex], hbonds->num_intrs);
           return FAILURE;
           }

           num_hbonds += MIN( (end_index [my_atoms[i].Hindex] - index [my_atoms[i].Hindex]) * 2, 
           2 * MIN_HBONDS);
           num_hbonds = MAX( num_hbonds, MIN_CAP*MIN_HBONDS );
           realloc->num_hbonds = num_hbonds;
         */

        max_hbonds = 0;
        for (i = 0; i < N; i++)
        {
            if (end_index[i] - index[i] >= system->max_hbonds)
            {
                //TODO: update
//                fprintf( stderr, "step%d-hbondchk failed: i=%d start(i)=%d end(i)=%d max_hbonds=%d\n",
//                        step, i, index[i], end_index[i], system->max_hbonds );
//                return FAILURE;
            }
            if (end_index[i] - index[i] >= max_hbonds)
            {
                max_hbonds = end_index[i] - index[i];
            }
        }
        realloc->num_hbonds = max_hbonds;
    }

    return SUCCESS;
}


CUDA_GLOBAL void k_init_bond_orders( reax_atom *my_atoms, reax_list far_nbrs, 
        reax_list bonds, real *total_bond_order, int N )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= N)
    {
        return;
    }

    atom_i = &(my_atoms[i]);
    start_i = Dev_Start_Index(i, &far_nbrs);
    end_i = Dev_End_Index(i, &far_nbrs);

    for( pj = start_i; pj < end_i; ++pj )
    { 
        // nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
        // j = nbr_pj->nbr;
        // atom_j = &(my_atoms[j]);

        //total_bond_order [i] ++;
        //atom_i->Hindex ++;
    }
}


CUDA_GLOBAL void k_bond_mark( reax_list p_bonds, storage p_workspace, int N )
{
    reax_list *bonds = &( p_bonds );
    storage *workspace = &( p_workspace );
    int j;

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //if (i >= N) return;

    for (int i = 0; i < N; i++) 
    {
        for (int k = Dev_Start_Index (i, bonds); k < Dev_End_Index (i, bonds); k++)
        {
            bond_data *bdata = &( bonds->select.bond_list [k] );
            j = bdata->nbr;

            if (i < j )
            {
                if ( workspace->bond_mark [j] > (workspace->bond_mark [i] + 1) )
                {
                    workspace->bond_mark [j] = workspace->bond_mark [i] + 1;    
                }
                else if ( workspace->bond_mark [i] > (workspace->bond_mark [j] + 1) )
                {
                    workspace->bond_mark [i] = workspace->bond_mark [j] + 1;
                }
            }
        }
    }
}


int Cuda_Init_Forces( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    int init_blocks;
    int hblocks;

    //init the workspace (bond_mark)
    /*
       int blocks;
       cuda_memset (dev_workspace->bond_mark, 0, sizeof (int) * system->n, "bond_mark");

       blocks = (system->N - system->n) / DEF_BLOCK_SIZE + 
       (((system->N - system->n) % DEF_BLOCK_SIZE == 0) ? 0 : 1);
       k_init_bond_mark <<< blocks, DEF_BLOCK_SIZE >>>
       (system->n, (system->N - system->n), dev_workspace->bond_mark);
       hipDeviceSynchronize ();
       cudaCheckError ();
     */
    //validate total_bond_orders

    //main kernel
    init_blocks = (system->N) / DEF_BLOCK_SIZE + 
        (((system->N % DEF_BLOCK_SIZE) == 0) ? 0 : 1);
    //fprintf (stderr, " Total atoms: %d, blocks: %d \n", system->N, init_blocks );

    //    k_init_bond_orders <<<init_blocks, DEF_BLOCK_SIZE >>>
    //            ( system->d_my_atoms, *(*dev_lists + FAR_NBRS), *(*dev_lists + BONDS), 
    //                dev_workspace->total_bond_order, system->N);
    //    hipDeviceSynchronize ();
    //    cudaCheckError ();
    //    fprintf (stderr, " DONE WITH VALIDATION \n");

    k_init_forces <<<init_blocks, DEF_BLOCK_SIZE >>>
        (system->d_my_atoms, system->reax_param.d_sbp, 
         system->reax_param.d_tbp, *dev_workspace, 
         (control_params *)control->d_control_params, 
         *(*dev_lists + FAR_NBRS), *(*dev_lists + BONDS), *(*dev_lists + HBONDS), 
         d_LR, system->n, system->N, system->reax_param.num_atom_types, 
         //system->max_sparse_entries, ((data->step-data->prev_steps) % control->reneighbor));
        system->max_sparse_entries, (((data->step-data->prev_steps) % control->reneighbor) == 0), 
        system->max_bonds, system->max_hbonds);
    hipDeviceSynchronize( );
    cudaCheckError( );

    //fix - sym_index and dbond_index
    New_fix_sym_dbond_indices <<<init_blocks, BLOCK_SIZE>>> 
        (*(*dev_lists + BONDS), system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );

    ///////////////////////
    ///////////////////////
    // FIX - 4 - HBOND ISSUE
    if ((control->hbond_cut > 0 ) && (system->numH > 0))
    {
        //make hbond_list symmetric
        hblocks = (system->N * HB_KER_SYM_THREADS_PER_ATOM) / HB_SYM_BLOCK_SIZE + 
            ((((system->N * HB_KER_SYM_THREADS_PER_ATOM) % HB_SYM_BLOCK_SIZE) == 0) ? 0 : 1);
        //New_fix_sym_hbond_indices <<<hblocks, HB_BLOCK_SIZE >>> 
        New_fix_sym_hbond_indices <<<hblocks, HB_BLOCK_SIZE >>> 
            (system->d_my_atoms, *(*dev_lists + HBONDS), system->N);
        hipDeviceSynchronize( );
        cudaCheckError( );
    }

    //update bond_mark
    //k_bond_mark <<< init_blocks, DEF_BLOCK_SIZE>>>
    /*
       k_bond_mark <<< 1, 1>>>
       ( *(*dev_lists + BONDS), *dev_workspace, system->N);
       hipDeviceSynchronize ();
       cudaCheckError ();
     */

    //TODO
    //1. update the sparse matrix count for reallocation
    //2. update the bonds count for reallocation
    //3. update the hydrogen bonds count for reallocation

    //Validate lists here.
    return Cuda_Validate_Lists( system, workspace, dev_lists, control, 
            data->step, system->n, system->N, system->numH );
}


int Cuda_Init_Forces_No_Charges( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    //TODO Implement later
    // when you figure out the bond_mark usage.

    return FAILURE;
}


void Cuda_Compute_Bonded_Forces (reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control )
{
    real t_start, t_elapsed;
    real *spad = (real *) scratch;
    rvec *rvec_spad;

    //1. Bond Order Interactions. - bond_orders.c
    t_start = Get_Time( );
    //fprintf (stderr, " Begin Bonded Forces ... %d x %d\n", BLOCKS_N, BLOCK_SIZE);
    Cuda_Calculate_BO_init  <<< BLOCKS_N, BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, 
          *dev_workspace, 
          system->N );
    hipDeviceSynchronize ();
    cudaCheckError ();

    Cuda_Calculate_BO <<< BLOCKS_N, BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, 
          system->reax_param.d_tbp, *dev_workspace, 
          *(*dev_lists + BONDS),
          system->reax_param.num_atom_types, system->N );
    hipDeviceSynchronize ();
    cudaCheckError ();


    Cuda_Update_Uncorrected_BO <<<BLOCKS_N, BLOCK_SIZE>>>
        (*dev_workspace, *(*dev_lists + BONDS), system->N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    Cuda_Update_Workspace_After_BO <<<BLOCKS_N, BLOCK_SIZE>>>
        (system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, 
         *dev_workspace, system->N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "Bond Orders... return value --> %d --- Timing %lf \n", hipGetLastError (), t_elapsed );
    //fprintf (stderr, "Cuda_Calculate_Bond_Orders Done... \n");

    //2. Bond Energy Interactions. - bonds.c
    t_start = Get_Time( );
    cuda_memset (spad, 0, system->N * ( 2 * sizeof (real)) , "scratch");

    Cuda_Bonds <<< BLOCKS, BLOCK_SIZE, sizeof (real)* BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, system->reax_param.d_tbp,
          *dev_workspace, *(*dev_lists + BONDS), 
          system->n, system->reax_param.num_atom_types, spad );
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for E_BE
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>  
        (spad, spad + system->n,  system->n);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2>>> 
        (spad + system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_bond, BLOCKS_POW_2);
    hipDeviceSynchronize ();
    cudaCheckError ();

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "Cuda_Bond_Energy ... return value --> %d --- Timing %lf \n", hipGetLastError (), t_elapsed );
    //fprintf (stderr, "Cuda_Bond_Energy Done... \n");


    //3. Atom Energy Interactions. 
    t_start = Get_Time( );
    cuda_memset (spad, 0, ( 6 * sizeof (real) * system->n ), "scratch");

    Cuda_Atom_Energy <<<BLOCKS, BLOCK_SIZE>>>( system->d_my_atoms, system->reax_param.d_gp, 
            system->reax_param.d_sbp, system->reax_param.d_tbp, 
            *dev_workspace, 
            *(*dev_lists + BONDS), system->n, system->reax_param.num_atom_types, 
            spad, spad + 2 * system->n, spad + 4*system->n);
    hipDeviceSynchronize ();
    cudaCheckError ();

    //CHANGE ORIGINAL
    //Cuda_Atom_Energy_PostProcess     <<<BLOCKS, BLOCK_SIZE >>>
    //                    ( *(*dev_lists + BONDS), *dev_workspace, system->n );
    Cuda_Atom_Energy_PostProcess     <<<BLOCKS_N, BLOCK_SIZE >>>
        ( *(*dev_lists + BONDS), *dev_workspace, system->N );
    //CHANGE ORIGINAL
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for E_Lp
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>  
        (spad, spad + system->n,  system->n);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>  
        (spad + system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_lp, BLOCKS);
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for E_Ov
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>  
        (spad + 2*system->n, spad + 3*system->n,  system->n);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>  
        (spad + 3*system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_ov, BLOCKS);
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for E_Un
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>  
        (spad + 4*system->n, spad + 5*system->n,  system->n);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>  
        (spad + 5*system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_un, BLOCKS);
    hipDeviceSynchronize ();
    cudaCheckError ();

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "test_LonePair_postprocess ... return value --> %d --- Timing %lf \n", hipGetLastError (), t_elapsed );
    //fprintf (stderr, "test_LonePair_postprocess Done... \n");


    //4. Valence Angles Interactions. 
    t_start = Get_Time( );

    //THREE BODY CHANGES HERE
    cuda_memset(spad, 0, (*dev_lists + BONDS)->num_intrs * sizeof (int), "scratch");
    Estimate_Cuda_Valence_Angles <<<BLOCKS_N, BLOCK_SIZE>>>
        (system->d_my_atoms, 
         (control_params *)control->d_control_params, 
         *(*dev_lists + BONDS),
         system->n, system->N, (int *)spad);
    hipDeviceSynchronize ();
    cudaCheckError ();


    int *thbody = (int *) host_scratch;
    memset (thbody, 0, sizeof (int) * (*dev_lists + BONDS)->num_intrs);
    copy_host_device (thbody, spad, (*dev_lists + BONDS)->num_intrs * sizeof (int), hipMemcpyDeviceToHost, "thb:offsets");

    int total_3body = thbody [0] * SAFE_ZONE;
    for (int x = 1; x < (*dev_lists + BONDS)->num_intrs; x++)
    {
        total_3body += thbody [x]*SAFE_ZONE;
        thbody [x] += thbody [x-1];
    }

    system->num_thbodies = thbody [(*dev_lists+BONDS)->num_intrs-1];
    if (!system->init_thblist) 
    {
        system->init_thblist = true;
        Dev_Make_List( (*dev_lists+BONDS)->num_intrs, total_3body, TYP_THREE_BODY, (*dev_lists + THREE_BODIES) );
        Make_List( (*dev_lists+BONDS)->num_intrs, total_3body, TYP_THREE_BODY, (*lists + THREE_BODIES) );

#ifdef __CUDA_MEM__
        fprintf (stderr, "Device memory allocated: three body list = %d MB\n", 
                sizeof (three_body_interaction_data) * total_3body / (1024*1024));
#endif
    }
    else
    {
        //if (((dev_workspace->realloc.num_bonds * DANGER_ZONE) >= (*dev_lists+BONDS)->num_intrs) || 
        //        (system->num_thbodies > (*dev_lists+THREE_BODIES)->num_intrs )) { 
        //int size = dev_workspace->realloc.num_bonds;
        if ((system->num_thbodies >= (*dev_lists+THREE_BODIES)->num_intrs ) || 
                ((*dev_lists+THREE_BODIES)->n < (*dev_lists+BONDS)->num_intrs) )
        {
            int size = (*dev_lists + BONDS)->num_intrs;

            /*Delete Three-body list*/
            Dev_Delete_List( *dev_lists + THREE_BODIES );
            Delete_List ( *lists + THREE_BODIES );

            fprintf( stderr, "p%d ***** Reallocating the Three-body list threebody.n: %d, bonds.num_intrs: %d, num_thb: %d, thb_entries: %d \n", 
                    system->my_rank, (*dev_lists+THREE_BODIES)->n, (*dev_lists+BONDS)->num_intrs, 
                    system->num_thbodies, (*dev_lists+THREE_BODIES)->num_intrs );

#ifdef __CUDA_MEM__
            fprintf( stderr, "Reallocating Three-body list: step: %d n - %d num_intrs - %d used: %d \n", 
                    data->step, dev_workspace->realloc.num_bonds, total_3body, system->num_thbodies );
#endif

            /*Recreate Three-body list */
            Dev_Make_List( size, total_3body, TYP_THREE_BODY, *dev_lists + THREE_BODIES );
            Make_List( size, total_3body, TYP_THREE_BODY, *lists + THREE_BODIES );
        }
    }

    //copy the indexes into the thb list;
    copy_host_device( thbody, ((*dev_lists + THREE_BODIES)->index + 1), sizeof (int) * ((*dev_lists+BONDS)->num_intrs - 1),
            hipMemcpyHostToDevice, "thb:index" );
    copy_host_device( thbody, ((*dev_lists + THREE_BODIES)->end_index + 1), sizeof (int) * ((*dev_lists+BONDS)->num_intrs - 1),
            hipMemcpyHostToDevice, "thb:end_index" );
    //THREE_BODY CHANGES HERE

    cuda_memset (spad, 0, ( 6 * sizeof (real) * system->N + sizeof (rvec) * system->N * 2), "scratch");
    Cuda_Valence_Angles <<< BLOCKS_N, BLOCK_SIZE >>>
        ( system->d_my_atoms,
          system->reax_param.d_gp, 
          system->reax_param.d_sbp, system->reax_param.d_thbp, 
          (control_params *)control->d_control_params,
          *dev_workspace, 
          *(*dev_lists + BONDS), *(*dev_lists + THREE_BODIES),
          system->n, system->N, system->reax_param.num_atom_types, 
          spad, spad + 2*system->N, spad + 4*system->N, (rvec *)(spad + 6*system->N));
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for E_Ang
    k_reduction <<<BLOCKS_N, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>  
        (spad, spad + system->N,  system->N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction <<<1, BLOCKS_POW_2_N, sizeof (real) * BLOCKS_POW_2_N >>>
        (spad + system->N, &((simulation_data *)data->d_simulation_data)->my_en.e_ang, BLOCKS_N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for E_Pen
    k_reduction <<<BLOCKS_N, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>
        (spad + 2*system->N, spad + 3*system->N,  system->N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction <<<1, BLOCKS_POW_2_N, sizeof (real) * BLOCKS_POW_2_N >>>
        (spad + 3*system->N, &((simulation_data *)data->d_simulation_data)->my_en.e_pen, BLOCKS_N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for E_Coa
    k_reduction <<<BLOCKS_N, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>
        (spad + 4*system->N, spad + 5*system->N,  system->N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction <<<1, BLOCKS_POW_2_N, sizeof (real) * BLOCKS_POW_2_N >>>
        (spad + 5*system->N, &((simulation_data *)data->d_simulation_data)->my_en.e_coa, BLOCKS_N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for ext_pres
    rvec_spad = (rvec *) (spad + 6*system->N);
    k_reduction_rvec <<<BLOCKS_N, BLOCK_SIZE, sizeof (rvec) * BLOCK_SIZE >>>
        (rvec_spad, rvec_spad + system->N,  system->N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    k_reduction_rvec <<<1, BLOCKS_POW_2_N, sizeof (rvec) * BLOCKS_POW_2_N >>>
        (rvec_spad + system->N, &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS_N);
    hipDeviceSynchronize ();
    cudaCheckError ();

    Cuda_Valence_Angles_PostProcess <<< BLOCKS_N, BLOCK_SIZE >>>
        (  system->d_my_atoms,
           (control_params *)control->d_control_params,
           *dev_workspace,
           *(*dev_lists + BONDS),
           system->N );
    hipDeviceSynchronize ();
    cudaCheckError ();

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "Three_Body_Interactions ...  Timing %lf \n", t_elapsed );
    //fprintf (stderr, "Three_Body_Interactions Done... \n");

    //5. Torsion Angles Interactions. 
    t_start = Get_Time( );
    cuda_memset (spad, 0, ( 4 * sizeof (real) * system->n + sizeof (rvec) * system->n * 2), "scratch");
    Cuda_Torsion_Angles <<< BLOCKS, BLOCK_SIZE >>>
        ( system->d_my_atoms,
          system->reax_param.d_gp,
          system->reax_param.d_fbp,
          (control_params *)control->d_control_params,
          *(*dev_lists + BONDS), *(*dev_lists + THREE_BODIES),
          *dev_workspace,
          system->n, system->reax_param.num_atom_types, 
          spad, spad + 2*system->n, (rvec *) (spad + 4*system->n));
    hipDeviceSynchronize ();
    cudaCheckError ();

    //Reduction for E_Tor
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>
        (spad, spad + system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>
        (spad + system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_tor, BLOCKS);
    hipDeviceSynchronize( );
    cudaCheckError( );

    //Reduction for E_Con
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>
        (spad + 2*system->n, spad + 3*system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>
        (spad + 3*system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_con, BLOCKS);
    hipDeviceSynchronize( );
    cudaCheckError( );

    //Reduction for ext_pres
    rvec_spad = (rvec *) (spad + 4*system->n);
    k_reduction_rvec <<<BLOCKS, BLOCK_SIZE, sizeof (rvec) * BLOCK_SIZE >>>
        (rvec_spad, rvec_spad + system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );

    k_reduction_rvec <<<1, BLOCKS_POW_2, sizeof (rvec) * BLOCKS_POW_2 >>>
            ( rvec_spad + system->n,
            &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS );
    hipDeviceSynchronize( );
    cudaCheckError( );

    //Post process here
    Cuda_Torsion_Angles_PostProcess   <<< BLOCKS_N, BLOCK_SIZE >>>
            ( system->d_my_atoms, *dev_workspace, *(*dev_lists + BONDS),
            system->N );
    hipDeviceSynchronize( );
    cudaCheckError( );

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "Four_Body_post process return value --> %d --- Four body Timing %lf \n", hipGetLastError (), t_elapsed );
    //fprintf (stderr, " Four_Body_ Done... \n");

    //6. Hydrogen Bonds Interactions.
    // FIX - 4 - Added additional check here
    if ((control->hbond_cut > 0) && (system->numH > 0))
    {
        t_start = Get_Time( );
        cuda_memset( spad, 0,
                2 * sizeof(real) * system->n + sizeof(rvec) * system->n * 2, "scratch" );

        int hbs = ((system->n * HB_KER_THREADS_PER_ATOM)/ HB_BLOCK_SIZE) + 
            (((system->n * HB_KER_THREADS_PER_ATOM) % HB_BLOCK_SIZE) == 0 ? 0 : 1);
        Cuda_Hydrogen_Bonds_MT <<<hbs, HB_BLOCK_SIZE, 
                HB_BLOCK_SIZE * (2 * sizeof(real) + 2 * sizeof(rvec)) >>>
        //Cuda_Hydrogen_Bonds <<< BLOCKS, BLOCK_SIZE>>>
                        ( system->d_my_atoms, system->reax_param.d_sbp,
                        system->reax_param.d_hbp, system->reax_param.d_gp,
                        (control_params *)control->d_control_params,
                        *dev_workspace, *(*dev_lists + BONDS), *(*dev_lists + HBONDS),
                        system->n, system->reax_param.num_atom_types,
                        spad, (rvec *) (spad + 2*system->n));
        hipDeviceSynchronize( );
        cudaCheckError( );

        //Reduction for E_HB
        k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof(real) * BLOCK_SIZE >>>
            (spad, spad + system->n,  system->n);
        hipDeviceSynchronize( );
        cudaCheckError( );

        k_reduction <<<1, BLOCKS_POW_2, sizeof(real) * BLOCKS_POW_2 >>>
            (spad + system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_hb, BLOCKS);
        hipDeviceSynchronize( );
        cudaCheckError( );


        //Reduction for ext_pres
        rvec_spad = (rvec *) (spad + 2*system->n);
        k_reduction_rvec <<<BLOCKS, BLOCK_SIZE, sizeof (rvec) * BLOCK_SIZE >>>
            (rvec_spad, rvec_spad + system->n,  system->n);
        hipDeviceSynchronize( );
        cudaCheckError( );

        k_reduction_rvec <<<1, BLOCKS_POW_2, sizeof (rvec) * BLOCKS_POW_2 >>>
            (rvec_spad + system->n, &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS);
        hipDeviceSynchronize( );
        cudaCheckError( );

        //post process step1:
        Cuda_Hydrogen_Bonds_PostProcess <<< BLOCKS_N, BLOCK_SIZE, BLOCK_SIZE * sizeof (rvec) >>>
            (  system->d_my_atoms, *dev_workspace,
               *(*dev_lists + BONDS), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );

        //post process step2:
        /*
           Cuda_Hydrogen_Bonds_HNbrs <<< system->N, 32, 32 * sizeof (rvec)>>>
           (  system->d_my_atoms,
         *dev_workspace,
         *(*dev_lists + HBONDS));
         */
        int hnbrs_bl = ((system->N * HB_POST_PROC_KER_THREADS_PER_ATOM)/ HB_POST_PROC_BLOCK_SIZE) + 
            (((system->N * HB_POST_PROC_KER_THREADS_PER_ATOM) % HB_POST_PROC_BLOCK_SIZE) == 0 ? 0 : 1);
        Cuda_Hydrogen_Bonds_HNbrs_BL <<< hnbrs_bl, HB_POST_PROC_BLOCK_SIZE, 
                HB_POST_PROC_BLOCK_SIZE * sizeof (rvec)>>>
                        ( system->d_my_atoms, *dev_workspace,
                        *(*dev_lists + HBONDS), system->N);
        hipDeviceSynchronize( );
        cudaCheckError( );

        t_elapsed = Get_Timing_Info( t_start );
        //fprintf (stderr, "Hydrogen bonds return value --> %d --- HydrogenBonds Timing %lf \n", hipGetLastError (), t_elapsed );
        //fprintf (stderr, "Hydrogen_Bond Done... \n");    
    }
}


void Cuda_Compute_NonBonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control,
        mpi_datatypes *mpi_data )
{
    /* van der Waals and Coulomb interactions */
    Cuda_NonBonded_Energy( system, control, workspace, data,
            lists, out_control, (control->tabulate == 0) ? false: true );
}

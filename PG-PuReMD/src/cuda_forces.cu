#include "hip/hip_runtime.h"

#include "cuda_forces.h"

#include "reax_types.h"
#include "reax_types.h"
#include "dev_list.h"
#include "cuda_utils.h"
#include "cuda_helpers.h"
#include "index_utils.h"
#include "vector.h"

#include "cuda_neighbors.h"

#include "forces.h"
#include "cuda_bond_orders.h"
#include "reduction.h"
#include "cuda_bonds.h"
#include "cuda_multi_body.h"
#include "cuda_valence_angles.h"
#include "cuda_torsion_angles.h"
#include "cuda_hydrogen_bonds.h"
#include "tool_box.h"
#include "cuda_nonbonded.h"


extern "C" void Make_List( int, int, int, reax_list* );
extern "C" void Delete_List( reax_list* );


CUDA_GLOBAL void k_disable_hydrogen_bonding( control_params *control )
{
    control->hbond_cut = 0.0;
}


CUDA_GLOBAL void k_estimate_storages( reax_atom *my_atoms, 
        single_body_parameters *sbp, two_body_parameters *tbp,
        control_params *control, reax_list far_nbrs, 
        int num_atom_types, int n, int N, int Hcap, int total_cap,
        int *Htop, int *bond_top, int *hb_top )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    int ihb, jhb;
    int local;
    real cutoff;
    real r_ij, r2; 
    real C12, C34, C56;
    real BO, BO_s, BO_pi, BO_pi2;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    atom_i = &(my_atoms[i]);
    type_i  = atom_i->type;
    start_i = Dev_Start_Index(i, &far_nbrs);
    end_i = Dev_End_Index(i, &far_nbrs);
    sbp_i = &(sbp[type_i]);

    if ( i < n )
    { 
        local = 1;
        cutoff = control->nonb_cut;
        atomicAdd( Htop, 1 );
        ihb = sbp_i->p_hbond;
    }   
    else
    {
        local = 0;
        cutoff = control->bond_cut;
        ihb = -1; 
    } 

    for ( pj = start_i; pj < end_i; ++pj )
    { 
        nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
        j = nbr_pj->nbr;
        atom_j = &(my_atoms[j]);

        if ( nbr_pj->d <= control->nonb_cut )
        {
            type_j = my_atoms[j].type;
            sbp_j = &(sbp[type_j]);
            ihb = sbp_i->p_hbond;
            jhb = sbp_j->p_hbond;
            if ( (control->hbond_cut > 0.1) 
                    && (nbr_pj->d <= control->hbond_cut) 
                    && (ihb == 2) && (jhb == 1) && (j < n) && (i > n) )
            {
                atomicAdd( &hb_top[i], 1 );
            }

            if ( i >= n )
            {
                ihb = -1;
            }
        }

        if ( nbr_pj->d <= cutoff )
        {
            type_j = my_atoms[j].type;
            r_ij = nbr_pj->d;
            sbp_j = &(sbp[type_j]);
            twbp = &(tbp[index_tbp (type_i,type_j,num_atom_types)]);

            if ( local )
            {
                if ( j < n || atom_i->orig_id < atom_j->orig_id ) //tryQEq ||1
                {
                    atomicAdd( Htop, 1 );
                }
                else if ( j < n || atom_i->orig_id > atom_j->orig_id ) //tryQEq ||1
                {
                    atomicAdd( Htop, 1 );
                }

                if ( control->hbond_cut > 0.1 && (ihb==1 || ihb==2) &&
                        nbr_pj->d <= control->hbond_cut )
                {
                    jhb = sbp_j->p_hbond;
                    if( ihb == 1 && jhb == 2 )
                    {
                        atomicAdd( &hb_top[i], 1 );
                    }
                    else if( ihb == 2 && jhb == 1 && j < n )
                    {
                        atomicAdd( &hb_top[i], 1 );
                    }
                }
            }

            // uncorrected bond orders 
            if ( nbr_pj->d <= control->bond_cut )
            {
                r2 = SQR( r_ij );

                if ( sbp_i->r_s > 0.0 && sbp_j->r_s > 0.0 )
                {
                    C12 = twbp->p_bo1 * POW( r_ij / twbp->r_s, twbp->p_bo2 );
                    BO_s = (1.0 + control->bo_cut) * EXP( C12 );
                }
                else
                {
                    BO_s = C12 = 0.0;
                }

                if ( sbp_i->r_pi > 0.0 && sbp_j->r_pi > 0.0 )
                {
                    C34 = twbp->p_bo3 * POW( r_ij / twbp->r_p, twbp->p_bo4 );
                    BO_pi = EXP( C34 );
                }
                else
                {
                    BO_pi = C34 = 0.0;
                }

                if ( sbp_i->r_pi_pi > 0.0 && sbp_j->r_pi_pi > 0.0 )
                {
                    C56 = twbp->p_bo5 * POW( r_ij / twbp->r_pp, twbp->p_bo6 );
                    BO_pi2= EXP( C56 );
                }
                else
                {
                    BO_pi2 = C56 = 0.0;
                }

                // Initially BO values are the uncorrected ones, page 1 
                BO = BO_s + BO_pi + BO_pi2;

                if ( BO >= control->bo_cut )
                {
                    atomicAdd( &bond_top[i], 1 );
                    //atomicAdd( &bond_top[j], 1 );
                }
            }
        }
    }
}


CUDA_GLOBAL void k_init_system_atoms( reax_atom *my_atoms, int N, 
        int *hb_top, int *bond_top )
{
    int i;
    
    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    my_atoms[i].num_bonds = bond_top[i];
    my_atoms[i].num_hbonds = hb_top[i];
}


void Cuda_Estimate_Storages( reax_system *system, control_params *control, 
        reax_list **lists, int *Htop, int *hb_top, int *bond_top )
{
    int i;
    int blocks = 0;
    int *d_Htop, *d_hb_top, *d_bond_top;
    int * tmp = (int*) scratch;
    int bond_count = 0;
    int hbond_count = 0;
    int max_hbonds = 0, min_hbonds = 999999;

    *Htop = 0;
    memset( hb_top, 0, sizeof(int) * system->total_cap );
    memset( bond_top, 0, sizeof(int) * system->total_cap );
	
    cuda_memset( tmp, 0, sizeof(int) *
            (1 + system->total_cap + system->total_cap), "Cuda_Estimate_Storages" );
 
    d_Htop = tmp; 
    d_hb_top = d_Htop + 1;
    d_bond_top = d_hb_top + system->total_cap;
   
    blocks = (int) CEIL((real)system->N / ST_BLOCK_SIZE);

    k_estimate_storages <<< blocks, ST_BLOCK_SIZE>>>
        (system->d_my_atoms, system->reax_param.d_sbp, system->reax_param.d_tbp, 
         (control_params *)control->d_control_params,
         *(*dev_lists + FAR_NBRS), system->reax_param.num_atom_types,
         system->n, system->N, system->Hcap, system->total_cap, 
         d_Htop, d_bond_top, d_hb_top );
    hipDeviceSynchronize( );
    cudaCheckError( );

    copy_host_device( Htop, d_Htop, sizeof(int),
            hipMemcpyDeviceToHost, "Htop");
    copy_host_device( hb_top, d_hb_top, sizeof(int) * system->total_cap,
            hipMemcpyDeviceToHost, "hb_top");
    copy_host_device( bond_top, d_bond_top, sizeof(int) * system->total_cap,
            hipMemcpyDeviceToHost, "bond_top");

    for ( i = 0; i < system->total_cap; i++ )
    {
        system->max_bonds[i] = MAX( (int)CEIL( bond_top[i] * SAFE_ZONE ), MIN_BONDS );
        bond_count += system->max_bonds[i];
    }

    //TODO: change
    for ( i = 0; i < system->N; i++ )
    {
        if ( hb_top[i] >= max_hbonds )
        {
            max_hbonds = hb_top[i];
        }
        if ( hb_top[i] <= min_hbonds )
        {
            min_hbonds = hb_top[i];
        }

        hbond_count += hb_top[i];
    }
    system->max_hbonds = max_hbonds * SAFER_ZONE;

#if defined(DEBUG)
    fprintf( stderr, "p:%d -->\n", system->my_rank );
    fprintf( stderr, " TOTAL DEVICE BOND COUNT: %d \n", bond_count );
    fprintf( stderr, " TOTAL DEVICE HBOND COUNT: %d \n", hbond_count );
    fprintf( stderr, " TOTAL DEVICE SPARSE COUNT: %d \n", *Htop );
#endif

    /* if number of hydrogen atoms is 0, disable hydrogen bond functionality */
    if ( hbond_count == 0 )
    {
        control->hbond_cut = 0.0;
        k_disable_hydrogen_bonding <<<1,1>>> ( (control_params *)control->d_control_params );
    }

    k_init_system_atoms <<<blocks, ST_BLOCK_SIZE>>>
        (system->d_my_atoms, system->N, d_hb_top, d_bond_top );

    hipDeviceSynchronize( );
    cudaCheckError( );
}


void Cuda_Estimate_Storages_Three_Body( reax_system *system, control_params *control, 
        reax_list **lists, int *num_3body, int *thbody )
{
    int i;
    real *spad = (real *) scratch;

    cuda_memset( spad, 0, (*dev_lists + BONDS)->num_intrs * sizeof(int), "scratch" );

    Estimate_Cuda_Valence_Angles <<<BLOCKS_N, BLOCK_SIZE>>>
        ( system->d_my_atoms, (control_params *)control->d_control_params, 
          *(*dev_lists + BONDS), system->n, system->N, (int *)spad );
    hipDeviceSynchronize( );
    cudaCheckError( );

    copy_host_device( thbody, spad, (*dev_lists + BONDS)->num_intrs * sizeof(int),
            hipMemcpyDeviceToHost, "thb:offsets" );

    *num_3body = 0;
    for ( i = 0; i < (*dev_lists + BONDS)->num_intrs; i++ )
    {
        *num_3body += thbody[i];
        thbody[i] += thbody[i - 1];
    }

    system->num_thbodies = thbody[(*dev_lists + BONDS)->num_intrs - 1];
}


CUDA_DEVICE real Compute_H( real r, real gamma, real *ctap )
{
    real taper, dr3gamij_1, dr3gamij_3;

    taper = ctap[7] * r + ctap[6];
    taper = taper * r + ctap[5];
    taper = taper * r + ctap[4];
    taper = taper * r + ctap[3];
    taper = taper * r + ctap[2];
    taper = taper * r + ctap[1];
    taper = taper * r + ctap[0];    

    dr3gamij_1 = ( r*r*r + gamma );
    dr3gamij_3 = POW( dr3gamij_1 , 1.0 / 3.0 );

    return taper * EV_to_KCALpMOL / dr3gamij_3;
}


CUDA_DEVICE real Compute_tabH( LR_lookup_table *t_LR, real r_ij, int ti, int tj, int num_atom_types )
{
    int r, tmin, tmax;
    real val, dif, base;
    LR_lookup_table *t; 

    tmin  = MIN( ti, tj );
    tmax  = MAX( ti, tj );
    t = &( t_LR[index_lr (tmin,tmax, num_atom_types)] );    

    /* cubic spline interpolation */
    r = (int)(r_ij * t->inv_dx);
    if( r == 0 )
    {
        ++r;
    }
    base = (real)(r+1) * t->dx;
    dif = r_ij - base;
    val = ((t->ele[r].d*dif + t->ele[r].c)*dif + t->ele[r].b)*dif
            + t->ele[r].a;
    val *= EV_to_KCALpMOL / C_ele;

    return val;
}


CUDA_GLOBAL void k_estimate_sparse_matrix (reax_atom *my_atoms, control_params *control, 
        reax_list p_far_nbrs, int n, int N, int renbr, int *indices)
{
    int i, j, pj;
    int start_i, end_i;
    int flag;
    real cutoff;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;
    reax_list *far_nbrs = &( p_far_nbrs );

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    atom_i = &(my_atoms[i]);
    start_i = Dev_Start_Index(i, far_nbrs);
    end_i   = Dev_End_Index(i, far_nbrs);

    cutoff = control->nonb_cut;

    //++Htop;
    if ( i < n )
    {
        indices[i]++;
    }

    /* update i-j distance - check if j is within cutoff */
    for( pj = start_i; pj < end_i; ++pj )
    {
        nbr_pj = &( far_nbrs->select.far_nbr_list[pj] );
        j = nbr_pj->nbr;
        atom_j = &(my_atoms[j]);
        if( renbr )
        {
            if(nbr_pj->d <= cutoff)
            {
                flag = 1;
            }
            else
            {
                flag = 0;
            }
        }
        else
        {
            if (i < j)
            {
                nbr_pj->dvec[0] = atom_j->x[0] - atom_i->x[0];
                nbr_pj->dvec[1] = atom_j->x[1] - atom_i->x[1];
                nbr_pj->dvec[2] = atom_j->x[2] - atom_i->x[2];
            }
            else
            {
                nbr_pj->dvec[0] = atom_i->x[0] - atom_j->x[0];
                nbr_pj->dvec[1] = atom_i->x[1] - atom_j->x[1];
                nbr_pj->dvec[2] = atom_i->x[2] - atom_j->x[2];
            }
            nbr_pj->d = rvec_Norm_Sqr( nbr_pj->dvec );
            //TODO
            //if( nbr_pj->d <= (cutoff) ) {
            if( nbr_pj->d <= SQR(cutoff) )
            {
                nbr_pj->d = sqrt(nbr_pj->d);
                flag = 1;
            }
            else
            {
                flag = 0;
            }
        }

        if( flag )
        {
            /* H matrix entry */
            //if( j < n || atom_i->orig_id < atom_j->orig_id )
            //++Htop;
            //    indices [i] ++;
            //else if (j < n || atom_i->orig_id > atom_j->orig_id )
            //    indices [i] ++;

            //if ((i < n) || (j < n))
            //    indices [i] ++;
            //if ((i < n) && (i < j) && ((j < n) || atom_i->orig_id < atom_j->orig_id))
            //    indices [i] ++;
            //if ( i >= n && j < n && atom_i->orig_id > atom_j->orig_id)
            //    indices [i] ++;
            //else if ((i >=n) && (i > j) && ((j < n) || (atom_i->orig_id > atom_j->orig_id)))
            //    indices [i] ++;
            //THIS IS THE HOST CONDITION
            //if (i < n && i < j && ( j < n || atom_i->orig_id < atom_j->orig_id ))
            //if (i < n && i < j && atom_i->orig_id < atom_j->orig_id && j >=n)
            //    indices [i] ++;
            //THIS IS THE DEVICE CONDITION
            //if ( i > j && i >= n && j < n && atom_j->orig_id < atom_i->orig_id)
            //    indices [i] ++;

            //this is the working condition
            if (i < j && i < n && ( j < n || atom_i->orig_id < atom_j->orig_id))
            {
                indices[i]++;
            }
            else if (i > j && i >= n && j < n && atom_j->orig_id < atom_i->orig_id)
            {
                indices[i]++;
            }
            else if (i > j && i < n && ( j < n || atom_j->orig_id < atom_i->orig_id ))
            {
                indices[i]++;
            }
        }
    }
}


int Cuda_Estimate_Sparse_Matrix( reax_system *system, control_params *control, 
        simulation_data *data, reax_list **lists )
{
    int blocks, max_sp_entries;
    int *indices = (int *) scratch;
    int *h_indices = (int *) host_scratch;
    int total_sparse = 0;

    cuda_memset( indices, 0, sizeof(int) * system->N, "sp_matrix:indices" );

    blocks = system->N / DEF_BLOCK_SIZE + 
        ((system->N % DEF_BLOCK_SIZE == 0) ? 0 : 1);

    //TODO
    k_estimate_sparse_matrix  <<< blocks, DEF_BLOCK_SIZE >>>
        (system->d_my_atoms, (control_params *)control->d_control_params, 
         *(*dev_lists + FAR_NBRS), system->n, system->N, 
         (((data->step-data->prev_steps) % control->reneighbor) == 0), indices);
    hipDeviceSynchronize( );
    cudaCheckError( );

    copy_host_device( h_indices, indices, sizeof(int) * system->N, 
            hipMemcpyDeviceToHost, "sp_matrix:indices" );
    max_sp_entries = 0;    
    for (int i = 0; i < system->N; i++)
    {
        total_sparse += h_indices [i];
        if (max_sp_entries < h_indices[i])
        {
            max_sp_entries = h_indices[i];
        }
    }

    //fprintf (stderr, " TOTAL DEVICE SPARSE ENTRIES: %d \n", total_sparse );
    //fprintf (stderr, "p%d: Max sparse entries -> %d \n", system->my_rank, max_sp_entries );
    system->max_sparse_entries = max_sp_entries * SAFE_ZONE;

    return SUCCESS;
}


CUDA_GLOBAL void k_init_forces( reax_atom *my_atoms, single_body_parameters *sbp, 
        two_body_parameters *tbp, storage workspace, control_params *control, 
        reax_list far_nbrs, reax_list bonds, reax_list hbonds, 
        LR_lookup_table *t_LR, int n, int N, int num_atom_types, 
        int max_sparse_entries, int renbr, int max_hbonds )
{
    int i, j, pj;
    int start_i, end_i;
    int type_i, type_j;
    int Htop;
    int btop_i, ihb, jhb, ihb_top;
    //int btop_j, jhb, jhb_top;
    int local, flag, flag2, flag3;
    real r_ij, cutoff;
    //reax_list *far_nbrs, *bonds, *hbonds;
    single_body_parameters *sbp_i, *sbp_j;
    two_body_parameters *twbp;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;
    sparse_matrix *H = &(workspace.H);

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    Htop = i * max_sparse_entries;
    btop_i = 0;

    //Commented for CUDA KERNEL
    //for( i = 0; i < system->N; ++i ) {
    atom_i = &(my_atoms[i]);
    type_i = atom_i->type;
    start_i = Dev_Start_Index( i, &far_nbrs );
    end_i = Dev_End_Index( i, &far_nbrs );
    btop_i = Dev_Start_Index( i, &bonds );

    sbp_i = &(sbp[type_i]);

    if( i < n )
    {
        local = 1;
        cutoff = control->nonb_cut;

        //update bond mark here
        workspace.bond_mark[i] = 0;
    }
    else
    {
        local = 0;
        cutoff = control->bond_cut;

        //update bond mark here
        workspace.bond_mark[i] = 1000;
    }

    ihb = -1;
    ihb_top = -1;
    //CHANGE ORIGINAL
    H->start[i] = Htop;

    if( local )
    {
        H->entries[Htop].j = i;
        H->entries[Htop].val = sbp_i->eta;
        ++Htop;
    }
    //CHANGE ORIGINAL

    if( control->hbond_cut > 0.0 )
    {
        ihb = sbp_i->p_hbond;
        //CHANGE ORIGINAL
        if( ihb == 1  || ihb == 2)
        {
            //CHANGE ORIGINAL
            //ihb_top = Dev_Start_Index( atom_i->Hindex, &hbonds );
            ihb_top = i * max_hbonds;
            Dev_Set_Start_Index( atom_i->Hindex, ihb_top, &hbonds );
        }
        else
        {
            ihb_top = -1;
        }
    }

    /* update i-j distance - check if j is within cutoff */
    for( pj = start_i; pj < end_i; ++pj )
    {
        nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
        j = nbr_pj->nbr;
        atom_j = &(my_atoms[j]);
        if( renbr )
        {
            if(nbr_pj->d <= cutoff)
            {
                flag = TRUE;
            }
            else
            {
                flag = FALSE;
            }

            if(nbr_pj->d <= control->nonb_cut)
            {
                flag2 = TRUE;
            }
            else
            {
                flag2 = FALSE;
            }

        }
        else
        {
            if (i < j)
            {
                nbr_pj->dvec[0] = atom_j->x[0] - atom_i->x[0];
                nbr_pj->dvec[1] = atom_j->x[1] - atom_i->x[1];
                nbr_pj->dvec[2] = atom_j->x[2] - atom_i->x[2];
                nbr_pj->d = rvec_Norm_Sqr( nbr_pj->dvec );
            }
            else
            {
                nbr_pj->dvec[0] = atom_i->x[0] - atom_j->x[0];
                nbr_pj->dvec[1] = atom_i->x[1] - atom_j->x[1];
                nbr_pj->dvec[2] = atom_i->x[2] - atom_j->x[2];
                nbr_pj->d = rvec_Norm_Sqr( nbr_pj->dvec );
            }

            if( nbr_pj->d <= SQR( control->nonb_cut ) )
            {
                flag2 = TRUE;
            }
            else
            {
                flag2 = FALSE;
            }

            if( nbr_pj->d <= SQR( control->nonb_cut ) )
            {
                nbr_pj->d = SQRT( nbr_pj->d );
                flag = TRUE;
            }
            else
            {
                flag = FALSE;
            }
        }
        if ( flag2 == TRUE )
        {
            ihb = sbp_i->p_hbond;
            type_j = atom_j->type;
            sbp_j = &(sbp[type_j]);
            jhb = sbp_j->p_hbond;
            if( control->hbond_cut > 0.0 && nbr_pj->d <= control->hbond_cut
                    && ihb == 2 && jhb == 1 && i >= n && j < n ) 
            {
                hbonds.select.hbond_list[ihb_top].nbr = j;
                hbonds.select.hbond_list[ihb_top].scl = -1;
                hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                //CUDA SPECIFIC
                hbonds.select.hbond_list[ihb_top].sym_index = -1;
                rvec_MakeZero( hbonds.select.hbond_list[ihb_top].hb_f );

                ++ihb_top;
            }

            //if ((i < n) || (j < n))
            //if (local || ((i >= n) &&(j < n)))

            flag3 = FALSE;
            if (i < j && i < n && ( j < n || atom_i->orig_id < atom_j->orig_id))
            {
                flag3 = TRUE;
            }
            else if (i > j && i >= n && j < n && atom_j->orig_id < atom_i->orig_id)
            {
                flag3 = TRUE;
            }
            else if (i > j && i < n && ( j < n || atom_j->orig_id < atom_i->orig_id ))
            {
                flag3 = TRUE;
            }

            if ( flag3 == TRUE )
            {
                twbp = &(tbp[ index_tbp (type_i,type_j,num_atom_types)]);
                r_ij = nbr_pj->d;

                //if (renbr) {
                H->entries[Htop].j = j;
                if( control->tabulate == 0 )
                {
                    H->entries[Htop].val = Compute_H(r_ij,twbp->gamma,workspace.Tap);
                }
                else
                {
                    H->entries[Htop].val = Compute_tabH(t_LR, r_ij, type_i, type_j,num_atom_types);
                }
                //}
                ++Htop;
            }
        }

        if( flag == TRUE )
        {
            type_j = atom_j->type;
            r_ij = nbr_pj->d;
            sbp_j = &(sbp[type_j]);
            twbp = &(tbp[ index_tbp (type_i,type_j,num_atom_types)]);

            if ( local )
            {
                /* H matrix entry */
                /*
                   if( j < n || atom_i->orig_id < atom_j->orig_id ) {//tryQEq||1
                   H->entries[Htop].j = j;
                   if( control->tabulate == 0 )
                   H->entries[Htop].val = Compute_H(r_ij,twbp->gamma,workspace.Tap);
                   else H->entries[Htop].val = Compute_tabH(t_LR, r_ij, type_i, type_j,num_atom_types);
                   ++Htop;
                   } 
                   else if( j < n || atom_i->orig_id > atom_j->orig_id ) {//tryQEq||1
                   H->entries[Htop].j = j;
                   if( control->tabulate == 0 )
                   H->entries[Htop].val = Compute_H(r_ij,twbp->gamma,workspace.Tap);
                   else H->entries[Htop].val = Compute_tabH(t_LR, r_ij, type_i, type_j,num_atom_types);
                   ++Htop;
                   } 
                 */

                //bool condition = !((i >= n) && (j >= n));
                /* hydrogen bond lists */
                if( control->hbond_cut > 0 && (ihb==1 || ihb==2) &&
                        nbr_pj->d <= control->hbond_cut // && i < j
                  )
                {
                    jhb = sbp_j->p_hbond;
                    if( ihb == 1 && jhb == 2 )
                    {
                        hbonds.select.hbond_list[ihb_top].nbr = j;
                        if (i < j) 
                        {
                            hbonds.select.hbond_list[ihb_top].scl = 1;
                        }
                        else
                        {
                            hbonds.select.hbond_list[ihb_top].scl = -1;
                        }
                        hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                        //CUDA SPECIFIC
                        hbonds.select.hbond_list[ihb_top].sym_index = -1;
                        rvec_MakeZero( hbonds.select.hbond_list[ihb_top].hb_f );

                        ++ihb_top;
                    }
                    else if( ihb == 2 && jhb == 1 && j < n )
                    {
                        //jhb_top = End_Index( atom_j->Hindex, hbonds );
                        hbonds.select.hbond_list[ihb_top].nbr = j;
                        hbonds.select.hbond_list[ihb_top].scl = -1;
                        hbonds.select.hbond_list[ihb_top].ptr = nbr_pj;

                        //CUDA SPECIFIC
                        hbonds.select.hbond_list[ihb_top].sym_index = -1;
                        rvec_MakeZero (hbonds.select.hbond_list[ihb_top].hb_f);

                        ++ihb_top;

                        //Set_End_Index( atom_j->Hindex, jhb_top+1, hbonds );
                        //++num_hbonds;
                    }
                }
            }

            /* uncorrected bond orders */
            if( nbr_pj->d <= control->bond_cut &&
                    Dev_BOp( bonds, control->bo_cut, i , btop_i, nbr_pj,
                        sbp_i, sbp_j, twbp, workspace.dDeltap_self,
                        workspace.total_bond_order ) == TRUE )
            {
                //num_bonds += 2;
                ++btop_i;

                /* Need to do later... since i and j are parallel
                   if( workspace->bond_mark[j] > workspace->bond_mark[i] + 1 )
                   workspace->bond_mark[j] = workspace->bond_mark[i] + 1;
                   else if( workspace->bond_mark[i] > workspace->bond_mark[j] + 1 ) {
                   workspace->bond_mark[i] = workspace->bond_mark[j] + 1;
                   }
                 */
            }
        }
    }

    Dev_Set_End_Index( i, btop_i, &bonds );
    //    if( local ) {
    H->end[i] = Htop;
    //   }
    //CHANGE ORIGINAL
    if ( ( ihb == 1 || ihb == 2 ) && ihb_top > 0 && control->hbond_cut > 0.0 )
    {
        Dev_Set_End_Index( atom_i->Hindex, ihb_top, &hbonds );
    }
    //} Commented for cuda kernel
}


CUDA_GLOBAL void k_init_bond_mark( int offset, int n, int *bond_mark )
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
    {
        return;
    }

    bond_mark[offset + threadIdx.x] = 1000;
}


CUDA_GLOBAL void New_fix_sym_dbond_indices( reax_list pbonds, int N )
{
    int i, j, k, nbr;
    bond_data *ibond, *jbond;
    int atom_j;

    reax_list *bonds = &pbonds;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
    {
        return;
    }

    for ( j = Dev_Start_Index(i, bonds); j < Dev_End_Index(i, bonds); j++ )
    {
        ibond = &( bonds->select.bond_list [j] );
        nbr = ibond->nbr;

        for ( k = Dev_Start_Index(nbr, bonds); k < Dev_End_Index(nbr, bonds); k++ )
        {
            jbond = &( bonds->select.bond_list[k] );
            atom_j = jbond->nbr;

            if ( atom_j == i )
            {
                if ( i > nbr )
                {
                    ibond->dbond_index = j;
                    jbond->dbond_index = j;

                    ibond->sym_index = k;
                    jbond->sym_index = j;
                }
            }
        }
    }
}


CUDA_GLOBAL void New_fix_sym_hbond_indices( reax_atom *my_atoms, reax_list hbonds, int N )
{
    int i, j, k;
    int nbr, nbrstart, nbrend;
    int start, end;
    hbond_data *ihbond, *jhbond;
    int __THREADS_PER_ATOM__ = HB_KER_SYM_THREADS_PER_ATOM;
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = thread_id / __THREADS_PER_ATOM__;
    int lane_id = thread_id & (__THREADS_PER_ATOM__ - 1);
    int my_bucket = threadIdx.x / __THREADS_PER_ATOM__;

    if ( warp_id > N )
    {
        return;
    }

    i = warp_id;
    j = start + lane_id;
    start = Dev_Start_Index( my_atoms[i].Hindex, &hbonds );
    end = Dev_End_Index( my_atoms[i].Hindex, &hbonds );

    while ( j < end )
    {
        ihbond = &( hbonds.select.hbond_list [j] );
        nbr = ihbond->nbr;

        nbrstart = Dev_Start_Index (my_atoms[nbr].Hindex, &hbonds);
        nbrend = Dev_End_Index (my_atoms[nbr].Hindex, &hbonds);

        for ( k = nbrstart; k < nbrend; k++ )
        {
            jhbond = &( hbonds.select.hbond_list [k] );

            if ( jhbond->nbr == i )
            {
                ihbond->sym_index = k;
                jhbond->sym_index = j;
                break;
            }
        }

        j += __THREADS_PER_ATOM__;
    }
}


////////////////////////
// HBOND ISSUE
CUDA_GLOBAL void k_update_bonds( reax_atom *my_atoms, 
        reax_list bonds, int n )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n)
    {
        return;
    }

    my_atoms[i].num_bonds = 
        MAX( Dev_Num_Entries(i, &bonds) * 2, MIN_BONDS );
}


CUDA_GLOBAL void k_update_hbonds( reax_atom *my_atoms, 
        reax_list hbonds, int n )
{
    int Hindex;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n)
    {
        return;
    }

    Hindex = my_atoms[i].Hindex;
    my_atoms[i].num_hbonds = 
        MAX( Dev_Num_Entries(Hindex, &hbonds) * SAFER_ZONE, MIN_HBONDS );
}
////////////////////////
////////////////////////
////////////////////////


int Cuda_Validate_Lists( reax_system *system, storage *workspace,
        reax_list **lists, control_params *control, 
        int step, int numH )
{
    int blocks;
    int i, comp, Hindex, ret;
    int total_3body, *thbody;
    int *index, *end_index;
    reax_list *bonds, *hbonds;
    reallocate_data *realloc;
    int max_sp_entries, num_hbonds, num_bonds;
    int total_sp_entries;
    int max_hbonds;
    real *spad = (real *) scratch;

    realloc = &( dev_workspace->realloc );
    blocks = system->n / DEF_BLOCK_SIZE + 
        ((system->n % DEF_BLOCK_SIZE == 0) ? 0 : 1);
    ret = SUCCESS;

    k_update_bonds <<< blocks, DEF_BLOCK_SIZE >>>
        (system->d_my_atoms, *(*lists + BONDS), 
         system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );

    ////////////////////////
    // HBOND ISSUE
    //FIX - 4 - Added this check for hydrogen bond issue
    if ( control->hbond_cut > 0.0 && system->numH > 0 )
    {
        k_update_hbonds <<< blocks, DEF_BLOCK_SIZE >>>
            (system->d_my_atoms, *(*lists + HBONDS), 
             system->n);
        hipDeviceSynchronize( );
        cudaCheckError( );
    }

    /* validate charge matrix */
    memset( host_scratch, 0, 2 * system->N * sizeof(int) );
    index = (int *) host_scratch;
    end_index = index + system->N;
    copy_host_device( index, dev_workspace->H.start, system->N * sizeof(int), 
            hipMemcpyDeviceToHost, "sparse_matrix:start" );
    copy_host_device( end_index, dev_workspace->H.end, system->N * sizeof(int), 
            hipMemcpyDeviceToHost, "sparse_matrix:end" );
    max_sp_entries = total_sp_entries = 0;
    for (i = 0; i < system->N; i++ )
    {
        //if (i < system->N-1)
        //    comp = index [i+1];
        //else
        //    comp = dev_workspace->H.m;

        total_sp_entries += end_index [i] - index[i];
        if ( end_index[i] - index[i] > system->max_sparse_entries )
        {
            fprintf( stderr, "step%d-sparsemat-chk failed: i=%d start(i)=%d end(i)=%d \n",
                    step, i, index[i], end_index[i] );
            ret = FAILURE;
        }
        else if ( end_index[i] >= dev_workspace->H.m )
        {
            //SUDHIR_FIX_SPARSE_MATRIX
            //TODO move this carver
            fprintf( stderr, "p:%d - step%d-sparsemat-chk failed (exceed limits): i=%d start(i)=%d end(i)=%d \n", 
                    system->my_rank, step, i, index[i], end_index[i] );
            //TODO move this carver
            ret = FAILURE;
        }
        else
        {
            if ( max_sp_entries <= end_index[i] - index[i] )
            {
                max_sp_entries = end_index[i] - index[i];
            }
        }
    }
    //if (max_sp_entries <= end_index[i] - index[i])
    //    max_sp_entries = end_index[i] - index[i];

    //update the current step max_sp_entries;
    realloc->Htop = max_sp_entries;

#if defined(DEBUG)
    fprintf( stderr, "p:%d - Cuda_Reallocate: Total H matrix entries: %d, cap: %d, used: %d \n", 
            system->my_rank, dev_workspace->H.n, dev_workspace->H.m, total_sp_entries );
#endif

    if (total_sp_entries >= dev_workspace->H.m)
    {
        fprintf( stderr, "p:%d - **ran out of space for sparse matrix: step: %d, allocated: %d, used: %d \n", 
                system->my_rank, step, dev_workspace->H.m, total_sp_entries );

        ret = FAILURE;
    }

    /* validate bond list */
    if ( system->N > 0 )
    {
        num_bonds = 0;

        bonds = *lists + BONDS;
        memset( host_scratch, 0, 2 * system->N * sizeof(int) );
        index = (int *) host_scratch;
        end_index = index + system->N;

        copy_host_device( index, bonds->index, system->N * sizeof(int), 
                hipMemcpyDeviceToHost, "dev_bonds:index" );
        copy_host_device( end_index, bonds->end_index, system->N * sizeof(int), 
                hipMemcpyDeviceToHost, "dev_bonds:end_index" );

        for ( i = 0; i < system->N; i++ )
        {
            if ( (end_index[i] - index[i]) >= system->max_bonds[i] )
            {
                fprintf( stderr, "p:%d - **ran out of space for bonds: atom: %d, allocated: %d, used: %d \n",
                        system->my_rank, i, system->max_bonds[i], end_index[i] - index[i] );
                system->max_bonds[i] =
                        MAX( (int)CEIL( (end_index[i] - index[i]) * SAFE_ZONE ), MIN_BONDS );
                ret = FAILURE;
                realloc->bonds = TRUE;
            }
        }
        for ( i = system->N; i < system->total_cap; i++ )
        {
            system->max_bonds[i] = MIN_NBRS;
        }
    }

    /* validate Hbonds list */
    num_hbonds = 0;
    // FIX - 4 - added additional check here
    if ( numH > 0 && control->hbond_cut > 0.0 )
    {
        hbonds = *lists + HBONDS;
        memset( host_scratch, 0, 2 * hbonds->n * sizeof(int) );
        index = (int *) host_scratch;
        end_index = index + hbonds->n;

        copy_host_device( index, hbonds->index, hbonds->n * sizeof(int), 
                hipMemcpyDeviceToHost, "hbonds:index" );
        copy_host_device( end_index, hbonds->end_index, hbonds->n * sizeof(int), 
                hipMemcpyDeviceToHost, "hbonds:end_index" );

        /*
           for (i = 0; i < N-1; i++) {
           Hindex = my_atoms [i].Hindex;
           if (Hindex > -1) 
           comp = index [Hindex + 1];
           else
           comp = hbonds->num_intrs;

           if (end_index [Hindex] > comp) {
           fprintf(stderr,"step%d-atom:%d hbondchk failed: H=%d start(H)=%d end(H)=%d str(H+1)=%d\n",
           step, i, Hindex, index[Hindex], end_index[Hindex], comp );
           return FAILURE;
           }

           num_hbonds += MAX( (end_index [Hindex] - index [Hindex]) * 2, MIN_HBONDS * 2);
           }
           if (end_index [my_atoms[i].Hindex] > hbonds->num_intrs) {
           fprintf(stderr,"step%d-atom:%d hbondchk failed: H=%d start(H)=%d end(H)=%d num_intrs=%d\n",
           step, i, Hindex, index[Hindex], end_index[Hindex], hbonds->num_intrs);
           return FAILURE;
           }

           num_hbonds += MIN( (end_index [my_atoms[i].Hindex] - index [my_atoms[i].Hindex]) * 2, 
           2 * MIN_HBONDS);
           num_hbonds = MAX( num_hbonds, MIN_CAP*MIN_HBONDS );
           realloc->num_hbonds = num_hbonds;
         */

        max_hbonds = 0;
        for (i = 0; i < system->N; i++)
        {
            if (end_index[i] - index[i] >= system->max_hbonds)
            {
                //TODO: update
//                fprintf( stderr, "step%d-hbondchk failed: i=%d start(i)=%d end(i)=%d max_hbonds=%d\n",
//                        step, i, index[i], end_index[i], system->max_hbonds );
//                return FAILURE;
            }
            if (end_index[i] - index[i] >= max_hbonds)
            {
                max_hbonds = end_index[i] - index[i];
            }
        }
        realloc->num_hbonds = max_hbonds;
    }

    /* three body interactions */
    cuda_memset( spad, 0, (*lists + BONDS)->num_intrs * sizeof (int), "scratch" );
    Estimate_Cuda_Valence_Angles <<<BLOCKS_N, BLOCK_SIZE>>>
        ( system->d_my_atoms, (control_params *)control->d_control_params, 
          *(*lists + BONDS), system->n, system->N, (int *)spad);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [ESTIMATE_CUDA_VALENCE_ANGLES]\n" );

    thbody = (int *) host_scratch;
    memset( thbody, 0, sizeof(int) * (*lists + BONDS)->num_intrs );
    copy_host_device( thbody, spad, (*lists + BONDS)->num_intrs * sizeof(int),
            hipMemcpyDeviceToHost, "thb:offsets" );

    total_3body = 0;
    for (i = 0; i < (*lists + BONDS)->num_intrs; i++)
    {
        total_3body += thbody[i];
        thbody[i] += thbody[i - 1];
    }

    system->num_thbodies = thbody[(*lists + BONDS)->num_intrs - 1];

    if ( system->num_thbodies > (*lists + THREE_BODIES)->num_intrs ||
                (*lists + THREE_BODIES)->n < (*lists + BONDS)->num_intrs )
    {
        realloc->num_3body = total_3body;
        system->num_thbodies = total_3body;
        ret = FAILURE;
    }
    fprintf( stderr, "system->num_thbodies = %d, lists:THREE_BODIES->num_intrs = %d,\n",
            system->num_thbodies, (*lists + THREE_BODIES)->num_intrs );
    fprintf( stderr, "lists:THREE_BODIES->n = %d, lists:BONDS->num_intrs = %d,\n",
            (*lists + THREE_BODIES)->n, (*lists + BONDS)->num_intrs );
    fprintf( stderr, "total_3body = %d\n", total_3body );

    if ( ret == SUCCESS )
    {
        /* copy the indexes into the thb list */
        copy_host_device( thbody, (*lists + THREE_BODIES)->index + 1,
                sizeof(int) * ((*lists + BONDS)->num_intrs - 1),
                hipMemcpyHostToDevice, "dev_thb:index" );
        copy_host_device( thbody, (*lists + THREE_BODIES)->end_index + 1,
                sizeof(int) * ((*lists + BONDS)->num_intrs - 1),
                hipMemcpyHostToDevice, "dev_thb:end_index" );
    }


    return ret;
}


CUDA_GLOBAL void k_init_bond_orders( reax_atom *my_atoms, reax_list far_nbrs, 
        reax_list bonds, real *total_bond_order, int N )
{
    int i, j, pj; 
    int start_i, end_i;
    int type_i, type_j;
    far_neighbor_data *nbr_pj;
    reax_atom *atom_i, *atom_j;

    i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= N)
    {
        return;
    }

    atom_i = &(my_atoms[i]);
    start_i = Dev_Start_Index(i, &far_nbrs);
    end_i = Dev_End_Index(i, &far_nbrs);

    for( pj = start_i; pj < end_i; ++pj )
    { 
        // nbr_pj = &( far_nbrs.select.far_nbr_list[pj] );
        // j = nbr_pj->nbr;
        // atom_j = &(my_atoms[j]);

        //total_bond_order [i] ++;
        //atom_i->Hindex ++;
    }
}


CUDA_GLOBAL void k_bond_mark( reax_list p_bonds, storage p_workspace, int N )
{
    reax_list *bonds = &( p_bonds );
    storage *workspace = &( p_workspace );
    int j;

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    //if (i >= N) return;

    for (int i = 0; i < N; i++) 
    {
        for (int k = Dev_Start_Index (i, bonds); k < Dev_End_Index (i, bonds); k++)
        {
            bond_data *bdata = &( bonds->select.bond_list [k] );
            j = bdata->nbr;

            if (i < j )
            {
                if ( workspace->bond_mark [j] > (workspace->bond_mark [i] + 1) )
                {
                    workspace->bond_mark [j] = workspace->bond_mark [i] + 1;    
                }
                else if ( workspace->bond_mark [i] > (workspace->bond_mark [j] + 1) )
                {
                    workspace->bond_mark [i] = workspace->bond_mark [j] + 1;
                }
            }
        }
    }
}


int Cuda_Init_Forces( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    int i, ret;
    int init_blocks, hblocks;

    //init the workspace (bond_mark)
    /*
       int blocks;
       cuda_memset (dev_workspace->bond_mark, 0, sizeof (int) * system->n, "bond_mark");

       blocks = (system->N - system->n) / DEF_BLOCK_SIZE + 
       (((system->N - system->n) % DEF_BLOCK_SIZE == 0) ? 0 : 1);
       k_init_bond_mark <<< blocks, DEF_BLOCK_SIZE >>>
       (system->n, (system->N - system->n), dev_workspace->bond_mark);
       hipDeviceSynchronize ();
       cudaCheckError ();
     */
    //validate total_bond_orders

    //main kernel
    init_blocks = (system->N) / DEF_BLOCK_SIZE + 
        (((system->N % DEF_BLOCK_SIZE) == 0) ? 0 : 1);
    //fprintf (stderr, " Total atoms: %d, blocks: %d \n", system->N, init_blocks );

    //    k_init_bond_orders <<<init_blocks, DEF_BLOCK_SIZE >>>
    //            ( system->d_my_atoms, *(*dev_lists + FAR_NBRS), *(*dev_lists + BONDS), 
    //                dev_workspace->total_bond_order, system->N);
    //    hipDeviceSynchronize ();
    //    cudaCheckError ();
    //    fprintf (stderr, " DONE WITH VALIDATION \n");

    k_init_forces <<<init_blocks, DEF_BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp,
          system->reax_param.d_tbp, *dev_workspace,
          (control_params *)control->d_control_params,
          *(*dev_lists + FAR_NBRS), *(*dev_lists + BONDS),
          *(*dev_lists + HBONDS), d_LR, system->n,
          system->N, system->reax_param.num_atom_types,
          system->max_sparse_entries, (((data->step-data->prev_steps) %
                  control->reneighbor) == 0), system->max_hbonds );
    hipDeviceSynchronize( );
    cudaCheckError( );

    //fix - sym_index and dbond_index
    New_fix_sym_dbond_indices <<<init_blocks, BLOCK_SIZE>>> 
        (*(*dev_lists + BONDS), system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );

    ///////////////////////
    ///////////////////////
    // FIX - 4 - HBOND ISSUE
    if ( control->hbond_cut > 0 && system->numH > 0 )
    {
        //make hbond_list symmetric
        hblocks = (system->N * HB_KER_SYM_THREADS_PER_ATOM) / HB_SYM_BLOCK_SIZE + 
            ((((system->N * HB_KER_SYM_THREADS_PER_ATOM) % HB_SYM_BLOCK_SIZE) == 0) ? 0 : 1);
        //New_fix_sym_hbond_indices <<<hblocks, HB_BLOCK_SIZE >>> 
        New_fix_sym_hbond_indices <<<hblocks, HB_BLOCK_SIZE >>> 
            (system->d_my_atoms, *(*dev_lists + HBONDS), system->N);
        hipDeviceSynchronize( );
        cudaCheckError( );
    }

    //update bond_mark
    //k_bond_mark <<< init_blocks, DEF_BLOCK_SIZE>>>
    /*
       k_bond_mark <<< 1, 1>>>
       ( *(*dev_lists + BONDS), *dev_workspace, system->N);
       hipDeviceSynchronize ();
       cudaCheckError ();
     */

    //TODO
    //1. update the sparse matrix count for reallocation
    //2. update the bonds count for reallocation
    //3. update the hydrogen bonds count for reallocation

    /* validate lists */
    ret = Cuda_Validate_Lists( system, workspace, dev_lists, control,
            data->step, system->numH );

    return ret;
}


int Cuda_Init_Forces_No_Charges( reax_system *system, control_params *control,
        simulation_data *data, storage *workspace,
        reax_list **lists, output_controls *out_control ) 
{
    //TODO Implement later
    // when you figure out the bond_mark usage.

    return FAILURE;
}


void Cuda_Compute_Bonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control )
{
    int i;
    real t_start, t_elapsed;
    real *spad = (real *) scratch;
    rvec *rvec_spad;

    //1. Bond Order Interactions. - bond_orders.c
    t_start = Get_Time( );
    //fprintf (stderr, " Begin Bonded Forces ... %d x %d\n", BLOCKS_N, BLOCK_SIZE);
    Cuda_Calculate_BO_init  <<< BLOCKS_N, BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_sbp, 
          *dev_workspace, system->N );
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_CALCULATE_BO_INIT]\n" );

    Cuda_Calculate_BO <<< BLOCKS_N, BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, 
          system->reax_param.d_tbp, *dev_workspace, 
          *(*dev_lists + BONDS),
          system->reax_param.num_atom_types, system->N );
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_CALCULATE_BO]\n" );


    Cuda_Update_Uncorrected_BO <<<BLOCKS_N, BLOCK_SIZE>>>
        (*dev_workspace, *(*dev_lists + BONDS), system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_UPDATE_UNCORRECTED_BO]\n" );

    Cuda_Update_Workspace_After_BO <<<BLOCKS_N, BLOCK_SIZE>>>
        (system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, 
         *dev_workspace, system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_UPDATE_WORKSPACE_AFTER_BO]\n" );

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "Bond Orders... return value --> %d --- Timing %lf \n", hipGetLastError (), t_elapsed );
    //fprintf (stderr, "Cuda_Calculate_Bond_Orders Done... \n");

    //2. Bond Energy Interactions. - bonds.c
    t_start = Get_Time( );
    cuda_memset( spad, 0, system->N * (2 * sizeof(real)) , "scratch" );

    Cuda_Bonds <<< BLOCKS, BLOCK_SIZE, sizeof(real)* BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_gp, system->reax_param.d_sbp, system->reax_param.d_tbp,
          *dev_workspace, *(*dev_lists + BONDS), 
          system->n, system->reax_param.num_atom_types, spad );
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_BONDS]\n" );

    //Reduction for E_BE
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof(real) * BLOCK_SIZE >>>  
        (spad, spad + system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2, sizeof(real) * BLOCKS_POW_2>>> 
        (spad + system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_bond, BLOCKS_POW_2);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "Cuda_Bond_Energy ... return value --> %d --- Timing %lf \n", hipGetLastError (), t_elapsed );
    //fprintf (stderr, "Cuda_Bond_Energy Done... \n");

    //3. Atom Energy Interactions. 
    t_start = Get_Time( );
    cuda_memset( spad, 0, ( 6 * sizeof(real) * system->n ), "scratch");

    Cuda_Atom_Energy <<<BLOCKS, BLOCK_SIZE>>>( system->d_my_atoms, system->reax_param.d_gp, 
            system->reax_param.d_sbp, system->reax_param.d_tbp, 
            *dev_workspace, 
            *(*dev_lists + BONDS), system->n, system->reax_param.num_atom_types, 
            spad, spad + 2 * system->n, spad + 4*system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_ATOM_ENERGY]\n" );

    //CHANGE ORIGINAL
    //Cuda_Atom_Energy_PostProcess <<<BLOCKS, BLOCK_SIZE >>>
    //                    ( *(*dev_lists + BONDS), *dev_workspace, system->n );
    Cuda_Atom_Energy_PostProcess <<<BLOCKS_N, BLOCK_SIZE >>>
        ( *(*dev_lists + BONDS), *dev_workspace, system->N );
    //CHANGE ORIGINAL
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_ATOM_ENERGY_POST_PROCESS]\n" );

    //Reduction for E_Lp
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof(real) * BLOCK_SIZE >>>  
        (spad, spad + system->n, system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2, sizeof(real) * BLOCKS_POW_2 >>>  
        (spad + system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_lp, BLOCKS);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    //Reduction for E_Ov
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>  
        (spad + 2*system->n, spad + 3*system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>  
        (spad + 3*system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_ov, BLOCKS);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    //Reduction for E_Un
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>  
        (spad + 4*system->n, spad + 5*system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>  
        (spad + 5*system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_un, BLOCKS);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "test_LonePair_postprocess ... return value --> %d --- Timing %lf \n", hipGetLastError (), t_elapsed );
    //fprintf (stderr, "test_LonePair_postprocess Done... \n");

    //4. Valence Angles Interactions. 
    t_start = Get_Time( );

    cuda_memset( spad, 0, 6 * sizeof(real) * system->N + sizeof(rvec) * system->N * 2, "scratch" );
    Cuda_Valence_Angles <<< BLOCKS_N, BLOCK_SIZE >>>
        ( system->d_my_atoms, system->reax_param.d_gp, 
          system->reax_param.d_sbp, system->reax_param.d_thbp, 
          (control_params *)control->d_control_params,
          *dev_workspace, *(*dev_lists + BONDS), *(*dev_lists + THREE_BODIES),
          system->n, system->N, system->reax_param.num_atom_types, 
          spad, spad + 2*system->N, spad + 4*system->N, (rvec *)(spad + 6*system->N));
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_VALENCE_ANGLES]\n" );

    //Reduction for E_Ang
    k_reduction <<<BLOCKS_N, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>  
        (spad, spad + system->N,  system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2_N, sizeof (real) * BLOCKS_POW_2_N >>>
        (spad + system->N, &((simulation_data *)data->d_simulation_data)->my_en.e_ang, BLOCKS_N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    //Reduction for E_Pen
    k_reduction <<<BLOCKS_N, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>
        (spad + 2*system->N, spad + 3*system->N,  system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2_N, sizeof (real) * BLOCKS_POW_2_N >>>
        (spad + 3*system->N, &((simulation_data *)data->d_simulation_data)->my_en.e_pen, BLOCKS_N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    //Reduction for E_Coa
    k_reduction <<<BLOCKS_N, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>
        (spad + 4*system->N, spad + 5*system->N,  system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2_N, sizeof (real) * BLOCKS_POW_2_N >>>
        (spad + 5*system->N, &((simulation_data *)data->d_simulation_data)->my_en.e_coa, BLOCKS_N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    //Reduction for ext_pres
    rvec_spad = (rvec *) (spad + 6*system->N);
    k_reduction_rvec <<<BLOCKS_N, BLOCK_SIZE, sizeof (rvec) * BLOCK_SIZE >>>
        (rvec_spad, rvec_spad + system->N,  system->N);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction_rvec <<<1, BLOCKS_POW_2_N, sizeof (rvec) * BLOCKS_POW_2_N >>>
        (rvec_spad + system->N, &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS_N);
    hipDeviceSynchronize ();
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    Cuda_Valence_Angles_PostProcess <<< BLOCKS_N, BLOCK_SIZE >>>
        (  system->d_my_atoms,
           (control_params *)control->d_control_params,
           *dev_workspace,
           *(*dev_lists + BONDS),
           system->N );
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_VALENCE_ANGLES_POST_PROCESS]\n" );

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "Three_Body_Interactions ...  Timing %lf \n", t_elapsed );
    //fprintf (stderr, "Three_Body_Interactions Done... \n");

    //5. Torsion Angles Interactions. 
    t_start = Get_Time( );
    cuda_memset (spad, 0, ( 4 * sizeof (real) * system->n + sizeof (rvec) * system->n * 2), "scratch");
    Cuda_Torsion_Angles <<< BLOCKS, BLOCK_SIZE >>>
        ( system->d_my_atoms,
          system->reax_param.d_gp,
          system->reax_param.d_fbp,
          (control_params *)control->d_control_params,
          *(*dev_lists + BONDS), *(*dev_lists + THREE_BODIES),
          *dev_workspace,
          system->n, system->reax_param.num_atom_types, 
          spad, spad + 2*system->n, (rvec *) (spad + 4*system->n));
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_TORSION_ANGLES]\n" );

    //Reduction for E_Tor
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>
        (spad, spad + system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>
        (spad + system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_tor, BLOCKS);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    //Reduction for E_Con
    k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof (real) * BLOCK_SIZE >>>
        (spad + 2*system->n, spad + 3*system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction <<<1, BLOCKS_POW_2, sizeof (real) * BLOCKS_POW_2 >>>
        (spad + 3*system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_con, BLOCKS);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    //Reduction for ext_pres
    rvec_spad = (rvec *) (spad + 4*system->n);
    k_reduction_rvec <<<BLOCKS, BLOCK_SIZE, sizeof (rvec) * BLOCK_SIZE >>>
        (rvec_spad, rvec_spad + system->n,  system->n);
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    k_reduction_rvec <<<1, BLOCKS_POW_2, sizeof (rvec) * BLOCKS_POW_2 >>>
            ( rvec_spad + system->n,
            &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS );
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [REDUCTION]\n" );

    //Post process here
    Cuda_Torsion_Angles_PostProcess   <<< BLOCKS_N, BLOCK_SIZE >>>
            ( system->d_my_atoms, *dev_workspace, *(*dev_lists + BONDS),
            system->N );
    hipDeviceSynchronize( );
    cudaCheckError( );
    fprintf( stderr, "      [CUDA_TORSION_POST_PROCESS]\n" );

    t_elapsed = Get_Timing_Info( t_start );
    //fprintf (stderr, "Four_Body_post process return value --> %d --- Four body Timing %lf \n", hipGetLastError (), t_elapsed );
    //fprintf (stderr, " Four_Body_ Done... \n");

    //6. Hydrogen Bonds Interactions.
    // FIX - 4 - Added additional check here
    if ((control->hbond_cut > 0) && (system->numH > 0))
    {
        t_start = Get_Time( );
        cuda_memset( spad, 0,
                2 * sizeof(real) * system->n + sizeof(rvec) * system->n * 2, "scratch" );

        int hbs = ((system->n * HB_KER_THREADS_PER_ATOM)/ HB_BLOCK_SIZE) + 
            (((system->n * HB_KER_THREADS_PER_ATOM) % HB_BLOCK_SIZE) == 0 ? 0 : 1);
        Cuda_Hydrogen_Bonds_MT <<<hbs, HB_BLOCK_SIZE, 
                HB_BLOCK_SIZE * (2 * sizeof(real) + 2 * sizeof(rvec)) >>>
        //Cuda_Hydrogen_Bonds <<< BLOCKS, BLOCK_SIZE>>>
                        ( system->d_my_atoms, system->reax_param.d_sbp,
                        system->reax_param.d_hbp, system->reax_param.d_gp,
                        (control_params *)control->d_control_params,
                        *dev_workspace, *(*dev_lists + BONDS), *(*dev_lists + HBONDS),
                        system->n, system->reax_param.num_atom_types,
                        spad, (rvec *) (spad + 2*system->n));
        hipDeviceSynchronize( );
        cudaCheckError( );
        fprintf( stderr, "      [CUDA_HYDROGEN_BONDS_MT]\n" );

        //Reduction for E_HB
        k_reduction <<<BLOCKS, BLOCK_SIZE, sizeof(real) * BLOCK_SIZE >>>
            (spad, spad + system->n,  system->n);
        hipDeviceSynchronize( );
        cudaCheckError( );
        fprintf( stderr, "      [REDUCTION]\n" );

        k_reduction <<<1, BLOCKS_POW_2, sizeof(real) * BLOCKS_POW_2 >>>
            (spad + system->n, &((simulation_data *)data->d_simulation_data)->my_en.e_hb, BLOCKS);
        hipDeviceSynchronize( );
        cudaCheckError( );
        fprintf( stderr, "      [REDUCTION]\n" );

        //Reduction for ext_pres
        rvec_spad = (rvec *) (spad + 2*system->n);
        k_reduction_rvec <<<BLOCKS, BLOCK_SIZE, sizeof (rvec) * BLOCK_SIZE >>>
            (rvec_spad, rvec_spad + system->n,  system->n);
        hipDeviceSynchronize( );
        cudaCheckError( );
        fprintf( stderr, "      [REDUCTION]\n" );

        k_reduction_rvec <<<1, BLOCKS_POW_2, sizeof (rvec) * BLOCKS_POW_2 >>>
            (rvec_spad + system->n, &((simulation_data *)data->d_simulation_data)->my_ext_press, BLOCKS);
        hipDeviceSynchronize( );
        cudaCheckError( );
        fprintf( stderr, "      [REDUCTION]\n" );

        //post process step1:
        Cuda_Hydrogen_Bonds_PostProcess <<< BLOCKS_N, BLOCK_SIZE, BLOCK_SIZE * sizeof (rvec) >>>
            (  system->d_my_atoms, *dev_workspace,
               *(*dev_lists + BONDS), system->N );
        hipDeviceSynchronize( );
        cudaCheckError( );
        fprintf( stderr, "      [CUDA_HYDROGEN_BONDS_POST_PROCESS]\n" );

        //post process step2:
        /*
           Cuda_Hydrogen_Bonds_HNbrs <<< system->N, 32, 32 * sizeof (rvec)>>>
           (  system->d_my_atoms,
         *dev_workspace,
         *(*dev_lists + HBONDS));
         */
        int hnbrs_bl = ((system->N * HB_POST_PROC_KER_THREADS_PER_ATOM)/ HB_POST_PROC_BLOCK_SIZE) + 
            (((system->N * HB_POST_PROC_KER_THREADS_PER_ATOM) % HB_POST_PROC_BLOCK_SIZE) == 0 ? 0 : 1);
        Cuda_Hydrogen_Bonds_HNbrs_BL <<< hnbrs_bl, HB_POST_PROC_BLOCK_SIZE, 
                HB_POST_PROC_BLOCK_SIZE * sizeof (rvec)>>>
                        ( system->d_my_atoms, *dev_workspace,
                        *(*dev_lists + HBONDS), system->N);
        hipDeviceSynchronize( );
        cudaCheckError( );
        fprintf( stderr, "      [CUDA_HYDROGEN_BONDS_HNBRS_BL]\n" );

        t_elapsed = Get_Timing_Info( t_start );
        //fprintf (stderr, "Hydrogen bonds return value --> %d --- HydrogenBonds Timing %lf \n", hipGetLastError (), t_elapsed );
        //fprintf (stderr, "Hydrogen_Bond Done... \n");    
    }
}


void Cuda_Compute_NonBonded_Forces( reax_system *system, control_params *control, 
        simulation_data *data, storage *workspace, 
        reax_list **lists, output_controls *out_control,
        mpi_datatypes *mpi_data )
{
    /* van der Waals and Coulomb interactions */
    Cuda_NonBonded_Energy( system, control, workspace, data,
            lists, out_control, (control->tabulate == 0) ? false: true );
}

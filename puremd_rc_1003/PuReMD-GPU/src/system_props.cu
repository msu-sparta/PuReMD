#include "hip/hip_runtime.h"
/*----------------------------------------------------------------------
  PuReMD-GPU - Reax Force Field Simulator
      
  Copyright (2014) Purdue University
  Sudhir Kylasa, skylasa@purdue.edu
  Hasan Metin Aktulga, haktulga@cs.purdue.edu
  Ananth Y Grama, ayg@cs.purdue.edu
 
  This program is free software; you can redistribute it and/or
  modify it under the terms of the GNU General Public License as
  published by the Free Software Foundation; either version 2 of 
  the License, or (at your option) any later version.
               
  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
  See the GNU General Public License for more details:
  <http://www.gnu.org/licenses/>.
  ----------------------------------------------------------------------*/

#include "system_props.h"
#include "box.h"
#include "vector.h"

#include "cuda_utils.h"
#include "cuda_copy.h"
#include "reduction.h"
#include "center_mass.h"
#include "validation.h"


real Get_Time( )
{
  struct timeval tim;
  
  gettimeofday(&tim, NULL );
  return( tim.tv_sec + (tim.tv_usec / 1000000.0) );
}


real Get_Timing_Info( real t_start )
{
  struct timeval tim;
  real t_end;
  
  gettimeofday(&tim, NULL );
  t_end = tim.tv_sec + (tim.tv_usec / 1000000.0);
  return (t_end - t_start);
}


void Temperature_Control( control_params *control, simulation_data *data, 
			  output_controls *out_control )
{
  real tmp;

  if( control->T_mode == 1 ) { // step-wise temperature control
    if( (data->step - data->prev_steps) % 
	((int)(control->T_freq / control->dt)) == 0 ) {
      if( fabs( control->T - control->T_final ) >= fabs( control->T_rate ) )
	control->T += control->T_rate;
      else control->T = control->T_final;	 
    }
  }
  else if( control->T_mode == 2 ) { // constant slope control
    tmp = control->T_rate * control->dt / control->T_freq;
    
    if( fabs( control->T - control->T_final ) >= fabs( tmp ) )
      control->T += tmp;       
  }
}

void prep_dev_system (reax_system *system) 
{
	//copy the system atoms to the device
	Sync_Host_Device ( system, hipMemcpyHostToDevice );
}


void Compute_Total_Mass( reax_system *system, simulation_data *data )
{
  int i;
  int blocks;
  int block_size;
  real	*partial_sums = 0;
  
  data->M = 0;

  for( i = 0; i < system->N; i++ ) 
    data->M += system->reaxprm.sbp[ system->atoms[i].type ].mass;  

  data->inv_M = 1. / data->M;    
}

void Cuda_Compute_Total_Mass( reax_system *system, simulation_data *data )
{
  	real	*partial_sums = (real *) scratch;
  	//data->M = 0;

	//cuda_malloc ((void **)&partial_sums, sizeof (real) * (blocks + 1), 1, 0);
	cuda_memset (partial_sums, 0, REAL_SIZE * (BLOCKS_POW_2 + 1), RES_SCRATCH );

	Compute_Total_Mass <<<BLOCKS_POW_2, BLOCK_SIZE, REAL_SIZE * BLOCK_SIZE >>> 
   							(system->reaxprm.d_sbp, system->d_atoms, partial_sums, system->N);
	hipDeviceSynchronize ();
	cudaCheckError ();

	Cuda_reduction <<<1, BLOCKS_POW_2, REAL_SIZE * BLOCKS_POW_2>>> 
   							(partial_sums, partial_sums + BLOCKS_POW_2, BLOCKS_POW_2);
   							//(partial_sums, &((simulation_data *)data->d_simulation_data)->M, BLOCKS_POW_2);
	hipDeviceSynchronize ();
	cudaCheckError ();

//#ifdef __BUILD_DEBUG__
//	validate_data ( system, data );
//#endif

	//copy_host_device (&data->M, &((simulation_data *)data->d_simulation_data)->M, 
//#ifdef __BUILD_DEBUG__
//	t_data_M = data->M;
//#endif
	copy_host_device (&data->M, partial_sums + BLOCKS_POW_2, 
							REAL_SIZE, hipMemcpyDeviceToHost, __LINE__);

//#ifdef __BUILD_DEBUG__
//	if (check_zero (t_data, data->M))
//	{
//		fprintf (stderr, "SimulationData:M does not match on host and device (%f %f) \n", t_data, data->M );
//		exit (0);
//	}
//#endif
  	data->inv_M = 1. / data->M;    
}


GLOBAL void Compute_Total_Mass (single_body_parameters *sbp, reax_atom *atoms, real *per_block_results, size_t n) 
{
  extern __shared__ real sdata[];
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  real x = 0; 

  if(i < n) 
    	x = sbp [ atoms[ i ].type ].mass;

  sdata[threadIdx.x] = x; 
  __syncthreads();

  for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) 
  {  
    if(threadIdx.x < offset)
    {  
      sdata[threadIdx.x] += sdata[threadIdx.x + offset];
    }  
    __syncthreads();
  }  

  if(threadIdx.x == 0) 
  {  
    per_block_results[blockIdx.x] = sdata[0];
  }
}


void Compute_Center_of_Mass( reax_system *system, simulation_data *data, 
			     FILE *fout )
{
  int i;
  real m, xx, xy, xz, yy, yz, zz, det;
  rvec tvec, diff;
  rtensor mat, inv;

  int blocks;
  int block_size;
  rvec *l_xcm, *l_vcm, *l_amcm;
	real t_start, t_end;

  rvec_MakeZero( data->xcm );  // position of CoM
  rvec_MakeZero( data->vcm );  // velocity of CoM
  rvec_MakeZero( data->amcm ); // angular momentum of CoM
  rvec_MakeZero( data->avcm ); // angular velocity of CoM

  /* Compute the position, velocity and angular momentum about the CoM */
  for( i = 0; i < system->N; ++i ) {
    m = system->reaxprm.sbp[ system->atoms[i].type ].mass;
    
    rvec_ScaledAdd( data->xcm, m, system->atoms[i].x );
    rvec_ScaledAdd( data->vcm, m, system->atoms[i].v );
    
    rvec_Cross( tvec, system->atoms[i].x, system->atoms[i].v );
    rvec_ScaledAdd( data->amcm, m, tvec );
    
    /*fprintf( fout,"%3d  %g %g %g\n",
      i+1, 
      system->atoms[i].v[0], system->atoms[i].v[1], system->atoms[i].v[2]  );
      fprintf( fout, "vcm:  %g %g %g\n", 
      data->vcm[0], data->vcm[1], data->vcm[2] );  
    */
  }

  rvec_Scale( data->xcm, data->inv_M, data->xcm );
  rvec_Scale( data->vcm, data->inv_M, data->vcm );

  rvec_Cross( tvec, data->xcm, data->vcm );
  rvec_ScaledAdd( data->amcm, -data->M, tvec );
  
  data->etran_cm = 0.5 * data->M * rvec_Norm_Sqr( data->vcm );

  /* Calculate and then invert the inertial tensor */
  xx = xy = xz = yy = yz = zz = 0;

  for( i = 0; i < system->N; ++i ) {
    m = system->reaxprm.sbp[ system->atoms[i].type ].mass;
    
    rvec_ScaledSum( diff, 1., system->atoms[i].x, -1., data->xcm );
    xx += diff[0] * diff[0] * m;
    xy += diff[0] * diff[1] * m;
    xz += diff[0] * diff[2] * m;
    yy += diff[1] * diff[1] * m;
    yz += diff[1] * diff[2] * m;
    zz += diff[2] * diff[2] * m;      
  }

#ifdef __DEBUG_CUDA__
  fprintf (stderr, " xx: %f \n", xx);
  fprintf (stderr, " xy: %f \n", xy);
  fprintf (stderr, " xz: %f \n", xz);
  fprintf (stderr, " yy: %f \n", yy);
  fprintf (stderr, " yz: %f \n", yz);
  fprintf (stderr, " zz: %f \n", zz);
#endif

  mat[0][0] = yy + zz;     
  mat[0][1] = mat[1][0] = -xy;
  mat[0][2] = mat[2][0] = -xz;
  mat[1][1] = xx + zz;
  mat[2][1] = mat[1][2] = -yz;
  mat[2][2] = xx + yy;

  /* invert the inertial tensor */
  det = ( mat[0][0] * mat[1][1] * mat[2][2] + 
	  mat[0][1] * mat[1][2] * mat[2][0] + 
	  mat[0][2] * mat[1][0] * mat[2][1] ) -
    ( mat[0][0] * mat[1][2] * mat[2][1] + 
      mat[0][1] * mat[1][0] * mat[2][2] + 
      mat[0][2] * mat[1][1] * mat[2][0] );

  inv[0][0] = mat[1][1] * mat[2][2] - mat[1][2] * mat[2][1];
  inv[0][1] = mat[0][2] * mat[2][1] - mat[0][1] * mat[2][2];
  inv[0][2] = mat[0][1] * mat[1][2] - mat[0][2] * mat[1][1];
  inv[1][0] = mat[1][2] * mat[2][0] - mat[1][0] * mat[2][2];
  inv[1][1] = mat[0][0] * mat[2][2] - mat[0][2] * mat[2][0];
  inv[1][2] = mat[0][2] * mat[1][0] - mat[0][0] * mat[1][2];
  inv[2][0] = mat[1][0] * mat[2][1] - mat[2][0] * mat[1][1];
  inv[2][1] = mat[2][0] * mat[0][1] - mat[0][0] * mat[2][1];
  inv[2][2] = mat[0][0] * mat[1][1] - mat[1][0] * mat[0][1];

  if( fabs(det) > ALMOST_ZERO )
    rtensor_Scale( inv, 1./det, inv );
  else 
    rtensor_MakeZero( inv );

  /* Compute the angular velocity about the centre of mass */
  rtensor_MatVec( data->avcm, inv, data->amcm );  
  data->erot_cm = 0.5 * E_CONV * rvec_Dot( data->avcm, data->amcm );

#if defined(DEBUG)
  fprintf( stderr, "xcm:  %24.15e %24.15e %24.15e\n",  
	   data->xcm[0], data->xcm[1], data->xcm[2] );
  fprintf( stderr, "vcm:  %24.15e %24.15e %24.15e\n", 
	   data->vcm[0], data->vcm[1], data->vcm[2] );
  fprintf( stderr, "amcm: %24.15e %24.15e %24.15e\n", 
	   data->amcm[0], data->amcm[1], data->amcm[2] );
  /* fprintf( fout, "mat:  %f %f %f\n     %f %f %f\n     %f %f %f\n",
     mat[0][0], mat[0][1], mat[0][2], 
     mat[1][0], mat[1][1], mat[1][2], 
     mat[2][0], mat[2][1], mat[2][2] );
     fprintf( fout, "inv:  %g %g %g\n     %g %g %g\n     %g %g %g\n",
     inv[0][0], inv[0][1], inv[0][2], 
     inv[1][0], inv[1][1], inv[1][2], 
     inv[2][0], inv[2][1], inv[2][2] );
     fflush( fout ); */
  fprintf( stderr, "avcm:  %24.15e %24.15e %24.15e\n", 
	   data->avcm[0], data->avcm[1], data->avcm[2] );
#endif
}


void Cuda_Compute_Center_of_Mass( reax_system *system, simulation_data *data, 
			     FILE *fout )
{
  int i;
  real m, xx, xy, xz, yy, yz, zz, det;
  rvec tvec, diff;
  rtensor mat, inv;

  int blocks;
  int block_size;
  rvec *l_xcm, *l_vcm, *l_amcm;
	real t_start, t_end;

	rvec t_xcm, t_vcm, t_amcm;

	rvec *r_scratch = (rvec *)scratch;

  //rvec_MakeZero( data->xcm );  // position of CoM
  //rvec_MakeZero( data->vcm );  // velocity of CoM
  //rvec_MakeZero( data->amcm ); // angular momentum of CoM
  //rvec_MakeZero( data->avcm ); // angular velocity of CoM

	//cuda_malloc ((void **)&l_xcm, RVEC_SIZE * (blocks + 1), 1, 0);
	//cuda_malloc ((void **)&l_vcm, RVEC_SIZE * (blocks + 1), 1, 0);
	//cuda_malloc ((void **)&l_amcm, RVEC_SIZE * (blocks + 1), 1, 0);
	
	cuda_memset ( scratch, 0, 3 * RVEC_SIZE * (BLOCKS_POW_2 + 1), RES_SCRATCH );
	l_xcm = r_scratch;
	l_vcm = r_scratch + (BLOCKS_POW_2 + 1);
	l_amcm = r_scratch + 2 * (BLOCKS_POW_2 + 1);

	center_of_mass_blocks <<<BLOCKS_POW_2, BLOCK_SIZE, 3 * (RVEC_SIZE * BLOCK_SIZE) >>> 
   							(system->reaxprm.d_sbp, system->d_atoms, l_xcm, l_vcm, l_amcm, system->N);
	hipDeviceSynchronize ();
	cudaCheckError ();

	center_of_mass <<<1, BLOCKS_POW_2, 3 * (RVEC_SIZE * BLOCKS_POW_2) >>> 
   							(l_xcm, l_vcm, l_amcm, 
   							l_xcm + BLOCKS_POW_2, 
								l_vcm + BLOCKS_POW_2, 
								l_amcm + BLOCKS_POW_2, 
								BLOCKS_POW_2);
	hipDeviceSynchronize ();
	cudaCheckError ();

//#ifdef __BUILD_DEBUG
//	validate_data ( system, data );
//#endif

//#ifdef __BUILD_DEBUG__
//	rvec_MakeZero (t_xcm);
//	rvec_MakeZero (t_vcm);
//	rvec_MakeZero (t_amcm);
//
//	rvec_Copy (t_xcm, data->xcm);
//	rvec_Copy (t_vcm, data->vcm);
//	rvec_Copy (t_amcm, data->amcm);
//#endif

	copy_host_device (data->xcm, l_xcm + BLOCKS_POW_2, RVEC_SIZE, hipMemcpyDeviceToHost, __LINE__);
	copy_host_device (data->vcm, l_vcm + BLOCKS_POW_2, RVEC_SIZE, hipMemcpyDeviceToHost, __LINE__);
	copy_host_device (data->amcm, l_amcm + BLOCKS_POW_2, RVEC_SIZE, hipMemcpyDeviceToHost, __LINE__);

  rvec_Scale( data->xcm, data->inv_M, data->xcm );
  rvec_Scale( data->vcm, data->inv_M, data->vcm );

  rvec_Cross( tvec, data->xcm, data->vcm );
  rvec_ScaledAdd( data->amcm, -data->M, tvec );

//#ifdef __BUILD_DEBUG__
//	if (check_zero (t_xcm, data->xcm) || 
//		check_zero (t_vcm, data->vcm) ||
//		check_zero (t_amcm, data->amcm)){
//			fprintf (stderr, "SimulationData (xcm, vcm, amcm) does not match between device and host \n");
//			exit (0);
//		}
//#endif
  
  data->etran_cm = 0.5 * data->M * rvec_Norm_Sqr( data->vcm );

  /* Calculate and then invert the inertial tensor */
  xx = xy = xz = yy = yz = zz = 0;

#ifdef __BUILD_DEBUG__

  for( i = 0; i < system->N; ++i ) {
    m = system->reaxprm.sbp[ system->atoms[i].type ].mass;
    
    rvec_ScaledSum( diff, 1., system->atoms[i].x, -1., data->xcm );
    xx += diff[0] * diff[0] * m;
    xy += diff[0] * diff[1] * m;
    xz += diff[0] * diff[2] * m;
    yy += diff[1] * diff[1] * m;
    yz += diff[1] * diff[2] * m;
    zz += diff[2] * diff[2] * m;      
  }

#endif

	real *partial_results = (real *) scratch;
	real *local_results;

	//cuda_malloc ((void **)&partial_results, 6 * sizeof (real) * (blocks + 1), 1, 0);
	cuda_memset (partial_results, 0, REAL_SIZE * 6 * (BLOCKS_POW_2 + 1), RES_SCRATCH );
	local_results = (real *) malloc (REAL_SIZE * 6 *(BLOCKS_POW_2+ 1));

	compute_center_mass <<<BLOCKS_POW_2, BLOCK_SIZE, 6 * (REAL_SIZE * BLOCK_SIZE) >>> 
   							(system->reaxprm.d_sbp, system->d_atoms, partial_results, 
							data->xcm[0], data->xcm[1], data->xcm[2], system->N);
	hipDeviceSynchronize ();
	cudaCheckError ();

	compute_center_mass <<<1, BLOCKS_POW_2, 6 * (REAL_SIZE * BLOCKS_POW_2) >>> 
   							(partial_results, partial_results + (BLOCKS_POW_2 * 6), BLOCKS_POW_2);
	hipDeviceSynchronize ();
	cudaCheckError ();

	copy_host_device (local_results, partial_results + 6 * BLOCKS_POW_2, REAL_SIZE * 6, hipMemcpyDeviceToHost, __LINE__);

#ifdef __BUILD_DEBUG__
	if (check_zero (local_results[0],xx) ||
		check_zero (local_results[1],xy) ||
		check_zero (local_results[2],xz) ||
		check_zero (local_results[3],yy) ||
		check_zero (local_results[4],yz) ||
		check_zero (local_results[5],zz) )
		{
			fprintf (stderr, " xx (%4.15f %4.15f) \n", xx, local_results[0]);
			fprintf (stderr, " xy (%4.15f %4.15f) \n", xy, local_results[1]);
			fprintf (stderr, " xz (%4.15f %4.15f) \n", xz, local_results[2]);
			fprintf (stderr, " yy (%4.15f %4.15f) \n", yy, local_results[3]);
			fprintf (stderr, " yz (%4.15f %4.15f) \n", yz, local_results[4]);
			fprintf (stderr, " zz (%4.15f %4.15f) \n", zz, local_results[5]);
			fprintf (stderr, " Failed to compute the center of mass \n");
			exit (1);
		}
#endif

	xx = local_results[0];
	xy = local_results[1];
	xz = local_results[2];
	yy = local_results[3];
	yz = local_results[4];
	zz = local_results[5];

  mat[0][0] = yy + zz;     
  mat[0][1] = mat[1][0] = -xy;
  mat[0][2] = mat[2][0] = -xz;
  mat[1][1] = xx + zz;
  mat[2][1] = mat[1][2] = -yz;
  mat[2][2] = xx + yy;

  /* invert the inertial tensor */
  det = ( mat[0][0] * mat[1][1] * mat[2][2] + 
	  mat[0][1] * mat[1][2] * mat[2][0] + 
	  mat[0][2] * mat[1][0] * mat[2][1] ) -
    ( mat[0][0] * mat[1][2] * mat[2][1] + 
      mat[0][1] * mat[1][0] * mat[2][2] + 
      mat[0][2] * mat[1][1] * mat[2][0] );

  inv[0][0] = mat[1][1] * mat[2][2] - mat[1][2] * mat[2][1];
  inv[0][1] = mat[0][2] * mat[2][1] - mat[0][1] * mat[2][2];
  inv[0][2] = mat[0][1] * mat[1][2] - mat[0][2] * mat[1][1];
  inv[1][0] = mat[1][2] * mat[2][0] - mat[1][0] * mat[2][2];
  inv[1][1] = mat[0][0] * mat[2][2] - mat[0][2] * mat[2][0];
  inv[1][2] = mat[0][2] * mat[1][0] - mat[0][0] * mat[1][2];
  inv[2][0] = mat[1][0] * mat[2][1] - mat[2][0] * mat[1][1];
  inv[2][1] = mat[2][0] * mat[0][1] - mat[0][0] * mat[2][1];
  inv[2][2] = mat[0][0] * mat[1][1] - mat[1][0] * mat[0][1];

  if( fabs(det) > ALMOST_ZERO )
    rtensor_Scale( inv, 1./det, inv );
  else 
    rtensor_MakeZero( inv );

  /* Compute the angular velocity about the centre of mass */
  rtensor_MatVec( data->avcm, inv, data->amcm );  
  data->erot_cm = 0.5 * E_CONV * rvec_Dot( data->avcm, data->amcm );

  //free the resources
  free (local_results);

#if defined(DEBUG)
  fprintf( stderr, "xcm:  %24.15e %24.15e %24.15e\n",  
	   data->xcm[0], data->xcm[1], data->xcm[2] );
  fprintf( stderr, "vcm:  %24.15e %24.15e %24.15e\n", 
	   data->vcm[0], data->vcm[1], data->vcm[2] );
  fprintf( stderr, "amcm: %24.15e %24.15e %24.15e\n", 
	   data->amcm[0], data->amcm[1], data->amcm[2] );
  /* fprintf( fout, "mat:  %f %f %f\n     %f %f %f\n     %f %f %f\n",
     mat[0][0], mat[0][1], mat[0][2], 
     mat[1][0], mat[1][1], mat[1][2], 
     mat[2][0], mat[2][1], mat[2][2] );
     fprintf( fout, "inv:  %g %g %g\n     %g %g %g\n     %g %g %g\n",
     inv[0][0], inv[0][1], inv[0][2], 
     inv[1][0], inv[1][1], inv[1][2], 
     inv[2][0], inv[2][1], inv[2][2] );
     fflush( fout ); */
  fprintf( stderr, "avcm:  %24.15e %24.15e %24.15e\n", 
	   data->avcm[0], data->avcm[1], data->avcm[2] );
#endif
}



void Compute_Kinetic_Energy( reax_system* system, simulation_data* data )
{
  int i;
  rvec p;
  real m;

  data->E_Kin = 0.0;

  for (i=0; i < system->N; i++) {
    m = system->reaxprm.sbp[system->atoms[i].type].mass;
    
    rvec_Scale( p, m, system->atoms[i].v );
    data->E_Kin += 0.5 * rvec_Dot( p, system->atoms[i].v );
    
    /* fprintf(stderr,"%d, %lf, %lf, %lf %lf\n",
       i,system->atoms[i].v[0], system->atoms[i].v[1], system->atoms[i].v[2],
       system->reaxprm.sbp[system->atoms[i].type].mass); */
  }
  
  data->therm.T = (2. * data->E_Kin) / (data->N_f * K_B);
  
  if ( fabs(data->therm.T) < ALMOST_ZERO ) /* avoid T being an absolute zero! */
    data->therm.T = ALMOST_ZERO;
}


GLOBAL void Compute_Kinetic_Energy( single_body_parameters* sbp, reax_atom* atoms, 
							unsigned int N, real *output)
{
  	extern __shared__ real sh_ekin[];
  	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  	rvec p;
  	real m, tmp;

	tmp = 0;
	m = 0;
	if (index < N) {
    	m = sbp[atoms[index].type].mass;
    	rvec_Scale( p, m, atoms[index].v );
		tmp = 0.5 * rvec_Dot( p, atoms[index].v );
	}
	sh_ekin[threadIdx.x] = tmp;
	__syncthreads ();

	for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
		if (threadIdx.x < offset ) {
			index = threadIdx.x + offset;
			sh_ekin[threadIdx.x] += sh_ekin[ index ];
		}
		__syncthreads ();
	}

	if (threadIdx.x == 0) {
		output [ blockIdx.x ] = sh_ekin [ 0 ];
	}
}

GLOBAL void Kinetic_Energy_Reduction (simulation_data *data,
												real *input, int n)
{
  extern __shared__ real sdata[];
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  real x = 0;

  if(i < n)
  {
    x = input[i];
  }
  sdata[threadIdx.x] = x;
  __syncthreads();

  for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
  {
    if(threadIdx.x < offset)
    {   
      sdata[threadIdx.x] += sdata[threadIdx.x + offset];
    }   

    __syncthreads();
  }

  if(threadIdx.x == 0)
  {
    	//per_block_results[blockIdx.x] = sdata[0];
		data->E_Kin = sdata[0];
  		data->therm.T = (2. * data->E_Kin) / (data->N_f * K_B);
  
  		if ( fabs(data->therm.T) < ALMOST_ZERO ) // avoid T being an absolute zero! 
    		data->therm.T = ALMOST_ZERO;
  }
}

void Cuda_Compute_Kinetic_Energy (reax_system *system, simulation_data *data)
{
   real *results = (real *) scratch;
	cuda_memset (results, 0, REAL_SIZE * BLOCKS_POW_2, RES_SCRATCH);
	Compute_Kinetic_Energy <<<BLOCKS_POW_2, BLOCK_SIZE, REAL_SIZE * BLOCK_SIZE>>>
			                 (system->reaxprm.d_sbp, system->d_atoms, system->N, (real *) results);
	hipDeviceSynchronize (); 
	cudaCheckError ();

	Kinetic_Energy_Reduction <<< 1, BLOCKS_POW_2, REAL_SIZE * BLOCKS_POW_2 >>>
									((simulation_data *)data->d_simulation_data, results, BLOCKS_POW_2);
	hipDeviceSynchronize (); 
	cudaCheckError ();
}

/*
GLOBAL void Compute_Kinetic_Energy( single_body_parameters* sbp, reax_atom* atoms, 
							unsigned int N, simulation_data *data, 
							real *output)
{
  unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
  rvec p;
  real m, tmp;

  extern __shared__ real sh_ekin[];

	tmp = 0;
	m = 0;
	if (index < N) {
    	m = sbp[atoms[index].type].mass;
    	rvec_Scale( p, m, atoms[index].v );
		tmp = 0.5 * rvec_Dot( p, atoms[index].v );
	}
	sh_ekin[threadIdx.x] = tmp;
	__syncthreads ();

	for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
		if (threadIdx.x < offset ) {
			index = threadIdx.x + offset;
			sh_ekin[threadIdx.x] += sh_ekin[ index ];
		}
		__syncthreads ();
	}

	if (threadIdx.x == 0) {
		output [ blockIdx.x ] = sh_ekin [ 0 ];
	}
	__syncthreads ();

  	//if ((blockIdx.x == 0) && (threadIdx.x < gridDim.x)) {
	//	sh_ekin [ threadIdx.x ] = output [ threadIdx.x ];
	//}
	//__syncthreads ();
	

	//gridDim indicates number of blocks configured for this invokation.
	// in this case it will be BLOCKS_POW_2 == 16
	for (int offset = gridDim.x/2; offset > 0; offset >>= 1) {
		if ((threadIdx.x < offset ) && (blockIdx.x == 0)) {
			index = threadIdx.x + offset;
			//sh_ekin[threadIdx.x] += sh_ekin[ index ];
			output [threadIdx.x] += output [index];
		}
		__syncthreads ();
	}
	__syncthreads ();

  	if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
		data->E_Kin = output[0];
  		data->therm.T = (2. * data->E_Kin) / (data->N_f * K_B);
  
  		if ( fabs(data->therm.T) < ALMOST_ZERO ) // avoid T being an absolute zero! 
    		data->therm.T = ALMOST_ZERO;
	}
}
*/


/* IMPORTANT: This function assumes that current kinetic energy and 
 *  the center of mass of the system is already computed before. 
 *
 * IMPORTANT: In Klein's paper, it is stated that a dU/dV term needs 
 *  to be added when there are long-range interactions or long-range 
 *  corrections to short-range interactions present.
 *  We may want to add that for more accuracy. 
 */
void Compute_Pressure_Isotropic( reax_system* system, control_params *control, 
				 simulation_data* data, 
				 output_controls *out_control )
{
  int i;
  reax_atom *p_atom;
  rvec tx;
  rvec tmp;
  simulation_box *box = &(system->box);

  /* Calculate internal pressure */
  rvec_MakeZero( data->int_press );

  // 0: both int and ext, 1: ext only, 2: int only
  if( control->press_mode == 0 || control->press_mode == 2 ) {
    for( i = 0; i < system->N; ++i ) {
      p_atom = &( system->atoms[i] );
	  
      /* transform x into unitbox coordinates */
      Transform_to_UnitBox( p_atom->x, box, 1, tx );
      
      /* this atom's contribution to internal pressure */
      rvec_Multiply( tmp, p_atom->f, tx );
      rvec_Add( data->int_press, tmp );
      
      if( out_control->debug_level > 0 ) {
	fprintf( out_control->prs, "%-8d%8.2f%8.2f%8.2f", 
		 i+1, p_atom->x[0], p_atom->x[1], p_atom->x[2] );
	fprintf( out_control->prs, "%8.2f%8.2f%8.2f", 
		 p_atom->f[0], p_atom->f[1], p_atom->f[2] );
	fprintf( out_control->prs, "%8.2f%8.2f%8.2f\n", 
		 data->int_press[0],data->int_press[1],data->int_press[2]);
      }
    }
  }
  
  /* kinetic contribution */
  data->kin_press = 2. * (E_CONV * data->E_Kin) / ( 3. * box->volume * P_CONV );
  
  /* Calculate total pressure in each direction */  
  data->tot_press[0] = data->kin_press - 
    ((data->int_press[0] + data->ext_press[0]) /
     (box->box_norms[1] * box->box_norms[2] * P_CONV));

  data->tot_press[1] = data->kin_press - 
    ((data->int_press[1] + data->ext_press[1])/
     (box->box_norms[0] * box->box_norms[2] * P_CONV));

  data->tot_press[2] = data->kin_press - 
    ((data->int_press[2] + data->ext_press[2])/
     (box->box_norms[0] * box->box_norms[1] * P_CONV));

  /* Average pressure for the whole box */
  data->iso_bar.P=(data->tot_press[0]+data->tot_press[1]+data->tot_press[2])/3;
}


void Compute_Pressure_Isotropic_Klein( reax_system* system, 
				       simulation_data* data )
{
  int i;
  reax_atom *p_atom;
  rvec dx;

  // IMPORTANT: This function assumes that current kinetic energy and 
  // the center of mass of the system is already computed before.
  data->iso_bar.P = 2.0 * data->E_Kin;

  for( i = 0; i < system->N; ++i )
    {
      p_atom = &( system->atoms[i] );
      rvec_ScaledSum(dx,1.0,p_atom->x,-1.0,data->xcm);
      data->iso_bar.P += ( -F_CONV * rvec_Dot(p_atom->f, dx) );
    }

  data->iso_bar.P /= (3.0 * system->box.volume);

  // IMPORTANT: In Klein's paper, it is stated that a dU/dV term needs 
  // to be added when there are long-range interactions or long-range 
  // corrections to short-range interactions present.
  // We may want to add that for more accuracy.
}


void Compute_Pressure( reax_system* system, simulation_data* data, 
		       static_storage *workspace )
{
  int i;
  reax_atom *p_atom;
  rtensor temp;

  rtensor_MakeZero( data->flex_bar.P );
  
  for( i = 0; i < system->N; ++i ) {
    p_atom = &( system->atoms[i] );
    // Distance_on_T3_Gen( data->rcm, p_atom->x, &(system->box), &dx );
    rvec_OuterProduct( temp, p_atom->v, p_atom->v );
    rtensor_ScaledAdd( data->flex_bar.P, 
		       system->reaxprm.sbp[ p_atom->type ].mass, temp );
    // rvec_OuterProduct(temp, workspace->virial_forces[i], p_atom->x ); 
    rtensor_ScaledAdd( data->flex_bar.P, -F_CONV, temp );
  }
  
  rtensor_Scale( data->flex_bar.P, 1.0 / system->box.volume, data->flex_bar.P );
  data->iso_bar.P = rtensor_Trace( data->flex_bar.P ) / 3.0;
}
